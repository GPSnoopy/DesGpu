#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

#ifdef DESGPU_COMPILE_ALL_SALTS

void des_25_encrypt_salt1152(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 5, 6, 23, 8, 15, 16, 17, 18, 19, 20, 19, 4, 21, 22, 7, 24, 63, 32, 33, 34, 35, 36, 35, 52, 37, 38, 55, 40, 47, 48, 49, 50, 51, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1153(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 5, 6, 23, 8, 31, 16, 17, 18, 19, 20, 19, 4, 21, 22, 7, 24, 47, 32, 33, 34, 35, 36, 35, 52, 37, 38, 55, 40, 63, 48, 49, 50, 51, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1154(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 5, 6, 23, 8, 15, 0, 17, 18, 19, 20, 19, 4, 21, 22, 7, 24, 63, 48, 33, 34, 35, 36, 35, 52, 37, 38, 55, 40, 47, 32, 49, 50, 51, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1155(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 5, 6, 23, 8, 31, 0, 17, 18, 19, 20, 19, 4, 21, 22, 7, 24, 47, 48, 33, 34, 35, 36, 35, 52, 37, 38, 55, 40, 63, 32, 49, 50, 51, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1156(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 5, 6, 23, 8, 15, 16, 1, 18, 19, 20, 19, 4, 21, 22, 7, 24, 63, 32, 49, 34, 35, 36, 35, 52, 37, 38, 55, 40, 47, 48, 33, 50, 51, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1157(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 5, 6, 23, 8, 31, 16, 1, 18, 19, 20, 19, 4, 21, 22, 7, 24, 47, 32, 49, 34, 35, 36, 35, 52, 37, 38, 55, 40, 63, 48, 33, 50, 51, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1158(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 5, 6, 23, 8, 15, 0, 1, 18, 19, 20, 19, 4, 21, 22, 7, 24, 63, 48, 49, 34, 35, 36, 35, 52, 37, 38, 55, 40, 47, 32, 33, 50, 51, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1159(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 5, 6, 23, 8, 31, 0, 1, 18, 19, 20, 19, 4, 21, 22, 7, 24, 47, 48, 49, 34, 35, 36, 35, 52, 37, 38, 55, 40, 63, 32, 33, 50, 51, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1160(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 5, 6, 23, 8, 15, 16, 17, 2, 19, 20, 19, 4, 21, 22, 7, 24, 63, 32, 33, 50, 35, 36, 35, 52, 37, 38, 55, 40, 47, 48, 49, 34, 51, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1161(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 5, 6, 23, 8, 31, 16, 17, 2, 19, 20, 19, 4, 21, 22, 7, 24, 47, 32, 33, 50, 35, 36, 35, 52, 37, 38, 55, 40, 63, 48, 49, 34, 51, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1162(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 5, 6, 23, 8, 15, 0, 17, 2, 19, 20, 19, 4, 21, 22, 7, 24, 63, 48, 33, 50, 35, 36, 35, 52, 37, 38, 55, 40, 47, 32, 49, 34, 51, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1163(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 5, 6, 23, 8, 31, 0, 17, 2, 19, 20, 19, 4, 21, 22, 7, 24, 47, 48, 33, 50, 35, 36, 35, 52, 37, 38, 55, 40, 63, 32, 49, 34, 51, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1164(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 5, 6, 23, 8, 15, 16, 1, 2, 19, 20, 19, 4, 21, 22, 7, 24, 63, 32, 49, 50, 35, 36, 35, 52, 37, 38, 55, 40, 47, 48, 33, 34, 51, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1165(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 5, 6, 23, 8, 31, 16, 1, 2, 19, 20, 19, 4, 21, 22, 7, 24, 47, 32, 49, 50, 35, 36, 35, 52, 37, 38, 55, 40, 63, 48, 33, 34, 51, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1166(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 5, 6, 23, 8, 15, 0, 1, 2, 19, 20, 19, 4, 21, 22, 7, 24, 63, 48, 49, 50, 35, 36, 35, 52, 37, 38, 55, 40, 47, 32, 33, 34, 51, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1167(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 5, 6, 23, 8, 31, 0, 1, 2, 19, 20, 19, 4, 21, 22, 7, 24, 47, 48, 49, 50, 35, 36, 35, 52, 37, 38, 55, 40, 63, 32, 33, 34, 51, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1168(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 5, 6, 23, 8, 15, 16, 17, 18, 3, 20, 19, 4, 21, 22, 7, 24, 63, 32, 33, 34, 51, 36, 35, 52, 37, 38, 55, 40, 47, 48, 49, 50, 35, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1169(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 5, 6, 23, 8, 31, 16, 17, 18, 3, 20, 19, 4, 21, 22, 7, 24, 47, 32, 33, 34, 51, 36, 35, 52, 37, 38, 55, 40, 63, 48, 49, 50, 35, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1170(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 5, 6, 23, 8, 15, 0, 17, 18, 3, 20, 19, 4, 21, 22, 7, 24, 63, 48, 33, 34, 51, 36, 35, 52, 37, 38, 55, 40, 47, 32, 49, 50, 35, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1171(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 5, 6, 23, 8, 31, 0, 17, 18, 3, 20, 19, 4, 21, 22, 7, 24, 47, 48, 33, 34, 51, 36, 35, 52, 37, 38, 55, 40, 63, 32, 49, 50, 35, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1172(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 5, 6, 23, 8, 15, 16, 1, 18, 3, 20, 19, 4, 21, 22, 7, 24, 63, 32, 49, 34, 51, 36, 35, 52, 37, 38, 55, 40, 47, 48, 33, 50, 35, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1173(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 5, 6, 23, 8, 31, 16, 1, 18, 3, 20, 19, 4, 21, 22, 7, 24, 47, 32, 49, 34, 51, 36, 35, 52, 37, 38, 55, 40, 63, 48, 33, 50, 35, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1174(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 5, 6, 23, 8, 15, 0, 1, 18, 3, 20, 19, 4, 21, 22, 7, 24, 63, 48, 49, 34, 51, 36, 35, 52, 37, 38, 55, 40, 47, 32, 33, 50, 35, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1175(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 5, 6, 23, 8, 31, 0, 1, 18, 3, 20, 19, 4, 21, 22, 7, 24, 47, 48, 49, 34, 51, 36, 35, 52, 37, 38, 55, 40, 63, 32, 33, 50, 35, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1176(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 5, 6, 23, 8, 15, 16, 17, 2, 3, 20, 19, 4, 21, 22, 7, 24, 63, 32, 33, 50, 51, 36, 35, 52, 37, 38, 55, 40, 47, 48, 49, 34, 35, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1177(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 5, 6, 23, 8, 31, 16, 17, 2, 3, 20, 19, 4, 21, 22, 7, 24, 47, 32, 33, 50, 51, 36, 35, 52, 37, 38, 55, 40, 63, 48, 49, 34, 35, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1178(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 5, 6, 23, 8, 15, 0, 17, 2, 3, 20, 19, 4, 21, 22, 7, 24, 63, 48, 33, 50, 51, 36, 35, 52, 37, 38, 55, 40, 47, 32, 49, 34, 35, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1179(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 5, 6, 23, 8, 31, 0, 17, 2, 3, 20, 19, 4, 21, 22, 7, 24, 47, 48, 33, 50, 51, 36, 35, 52, 37, 38, 55, 40, 63, 32, 49, 34, 35, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1180(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 5, 6, 23, 8, 15, 16, 1, 2, 3, 20, 19, 4, 21, 22, 7, 24, 63, 32, 49, 50, 51, 36, 35, 52, 37, 38, 55, 40, 47, 48, 33, 34, 35, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1181(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 5, 6, 23, 8, 31, 16, 1, 2, 3, 20, 19, 4, 21, 22, 7, 24, 47, 32, 49, 50, 51, 36, 35, 52, 37, 38, 55, 40, 63, 48, 33, 34, 35, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1182(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 5, 6, 23, 8, 15, 0, 1, 2, 3, 20, 19, 4, 21, 22, 7, 24, 63, 48, 49, 50, 51, 36, 35, 52, 37, 38, 55, 40, 47, 32, 33, 34, 35, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1183(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 5, 6, 23, 8, 31, 0, 1, 2, 3, 20, 19, 4, 21, 22, 7, 24, 47, 48, 49, 50, 51, 36, 35, 52, 37, 38, 55, 40, 63, 32, 33, 34, 35, 52, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1184(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 5, 6, 23, 8, 15, 16, 17, 18, 19, 4, 19, 4, 21, 22, 7, 24, 63, 32, 33, 34, 35, 52, 35, 52, 37, 38, 55, 40, 47, 48, 49, 50, 51, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1185(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 5, 6, 23, 8, 31, 16, 17, 18, 19, 4, 19, 4, 21, 22, 7, 24, 47, 32, 33, 34, 35, 52, 35, 52, 37, 38, 55, 40, 63, 48, 49, 50, 51, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1186(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 5, 6, 23, 8, 15, 0, 17, 18, 19, 4, 19, 4, 21, 22, 7, 24, 63, 48, 33, 34, 35, 52, 35, 52, 37, 38, 55, 40, 47, 32, 49, 50, 51, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1187(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 5, 6, 23, 8, 31, 0, 17, 18, 19, 4, 19, 4, 21, 22, 7, 24, 47, 48, 33, 34, 35, 52, 35, 52, 37, 38, 55, 40, 63, 32, 49, 50, 51, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1188(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 5, 6, 23, 8, 15, 16, 1, 18, 19, 4, 19, 4, 21, 22, 7, 24, 63, 32, 49, 34, 35, 52, 35, 52, 37, 38, 55, 40, 47, 48, 33, 50, 51, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1189(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 5, 6, 23, 8, 31, 16, 1, 18, 19, 4, 19, 4, 21, 22, 7, 24, 47, 32, 49, 34, 35, 52, 35, 52, 37, 38, 55, 40, 63, 48, 33, 50, 51, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1190(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 5, 6, 23, 8, 15, 0, 1, 18, 19, 4, 19, 4, 21, 22, 7, 24, 63, 48, 49, 34, 35, 52, 35, 52, 37, 38, 55, 40, 47, 32, 33, 50, 51, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1191(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 5, 6, 23, 8, 31, 0, 1, 18, 19, 4, 19, 4, 21, 22, 7, 24, 47, 48, 49, 34, 35, 52, 35, 52, 37, 38, 55, 40, 63, 32, 33, 50, 51, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1192(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 5, 6, 23, 8, 15, 16, 17, 2, 19, 4, 19, 4, 21, 22, 7, 24, 63, 32, 33, 50, 35, 52, 35, 52, 37, 38, 55, 40, 47, 48, 49, 34, 51, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1193(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 5, 6, 23, 8, 31, 16, 17, 2, 19, 4, 19, 4, 21, 22, 7, 24, 47, 32, 33, 50, 35, 52, 35, 52, 37, 38, 55, 40, 63, 48, 49, 34, 51, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1194(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 5, 6, 23, 8, 15, 0, 17, 2, 19, 4, 19, 4, 21, 22, 7, 24, 63, 48, 33, 50, 35, 52, 35, 52, 37, 38, 55, 40, 47, 32, 49, 34, 51, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1195(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 5, 6, 23, 8, 31, 0, 17, 2, 19, 4, 19, 4, 21, 22, 7, 24, 47, 48, 33, 50, 35, 52, 35, 52, 37, 38, 55, 40, 63, 32, 49, 34, 51, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1196(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 5, 6, 23, 8, 15, 16, 1, 2, 19, 4, 19, 4, 21, 22, 7, 24, 63, 32, 49, 50, 35, 52, 35, 52, 37, 38, 55, 40, 47, 48, 33, 34, 51, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1197(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 5, 6, 23, 8, 31, 16, 1, 2, 19, 4, 19, 4, 21, 22, 7, 24, 47, 32, 49, 50, 35, 52, 35, 52, 37, 38, 55, 40, 63, 48, 33, 34, 51, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1198(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 5, 6, 23, 8, 15, 0, 1, 2, 19, 4, 19, 4, 21, 22, 7, 24, 63, 48, 49, 50, 35, 52, 35, 52, 37, 38, 55, 40, 47, 32, 33, 34, 51, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1199(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 5, 6, 23, 8, 31, 0, 1, 2, 19, 4, 19, 4, 21, 22, 7, 24, 47, 48, 49, 50, 35, 52, 35, 52, 37, 38, 55, 40, 63, 32, 33, 34, 51, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1200(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 5, 6, 23, 8, 15, 16, 17, 18, 3, 4, 19, 4, 21, 22, 7, 24, 63, 32, 33, 34, 51, 52, 35, 52, 37, 38, 55, 40, 47, 48, 49, 50, 35, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1201(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 5, 6, 23, 8, 31, 16, 17, 18, 3, 4, 19, 4, 21, 22, 7, 24, 47, 32, 33, 34, 51, 52, 35, 52, 37, 38, 55, 40, 63, 48, 49, 50, 35, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1202(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 5, 6, 23, 8, 15, 0, 17, 18, 3, 4, 19, 4, 21, 22, 7, 24, 63, 48, 33, 34, 51, 52, 35, 52, 37, 38, 55, 40, 47, 32, 49, 50, 35, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1203(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 5, 6, 23, 8, 31, 0, 17, 18, 3, 4, 19, 4, 21, 22, 7, 24, 47, 48, 33, 34, 51, 52, 35, 52, 37, 38, 55, 40, 63, 32, 49, 50, 35, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1204(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 5, 6, 23, 8, 15, 16, 1, 18, 3, 4, 19, 4, 21, 22, 7, 24, 63, 32, 49, 34, 51, 52, 35, 52, 37, 38, 55, 40, 47, 48, 33, 50, 35, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1205(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 5, 6, 23, 8, 31, 16, 1, 18, 3, 4, 19, 4, 21, 22, 7, 24, 47, 32, 49, 34, 51, 52, 35, 52, 37, 38, 55, 40, 63, 48, 33, 50, 35, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1206(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 5, 6, 23, 8, 15, 0, 1, 18, 3, 4, 19, 4, 21, 22, 7, 24, 63, 48, 49, 34, 51, 52, 35, 52, 37, 38, 55, 40, 47, 32, 33, 50, 35, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1207(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 5, 6, 23, 8, 31, 0, 1, 18, 3, 4, 19, 4, 21, 22, 7, 24, 47, 48, 49, 34, 51, 52, 35, 52, 37, 38, 55, 40, 63, 32, 33, 50, 35, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1208(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 5, 6, 23, 8, 15, 16, 17, 2, 3, 4, 19, 4, 21, 22, 7, 24, 63, 32, 33, 50, 51, 52, 35, 52, 37, 38, 55, 40, 47, 48, 49, 34, 35, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1209(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 5, 6, 23, 8, 31, 16, 17, 2, 3, 4, 19, 4, 21, 22, 7, 24, 47, 32, 33, 50, 51, 52, 35, 52, 37, 38, 55, 40, 63, 48, 49, 34, 35, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1210(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 5, 6, 23, 8, 15, 0, 17, 2, 3, 4, 19, 4, 21, 22, 7, 24, 63, 48, 33, 50, 51, 52, 35, 52, 37, 38, 55, 40, 47, 32, 49, 34, 35, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1211(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 5, 6, 23, 8, 31, 0, 17, 2, 3, 4, 19, 4, 21, 22, 7, 24, 47, 48, 33, 50, 51, 52, 35, 52, 37, 38, 55, 40, 63, 32, 49, 34, 35, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1212(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 5, 6, 23, 8, 15, 16, 1, 2, 3, 4, 19, 4, 21, 22, 7, 24, 63, 32, 49, 50, 51, 52, 35, 52, 37, 38, 55, 40, 47, 48, 33, 34, 35, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1213(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 5, 6, 23, 8, 31, 16, 1, 2, 3, 4, 19, 4, 21, 22, 7, 24, 47, 32, 49, 50, 51, 52, 35, 52, 37, 38, 55, 40, 63, 48, 33, 34, 35, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1214(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 5, 6, 23, 8, 15, 0, 1, 2, 3, 4, 19, 4, 21, 22, 7, 24, 63, 48, 49, 50, 51, 52, 35, 52, 37, 38, 55, 40, 47, 32, 33, 34, 35, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1215(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 5, 6, 23, 8, 31, 0, 1, 2, 3, 4, 19, 4, 21, 22, 7, 24, 47, 48, 49, 50, 51, 52, 35, 52, 37, 38, 55, 40, 63, 32, 33, 34, 35, 36, 51, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1216(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 5, 6, 23, 8, 15, 16, 17, 18, 19, 20, 3, 4, 21, 22, 7, 24, 63, 32, 33, 34, 35, 36, 51, 52, 37, 38, 55, 40, 47, 48, 49, 50, 51, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1217(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 5, 6, 23, 8, 31, 16, 17, 18, 19, 20, 3, 4, 21, 22, 7, 24, 47, 32, 33, 34, 35, 36, 51, 52, 37, 38, 55, 40, 63, 48, 49, 50, 51, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1218(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 5, 6, 23, 8, 15, 0, 17, 18, 19, 20, 3, 4, 21, 22, 7, 24, 63, 48, 33, 34, 35, 36, 51, 52, 37, 38, 55, 40, 47, 32, 49, 50, 51, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1219(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 5, 6, 23, 8, 31, 0, 17, 18, 19, 20, 3, 4, 21, 22, 7, 24, 47, 48, 33, 34, 35, 36, 51, 52, 37, 38, 55, 40, 63, 32, 49, 50, 51, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1220(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 5, 6, 23, 8, 15, 16, 1, 18, 19, 20, 3, 4, 21, 22, 7, 24, 63, 32, 49, 34, 35, 36, 51, 52, 37, 38, 55, 40, 47, 48, 33, 50, 51, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1221(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 5, 6, 23, 8, 31, 16, 1, 18, 19, 20, 3, 4, 21, 22, 7, 24, 47, 32, 49, 34, 35, 36, 51, 52, 37, 38, 55, 40, 63, 48, 33, 50, 51, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1222(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 5, 6, 23, 8, 15, 0, 1, 18, 19, 20, 3, 4, 21, 22, 7, 24, 63, 48, 49, 34, 35, 36, 51, 52, 37, 38, 55, 40, 47, 32, 33, 50, 51, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1223(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 5, 6, 23, 8, 31, 0, 1, 18, 19, 20, 3, 4, 21, 22, 7, 24, 47, 48, 49, 34, 35, 36, 51, 52, 37, 38, 55, 40, 63, 32, 33, 50, 51, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1224(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 5, 6, 23, 8, 15, 16, 17, 2, 19, 20, 3, 4, 21, 22, 7, 24, 63, 32, 33, 50, 35, 36, 51, 52, 37, 38, 55, 40, 47, 48, 49, 34, 51, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1225(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 5, 6, 23, 8, 31, 16, 17, 2, 19, 20, 3, 4, 21, 22, 7, 24, 47, 32, 33, 50, 35, 36, 51, 52, 37, 38, 55, 40, 63, 48, 49, 34, 51, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1226(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 5, 6, 23, 8, 15, 0, 17, 2, 19, 20, 3, 4, 21, 22, 7, 24, 63, 48, 33, 50, 35, 36, 51, 52, 37, 38, 55, 40, 47, 32, 49, 34, 51, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1227(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 5, 6, 23, 8, 31, 0, 17, 2, 19, 20, 3, 4, 21, 22, 7, 24, 47, 48, 33, 50, 35, 36, 51, 52, 37, 38, 55, 40, 63, 32, 49, 34, 51, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1228(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 5, 6, 23, 8, 15, 16, 1, 2, 19, 20, 3, 4, 21, 22, 7, 24, 63, 32, 49, 50, 35, 36, 51, 52, 37, 38, 55, 40, 47, 48, 33, 34, 51, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1229(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 5, 6, 23, 8, 31, 16, 1, 2, 19, 20, 3, 4, 21, 22, 7, 24, 47, 32, 49, 50, 35, 36, 51, 52, 37, 38, 55, 40, 63, 48, 33, 34, 51, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1230(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 5, 6, 23, 8, 15, 0, 1, 2, 19, 20, 3, 4, 21, 22, 7, 24, 63, 48, 49, 50, 35, 36, 51, 52, 37, 38, 55, 40, 47, 32, 33, 34, 51, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1231(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 5, 6, 23, 8, 31, 0, 1, 2, 19, 20, 3, 4, 21, 22, 7, 24, 47, 48, 49, 50, 35, 36, 51, 52, 37, 38, 55, 40, 63, 32, 33, 34, 51, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1232(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 5, 6, 23, 8, 15, 16, 17, 18, 3, 20, 3, 4, 21, 22, 7, 24, 63, 32, 33, 34, 51, 36, 51, 52, 37, 38, 55, 40, 47, 48, 49, 50, 35, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1233(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 5, 6, 23, 8, 31, 16, 17, 18, 3, 20, 3, 4, 21, 22, 7, 24, 47, 32, 33, 34, 51, 36, 51, 52, 37, 38, 55, 40, 63, 48, 49, 50, 35, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1234(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 5, 6, 23, 8, 15, 0, 17, 18, 3, 20, 3, 4, 21, 22, 7, 24, 63, 48, 33, 34, 51, 36, 51, 52, 37, 38, 55, 40, 47, 32, 49, 50, 35, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1235(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 5, 6, 23, 8, 31, 0, 17, 18, 3, 20, 3, 4, 21, 22, 7, 24, 47, 48, 33, 34, 51, 36, 51, 52, 37, 38, 55, 40, 63, 32, 49, 50, 35, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1236(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 5, 6, 23, 8, 15, 16, 1, 18, 3, 20, 3, 4, 21, 22, 7, 24, 63, 32, 49, 34, 51, 36, 51, 52, 37, 38, 55, 40, 47, 48, 33, 50, 35, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1237(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 5, 6, 23, 8, 31, 16, 1, 18, 3, 20, 3, 4, 21, 22, 7, 24, 47, 32, 49, 34, 51, 36, 51, 52, 37, 38, 55, 40, 63, 48, 33, 50, 35, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1238(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 5, 6, 23, 8, 15, 0, 1, 18, 3, 20, 3, 4, 21, 22, 7, 24, 63, 48, 49, 34, 51, 36, 51, 52, 37, 38, 55, 40, 47, 32, 33, 50, 35, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1239(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 5, 6, 23, 8, 31, 0, 1, 18, 3, 20, 3, 4, 21, 22, 7, 24, 47, 48, 49, 34, 51, 36, 51, 52, 37, 38, 55, 40, 63, 32, 33, 50, 35, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1240(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 5, 6, 23, 8, 15, 16, 17, 2, 3, 20, 3, 4, 21, 22, 7, 24, 63, 32, 33, 50, 51, 36, 51, 52, 37, 38, 55, 40, 47, 48, 49, 34, 35, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1241(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 5, 6, 23, 8, 31, 16, 17, 2, 3, 20, 3, 4, 21, 22, 7, 24, 47, 32, 33, 50, 51, 36, 51, 52, 37, 38, 55, 40, 63, 48, 49, 34, 35, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1242(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 5, 6, 23, 8, 15, 0, 17, 2, 3, 20, 3, 4, 21, 22, 7, 24, 63, 48, 33, 50, 51, 36, 51, 52, 37, 38, 55, 40, 47, 32, 49, 34, 35, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1243(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 5, 6, 23, 8, 31, 0, 17, 2, 3, 20, 3, 4, 21, 22, 7, 24, 47, 48, 33, 50, 51, 36, 51, 52, 37, 38, 55, 40, 63, 32, 49, 34, 35, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1244(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 5, 6, 23, 8, 15, 16, 1, 2, 3, 20, 3, 4, 21, 22, 7, 24, 63, 32, 49, 50, 51, 36, 51, 52, 37, 38, 55, 40, 47, 48, 33, 34, 35, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1245(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 5, 6, 23, 8, 31, 16, 1, 2, 3, 20, 3, 4, 21, 22, 7, 24, 47, 32, 49, 50, 51, 36, 51, 52, 37, 38, 55, 40, 63, 48, 33, 34, 35, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1246(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 5, 6, 23, 8, 15, 0, 1, 2, 3, 20, 3, 4, 21, 22, 7, 24, 63, 48, 49, 50, 51, 36, 51, 52, 37, 38, 55, 40, 47, 32, 33, 34, 35, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1247(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 5, 6, 23, 8, 31, 0, 1, 2, 3, 20, 3, 4, 21, 22, 7, 24, 47, 48, 49, 50, 51, 36, 51, 52, 37, 38, 55, 40, 63, 32, 33, 34, 35, 52, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1248(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 5, 6, 23, 8, 15, 16, 17, 18, 19, 4, 3, 4, 21, 22, 7, 24, 63, 32, 33, 34, 35, 52, 51, 52, 37, 38, 55, 40, 47, 48, 49, 50, 51, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1249(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 5, 6, 23, 8, 31, 16, 17, 18, 19, 4, 3, 4, 21, 22, 7, 24, 47, 32, 33, 34, 35, 52, 51, 52, 37, 38, 55, 40, 63, 48, 49, 50, 51, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1250(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 5, 6, 23, 8, 15, 0, 17, 18, 19, 4, 3, 4, 21, 22, 7, 24, 63, 48, 33, 34, 35, 52, 51, 52, 37, 38, 55, 40, 47, 32, 49, 50, 51, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1251(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 5, 6, 23, 8, 31, 0, 17, 18, 19, 4, 3, 4, 21, 22, 7, 24, 47, 48, 33, 34, 35, 52, 51, 52, 37, 38, 55, 40, 63, 32, 49, 50, 51, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1252(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 5, 6, 23, 8, 15, 16, 1, 18, 19, 4, 3, 4, 21, 22, 7, 24, 63, 32, 49, 34, 35, 52, 51, 52, 37, 38, 55, 40, 47, 48, 33, 50, 51, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1253(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 5, 6, 23, 8, 31, 16, 1, 18, 19, 4, 3, 4, 21, 22, 7, 24, 47, 32, 49, 34, 35, 52, 51, 52, 37, 38, 55, 40, 63, 48, 33, 50, 51, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1254(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 5, 6, 23, 8, 15, 0, 1, 18, 19, 4, 3, 4, 21, 22, 7, 24, 63, 48, 49, 34, 35, 52, 51, 52, 37, 38, 55, 40, 47, 32, 33, 50, 51, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1255(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 5, 6, 23, 8, 31, 0, 1, 18, 19, 4, 3, 4, 21, 22, 7, 24, 47, 48, 49, 34, 35, 52, 51, 52, 37, 38, 55, 40, 63, 32, 33, 50, 51, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1256(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 5, 6, 23, 8, 15, 16, 17, 2, 19, 4, 3, 4, 21, 22, 7, 24, 63, 32, 33, 50, 35, 52, 51, 52, 37, 38, 55, 40, 47, 48, 49, 34, 51, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1257(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 5, 6, 23, 8, 31, 16, 17, 2, 19, 4, 3, 4, 21, 22, 7, 24, 47, 32, 33, 50, 35, 52, 51, 52, 37, 38, 55, 40, 63, 48, 49, 34, 51, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1258(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 5, 6, 23, 8, 15, 0, 17, 2, 19, 4, 3, 4, 21, 22, 7, 24, 63, 48, 33, 50, 35, 52, 51, 52, 37, 38, 55, 40, 47, 32, 49, 34, 51, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1259(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 5, 6, 23, 8, 31, 0, 17, 2, 19, 4, 3, 4, 21, 22, 7, 24, 47, 48, 33, 50, 35, 52, 51, 52, 37, 38, 55, 40, 63, 32, 49, 34, 51, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1260(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 5, 6, 23, 8, 15, 16, 1, 2, 19, 4, 3, 4, 21, 22, 7, 24, 63, 32, 49, 50, 35, 52, 51, 52, 37, 38, 55, 40, 47, 48, 33, 34, 51, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1261(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 5, 6, 23, 8, 31, 16, 1, 2, 19, 4, 3, 4, 21, 22, 7, 24, 47, 32, 49, 50, 35, 52, 51, 52, 37, 38, 55, 40, 63, 48, 33, 34, 51, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1262(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 5, 6, 23, 8, 15, 0, 1, 2, 19, 4, 3, 4, 21, 22, 7, 24, 63, 48, 49, 50, 35, 52, 51, 52, 37, 38, 55, 40, 47, 32, 33, 34, 51, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1263(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 5, 6, 23, 8, 31, 0, 1, 2, 19, 4, 3, 4, 21, 22, 7, 24, 47, 48, 49, 50, 35, 52, 51, 52, 37, 38, 55, 40, 63, 32, 33, 34, 51, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1264(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 5, 6, 23, 8, 15, 16, 17, 18, 3, 4, 3, 4, 21, 22, 7, 24, 63, 32, 33, 34, 51, 52, 51, 52, 37, 38, 55, 40, 47, 48, 49, 50, 35, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1265(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 5, 6, 23, 8, 31, 16, 17, 18, 3, 4, 3, 4, 21, 22, 7, 24, 47, 32, 33, 34, 51, 52, 51, 52, 37, 38, 55, 40, 63, 48, 49, 50, 35, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1266(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 5, 6, 23, 8, 15, 0, 17, 18, 3, 4, 3, 4, 21, 22, 7, 24, 63, 48, 33, 34, 51, 52, 51, 52, 37, 38, 55, 40, 47, 32, 49, 50, 35, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1267(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 5, 6, 23, 8, 31, 0, 17, 18, 3, 4, 3, 4, 21, 22, 7, 24, 47, 48, 33, 34, 51, 52, 51, 52, 37, 38, 55, 40, 63, 32, 49, 50, 35, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1268(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 5, 6, 23, 8, 15, 16, 1, 18, 3, 4, 3, 4, 21, 22, 7, 24, 63, 32, 49, 34, 51, 52, 51, 52, 37, 38, 55, 40, 47, 48, 33, 50, 35, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1269(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 5, 6, 23, 8, 31, 16, 1, 18, 3, 4, 3, 4, 21, 22, 7, 24, 47, 32, 49, 34, 51, 52, 51, 52, 37, 38, 55, 40, 63, 48, 33, 50, 35, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1270(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 5, 6, 23, 8, 15, 0, 1, 18, 3, 4, 3, 4, 21, 22, 7, 24, 63, 48, 49, 34, 51, 52, 51, 52, 37, 38, 55, 40, 47, 32, 33, 50, 35, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1271(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 5, 6, 23, 8, 31, 0, 1, 18, 3, 4, 3, 4, 21, 22, 7, 24, 47, 48, 49, 34, 51, 52, 51, 52, 37, 38, 55, 40, 63, 32, 33, 50, 35, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1272(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 5, 6, 23, 8, 15, 16, 17, 2, 3, 4, 3, 4, 21, 22, 7, 24, 63, 32, 33, 50, 51, 52, 51, 52, 37, 38, 55, 40, 47, 48, 49, 34, 35, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1273(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 5, 6, 23, 8, 31, 16, 17, 2, 3, 4, 3, 4, 21, 22, 7, 24, 47, 32, 33, 50, 51, 52, 51, 52, 37, 38, 55, 40, 63, 48, 49, 34, 35, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1274(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 5, 6, 23, 8, 15, 0, 17, 2, 3, 4, 3, 4, 21, 22, 7, 24, 63, 48, 33, 50, 51, 52, 51, 52, 37, 38, 55, 40, 47, 32, 49, 34, 35, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1275(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 5, 6, 23, 8, 31, 0, 17, 2, 3, 4, 3, 4, 21, 22, 7, 24, 47, 48, 33, 50, 51, 52, 51, 52, 37, 38, 55, 40, 63, 32, 49, 34, 35, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1276(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 5, 6, 23, 8, 15, 16, 1, 2, 3, 4, 3, 4, 21, 22, 7, 24, 63, 32, 49, 50, 51, 52, 51, 52, 37, 38, 55, 40, 47, 48, 33, 34, 35, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1277(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 5, 6, 23, 8, 31, 16, 1, 2, 3, 4, 3, 4, 21, 22, 7, 24, 47, 32, 49, 50, 51, 52, 51, 52, 37, 38, 55, 40, 63, 48, 33, 34, 35, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1278(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 5, 6, 23, 8, 15, 0, 1, 2, 3, 4, 3, 4, 21, 22, 7, 24, 63, 48, 49, 50, 51, 52, 51, 52, 37, 38, 55, 40, 47, 32, 33, 34, 35, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1279(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 5, 6, 23, 8, 31, 0, 1, 2, 3, 4, 3, 4, 21, 22, 7, 24, 47, 48, 49, 50, 51, 52, 51, 52, 37, 38, 55, 40, 63, 32, 33, 34, 35, 36, 35, 36, 53, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

#endif // DESGPU_COMPILE_ALL_SALTS


