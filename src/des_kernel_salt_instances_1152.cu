#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt1152(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 5, 6, 23, 8, 15, 16, 17, 18, 19, 20, 19, 4, 21, 22, 7, 24, 63, 32, 33, 34, 35, 36, 35, 52, 37, 38, 55, 40, 47, 48, 49, 50, 51, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1153(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 5, 6, 23, 8, 31, 16, 17, 18, 19, 20, 19, 4, 21, 22, 7, 24, 47, 32, 33, 34, 35, 36, 35, 52, 37, 38, 55, 40, 63, 48, 49, 50, 51, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1154(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 5, 6, 23, 8, 15, 0, 17, 18, 19, 20, 19, 4, 21, 22, 7, 24, 63, 48, 33, 34, 35, 36, 35, 52, 37, 38, 55, 40, 47, 32, 49, 50, 51, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1155(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 5, 6, 23, 8, 31, 0, 17, 18, 19, 20, 19, 4, 21, 22, 7, 24, 47, 48, 33, 34, 35, 36, 35, 52, 37, 38, 55, 40, 63, 32, 49, 50, 51, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1156(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 5, 6, 23, 8, 15, 16, 1, 18, 19, 20, 19, 4, 21, 22, 7, 24, 63, 32, 49, 34, 35, 36, 35, 52, 37, 38, 55, 40, 47, 48, 33, 50, 51, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1157(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 5, 6, 23, 8, 31, 16, 1, 18, 19, 20, 19, 4, 21, 22, 7, 24, 47, 32, 49, 34, 35, 36, 35, 52, 37, 38, 55, 40, 63, 48, 33, 50, 51, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1158(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 5, 6, 23, 8, 15, 0, 1, 18, 19, 20, 19, 4, 21, 22, 7, 24, 63, 48, 49, 34, 35, 36, 35, 52, 37, 38, 55, 40, 47, 32, 33, 50, 51, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1159(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 5, 6, 23, 8, 31, 0, 1, 18, 19, 20, 19, 4, 21, 22, 7, 24, 47, 48, 49, 34, 35, 36, 35, 52, 37, 38, 55, 40, 63, 32, 33, 50, 51, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1160(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 5, 6, 23, 8, 15, 16, 17, 2, 19, 20, 19, 4, 21, 22, 7, 24, 63, 32, 33, 50, 35, 36, 35, 52, 37, 38, 55, 40, 47, 48, 49, 34, 51, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1161(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 5, 6, 23, 8, 31, 16, 17, 2, 19, 20, 19, 4, 21, 22, 7, 24, 47, 32, 33, 50, 35, 36, 35, 52, 37, 38, 55, 40, 63, 48, 49, 34, 51, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1162(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 5, 6, 23, 8, 15, 0, 17, 2, 19, 20, 19, 4, 21, 22, 7, 24, 63, 48, 33, 50, 35, 36, 35, 52, 37, 38, 55, 40, 47, 32, 49, 34, 51, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1163(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 5, 6, 23, 8, 31, 0, 17, 2, 19, 20, 19, 4, 21, 22, 7, 24, 47, 48, 33, 50, 35, 36, 35, 52, 37, 38, 55, 40, 63, 32, 49, 34, 51, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1164(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 5, 6, 23, 8, 15, 16, 1, 2, 19, 20, 19, 4, 21, 22, 7, 24, 63, 32, 49, 50, 35, 36, 35, 52, 37, 38, 55, 40, 47, 48, 33, 34, 51, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1165(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 5, 6, 23, 8, 31, 16, 1, 2, 19, 20, 19, 4, 21, 22, 7, 24, 47, 32, 49, 50, 35, 36, 35, 52, 37, 38, 55, 40, 63, 48, 33, 34, 51, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1166(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 5, 6, 23, 8, 15, 0, 1, 2, 19, 20, 19, 4, 21, 22, 7, 24, 63, 48, 49, 50, 35, 36, 35, 52, 37, 38, 55, 40, 47, 32, 33, 34, 51, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1167(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 5, 6, 23, 8, 31, 0, 1, 2, 19, 20, 19, 4, 21, 22, 7, 24, 47, 48, 49, 50, 35, 36, 35, 52, 37, 38, 55, 40, 63, 32, 33, 34, 51, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1168(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 5, 6, 23, 8, 15, 16, 17, 18, 3, 20, 19, 4, 21, 22, 7, 24, 63, 32, 33, 34, 51, 36, 35, 52, 37, 38, 55, 40, 47, 48, 49, 50, 35, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1169(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 5, 6, 23, 8, 31, 16, 17, 18, 3, 20, 19, 4, 21, 22, 7, 24, 47, 32, 33, 34, 51, 36, 35, 52, 37, 38, 55, 40, 63, 48, 49, 50, 35, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1170(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 5, 6, 23, 8, 15, 0, 17, 18, 3, 20, 19, 4, 21, 22, 7, 24, 63, 48, 33, 34, 51, 36, 35, 52, 37, 38, 55, 40, 47, 32, 49, 50, 35, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1171(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 5, 6, 23, 8, 31, 0, 17, 18, 3, 20, 19, 4, 21, 22, 7, 24, 47, 48, 33, 34, 51, 36, 35, 52, 37, 38, 55, 40, 63, 32, 49, 50, 35, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1172(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 5, 6, 23, 8, 15, 16, 1, 18, 3, 20, 19, 4, 21, 22, 7, 24, 63, 32, 49, 34, 51, 36, 35, 52, 37, 38, 55, 40, 47, 48, 33, 50, 35, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1173(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 5, 6, 23, 8, 31, 16, 1, 18, 3, 20, 19, 4, 21, 22, 7, 24, 47, 32, 49, 34, 51, 36, 35, 52, 37, 38, 55, 40, 63, 48, 33, 50, 35, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1174(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 5, 6, 23, 8, 15, 0, 1, 18, 3, 20, 19, 4, 21, 22, 7, 24, 63, 48, 49, 34, 51, 36, 35, 52, 37, 38, 55, 40, 47, 32, 33, 50, 35, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1175(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 5, 6, 23, 8, 31, 0, 1, 18, 3, 20, 19, 4, 21, 22, 7, 24, 47, 48, 49, 34, 51, 36, 35, 52, 37, 38, 55, 40, 63, 32, 33, 50, 35, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1176(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 5, 6, 23, 8, 15, 16, 17, 2, 3, 20, 19, 4, 21, 22, 7, 24, 63, 32, 33, 50, 51, 36, 35, 52, 37, 38, 55, 40, 47, 48, 49, 34, 35, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1177(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 5, 6, 23, 8, 31, 16, 17, 2, 3, 20, 19, 4, 21, 22, 7, 24, 47, 32, 33, 50, 51, 36, 35, 52, 37, 38, 55, 40, 63, 48, 49, 34, 35, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1178(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 5, 6, 23, 8, 15, 0, 17, 2, 3, 20, 19, 4, 21, 22, 7, 24, 63, 48, 33, 50, 51, 36, 35, 52, 37, 38, 55, 40, 47, 32, 49, 34, 35, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1179(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 5, 6, 23, 8, 31, 0, 17, 2, 3, 20, 19, 4, 21, 22, 7, 24, 47, 48, 33, 50, 51, 36, 35, 52, 37, 38, 55, 40, 63, 32, 49, 34, 35, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1180(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 5, 6, 23, 8, 15, 16, 1, 2, 3, 20, 19, 4, 21, 22, 7, 24, 63, 32, 49, 50, 51, 36, 35, 52, 37, 38, 55, 40, 47, 48, 33, 34, 35, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1181(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 5, 6, 23, 8, 31, 16, 1, 2, 3, 20, 19, 4, 21, 22, 7, 24, 47, 32, 49, 50, 51, 36, 35, 52, 37, 38, 55, 40, 63, 48, 33, 34, 35, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1182(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 5, 6, 23, 8, 15, 0, 1, 2, 3, 20, 19, 4, 21, 22, 7, 24, 63, 48, 49, 50, 51, 36, 35, 52, 37, 38, 55, 40, 47, 32, 33, 34, 35, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1183(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 5, 6, 23, 8, 31, 0, 1, 2, 3, 20, 19, 4, 21, 22, 7, 24, 47, 48, 49, 50, 51, 36, 35, 52, 37, 38, 55, 40, 63, 32, 33, 34, 35, 52, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1184(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 5, 6, 23, 8, 15, 16, 17, 18, 19, 4, 19, 4, 21, 22, 7, 24, 63, 32, 33, 34, 35, 52, 35, 52, 37, 38, 55, 40, 47, 48, 49, 50, 51, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1185(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 5, 6, 23, 8, 31, 16, 17, 18, 19, 4, 19, 4, 21, 22, 7, 24, 47, 32, 33, 34, 35, 52, 35, 52, 37, 38, 55, 40, 63, 48, 49, 50, 51, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1186(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 5, 6, 23, 8, 15, 0, 17, 18, 19, 4, 19, 4, 21, 22, 7, 24, 63, 48, 33, 34, 35, 52, 35, 52, 37, 38, 55, 40, 47, 32, 49, 50, 51, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1187(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 5, 6, 23, 8, 31, 0, 17, 18, 19, 4, 19, 4, 21, 22, 7, 24, 47, 48, 33, 34, 35, 52, 35, 52, 37, 38, 55, 40, 63, 32, 49, 50, 51, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1188(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 5, 6, 23, 8, 15, 16, 1, 18, 19, 4, 19, 4, 21, 22, 7, 24, 63, 32, 49, 34, 35, 52, 35, 52, 37, 38, 55, 40, 47, 48, 33, 50, 51, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1189(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 5, 6, 23, 8, 31, 16, 1, 18, 19, 4, 19, 4, 21, 22, 7, 24, 47, 32, 49, 34, 35, 52, 35, 52, 37, 38, 55, 40, 63, 48, 33, 50, 51, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1190(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 5, 6, 23, 8, 15, 0, 1, 18, 19, 4, 19, 4, 21, 22, 7, 24, 63, 48, 49, 34, 35, 52, 35, 52, 37, 38, 55, 40, 47, 32, 33, 50, 51, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1191(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 5, 6, 23, 8, 31, 0, 1, 18, 19, 4, 19, 4, 21, 22, 7, 24, 47, 48, 49, 34, 35, 52, 35, 52, 37, 38, 55, 40, 63, 32, 33, 50, 51, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1192(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 5, 6, 23, 8, 15, 16, 17, 2, 19, 4, 19, 4, 21, 22, 7, 24, 63, 32, 33, 50, 35, 52, 35, 52, 37, 38, 55, 40, 47, 48, 49, 34, 51, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1193(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 5, 6, 23, 8, 31, 16, 17, 2, 19, 4, 19, 4, 21, 22, 7, 24, 47, 32, 33, 50, 35, 52, 35, 52, 37, 38, 55, 40, 63, 48, 49, 34, 51, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1194(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 5, 6, 23, 8, 15, 0, 17, 2, 19, 4, 19, 4, 21, 22, 7, 24, 63, 48, 33, 50, 35, 52, 35, 52, 37, 38, 55, 40, 47, 32, 49, 34, 51, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1195(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 5, 6, 23, 8, 31, 0, 17, 2, 19, 4, 19, 4, 21, 22, 7, 24, 47, 48, 33, 50, 35, 52, 35, 52, 37, 38, 55, 40, 63, 32, 49, 34, 51, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1196(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 5, 6, 23, 8, 15, 16, 1, 2, 19, 4, 19, 4, 21, 22, 7, 24, 63, 32, 49, 50, 35, 52, 35, 52, 37, 38, 55, 40, 47, 48, 33, 34, 51, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1197(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 5, 6, 23, 8, 31, 16, 1, 2, 19, 4, 19, 4, 21, 22, 7, 24, 47, 32, 49, 50, 35, 52, 35, 52, 37, 38, 55, 40, 63, 48, 33, 34, 51, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1198(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 5, 6, 23, 8, 15, 0, 1, 2, 19, 4, 19, 4, 21, 22, 7, 24, 63, 48, 49, 50, 35, 52, 35, 52, 37, 38, 55, 40, 47, 32, 33, 34, 51, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1199(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 5, 6, 23, 8, 31, 0, 1, 2, 19, 4, 19, 4, 21, 22, 7, 24, 47, 48, 49, 50, 35, 52, 35, 52, 37, 38, 55, 40, 63, 32, 33, 34, 51, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1200(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 5, 6, 23, 8, 15, 16, 17, 18, 3, 4, 19, 4, 21, 22, 7, 24, 63, 32, 33, 34, 51, 52, 35, 52, 37, 38, 55, 40, 47, 48, 49, 50, 35, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1201(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 5, 6, 23, 8, 31, 16, 17, 18, 3, 4, 19, 4, 21, 22, 7, 24, 47, 32, 33, 34, 51, 52, 35, 52, 37, 38, 55, 40, 63, 48, 49, 50, 35, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1202(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 5, 6, 23, 8, 15, 0, 17, 18, 3, 4, 19, 4, 21, 22, 7, 24, 63, 48, 33, 34, 51, 52, 35, 52, 37, 38, 55, 40, 47, 32, 49, 50, 35, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1203(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 5, 6, 23, 8, 31, 0, 17, 18, 3, 4, 19, 4, 21, 22, 7, 24, 47, 48, 33, 34, 51, 52, 35, 52, 37, 38, 55, 40, 63, 32, 49, 50, 35, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1204(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 5, 6, 23, 8, 15, 16, 1, 18, 3, 4, 19, 4, 21, 22, 7, 24, 63, 32, 49, 34, 51, 52, 35, 52, 37, 38, 55, 40, 47, 48, 33, 50, 35, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1205(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 5, 6, 23, 8, 31, 16, 1, 18, 3, 4, 19, 4, 21, 22, 7, 24, 47, 32, 49, 34, 51, 52, 35, 52, 37, 38, 55, 40, 63, 48, 33, 50, 35, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1206(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 5, 6, 23, 8, 15, 0, 1, 18, 3, 4, 19, 4, 21, 22, 7, 24, 63, 48, 49, 34, 51, 52, 35, 52, 37, 38, 55, 40, 47, 32, 33, 50, 35, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1207(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 5, 6, 23, 8, 31, 0, 1, 18, 3, 4, 19, 4, 21, 22, 7, 24, 47, 48, 49, 34, 51, 52, 35, 52, 37, 38, 55, 40, 63, 32, 33, 50, 35, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1208(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 5, 6, 23, 8, 15, 16, 17, 2, 3, 4, 19, 4, 21, 22, 7, 24, 63, 32, 33, 50, 51, 52, 35, 52, 37, 38, 55, 40, 47, 48, 49, 34, 35, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1209(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 5, 6, 23, 8, 31, 16, 17, 2, 3, 4, 19, 4, 21, 22, 7, 24, 47, 32, 33, 50, 51, 52, 35, 52, 37, 38, 55, 40, 63, 48, 49, 34, 35, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1210(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 5, 6, 23, 8, 15, 0, 17, 2, 3, 4, 19, 4, 21, 22, 7, 24, 63, 48, 33, 50, 51, 52, 35, 52, 37, 38, 55, 40, 47, 32, 49, 34, 35, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1211(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 5, 6, 23, 8, 31, 0, 17, 2, 3, 4, 19, 4, 21, 22, 7, 24, 47, 48, 33, 50, 51, 52, 35, 52, 37, 38, 55, 40, 63, 32, 49, 34, 35, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1212(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 5, 6, 23, 8, 15, 16, 1, 2, 3, 4, 19, 4, 21, 22, 7, 24, 63, 32, 49, 50, 51, 52, 35, 52, 37, 38, 55, 40, 47, 48, 33, 34, 35, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1213(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 5, 6, 23, 8, 31, 16, 1, 2, 3, 4, 19, 4, 21, 22, 7, 24, 47, 32, 49, 50, 51, 52, 35, 52, 37, 38, 55, 40, 63, 48, 33, 34, 35, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1214(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 5, 6, 23, 8, 15, 0, 1, 2, 3, 4, 19, 4, 21, 22, 7, 24, 63, 48, 49, 50, 51, 52, 35, 52, 37, 38, 55, 40, 47, 32, 33, 34, 35, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1215(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 5, 6, 23, 8, 31, 0, 1, 2, 3, 4, 19, 4, 21, 22, 7, 24, 47, 48, 49, 50, 51, 52, 35, 52, 37, 38, 55, 40, 63, 32, 33, 34, 35, 36, 51, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1216(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 5, 6, 23, 8, 15, 16, 17, 18, 19, 20, 3, 4, 21, 22, 7, 24, 63, 32, 33, 34, 35, 36, 51, 52, 37, 38, 55, 40, 47, 48, 49, 50, 51, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1217(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 5, 6, 23, 8, 31, 16, 17, 18, 19, 20, 3, 4, 21, 22, 7, 24, 47, 32, 33, 34, 35, 36, 51, 52, 37, 38, 55, 40, 63, 48, 49, 50, 51, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1218(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 5, 6, 23, 8, 15, 0, 17, 18, 19, 20, 3, 4, 21, 22, 7, 24, 63, 48, 33, 34, 35, 36, 51, 52, 37, 38, 55, 40, 47, 32, 49, 50, 51, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1219(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 5, 6, 23, 8, 31, 0, 17, 18, 19, 20, 3, 4, 21, 22, 7, 24, 47, 48, 33, 34, 35, 36, 51, 52, 37, 38, 55, 40, 63, 32, 49, 50, 51, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1220(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 5, 6, 23, 8, 15, 16, 1, 18, 19, 20, 3, 4, 21, 22, 7, 24, 63, 32, 49, 34, 35, 36, 51, 52, 37, 38, 55, 40, 47, 48, 33, 50, 51, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1221(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 5, 6, 23, 8, 31, 16, 1, 18, 19, 20, 3, 4, 21, 22, 7, 24, 47, 32, 49, 34, 35, 36, 51, 52, 37, 38, 55, 40, 63, 48, 33, 50, 51, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1222(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 5, 6, 23, 8, 15, 0, 1, 18, 19, 20, 3, 4, 21, 22, 7, 24, 63, 48, 49, 34, 35, 36, 51, 52, 37, 38, 55, 40, 47, 32, 33, 50, 51, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1223(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 5, 6, 23, 8, 31, 0, 1, 18, 19, 20, 3, 4, 21, 22, 7, 24, 47, 48, 49, 34, 35, 36, 51, 52, 37, 38, 55, 40, 63, 32, 33, 50, 51, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1224(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 5, 6, 23, 8, 15, 16, 17, 2, 19, 20, 3, 4, 21, 22, 7, 24, 63, 32, 33, 50, 35, 36, 51, 52, 37, 38, 55, 40, 47, 48, 49, 34, 51, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1225(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 5, 6, 23, 8, 31, 16, 17, 2, 19, 20, 3, 4, 21, 22, 7, 24, 47, 32, 33, 50, 35, 36, 51, 52, 37, 38, 55, 40, 63, 48, 49, 34, 51, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1226(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 5, 6, 23, 8, 15, 0, 17, 2, 19, 20, 3, 4, 21, 22, 7, 24, 63, 48, 33, 50, 35, 36, 51, 52, 37, 38, 55, 40, 47, 32, 49, 34, 51, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1227(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 5, 6, 23, 8, 31, 0, 17, 2, 19, 20, 3, 4, 21, 22, 7, 24, 47, 48, 33, 50, 35, 36, 51, 52, 37, 38, 55, 40, 63, 32, 49, 34, 51, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1228(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 5, 6, 23, 8, 15, 16, 1, 2, 19, 20, 3, 4, 21, 22, 7, 24, 63, 32, 49, 50, 35, 36, 51, 52, 37, 38, 55, 40, 47, 48, 33, 34, 51, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1229(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 5, 6, 23, 8, 31, 16, 1, 2, 19, 20, 3, 4, 21, 22, 7, 24, 47, 32, 49, 50, 35, 36, 51, 52, 37, 38, 55, 40, 63, 48, 33, 34, 51, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1230(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 5, 6, 23, 8, 15, 0, 1, 2, 19, 20, 3, 4, 21, 22, 7, 24, 63, 48, 49, 50, 35, 36, 51, 52, 37, 38, 55, 40, 47, 32, 33, 34, 51, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1231(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 5, 6, 23, 8, 31, 0, 1, 2, 19, 20, 3, 4, 21, 22, 7, 24, 47, 48, 49, 50, 35, 36, 51, 52, 37, 38, 55, 40, 63, 32, 33, 34, 51, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1232(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 5, 6, 23, 8, 15, 16, 17, 18, 3, 20, 3, 4, 21, 22, 7, 24, 63, 32, 33, 34, 51, 36, 51, 52, 37, 38, 55, 40, 47, 48, 49, 50, 35, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1233(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 5, 6, 23, 8, 31, 16, 17, 18, 3, 20, 3, 4, 21, 22, 7, 24, 47, 32, 33, 34, 51, 36, 51, 52, 37, 38, 55, 40, 63, 48, 49, 50, 35, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1234(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 5, 6, 23, 8, 15, 0, 17, 18, 3, 20, 3, 4, 21, 22, 7, 24, 63, 48, 33, 34, 51, 36, 51, 52, 37, 38, 55, 40, 47, 32, 49, 50, 35, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1235(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 5, 6, 23, 8, 31, 0, 17, 18, 3, 20, 3, 4, 21, 22, 7, 24, 47, 48, 33, 34, 51, 36, 51, 52, 37, 38, 55, 40, 63, 32, 49, 50, 35, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1236(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 5, 6, 23, 8, 15, 16, 1, 18, 3, 20, 3, 4, 21, 22, 7, 24, 63, 32, 49, 34, 51, 36, 51, 52, 37, 38, 55, 40, 47, 48, 33, 50, 35, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1237(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 5, 6, 23, 8, 31, 16, 1, 18, 3, 20, 3, 4, 21, 22, 7, 24, 47, 32, 49, 34, 51, 36, 51, 52, 37, 38, 55, 40, 63, 48, 33, 50, 35, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1238(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 5, 6, 23, 8, 15, 0, 1, 18, 3, 20, 3, 4, 21, 22, 7, 24, 63, 48, 49, 34, 51, 36, 51, 52, 37, 38, 55, 40, 47, 32, 33, 50, 35, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1239(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 5, 6, 23, 8, 31, 0, 1, 18, 3, 20, 3, 4, 21, 22, 7, 24, 47, 48, 49, 34, 51, 36, 51, 52, 37, 38, 55, 40, 63, 32, 33, 50, 35, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1240(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 5, 6, 23, 8, 15, 16, 17, 2, 3, 20, 3, 4, 21, 22, 7, 24, 63, 32, 33, 50, 51, 36, 51, 52, 37, 38, 55, 40, 47, 48, 49, 34, 35, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1241(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 5, 6, 23, 8, 31, 16, 17, 2, 3, 20, 3, 4, 21, 22, 7, 24, 47, 32, 33, 50, 51, 36, 51, 52, 37, 38, 55, 40, 63, 48, 49, 34, 35, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1242(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 5, 6, 23, 8, 15, 0, 17, 2, 3, 20, 3, 4, 21, 22, 7, 24, 63, 48, 33, 50, 51, 36, 51, 52, 37, 38, 55, 40, 47, 32, 49, 34, 35, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1243(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 5, 6, 23, 8, 31, 0, 17, 2, 3, 20, 3, 4, 21, 22, 7, 24, 47, 48, 33, 50, 51, 36, 51, 52, 37, 38, 55, 40, 63, 32, 49, 34, 35, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1244(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 5, 6, 23, 8, 15, 16, 1, 2, 3, 20, 3, 4, 21, 22, 7, 24, 63, 32, 49, 50, 51, 36, 51, 52, 37, 38, 55, 40, 47, 48, 33, 34, 35, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1245(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 5, 6, 23, 8, 31, 16, 1, 2, 3, 20, 3, 4, 21, 22, 7, 24, 47, 32, 49, 50, 51, 36, 51, 52, 37, 38, 55, 40, 63, 48, 33, 34, 35, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1246(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 5, 6, 23, 8, 15, 0, 1, 2, 3, 20, 3, 4, 21, 22, 7, 24, 63, 48, 49, 50, 51, 36, 51, 52, 37, 38, 55, 40, 47, 32, 33, 34, 35, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1247(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 5, 6, 23, 8, 31, 0, 1, 2, 3, 20, 3, 4, 21, 22, 7, 24, 47, 48, 49, 50, 51, 36, 51, 52, 37, 38, 55, 40, 63, 32, 33, 34, 35, 52, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1248(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 5, 6, 23, 8, 15, 16, 17, 18, 19, 4, 3, 4, 21, 22, 7, 24, 63, 32, 33, 34, 35, 52, 51, 52, 37, 38, 55, 40, 47, 48, 49, 50, 51, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1249(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 5, 6, 23, 8, 31, 16, 17, 18, 19, 4, 3, 4, 21, 22, 7, 24, 47, 32, 33, 34, 35, 52, 51, 52, 37, 38, 55, 40, 63, 48, 49, 50, 51, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1250(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 5, 6, 23, 8, 15, 0, 17, 18, 19, 4, 3, 4, 21, 22, 7, 24, 63, 48, 33, 34, 35, 52, 51, 52, 37, 38, 55, 40, 47, 32, 49, 50, 51, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1251(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 5, 6, 23, 8, 31, 0, 17, 18, 19, 4, 3, 4, 21, 22, 7, 24, 47, 48, 33, 34, 35, 52, 51, 52, 37, 38, 55, 40, 63, 32, 49, 50, 51, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1252(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 5, 6, 23, 8, 15, 16, 1, 18, 19, 4, 3, 4, 21, 22, 7, 24, 63, 32, 49, 34, 35, 52, 51, 52, 37, 38, 55, 40, 47, 48, 33, 50, 51, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1253(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 5, 6, 23, 8, 31, 16, 1, 18, 19, 4, 3, 4, 21, 22, 7, 24, 47, 32, 49, 34, 35, 52, 51, 52, 37, 38, 55, 40, 63, 48, 33, 50, 51, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1254(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 5, 6, 23, 8, 15, 0, 1, 18, 19, 4, 3, 4, 21, 22, 7, 24, 63, 48, 49, 34, 35, 52, 51, 52, 37, 38, 55, 40, 47, 32, 33, 50, 51, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1255(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 5, 6, 23, 8, 31, 0, 1, 18, 19, 4, 3, 4, 21, 22, 7, 24, 47, 48, 49, 34, 35, 52, 51, 52, 37, 38, 55, 40, 63, 32, 33, 50, 51, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1256(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 5, 6, 23, 8, 15, 16, 17, 2, 19, 4, 3, 4, 21, 22, 7, 24, 63, 32, 33, 50, 35, 52, 51, 52, 37, 38, 55, 40, 47, 48, 49, 34, 51, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1257(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 5, 6, 23, 8, 31, 16, 17, 2, 19, 4, 3, 4, 21, 22, 7, 24, 47, 32, 33, 50, 35, 52, 51, 52, 37, 38, 55, 40, 63, 48, 49, 34, 51, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1258(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 5, 6, 23, 8, 15, 0, 17, 2, 19, 4, 3, 4, 21, 22, 7, 24, 63, 48, 33, 50, 35, 52, 51, 52, 37, 38, 55, 40, 47, 32, 49, 34, 51, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1259(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 5, 6, 23, 8, 31, 0, 17, 2, 19, 4, 3, 4, 21, 22, 7, 24, 47, 48, 33, 50, 35, 52, 51, 52, 37, 38, 55, 40, 63, 32, 49, 34, 51, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1260(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 5, 6, 23, 8, 15, 16, 1, 2, 19, 4, 3, 4, 21, 22, 7, 24, 63, 32, 49, 50, 35, 52, 51, 52, 37, 38, 55, 40, 47, 48, 33, 34, 51, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1261(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 5, 6, 23, 8, 31, 16, 1, 2, 19, 4, 3, 4, 21, 22, 7, 24, 47, 32, 49, 50, 35, 52, 51, 52, 37, 38, 55, 40, 63, 48, 33, 34, 51, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1262(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 5, 6, 23, 8, 15, 0, 1, 2, 19, 4, 3, 4, 21, 22, 7, 24, 63, 48, 49, 50, 35, 52, 51, 52, 37, 38, 55, 40, 47, 32, 33, 34, 51, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1263(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 5, 6, 23, 8, 31, 0, 1, 2, 19, 4, 3, 4, 21, 22, 7, 24, 47, 48, 49, 50, 35, 52, 51, 52, 37, 38, 55, 40, 63, 32, 33, 34, 51, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1264(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 5, 6, 23, 8, 15, 16, 17, 18, 3, 4, 3, 4, 21, 22, 7, 24, 63, 32, 33, 34, 51, 52, 51, 52, 37, 38, 55, 40, 47, 48, 49, 50, 35, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1265(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 5, 6, 23, 8, 31, 16, 17, 18, 3, 4, 3, 4, 21, 22, 7, 24, 47, 32, 33, 34, 51, 52, 51, 52, 37, 38, 55, 40, 63, 48, 49, 50, 35, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1266(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 5, 6, 23, 8, 15, 0, 17, 18, 3, 4, 3, 4, 21, 22, 7, 24, 63, 48, 33, 34, 51, 52, 51, 52, 37, 38, 55, 40, 47, 32, 49, 50, 35, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1267(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 5, 6, 23, 8, 31, 0, 17, 18, 3, 4, 3, 4, 21, 22, 7, 24, 47, 48, 33, 34, 51, 52, 51, 52, 37, 38, 55, 40, 63, 32, 49, 50, 35, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1268(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 5, 6, 23, 8, 15, 16, 1, 18, 3, 4, 3, 4, 21, 22, 7, 24, 63, 32, 49, 34, 51, 52, 51, 52, 37, 38, 55, 40, 47, 48, 33, 50, 35, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1269(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 5, 6, 23, 8, 31, 16, 1, 18, 3, 4, 3, 4, 21, 22, 7, 24, 47, 32, 49, 34, 51, 52, 51, 52, 37, 38, 55, 40, 63, 48, 33, 50, 35, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1270(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 5, 6, 23, 8, 15, 0, 1, 18, 3, 4, 3, 4, 21, 22, 7, 24, 63, 48, 49, 34, 51, 52, 51, 52, 37, 38, 55, 40, 47, 32, 33, 50, 35, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1271(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 5, 6, 23, 8, 31, 0, 1, 18, 3, 4, 3, 4, 21, 22, 7, 24, 47, 48, 49, 34, 51, 52, 51, 52, 37, 38, 55, 40, 63, 32, 33, 50, 35, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1272(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 5, 6, 23, 8, 15, 16, 17, 2, 3, 4, 3, 4, 21, 22, 7, 24, 63, 32, 33, 50, 51, 52, 51, 52, 37, 38, 55, 40, 47, 48, 49, 34, 35, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1273(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 5, 6, 23, 8, 31, 16, 17, 2, 3, 4, 3, 4, 21, 22, 7, 24, 47, 32, 33, 50, 51, 52, 51, 52, 37, 38, 55, 40, 63, 48, 49, 34, 35, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1274(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 5, 6, 23, 8, 15, 0, 17, 2, 3, 4, 3, 4, 21, 22, 7, 24, 63, 48, 33, 50, 51, 52, 51, 52, 37, 38, 55, 40, 47, 32, 49, 34, 35, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1275(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 5, 6, 23, 8, 31, 0, 17, 2, 3, 4, 3, 4, 21, 22, 7, 24, 47, 48, 33, 50, 51, 52, 51, 52, 37, 38, 55, 40, 63, 32, 49, 34, 35, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1276(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 5, 6, 23, 8, 15, 16, 1, 2, 3, 4, 3, 4, 21, 22, 7, 24, 63, 32, 49, 50, 51, 52, 51, 52, 37, 38, 55, 40, 47, 48, 33, 34, 35, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1277(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 5, 6, 23, 8, 31, 16, 1, 2, 3, 4, 3, 4, 21, 22, 7, 24, 47, 32, 49, 50, 51, 52, 51, 52, 37, 38, 55, 40, 63, 48, 33, 34, 35, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1278(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 5, 6, 23, 8, 15, 0, 1, 2, 3, 4, 3, 4, 21, 22, 7, 24, 63, 48, 49, 50, 51, 52, 51, 52, 37, 38, 55, 40, 47, 32, 33, 34, 35, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1279(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 5, 6, 23, 8, 31, 0, 1, 2, 3, 4, 3, 4, 21, 22, 7, 24, 47, 48, 49, 50, 51, 52, 51, 52, 37, 38, 55, 40, 63, 32, 33, 34, 35, 36, 35, 36, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}


