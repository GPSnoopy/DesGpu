#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

#ifdef DESGPU_COMPILE_ALL_SALTS

void des_25_encrypt_salt3584(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 4, 5, 22, 23, 24, 15, 16, 17, 18, 19, 20, 19, 20, 21, 6, 7, 8, 63, 32, 33, 34, 35, 36, 35, 36, 37, 54, 55, 56, 47, 48, 49, 50, 51, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3585(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 4, 5, 22, 23, 24, 31, 16, 17, 18, 19, 20, 19, 20, 21, 6, 7, 8, 47, 32, 33, 34, 35, 36, 35, 36, 37, 54, 55, 56, 63, 48, 49, 50, 51, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3586(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 4, 5, 22, 23, 24, 15, 0, 17, 18, 19, 20, 19, 20, 21, 6, 7, 8, 63, 48, 33, 34, 35, 36, 35, 36, 37, 54, 55, 56, 47, 32, 49, 50, 51, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3587(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 4, 5, 22, 23, 24, 31, 0, 17, 18, 19, 20, 19, 20, 21, 6, 7, 8, 47, 48, 33, 34, 35, 36, 35, 36, 37, 54, 55, 56, 63, 32, 49, 50, 51, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3588(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 4, 5, 22, 23, 24, 15, 16, 1, 18, 19, 20, 19, 20, 21, 6, 7, 8, 63, 32, 49, 34, 35, 36, 35, 36, 37, 54, 55, 56, 47, 48, 33, 50, 51, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3589(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 4, 5, 22, 23, 24, 31, 16, 1, 18, 19, 20, 19, 20, 21, 6, 7, 8, 47, 32, 49, 34, 35, 36, 35, 36, 37, 54, 55, 56, 63, 48, 33, 50, 51, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3590(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 4, 5, 22, 23, 24, 15, 0, 1, 18, 19, 20, 19, 20, 21, 6, 7, 8, 63, 48, 49, 34, 35, 36, 35, 36, 37, 54, 55, 56, 47, 32, 33, 50, 51, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3591(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 4, 5, 22, 23, 24, 31, 0, 1, 18, 19, 20, 19, 20, 21, 6, 7, 8, 47, 48, 49, 34, 35, 36, 35, 36, 37, 54, 55, 56, 63, 32, 33, 50, 51, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3592(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 4, 5, 22, 23, 24, 15, 16, 17, 2, 19, 20, 19, 20, 21, 6, 7, 8, 63, 32, 33, 50, 35, 36, 35, 36, 37, 54, 55, 56, 47, 48, 49, 34, 51, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3593(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 4, 5, 22, 23, 24, 31, 16, 17, 2, 19, 20, 19, 20, 21, 6, 7, 8, 47, 32, 33, 50, 35, 36, 35, 36, 37, 54, 55, 56, 63, 48, 49, 34, 51, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3594(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 4, 5, 22, 23, 24, 15, 0, 17, 2, 19, 20, 19, 20, 21, 6, 7, 8, 63, 48, 33, 50, 35, 36, 35, 36, 37, 54, 55, 56, 47, 32, 49, 34, 51, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3595(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 4, 5, 22, 23, 24, 31, 0, 17, 2, 19, 20, 19, 20, 21, 6, 7, 8, 47, 48, 33, 50, 35, 36, 35, 36, 37, 54, 55, 56, 63, 32, 49, 34, 51, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3596(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 4, 5, 22, 23, 24, 15, 16, 1, 2, 19, 20, 19, 20, 21, 6, 7, 8, 63, 32, 49, 50, 35, 36, 35, 36, 37, 54, 55, 56, 47, 48, 33, 34, 51, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3597(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 4, 5, 22, 23, 24, 31, 16, 1, 2, 19, 20, 19, 20, 21, 6, 7, 8, 47, 32, 49, 50, 35, 36, 35, 36, 37, 54, 55, 56, 63, 48, 33, 34, 51, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3598(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 4, 5, 22, 23, 24, 15, 0, 1, 2, 19, 20, 19, 20, 21, 6, 7, 8, 63, 48, 49, 50, 35, 36, 35, 36, 37, 54, 55, 56, 47, 32, 33, 34, 51, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3599(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 4, 5, 22, 23, 24, 31, 0, 1, 2, 19, 20, 19, 20, 21, 6, 7, 8, 47, 48, 49, 50, 35, 36, 35, 36, 37, 54, 55, 56, 63, 32, 33, 34, 51, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3600(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 4, 5, 22, 23, 24, 15, 16, 17, 18, 3, 20, 19, 20, 21, 6, 7, 8, 63, 32, 33, 34, 51, 36, 35, 36, 37, 54, 55, 56, 47, 48, 49, 50, 35, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3601(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 4, 5, 22, 23, 24, 31, 16, 17, 18, 3, 20, 19, 20, 21, 6, 7, 8, 47, 32, 33, 34, 51, 36, 35, 36, 37, 54, 55, 56, 63, 48, 49, 50, 35, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3602(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 4, 5, 22, 23, 24, 15, 0, 17, 18, 3, 20, 19, 20, 21, 6, 7, 8, 63, 48, 33, 34, 51, 36, 35, 36, 37, 54, 55, 56, 47, 32, 49, 50, 35, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3603(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 4, 5, 22, 23, 24, 31, 0, 17, 18, 3, 20, 19, 20, 21, 6, 7, 8, 47, 48, 33, 34, 51, 36, 35, 36, 37, 54, 55, 56, 63, 32, 49, 50, 35, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3604(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 4, 5, 22, 23, 24, 15, 16, 1, 18, 3, 20, 19, 20, 21, 6, 7, 8, 63, 32, 49, 34, 51, 36, 35, 36, 37, 54, 55, 56, 47, 48, 33, 50, 35, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3605(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 4, 5, 22, 23, 24, 31, 16, 1, 18, 3, 20, 19, 20, 21, 6, 7, 8, 47, 32, 49, 34, 51, 36, 35, 36, 37, 54, 55, 56, 63, 48, 33, 50, 35, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3606(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 4, 5, 22, 23, 24, 15, 0, 1, 18, 3, 20, 19, 20, 21, 6, 7, 8, 63, 48, 49, 34, 51, 36, 35, 36, 37, 54, 55, 56, 47, 32, 33, 50, 35, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3607(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 4, 5, 22, 23, 24, 31, 0, 1, 18, 3, 20, 19, 20, 21, 6, 7, 8, 47, 48, 49, 34, 51, 36, 35, 36, 37, 54, 55, 56, 63, 32, 33, 50, 35, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3608(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 4, 5, 22, 23, 24, 15, 16, 17, 2, 3, 20, 19, 20, 21, 6, 7, 8, 63, 32, 33, 50, 51, 36, 35, 36, 37, 54, 55, 56, 47, 48, 49, 34, 35, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3609(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 4, 5, 22, 23, 24, 31, 16, 17, 2, 3, 20, 19, 20, 21, 6, 7, 8, 47, 32, 33, 50, 51, 36, 35, 36, 37, 54, 55, 56, 63, 48, 49, 34, 35, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3610(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 4, 5, 22, 23, 24, 15, 0, 17, 2, 3, 20, 19, 20, 21, 6, 7, 8, 63, 48, 33, 50, 51, 36, 35, 36, 37, 54, 55, 56, 47, 32, 49, 34, 35, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3611(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 4, 5, 22, 23, 24, 31, 0, 17, 2, 3, 20, 19, 20, 21, 6, 7, 8, 47, 48, 33, 50, 51, 36, 35, 36, 37, 54, 55, 56, 63, 32, 49, 34, 35, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3612(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 4, 5, 22, 23, 24, 15, 16, 1, 2, 3, 20, 19, 20, 21, 6, 7, 8, 63, 32, 49, 50, 51, 36, 35, 36, 37, 54, 55, 56, 47, 48, 33, 34, 35, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3613(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 4, 5, 22, 23, 24, 31, 16, 1, 2, 3, 20, 19, 20, 21, 6, 7, 8, 47, 32, 49, 50, 51, 36, 35, 36, 37, 54, 55, 56, 63, 48, 33, 34, 35, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3614(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 4, 5, 22, 23, 24, 15, 0, 1, 2, 3, 20, 19, 20, 21, 6, 7, 8, 63, 48, 49, 50, 51, 36, 35, 36, 37, 54, 55, 56, 47, 32, 33, 34, 35, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3615(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 4, 5, 22, 23, 24, 31, 0, 1, 2, 3, 20, 19, 20, 21, 6, 7, 8, 47, 48, 49, 50, 51, 36, 35, 36, 37, 54, 55, 56, 63, 32, 33, 34, 35, 52, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3616(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 4, 5, 22, 23, 24, 15, 16, 17, 18, 19, 4, 19, 20, 21, 6, 7, 8, 63, 32, 33, 34, 35, 52, 35, 36, 37, 54, 55, 56, 47, 48, 49, 50, 51, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3617(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 4, 5, 22, 23, 24, 31, 16, 17, 18, 19, 4, 19, 20, 21, 6, 7, 8, 47, 32, 33, 34, 35, 52, 35, 36, 37, 54, 55, 56, 63, 48, 49, 50, 51, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3618(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 4, 5, 22, 23, 24, 15, 0, 17, 18, 19, 4, 19, 20, 21, 6, 7, 8, 63, 48, 33, 34, 35, 52, 35, 36, 37, 54, 55, 56, 47, 32, 49, 50, 51, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3619(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 4, 5, 22, 23, 24, 31, 0, 17, 18, 19, 4, 19, 20, 21, 6, 7, 8, 47, 48, 33, 34, 35, 52, 35, 36, 37, 54, 55, 56, 63, 32, 49, 50, 51, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3620(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 4, 5, 22, 23, 24, 15, 16, 1, 18, 19, 4, 19, 20, 21, 6, 7, 8, 63, 32, 49, 34, 35, 52, 35, 36, 37, 54, 55, 56, 47, 48, 33, 50, 51, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3621(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 4, 5, 22, 23, 24, 31, 16, 1, 18, 19, 4, 19, 20, 21, 6, 7, 8, 47, 32, 49, 34, 35, 52, 35, 36, 37, 54, 55, 56, 63, 48, 33, 50, 51, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3622(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 4, 5, 22, 23, 24, 15, 0, 1, 18, 19, 4, 19, 20, 21, 6, 7, 8, 63, 48, 49, 34, 35, 52, 35, 36, 37, 54, 55, 56, 47, 32, 33, 50, 51, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3623(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 4, 5, 22, 23, 24, 31, 0, 1, 18, 19, 4, 19, 20, 21, 6, 7, 8, 47, 48, 49, 34, 35, 52, 35, 36, 37, 54, 55, 56, 63, 32, 33, 50, 51, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3624(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 4, 5, 22, 23, 24, 15, 16, 17, 2, 19, 4, 19, 20, 21, 6, 7, 8, 63, 32, 33, 50, 35, 52, 35, 36, 37, 54, 55, 56, 47, 48, 49, 34, 51, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3625(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 4, 5, 22, 23, 24, 31, 16, 17, 2, 19, 4, 19, 20, 21, 6, 7, 8, 47, 32, 33, 50, 35, 52, 35, 36, 37, 54, 55, 56, 63, 48, 49, 34, 51, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3626(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 4, 5, 22, 23, 24, 15, 0, 17, 2, 19, 4, 19, 20, 21, 6, 7, 8, 63, 48, 33, 50, 35, 52, 35, 36, 37, 54, 55, 56, 47, 32, 49, 34, 51, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3627(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 4, 5, 22, 23, 24, 31, 0, 17, 2, 19, 4, 19, 20, 21, 6, 7, 8, 47, 48, 33, 50, 35, 52, 35, 36, 37, 54, 55, 56, 63, 32, 49, 34, 51, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3628(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 4, 5, 22, 23, 24, 15, 16, 1, 2, 19, 4, 19, 20, 21, 6, 7, 8, 63, 32, 49, 50, 35, 52, 35, 36, 37, 54, 55, 56, 47, 48, 33, 34, 51, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3629(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 4, 5, 22, 23, 24, 31, 16, 1, 2, 19, 4, 19, 20, 21, 6, 7, 8, 47, 32, 49, 50, 35, 52, 35, 36, 37, 54, 55, 56, 63, 48, 33, 34, 51, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3630(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 4, 5, 22, 23, 24, 15, 0, 1, 2, 19, 4, 19, 20, 21, 6, 7, 8, 63, 48, 49, 50, 35, 52, 35, 36, 37, 54, 55, 56, 47, 32, 33, 34, 51, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3631(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 4, 5, 22, 23, 24, 31, 0, 1, 2, 19, 4, 19, 20, 21, 6, 7, 8, 47, 48, 49, 50, 35, 52, 35, 36, 37, 54, 55, 56, 63, 32, 33, 34, 51, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3632(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 4, 5, 22, 23, 24, 15, 16, 17, 18, 3, 4, 19, 20, 21, 6, 7, 8, 63, 32, 33, 34, 51, 52, 35, 36, 37, 54, 55, 56, 47, 48, 49, 50, 35, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3633(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 4, 5, 22, 23, 24, 31, 16, 17, 18, 3, 4, 19, 20, 21, 6, 7, 8, 47, 32, 33, 34, 51, 52, 35, 36, 37, 54, 55, 56, 63, 48, 49, 50, 35, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3634(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 4, 5, 22, 23, 24, 15, 0, 17, 18, 3, 4, 19, 20, 21, 6, 7, 8, 63, 48, 33, 34, 51, 52, 35, 36, 37, 54, 55, 56, 47, 32, 49, 50, 35, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3635(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 4, 5, 22, 23, 24, 31, 0, 17, 18, 3, 4, 19, 20, 21, 6, 7, 8, 47, 48, 33, 34, 51, 52, 35, 36, 37, 54, 55, 56, 63, 32, 49, 50, 35, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3636(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 4, 5, 22, 23, 24, 15, 16, 1, 18, 3, 4, 19, 20, 21, 6, 7, 8, 63, 32, 49, 34, 51, 52, 35, 36, 37, 54, 55, 56, 47, 48, 33, 50, 35, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3637(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 4, 5, 22, 23, 24, 31, 16, 1, 18, 3, 4, 19, 20, 21, 6, 7, 8, 47, 32, 49, 34, 51, 52, 35, 36, 37, 54, 55, 56, 63, 48, 33, 50, 35, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3638(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 4, 5, 22, 23, 24, 15, 0, 1, 18, 3, 4, 19, 20, 21, 6, 7, 8, 63, 48, 49, 34, 51, 52, 35, 36, 37, 54, 55, 56, 47, 32, 33, 50, 35, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3639(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 4, 5, 22, 23, 24, 31, 0, 1, 18, 3, 4, 19, 20, 21, 6, 7, 8, 47, 48, 49, 34, 51, 52, 35, 36, 37, 54, 55, 56, 63, 32, 33, 50, 35, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3640(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 4, 5, 22, 23, 24, 15, 16, 17, 2, 3, 4, 19, 20, 21, 6, 7, 8, 63, 32, 33, 50, 51, 52, 35, 36, 37, 54, 55, 56, 47, 48, 49, 34, 35, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3641(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 4, 5, 22, 23, 24, 31, 16, 17, 2, 3, 4, 19, 20, 21, 6, 7, 8, 47, 32, 33, 50, 51, 52, 35, 36, 37, 54, 55, 56, 63, 48, 49, 34, 35, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3642(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 4, 5, 22, 23, 24, 15, 0, 17, 2, 3, 4, 19, 20, 21, 6, 7, 8, 63, 48, 33, 50, 51, 52, 35, 36, 37, 54, 55, 56, 47, 32, 49, 34, 35, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3643(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 4, 5, 22, 23, 24, 31, 0, 17, 2, 3, 4, 19, 20, 21, 6, 7, 8, 47, 48, 33, 50, 51, 52, 35, 36, 37, 54, 55, 56, 63, 32, 49, 34, 35, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3644(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 4, 5, 22, 23, 24, 15, 16, 1, 2, 3, 4, 19, 20, 21, 6, 7, 8, 63, 32, 49, 50, 51, 52, 35, 36, 37, 54, 55, 56, 47, 48, 33, 34, 35, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3645(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 4, 5, 22, 23, 24, 31, 16, 1, 2, 3, 4, 19, 20, 21, 6, 7, 8, 47, 32, 49, 50, 51, 52, 35, 36, 37, 54, 55, 56, 63, 48, 33, 34, 35, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3646(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 4, 5, 22, 23, 24, 15, 0, 1, 2, 3, 4, 19, 20, 21, 6, 7, 8, 63, 48, 49, 50, 51, 52, 35, 36, 37, 54, 55, 56, 47, 32, 33, 34, 35, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3647(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 4, 5, 22, 23, 24, 31, 0, 1, 2, 3, 4, 19, 20, 21, 6, 7, 8, 47, 48, 49, 50, 51, 52, 35, 36, 37, 54, 55, 56, 63, 32, 33, 34, 35, 36, 51, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3648(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 4, 5, 22, 23, 24, 15, 16, 17, 18, 19, 20, 3, 20, 21, 6, 7, 8, 63, 32, 33, 34, 35, 36, 51, 36, 37, 54, 55, 56, 47, 48, 49, 50, 51, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3649(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 4, 5, 22, 23, 24, 31, 16, 17, 18, 19, 20, 3, 20, 21, 6, 7, 8, 47, 32, 33, 34, 35, 36, 51, 36, 37, 54, 55, 56, 63, 48, 49, 50, 51, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3650(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 4, 5, 22, 23, 24, 15, 0, 17, 18, 19, 20, 3, 20, 21, 6, 7, 8, 63, 48, 33, 34, 35, 36, 51, 36, 37, 54, 55, 56, 47, 32, 49, 50, 51, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3651(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 4, 5, 22, 23, 24, 31, 0, 17, 18, 19, 20, 3, 20, 21, 6, 7, 8, 47, 48, 33, 34, 35, 36, 51, 36, 37, 54, 55, 56, 63, 32, 49, 50, 51, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3652(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 4, 5, 22, 23, 24, 15, 16, 1, 18, 19, 20, 3, 20, 21, 6, 7, 8, 63, 32, 49, 34, 35, 36, 51, 36, 37, 54, 55, 56, 47, 48, 33, 50, 51, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3653(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 4, 5, 22, 23, 24, 31, 16, 1, 18, 19, 20, 3, 20, 21, 6, 7, 8, 47, 32, 49, 34, 35, 36, 51, 36, 37, 54, 55, 56, 63, 48, 33, 50, 51, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3654(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 4, 5, 22, 23, 24, 15, 0, 1, 18, 19, 20, 3, 20, 21, 6, 7, 8, 63, 48, 49, 34, 35, 36, 51, 36, 37, 54, 55, 56, 47, 32, 33, 50, 51, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3655(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 4, 5, 22, 23, 24, 31, 0, 1, 18, 19, 20, 3, 20, 21, 6, 7, 8, 47, 48, 49, 34, 35, 36, 51, 36, 37, 54, 55, 56, 63, 32, 33, 50, 51, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3656(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 4, 5, 22, 23, 24, 15, 16, 17, 2, 19, 20, 3, 20, 21, 6, 7, 8, 63, 32, 33, 50, 35, 36, 51, 36, 37, 54, 55, 56, 47, 48, 49, 34, 51, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3657(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 4, 5, 22, 23, 24, 31, 16, 17, 2, 19, 20, 3, 20, 21, 6, 7, 8, 47, 32, 33, 50, 35, 36, 51, 36, 37, 54, 55, 56, 63, 48, 49, 34, 51, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3658(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 4, 5, 22, 23, 24, 15, 0, 17, 2, 19, 20, 3, 20, 21, 6, 7, 8, 63, 48, 33, 50, 35, 36, 51, 36, 37, 54, 55, 56, 47, 32, 49, 34, 51, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3659(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 4, 5, 22, 23, 24, 31, 0, 17, 2, 19, 20, 3, 20, 21, 6, 7, 8, 47, 48, 33, 50, 35, 36, 51, 36, 37, 54, 55, 56, 63, 32, 49, 34, 51, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3660(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 4, 5, 22, 23, 24, 15, 16, 1, 2, 19, 20, 3, 20, 21, 6, 7, 8, 63, 32, 49, 50, 35, 36, 51, 36, 37, 54, 55, 56, 47, 48, 33, 34, 51, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3661(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 4, 5, 22, 23, 24, 31, 16, 1, 2, 19, 20, 3, 20, 21, 6, 7, 8, 47, 32, 49, 50, 35, 36, 51, 36, 37, 54, 55, 56, 63, 48, 33, 34, 51, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3662(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 4, 5, 22, 23, 24, 15, 0, 1, 2, 19, 20, 3, 20, 21, 6, 7, 8, 63, 48, 49, 50, 35, 36, 51, 36, 37, 54, 55, 56, 47, 32, 33, 34, 51, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3663(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 4, 5, 22, 23, 24, 31, 0, 1, 2, 19, 20, 3, 20, 21, 6, 7, 8, 47, 48, 49, 50, 35, 36, 51, 36, 37, 54, 55, 56, 63, 32, 33, 34, 51, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3664(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 4, 5, 22, 23, 24, 15, 16, 17, 18, 3, 20, 3, 20, 21, 6, 7, 8, 63, 32, 33, 34, 51, 36, 51, 36, 37, 54, 55, 56, 47, 48, 49, 50, 35, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3665(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 4, 5, 22, 23, 24, 31, 16, 17, 18, 3, 20, 3, 20, 21, 6, 7, 8, 47, 32, 33, 34, 51, 36, 51, 36, 37, 54, 55, 56, 63, 48, 49, 50, 35, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3666(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 4, 5, 22, 23, 24, 15, 0, 17, 18, 3, 20, 3, 20, 21, 6, 7, 8, 63, 48, 33, 34, 51, 36, 51, 36, 37, 54, 55, 56, 47, 32, 49, 50, 35, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3667(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 4, 5, 22, 23, 24, 31, 0, 17, 18, 3, 20, 3, 20, 21, 6, 7, 8, 47, 48, 33, 34, 51, 36, 51, 36, 37, 54, 55, 56, 63, 32, 49, 50, 35, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3668(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 4, 5, 22, 23, 24, 15, 16, 1, 18, 3, 20, 3, 20, 21, 6, 7, 8, 63, 32, 49, 34, 51, 36, 51, 36, 37, 54, 55, 56, 47, 48, 33, 50, 35, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3669(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 4, 5, 22, 23, 24, 31, 16, 1, 18, 3, 20, 3, 20, 21, 6, 7, 8, 47, 32, 49, 34, 51, 36, 51, 36, 37, 54, 55, 56, 63, 48, 33, 50, 35, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3670(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 4, 5, 22, 23, 24, 15, 0, 1, 18, 3, 20, 3, 20, 21, 6, 7, 8, 63, 48, 49, 34, 51, 36, 51, 36, 37, 54, 55, 56, 47, 32, 33, 50, 35, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3671(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 4, 5, 22, 23, 24, 31, 0, 1, 18, 3, 20, 3, 20, 21, 6, 7, 8, 47, 48, 49, 34, 51, 36, 51, 36, 37, 54, 55, 56, 63, 32, 33, 50, 35, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3672(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 4, 5, 22, 23, 24, 15, 16, 17, 2, 3, 20, 3, 20, 21, 6, 7, 8, 63, 32, 33, 50, 51, 36, 51, 36, 37, 54, 55, 56, 47, 48, 49, 34, 35, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3673(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 4, 5, 22, 23, 24, 31, 16, 17, 2, 3, 20, 3, 20, 21, 6, 7, 8, 47, 32, 33, 50, 51, 36, 51, 36, 37, 54, 55, 56, 63, 48, 49, 34, 35, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3674(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 4, 5, 22, 23, 24, 15, 0, 17, 2, 3, 20, 3, 20, 21, 6, 7, 8, 63, 48, 33, 50, 51, 36, 51, 36, 37, 54, 55, 56, 47, 32, 49, 34, 35, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3675(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 4, 5, 22, 23, 24, 31, 0, 17, 2, 3, 20, 3, 20, 21, 6, 7, 8, 47, 48, 33, 50, 51, 36, 51, 36, 37, 54, 55, 56, 63, 32, 49, 34, 35, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3676(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 4, 5, 22, 23, 24, 15, 16, 1, 2, 3, 20, 3, 20, 21, 6, 7, 8, 63, 32, 49, 50, 51, 36, 51, 36, 37, 54, 55, 56, 47, 48, 33, 34, 35, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3677(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 4, 5, 22, 23, 24, 31, 16, 1, 2, 3, 20, 3, 20, 21, 6, 7, 8, 47, 32, 49, 50, 51, 36, 51, 36, 37, 54, 55, 56, 63, 48, 33, 34, 35, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3678(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 4, 5, 22, 23, 24, 15, 0, 1, 2, 3, 20, 3, 20, 21, 6, 7, 8, 63, 48, 49, 50, 51, 36, 51, 36, 37, 54, 55, 56, 47, 32, 33, 34, 35, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3679(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 4, 5, 22, 23, 24, 31, 0, 1, 2, 3, 20, 3, 20, 21, 6, 7, 8, 47, 48, 49, 50, 51, 36, 51, 36, 37, 54, 55, 56, 63, 32, 33, 34, 35, 52, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3680(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 4, 5, 22, 23, 24, 15, 16, 17, 18, 19, 4, 3, 20, 21, 6, 7, 8, 63, 32, 33, 34, 35, 52, 51, 36, 37, 54, 55, 56, 47, 48, 49, 50, 51, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3681(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 4, 5, 22, 23, 24, 31, 16, 17, 18, 19, 4, 3, 20, 21, 6, 7, 8, 47, 32, 33, 34, 35, 52, 51, 36, 37, 54, 55, 56, 63, 48, 49, 50, 51, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3682(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 4, 5, 22, 23, 24, 15, 0, 17, 18, 19, 4, 3, 20, 21, 6, 7, 8, 63, 48, 33, 34, 35, 52, 51, 36, 37, 54, 55, 56, 47, 32, 49, 50, 51, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3683(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 4, 5, 22, 23, 24, 31, 0, 17, 18, 19, 4, 3, 20, 21, 6, 7, 8, 47, 48, 33, 34, 35, 52, 51, 36, 37, 54, 55, 56, 63, 32, 49, 50, 51, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3684(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 4, 5, 22, 23, 24, 15, 16, 1, 18, 19, 4, 3, 20, 21, 6, 7, 8, 63, 32, 49, 34, 35, 52, 51, 36, 37, 54, 55, 56, 47, 48, 33, 50, 51, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3685(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 4, 5, 22, 23, 24, 31, 16, 1, 18, 19, 4, 3, 20, 21, 6, 7, 8, 47, 32, 49, 34, 35, 52, 51, 36, 37, 54, 55, 56, 63, 48, 33, 50, 51, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3686(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 4, 5, 22, 23, 24, 15, 0, 1, 18, 19, 4, 3, 20, 21, 6, 7, 8, 63, 48, 49, 34, 35, 52, 51, 36, 37, 54, 55, 56, 47, 32, 33, 50, 51, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3687(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 4, 5, 22, 23, 24, 31, 0, 1, 18, 19, 4, 3, 20, 21, 6, 7, 8, 47, 48, 49, 34, 35, 52, 51, 36, 37, 54, 55, 56, 63, 32, 33, 50, 51, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3688(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 4, 5, 22, 23, 24, 15, 16, 17, 2, 19, 4, 3, 20, 21, 6, 7, 8, 63, 32, 33, 50, 35, 52, 51, 36, 37, 54, 55, 56, 47, 48, 49, 34, 51, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3689(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 4, 5, 22, 23, 24, 31, 16, 17, 2, 19, 4, 3, 20, 21, 6, 7, 8, 47, 32, 33, 50, 35, 52, 51, 36, 37, 54, 55, 56, 63, 48, 49, 34, 51, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3690(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 4, 5, 22, 23, 24, 15, 0, 17, 2, 19, 4, 3, 20, 21, 6, 7, 8, 63, 48, 33, 50, 35, 52, 51, 36, 37, 54, 55, 56, 47, 32, 49, 34, 51, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3691(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 4, 5, 22, 23, 24, 31, 0, 17, 2, 19, 4, 3, 20, 21, 6, 7, 8, 47, 48, 33, 50, 35, 52, 51, 36, 37, 54, 55, 56, 63, 32, 49, 34, 51, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3692(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 4, 5, 22, 23, 24, 15, 16, 1, 2, 19, 4, 3, 20, 21, 6, 7, 8, 63, 32, 49, 50, 35, 52, 51, 36, 37, 54, 55, 56, 47, 48, 33, 34, 51, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3693(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 4, 5, 22, 23, 24, 31, 16, 1, 2, 19, 4, 3, 20, 21, 6, 7, 8, 47, 32, 49, 50, 35, 52, 51, 36, 37, 54, 55, 56, 63, 48, 33, 34, 51, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3694(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 4, 5, 22, 23, 24, 15, 0, 1, 2, 19, 4, 3, 20, 21, 6, 7, 8, 63, 48, 49, 50, 35, 52, 51, 36, 37, 54, 55, 56, 47, 32, 33, 34, 51, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3695(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 4, 5, 22, 23, 24, 31, 0, 1, 2, 19, 4, 3, 20, 21, 6, 7, 8, 47, 48, 49, 50, 35, 52, 51, 36, 37, 54, 55, 56, 63, 32, 33, 34, 51, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3696(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 4, 5, 22, 23, 24, 15, 16, 17, 18, 3, 4, 3, 20, 21, 6, 7, 8, 63, 32, 33, 34, 51, 52, 51, 36, 37, 54, 55, 56, 47, 48, 49, 50, 35, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3697(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 4, 5, 22, 23, 24, 31, 16, 17, 18, 3, 4, 3, 20, 21, 6, 7, 8, 47, 32, 33, 34, 51, 52, 51, 36, 37, 54, 55, 56, 63, 48, 49, 50, 35, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3698(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 4, 5, 22, 23, 24, 15, 0, 17, 18, 3, 4, 3, 20, 21, 6, 7, 8, 63, 48, 33, 34, 51, 52, 51, 36, 37, 54, 55, 56, 47, 32, 49, 50, 35, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3699(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 4, 5, 22, 23, 24, 31, 0, 17, 18, 3, 4, 3, 20, 21, 6, 7, 8, 47, 48, 33, 34, 51, 52, 51, 36, 37, 54, 55, 56, 63, 32, 49, 50, 35, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3700(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 4, 5, 22, 23, 24, 15, 16, 1, 18, 3, 4, 3, 20, 21, 6, 7, 8, 63, 32, 49, 34, 51, 52, 51, 36, 37, 54, 55, 56, 47, 48, 33, 50, 35, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3701(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 4, 5, 22, 23, 24, 31, 16, 1, 18, 3, 4, 3, 20, 21, 6, 7, 8, 47, 32, 49, 34, 51, 52, 51, 36, 37, 54, 55, 56, 63, 48, 33, 50, 35, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3702(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 4, 5, 22, 23, 24, 15, 0, 1, 18, 3, 4, 3, 20, 21, 6, 7, 8, 63, 48, 49, 34, 51, 52, 51, 36, 37, 54, 55, 56, 47, 32, 33, 50, 35, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3703(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 4, 5, 22, 23, 24, 31, 0, 1, 18, 3, 4, 3, 20, 21, 6, 7, 8, 47, 48, 49, 34, 51, 52, 51, 36, 37, 54, 55, 56, 63, 32, 33, 50, 35, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3704(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 4, 5, 22, 23, 24, 15, 16, 17, 2, 3, 4, 3, 20, 21, 6, 7, 8, 63, 32, 33, 50, 51, 52, 51, 36, 37, 54, 55, 56, 47, 48, 49, 34, 35, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3705(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 4, 5, 22, 23, 24, 31, 16, 17, 2, 3, 4, 3, 20, 21, 6, 7, 8, 47, 32, 33, 50, 51, 52, 51, 36, 37, 54, 55, 56, 63, 48, 49, 34, 35, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3706(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 4, 5, 22, 23, 24, 15, 0, 17, 2, 3, 4, 3, 20, 21, 6, 7, 8, 63, 48, 33, 50, 51, 52, 51, 36, 37, 54, 55, 56, 47, 32, 49, 34, 35, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3707(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 4, 5, 22, 23, 24, 31, 0, 17, 2, 3, 4, 3, 20, 21, 6, 7, 8, 47, 48, 33, 50, 51, 52, 51, 36, 37, 54, 55, 56, 63, 32, 49, 34, 35, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3708(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 4, 5, 22, 23, 24, 15, 16, 1, 2, 3, 4, 3, 20, 21, 6, 7, 8, 63, 32, 49, 50, 51, 52, 51, 36, 37, 54, 55, 56, 47, 48, 33, 34, 35, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3709(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 4, 5, 22, 23, 24, 31, 16, 1, 2, 3, 4, 3, 20, 21, 6, 7, 8, 47, 32, 49, 50, 51, 52, 51, 36, 37, 54, 55, 56, 63, 48, 33, 34, 35, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3710(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 4, 5, 22, 23, 24, 15, 0, 1, 2, 3, 4, 3, 20, 21, 6, 7, 8, 63, 48, 49, 50, 51, 52, 51, 36, 37, 54, 55, 56, 47, 32, 33, 34, 35, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3711(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 4, 5, 22, 23, 24, 31, 0, 1, 2, 3, 4, 3, 20, 21, 6, 7, 8, 47, 48, 49, 50, 51, 52, 51, 36, 37, 54, 55, 56, 63, 32, 33, 34, 35, 36, 35, 52, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

#endif // DESGPU_COMPILE_ALL_SALTS


