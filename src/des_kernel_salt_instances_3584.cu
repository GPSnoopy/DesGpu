#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt3584(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 4, 5, 22, 23, 24, 15, 16, 17, 18, 19, 20, 19, 20, 21, 6, 7, 8, 63, 32, 33, 34, 35, 36, 35, 36, 37, 54, 55, 56, 47, 48, 49, 50, 51, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3585(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 4, 5, 22, 23, 24, 31, 16, 17, 18, 19, 20, 19, 20, 21, 6, 7, 8, 47, 32, 33, 34, 35, 36, 35, 36, 37, 54, 55, 56, 63, 48, 49, 50, 51, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3586(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 4, 5, 22, 23, 24, 15, 0, 17, 18, 19, 20, 19, 20, 21, 6, 7, 8, 63, 48, 33, 34, 35, 36, 35, 36, 37, 54, 55, 56, 47, 32, 49, 50, 51, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3587(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 4, 5, 22, 23, 24, 31, 0, 17, 18, 19, 20, 19, 20, 21, 6, 7, 8, 47, 48, 33, 34, 35, 36, 35, 36, 37, 54, 55, 56, 63, 32, 49, 50, 51, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3588(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 4, 5, 22, 23, 24, 15, 16, 1, 18, 19, 20, 19, 20, 21, 6, 7, 8, 63, 32, 49, 34, 35, 36, 35, 36, 37, 54, 55, 56, 47, 48, 33, 50, 51, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3589(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 4, 5, 22, 23, 24, 31, 16, 1, 18, 19, 20, 19, 20, 21, 6, 7, 8, 47, 32, 49, 34, 35, 36, 35, 36, 37, 54, 55, 56, 63, 48, 33, 50, 51, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3590(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 4, 5, 22, 23, 24, 15, 0, 1, 18, 19, 20, 19, 20, 21, 6, 7, 8, 63, 48, 49, 34, 35, 36, 35, 36, 37, 54, 55, 56, 47, 32, 33, 50, 51, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3591(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 4, 5, 22, 23, 24, 31, 0, 1, 18, 19, 20, 19, 20, 21, 6, 7, 8, 47, 48, 49, 34, 35, 36, 35, 36, 37, 54, 55, 56, 63, 32, 33, 50, 51, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3592(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 4, 5, 22, 23, 24, 15, 16, 17, 2, 19, 20, 19, 20, 21, 6, 7, 8, 63, 32, 33, 50, 35, 36, 35, 36, 37, 54, 55, 56, 47, 48, 49, 34, 51, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3593(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 4, 5, 22, 23, 24, 31, 16, 17, 2, 19, 20, 19, 20, 21, 6, 7, 8, 47, 32, 33, 50, 35, 36, 35, 36, 37, 54, 55, 56, 63, 48, 49, 34, 51, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3594(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 4, 5, 22, 23, 24, 15, 0, 17, 2, 19, 20, 19, 20, 21, 6, 7, 8, 63, 48, 33, 50, 35, 36, 35, 36, 37, 54, 55, 56, 47, 32, 49, 34, 51, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3595(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 4, 5, 22, 23, 24, 31, 0, 17, 2, 19, 20, 19, 20, 21, 6, 7, 8, 47, 48, 33, 50, 35, 36, 35, 36, 37, 54, 55, 56, 63, 32, 49, 34, 51, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3596(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 4, 5, 22, 23, 24, 15, 16, 1, 2, 19, 20, 19, 20, 21, 6, 7, 8, 63, 32, 49, 50, 35, 36, 35, 36, 37, 54, 55, 56, 47, 48, 33, 34, 51, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3597(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 4, 5, 22, 23, 24, 31, 16, 1, 2, 19, 20, 19, 20, 21, 6, 7, 8, 47, 32, 49, 50, 35, 36, 35, 36, 37, 54, 55, 56, 63, 48, 33, 34, 51, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3598(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 4, 5, 22, 23, 24, 15, 0, 1, 2, 19, 20, 19, 20, 21, 6, 7, 8, 63, 48, 49, 50, 35, 36, 35, 36, 37, 54, 55, 56, 47, 32, 33, 34, 51, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3599(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 4, 5, 22, 23, 24, 31, 0, 1, 2, 19, 20, 19, 20, 21, 6, 7, 8, 47, 48, 49, 50, 35, 36, 35, 36, 37, 54, 55, 56, 63, 32, 33, 34, 51, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3600(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 4, 5, 22, 23, 24, 15, 16, 17, 18, 3, 20, 19, 20, 21, 6, 7, 8, 63, 32, 33, 34, 51, 36, 35, 36, 37, 54, 55, 56, 47, 48, 49, 50, 35, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3601(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 4, 5, 22, 23, 24, 31, 16, 17, 18, 3, 20, 19, 20, 21, 6, 7, 8, 47, 32, 33, 34, 51, 36, 35, 36, 37, 54, 55, 56, 63, 48, 49, 50, 35, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3602(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 4, 5, 22, 23, 24, 15, 0, 17, 18, 3, 20, 19, 20, 21, 6, 7, 8, 63, 48, 33, 34, 51, 36, 35, 36, 37, 54, 55, 56, 47, 32, 49, 50, 35, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3603(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 4, 5, 22, 23, 24, 31, 0, 17, 18, 3, 20, 19, 20, 21, 6, 7, 8, 47, 48, 33, 34, 51, 36, 35, 36, 37, 54, 55, 56, 63, 32, 49, 50, 35, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3604(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 4, 5, 22, 23, 24, 15, 16, 1, 18, 3, 20, 19, 20, 21, 6, 7, 8, 63, 32, 49, 34, 51, 36, 35, 36, 37, 54, 55, 56, 47, 48, 33, 50, 35, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3605(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 4, 5, 22, 23, 24, 31, 16, 1, 18, 3, 20, 19, 20, 21, 6, 7, 8, 47, 32, 49, 34, 51, 36, 35, 36, 37, 54, 55, 56, 63, 48, 33, 50, 35, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3606(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 4, 5, 22, 23, 24, 15, 0, 1, 18, 3, 20, 19, 20, 21, 6, 7, 8, 63, 48, 49, 34, 51, 36, 35, 36, 37, 54, 55, 56, 47, 32, 33, 50, 35, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3607(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 4, 5, 22, 23, 24, 31, 0, 1, 18, 3, 20, 19, 20, 21, 6, 7, 8, 47, 48, 49, 34, 51, 36, 35, 36, 37, 54, 55, 56, 63, 32, 33, 50, 35, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3608(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 4, 5, 22, 23, 24, 15, 16, 17, 2, 3, 20, 19, 20, 21, 6, 7, 8, 63, 32, 33, 50, 51, 36, 35, 36, 37, 54, 55, 56, 47, 48, 49, 34, 35, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3609(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 4, 5, 22, 23, 24, 31, 16, 17, 2, 3, 20, 19, 20, 21, 6, 7, 8, 47, 32, 33, 50, 51, 36, 35, 36, 37, 54, 55, 56, 63, 48, 49, 34, 35, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3610(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 4, 5, 22, 23, 24, 15, 0, 17, 2, 3, 20, 19, 20, 21, 6, 7, 8, 63, 48, 33, 50, 51, 36, 35, 36, 37, 54, 55, 56, 47, 32, 49, 34, 35, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3611(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 4, 5, 22, 23, 24, 31, 0, 17, 2, 3, 20, 19, 20, 21, 6, 7, 8, 47, 48, 33, 50, 51, 36, 35, 36, 37, 54, 55, 56, 63, 32, 49, 34, 35, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3612(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 4, 5, 22, 23, 24, 15, 16, 1, 2, 3, 20, 19, 20, 21, 6, 7, 8, 63, 32, 49, 50, 51, 36, 35, 36, 37, 54, 55, 56, 47, 48, 33, 34, 35, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3613(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 4, 5, 22, 23, 24, 31, 16, 1, 2, 3, 20, 19, 20, 21, 6, 7, 8, 47, 32, 49, 50, 51, 36, 35, 36, 37, 54, 55, 56, 63, 48, 33, 34, 35, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3614(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 4, 5, 22, 23, 24, 15, 0, 1, 2, 3, 20, 19, 20, 21, 6, 7, 8, 63, 48, 49, 50, 51, 36, 35, 36, 37, 54, 55, 56, 47, 32, 33, 34, 35, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3615(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 4, 5, 22, 23, 24, 31, 0, 1, 2, 3, 20, 19, 20, 21, 6, 7, 8, 47, 48, 49, 50, 51, 36, 35, 36, 37, 54, 55, 56, 63, 32, 33, 34, 35, 52, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3616(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 4, 5, 22, 23, 24, 15, 16, 17, 18, 19, 4, 19, 20, 21, 6, 7, 8, 63, 32, 33, 34, 35, 52, 35, 36, 37, 54, 55, 56, 47, 48, 49, 50, 51, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3617(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 4, 5, 22, 23, 24, 31, 16, 17, 18, 19, 4, 19, 20, 21, 6, 7, 8, 47, 32, 33, 34, 35, 52, 35, 36, 37, 54, 55, 56, 63, 48, 49, 50, 51, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3618(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 4, 5, 22, 23, 24, 15, 0, 17, 18, 19, 4, 19, 20, 21, 6, 7, 8, 63, 48, 33, 34, 35, 52, 35, 36, 37, 54, 55, 56, 47, 32, 49, 50, 51, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3619(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 4, 5, 22, 23, 24, 31, 0, 17, 18, 19, 4, 19, 20, 21, 6, 7, 8, 47, 48, 33, 34, 35, 52, 35, 36, 37, 54, 55, 56, 63, 32, 49, 50, 51, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3620(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 4, 5, 22, 23, 24, 15, 16, 1, 18, 19, 4, 19, 20, 21, 6, 7, 8, 63, 32, 49, 34, 35, 52, 35, 36, 37, 54, 55, 56, 47, 48, 33, 50, 51, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3621(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 4, 5, 22, 23, 24, 31, 16, 1, 18, 19, 4, 19, 20, 21, 6, 7, 8, 47, 32, 49, 34, 35, 52, 35, 36, 37, 54, 55, 56, 63, 48, 33, 50, 51, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3622(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 4, 5, 22, 23, 24, 15, 0, 1, 18, 19, 4, 19, 20, 21, 6, 7, 8, 63, 48, 49, 34, 35, 52, 35, 36, 37, 54, 55, 56, 47, 32, 33, 50, 51, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3623(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 4, 5, 22, 23, 24, 31, 0, 1, 18, 19, 4, 19, 20, 21, 6, 7, 8, 47, 48, 49, 34, 35, 52, 35, 36, 37, 54, 55, 56, 63, 32, 33, 50, 51, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3624(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 4, 5, 22, 23, 24, 15, 16, 17, 2, 19, 4, 19, 20, 21, 6, 7, 8, 63, 32, 33, 50, 35, 52, 35, 36, 37, 54, 55, 56, 47, 48, 49, 34, 51, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3625(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 4, 5, 22, 23, 24, 31, 16, 17, 2, 19, 4, 19, 20, 21, 6, 7, 8, 47, 32, 33, 50, 35, 52, 35, 36, 37, 54, 55, 56, 63, 48, 49, 34, 51, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3626(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 4, 5, 22, 23, 24, 15, 0, 17, 2, 19, 4, 19, 20, 21, 6, 7, 8, 63, 48, 33, 50, 35, 52, 35, 36, 37, 54, 55, 56, 47, 32, 49, 34, 51, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3627(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 4, 5, 22, 23, 24, 31, 0, 17, 2, 19, 4, 19, 20, 21, 6, 7, 8, 47, 48, 33, 50, 35, 52, 35, 36, 37, 54, 55, 56, 63, 32, 49, 34, 51, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3628(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 4, 5, 22, 23, 24, 15, 16, 1, 2, 19, 4, 19, 20, 21, 6, 7, 8, 63, 32, 49, 50, 35, 52, 35, 36, 37, 54, 55, 56, 47, 48, 33, 34, 51, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3629(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 4, 5, 22, 23, 24, 31, 16, 1, 2, 19, 4, 19, 20, 21, 6, 7, 8, 47, 32, 49, 50, 35, 52, 35, 36, 37, 54, 55, 56, 63, 48, 33, 34, 51, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3630(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 4, 5, 22, 23, 24, 15, 0, 1, 2, 19, 4, 19, 20, 21, 6, 7, 8, 63, 48, 49, 50, 35, 52, 35, 36, 37, 54, 55, 56, 47, 32, 33, 34, 51, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3631(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 4, 5, 22, 23, 24, 31, 0, 1, 2, 19, 4, 19, 20, 21, 6, 7, 8, 47, 48, 49, 50, 35, 52, 35, 36, 37, 54, 55, 56, 63, 32, 33, 34, 51, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3632(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 4, 5, 22, 23, 24, 15, 16, 17, 18, 3, 4, 19, 20, 21, 6, 7, 8, 63, 32, 33, 34, 51, 52, 35, 36, 37, 54, 55, 56, 47, 48, 49, 50, 35, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3633(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 4, 5, 22, 23, 24, 31, 16, 17, 18, 3, 4, 19, 20, 21, 6, 7, 8, 47, 32, 33, 34, 51, 52, 35, 36, 37, 54, 55, 56, 63, 48, 49, 50, 35, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3634(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 4, 5, 22, 23, 24, 15, 0, 17, 18, 3, 4, 19, 20, 21, 6, 7, 8, 63, 48, 33, 34, 51, 52, 35, 36, 37, 54, 55, 56, 47, 32, 49, 50, 35, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3635(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 4, 5, 22, 23, 24, 31, 0, 17, 18, 3, 4, 19, 20, 21, 6, 7, 8, 47, 48, 33, 34, 51, 52, 35, 36, 37, 54, 55, 56, 63, 32, 49, 50, 35, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3636(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 4, 5, 22, 23, 24, 15, 16, 1, 18, 3, 4, 19, 20, 21, 6, 7, 8, 63, 32, 49, 34, 51, 52, 35, 36, 37, 54, 55, 56, 47, 48, 33, 50, 35, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3637(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 4, 5, 22, 23, 24, 31, 16, 1, 18, 3, 4, 19, 20, 21, 6, 7, 8, 47, 32, 49, 34, 51, 52, 35, 36, 37, 54, 55, 56, 63, 48, 33, 50, 35, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3638(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 4, 5, 22, 23, 24, 15, 0, 1, 18, 3, 4, 19, 20, 21, 6, 7, 8, 63, 48, 49, 34, 51, 52, 35, 36, 37, 54, 55, 56, 47, 32, 33, 50, 35, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3639(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 4, 5, 22, 23, 24, 31, 0, 1, 18, 3, 4, 19, 20, 21, 6, 7, 8, 47, 48, 49, 34, 51, 52, 35, 36, 37, 54, 55, 56, 63, 32, 33, 50, 35, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3640(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 4, 5, 22, 23, 24, 15, 16, 17, 2, 3, 4, 19, 20, 21, 6, 7, 8, 63, 32, 33, 50, 51, 52, 35, 36, 37, 54, 55, 56, 47, 48, 49, 34, 35, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3641(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 4, 5, 22, 23, 24, 31, 16, 17, 2, 3, 4, 19, 20, 21, 6, 7, 8, 47, 32, 33, 50, 51, 52, 35, 36, 37, 54, 55, 56, 63, 48, 49, 34, 35, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3642(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 4, 5, 22, 23, 24, 15, 0, 17, 2, 3, 4, 19, 20, 21, 6, 7, 8, 63, 48, 33, 50, 51, 52, 35, 36, 37, 54, 55, 56, 47, 32, 49, 34, 35, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3643(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 4, 5, 22, 23, 24, 31, 0, 17, 2, 3, 4, 19, 20, 21, 6, 7, 8, 47, 48, 33, 50, 51, 52, 35, 36, 37, 54, 55, 56, 63, 32, 49, 34, 35, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3644(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 4, 5, 22, 23, 24, 15, 16, 1, 2, 3, 4, 19, 20, 21, 6, 7, 8, 63, 32, 49, 50, 51, 52, 35, 36, 37, 54, 55, 56, 47, 48, 33, 34, 35, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3645(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 4, 5, 22, 23, 24, 31, 16, 1, 2, 3, 4, 19, 20, 21, 6, 7, 8, 47, 32, 49, 50, 51, 52, 35, 36, 37, 54, 55, 56, 63, 48, 33, 34, 35, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3646(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 4, 5, 22, 23, 24, 15, 0, 1, 2, 3, 4, 19, 20, 21, 6, 7, 8, 63, 48, 49, 50, 51, 52, 35, 36, 37, 54, 55, 56, 47, 32, 33, 34, 35, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3647(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 4, 5, 22, 23, 24, 31, 0, 1, 2, 3, 4, 19, 20, 21, 6, 7, 8, 47, 48, 49, 50, 51, 52, 35, 36, 37, 54, 55, 56, 63, 32, 33, 34, 35, 36, 51, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3648(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 4, 5, 22, 23, 24, 15, 16, 17, 18, 19, 20, 3, 20, 21, 6, 7, 8, 63, 32, 33, 34, 35, 36, 51, 36, 37, 54, 55, 56, 47, 48, 49, 50, 51, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3649(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 4, 5, 22, 23, 24, 31, 16, 17, 18, 19, 20, 3, 20, 21, 6, 7, 8, 47, 32, 33, 34, 35, 36, 51, 36, 37, 54, 55, 56, 63, 48, 49, 50, 51, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3650(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 4, 5, 22, 23, 24, 15, 0, 17, 18, 19, 20, 3, 20, 21, 6, 7, 8, 63, 48, 33, 34, 35, 36, 51, 36, 37, 54, 55, 56, 47, 32, 49, 50, 51, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3651(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 4, 5, 22, 23, 24, 31, 0, 17, 18, 19, 20, 3, 20, 21, 6, 7, 8, 47, 48, 33, 34, 35, 36, 51, 36, 37, 54, 55, 56, 63, 32, 49, 50, 51, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3652(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 4, 5, 22, 23, 24, 15, 16, 1, 18, 19, 20, 3, 20, 21, 6, 7, 8, 63, 32, 49, 34, 35, 36, 51, 36, 37, 54, 55, 56, 47, 48, 33, 50, 51, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3653(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 4, 5, 22, 23, 24, 31, 16, 1, 18, 19, 20, 3, 20, 21, 6, 7, 8, 47, 32, 49, 34, 35, 36, 51, 36, 37, 54, 55, 56, 63, 48, 33, 50, 51, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3654(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 4, 5, 22, 23, 24, 15, 0, 1, 18, 19, 20, 3, 20, 21, 6, 7, 8, 63, 48, 49, 34, 35, 36, 51, 36, 37, 54, 55, 56, 47, 32, 33, 50, 51, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3655(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 4, 5, 22, 23, 24, 31, 0, 1, 18, 19, 20, 3, 20, 21, 6, 7, 8, 47, 48, 49, 34, 35, 36, 51, 36, 37, 54, 55, 56, 63, 32, 33, 50, 51, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3656(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 4, 5, 22, 23, 24, 15, 16, 17, 2, 19, 20, 3, 20, 21, 6, 7, 8, 63, 32, 33, 50, 35, 36, 51, 36, 37, 54, 55, 56, 47, 48, 49, 34, 51, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3657(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 4, 5, 22, 23, 24, 31, 16, 17, 2, 19, 20, 3, 20, 21, 6, 7, 8, 47, 32, 33, 50, 35, 36, 51, 36, 37, 54, 55, 56, 63, 48, 49, 34, 51, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3658(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 4, 5, 22, 23, 24, 15, 0, 17, 2, 19, 20, 3, 20, 21, 6, 7, 8, 63, 48, 33, 50, 35, 36, 51, 36, 37, 54, 55, 56, 47, 32, 49, 34, 51, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3659(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 4, 5, 22, 23, 24, 31, 0, 17, 2, 19, 20, 3, 20, 21, 6, 7, 8, 47, 48, 33, 50, 35, 36, 51, 36, 37, 54, 55, 56, 63, 32, 49, 34, 51, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3660(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 4, 5, 22, 23, 24, 15, 16, 1, 2, 19, 20, 3, 20, 21, 6, 7, 8, 63, 32, 49, 50, 35, 36, 51, 36, 37, 54, 55, 56, 47, 48, 33, 34, 51, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3661(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 4, 5, 22, 23, 24, 31, 16, 1, 2, 19, 20, 3, 20, 21, 6, 7, 8, 47, 32, 49, 50, 35, 36, 51, 36, 37, 54, 55, 56, 63, 48, 33, 34, 51, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3662(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 4, 5, 22, 23, 24, 15, 0, 1, 2, 19, 20, 3, 20, 21, 6, 7, 8, 63, 48, 49, 50, 35, 36, 51, 36, 37, 54, 55, 56, 47, 32, 33, 34, 51, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3663(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 4, 5, 22, 23, 24, 31, 0, 1, 2, 19, 20, 3, 20, 21, 6, 7, 8, 47, 48, 49, 50, 35, 36, 51, 36, 37, 54, 55, 56, 63, 32, 33, 34, 51, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3664(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 4, 5, 22, 23, 24, 15, 16, 17, 18, 3, 20, 3, 20, 21, 6, 7, 8, 63, 32, 33, 34, 51, 36, 51, 36, 37, 54, 55, 56, 47, 48, 49, 50, 35, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3665(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 4, 5, 22, 23, 24, 31, 16, 17, 18, 3, 20, 3, 20, 21, 6, 7, 8, 47, 32, 33, 34, 51, 36, 51, 36, 37, 54, 55, 56, 63, 48, 49, 50, 35, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3666(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 4, 5, 22, 23, 24, 15, 0, 17, 18, 3, 20, 3, 20, 21, 6, 7, 8, 63, 48, 33, 34, 51, 36, 51, 36, 37, 54, 55, 56, 47, 32, 49, 50, 35, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3667(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 4, 5, 22, 23, 24, 31, 0, 17, 18, 3, 20, 3, 20, 21, 6, 7, 8, 47, 48, 33, 34, 51, 36, 51, 36, 37, 54, 55, 56, 63, 32, 49, 50, 35, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3668(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 4, 5, 22, 23, 24, 15, 16, 1, 18, 3, 20, 3, 20, 21, 6, 7, 8, 63, 32, 49, 34, 51, 36, 51, 36, 37, 54, 55, 56, 47, 48, 33, 50, 35, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3669(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 4, 5, 22, 23, 24, 31, 16, 1, 18, 3, 20, 3, 20, 21, 6, 7, 8, 47, 32, 49, 34, 51, 36, 51, 36, 37, 54, 55, 56, 63, 48, 33, 50, 35, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3670(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 4, 5, 22, 23, 24, 15, 0, 1, 18, 3, 20, 3, 20, 21, 6, 7, 8, 63, 48, 49, 34, 51, 36, 51, 36, 37, 54, 55, 56, 47, 32, 33, 50, 35, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3671(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 4, 5, 22, 23, 24, 31, 0, 1, 18, 3, 20, 3, 20, 21, 6, 7, 8, 47, 48, 49, 34, 51, 36, 51, 36, 37, 54, 55, 56, 63, 32, 33, 50, 35, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3672(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 4, 5, 22, 23, 24, 15, 16, 17, 2, 3, 20, 3, 20, 21, 6, 7, 8, 63, 32, 33, 50, 51, 36, 51, 36, 37, 54, 55, 56, 47, 48, 49, 34, 35, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3673(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 4, 5, 22, 23, 24, 31, 16, 17, 2, 3, 20, 3, 20, 21, 6, 7, 8, 47, 32, 33, 50, 51, 36, 51, 36, 37, 54, 55, 56, 63, 48, 49, 34, 35, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3674(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 4, 5, 22, 23, 24, 15, 0, 17, 2, 3, 20, 3, 20, 21, 6, 7, 8, 63, 48, 33, 50, 51, 36, 51, 36, 37, 54, 55, 56, 47, 32, 49, 34, 35, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3675(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 4, 5, 22, 23, 24, 31, 0, 17, 2, 3, 20, 3, 20, 21, 6, 7, 8, 47, 48, 33, 50, 51, 36, 51, 36, 37, 54, 55, 56, 63, 32, 49, 34, 35, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3676(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 4, 5, 22, 23, 24, 15, 16, 1, 2, 3, 20, 3, 20, 21, 6, 7, 8, 63, 32, 49, 50, 51, 36, 51, 36, 37, 54, 55, 56, 47, 48, 33, 34, 35, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3677(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 4, 5, 22, 23, 24, 31, 16, 1, 2, 3, 20, 3, 20, 21, 6, 7, 8, 47, 32, 49, 50, 51, 36, 51, 36, 37, 54, 55, 56, 63, 48, 33, 34, 35, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3678(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 4, 5, 22, 23, 24, 15, 0, 1, 2, 3, 20, 3, 20, 21, 6, 7, 8, 63, 48, 49, 50, 51, 36, 51, 36, 37, 54, 55, 56, 47, 32, 33, 34, 35, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3679(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 4, 5, 22, 23, 24, 31, 0, 1, 2, 3, 20, 3, 20, 21, 6, 7, 8, 47, 48, 49, 50, 51, 36, 51, 36, 37, 54, 55, 56, 63, 32, 33, 34, 35, 52, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3680(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 4, 5, 22, 23, 24, 15, 16, 17, 18, 19, 4, 3, 20, 21, 6, 7, 8, 63, 32, 33, 34, 35, 52, 51, 36, 37, 54, 55, 56, 47, 48, 49, 50, 51, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3681(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 4, 5, 22, 23, 24, 31, 16, 17, 18, 19, 4, 3, 20, 21, 6, 7, 8, 47, 32, 33, 34, 35, 52, 51, 36, 37, 54, 55, 56, 63, 48, 49, 50, 51, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3682(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 4, 5, 22, 23, 24, 15, 0, 17, 18, 19, 4, 3, 20, 21, 6, 7, 8, 63, 48, 33, 34, 35, 52, 51, 36, 37, 54, 55, 56, 47, 32, 49, 50, 51, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3683(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 4, 5, 22, 23, 24, 31, 0, 17, 18, 19, 4, 3, 20, 21, 6, 7, 8, 47, 48, 33, 34, 35, 52, 51, 36, 37, 54, 55, 56, 63, 32, 49, 50, 51, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3684(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 4, 5, 22, 23, 24, 15, 16, 1, 18, 19, 4, 3, 20, 21, 6, 7, 8, 63, 32, 49, 34, 35, 52, 51, 36, 37, 54, 55, 56, 47, 48, 33, 50, 51, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3685(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 4, 5, 22, 23, 24, 31, 16, 1, 18, 19, 4, 3, 20, 21, 6, 7, 8, 47, 32, 49, 34, 35, 52, 51, 36, 37, 54, 55, 56, 63, 48, 33, 50, 51, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3686(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 4, 5, 22, 23, 24, 15, 0, 1, 18, 19, 4, 3, 20, 21, 6, 7, 8, 63, 48, 49, 34, 35, 52, 51, 36, 37, 54, 55, 56, 47, 32, 33, 50, 51, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3687(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 4, 5, 22, 23, 24, 31, 0, 1, 18, 19, 4, 3, 20, 21, 6, 7, 8, 47, 48, 49, 34, 35, 52, 51, 36, 37, 54, 55, 56, 63, 32, 33, 50, 51, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3688(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 4, 5, 22, 23, 24, 15, 16, 17, 2, 19, 4, 3, 20, 21, 6, 7, 8, 63, 32, 33, 50, 35, 52, 51, 36, 37, 54, 55, 56, 47, 48, 49, 34, 51, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3689(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 4, 5, 22, 23, 24, 31, 16, 17, 2, 19, 4, 3, 20, 21, 6, 7, 8, 47, 32, 33, 50, 35, 52, 51, 36, 37, 54, 55, 56, 63, 48, 49, 34, 51, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3690(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 4, 5, 22, 23, 24, 15, 0, 17, 2, 19, 4, 3, 20, 21, 6, 7, 8, 63, 48, 33, 50, 35, 52, 51, 36, 37, 54, 55, 56, 47, 32, 49, 34, 51, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3691(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 4, 5, 22, 23, 24, 31, 0, 17, 2, 19, 4, 3, 20, 21, 6, 7, 8, 47, 48, 33, 50, 35, 52, 51, 36, 37, 54, 55, 56, 63, 32, 49, 34, 51, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3692(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 4, 5, 22, 23, 24, 15, 16, 1, 2, 19, 4, 3, 20, 21, 6, 7, 8, 63, 32, 49, 50, 35, 52, 51, 36, 37, 54, 55, 56, 47, 48, 33, 34, 51, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3693(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 4, 5, 22, 23, 24, 31, 16, 1, 2, 19, 4, 3, 20, 21, 6, 7, 8, 47, 32, 49, 50, 35, 52, 51, 36, 37, 54, 55, 56, 63, 48, 33, 34, 51, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3694(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 4, 5, 22, 23, 24, 15, 0, 1, 2, 19, 4, 3, 20, 21, 6, 7, 8, 63, 48, 49, 50, 35, 52, 51, 36, 37, 54, 55, 56, 47, 32, 33, 34, 51, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3695(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 4, 5, 22, 23, 24, 31, 0, 1, 2, 19, 4, 3, 20, 21, 6, 7, 8, 47, 48, 49, 50, 35, 52, 51, 36, 37, 54, 55, 56, 63, 32, 33, 34, 51, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3696(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 4, 5, 22, 23, 24, 15, 16, 17, 18, 3, 4, 3, 20, 21, 6, 7, 8, 63, 32, 33, 34, 51, 52, 51, 36, 37, 54, 55, 56, 47, 48, 49, 50, 35, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3697(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 4, 5, 22, 23, 24, 31, 16, 17, 18, 3, 4, 3, 20, 21, 6, 7, 8, 47, 32, 33, 34, 51, 52, 51, 36, 37, 54, 55, 56, 63, 48, 49, 50, 35, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3698(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 4, 5, 22, 23, 24, 15, 0, 17, 18, 3, 4, 3, 20, 21, 6, 7, 8, 63, 48, 33, 34, 51, 52, 51, 36, 37, 54, 55, 56, 47, 32, 49, 50, 35, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3699(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 4, 5, 22, 23, 24, 31, 0, 17, 18, 3, 4, 3, 20, 21, 6, 7, 8, 47, 48, 33, 34, 51, 52, 51, 36, 37, 54, 55, 56, 63, 32, 49, 50, 35, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3700(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 4, 5, 22, 23, 24, 15, 16, 1, 18, 3, 4, 3, 20, 21, 6, 7, 8, 63, 32, 49, 34, 51, 52, 51, 36, 37, 54, 55, 56, 47, 48, 33, 50, 35, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3701(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 4, 5, 22, 23, 24, 31, 16, 1, 18, 3, 4, 3, 20, 21, 6, 7, 8, 47, 32, 49, 34, 51, 52, 51, 36, 37, 54, 55, 56, 63, 48, 33, 50, 35, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3702(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 4, 5, 22, 23, 24, 15, 0, 1, 18, 3, 4, 3, 20, 21, 6, 7, 8, 63, 48, 49, 34, 51, 52, 51, 36, 37, 54, 55, 56, 47, 32, 33, 50, 35, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3703(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 4, 5, 22, 23, 24, 31, 0, 1, 18, 3, 4, 3, 20, 21, 6, 7, 8, 47, 48, 49, 34, 51, 52, 51, 36, 37, 54, 55, 56, 63, 32, 33, 50, 35, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3704(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 4, 5, 22, 23, 24, 15, 16, 17, 2, 3, 4, 3, 20, 21, 6, 7, 8, 63, 32, 33, 50, 51, 52, 51, 36, 37, 54, 55, 56, 47, 48, 49, 34, 35, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3705(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 4, 5, 22, 23, 24, 31, 16, 17, 2, 3, 4, 3, 20, 21, 6, 7, 8, 47, 32, 33, 50, 51, 52, 51, 36, 37, 54, 55, 56, 63, 48, 49, 34, 35, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3706(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 4, 5, 22, 23, 24, 15, 0, 17, 2, 3, 4, 3, 20, 21, 6, 7, 8, 63, 48, 33, 50, 51, 52, 51, 36, 37, 54, 55, 56, 47, 32, 49, 34, 35, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3707(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 4, 5, 22, 23, 24, 31, 0, 17, 2, 3, 4, 3, 20, 21, 6, 7, 8, 47, 48, 33, 50, 51, 52, 51, 36, 37, 54, 55, 56, 63, 32, 49, 34, 35, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3708(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 4, 5, 22, 23, 24, 15, 16, 1, 2, 3, 4, 3, 20, 21, 6, 7, 8, 63, 32, 49, 50, 51, 52, 51, 36, 37, 54, 55, 56, 47, 48, 33, 34, 35, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3709(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 4, 5, 22, 23, 24, 31, 16, 1, 2, 3, 4, 3, 20, 21, 6, 7, 8, 47, 32, 49, 50, 51, 52, 51, 36, 37, 54, 55, 56, 63, 48, 33, 34, 35, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3710(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 4, 5, 22, 23, 24, 15, 0, 1, 2, 3, 4, 3, 20, 21, 6, 7, 8, 63, 48, 49, 50, 51, 52, 51, 36, 37, 54, 55, 56, 47, 32, 33, 34, 35, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3711(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 4, 5, 22, 23, 24, 31, 0, 1, 2, 3, 4, 3, 20, 21, 6, 7, 8, 47, 48, 49, 50, 51, 52, 51, 36, 37, 54, 55, 56, 63, 32, 33, 34, 35, 36, 35, 52, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}


