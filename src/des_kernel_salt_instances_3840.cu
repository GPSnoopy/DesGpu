#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

#ifdef DESGPU_COMPILE_ALL_SALTS

void des_25_encrypt_salt3840(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 4, 21, 22, 23, 24, 15, 16, 17, 18, 19, 20, 19, 20, 5, 6, 7, 8, 63, 32, 33, 34, 35, 36, 35, 36, 53, 54, 55, 56, 47, 48, 49, 50, 51, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3841(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 4, 21, 22, 23, 24, 31, 16, 17, 18, 19, 20, 19, 20, 5, 6, 7, 8, 47, 32, 33, 34, 35, 36, 35, 36, 53, 54, 55, 56, 63, 48, 49, 50, 51, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3842(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 4, 21, 22, 23, 24, 15, 0, 17, 18, 19, 20, 19, 20, 5, 6, 7, 8, 63, 48, 33, 34, 35, 36, 35, 36, 53, 54, 55, 56, 47, 32, 49, 50, 51, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3843(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 4, 21, 22, 23, 24, 31, 0, 17, 18, 19, 20, 19, 20, 5, 6, 7, 8, 47, 48, 33, 34, 35, 36, 35, 36, 53, 54, 55, 56, 63, 32, 49, 50, 51, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3844(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 4, 21, 22, 23, 24, 15, 16, 1, 18, 19, 20, 19, 20, 5, 6, 7, 8, 63, 32, 49, 34, 35, 36, 35, 36, 53, 54, 55, 56, 47, 48, 33, 50, 51, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3845(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 4, 21, 22, 23, 24, 31, 16, 1, 18, 19, 20, 19, 20, 5, 6, 7, 8, 47, 32, 49, 34, 35, 36, 35, 36, 53, 54, 55, 56, 63, 48, 33, 50, 51, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3846(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 4, 21, 22, 23, 24, 15, 0, 1, 18, 19, 20, 19, 20, 5, 6, 7, 8, 63, 48, 49, 34, 35, 36, 35, 36, 53, 54, 55, 56, 47, 32, 33, 50, 51, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3847(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 4, 21, 22, 23, 24, 31, 0, 1, 18, 19, 20, 19, 20, 5, 6, 7, 8, 47, 48, 49, 34, 35, 36, 35, 36, 53, 54, 55, 56, 63, 32, 33, 50, 51, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3848(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 4, 21, 22, 23, 24, 15, 16, 17, 2, 19, 20, 19, 20, 5, 6, 7, 8, 63, 32, 33, 50, 35, 36, 35, 36, 53, 54, 55, 56, 47, 48, 49, 34, 51, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3849(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 4, 21, 22, 23, 24, 31, 16, 17, 2, 19, 20, 19, 20, 5, 6, 7, 8, 47, 32, 33, 50, 35, 36, 35, 36, 53, 54, 55, 56, 63, 48, 49, 34, 51, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3850(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 4, 21, 22, 23, 24, 15, 0, 17, 2, 19, 20, 19, 20, 5, 6, 7, 8, 63, 48, 33, 50, 35, 36, 35, 36, 53, 54, 55, 56, 47, 32, 49, 34, 51, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3851(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 4, 21, 22, 23, 24, 31, 0, 17, 2, 19, 20, 19, 20, 5, 6, 7, 8, 47, 48, 33, 50, 35, 36, 35, 36, 53, 54, 55, 56, 63, 32, 49, 34, 51, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3852(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 4, 21, 22, 23, 24, 15, 16, 1, 2, 19, 20, 19, 20, 5, 6, 7, 8, 63, 32, 49, 50, 35, 36, 35, 36, 53, 54, 55, 56, 47, 48, 33, 34, 51, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3853(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 4, 21, 22, 23, 24, 31, 16, 1, 2, 19, 20, 19, 20, 5, 6, 7, 8, 47, 32, 49, 50, 35, 36, 35, 36, 53, 54, 55, 56, 63, 48, 33, 34, 51, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3854(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 4, 21, 22, 23, 24, 15, 0, 1, 2, 19, 20, 19, 20, 5, 6, 7, 8, 63, 48, 49, 50, 35, 36, 35, 36, 53, 54, 55, 56, 47, 32, 33, 34, 51, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3855(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 4, 21, 22, 23, 24, 31, 0, 1, 2, 19, 20, 19, 20, 5, 6, 7, 8, 47, 48, 49, 50, 35, 36, 35, 36, 53, 54, 55, 56, 63, 32, 33, 34, 51, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3856(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 4, 21, 22, 23, 24, 15, 16, 17, 18, 3, 20, 19, 20, 5, 6, 7, 8, 63, 32, 33, 34, 51, 36, 35, 36, 53, 54, 55, 56, 47, 48, 49, 50, 35, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3857(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 4, 21, 22, 23, 24, 31, 16, 17, 18, 3, 20, 19, 20, 5, 6, 7, 8, 47, 32, 33, 34, 51, 36, 35, 36, 53, 54, 55, 56, 63, 48, 49, 50, 35, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3858(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 4, 21, 22, 23, 24, 15, 0, 17, 18, 3, 20, 19, 20, 5, 6, 7, 8, 63, 48, 33, 34, 51, 36, 35, 36, 53, 54, 55, 56, 47, 32, 49, 50, 35, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3859(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 4, 21, 22, 23, 24, 31, 0, 17, 18, 3, 20, 19, 20, 5, 6, 7, 8, 47, 48, 33, 34, 51, 36, 35, 36, 53, 54, 55, 56, 63, 32, 49, 50, 35, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3860(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 4, 21, 22, 23, 24, 15, 16, 1, 18, 3, 20, 19, 20, 5, 6, 7, 8, 63, 32, 49, 34, 51, 36, 35, 36, 53, 54, 55, 56, 47, 48, 33, 50, 35, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3861(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 4, 21, 22, 23, 24, 31, 16, 1, 18, 3, 20, 19, 20, 5, 6, 7, 8, 47, 32, 49, 34, 51, 36, 35, 36, 53, 54, 55, 56, 63, 48, 33, 50, 35, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3862(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 4, 21, 22, 23, 24, 15, 0, 1, 18, 3, 20, 19, 20, 5, 6, 7, 8, 63, 48, 49, 34, 51, 36, 35, 36, 53, 54, 55, 56, 47, 32, 33, 50, 35, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3863(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 4, 21, 22, 23, 24, 31, 0, 1, 18, 3, 20, 19, 20, 5, 6, 7, 8, 47, 48, 49, 34, 51, 36, 35, 36, 53, 54, 55, 56, 63, 32, 33, 50, 35, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3864(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 4, 21, 22, 23, 24, 15, 16, 17, 2, 3, 20, 19, 20, 5, 6, 7, 8, 63, 32, 33, 50, 51, 36, 35, 36, 53, 54, 55, 56, 47, 48, 49, 34, 35, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3865(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 4, 21, 22, 23, 24, 31, 16, 17, 2, 3, 20, 19, 20, 5, 6, 7, 8, 47, 32, 33, 50, 51, 36, 35, 36, 53, 54, 55, 56, 63, 48, 49, 34, 35, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3866(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 4, 21, 22, 23, 24, 15, 0, 17, 2, 3, 20, 19, 20, 5, 6, 7, 8, 63, 48, 33, 50, 51, 36, 35, 36, 53, 54, 55, 56, 47, 32, 49, 34, 35, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3867(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 4, 21, 22, 23, 24, 31, 0, 17, 2, 3, 20, 19, 20, 5, 6, 7, 8, 47, 48, 33, 50, 51, 36, 35, 36, 53, 54, 55, 56, 63, 32, 49, 34, 35, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3868(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 4, 21, 22, 23, 24, 15, 16, 1, 2, 3, 20, 19, 20, 5, 6, 7, 8, 63, 32, 49, 50, 51, 36, 35, 36, 53, 54, 55, 56, 47, 48, 33, 34, 35, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3869(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 4, 21, 22, 23, 24, 31, 16, 1, 2, 3, 20, 19, 20, 5, 6, 7, 8, 47, 32, 49, 50, 51, 36, 35, 36, 53, 54, 55, 56, 63, 48, 33, 34, 35, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3870(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 4, 21, 22, 23, 24, 15, 0, 1, 2, 3, 20, 19, 20, 5, 6, 7, 8, 63, 48, 49, 50, 51, 36, 35, 36, 53, 54, 55, 56, 47, 32, 33, 34, 35, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3871(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 4, 21, 22, 23, 24, 31, 0, 1, 2, 3, 20, 19, 20, 5, 6, 7, 8, 47, 48, 49, 50, 51, 36, 35, 36, 53, 54, 55, 56, 63, 32, 33, 34, 35, 52, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3872(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 4, 21, 22, 23, 24, 15, 16, 17, 18, 19, 4, 19, 20, 5, 6, 7, 8, 63, 32, 33, 34, 35, 52, 35, 36, 53, 54, 55, 56, 47, 48, 49, 50, 51, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3873(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 4, 21, 22, 23, 24, 31, 16, 17, 18, 19, 4, 19, 20, 5, 6, 7, 8, 47, 32, 33, 34, 35, 52, 35, 36, 53, 54, 55, 56, 63, 48, 49, 50, 51, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3874(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 4, 21, 22, 23, 24, 15, 0, 17, 18, 19, 4, 19, 20, 5, 6, 7, 8, 63, 48, 33, 34, 35, 52, 35, 36, 53, 54, 55, 56, 47, 32, 49, 50, 51, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3875(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 4, 21, 22, 23, 24, 31, 0, 17, 18, 19, 4, 19, 20, 5, 6, 7, 8, 47, 48, 33, 34, 35, 52, 35, 36, 53, 54, 55, 56, 63, 32, 49, 50, 51, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3876(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 4, 21, 22, 23, 24, 15, 16, 1, 18, 19, 4, 19, 20, 5, 6, 7, 8, 63, 32, 49, 34, 35, 52, 35, 36, 53, 54, 55, 56, 47, 48, 33, 50, 51, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3877(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 4, 21, 22, 23, 24, 31, 16, 1, 18, 19, 4, 19, 20, 5, 6, 7, 8, 47, 32, 49, 34, 35, 52, 35, 36, 53, 54, 55, 56, 63, 48, 33, 50, 51, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3878(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 4, 21, 22, 23, 24, 15, 0, 1, 18, 19, 4, 19, 20, 5, 6, 7, 8, 63, 48, 49, 34, 35, 52, 35, 36, 53, 54, 55, 56, 47, 32, 33, 50, 51, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3879(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 4, 21, 22, 23, 24, 31, 0, 1, 18, 19, 4, 19, 20, 5, 6, 7, 8, 47, 48, 49, 34, 35, 52, 35, 36, 53, 54, 55, 56, 63, 32, 33, 50, 51, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3880(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 4, 21, 22, 23, 24, 15, 16, 17, 2, 19, 4, 19, 20, 5, 6, 7, 8, 63, 32, 33, 50, 35, 52, 35, 36, 53, 54, 55, 56, 47, 48, 49, 34, 51, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3881(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 4, 21, 22, 23, 24, 31, 16, 17, 2, 19, 4, 19, 20, 5, 6, 7, 8, 47, 32, 33, 50, 35, 52, 35, 36, 53, 54, 55, 56, 63, 48, 49, 34, 51, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3882(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 4, 21, 22, 23, 24, 15, 0, 17, 2, 19, 4, 19, 20, 5, 6, 7, 8, 63, 48, 33, 50, 35, 52, 35, 36, 53, 54, 55, 56, 47, 32, 49, 34, 51, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3883(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 4, 21, 22, 23, 24, 31, 0, 17, 2, 19, 4, 19, 20, 5, 6, 7, 8, 47, 48, 33, 50, 35, 52, 35, 36, 53, 54, 55, 56, 63, 32, 49, 34, 51, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3884(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 4, 21, 22, 23, 24, 15, 16, 1, 2, 19, 4, 19, 20, 5, 6, 7, 8, 63, 32, 49, 50, 35, 52, 35, 36, 53, 54, 55, 56, 47, 48, 33, 34, 51, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3885(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 4, 21, 22, 23, 24, 31, 16, 1, 2, 19, 4, 19, 20, 5, 6, 7, 8, 47, 32, 49, 50, 35, 52, 35, 36, 53, 54, 55, 56, 63, 48, 33, 34, 51, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3886(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 4, 21, 22, 23, 24, 15, 0, 1, 2, 19, 4, 19, 20, 5, 6, 7, 8, 63, 48, 49, 50, 35, 52, 35, 36, 53, 54, 55, 56, 47, 32, 33, 34, 51, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3887(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 4, 21, 22, 23, 24, 31, 0, 1, 2, 19, 4, 19, 20, 5, 6, 7, 8, 47, 48, 49, 50, 35, 52, 35, 36, 53, 54, 55, 56, 63, 32, 33, 34, 51, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3888(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 4, 21, 22, 23, 24, 15, 16, 17, 18, 3, 4, 19, 20, 5, 6, 7, 8, 63, 32, 33, 34, 51, 52, 35, 36, 53, 54, 55, 56, 47, 48, 49, 50, 35, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3889(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 4, 21, 22, 23, 24, 31, 16, 17, 18, 3, 4, 19, 20, 5, 6, 7, 8, 47, 32, 33, 34, 51, 52, 35, 36, 53, 54, 55, 56, 63, 48, 49, 50, 35, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3890(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 4, 21, 22, 23, 24, 15, 0, 17, 18, 3, 4, 19, 20, 5, 6, 7, 8, 63, 48, 33, 34, 51, 52, 35, 36, 53, 54, 55, 56, 47, 32, 49, 50, 35, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3891(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 4, 21, 22, 23, 24, 31, 0, 17, 18, 3, 4, 19, 20, 5, 6, 7, 8, 47, 48, 33, 34, 51, 52, 35, 36, 53, 54, 55, 56, 63, 32, 49, 50, 35, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3892(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 4, 21, 22, 23, 24, 15, 16, 1, 18, 3, 4, 19, 20, 5, 6, 7, 8, 63, 32, 49, 34, 51, 52, 35, 36, 53, 54, 55, 56, 47, 48, 33, 50, 35, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3893(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 4, 21, 22, 23, 24, 31, 16, 1, 18, 3, 4, 19, 20, 5, 6, 7, 8, 47, 32, 49, 34, 51, 52, 35, 36, 53, 54, 55, 56, 63, 48, 33, 50, 35, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3894(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 4, 21, 22, 23, 24, 15, 0, 1, 18, 3, 4, 19, 20, 5, 6, 7, 8, 63, 48, 49, 34, 51, 52, 35, 36, 53, 54, 55, 56, 47, 32, 33, 50, 35, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3895(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 4, 21, 22, 23, 24, 31, 0, 1, 18, 3, 4, 19, 20, 5, 6, 7, 8, 47, 48, 49, 34, 51, 52, 35, 36, 53, 54, 55, 56, 63, 32, 33, 50, 35, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3896(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 4, 21, 22, 23, 24, 15, 16, 17, 2, 3, 4, 19, 20, 5, 6, 7, 8, 63, 32, 33, 50, 51, 52, 35, 36, 53, 54, 55, 56, 47, 48, 49, 34, 35, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3897(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 4, 21, 22, 23, 24, 31, 16, 17, 2, 3, 4, 19, 20, 5, 6, 7, 8, 47, 32, 33, 50, 51, 52, 35, 36, 53, 54, 55, 56, 63, 48, 49, 34, 35, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3898(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 4, 21, 22, 23, 24, 15, 0, 17, 2, 3, 4, 19, 20, 5, 6, 7, 8, 63, 48, 33, 50, 51, 52, 35, 36, 53, 54, 55, 56, 47, 32, 49, 34, 35, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3899(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 4, 21, 22, 23, 24, 31, 0, 17, 2, 3, 4, 19, 20, 5, 6, 7, 8, 47, 48, 33, 50, 51, 52, 35, 36, 53, 54, 55, 56, 63, 32, 49, 34, 35, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3900(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 4, 21, 22, 23, 24, 15, 16, 1, 2, 3, 4, 19, 20, 5, 6, 7, 8, 63, 32, 49, 50, 51, 52, 35, 36, 53, 54, 55, 56, 47, 48, 33, 34, 35, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3901(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 4, 21, 22, 23, 24, 31, 16, 1, 2, 3, 4, 19, 20, 5, 6, 7, 8, 47, 32, 49, 50, 51, 52, 35, 36, 53, 54, 55, 56, 63, 48, 33, 34, 35, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3902(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 4, 21, 22, 23, 24, 15, 0, 1, 2, 3, 4, 19, 20, 5, 6, 7, 8, 63, 48, 49, 50, 51, 52, 35, 36, 53, 54, 55, 56, 47, 32, 33, 34, 35, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3903(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 4, 21, 22, 23, 24, 31, 0, 1, 2, 3, 4, 19, 20, 5, 6, 7, 8, 47, 48, 49, 50, 51, 52, 35, 36, 53, 54, 55, 56, 63, 32, 33, 34, 35, 36, 51, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3904(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 4, 21, 22, 23, 24, 15, 16, 17, 18, 19, 20, 3, 20, 5, 6, 7, 8, 63, 32, 33, 34, 35, 36, 51, 36, 53, 54, 55, 56, 47, 48, 49, 50, 51, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3905(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 4, 21, 22, 23, 24, 31, 16, 17, 18, 19, 20, 3, 20, 5, 6, 7, 8, 47, 32, 33, 34, 35, 36, 51, 36, 53, 54, 55, 56, 63, 48, 49, 50, 51, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3906(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 4, 21, 22, 23, 24, 15, 0, 17, 18, 19, 20, 3, 20, 5, 6, 7, 8, 63, 48, 33, 34, 35, 36, 51, 36, 53, 54, 55, 56, 47, 32, 49, 50, 51, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3907(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 4, 21, 22, 23, 24, 31, 0, 17, 18, 19, 20, 3, 20, 5, 6, 7, 8, 47, 48, 33, 34, 35, 36, 51, 36, 53, 54, 55, 56, 63, 32, 49, 50, 51, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3908(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 4, 21, 22, 23, 24, 15, 16, 1, 18, 19, 20, 3, 20, 5, 6, 7, 8, 63, 32, 49, 34, 35, 36, 51, 36, 53, 54, 55, 56, 47, 48, 33, 50, 51, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3909(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 4, 21, 22, 23, 24, 31, 16, 1, 18, 19, 20, 3, 20, 5, 6, 7, 8, 47, 32, 49, 34, 35, 36, 51, 36, 53, 54, 55, 56, 63, 48, 33, 50, 51, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3910(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 4, 21, 22, 23, 24, 15, 0, 1, 18, 19, 20, 3, 20, 5, 6, 7, 8, 63, 48, 49, 34, 35, 36, 51, 36, 53, 54, 55, 56, 47, 32, 33, 50, 51, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3911(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 4, 21, 22, 23, 24, 31, 0, 1, 18, 19, 20, 3, 20, 5, 6, 7, 8, 47, 48, 49, 34, 35, 36, 51, 36, 53, 54, 55, 56, 63, 32, 33, 50, 51, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3912(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 4, 21, 22, 23, 24, 15, 16, 17, 2, 19, 20, 3, 20, 5, 6, 7, 8, 63, 32, 33, 50, 35, 36, 51, 36, 53, 54, 55, 56, 47, 48, 49, 34, 51, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3913(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 4, 21, 22, 23, 24, 31, 16, 17, 2, 19, 20, 3, 20, 5, 6, 7, 8, 47, 32, 33, 50, 35, 36, 51, 36, 53, 54, 55, 56, 63, 48, 49, 34, 51, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3914(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 4, 21, 22, 23, 24, 15, 0, 17, 2, 19, 20, 3, 20, 5, 6, 7, 8, 63, 48, 33, 50, 35, 36, 51, 36, 53, 54, 55, 56, 47, 32, 49, 34, 51, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3915(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 4, 21, 22, 23, 24, 31, 0, 17, 2, 19, 20, 3, 20, 5, 6, 7, 8, 47, 48, 33, 50, 35, 36, 51, 36, 53, 54, 55, 56, 63, 32, 49, 34, 51, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3916(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 4, 21, 22, 23, 24, 15, 16, 1, 2, 19, 20, 3, 20, 5, 6, 7, 8, 63, 32, 49, 50, 35, 36, 51, 36, 53, 54, 55, 56, 47, 48, 33, 34, 51, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3917(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 4, 21, 22, 23, 24, 31, 16, 1, 2, 19, 20, 3, 20, 5, 6, 7, 8, 47, 32, 49, 50, 35, 36, 51, 36, 53, 54, 55, 56, 63, 48, 33, 34, 51, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3918(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 4, 21, 22, 23, 24, 15, 0, 1, 2, 19, 20, 3, 20, 5, 6, 7, 8, 63, 48, 49, 50, 35, 36, 51, 36, 53, 54, 55, 56, 47, 32, 33, 34, 51, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3919(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 4, 21, 22, 23, 24, 31, 0, 1, 2, 19, 20, 3, 20, 5, 6, 7, 8, 47, 48, 49, 50, 35, 36, 51, 36, 53, 54, 55, 56, 63, 32, 33, 34, 51, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3920(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 4, 21, 22, 23, 24, 15, 16, 17, 18, 3, 20, 3, 20, 5, 6, 7, 8, 63, 32, 33, 34, 51, 36, 51, 36, 53, 54, 55, 56, 47, 48, 49, 50, 35, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3921(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 4, 21, 22, 23, 24, 31, 16, 17, 18, 3, 20, 3, 20, 5, 6, 7, 8, 47, 32, 33, 34, 51, 36, 51, 36, 53, 54, 55, 56, 63, 48, 49, 50, 35, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3922(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 4, 21, 22, 23, 24, 15, 0, 17, 18, 3, 20, 3, 20, 5, 6, 7, 8, 63, 48, 33, 34, 51, 36, 51, 36, 53, 54, 55, 56, 47, 32, 49, 50, 35, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3923(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 4, 21, 22, 23, 24, 31, 0, 17, 18, 3, 20, 3, 20, 5, 6, 7, 8, 47, 48, 33, 34, 51, 36, 51, 36, 53, 54, 55, 56, 63, 32, 49, 50, 35, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3924(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 4, 21, 22, 23, 24, 15, 16, 1, 18, 3, 20, 3, 20, 5, 6, 7, 8, 63, 32, 49, 34, 51, 36, 51, 36, 53, 54, 55, 56, 47, 48, 33, 50, 35, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3925(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 4, 21, 22, 23, 24, 31, 16, 1, 18, 3, 20, 3, 20, 5, 6, 7, 8, 47, 32, 49, 34, 51, 36, 51, 36, 53, 54, 55, 56, 63, 48, 33, 50, 35, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3926(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 4, 21, 22, 23, 24, 15, 0, 1, 18, 3, 20, 3, 20, 5, 6, 7, 8, 63, 48, 49, 34, 51, 36, 51, 36, 53, 54, 55, 56, 47, 32, 33, 50, 35, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3927(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 4, 21, 22, 23, 24, 31, 0, 1, 18, 3, 20, 3, 20, 5, 6, 7, 8, 47, 48, 49, 34, 51, 36, 51, 36, 53, 54, 55, 56, 63, 32, 33, 50, 35, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3928(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 4, 21, 22, 23, 24, 15, 16, 17, 2, 3, 20, 3, 20, 5, 6, 7, 8, 63, 32, 33, 50, 51, 36, 51, 36, 53, 54, 55, 56, 47, 48, 49, 34, 35, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3929(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 4, 21, 22, 23, 24, 31, 16, 17, 2, 3, 20, 3, 20, 5, 6, 7, 8, 47, 32, 33, 50, 51, 36, 51, 36, 53, 54, 55, 56, 63, 48, 49, 34, 35, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3930(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 4, 21, 22, 23, 24, 15, 0, 17, 2, 3, 20, 3, 20, 5, 6, 7, 8, 63, 48, 33, 50, 51, 36, 51, 36, 53, 54, 55, 56, 47, 32, 49, 34, 35, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3931(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 4, 21, 22, 23, 24, 31, 0, 17, 2, 3, 20, 3, 20, 5, 6, 7, 8, 47, 48, 33, 50, 51, 36, 51, 36, 53, 54, 55, 56, 63, 32, 49, 34, 35, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3932(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 4, 21, 22, 23, 24, 15, 16, 1, 2, 3, 20, 3, 20, 5, 6, 7, 8, 63, 32, 49, 50, 51, 36, 51, 36, 53, 54, 55, 56, 47, 48, 33, 34, 35, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3933(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 4, 21, 22, 23, 24, 31, 16, 1, 2, 3, 20, 3, 20, 5, 6, 7, 8, 47, 32, 49, 50, 51, 36, 51, 36, 53, 54, 55, 56, 63, 48, 33, 34, 35, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3934(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 4, 21, 22, 23, 24, 15, 0, 1, 2, 3, 20, 3, 20, 5, 6, 7, 8, 63, 48, 49, 50, 51, 36, 51, 36, 53, 54, 55, 56, 47, 32, 33, 34, 35, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3935(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 4, 21, 22, 23, 24, 31, 0, 1, 2, 3, 20, 3, 20, 5, 6, 7, 8, 47, 48, 49, 50, 51, 36, 51, 36, 53, 54, 55, 56, 63, 32, 33, 34, 35, 52, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3936(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 4, 21, 22, 23, 24, 15, 16, 17, 18, 19, 4, 3, 20, 5, 6, 7, 8, 63, 32, 33, 34, 35, 52, 51, 36, 53, 54, 55, 56, 47, 48, 49, 50, 51, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3937(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 4, 21, 22, 23, 24, 31, 16, 17, 18, 19, 4, 3, 20, 5, 6, 7, 8, 47, 32, 33, 34, 35, 52, 51, 36, 53, 54, 55, 56, 63, 48, 49, 50, 51, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3938(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 4, 21, 22, 23, 24, 15, 0, 17, 18, 19, 4, 3, 20, 5, 6, 7, 8, 63, 48, 33, 34, 35, 52, 51, 36, 53, 54, 55, 56, 47, 32, 49, 50, 51, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3939(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 4, 21, 22, 23, 24, 31, 0, 17, 18, 19, 4, 3, 20, 5, 6, 7, 8, 47, 48, 33, 34, 35, 52, 51, 36, 53, 54, 55, 56, 63, 32, 49, 50, 51, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3940(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 4, 21, 22, 23, 24, 15, 16, 1, 18, 19, 4, 3, 20, 5, 6, 7, 8, 63, 32, 49, 34, 35, 52, 51, 36, 53, 54, 55, 56, 47, 48, 33, 50, 51, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3941(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 4, 21, 22, 23, 24, 31, 16, 1, 18, 19, 4, 3, 20, 5, 6, 7, 8, 47, 32, 49, 34, 35, 52, 51, 36, 53, 54, 55, 56, 63, 48, 33, 50, 51, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3942(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 4, 21, 22, 23, 24, 15, 0, 1, 18, 19, 4, 3, 20, 5, 6, 7, 8, 63, 48, 49, 34, 35, 52, 51, 36, 53, 54, 55, 56, 47, 32, 33, 50, 51, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3943(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 4, 21, 22, 23, 24, 31, 0, 1, 18, 19, 4, 3, 20, 5, 6, 7, 8, 47, 48, 49, 34, 35, 52, 51, 36, 53, 54, 55, 56, 63, 32, 33, 50, 51, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3944(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 4, 21, 22, 23, 24, 15, 16, 17, 2, 19, 4, 3, 20, 5, 6, 7, 8, 63, 32, 33, 50, 35, 52, 51, 36, 53, 54, 55, 56, 47, 48, 49, 34, 51, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3945(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 4, 21, 22, 23, 24, 31, 16, 17, 2, 19, 4, 3, 20, 5, 6, 7, 8, 47, 32, 33, 50, 35, 52, 51, 36, 53, 54, 55, 56, 63, 48, 49, 34, 51, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3946(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 4, 21, 22, 23, 24, 15, 0, 17, 2, 19, 4, 3, 20, 5, 6, 7, 8, 63, 48, 33, 50, 35, 52, 51, 36, 53, 54, 55, 56, 47, 32, 49, 34, 51, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3947(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 4, 21, 22, 23, 24, 31, 0, 17, 2, 19, 4, 3, 20, 5, 6, 7, 8, 47, 48, 33, 50, 35, 52, 51, 36, 53, 54, 55, 56, 63, 32, 49, 34, 51, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3948(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 4, 21, 22, 23, 24, 15, 16, 1, 2, 19, 4, 3, 20, 5, 6, 7, 8, 63, 32, 49, 50, 35, 52, 51, 36, 53, 54, 55, 56, 47, 48, 33, 34, 51, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3949(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 4, 21, 22, 23, 24, 31, 16, 1, 2, 19, 4, 3, 20, 5, 6, 7, 8, 47, 32, 49, 50, 35, 52, 51, 36, 53, 54, 55, 56, 63, 48, 33, 34, 51, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3950(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 4, 21, 22, 23, 24, 15, 0, 1, 2, 19, 4, 3, 20, 5, 6, 7, 8, 63, 48, 49, 50, 35, 52, 51, 36, 53, 54, 55, 56, 47, 32, 33, 34, 51, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3951(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 4, 21, 22, 23, 24, 31, 0, 1, 2, 19, 4, 3, 20, 5, 6, 7, 8, 47, 48, 49, 50, 35, 52, 51, 36, 53, 54, 55, 56, 63, 32, 33, 34, 51, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3952(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 4, 21, 22, 23, 24, 15, 16, 17, 18, 3, 4, 3, 20, 5, 6, 7, 8, 63, 32, 33, 34, 51, 52, 51, 36, 53, 54, 55, 56, 47, 48, 49, 50, 35, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3953(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 4, 21, 22, 23, 24, 31, 16, 17, 18, 3, 4, 3, 20, 5, 6, 7, 8, 47, 32, 33, 34, 51, 52, 51, 36, 53, 54, 55, 56, 63, 48, 49, 50, 35, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3954(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 4, 21, 22, 23, 24, 15, 0, 17, 18, 3, 4, 3, 20, 5, 6, 7, 8, 63, 48, 33, 34, 51, 52, 51, 36, 53, 54, 55, 56, 47, 32, 49, 50, 35, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3955(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 4, 21, 22, 23, 24, 31, 0, 17, 18, 3, 4, 3, 20, 5, 6, 7, 8, 47, 48, 33, 34, 51, 52, 51, 36, 53, 54, 55, 56, 63, 32, 49, 50, 35, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3956(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 4, 21, 22, 23, 24, 15, 16, 1, 18, 3, 4, 3, 20, 5, 6, 7, 8, 63, 32, 49, 34, 51, 52, 51, 36, 53, 54, 55, 56, 47, 48, 33, 50, 35, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3957(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 4, 21, 22, 23, 24, 31, 16, 1, 18, 3, 4, 3, 20, 5, 6, 7, 8, 47, 32, 49, 34, 51, 52, 51, 36, 53, 54, 55, 56, 63, 48, 33, 50, 35, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3958(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 4, 21, 22, 23, 24, 15, 0, 1, 18, 3, 4, 3, 20, 5, 6, 7, 8, 63, 48, 49, 34, 51, 52, 51, 36, 53, 54, 55, 56, 47, 32, 33, 50, 35, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3959(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 4, 21, 22, 23, 24, 31, 0, 1, 18, 3, 4, 3, 20, 5, 6, 7, 8, 47, 48, 49, 34, 51, 52, 51, 36, 53, 54, 55, 56, 63, 32, 33, 50, 35, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3960(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 4, 21, 22, 23, 24, 15, 16, 17, 2, 3, 4, 3, 20, 5, 6, 7, 8, 63, 32, 33, 50, 51, 52, 51, 36, 53, 54, 55, 56, 47, 48, 49, 34, 35, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3961(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 4, 21, 22, 23, 24, 31, 16, 17, 2, 3, 4, 3, 20, 5, 6, 7, 8, 47, 32, 33, 50, 51, 52, 51, 36, 53, 54, 55, 56, 63, 48, 49, 34, 35, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3962(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 4, 21, 22, 23, 24, 15, 0, 17, 2, 3, 4, 3, 20, 5, 6, 7, 8, 63, 48, 33, 50, 51, 52, 51, 36, 53, 54, 55, 56, 47, 32, 49, 34, 35, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3963(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 4, 21, 22, 23, 24, 31, 0, 17, 2, 3, 4, 3, 20, 5, 6, 7, 8, 47, 48, 33, 50, 51, 52, 51, 36, 53, 54, 55, 56, 63, 32, 49, 34, 35, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3964(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 4, 21, 22, 23, 24, 15, 16, 1, 2, 3, 4, 3, 20, 5, 6, 7, 8, 63, 32, 49, 50, 51, 52, 51, 36, 53, 54, 55, 56, 47, 48, 33, 34, 35, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3965(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 4, 21, 22, 23, 24, 31, 16, 1, 2, 3, 4, 3, 20, 5, 6, 7, 8, 47, 32, 49, 50, 51, 52, 51, 36, 53, 54, 55, 56, 63, 48, 33, 34, 35, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3966(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 4, 21, 22, 23, 24, 15, 0, 1, 2, 3, 4, 3, 20, 5, 6, 7, 8, 63, 48, 49, 50, 51, 52, 51, 36, 53, 54, 55, 56, 47, 32, 33, 34, 35, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3967(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 4, 21, 22, 23, 24, 31, 0, 1, 2, 3, 4, 3, 20, 5, 6, 7, 8, 47, 48, 49, 50, 51, 52, 51, 36, 53, 54, 55, 56, 63, 32, 33, 34, 35, 36, 35, 52, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

#endif // DESGPU_COMPILE_ALL_SALTS


