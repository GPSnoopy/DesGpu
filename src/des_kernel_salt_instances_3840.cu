#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt3840(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 4, 21, 22, 23, 24, 15, 16, 17, 18, 19, 20, 19, 20, 5, 6, 7, 8, 63, 32, 33, 34, 35, 36, 35, 36, 53, 54, 55, 56, 47, 48, 49, 50, 51, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3841(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 4, 21, 22, 23, 24, 31, 16, 17, 18, 19, 20, 19, 20, 5, 6, 7, 8, 47, 32, 33, 34, 35, 36, 35, 36, 53, 54, 55, 56, 63, 48, 49, 50, 51, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3842(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 4, 21, 22, 23, 24, 15, 0, 17, 18, 19, 20, 19, 20, 5, 6, 7, 8, 63, 48, 33, 34, 35, 36, 35, 36, 53, 54, 55, 56, 47, 32, 49, 50, 51, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3843(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 4, 21, 22, 23, 24, 31, 0, 17, 18, 19, 20, 19, 20, 5, 6, 7, 8, 47, 48, 33, 34, 35, 36, 35, 36, 53, 54, 55, 56, 63, 32, 49, 50, 51, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3844(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 4, 21, 22, 23, 24, 15, 16, 1, 18, 19, 20, 19, 20, 5, 6, 7, 8, 63, 32, 49, 34, 35, 36, 35, 36, 53, 54, 55, 56, 47, 48, 33, 50, 51, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3845(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 4, 21, 22, 23, 24, 31, 16, 1, 18, 19, 20, 19, 20, 5, 6, 7, 8, 47, 32, 49, 34, 35, 36, 35, 36, 53, 54, 55, 56, 63, 48, 33, 50, 51, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3846(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 4, 21, 22, 23, 24, 15, 0, 1, 18, 19, 20, 19, 20, 5, 6, 7, 8, 63, 48, 49, 34, 35, 36, 35, 36, 53, 54, 55, 56, 47, 32, 33, 50, 51, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3847(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 4, 21, 22, 23, 24, 31, 0, 1, 18, 19, 20, 19, 20, 5, 6, 7, 8, 47, 48, 49, 34, 35, 36, 35, 36, 53, 54, 55, 56, 63, 32, 33, 50, 51, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3848(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 4, 21, 22, 23, 24, 15, 16, 17, 2, 19, 20, 19, 20, 5, 6, 7, 8, 63, 32, 33, 50, 35, 36, 35, 36, 53, 54, 55, 56, 47, 48, 49, 34, 51, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3849(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 4, 21, 22, 23, 24, 31, 16, 17, 2, 19, 20, 19, 20, 5, 6, 7, 8, 47, 32, 33, 50, 35, 36, 35, 36, 53, 54, 55, 56, 63, 48, 49, 34, 51, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3850(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 4, 21, 22, 23, 24, 15, 0, 17, 2, 19, 20, 19, 20, 5, 6, 7, 8, 63, 48, 33, 50, 35, 36, 35, 36, 53, 54, 55, 56, 47, 32, 49, 34, 51, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3851(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 4, 21, 22, 23, 24, 31, 0, 17, 2, 19, 20, 19, 20, 5, 6, 7, 8, 47, 48, 33, 50, 35, 36, 35, 36, 53, 54, 55, 56, 63, 32, 49, 34, 51, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3852(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 4, 21, 22, 23, 24, 15, 16, 1, 2, 19, 20, 19, 20, 5, 6, 7, 8, 63, 32, 49, 50, 35, 36, 35, 36, 53, 54, 55, 56, 47, 48, 33, 34, 51, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3853(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 4, 21, 22, 23, 24, 31, 16, 1, 2, 19, 20, 19, 20, 5, 6, 7, 8, 47, 32, 49, 50, 35, 36, 35, 36, 53, 54, 55, 56, 63, 48, 33, 34, 51, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3854(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 4, 21, 22, 23, 24, 15, 0, 1, 2, 19, 20, 19, 20, 5, 6, 7, 8, 63, 48, 49, 50, 35, 36, 35, 36, 53, 54, 55, 56, 47, 32, 33, 34, 51, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3855(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 4, 21, 22, 23, 24, 31, 0, 1, 2, 19, 20, 19, 20, 5, 6, 7, 8, 47, 48, 49, 50, 35, 36, 35, 36, 53, 54, 55, 56, 63, 32, 33, 34, 51, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3856(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 4, 21, 22, 23, 24, 15, 16, 17, 18, 3, 20, 19, 20, 5, 6, 7, 8, 63, 32, 33, 34, 51, 36, 35, 36, 53, 54, 55, 56, 47, 48, 49, 50, 35, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3857(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 4, 21, 22, 23, 24, 31, 16, 17, 18, 3, 20, 19, 20, 5, 6, 7, 8, 47, 32, 33, 34, 51, 36, 35, 36, 53, 54, 55, 56, 63, 48, 49, 50, 35, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3858(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 4, 21, 22, 23, 24, 15, 0, 17, 18, 3, 20, 19, 20, 5, 6, 7, 8, 63, 48, 33, 34, 51, 36, 35, 36, 53, 54, 55, 56, 47, 32, 49, 50, 35, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3859(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 4, 21, 22, 23, 24, 31, 0, 17, 18, 3, 20, 19, 20, 5, 6, 7, 8, 47, 48, 33, 34, 51, 36, 35, 36, 53, 54, 55, 56, 63, 32, 49, 50, 35, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3860(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 4, 21, 22, 23, 24, 15, 16, 1, 18, 3, 20, 19, 20, 5, 6, 7, 8, 63, 32, 49, 34, 51, 36, 35, 36, 53, 54, 55, 56, 47, 48, 33, 50, 35, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3861(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 4, 21, 22, 23, 24, 31, 16, 1, 18, 3, 20, 19, 20, 5, 6, 7, 8, 47, 32, 49, 34, 51, 36, 35, 36, 53, 54, 55, 56, 63, 48, 33, 50, 35, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3862(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 4, 21, 22, 23, 24, 15, 0, 1, 18, 3, 20, 19, 20, 5, 6, 7, 8, 63, 48, 49, 34, 51, 36, 35, 36, 53, 54, 55, 56, 47, 32, 33, 50, 35, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3863(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 4, 21, 22, 23, 24, 31, 0, 1, 18, 3, 20, 19, 20, 5, 6, 7, 8, 47, 48, 49, 34, 51, 36, 35, 36, 53, 54, 55, 56, 63, 32, 33, 50, 35, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3864(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 4, 21, 22, 23, 24, 15, 16, 17, 2, 3, 20, 19, 20, 5, 6, 7, 8, 63, 32, 33, 50, 51, 36, 35, 36, 53, 54, 55, 56, 47, 48, 49, 34, 35, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3865(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 4, 21, 22, 23, 24, 31, 16, 17, 2, 3, 20, 19, 20, 5, 6, 7, 8, 47, 32, 33, 50, 51, 36, 35, 36, 53, 54, 55, 56, 63, 48, 49, 34, 35, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3866(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 4, 21, 22, 23, 24, 15, 0, 17, 2, 3, 20, 19, 20, 5, 6, 7, 8, 63, 48, 33, 50, 51, 36, 35, 36, 53, 54, 55, 56, 47, 32, 49, 34, 35, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3867(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 4, 21, 22, 23, 24, 31, 0, 17, 2, 3, 20, 19, 20, 5, 6, 7, 8, 47, 48, 33, 50, 51, 36, 35, 36, 53, 54, 55, 56, 63, 32, 49, 34, 35, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3868(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 4, 21, 22, 23, 24, 15, 16, 1, 2, 3, 20, 19, 20, 5, 6, 7, 8, 63, 32, 49, 50, 51, 36, 35, 36, 53, 54, 55, 56, 47, 48, 33, 34, 35, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3869(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 4, 21, 22, 23, 24, 31, 16, 1, 2, 3, 20, 19, 20, 5, 6, 7, 8, 47, 32, 49, 50, 51, 36, 35, 36, 53, 54, 55, 56, 63, 48, 33, 34, 35, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3870(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 4, 21, 22, 23, 24, 15, 0, 1, 2, 3, 20, 19, 20, 5, 6, 7, 8, 63, 48, 49, 50, 51, 36, 35, 36, 53, 54, 55, 56, 47, 32, 33, 34, 35, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3871(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 4, 21, 22, 23, 24, 31, 0, 1, 2, 3, 20, 19, 20, 5, 6, 7, 8, 47, 48, 49, 50, 51, 36, 35, 36, 53, 54, 55, 56, 63, 32, 33, 34, 35, 52, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3872(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 4, 21, 22, 23, 24, 15, 16, 17, 18, 19, 4, 19, 20, 5, 6, 7, 8, 63, 32, 33, 34, 35, 52, 35, 36, 53, 54, 55, 56, 47, 48, 49, 50, 51, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3873(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 4, 21, 22, 23, 24, 31, 16, 17, 18, 19, 4, 19, 20, 5, 6, 7, 8, 47, 32, 33, 34, 35, 52, 35, 36, 53, 54, 55, 56, 63, 48, 49, 50, 51, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3874(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 4, 21, 22, 23, 24, 15, 0, 17, 18, 19, 4, 19, 20, 5, 6, 7, 8, 63, 48, 33, 34, 35, 52, 35, 36, 53, 54, 55, 56, 47, 32, 49, 50, 51, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3875(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 4, 21, 22, 23, 24, 31, 0, 17, 18, 19, 4, 19, 20, 5, 6, 7, 8, 47, 48, 33, 34, 35, 52, 35, 36, 53, 54, 55, 56, 63, 32, 49, 50, 51, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3876(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 4, 21, 22, 23, 24, 15, 16, 1, 18, 19, 4, 19, 20, 5, 6, 7, 8, 63, 32, 49, 34, 35, 52, 35, 36, 53, 54, 55, 56, 47, 48, 33, 50, 51, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3877(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 4, 21, 22, 23, 24, 31, 16, 1, 18, 19, 4, 19, 20, 5, 6, 7, 8, 47, 32, 49, 34, 35, 52, 35, 36, 53, 54, 55, 56, 63, 48, 33, 50, 51, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3878(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 4, 21, 22, 23, 24, 15, 0, 1, 18, 19, 4, 19, 20, 5, 6, 7, 8, 63, 48, 49, 34, 35, 52, 35, 36, 53, 54, 55, 56, 47, 32, 33, 50, 51, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3879(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 4, 21, 22, 23, 24, 31, 0, 1, 18, 19, 4, 19, 20, 5, 6, 7, 8, 47, 48, 49, 34, 35, 52, 35, 36, 53, 54, 55, 56, 63, 32, 33, 50, 51, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3880(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 4, 21, 22, 23, 24, 15, 16, 17, 2, 19, 4, 19, 20, 5, 6, 7, 8, 63, 32, 33, 50, 35, 52, 35, 36, 53, 54, 55, 56, 47, 48, 49, 34, 51, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3881(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 4, 21, 22, 23, 24, 31, 16, 17, 2, 19, 4, 19, 20, 5, 6, 7, 8, 47, 32, 33, 50, 35, 52, 35, 36, 53, 54, 55, 56, 63, 48, 49, 34, 51, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3882(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 4, 21, 22, 23, 24, 15, 0, 17, 2, 19, 4, 19, 20, 5, 6, 7, 8, 63, 48, 33, 50, 35, 52, 35, 36, 53, 54, 55, 56, 47, 32, 49, 34, 51, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3883(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 4, 21, 22, 23, 24, 31, 0, 17, 2, 19, 4, 19, 20, 5, 6, 7, 8, 47, 48, 33, 50, 35, 52, 35, 36, 53, 54, 55, 56, 63, 32, 49, 34, 51, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3884(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 4, 21, 22, 23, 24, 15, 16, 1, 2, 19, 4, 19, 20, 5, 6, 7, 8, 63, 32, 49, 50, 35, 52, 35, 36, 53, 54, 55, 56, 47, 48, 33, 34, 51, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3885(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 4, 21, 22, 23, 24, 31, 16, 1, 2, 19, 4, 19, 20, 5, 6, 7, 8, 47, 32, 49, 50, 35, 52, 35, 36, 53, 54, 55, 56, 63, 48, 33, 34, 51, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3886(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 4, 21, 22, 23, 24, 15, 0, 1, 2, 19, 4, 19, 20, 5, 6, 7, 8, 63, 48, 49, 50, 35, 52, 35, 36, 53, 54, 55, 56, 47, 32, 33, 34, 51, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3887(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 4, 21, 22, 23, 24, 31, 0, 1, 2, 19, 4, 19, 20, 5, 6, 7, 8, 47, 48, 49, 50, 35, 52, 35, 36, 53, 54, 55, 56, 63, 32, 33, 34, 51, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3888(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 4, 21, 22, 23, 24, 15, 16, 17, 18, 3, 4, 19, 20, 5, 6, 7, 8, 63, 32, 33, 34, 51, 52, 35, 36, 53, 54, 55, 56, 47, 48, 49, 50, 35, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3889(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 4, 21, 22, 23, 24, 31, 16, 17, 18, 3, 4, 19, 20, 5, 6, 7, 8, 47, 32, 33, 34, 51, 52, 35, 36, 53, 54, 55, 56, 63, 48, 49, 50, 35, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3890(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 4, 21, 22, 23, 24, 15, 0, 17, 18, 3, 4, 19, 20, 5, 6, 7, 8, 63, 48, 33, 34, 51, 52, 35, 36, 53, 54, 55, 56, 47, 32, 49, 50, 35, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3891(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 4, 21, 22, 23, 24, 31, 0, 17, 18, 3, 4, 19, 20, 5, 6, 7, 8, 47, 48, 33, 34, 51, 52, 35, 36, 53, 54, 55, 56, 63, 32, 49, 50, 35, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3892(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 4, 21, 22, 23, 24, 15, 16, 1, 18, 3, 4, 19, 20, 5, 6, 7, 8, 63, 32, 49, 34, 51, 52, 35, 36, 53, 54, 55, 56, 47, 48, 33, 50, 35, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3893(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 4, 21, 22, 23, 24, 31, 16, 1, 18, 3, 4, 19, 20, 5, 6, 7, 8, 47, 32, 49, 34, 51, 52, 35, 36, 53, 54, 55, 56, 63, 48, 33, 50, 35, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3894(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 4, 21, 22, 23, 24, 15, 0, 1, 18, 3, 4, 19, 20, 5, 6, 7, 8, 63, 48, 49, 34, 51, 52, 35, 36, 53, 54, 55, 56, 47, 32, 33, 50, 35, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3895(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 4, 21, 22, 23, 24, 31, 0, 1, 18, 3, 4, 19, 20, 5, 6, 7, 8, 47, 48, 49, 34, 51, 52, 35, 36, 53, 54, 55, 56, 63, 32, 33, 50, 35, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3896(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 4, 21, 22, 23, 24, 15, 16, 17, 2, 3, 4, 19, 20, 5, 6, 7, 8, 63, 32, 33, 50, 51, 52, 35, 36, 53, 54, 55, 56, 47, 48, 49, 34, 35, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3897(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 4, 21, 22, 23, 24, 31, 16, 17, 2, 3, 4, 19, 20, 5, 6, 7, 8, 47, 32, 33, 50, 51, 52, 35, 36, 53, 54, 55, 56, 63, 48, 49, 34, 35, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3898(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 4, 21, 22, 23, 24, 15, 0, 17, 2, 3, 4, 19, 20, 5, 6, 7, 8, 63, 48, 33, 50, 51, 52, 35, 36, 53, 54, 55, 56, 47, 32, 49, 34, 35, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3899(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 4, 21, 22, 23, 24, 31, 0, 17, 2, 3, 4, 19, 20, 5, 6, 7, 8, 47, 48, 33, 50, 51, 52, 35, 36, 53, 54, 55, 56, 63, 32, 49, 34, 35, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3900(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 4, 21, 22, 23, 24, 15, 16, 1, 2, 3, 4, 19, 20, 5, 6, 7, 8, 63, 32, 49, 50, 51, 52, 35, 36, 53, 54, 55, 56, 47, 48, 33, 34, 35, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3901(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 4, 21, 22, 23, 24, 31, 16, 1, 2, 3, 4, 19, 20, 5, 6, 7, 8, 47, 32, 49, 50, 51, 52, 35, 36, 53, 54, 55, 56, 63, 48, 33, 34, 35, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3902(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 4, 21, 22, 23, 24, 15, 0, 1, 2, 3, 4, 19, 20, 5, 6, 7, 8, 63, 48, 49, 50, 51, 52, 35, 36, 53, 54, 55, 56, 47, 32, 33, 34, 35, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3903(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 4, 21, 22, 23, 24, 31, 0, 1, 2, 3, 4, 19, 20, 5, 6, 7, 8, 47, 48, 49, 50, 51, 52, 35, 36, 53, 54, 55, 56, 63, 32, 33, 34, 35, 36, 51, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3904(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 4, 21, 22, 23, 24, 15, 16, 17, 18, 19, 20, 3, 20, 5, 6, 7, 8, 63, 32, 33, 34, 35, 36, 51, 36, 53, 54, 55, 56, 47, 48, 49, 50, 51, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3905(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 4, 21, 22, 23, 24, 31, 16, 17, 18, 19, 20, 3, 20, 5, 6, 7, 8, 47, 32, 33, 34, 35, 36, 51, 36, 53, 54, 55, 56, 63, 48, 49, 50, 51, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3906(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 4, 21, 22, 23, 24, 15, 0, 17, 18, 19, 20, 3, 20, 5, 6, 7, 8, 63, 48, 33, 34, 35, 36, 51, 36, 53, 54, 55, 56, 47, 32, 49, 50, 51, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3907(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 4, 21, 22, 23, 24, 31, 0, 17, 18, 19, 20, 3, 20, 5, 6, 7, 8, 47, 48, 33, 34, 35, 36, 51, 36, 53, 54, 55, 56, 63, 32, 49, 50, 51, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3908(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 4, 21, 22, 23, 24, 15, 16, 1, 18, 19, 20, 3, 20, 5, 6, 7, 8, 63, 32, 49, 34, 35, 36, 51, 36, 53, 54, 55, 56, 47, 48, 33, 50, 51, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3909(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 4, 21, 22, 23, 24, 31, 16, 1, 18, 19, 20, 3, 20, 5, 6, 7, 8, 47, 32, 49, 34, 35, 36, 51, 36, 53, 54, 55, 56, 63, 48, 33, 50, 51, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3910(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 4, 21, 22, 23, 24, 15, 0, 1, 18, 19, 20, 3, 20, 5, 6, 7, 8, 63, 48, 49, 34, 35, 36, 51, 36, 53, 54, 55, 56, 47, 32, 33, 50, 51, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3911(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 4, 21, 22, 23, 24, 31, 0, 1, 18, 19, 20, 3, 20, 5, 6, 7, 8, 47, 48, 49, 34, 35, 36, 51, 36, 53, 54, 55, 56, 63, 32, 33, 50, 51, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3912(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 4, 21, 22, 23, 24, 15, 16, 17, 2, 19, 20, 3, 20, 5, 6, 7, 8, 63, 32, 33, 50, 35, 36, 51, 36, 53, 54, 55, 56, 47, 48, 49, 34, 51, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3913(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 4, 21, 22, 23, 24, 31, 16, 17, 2, 19, 20, 3, 20, 5, 6, 7, 8, 47, 32, 33, 50, 35, 36, 51, 36, 53, 54, 55, 56, 63, 48, 49, 34, 51, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3914(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 4, 21, 22, 23, 24, 15, 0, 17, 2, 19, 20, 3, 20, 5, 6, 7, 8, 63, 48, 33, 50, 35, 36, 51, 36, 53, 54, 55, 56, 47, 32, 49, 34, 51, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3915(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 4, 21, 22, 23, 24, 31, 0, 17, 2, 19, 20, 3, 20, 5, 6, 7, 8, 47, 48, 33, 50, 35, 36, 51, 36, 53, 54, 55, 56, 63, 32, 49, 34, 51, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3916(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 4, 21, 22, 23, 24, 15, 16, 1, 2, 19, 20, 3, 20, 5, 6, 7, 8, 63, 32, 49, 50, 35, 36, 51, 36, 53, 54, 55, 56, 47, 48, 33, 34, 51, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3917(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 4, 21, 22, 23, 24, 31, 16, 1, 2, 19, 20, 3, 20, 5, 6, 7, 8, 47, 32, 49, 50, 35, 36, 51, 36, 53, 54, 55, 56, 63, 48, 33, 34, 51, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3918(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 4, 21, 22, 23, 24, 15, 0, 1, 2, 19, 20, 3, 20, 5, 6, 7, 8, 63, 48, 49, 50, 35, 36, 51, 36, 53, 54, 55, 56, 47, 32, 33, 34, 51, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3919(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 4, 21, 22, 23, 24, 31, 0, 1, 2, 19, 20, 3, 20, 5, 6, 7, 8, 47, 48, 49, 50, 35, 36, 51, 36, 53, 54, 55, 56, 63, 32, 33, 34, 51, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3920(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 4, 21, 22, 23, 24, 15, 16, 17, 18, 3, 20, 3, 20, 5, 6, 7, 8, 63, 32, 33, 34, 51, 36, 51, 36, 53, 54, 55, 56, 47, 48, 49, 50, 35, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3921(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 4, 21, 22, 23, 24, 31, 16, 17, 18, 3, 20, 3, 20, 5, 6, 7, 8, 47, 32, 33, 34, 51, 36, 51, 36, 53, 54, 55, 56, 63, 48, 49, 50, 35, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3922(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 4, 21, 22, 23, 24, 15, 0, 17, 18, 3, 20, 3, 20, 5, 6, 7, 8, 63, 48, 33, 34, 51, 36, 51, 36, 53, 54, 55, 56, 47, 32, 49, 50, 35, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3923(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 4, 21, 22, 23, 24, 31, 0, 17, 18, 3, 20, 3, 20, 5, 6, 7, 8, 47, 48, 33, 34, 51, 36, 51, 36, 53, 54, 55, 56, 63, 32, 49, 50, 35, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3924(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 4, 21, 22, 23, 24, 15, 16, 1, 18, 3, 20, 3, 20, 5, 6, 7, 8, 63, 32, 49, 34, 51, 36, 51, 36, 53, 54, 55, 56, 47, 48, 33, 50, 35, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3925(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 4, 21, 22, 23, 24, 31, 16, 1, 18, 3, 20, 3, 20, 5, 6, 7, 8, 47, 32, 49, 34, 51, 36, 51, 36, 53, 54, 55, 56, 63, 48, 33, 50, 35, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3926(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 4, 21, 22, 23, 24, 15, 0, 1, 18, 3, 20, 3, 20, 5, 6, 7, 8, 63, 48, 49, 34, 51, 36, 51, 36, 53, 54, 55, 56, 47, 32, 33, 50, 35, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3927(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 4, 21, 22, 23, 24, 31, 0, 1, 18, 3, 20, 3, 20, 5, 6, 7, 8, 47, 48, 49, 34, 51, 36, 51, 36, 53, 54, 55, 56, 63, 32, 33, 50, 35, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3928(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 4, 21, 22, 23, 24, 15, 16, 17, 2, 3, 20, 3, 20, 5, 6, 7, 8, 63, 32, 33, 50, 51, 36, 51, 36, 53, 54, 55, 56, 47, 48, 49, 34, 35, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3929(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 4, 21, 22, 23, 24, 31, 16, 17, 2, 3, 20, 3, 20, 5, 6, 7, 8, 47, 32, 33, 50, 51, 36, 51, 36, 53, 54, 55, 56, 63, 48, 49, 34, 35, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3930(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 4, 21, 22, 23, 24, 15, 0, 17, 2, 3, 20, 3, 20, 5, 6, 7, 8, 63, 48, 33, 50, 51, 36, 51, 36, 53, 54, 55, 56, 47, 32, 49, 34, 35, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3931(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 4, 21, 22, 23, 24, 31, 0, 17, 2, 3, 20, 3, 20, 5, 6, 7, 8, 47, 48, 33, 50, 51, 36, 51, 36, 53, 54, 55, 56, 63, 32, 49, 34, 35, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3932(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 4, 21, 22, 23, 24, 15, 16, 1, 2, 3, 20, 3, 20, 5, 6, 7, 8, 63, 32, 49, 50, 51, 36, 51, 36, 53, 54, 55, 56, 47, 48, 33, 34, 35, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3933(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 4, 21, 22, 23, 24, 31, 16, 1, 2, 3, 20, 3, 20, 5, 6, 7, 8, 47, 32, 49, 50, 51, 36, 51, 36, 53, 54, 55, 56, 63, 48, 33, 34, 35, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3934(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 4, 21, 22, 23, 24, 15, 0, 1, 2, 3, 20, 3, 20, 5, 6, 7, 8, 63, 48, 49, 50, 51, 36, 51, 36, 53, 54, 55, 56, 47, 32, 33, 34, 35, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3935(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 4, 21, 22, 23, 24, 31, 0, 1, 2, 3, 20, 3, 20, 5, 6, 7, 8, 47, 48, 49, 50, 51, 36, 51, 36, 53, 54, 55, 56, 63, 32, 33, 34, 35, 52, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3936(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 4, 21, 22, 23, 24, 15, 16, 17, 18, 19, 4, 3, 20, 5, 6, 7, 8, 63, 32, 33, 34, 35, 52, 51, 36, 53, 54, 55, 56, 47, 48, 49, 50, 51, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3937(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 4, 21, 22, 23, 24, 31, 16, 17, 18, 19, 4, 3, 20, 5, 6, 7, 8, 47, 32, 33, 34, 35, 52, 51, 36, 53, 54, 55, 56, 63, 48, 49, 50, 51, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3938(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 4, 21, 22, 23, 24, 15, 0, 17, 18, 19, 4, 3, 20, 5, 6, 7, 8, 63, 48, 33, 34, 35, 52, 51, 36, 53, 54, 55, 56, 47, 32, 49, 50, 51, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3939(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 4, 21, 22, 23, 24, 31, 0, 17, 18, 19, 4, 3, 20, 5, 6, 7, 8, 47, 48, 33, 34, 35, 52, 51, 36, 53, 54, 55, 56, 63, 32, 49, 50, 51, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3940(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 4, 21, 22, 23, 24, 15, 16, 1, 18, 19, 4, 3, 20, 5, 6, 7, 8, 63, 32, 49, 34, 35, 52, 51, 36, 53, 54, 55, 56, 47, 48, 33, 50, 51, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3941(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 4, 21, 22, 23, 24, 31, 16, 1, 18, 19, 4, 3, 20, 5, 6, 7, 8, 47, 32, 49, 34, 35, 52, 51, 36, 53, 54, 55, 56, 63, 48, 33, 50, 51, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3942(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 4, 21, 22, 23, 24, 15, 0, 1, 18, 19, 4, 3, 20, 5, 6, 7, 8, 63, 48, 49, 34, 35, 52, 51, 36, 53, 54, 55, 56, 47, 32, 33, 50, 51, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3943(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 4, 21, 22, 23, 24, 31, 0, 1, 18, 19, 4, 3, 20, 5, 6, 7, 8, 47, 48, 49, 34, 35, 52, 51, 36, 53, 54, 55, 56, 63, 32, 33, 50, 51, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3944(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 4, 21, 22, 23, 24, 15, 16, 17, 2, 19, 4, 3, 20, 5, 6, 7, 8, 63, 32, 33, 50, 35, 52, 51, 36, 53, 54, 55, 56, 47, 48, 49, 34, 51, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3945(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 4, 21, 22, 23, 24, 31, 16, 17, 2, 19, 4, 3, 20, 5, 6, 7, 8, 47, 32, 33, 50, 35, 52, 51, 36, 53, 54, 55, 56, 63, 48, 49, 34, 51, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3946(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 4, 21, 22, 23, 24, 15, 0, 17, 2, 19, 4, 3, 20, 5, 6, 7, 8, 63, 48, 33, 50, 35, 52, 51, 36, 53, 54, 55, 56, 47, 32, 49, 34, 51, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3947(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 4, 21, 22, 23, 24, 31, 0, 17, 2, 19, 4, 3, 20, 5, 6, 7, 8, 47, 48, 33, 50, 35, 52, 51, 36, 53, 54, 55, 56, 63, 32, 49, 34, 51, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3948(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 4, 21, 22, 23, 24, 15, 16, 1, 2, 19, 4, 3, 20, 5, 6, 7, 8, 63, 32, 49, 50, 35, 52, 51, 36, 53, 54, 55, 56, 47, 48, 33, 34, 51, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3949(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 4, 21, 22, 23, 24, 31, 16, 1, 2, 19, 4, 3, 20, 5, 6, 7, 8, 47, 32, 49, 50, 35, 52, 51, 36, 53, 54, 55, 56, 63, 48, 33, 34, 51, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3950(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 4, 21, 22, 23, 24, 15, 0, 1, 2, 19, 4, 3, 20, 5, 6, 7, 8, 63, 48, 49, 50, 35, 52, 51, 36, 53, 54, 55, 56, 47, 32, 33, 34, 51, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3951(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 4, 21, 22, 23, 24, 31, 0, 1, 2, 19, 4, 3, 20, 5, 6, 7, 8, 47, 48, 49, 50, 35, 52, 51, 36, 53, 54, 55, 56, 63, 32, 33, 34, 51, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3952(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 4, 21, 22, 23, 24, 15, 16, 17, 18, 3, 4, 3, 20, 5, 6, 7, 8, 63, 32, 33, 34, 51, 52, 51, 36, 53, 54, 55, 56, 47, 48, 49, 50, 35, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3953(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 4, 21, 22, 23, 24, 31, 16, 17, 18, 3, 4, 3, 20, 5, 6, 7, 8, 47, 32, 33, 34, 51, 52, 51, 36, 53, 54, 55, 56, 63, 48, 49, 50, 35, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3954(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 4, 21, 22, 23, 24, 15, 0, 17, 18, 3, 4, 3, 20, 5, 6, 7, 8, 63, 48, 33, 34, 51, 52, 51, 36, 53, 54, 55, 56, 47, 32, 49, 50, 35, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3955(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 4, 21, 22, 23, 24, 31, 0, 17, 18, 3, 4, 3, 20, 5, 6, 7, 8, 47, 48, 33, 34, 51, 52, 51, 36, 53, 54, 55, 56, 63, 32, 49, 50, 35, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3956(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 4, 21, 22, 23, 24, 15, 16, 1, 18, 3, 4, 3, 20, 5, 6, 7, 8, 63, 32, 49, 34, 51, 52, 51, 36, 53, 54, 55, 56, 47, 48, 33, 50, 35, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3957(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 4, 21, 22, 23, 24, 31, 16, 1, 18, 3, 4, 3, 20, 5, 6, 7, 8, 47, 32, 49, 34, 51, 52, 51, 36, 53, 54, 55, 56, 63, 48, 33, 50, 35, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3958(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 4, 21, 22, 23, 24, 15, 0, 1, 18, 3, 4, 3, 20, 5, 6, 7, 8, 63, 48, 49, 34, 51, 52, 51, 36, 53, 54, 55, 56, 47, 32, 33, 50, 35, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3959(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 4, 21, 22, 23, 24, 31, 0, 1, 18, 3, 4, 3, 20, 5, 6, 7, 8, 47, 48, 49, 34, 51, 52, 51, 36, 53, 54, 55, 56, 63, 32, 33, 50, 35, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3960(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 4, 21, 22, 23, 24, 15, 16, 17, 2, 3, 4, 3, 20, 5, 6, 7, 8, 63, 32, 33, 50, 51, 52, 51, 36, 53, 54, 55, 56, 47, 48, 49, 34, 35, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3961(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 4, 21, 22, 23, 24, 31, 16, 17, 2, 3, 4, 3, 20, 5, 6, 7, 8, 47, 32, 33, 50, 51, 52, 51, 36, 53, 54, 55, 56, 63, 48, 49, 34, 35, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3962(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 4, 21, 22, 23, 24, 15, 0, 17, 2, 3, 4, 3, 20, 5, 6, 7, 8, 63, 48, 33, 50, 51, 52, 51, 36, 53, 54, 55, 56, 47, 32, 49, 34, 35, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3963(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 4, 21, 22, 23, 24, 31, 0, 17, 2, 3, 4, 3, 20, 5, 6, 7, 8, 47, 48, 33, 50, 51, 52, 51, 36, 53, 54, 55, 56, 63, 32, 49, 34, 35, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3964(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 4, 21, 22, 23, 24, 15, 16, 1, 2, 3, 4, 3, 20, 5, 6, 7, 8, 63, 32, 49, 50, 51, 52, 51, 36, 53, 54, 55, 56, 47, 48, 33, 34, 35, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3965(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 4, 21, 22, 23, 24, 31, 16, 1, 2, 3, 4, 3, 20, 5, 6, 7, 8, 47, 32, 49, 50, 51, 52, 51, 36, 53, 54, 55, 56, 63, 48, 33, 34, 35, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3966(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 4, 21, 22, 23, 24, 15, 0, 1, 2, 3, 4, 3, 20, 5, 6, 7, 8, 63, 48, 49, 50, 51, 52, 51, 36, 53, 54, 55, 56, 47, 32, 33, 34, 35, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3967(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 4, 21, 22, 23, 24, 31, 0, 1, 2, 3, 4, 3, 20, 5, 6, 7, 8, 47, 48, 49, 50, 51, 52, 51, 36, 53, 54, 55, 56, 63, 32, 33, 34, 35, 36, 35, 52, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}


