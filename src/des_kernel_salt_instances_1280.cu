#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt1280(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 4, 21, 6, 23, 8, 15, 16, 17, 18, 19, 20, 19, 20, 5, 22, 7, 24, 63, 32, 33, 34, 35, 36, 35, 36, 53, 38, 55, 40, 47, 48, 49, 50, 51, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1281(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 4, 21, 6, 23, 8, 31, 16, 17, 18, 19, 20, 19, 20, 5, 22, 7, 24, 47, 32, 33, 34, 35, 36, 35, 36, 53, 38, 55, 40, 63, 48, 49, 50, 51, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1282(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 4, 21, 6, 23, 8, 15, 0, 17, 18, 19, 20, 19, 20, 5, 22, 7, 24, 63, 48, 33, 34, 35, 36, 35, 36, 53, 38, 55, 40, 47, 32, 49, 50, 51, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1283(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 4, 21, 6, 23, 8, 31, 0, 17, 18, 19, 20, 19, 20, 5, 22, 7, 24, 47, 48, 33, 34, 35, 36, 35, 36, 53, 38, 55, 40, 63, 32, 49, 50, 51, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1284(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 4, 21, 6, 23, 8, 15, 16, 1, 18, 19, 20, 19, 20, 5, 22, 7, 24, 63, 32, 49, 34, 35, 36, 35, 36, 53, 38, 55, 40, 47, 48, 33, 50, 51, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1285(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 4, 21, 6, 23, 8, 31, 16, 1, 18, 19, 20, 19, 20, 5, 22, 7, 24, 47, 32, 49, 34, 35, 36, 35, 36, 53, 38, 55, 40, 63, 48, 33, 50, 51, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1286(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 4, 21, 6, 23, 8, 15, 0, 1, 18, 19, 20, 19, 20, 5, 22, 7, 24, 63, 48, 49, 34, 35, 36, 35, 36, 53, 38, 55, 40, 47, 32, 33, 50, 51, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1287(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 4, 21, 6, 23, 8, 31, 0, 1, 18, 19, 20, 19, 20, 5, 22, 7, 24, 47, 48, 49, 34, 35, 36, 35, 36, 53, 38, 55, 40, 63, 32, 33, 50, 51, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1288(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 4, 21, 6, 23, 8, 15, 16, 17, 2, 19, 20, 19, 20, 5, 22, 7, 24, 63, 32, 33, 50, 35, 36, 35, 36, 53, 38, 55, 40, 47, 48, 49, 34, 51, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1289(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 4, 21, 6, 23, 8, 31, 16, 17, 2, 19, 20, 19, 20, 5, 22, 7, 24, 47, 32, 33, 50, 35, 36, 35, 36, 53, 38, 55, 40, 63, 48, 49, 34, 51, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1290(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 4, 21, 6, 23, 8, 15, 0, 17, 2, 19, 20, 19, 20, 5, 22, 7, 24, 63, 48, 33, 50, 35, 36, 35, 36, 53, 38, 55, 40, 47, 32, 49, 34, 51, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1291(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 4, 21, 6, 23, 8, 31, 0, 17, 2, 19, 20, 19, 20, 5, 22, 7, 24, 47, 48, 33, 50, 35, 36, 35, 36, 53, 38, 55, 40, 63, 32, 49, 34, 51, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1292(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 4, 21, 6, 23, 8, 15, 16, 1, 2, 19, 20, 19, 20, 5, 22, 7, 24, 63, 32, 49, 50, 35, 36, 35, 36, 53, 38, 55, 40, 47, 48, 33, 34, 51, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1293(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 4, 21, 6, 23, 8, 31, 16, 1, 2, 19, 20, 19, 20, 5, 22, 7, 24, 47, 32, 49, 50, 35, 36, 35, 36, 53, 38, 55, 40, 63, 48, 33, 34, 51, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1294(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 4, 21, 6, 23, 8, 15, 0, 1, 2, 19, 20, 19, 20, 5, 22, 7, 24, 63, 48, 49, 50, 35, 36, 35, 36, 53, 38, 55, 40, 47, 32, 33, 34, 51, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1295(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 4, 21, 6, 23, 8, 31, 0, 1, 2, 19, 20, 19, 20, 5, 22, 7, 24, 47, 48, 49, 50, 35, 36, 35, 36, 53, 38, 55, 40, 63, 32, 33, 34, 51, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1296(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 4, 21, 6, 23, 8, 15, 16, 17, 18, 3, 20, 19, 20, 5, 22, 7, 24, 63, 32, 33, 34, 51, 36, 35, 36, 53, 38, 55, 40, 47, 48, 49, 50, 35, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1297(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 4, 21, 6, 23, 8, 31, 16, 17, 18, 3, 20, 19, 20, 5, 22, 7, 24, 47, 32, 33, 34, 51, 36, 35, 36, 53, 38, 55, 40, 63, 48, 49, 50, 35, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1298(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 4, 21, 6, 23, 8, 15, 0, 17, 18, 3, 20, 19, 20, 5, 22, 7, 24, 63, 48, 33, 34, 51, 36, 35, 36, 53, 38, 55, 40, 47, 32, 49, 50, 35, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1299(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 4, 21, 6, 23, 8, 31, 0, 17, 18, 3, 20, 19, 20, 5, 22, 7, 24, 47, 48, 33, 34, 51, 36, 35, 36, 53, 38, 55, 40, 63, 32, 49, 50, 35, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1300(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 4, 21, 6, 23, 8, 15, 16, 1, 18, 3, 20, 19, 20, 5, 22, 7, 24, 63, 32, 49, 34, 51, 36, 35, 36, 53, 38, 55, 40, 47, 48, 33, 50, 35, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1301(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 4, 21, 6, 23, 8, 31, 16, 1, 18, 3, 20, 19, 20, 5, 22, 7, 24, 47, 32, 49, 34, 51, 36, 35, 36, 53, 38, 55, 40, 63, 48, 33, 50, 35, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1302(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 4, 21, 6, 23, 8, 15, 0, 1, 18, 3, 20, 19, 20, 5, 22, 7, 24, 63, 48, 49, 34, 51, 36, 35, 36, 53, 38, 55, 40, 47, 32, 33, 50, 35, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1303(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 4, 21, 6, 23, 8, 31, 0, 1, 18, 3, 20, 19, 20, 5, 22, 7, 24, 47, 48, 49, 34, 51, 36, 35, 36, 53, 38, 55, 40, 63, 32, 33, 50, 35, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1304(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 4, 21, 6, 23, 8, 15, 16, 17, 2, 3, 20, 19, 20, 5, 22, 7, 24, 63, 32, 33, 50, 51, 36, 35, 36, 53, 38, 55, 40, 47, 48, 49, 34, 35, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1305(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 4, 21, 6, 23, 8, 31, 16, 17, 2, 3, 20, 19, 20, 5, 22, 7, 24, 47, 32, 33, 50, 51, 36, 35, 36, 53, 38, 55, 40, 63, 48, 49, 34, 35, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1306(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 4, 21, 6, 23, 8, 15, 0, 17, 2, 3, 20, 19, 20, 5, 22, 7, 24, 63, 48, 33, 50, 51, 36, 35, 36, 53, 38, 55, 40, 47, 32, 49, 34, 35, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1307(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 4, 21, 6, 23, 8, 31, 0, 17, 2, 3, 20, 19, 20, 5, 22, 7, 24, 47, 48, 33, 50, 51, 36, 35, 36, 53, 38, 55, 40, 63, 32, 49, 34, 35, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1308(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 4, 21, 6, 23, 8, 15, 16, 1, 2, 3, 20, 19, 20, 5, 22, 7, 24, 63, 32, 49, 50, 51, 36, 35, 36, 53, 38, 55, 40, 47, 48, 33, 34, 35, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1309(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 4, 21, 6, 23, 8, 31, 16, 1, 2, 3, 20, 19, 20, 5, 22, 7, 24, 47, 32, 49, 50, 51, 36, 35, 36, 53, 38, 55, 40, 63, 48, 33, 34, 35, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1310(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 4, 21, 6, 23, 8, 15, 0, 1, 2, 3, 20, 19, 20, 5, 22, 7, 24, 63, 48, 49, 50, 51, 36, 35, 36, 53, 38, 55, 40, 47, 32, 33, 34, 35, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1311(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 4, 21, 6, 23, 8, 31, 0, 1, 2, 3, 20, 19, 20, 5, 22, 7, 24, 47, 48, 49, 50, 51, 36, 35, 36, 53, 38, 55, 40, 63, 32, 33, 34, 35, 52, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1312(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 4, 21, 6, 23, 8, 15, 16, 17, 18, 19, 4, 19, 20, 5, 22, 7, 24, 63, 32, 33, 34, 35, 52, 35, 36, 53, 38, 55, 40, 47, 48, 49, 50, 51, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1313(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 4, 21, 6, 23, 8, 31, 16, 17, 18, 19, 4, 19, 20, 5, 22, 7, 24, 47, 32, 33, 34, 35, 52, 35, 36, 53, 38, 55, 40, 63, 48, 49, 50, 51, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1314(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 4, 21, 6, 23, 8, 15, 0, 17, 18, 19, 4, 19, 20, 5, 22, 7, 24, 63, 48, 33, 34, 35, 52, 35, 36, 53, 38, 55, 40, 47, 32, 49, 50, 51, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1315(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 4, 21, 6, 23, 8, 31, 0, 17, 18, 19, 4, 19, 20, 5, 22, 7, 24, 47, 48, 33, 34, 35, 52, 35, 36, 53, 38, 55, 40, 63, 32, 49, 50, 51, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1316(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 4, 21, 6, 23, 8, 15, 16, 1, 18, 19, 4, 19, 20, 5, 22, 7, 24, 63, 32, 49, 34, 35, 52, 35, 36, 53, 38, 55, 40, 47, 48, 33, 50, 51, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1317(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 4, 21, 6, 23, 8, 31, 16, 1, 18, 19, 4, 19, 20, 5, 22, 7, 24, 47, 32, 49, 34, 35, 52, 35, 36, 53, 38, 55, 40, 63, 48, 33, 50, 51, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1318(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 4, 21, 6, 23, 8, 15, 0, 1, 18, 19, 4, 19, 20, 5, 22, 7, 24, 63, 48, 49, 34, 35, 52, 35, 36, 53, 38, 55, 40, 47, 32, 33, 50, 51, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1319(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 4, 21, 6, 23, 8, 31, 0, 1, 18, 19, 4, 19, 20, 5, 22, 7, 24, 47, 48, 49, 34, 35, 52, 35, 36, 53, 38, 55, 40, 63, 32, 33, 50, 51, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1320(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 4, 21, 6, 23, 8, 15, 16, 17, 2, 19, 4, 19, 20, 5, 22, 7, 24, 63, 32, 33, 50, 35, 52, 35, 36, 53, 38, 55, 40, 47, 48, 49, 34, 51, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1321(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 4, 21, 6, 23, 8, 31, 16, 17, 2, 19, 4, 19, 20, 5, 22, 7, 24, 47, 32, 33, 50, 35, 52, 35, 36, 53, 38, 55, 40, 63, 48, 49, 34, 51, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1322(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 4, 21, 6, 23, 8, 15, 0, 17, 2, 19, 4, 19, 20, 5, 22, 7, 24, 63, 48, 33, 50, 35, 52, 35, 36, 53, 38, 55, 40, 47, 32, 49, 34, 51, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1323(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 4, 21, 6, 23, 8, 31, 0, 17, 2, 19, 4, 19, 20, 5, 22, 7, 24, 47, 48, 33, 50, 35, 52, 35, 36, 53, 38, 55, 40, 63, 32, 49, 34, 51, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1324(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 4, 21, 6, 23, 8, 15, 16, 1, 2, 19, 4, 19, 20, 5, 22, 7, 24, 63, 32, 49, 50, 35, 52, 35, 36, 53, 38, 55, 40, 47, 48, 33, 34, 51, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1325(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 4, 21, 6, 23, 8, 31, 16, 1, 2, 19, 4, 19, 20, 5, 22, 7, 24, 47, 32, 49, 50, 35, 52, 35, 36, 53, 38, 55, 40, 63, 48, 33, 34, 51, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1326(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 4, 21, 6, 23, 8, 15, 0, 1, 2, 19, 4, 19, 20, 5, 22, 7, 24, 63, 48, 49, 50, 35, 52, 35, 36, 53, 38, 55, 40, 47, 32, 33, 34, 51, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1327(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 4, 21, 6, 23, 8, 31, 0, 1, 2, 19, 4, 19, 20, 5, 22, 7, 24, 47, 48, 49, 50, 35, 52, 35, 36, 53, 38, 55, 40, 63, 32, 33, 34, 51, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1328(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 4, 21, 6, 23, 8, 15, 16, 17, 18, 3, 4, 19, 20, 5, 22, 7, 24, 63, 32, 33, 34, 51, 52, 35, 36, 53, 38, 55, 40, 47, 48, 49, 50, 35, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1329(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 4, 21, 6, 23, 8, 31, 16, 17, 18, 3, 4, 19, 20, 5, 22, 7, 24, 47, 32, 33, 34, 51, 52, 35, 36, 53, 38, 55, 40, 63, 48, 49, 50, 35, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1330(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 4, 21, 6, 23, 8, 15, 0, 17, 18, 3, 4, 19, 20, 5, 22, 7, 24, 63, 48, 33, 34, 51, 52, 35, 36, 53, 38, 55, 40, 47, 32, 49, 50, 35, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1331(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 4, 21, 6, 23, 8, 31, 0, 17, 18, 3, 4, 19, 20, 5, 22, 7, 24, 47, 48, 33, 34, 51, 52, 35, 36, 53, 38, 55, 40, 63, 32, 49, 50, 35, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1332(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 4, 21, 6, 23, 8, 15, 16, 1, 18, 3, 4, 19, 20, 5, 22, 7, 24, 63, 32, 49, 34, 51, 52, 35, 36, 53, 38, 55, 40, 47, 48, 33, 50, 35, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1333(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 4, 21, 6, 23, 8, 31, 16, 1, 18, 3, 4, 19, 20, 5, 22, 7, 24, 47, 32, 49, 34, 51, 52, 35, 36, 53, 38, 55, 40, 63, 48, 33, 50, 35, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1334(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 4, 21, 6, 23, 8, 15, 0, 1, 18, 3, 4, 19, 20, 5, 22, 7, 24, 63, 48, 49, 34, 51, 52, 35, 36, 53, 38, 55, 40, 47, 32, 33, 50, 35, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1335(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 4, 21, 6, 23, 8, 31, 0, 1, 18, 3, 4, 19, 20, 5, 22, 7, 24, 47, 48, 49, 34, 51, 52, 35, 36, 53, 38, 55, 40, 63, 32, 33, 50, 35, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1336(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 4, 21, 6, 23, 8, 15, 16, 17, 2, 3, 4, 19, 20, 5, 22, 7, 24, 63, 32, 33, 50, 51, 52, 35, 36, 53, 38, 55, 40, 47, 48, 49, 34, 35, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1337(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 4, 21, 6, 23, 8, 31, 16, 17, 2, 3, 4, 19, 20, 5, 22, 7, 24, 47, 32, 33, 50, 51, 52, 35, 36, 53, 38, 55, 40, 63, 48, 49, 34, 35, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1338(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 4, 21, 6, 23, 8, 15, 0, 17, 2, 3, 4, 19, 20, 5, 22, 7, 24, 63, 48, 33, 50, 51, 52, 35, 36, 53, 38, 55, 40, 47, 32, 49, 34, 35, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1339(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 4, 21, 6, 23, 8, 31, 0, 17, 2, 3, 4, 19, 20, 5, 22, 7, 24, 47, 48, 33, 50, 51, 52, 35, 36, 53, 38, 55, 40, 63, 32, 49, 34, 35, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1340(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 4, 21, 6, 23, 8, 15, 16, 1, 2, 3, 4, 19, 20, 5, 22, 7, 24, 63, 32, 49, 50, 51, 52, 35, 36, 53, 38, 55, 40, 47, 48, 33, 34, 35, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1341(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 4, 21, 6, 23, 8, 31, 16, 1, 2, 3, 4, 19, 20, 5, 22, 7, 24, 47, 32, 49, 50, 51, 52, 35, 36, 53, 38, 55, 40, 63, 48, 33, 34, 35, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1342(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 4, 21, 6, 23, 8, 15, 0, 1, 2, 3, 4, 19, 20, 5, 22, 7, 24, 63, 48, 49, 50, 51, 52, 35, 36, 53, 38, 55, 40, 47, 32, 33, 34, 35, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1343(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 4, 21, 6, 23, 8, 31, 0, 1, 2, 3, 4, 19, 20, 5, 22, 7, 24, 47, 48, 49, 50, 51, 52, 35, 36, 53, 38, 55, 40, 63, 32, 33, 34, 35, 36, 51, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1344(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 4, 21, 6, 23, 8, 15, 16, 17, 18, 19, 20, 3, 20, 5, 22, 7, 24, 63, 32, 33, 34, 35, 36, 51, 36, 53, 38, 55, 40, 47, 48, 49, 50, 51, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1345(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 4, 21, 6, 23, 8, 31, 16, 17, 18, 19, 20, 3, 20, 5, 22, 7, 24, 47, 32, 33, 34, 35, 36, 51, 36, 53, 38, 55, 40, 63, 48, 49, 50, 51, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1346(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 4, 21, 6, 23, 8, 15, 0, 17, 18, 19, 20, 3, 20, 5, 22, 7, 24, 63, 48, 33, 34, 35, 36, 51, 36, 53, 38, 55, 40, 47, 32, 49, 50, 51, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1347(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 4, 21, 6, 23, 8, 31, 0, 17, 18, 19, 20, 3, 20, 5, 22, 7, 24, 47, 48, 33, 34, 35, 36, 51, 36, 53, 38, 55, 40, 63, 32, 49, 50, 51, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1348(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 4, 21, 6, 23, 8, 15, 16, 1, 18, 19, 20, 3, 20, 5, 22, 7, 24, 63, 32, 49, 34, 35, 36, 51, 36, 53, 38, 55, 40, 47, 48, 33, 50, 51, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1349(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 4, 21, 6, 23, 8, 31, 16, 1, 18, 19, 20, 3, 20, 5, 22, 7, 24, 47, 32, 49, 34, 35, 36, 51, 36, 53, 38, 55, 40, 63, 48, 33, 50, 51, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1350(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 4, 21, 6, 23, 8, 15, 0, 1, 18, 19, 20, 3, 20, 5, 22, 7, 24, 63, 48, 49, 34, 35, 36, 51, 36, 53, 38, 55, 40, 47, 32, 33, 50, 51, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1351(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 4, 21, 6, 23, 8, 31, 0, 1, 18, 19, 20, 3, 20, 5, 22, 7, 24, 47, 48, 49, 34, 35, 36, 51, 36, 53, 38, 55, 40, 63, 32, 33, 50, 51, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1352(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 4, 21, 6, 23, 8, 15, 16, 17, 2, 19, 20, 3, 20, 5, 22, 7, 24, 63, 32, 33, 50, 35, 36, 51, 36, 53, 38, 55, 40, 47, 48, 49, 34, 51, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1353(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 4, 21, 6, 23, 8, 31, 16, 17, 2, 19, 20, 3, 20, 5, 22, 7, 24, 47, 32, 33, 50, 35, 36, 51, 36, 53, 38, 55, 40, 63, 48, 49, 34, 51, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1354(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 4, 21, 6, 23, 8, 15, 0, 17, 2, 19, 20, 3, 20, 5, 22, 7, 24, 63, 48, 33, 50, 35, 36, 51, 36, 53, 38, 55, 40, 47, 32, 49, 34, 51, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1355(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 4, 21, 6, 23, 8, 31, 0, 17, 2, 19, 20, 3, 20, 5, 22, 7, 24, 47, 48, 33, 50, 35, 36, 51, 36, 53, 38, 55, 40, 63, 32, 49, 34, 51, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1356(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 4, 21, 6, 23, 8, 15, 16, 1, 2, 19, 20, 3, 20, 5, 22, 7, 24, 63, 32, 49, 50, 35, 36, 51, 36, 53, 38, 55, 40, 47, 48, 33, 34, 51, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1357(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 4, 21, 6, 23, 8, 31, 16, 1, 2, 19, 20, 3, 20, 5, 22, 7, 24, 47, 32, 49, 50, 35, 36, 51, 36, 53, 38, 55, 40, 63, 48, 33, 34, 51, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1358(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 4, 21, 6, 23, 8, 15, 0, 1, 2, 19, 20, 3, 20, 5, 22, 7, 24, 63, 48, 49, 50, 35, 36, 51, 36, 53, 38, 55, 40, 47, 32, 33, 34, 51, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1359(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 4, 21, 6, 23, 8, 31, 0, 1, 2, 19, 20, 3, 20, 5, 22, 7, 24, 47, 48, 49, 50, 35, 36, 51, 36, 53, 38, 55, 40, 63, 32, 33, 34, 51, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1360(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 4, 21, 6, 23, 8, 15, 16, 17, 18, 3, 20, 3, 20, 5, 22, 7, 24, 63, 32, 33, 34, 51, 36, 51, 36, 53, 38, 55, 40, 47, 48, 49, 50, 35, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1361(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 4, 21, 6, 23, 8, 31, 16, 17, 18, 3, 20, 3, 20, 5, 22, 7, 24, 47, 32, 33, 34, 51, 36, 51, 36, 53, 38, 55, 40, 63, 48, 49, 50, 35, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1362(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 4, 21, 6, 23, 8, 15, 0, 17, 18, 3, 20, 3, 20, 5, 22, 7, 24, 63, 48, 33, 34, 51, 36, 51, 36, 53, 38, 55, 40, 47, 32, 49, 50, 35, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1363(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 4, 21, 6, 23, 8, 31, 0, 17, 18, 3, 20, 3, 20, 5, 22, 7, 24, 47, 48, 33, 34, 51, 36, 51, 36, 53, 38, 55, 40, 63, 32, 49, 50, 35, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1364(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 4, 21, 6, 23, 8, 15, 16, 1, 18, 3, 20, 3, 20, 5, 22, 7, 24, 63, 32, 49, 34, 51, 36, 51, 36, 53, 38, 55, 40, 47, 48, 33, 50, 35, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1365(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 4, 21, 6, 23, 8, 31, 16, 1, 18, 3, 20, 3, 20, 5, 22, 7, 24, 47, 32, 49, 34, 51, 36, 51, 36, 53, 38, 55, 40, 63, 48, 33, 50, 35, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1366(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 4, 21, 6, 23, 8, 15, 0, 1, 18, 3, 20, 3, 20, 5, 22, 7, 24, 63, 48, 49, 34, 51, 36, 51, 36, 53, 38, 55, 40, 47, 32, 33, 50, 35, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1367(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 4, 21, 6, 23, 8, 31, 0, 1, 18, 3, 20, 3, 20, 5, 22, 7, 24, 47, 48, 49, 34, 51, 36, 51, 36, 53, 38, 55, 40, 63, 32, 33, 50, 35, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1368(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 4, 21, 6, 23, 8, 15, 16, 17, 2, 3, 20, 3, 20, 5, 22, 7, 24, 63, 32, 33, 50, 51, 36, 51, 36, 53, 38, 55, 40, 47, 48, 49, 34, 35, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1369(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 4, 21, 6, 23, 8, 31, 16, 17, 2, 3, 20, 3, 20, 5, 22, 7, 24, 47, 32, 33, 50, 51, 36, 51, 36, 53, 38, 55, 40, 63, 48, 49, 34, 35, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1370(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 4, 21, 6, 23, 8, 15, 0, 17, 2, 3, 20, 3, 20, 5, 22, 7, 24, 63, 48, 33, 50, 51, 36, 51, 36, 53, 38, 55, 40, 47, 32, 49, 34, 35, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1371(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 4, 21, 6, 23, 8, 31, 0, 17, 2, 3, 20, 3, 20, 5, 22, 7, 24, 47, 48, 33, 50, 51, 36, 51, 36, 53, 38, 55, 40, 63, 32, 49, 34, 35, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1372(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 4, 21, 6, 23, 8, 15, 16, 1, 2, 3, 20, 3, 20, 5, 22, 7, 24, 63, 32, 49, 50, 51, 36, 51, 36, 53, 38, 55, 40, 47, 48, 33, 34, 35, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1373(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 4, 21, 6, 23, 8, 31, 16, 1, 2, 3, 20, 3, 20, 5, 22, 7, 24, 47, 32, 49, 50, 51, 36, 51, 36, 53, 38, 55, 40, 63, 48, 33, 34, 35, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1374(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 4, 21, 6, 23, 8, 15, 0, 1, 2, 3, 20, 3, 20, 5, 22, 7, 24, 63, 48, 49, 50, 51, 36, 51, 36, 53, 38, 55, 40, 47, 32, 33, 34, 35, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1375(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 4, 21, 6, 23, 8, 31, 0, 1, 2, 3, 20, 3, 20, 5, 22, 7, 24, 47, 48, 49, 50, 51, 36, 51, 36, 53, 38, 55, 40, 63, 32, 33, 34, 35, 52, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1376(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 4, 21, 6, 23, 8, 15, 16, 17, 18, 19, 4, 3, 20, 5, 22, 7, 24, 63, 32, 33, 34, 35, 52, 51, 36, 53, 38, 55, 40, 47, 48, 49, 50, 51, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1377(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 4, 21, 6, 23, 8, 31, 16, 17, 18, 19, 4, 3, 20, 5, 22, 7, 24, 47, 32, 33, 34, 35, 52, 51, 36, 53, 38, 55, 40, 63, 48, 49, 50, 51, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1378(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 4, 21, 6, 23, 8, 15, 0, 17, 18, 19, 4, 3, 20, 5, 22, 7, 24, 63, 48, 33, 34, 35, 52, 51, 36, 53, 38, 55, 40, 47, 32, 49, 50, 51, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1379(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 4, 21, 6, 23, 8, 31, 0, 17, 18, 19, 4, 3, 20, 5, 22, 7, 24, 47, 48, 33, 34, 35, 52, 51, 36, 53, 38, 55, 40, 63, 32, 49, 50, 51, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1380(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 4, 21, 6, 23, 8, 15, 16, 1, 18, 19, 4, 3, 20, 5, 22, 7, 24, 63, 32, 49, 34, 35, 52, 51, 36, 53, 38, 55, 40, 47, 48, 33, 50, 51, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1381(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 4, 21, 6, 23, 8, 31, 16, 1, 18, 19, 4, 3, 20, 5, 22, 7, 24, 47, 32, 49, 34, 35, 52, 51, 36, 53, 38, 55, 40, 63, 48, 33, 50, 51, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1382(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 4, 21, 6, 23, 8, 15, 0, 1, 18, 19, 4, 3, 20, 5, 22, 7, 24, 63, 48, 49, 34, 35, 52, 51, 36, 53, 38, 55, 40, 47, 32, 33, 50, 51, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1383(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 4, 21, 6, 23, 8, 31, 0, 1, 18, 19, 4, 3, 20, 5, 22, 7, 24, 47, 48, 49, 34, 35, 52, 51, 36, 53, 38, 55, 40, 63, 32, 33, 50, 51, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1384(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 4, 21, 6, 23, 8, 15, 16, 17, 2, 19, 4, 3, 20, 5, 22, 7, 24, 63, 32, 33, 50, 35, 52, 51, 36, 53, 38, 55, 40, 47, 48, 49, 34, 51, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1385(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 4, 21, 6, 23, 8, 31, 16, 17, 2, 19, 4, 3, 20, 5, 22, 7, 24, 47, 32, 33, 50, 35, 52, 51, 36, 53, 38, 55, 40, 63, 48, 49, 34, 51, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1386(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 4, 21, 6, 23, 8, 15, 0, 17, 2, 19, 4, 3, 20, 5, 22, 7, 24, 63, 48, 33, 50, 35, 52, 51, 36, 53, 38, 55, 40, 47, 32, 49, 34, 51, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1387(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 4, 21, 6, 23, 8, 31, 0, 17, 2, 19, 4, 3, 20, 5, 22, 7, 24, 47, 48, 33, 50, 35, 52, 51, 36, 53, 38, 55, 40, 63, 32, 49, 34, 51, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1388(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 4, 21, 6, 23, 8, 15, 16, 1, 2, 19, 4, 3, 20, 5, 22, 7, 24, 63, 32, 49, 50, 35, 52, 51, 36, 53, 38, 55, 40, 47, 48, 33, 34, 51, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1389(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 4, 21, 6, 23, 8, 31, 16, 1, 2, 19, 4, 3, 20, 5, 22, 7, 24, 47, 32, 49, 50, 35, 52, 51, 36, 53, 38, 55, 40, 63, 48, 33, 34, 51, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1390(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 4, 21, 6, 23, 8, 15, 0, 1, 2, 19, 4, 3, 20, 5, 22, 7, 24, 63, 48, 49, 50, 35, 52, 51, 36, 53, 38, 55, 40, 47, 32, 33, 34, 51, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1391(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 4, 21, 6, 23, 8, 31, 0, 1, 2, 19, 4, 3, 20, 5, 22, 7, 24, 47, 48, 49, 50, 35, 52, 51, 36, 53, 38, 55, 40, 63, 32, 33, 34, 51, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1392(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 4, 21, 6, 23, 8, 15, 16, 17, 18, 3, 4, 3, 20, 5, 22, 7, 24, 63, 32, 33, 34, 51, 52, 51, 36, 53, 38, 55, 40, 47, 48, 49, 50, 35, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1393(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 4, 21, 6, 23, 8, 31, 16, 17, 18, 3, 4, 3, 20, 5, 22, 7, 24, 47, 32, 33, 34, 51, 52, 51, 36, 53, 38, 55, 40, 63, 48, 49, 50, 35, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1394(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 4, 21, 6, 23, 8, 15, 0, 17, 18, 3, 4, 3, 20, 5, 22, 7, 24, 63, 48, 33, 34, 51, 52, 51, 36, 53, 38, 55, 40, 47, 32, 49, 50, 35, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1395(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 4, 21, 6, 23, 8, 31, 0, 17, 18, 3, 4, 3, 20, 5, 22, 7, 24, 47, 48, 33, 34, 51, 52, 51, 36, 53, 38, 55, 40, 63, 32, 49, 50, 35, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1396(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 4, 21, 6, 23, 8, 15, 16, 1, 18, 3, 4, 3, 20, 5, 22, 7, 24, 63, 32, 49, 34, 51, 52, 51, 36, 53, 38, 55, 40, 47, 48, 33, 50, 35, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1397(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 4, 21, 6, 23, 8, 31, 16, 1, 18, 3, 4, 3, 20, 5, 22, 7, 24, 47, 32, 49, 34, 51, 52, 51, 36, 53, 38, 55, 40, 63, 48, 33, 50, 35, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1398(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 4, 21, 6, 23, 8, 15, 0, 1, 18, 3, 4, 3, 20, 5, 22, 7, 24, 63, 48, 49, 34, 51, 52, 51, 36, 53, 38, 55, 40, 47, 32, 33, 50, 35, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1399(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 4, 21, 6, 23, 8, 31, 0, 1, 18, 3, 4, 3, 20, 5, 22, 7, 24, 47, 48, 49, 34, 51, 52, 51, 36, 53, 38, 55, 40, 63, 32, 33, 50, 35, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1400(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 4, 21, 6, 23, 8, 15, 16, 17, 2, 3, 4, 3, 20, 5, 22, 7, 24, 63, 32, 33, 50, 51, 52, 51, 36, 53, 38, 55, 40, 47, 48, 49, 34, 35, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1401(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 4, 21, 6, 23, 8, 31, 16, 17, 2, 3, 4, 3, 20, 5, 22, 7, 24, 47, 32, 33, 50, 51, 52, 51, 36, 53, 38, 55, 40, 63, 48, 49, 34, 35, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1402(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 4, 21, 6, 23, 8, 15, 0, 17, 2, 3, 4, 3, 20, 5, 22, 7, 24, 63, 48, 33, 50, 51, 52, 51, 36, 53, 38, 55, 40, 47, 32, 49, 34, 35, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1403(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 4, 21, 6, 23, 8, 31, 0, 17, 2, 3, 4, 3, 20, 5, 22, 7, 24, 47, 48, 33, 50, 51, 52, 51, 36, 53, 38, 55, 40, 63, 32, 49, 34, 35, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1404(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 4, 21, 6, 23, 8, 15, 16, 1, 2, 3, 4, 3, 20, 5, 22, 7, 24, 63, 32, 49, 50, 51, 52, 51, 36, 53, 38, 55, 40, 47, 48, 33, 34, 35, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1405(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 4, 21, 6, 23, 8, 31, 16, 1, 2, 3, 4, 3, 20, 5, 22, 7, 24, 47, 32, 49, 50, 51, 52, 51, 36, 53, 38, 55, 40, 63, 48, 33, 34, 35, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1406(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 4, 21, 6, 23, 8, 15, 0, 1, 2, 3, 4, 3, 20, 5, 22, 7, 24, 63, 48, 49, 50, 51, 52, 51, 36, 53, 38, 55, 40, 47, 32, 33, 34, 35, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1407(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 4, 21, 6, 23, 8, 31, 0, 1, 2, 3, 4, 3, 20, 5, 22, 7, 24, 47, 48, 49, 50, 51, 52, 51, 36, 53, 38, 55, 40, 63, 32, 33, 34, 35, 36, 35, 52, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}


