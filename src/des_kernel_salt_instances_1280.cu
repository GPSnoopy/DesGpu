#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

#ifdef DESGPU_COMPILE_ALL_SALTS

void des_25_encrypt_salt1280(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 4, 21, 6, 23, 8, 15, 16, 17, 18, 19, 20, 19, 20, 5, 22, 7, 24, 63, 32, 33, 34, 35, 36, 35, 36, 53, 38, 55, 40, 47, 48, 49, 50, 51, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1281(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 4, 21, 6, 23, 8, 31, 16, 17, 18, 19, 20, 19, 20, 5, 22, 7, 24, 47, 32, 33, 34, 35, 36, 35, 36, 53, 38, 55, 40, 63, 48, 49, 50, 51, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1282(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 4, 21, 6, 23, 8, 15, 0, 17, 18, 19, 20, 19, 20, 5, 22, 7, 24, 63, 48, 33, 34, 35, 36, 35, 36, 53, 38, 55, 40, 47, 32, 49, 50, 51, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1283(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 4, 21, 6, 23, 8, 31, 0, 17, 18, 19, 20, 19, 20, 5, 22, 7, 24, 47, 48, 33, 34, 35, 36, 35, 36, 53, 38, 55, 40, 63, 32, 49, 50, 51, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1284(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 4, 21, 6, 23, 8, 15, 16, 1, 18, 19, 20, 19, 20, 5, 22, 7, 24, 63, 32, 49, 34, 35, 36, 35, 36, 53, 38, 55, 40, 47, 48, 33, 50, 51, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1285(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 4, 21, 6, 23, 8, 31, 16, 1, 18, 19, 20, 19, 20, 5, 22, 7, 24, 47, 32, 49, 34, 35, 36, 35, 36, 53, 38, 55, 40, 63, 48, 33, 50, 51, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1286(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 4, 21, 6, 23, 8, 15, 0, 1, 18, 19, 20, 19, 20, 5, 22, 7, 24, 63, 48, 49, 34, 35, 36, 35, 36, 53, 38, 55, 40, 47, 32, 33, 50, 51, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1287(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 4, 21, 6, 23, 8, 31, 0, 1, 18, 19, 20, 19, 20, 5, 22, 7, 24, 47, 48, 49, 34, 35, 36, 35, 36, 53, 38, 55, 40, 63, 32, 33, 50, 51, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1288(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 4, 21, 6, 23, 8, 15, 16, 17, 2, 19, 20, 19, 20, 5, 22, 7, 24, 63, 32, 33, 50, 35, 36, 35, 36, 53, 38, 55, 40, 47, 48, 49, 34, 51, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1289(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 4, 21, 6, 23, 8, 31, 16, 17, 2, 19, 20, 19, 20, 5, 22, 7, 24, 47, 32, 33, 50, 35, 36, 35, 36, 53, 38, 55, 40, 63, 48, 49, 34, 51, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1290(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 4, 21, 6, 23, 8, 15, 0, 17, 2, 19, 20, 19, 20, 5, 22, 7, 24, 63, 48, 33, 50, 35, 36, 35, 36, 53, 38, 55, 40, 47, 32, 49, 34, 51, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1291(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 4, 21, 6, 23, 8, 31, 0, 17, 2, 19, 20, 19, 20, 5, 22, 7, 24, 47, 48, 33, 50, 35, 36, 35, 36, 53, 38, 55, 40, 63, 32, 49, 34, 51, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1292(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 4, 21, 6, 23, 8, 15, 16, 1, 2, 19, 20, 19, 20, 5, 22, 7, 24, 63, 32, 49, 50, 35, 36, 35, 36, 53, 38, 55, 40, 47, 48, 33, 34, 51, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1293(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 4, 21, 6, 23, 8, 31, 16, 1, 2, 19, 20, 19, 20, 5, 22, 7, 24, 47, 32, 49, 50, 35, 36, 35, 36, 53, 38, 55, 40, 63, 48, 33, 34, 51, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1294(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 4, 21, 6, 23, 8, 15, 0, 1, 2, 19, 20, 19, 20, 5, 22, 7, 24, 63, 48, 49, 50, 35, 36, 35, 36, 53, 38, 55, 40, 47, 32, 33, 34, 51, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1295(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 4, 21, 6, 23, 8, 31, 0, 1, 2, 19, 20, 19, 20, 5, 22, 7, 24, 47, 48, 49, 50, 35, 36, 35, 36, 53, 38, 55, 40, 63, 32, 33, 34, 51, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1296(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 4, 21, 6, 23, 8, 15, 16, 17, 18, 3, 20, 19, 20, 5, 22, 7, 24, 63, 32, 33, 34, 51, 36, 35, 36, 53, 38, 55, 40, 47, 48, 49, 50, 35, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1297(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 4, 21, 6, 23, 8, 31, 16, 17, 18, 3, 20, 19, 20, 5, 22, 7, 24, 47, 32, 33, 34, 51, 36, 35, 36, 53, 38, 55, 40, 63, 48, 49, 50, 35, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1298(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 4, 21, 6, 23, 8, 15, 0, 17, 18, 3, 20, 19, 20, 5, 22, 7, 24, 63, 48, 33, 34, 51, 36, 35, 36, 53, 38, 55, 40, 47, 32, 49, 50, 35, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1299(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 4, 21, 6, 23, 8, 31, 0, 17, 18, 3, 20, 19, 20, 5, 22, 7, 24, 47, 48, 33, 34, 51, 36, 35, 36, 53, 38, 55, 40, 63, 32, 49, 50, 35, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1300(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 4, 21, 6, 23, 8, 15, 16, 1, 18, 3, 20, 19, 20, 5, 22, 7, 24, 63, 32, 49, 34, 51, 36, 35, 36, 53, 38, 55, 40, 47, 48, 33, 50, 35, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1301(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 4, 21, 6, 23, 8, 31, 16, 1, 18, 3, 20, 19, 20, 5, 22, 7, 24, 47, 32, 49, 34, 51, 36, 35, 36, 53, 38, 55, 40, 63, 48, 33, 50, 35, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1302(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 4, 21, 6, 23, 8, 15, 0, 1, 18, 3, 20, 19, 20, 5, 22, 7, 24, 63, 48, 49, 34, 51, 36, 35, 36, 53, 38, 55, 40, 47, 32, 33, 50, 35, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1303(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 4, 21, 6, 23, 8, 31, 0, 1, 18, 3, 20, 19, 20, 5, 22, 7, 24, 47, 48, 49, 34, 51, 36, 35, 36, 53, 38, 55, 40, 63, 32, 33, 50, 35, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1304(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 4, 21, 6, 23, 8, 15, 16, 17, 2, 3, 20, 19, 20, 5, 22, 7, 24, 63, 32, 33, 50, 51, 36, 35, 36, 53, 38, 55, 40, 47, 48, 49, 34, 35, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1305(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 4, 21, 6, 23, 8, 31, 16, 17, 2, 3, 20, 19, 20, 5, 22, 7, 24, 47, 32, 33, 50, 51, 36, 35, 36, 53, 38, 55, 40, 63, 48, 49, 34, 35, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1306(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 4, 21, 6, 23, 8, 15, 0, 17, 2, 3, 20, 19, 20, 5, 22, 7, 24, 63, 48, 33, 50, 51, 36, 35, 36, 53, 38, 55, 40, 47, 32, 49, 34, 35, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1307(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 4, 21, 6, 23, 8, 31, 0, 17, 2, 3, 20, 19, 20, 5, 22, 7, 24, 47, 48, 33, 50, 51, 36, 35, 36, 53, 38, 55, 40, 63, 32, 49, 34, 35, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1308(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 4, 21, 6, 23, 8, 15, 16, 1, 2, 3, 20, 19, 20, 5, 22, 7, 24, 63, 32, 49, 50, 51, 36, 35, 36, 53, 38, 55, 40, 47, 48, 33, 34, 35, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1309(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 4, 21, 6, 23, 8, 31, 16, 1, 2, 3, 20, 19, 20, 5, 22, 7, 24, 47, 32, 49, 50, 51, 36, 35, 36, 53, 38, 55, 40, 63, 48, 33, 34, 35, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1310(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 4, 21, 6, 23, 8, 15, 0, 1, 2, 3, 20, 19, 20, 5, 22, 7, 24, 63, 48, 49, 50, 51, 36, 35, 36, 53, 38, 55, 40, 47, 32, 33, 34, 35, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1311(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 4, 21, 6, 23, 8, 31, 0, 1, 2, 3, 20, 19, 20, 5, 22, 7, 24, 47, 48, 49, 50, 51, 36, 35, 36, 53, 38, 55, 40, 63, 32, 33, 34, 35, 52, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1312(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 4, 21, 6, 23, 8, 15, 16, 17, 18, 19, 4, 19, 20, 5, 22, 7, 24, 63, 32, 33, 34, 35, 52, 35, 36, 53, 38, 55, 40, 47, 48, 49, 50, 51, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1313(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 4, 21, 6, 23, 8, 31, 16, 17, 18, 19, 4, 19, 20, 5, 22, 7, 24, 47, 32, 33, 34, 35, 52, 35, 36, 53, 38, 55, 40, 63, 48, 49, 50, 51, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1314(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 4, 21, 6, 23, 8, 15, 0, 17, 18, 19, 4, 19, 20, 5, 22, 7, 24, 63, 48, 33, 34, 35, 52, 35, 36, 53, 38, 55, 40, 47, 32, 49, 50, 51, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1315(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 4, 21, 6, 23, 8, 31, 0, 17, 18, 19, 4, 19, 20, 5, 22, 7, 24, 47, 48, 33, 34, 35, 52, 35, 36, 53, 38, 55, 40, 63, 32, 49, 50, 51, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1316(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 4, 21, 6, 23, 8, 15, 16, 1, 18, 19, 4, 19, 20, 5, 22, 7, 24, 63, 32, 49, 34, 35, 52, 35, 36, 53, 38, 55, 40, 47, 48, 33, 50, 51, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1317(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 4, 21, 6, 23, 8, 31, 16, 1, 18, 19, 4, 19, 20, 5, 22, 7, 24, 47, 32, 49, 34, 35, 52, 35, 36, 53, 38, 55, 40, 63, 48, 33, 50, 51, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1318(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 4, 21, 6, 23, 8, 15, 0, 1, 18, 19, 4, 19, 20, 5, 22, 7, 24, 63, 48, 49, 34, 35, 52, 35, 36, 53, 38, 55, 40, 47, 32, 33, 50, 51, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1319(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 4, 21, 6, 23, 8, 31, 0, 1, 18, 19, 4, 19, 20, 5, 22, 7, 24, 47, 48, 49, 34, 35, 52, 35, 36, 53, 38, 55, 40, 63, 32, 33, 50, 51, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1320(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 4, 21, 6, 23, 8, 15, 16, 17, 2, 19, 4, 19, 20, 5, 22, 7, 24, 63, 32, 33, 50, 35, 52, 35, 36, 53, 38, 55, 40, 47, 48, 49, 34, 51, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1321(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 4, 21, 6, 23, 8, 31, 16, 17, 2, 19, 4, 19, 20, 5, 22, 7, 24, 47, 32, 33, 50, 35, 52, 35, 36, 53, 38, 55, 40, 63, 48, 49, 34, 51, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1322(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 4, 21, 6, 23, 8, 15, 0, 17, 2, 19, 4, 19, 20, 5, 22, 7, 24, 63, 48, 33, 50, 35, 52, 35, 36, 53, 38, 55, 40, 47, 32, 49, 34, 51, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1323(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 4, 21, 6, 23, 8, 31, 0, 17, 2, 19, 4, 19, 20, 5, 22, 7, 24, 47, 48, 33, 50, 35, 52, 35, 36, 53, 38, 55, 40, 63, 32, 49, 34, 51, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1324(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 4, 21, 6, 23, 8, 15, 16, 1, 2, 19, 4, 19, 20, 5, 22, 7, 24, 63, 32, 49, 50, 35, 52, 35, 36, 53, 38, 55, 40, 47, 48, 33, 34, 51, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1325(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 4, 21, 6, 23, 8, 31, 16, 1, 2, 19, 4, 19, 20, 5, 22, 7, 24, 47, 32, 49, 50, 35, 52, 35, 36, 53, 38, 55, 40, 63, 48, 33, 34, 51, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1326(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 4, 21, 6, 23, 8, 15, 0, 1, 2, 19, 4, 19, 20, 5, 22, 7, 24, 63, 48, 49, 50, 35, 52, 35, 36, 53, 38, 55, 40, 47, 32, 33, 34, 51, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1327(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 4, 21, 6, 23, 8, 31, 0, 1, 2, 19, 4, 19, 20, 5, 22, 7, 24, 47, 48, 49, 50, 35, 52, 35, 36, 53, 38, 55, 40, 63, 32, 33, 34, 51, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1328(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 4, 21, 6, 23, 8, 15, 16, 17, 18, 3, 4, 19, 20, 5, 22, 7, 24, 63, 32, 33, 34, 51, 52, 35, 36, 53, 38, 55, 40, 47, 48, 49, 50, 35, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1329(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 4, 21, 6, 23, 8, 31, 16, 17, 18, 3, 4, 19, 20, 5, 22, 7, 24, 47, 32, 33, 34, 51, 52, 35, 36, 53, 38, 55, 40, 63, 48, 49, 50, 35, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1330(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 4, 21, 6, 23, 8, 15, 0, 17, 18, 3, 4, 19, 20, 5, 22, 7, 24, 63, 48, 33, 34, 51, 52, 35, 36, 53, 38, 55, 40, 47, 32, 49, 50, 35, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1331(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 4, 21, 6, 23, 8, 31, 0, 17, 18, 3, 4, 19, 20, 5, 22, 7, 24, 47, 48, 33, 34, 51, 52, 35, 36, 53, 38, 55, 40, 63, 32, 49, 50, 35, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1332(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 4, 21, 6, 23, 8, 15, 16, 1, 18, 3, 4, 19, 20, 5, 22, 7, 24, 63, 32, 49, 34, 51, 52, 35, 36, 53, 38, 55, 40, 47, 48, 33, 50, 35, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1333(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 4, 21, 6, 23, 8, 31, 16, 1, 18, 3, 4, 19, 20, 5, 22, 7, 24, 47, 32, 49, 34, 51, 52, 35, 36, 53, 38, 55, 40, 63, 48, 33, 50, 35, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1334(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 4, 21, 6, 23, 8, 15, 0, 1, 18, 3, 4, 19, 20, 5, 22, 7, 24, 63, 48, 49, 34, 51, 52, 35, 36, 53, 38, 55, 40, 47, 32, 33, 50, 35, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1335(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 4, 21, 6, 23, 8, 31, 0, 1, 18, 3, 4, 19, 20, 5, 22, 7, 24, 47, 48, 49, 34, 51, 52, 35, 36, 53, 38, 55, 40, 63, 32, 33, 50, 35, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1336(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 4, 21, 6, 23, 8, 15, 16, 17, 2, 3, 4, 19, 20, 5, 22, 7, 24, 63, 32, 33, 50, 51, 52, 35, 36, 53, 38, 55, 40, 47, 48, 49, 34, 35, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1337(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 4, 21, 6, 23, 8, 31, 16, 17, 2, 3, 4, 19, 20, 5, 22, 7, 24, 47, 32, 33, 50, 51, 52, 35, 36, 53, 38, 55, 40, 63, 48, 49, 34, 35, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1338(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 4, 21, 6, 23, 8, 15, 0, 17, 2, 3, 4, 19, 20, 5, 22, 7, 24, 63, 48, 33, 50, 51, 52, 35, 36, 53, 38, 55, 40, 47, 32, 49, 34, 35, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1339(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 4, 21, 6, 23, 8, 31, 0, 17, 2, 3, 4, 19, 20, 5, 22, 7, 24, 47, 48, 33, 50, 51, 52, 35, 36, 53, 38, 55, 40, 63, 32, 49, 34, 35, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1340(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 4, 21, 6, 23, 8, 15, 16, 1, 2, 3, 4, 19, 20, 5, 22, 7, 24, 63, 32, 49, 50, 51, 52, 35, 36, 53, 38, 55, 40, 47, 48, 33, 34, 35, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1341(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 4, 21, 6, 23, 8, 31, 16, 1, 2, 3, 4, 19, 20, 5, 22, 7, 24, 47, 32, 49, 50, 51, 52, 35, 36, 53, 38, 55, 40, 63, 48, 33, 34, 35, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1342(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 4, 21, 6, 23, 8, 15, 0, 1, 2, 3, 4, 19, 20, 5, 22, 7, 24, 63, 48, 49, 50, 51, 52, 35, 36, 53, 38, 55, 40, 47, 32, 33, 34, 35, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1343(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 4, 21, 6, 23, 8, 31, 0, 1, 2, 3, 4, 19, 20, 5, 22, 7, 24, 47, 48, 49, 50, 51, 52, 35, 36, 53, 38, 55, 40, 63, 32, 33, 34, 35, 36, 51, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1344(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 4, 21, 6, 23, 8, 15, 16, 17, 18, 19, 20, 3, 20, 5, 22, 7, 24, 63, 32, 33, 34, 35, 36, 51, 36, 53, 38, 55, 40, 47, 48, 49, 50, 51, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1345(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 4, 21, 6, 23, 8, 31, 16, 17, 18, 19, 20, 3, 20, 5, 22, 7, 24, 47, 32, 33, 34, 35, 36, 51, 36, 53, 38, 55, 40, 63, 48, 49, 50, 51, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1346(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 4, 21, 6, 23, 8, 15, 0, 17, 18, 19, 20, 3, 20, 5, 22, 7, 24, 63, 48, 33, 34, 35, 36, 51, 36, 53, 38, 55, 40, 47, 32, 49, 50, 51, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1347(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 4, 21, 6, 23, 8, 31, 0, 17, 18, 19, 20, 3, 20, 5, 22, 7, 24, 47, 48, 33, 34, 35, 36, 51, 36, 53, 38, 55, 40, 63, 32, 49, 50, 51, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1348(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 4, 21, 6, 23, 8, 15, 16, 1, 18, 19, 20, 3, 20, 5, 22, 7, 24, 63, 32, 49, 34, 35, 36, 51, 36, 53, 38, 55, 40, 47, 48, 33, 50, 51, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1349(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 4, 21, 6, 23, 8, 31, 16, 1, 18, 19, 20, 3, 20, 5, 22, 7, 24, 47, 32, 49, 34, 35, 36, 51, 36, 53, 38, 55, 40, 63, 48, 33, 50, 51, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1350(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 4, 21, 6, 23, 8, 15, 0, 1, 18, 19, 20, 3, 20, 5, 22, 7, 24, 63, 48, 49, 34, 35, 36, 51, 36, 53, 38, 55, 40, 47, 32, 33, 50, 51, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1351(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 4, 21, 6, 23, 8, 31, 0, 1, 18, 19, 20, 3, 20, 5, 22, 7, 24, 47, 48, 49, 34, 35, 36, 51, 36, 53, 38, 55, 40, 63, 32, 33, 50, 51, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1352(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 4, 21, 6, 23, 8, 15, 16, 17, 2, 19, 20, 3, 20, 5, 22, 7, 24, 63, 32, 33, 50, 35, 36, 51, 36, 53, 38, 55, 40, 47, 48, 49, 34, 51, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1353(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 4, 21, 6, 23, 8, 31, 16, 17, 2, 19, 20, 3, 20, 5, 22, 7, 24, 47, 32, 33, 50, 35, 36, 51, 36, 53, 38, 55, 40, 63, 48, 49, 34, 51, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1354(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 4, 21, 6, 23, 8, 15, 0, 17, 2, 19, 20, 3, 20, 5, 22, 7, 24, 63, 48, 33, 50, 35, 36, 51, 36, 53, 38, 55, 40, 47, 32, 49, 34, 51, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1355(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 4, 21, 6, 23, 8, 31, 0, 17, 2, 19, 20, 3, 20, 5, 22, 7, 24, 47, 48, 33, 50, 35, 36, 51, 36, 53, 38, 55, 40, 63, 32, 49, 34, 51, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1356(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 4, 21, 6, 23, 8, 15, 16, 1, 2, 19, 20, 3, 20, 5, 22, 7, 24, 63, 32, 49, 50, 35, 36, 51, 36, 53, 38, 55, 40, 47, 48, 33, 34, 51, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1357(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 4, 21, 6, 23, 8, 31, 16, 1, 2, 19, 20, 3, 20, 5, 22, 7, 24, 47, 32, 49, 50, 35, 36, 51, 36, 53, 38, 55, 40, 63, 48, 33, 34, 51, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1358(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 4, 21, 6, 23, 8, 15, 0, 1, 2, 19, 20, 3, 20, 5, 22, 7, 24, 63, 48, 49, 50, 35, 36, 51, 36, 53, 38, 55, 40, 47, 32, 33, 34, 51, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1359(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 4, 21, 6, 23, 8, 31, 0, 1, 2, 19, 20, 3, 20, 5, 22, 7, 24, 47, 48, 49, 50, 35, 36, 51, 36, 53, 38, 55, 40, 63, 32, 33, 34, 51, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1360(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 4, 21, 6, 23, 8, 15, 16, 17, 18, 3, 20, 3, 20, 5, 22, 7, 24, 63, 32, 33, 34, 51, 36, 51, 36, 53, 38, 55, 40, 47, 48, 49, 50, 35, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1361(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 4, 21, 6, 23, 8, 31, 16, 17, 18, 3, 20, 3, 20, 5, 22, 7, 24, 47, 32, 33, 34, 51, 36, 51, 36, 53, 38, 55, 40, 63, 48, 49, 50, 35, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1362(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 4, 21, 6, 23, 8, 15, 0, 17, 18, 3, 20, 3, 20, 5, 22, 7, 24, 63, 48, 33, 34, 51, 36, 51, 36, 53, 38, 55, 40, 47, 32, 49, 50, 35, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1363(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 4, 21, 6, 23, 8, 31, 0, 17, 18, 3, 20, 3, 20, 5, 22, 7, 24, 47, 48, 33, 34, 51, 36, 51, 36, 53, 38, 55, 40, 63, 32, 49, 50, 35, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1364(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 4, 21, 6, 23, 8, 15, 16, 1, 18, 3, 20, 3, 20, 5, 22, 7, 24, 63, 32, 49, 34, 51, 36, 51, 36, 53, 38, 55, 40, 47, 48, 33, 50, 35, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1365(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 4, 21, 6, 23, 8, 31, 16, 1, 18, 3, 20, 3, 20, 5, 22, 7, 24, 47, 32, 49, 34, 51, 36, 51, 36, 53, 38, 55, 40, 63, 48, 33, 50, 35, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1366(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 4, 21, 6, 23, 8, 15, 0, 1, 18, 3, 20, 3, 20, 5, 22, 7, 24, 63, 48, 49, 34, 51, 36, 51, 36, 53, 38, 55, 40, 47, 32, 33, 50, 35, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1367(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 4, 21, 6, 23, 8, 31, 0, 1, 18, 3, 20, 3, 20, 5, 22, 7, 24, 47, 48, 49, 34, 51, 36, 51, 36, 53, 38, 55, 40, 63, 32, 33, 50, 35, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1368(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 4, 21, 6, 23, 8, 15, 16, 17, 2, 3, 20, 3, 20, 5, 22, 7, 24, 63, 32, 33, 50, 51, 36, 51, 36, 53, 38, 55, 40, 47, 48, 49, 34, 35, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1369(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 4, 21, 6, 23, 8, 31, 16, 17, 2, 3, 20, 3, 20, 5, 22, 7, 24, 47, 32, 33, 50, 51, 36, 51, 36, 53, 38, 55, 40, 63, 48, 49, 34, 35, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1370(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 4, 21, 6, 23, 8, 15, 0, 17, 2, 3, 20, 3, 20, 5, 22, 7, 24, 63, 48, 33, 50, 51, 36, 51, 36, 53, 38, 55, 40, 47, 32, 49, 34, 35, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1371(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 4, 21, 6, 23, 8, 31, 0, 17, 2, 3, 20, 3, 20, 5, 22, 7, 24, 47, 48, 33, 50, 51, 36, 51, 36, 53, 38, 55, 40, 63, 32, 49, 34, 35, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1372(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 4, 21, 6, 23, 8, 15, 16, 1, 2, 3, 20, 3, 20, 5, 22, 7, 24, 63, 32, 49, 50, 51, 36, 51, 36, 53, 38, 55, 40, 47, 48, 33, 34, 35, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1373(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 4, 21, 6, 23, 8, 31, 16, 1, 2, 3, 20, 3, 20, 5, 22, 7, 24, 47, 32, 49, 50, 51, 36, 51, 36, 53, 38, 55, 40, 63, 48, 33, 34, 35, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1374(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 4, 21, 6, 23, 8, 15, 0, 1, 2, 3, 20, 3, 20, 5, 22, 7, 24, 63, 48, 49, 50, 51, 36, 51, 36, 53, 38, 55, 40, 47, 32, 33, 34, 35, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1375(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 4, 21, 6, 23, 8, 31, 0, 1, 2, 3, 20, 3, 20, 5, 22, 7, 24, 47, 48, 49, 50, 51, 36, 51, 36, 53, 38, 55, 40, 63, 32, 33, 34, 35, 52, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1376(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 4, 21, 6, 23, 8, 15, 16, 17, 18, 19, 4, 3, 20, 5, 22, 7, 24, 63, 32, 33, 34, 35, 52, 51, 36, 53, 38, 55, 40, 47, 48, 49, 50, 51, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1377(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 4, 21, 6, 23, 8, 31, 16, 17, 18, 19, 4, 3, 20, 5, 22, 7, 24, 47, 32, 33, 34, 35, 52, 51, 36, 53, 38, 55, 40, 63, 48, 49, 50, 51, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1378(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 4, 21, 6, 23, 8, 15, 0, 17, 18, 19, 4, 3, 20, 5, 22, 7, 24, 63, 48, 33, 34, 35, 52, 51, 36, 53, 38, 55, 40, 47, 32, 49, 50, 51, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1379(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 4, 21, 6, 23, 8, 31, 0, 17, 18, 19, 4, 3, 20, 5, 22, 7, 24, 47, 48, 33, 34, 35, 52, 51, 36, 53, 38, 55, 40, 63, 32, 49, 50, 51, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1380(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 4, 21, 6, 23, 8, 15, 16, 1, 18, 19, 4, 3, 20, 5, 22, 7, 24, 63, 32, 49, 34, 35, 52, 51, 36, 53, 38, 55, 40, 47, 48, 33, 50, 51, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1381(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 4, 21, 6, 23, 8, 31, 16, 1, 18, 19, 4, 3, 20, 5, 22, 7, 24, 47, 32, 49, 34, 35, 52, 51, 36, 53, 38, 55, 40, 63, 48, 33, 50, 51, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1382(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 4, 21, 6, 23, 8, 15, 0, 1, 18, 19, 4, 3, 20, 5, 22, 7, 24, 63, 48, 49, 34, 35, 52, 51, 36, 53, 38, 55, 40, 47, 32, 33, 50, 51, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1383(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 4, 21, 6, 23, 8, 31, 0, 1, 18, 19, 4, 3, 20, 5, 22, 7, 24, 47, 48, 49, 34, 35, 52, 51, 36, 53, 38, 55, 40, 63, 32, 33, 50, 51, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1384(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 4, 21, 6, 23, 8, 15, 16, 17, 2, 19, 4, 3, 20, 5, 22, 7, 24, 63, 32, 33, 50, 35, 52, 51, 36, 53, 38, 55, 40, 47, 48, 49, 34, 51, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1385(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 4, 21, 6, 23, 8, 31, 16, 17, 2, 19, 4, 3, 20, 5, 22, 7, 24, 47, 32, 33, 50, 35, 52, 51, 36, 53, 38, 55, 40, 63, 48, 49, 34, 51, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1386(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 4, 21, 6, 23, 8, 15, 0, 17, 2, 19, 4, 3, 20, 5, 22, 7, 24, 63, 48, 33, 50, 35, 52, 51, 36, 53, 38, 55, 40, 47, 32, 49, 34, 51, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1387(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 4, 21, 6, 23, 8, 31, 0, 17, 2, 19, 4, 3, 20, 5, 22, 7, 24, 47, 48, 33, 50, 35, 52, 51, 36, 53, 38, 55, 40, 63, 32, 49, 34, 51, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1388(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 4, 21, 6, 23, 8, 15, 16, 1, 2, 19, 4, 3, 20, 5, 22, 7, 24, 63, 32, 49, 50, 35, 52, 51, 36, 53, 38, 55, 40, 47, 48, 33, 34, 51, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1389(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 4, 21, 6, 23, 8, 31, 16, 1, 2, 19, 4, 3, 20, 5, 22, 7, 24, 47, 32, 49, 50, 35, 52, 51, 36, 53, 38, 55, 40, 63, 48, 33, 34, 51, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1390(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 4, 21, 6, 23, 8, 15, 0, 1, 2, 19, 4, 3, 20, 5, 22, 7, 24, 63, 48, 49, 50, 35, 52, 51, 36, 53, 38, 55, 40, 47, 32, 33, 34, 51, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1391(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 4, 21, 6, 23, 8, 31, 0, 1, 2, 19, 4, 3, 20, 5, 22, 7, 24, 47, 48, 49, 50, 35, 52, 51, 36, 53, 38, 55, 40, 63, 32, 33, 34, 51, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1392(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 4, 21, 6, 23, 8, 15, 16, 17, 18, 3, 4, 3, 20, 5, 22, 7, 24, 63, 32, 33, 34, 51, 52, 51, 36, 53, 38, 55, 40, 47, 48, 49, 50, 35, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1393(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 4, 21, 6, 23, 8, 31, 16, 17, 18, 3, 4, 3, 20, 5, 22, 7, 24, 47, 32, 33, 34, 51, 52, 51, 36, 53, 38, 55, 40, 63, 48, 49, 50, 35, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1394(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 4, 21, 6, 23, 8, 15, 0, 17, 18, 3, 4, 3, 20, 5, 22, 7, 24, 63, 48, 33, 34, 51, 52, 51, 36, 53, 38, 55, 40, 47, 32, 49, 50, 35, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1395(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 4, 21, 6, 23, 8, 31, 0, 17, 18, 3, 4, 3, 20, 5, 22, 7, 24, 47, 48, 33, 34, 51, 52, 51, 36, 53, 38, 55, 40, 63, 32, 49, 50, 35, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1396(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 4, 21, 6, 23, 8, 15, 16, 1, 18, 3, 4, 3, 20, 5, 22, 7, 24, 63, 32, 49, 34, 51, 52, 51, 36, 53, 38, 55, 40, 47, 48, 33, 50, 35, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1397(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 4, 21, 6, 23, 8, 31, 16, 1, 18, 3, 4, 3, 20, 5, 22, 7, 24, 47, 32, 49, 34, 51, 52, 51, 36, 53, 38, 55, 40, 63, 48, 33, 50, 35, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1398(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 4, 21, 6, 23, 8, 15, 0, 1, 18, 3, 4, 3, 20, 5, 22, 7, 24, 63, 48, 49, 34, 51, 52, 51, 36, 53, 38, 55, 40, 47, 32, 33, 50, 35, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1399(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 4, 21, 6, 23, 8, 31, 0, 1, 18, 3, 4, 3, 20, 5, 22, 7, 24, 47, 48, 49, 34, 51, 52, 51, 36, 53, 38, 55, 40, 63, 32, 33, 50, 35, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1400(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 4, 21, 6, 23, 8, 15, 16, 17, 2, 3, 4, 3, 20, 5, 22, 7, 24, 63, 32, 33, 50, 51, 52, 51, 36, 53, 38, 55, 40, 47, 48, 49, 34, 35, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1401(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 4, 21, 6, 23, 8, 31, 16, 17, 2, 3, 4, 3, 20, 5, 22, 7, 24, 47, 32, 33, 50, 51, 52, 51, 36, 53, 38, 55, 40, 63, 48, 49, 34, 35, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1402(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 4, 21, 6, 23, 8, 15, 0, 17, 2, 3, 4, 3, 20, 5, 22, 7, 24, 63, 48, 33, 50, 51, 52, 51, 36, 53, 38, 55, 40, 47, 32, 49, 34, 35, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1403(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 4, 21, 6, 23, 8, 31, 0, 17, 2, 3, 4, 3, 20, 5, 22, 7, 24, 47, 48, 33, 50, 51, 52, 51, 36, 53, 38, 55, 40, 63, 32, 49, 34, 35, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1404(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 4, 21, 6, 23, 8, 15, 16, 1, 2, 3, 4, 3, 20, 5, 22, 7, 24, 63, 32, 49, 50, 51, 52, 51, 36, 53, 38, 55, 40, 47, 48, 33, 34, 35, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1405(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 4, 21, 6, 23, 8, 31, 16, 1, 2, 3, 4, 3, 20, 5, 22, 7, 24, 47, 32, 49, 50, 51, 52, 51, 36, 53, 38, 55, 40, 63, 48, 33, 34, 35, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1406(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 4, 21, 6, 23, 8, 15, 0, 1, 2, 3, 4, 3, 20, 5, 22, 7, 24, 63, 48, 49, 50, 51, 52, 51, 36, 53, 38, 55, 40, 47, 32, 33, 34, 35, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1407(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 4, 21, 6, 23, 8, 31, 0, 1, 2, 3, 4, 3, 20, 5, 22, 7, 24, 47, 48, 49, 50, 51, 52, 51, 36, 53, 38, 55, 40, 63, 32, 33, 34, 35, 36, 35, 52, 37, 54, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

#endif // DESGPU_COMPILE_ALL_SALTS


