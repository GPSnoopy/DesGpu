#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

#ifdef DESGPU_COMPILE_ALL_SALTS

void des_25_encrypt_salt2048(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 4, 5, 6, 7, 24, 15, 16, 17, 18, 19, 20, 19, 20, 21, 22, 23, 8, 63, 32, 33, 34, 35, 36, 35, 36, 37, 38, 39, 56, 47, 48, 49, 50, 51, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2049(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 4, 5, 6, 7, 24, 31, 16, 17, 18, 19, 20, 19, 20, 21, 22, 23, 8, 47, 32, 33, 34, 35, 36, 35, 36, 37, 38, 39, 56, 63, 48, 49, 50, 51, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2050(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 4, 5, 6, 7, 24, 15, 0, 17, 18, 19, 20, 19, 20, 21, 22, 23, 8, 63, 48, 33, 34, 35, 36, 35, 36, 37, 38, 39, 56, 47, 32, 49, 50, 51, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2051(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 4, 5, 6, 7, 24, 31, 0, 17, 18, 19, 20, 19, 20, 21, 22, 23, 8, 47, 48, 33, 34, 35, 36, 35, 36, 37, 38, 39, 56, 63, 32, 49, 50, 51, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2052(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 4, 5, 6, 7, 24, 15, 16, 1, 18, 19, 20, 19, 20, 21, 22, 23, 8, 63, 32, 49, 34, 35, 36, 35, 36, 37, 38, 39, 56, 47, 48, 33, 50, 51, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2053(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 4, 5, 6, 7, 24, 31, 16, 1, 18, 19, 20, 19, 20, 21, 22, 23, 8, 47, 32, 49, 34, 35, 36, 35, 36, 37, 38, 39, 56, 63, 48, 33, 50, 51, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2054(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 4, 5, 6, 7, 24, 15, 0, 1, 18, 19, 20, 19, 20, 21, 22, 23, 8, 63, 48, 49, 34, 35, 36, 35, 36, 37, 38, 39, 56, 47, 32, 33, 50, 51, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2055(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 4, 5, 6, 7, 24, 31, 0, 1, 18, 19, 20, 19, 20, 21, 22, 23, 8, 47, 48, 49, 34, 35, 36, 35, 36, 37, 38, 39, 56, 63, 32, 33, 50, 51, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2056(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 4, 5, 6, 7, 24, 15, 16, 17, 2, 19, 20, 19, 20, 21, 22, 23, 8, 63, 32, 33, 50, 35, 36, 35, 36, 37, 38, 39, 56, 47, 48, 49, 34, 51, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2057(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 4, 5, 6, 7, 24, 31, 16, 17, 2, 19, 20, 19, 20, 21, 22, 23, 8, 47, 32, 33, 50, 35, 36, 35, 36, 37, 38, 39, 56, 63, 48, 49, 34, 51, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2058(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 4, 5, 6, 7, 24, 15, 0, 17, 2, 19, 20, 19, 20, 21, 22, 23, 8, 63, 48, 33, 50, 35, 36, 35, 36, 37, 38, 39, 56, 47, 32, 49, 34, 51, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2059(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 4, 5, 6, 7, 24, 31, 0, 17, 2, 19, 20, 19, 20, 21, 22, 23, 8, 47, 48, 33, 50, 35, 36, 35, 36, 37, 38, 39, 56, 63, 32, 49, 34, 51, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2060(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 4, 5, 6, 7, 24, 15, 16, 1, 2, 19, 20, 19, 20, 21, 22, 23, 8, 63, 32, 49, 50, 35, 36, 35, 36, 37, 38, 39, 56, 47, 48, 33, 34, 51, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2061(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 4, 5, 6, 7, 24, 31, 16, 1, 2, 19, 20, 19, 20, 21, 22, 23, 8, 47, 32, 49, 50, 35, 36, 35, 36, 37, 38, 39, 56, 63, 48, 33, 34, 51, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2062(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 4, 5, 6, 7, 24, 15, 0, 1, 2, 19, 20, 19, 20, 21, 22, 23, 8, 63, 48, 49, 50, 35, 36, 35, 36, 37, 38, 39, 56, 47, 32, 33, 34, 51, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2063(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 4, 5, 6, 7, 24, 31, 0, 1, 2, 19, 20, 19, 20, 21, 22, 23, 8, 47, 48, 49, 50, 35, 36, 35, 36, 37, 38, 39, 56, 63, 32, 33, 34, 51, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2064(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 4, 5, 6, 7, 24, 15, 16, 17, 18, 3, 20, 19, 20, 21, 22, 23, 8, 63, 32, 33, 34, 51, 36, 35, 36, 37, 38, 39, 56, 47, 48, 49, 50, 35, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2065(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 4, 5, 6, 7, 24, 31, 16, 17, 18, 3, 20, 19, 20, 21, 22, 23, 8, 47, 32, 33, 34, 51, 36, 35, 36, 37, 38, 39, 56, 63, 48, 49, 50, 35, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2066(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 4, 5, 6, 7, 24, 15, 0, 17, 18, 3, 20, 19, 20, 21, 22, 23, 8, 63, 48, 33, 34, 51, 36, 35, 36, 37, 38, 39, 56, 47, 32, 49, 50, 35, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2067(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 4, 5, 6, 7, 24, 31, 0, 17, 18, 3, 20, 19, 20, 21, 22, 23, 8, 47, 48, 33, 34, 51, 36, 35, 36, 37, 38, 39, 56, 63, 32, 49, 50, 35, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2068(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 4, 5, 6, 7, 24, 15, 16, 1, 18, 3, 20, 19, 20, 21, 22, 23, 8, 63, 32, 49, 34, 51, 36, 35, 36, 37, 38, 39, 56, 47, 48, 33, 50, 35, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2069(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 4, 5, 6, 7, 24, 31, 16, 1, 18, 3, 20, 19, 20, 21, 22, 23, 8, 47, 32, 49, 34, 51, 36, 35, 36, 37, 38, 39, 56, 63, 48, 33, 50, 35, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2070(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 4, 5, 6, 7, 24, 15, 0, 1, 18, 3, 20, 19, 20, 21, 22, 23, 8, 63, 48, 49, 34, 51, 36, 35, 36, 37, 38, 39, 56, 47, 32, 33, 50, 35, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2071(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 4, 5, 6, 7, 24, 31, 0, 1, 18, 3, 20, 19, 20, 21, 22, 23, 8, 47, 48, 49, 34, 51, 36, 35, 36, 37, 38, 39, 56, 63, 32, 33, 50, 35, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2072(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 4, 5, 6, 7, 24, 15, 16, 17, 2, 3, 20, 19, 20, 21, 22, 23, 8, 63, 32, 33, 50, 51, 36, 35, 36, 37, 38, 39, 56, 47, 48, 49, 34, 35, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2073(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 4, 5, 6, 7, 24, 31, 16, 17, 2, 3, 20, 19, 20, 21, 22, 23, 8, 47, 32, 33, 50, 51, 36, 35, 36, 37, 38, 39, 56, 63, 48, 49, 34, 35, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2074(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 4, 5, 6, 7, 24, 15, 0, 17, 2, 3, 20, 19, 20, 21, 22, 23, 8, 63, 48, 33, 50, 51, 36, 35, 36, 37, 38, 39, 56, 47, 32, 49, 34, 35, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2075(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 4, 5, 6, 7, 24, 31, 0, 17, 2, 3, 20, 19, 20, 21, 22, 23, 8, 47, 48, 33, 50, 51, 36, 35, 36, 37, 38, 39, 56, 63, 32, 49, 34, 35, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2076(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 4, 5, 6, 7, 24, 15, 16, 1, 2, 3, 20, 19, 20, 21, 22, 23, 8, 63, 32, 49, 50, 51, 36, 35, 36, 37, 38, 39, 56, 47, 48, 33, 34, 35, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2077(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 4, 5, 6, 7, 24, 31, 16, 1, 2, 3, 20, 19, 20, 21, 22, 23, 8, 47, 32, 49, 50, 51, 36, 35, 36, 37, 38, 39, 56, 63, 48, 33, 34, 35, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2078(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 4, 5, 6, 7, 24, 15, 0, 1, 2, 3, 20, 19, 20, 21, 22, 23, 8, 63, 48, 49, 50, 51, 36, 35, 36, 37, 38, 39, 56, 47, 32, 33, 34, 35, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2079(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 4, 5, 6, 7, 24, 31, 0, 1, 2, 3, 20, 19, 20, 21, 22, 23, 8, 47, 48, 49, 50, 51, 36, 35, 36, 37, 38, 39, 56, 63, 32, 33, 34, 35, 52, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2080(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 4, 5, 6, 7, 24, 15, 16, 17, 18, 19, 4, 19, 20, 21, 22, 23, 8, 63, 32, 33, 34, 35, 52, 35, 36, 37, 38, 39, 56, 47, 48, 49, 50, 51, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2081(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 4, 5, 6, 7, 24, 31, 16, 17, 18, 19, 4, 19, 20, 21, 22, 23, 8, 47, 32, 33, 34, 35, 52, 35, 36, 37, 38, 39, 56, 63, 48, 49, 50, 51, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2082(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 4, 5, 6, 7, 24, 15, 0, 17, 18, 19, 4, 19, 20, 21, 22, 23, 8, 63, 48, 33, 34, 35, 52, 35, 36, 37, 38, 39, 56, 47, 32, 49, 50, 51, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2083(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 4, 5, 6, 7, 24, 31, 0, 17, 18, 19, 4, 19, 20, 21, 22, 23, 8, 47, 48, 33, 34, 35, 52, 35, 36, 37, 38, 39, 56, 63, 32, 49, 50, 51, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2084(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 4, 5, 6, 7, 24, 15, 16, 1, 18, 19, 4, 19, 20, 21, 22, 23, 8, 63, 32, 49, 34, 35, 52, 35, 36, 37, 38, 39, 56, 47, 48, 33, 50, 51, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2085(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 4, 5, 6, 7, 24, 31, 16, 1, 18, 19, 4, 19, 20, 21, 22, 23, 8, 47, 32, 49, 34, 35, 52, 35, 36, 37, 38, 39, 56, 63, 48, 33, 50, 51, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2086(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 4, 5, 6, 7, 24, 15, 0, 1, 18, 19, 4, 19, 20, 21, 22, 23, 8, 63, 48, 49, 34, 35, 52, 35, 36, 37, 38, 39, 56, 47, 32, 33, 50, 51, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2087(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 4, 5, 6, 7, 24, 31, 0, 1, 18, 19, 4, 19, 20, 21, 22, 23, 8, 47, 48, 49, 34, 35, 52, 35, 36, 37, 38, 39, 56, 63, 32, 33, 50, 51, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2088(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 4, 5, 6, 7, 24, 15, 16, 17, 2, 19, 4, 19, 20, 21, 22, 23, 8, 63, 32, 33, 50, 35, 52, 35, 36, 37, 38, 39, 56, 47, 48, 49, 34, 51, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2089(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 4, 5, 6, 7, 24, 31, 16, 17, 2, 19, 4, 19, 20, 21, 22, 23, 8, 47, 32, 33, 50, 35, 52, 35, 36, 37, 38, 39, 56, 63, 48, 49, 34, 51, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2090(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 4, 5, 6, 7, 24, 15, 0, 17, 2, 19, 4, 19, 20, 21, 22, 23, 8, 63, 48, 33, 50, 35, 52, 35, 36, 37, 38, 39, 56, 47, 32, 49, 34, 51, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2091(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 4, 5, 6, 7, 24, 31, 0, 17, 2, 19, 4, 19, 20, 21, 22, 23, 8, 47, 48, 33, 50, 35, 52, 35, 36, 37, 38, 39, 56, 63, 32, 49, 34, 51, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2092(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 4, 5, 6, 7, 24, 15, 16, 1, 2, 19, 4, 19, 20, 21, 22, 23, 8, 63, 32, 49, 50, 35, 52, 35, 36, 37, 38, 39, 56, 47, 48, 33, 34, 51, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2093(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 4, 5, 6, 7, 24, 31, 16, 1, 2, 19, 4, 19, 20, 21, 22, 23, 8, 47, 32, 49, 50, 35, 52, 35, 36, 37, 38, 39, 56, 63, 48, 33, 34, 51, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2094(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 4, 5, 6, 7, 24, 15, 0, 1, 2, 19, 4, 19, 20, 21, 22, 23, 8, 63, 48, 49, 50, 35, 52, 35, 36, 37, 38, 39, 56, 47, 32, 33, 34, 51, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2095(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 4, 5, 6, 7, 24, 31, 0, 1, 2, 19, 4, 19, 20, 21, 22, 23, 8, 47, 48, 49, 50, 35, 52, 35, 36, 37, 38, 39, 56, 63, 32, 33, 34, 51, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2096(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 4, 5, 6, 7, 24, 15, 16, 17, 18, 3, 4, 19, 20, 21, 22, 23, 8, 63, 32, 33, 34, 51, 52, 35, 36, 37, 38, 39, 56, 47, 48, 49, 50, 35, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2097(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 4, 5, 6, 7, 24, 31, 16, 17, 18, 3, 4, 19, 20, 21, 22, 23, 8, 47, 32, 33, 34, 51, 52, 35, 36, 37, 38, 39, 56, 63, 48, 49, 50, 35, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2098(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 4, 5, 6, 7, 24, 15, 0, 17, 18, 3, 4, 19, 20, 21, 22, 23, 8, 63, 48, 33, 34, 51, 52, 35, 36, 37, 38, 39, 56, 47, 32, 49, 50, 35, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2099(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 4, 5, 6, 7, 24, 31, 0, 17, 18, 3, 4, 19, 20, 21, 22, 23, 8, 47, 48, 33, 34, 51, 52, 35, 36, 37, 38, 39, 56, 63, 32, 49, 50, 35, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2100(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 4, 5, 6, 7, 24, 15, 16, 1, 18, 3, 4, 19, 20, 21, 22, 23, 8, 63, 32, 49, 34, 51, 52, 35, 36, 37, 38, 39, 56, 47, 48, 33, 50, 35, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2101(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 4, 5, 6, 7, 24, 31, 16, 1, 18, 3, 4, 19, 20, 21, 22, 23, 8, 47, 32, 49, 34, 51, 52, 35, 36, 37, 38, 39, 56, 63, 48, 33, 50, 35, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2102(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 4, 5, 6, 7, 24, 15, 0, 1, 18, 3, 4, 19, 20, 21, 22, 23, 8, 63, 48, 49, 34, 51, 52, 35, 36, 37, 38, 39, 56, 47, 32, 33, 50, 35, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2103(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 4, 5, 6, 7, 24, 31, 0, 1, 18, 3, 4, 19, 20, 21, 22, 23, 8, 47, 48, 49, 34, 51, 52, 35, 36, 37, 38, 39, 56, 63, 32, 33, 50, 35, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2104(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 4, 5, 6, 7, 24, 15, 16, 17, 2, 3, 4, 19, 20, 21, 22, 23, 8, 63, 32, 33, 50, 51, 52, 35, 36, 37, 38, 39, 56, 47, 48, 49, 34, 35, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2105(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 4, 5, 6, 7, 24, 31, 16, 17, 2, 3, 4, 19, 20, 21, 22, 23, 8, 47, 32, 33, 50, 51, 52, 35, 36, 37, 38, 39, 56, 63, 48, 49, 34, 35, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2106(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 4, 5, 6, 7, 24, 15, 0, 17, 2, 3, 4, 19, 20, 21, 22, 23, 8, 63, 48, 33, 50, 51, 52, 35, 36, 37, 38, 39, 56, 47, 32, 49, 34, 35, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2107(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 4, 5, 6, 7, 24, 31, 0, 17, 2, 3, 4, 19, 20, 21, 22, 23, 8, 47, 48, 33, 50, 51, 52, 35, 36, 37, 38, 39, 56, 63, 32, 49, 34, 35, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2108(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 4, 5, 6, 7, 24, 15, 16, 1, 2, 3, 4, 19, 20, 21, 22, 23, 8, 63, 32, 49, 50, 51, 52, 35, 36, 37, 38, 39, 56, 47, 48, 33, 34, 35, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2109(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 4, 5, 6, 7, 24, 31, 16, 1, 2, 3, 4, 19, 20, 21, 22, 23, 8, 47, 32, 49, 50, 51, 52, 35, 36, 37, 38, 39, 56, 63, 48, 33, 34, 35, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2110(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 4, 5, 6, 7, 24, 15, 0, 1, 2, 3, 4, 19, 20, 21, 22, 23, 8, 63, 48, 49, 50, 51, 52, 35, 36, 37, 38, 39, 56, 47, 32, 33, 34, 35, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2111(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 4, 5, 6, 7, 24, 31, 0, 1, 2, 3, 4, 19, 20, 21, 22, 23, 8, 47, 48, 49, 50, 51, 52, 35, 36, 37, 38, 39, 56, 63, 32, 33, 34, 35, 36, 51, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2112(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 4, 5, 6, 7, 24, 15, 16, 17, 18, 19, 20, 3, 20, 21, 22, 23, 8, 63, 32, 33, 34, 35, 36, 51, 36, 37, 38, 39, 56, 47, 48, 49, 50, 51, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2113(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 4, 5, 6, 7, 24, 31, 16, 17, 18, 19, 20, 3, 20, 21, 22, 23, 8, 47, 32, 33, 34, 35, 36, 51, 36, 37, 38, 39, 56, 63, 48, 49, 50, 51, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2114(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 4, 5, 6, 7, 24, 15, 0, 17, 18, 19, 20, 3, 20, 21, 22, 23, 8, 63, 48, 33, 34, 35, 36, 51, 36, 37, 38, 39, 56, 47, 32, 49, 50, 51, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2115(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 4, 5, 6, 7, 24, 31, 0, 17, 18, 19, 20, 3, 20, 21, 22, 23, 8, 47, 48, 33, 34, 35, 36, 51, 36, 37, 38, 39, 56, 63, 32, 49, 50, 51, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2116(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 4, 5, 6, 7, 24, 15, 16, 1, 18, 19, 20, 3, 20, 21, 22, 23, 8, 63, 32, 49, 34, 35, 36, 51, 36, 37, 38, 39, 56, 47, 48, 33, 50, 51, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2117(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 4, 5, 6, 7, 24, 31, 16, 1, 18, 19, 20, 3, 20, 21, 22, 23, 8, 47, 32, 49, 34, 35, 36, 51, 36, 37, 38, 39, 56, 63, 48, 33, 50, 51, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2118(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 4, 5, 6, 7, 24, 15, 0, 1, 18, 19, 20, 3, 20, 21, 22, 23, 8, 63, 48, 49, 34, 35, 36, 51, 36, 37, 38, 39, 56, 47, 32, 33, 50, 51, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2119(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 4, 5, 6, 7, 24, 31, 0, 1, 18, 19, 20, 3, 20, 21, 22, 23, 8, 47, 48, 49, 34, 35, 36, 51, 36, 37, 38, 39, 56, 63, 32, 33, 50, 51, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2120(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 4, 5, 6, 7, 24, 15, 16, 17, 2, 19, 20, 3, 20, 21, 22, 23, 8, 63, 32, 33, 50, 35, 36, 51, 36, 37, 38, 39, 56, 47, 48, 49, 34, 51, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2121(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 4, 5, 6, 7, 24, 31, 16, 17, 2, 19, 20, 3, 20, 21, 22, 23, 8, 47, 32, 33, 50, 35, 36, 51, 36, 37, 38, 39, 56, 63, 48, 49, 34, 51, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2122(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 4, 5, 6, 7, 24, 15, 0, 17, 2, 19, 20, 3, 20, 21, 22, 23, 8, 63, 48, 33, 50, 35, 36, 51, 36, 37, 38, 39, 56, 47, 32, 49, 34, 51, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2123(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 4, 5, 6, 7, 24, 31, 0, 17, 2, 19, 20, 3, 20, 21, 22, 23, 8, 47, 48, 33, 50, 35, 36, 51, 36, 37, 38, 39, 56, 63, 32, 49, 34, 51, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2124(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 4, 5, 6, 7, 24, 15, 16, 1, 2, 19, 20, 3, 20, 21, 22, 23, 8, 63, 32, 49, 50, 35, 36, 51, 36, 37, 38, 39, 56, 47, 48, 33, 34, 51, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2125(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 4, 5, 6, 7, 24, 31, 16, 1, 2, 19, 20, 3, 20, 21, 22, 23, 8, 47, 32, 49, 50, 35, 36, 51, 36, 37, 38, 39, 56, 63, 48, 33, 34, 51, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2126(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 4, 5, 6, 7, 24, 15, 0, 1, 2, 19, 20, 3, 20, 21, 22, 23, 8, 63, 48, 49, 50, 35, 36, 51, 36, 37, 38, 39, 56, 47, 32, 33, 34, 51, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2127(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 4, 5, 6, 7, 24, 31, 0, 1, 2, 19, 20, 3, 20, 21, 22, 23, 8, 47, 48, 49, 50, 35, 36, 51, 36, 37, 38, 39, 56, 63, 32, 33, 34, 51, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2128(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 4, 5, 6, 7, 24, 15, 16, 17, 18, 3, 20, 3, 20, 21, 22, 23, 8, 63, 32, 33, 34, 51, 36, 51, 36, 37, 38, 39, 56, 47, 48, 49, 50, 35, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2129(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 4, 5, 6, 7, 24, 31, 16, 17, 18, 3, 20, 3, 20, 21, 22, 23, 8, 47, 32, 33, 34, 51, 36, 51, 36, 37, 38, 39, 56, 63, 48, 49, 50, 35, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2130(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 4, 5, 6, 7, 24, 15, 0, 17, 18, 3, 20, 3, 20, 21, 22, 23, 8, 63, 48, 33, 34, 51, 36, 51, 36, 37, 38, 39, 56, 47, 32, 49, 50, 35, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2131(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 4, 5, 6, 7, 24, 31, 0, 17, 18, 3, 20, 3, 20, 21, 22, 23, 8, 47, 48, 33, 34, 51, 36, 51, 36, 37, 38, 39, 56, 63, 32, 49, 50, 35, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2132(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 4, 5, 6, 7, 24, 15, 16, 1, 18, 3, 20, 3, 20, 21, 22, 23, 8, 63, 32, 49, 34, 51, 36, 51, 36, 37, 38, 39, 56, 47, 48, 33, 50, 35, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2133(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 4, 5, 6, 7, 24, 31, 16, 1, 18, 3, 20, 3, 20, 21, 22, 23, 8, 47, 32, 49, 34, 51, 36, 51, 36, 37, 38, 39, 56, 63, 48, 33, 50, 35, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2134(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 4, 5, 6, 7, 24, 15, 0, 1, 18, 3, 20, 3, 20, 21, 22, 23, 8, 63, 48, 49, 34, 51, 36, 51, 36, 37, 38, 39, 56, 47, 32, 33, 50, 35, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2135(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 4, 5, 6, 7, 24, 31, 0, 1, 18, 3, 20, 3, 20, 21, 22, 23, 8, 47, 48, 49, 34, 51, 36, 51, 36, 37, 38, 39, 56, 63, 32, 33, 50, 35, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2136(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 4, 5, 6, 7, 24, 15, 16, 17, 2, 3, 20, 3, 20, 21, 22, 23, 8, 63, 32, 33, 50, 51, 36, 51, 36, 37, 38, 39, 56, 47, 48, 49, 34, 35, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2137(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 4, 5, 6, 7, 24, 31, 16, 17, 2, 3, 20, 3, 20, 21, 22, 23, 8, 47, 32, 33, 50, 51, 36, 51, 36, 37, 38, 39, 56, 63, 48, 49, 34, 35, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2138(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 4, 5, 6, 7, 24, 15, 0, 17, 2, 3, 20, 3, 20, 21, 22, 23, 8, 63, 48, 33, 50, 51, 36, 51, 36, 37, 38, 39, 56, 47, 32, 49, 34, 35, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2139(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 4, 5, 6, 7, 24, 31, 0, 17, 2, 3, 20, 3, 20, 21, 22, 23, 8, 47, 48, 33, 50, 51, 36, 51, 36, 37, 38, 39, 56, 63, 32, 49, 34, 35, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2140(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 4, 5, 6, 7, 24, 15, 16, 1, 2, 3, 20, 3, 20, 21, 22, 23, 8, 63, 32, 49, 50, 51, 36, 51, 36, 37, 38, 39, 56, 47, 48, 33, 34, 35, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2141(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 4, 5, 6, 7, 24, 31, 16, 1, 2, 3, 20, 3, 20, 21, 22, 23, 8, 47, 32, 49, 50, 51, 36, 51, 36, 37, 38, 39, 56, 63, 48, 33, 34, 35, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2142(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 4, 5, 6, 7, 24, 15, 0, 1, 2, 3, 20, 3, 20, 21, 22, 23, 8, 63, 48, 49, 50, 51, 36, 51, 36, 37, 38, 39, 56, 47, 32, 33, 34, 35, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2143(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 4, 5, 6, 7, 24, 31, 0, 1, 2, 3, 20, 3, 20, 21, 22, 23, 8, 47, 48, 49, 50, 51, 36, 51, 36, 37, 38, 39, 56, 63, 32, 33, 34, 35, 52, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2144(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 4, 5, 6, 7, 24, 15, 16, 17, 18, 19, 4, 3, 20, 21, 22, 23, 8, 63, 32, 33, 34, 35, 52, 51, 36, 37, 38, 39, 56, 47, 48, 49, 50, 51, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2145(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 4, 5, 6, 7, 24, 31, 16, 17, 18, 19, 4, 3, 20, 21, 22, 23, 8, 47, 32, 33, 34, 35, 52, 51, 36, 37, 38, 39, 56, 63, 48, 49, 50, 51, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2146(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 4, 5, 6, 7, 24, 15, 0, 17, 18, 19, 4, 3, 20, 21, 22, 23, 8, 63, 48, 33, 34, 35, 52, 51, 36, 37, 38, 39, 56, 47, 32, 49, 50, 51, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2147(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 4, 5, 6, 7, 24, 31, 0, 17, 18, 19, 4, 3, 20, 21, 22, 23, 8, 47, 48, 33, 34, 35, 52, 51, 36, 37, 38, 39, 56, 63, 32, 49, 50, 51, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2148(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 4, 5, 6, 7, 24, 15, 16, 1, 18, 19, 4, 3, 20, 21, 22, 23, 8, 63, 32, 49, 34, 35, 52, 51, 36, 37, 38, 39, 56, 47, 48, 33, 50, 51, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2149(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 4, 5, 6, 7, 24, 31, 16, 1, 18, 19, 4, 3, 20, 21, 22, 23, 8, 47, 32, 49, 34, 35, 52, 51, 36, 37, 38, 39, 56, 63, 48, 33, 50, 51, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2150(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 4, 5, 6, 7, 24, 15, 0, 1, 18, 19, 4, 3, 20, 21, 22, 23, 8, 63, 48, 49, 34, 35, 52, 51, 36, 37, 38, 39, 56, 47, 32, 33, 50, 51, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2151(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 4, 5, 6, 7, 24, 31, 0, 1, 18, 19, 4, 3, 20, 21, 22, 23, 8, 47, 48, 49, 34, 35, 52, 51, 36, 37, 38, 39, 56, 63, 32, 33, 50, 51, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2152(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 4, 5, 6, 7, 24, 15, 16, 17, 2, 19, 4, 3, 20, 21, 22, 23, 8, 63, 32, 33, 50, 35, 52, 51, 36, 37, 38, 39, 56, 47, 48, 49, 34, 51, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2153(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 4, 5, 6, 7, 24, 31, 16, 17, 2, 19, 4, 3, 20, 21, 22, 23, 8, 47, 32, 33, 50, 35, 52, 51, 36, 37, 38, 39, 56, 63, 48, 49, 34, 51, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2154(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 4, 5, 6, 7, 24, 15, 0, 17, 2, 19, 4, 3, 20, 21, 22, 23, 8, 63, 48, 33, 50, 35, 52, 51, 36, 37, 38, 39, 56, 47, 32, 49, 34, 51, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2155(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 4, 5, 6, 7, 24, 31, 0, 17, 2, 19, 4, 3, 20, 21, 22, 23, 8, 47, 48, 33, 50, 35, 52, 51, 36, 37, 38, 39, 56, 63, 32, 49, 34, 51, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2156(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 4, 5, 6, 7, 24, 15, 16, 1, 2, 19, 4, 3, 20, 21, 22, 23, 8, 63, 32, 49, 50, 35, 52, 51, 36, 37, 38, 39, 56, 47, 48, 33, 34, 51, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2157(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 4, 5, 6, 7, 24, 31, 16, 1, 2, 19, 4, 3, 20, 21, 22, 23, 8, 47, 32, 49, 50, 35, 52, 51, 36, 37, 38, 39, 56, 63, 48, 33, 34, 51, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2158(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 4, 5, 6, 7, 24, 15, 0, 1, 2, 19, 4, 3, 20, 21, 22, 23, 8, 63, 48, 49, 50, 35, 52, 51, 36, 37, 38, 39, 56, 47, 32, 33, 34, 51, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2159(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 4, 5, 6, 7, 24, 31, 0, 1, 2, 19, 4, 3, 20, 21, 22, 23, 8, 47, 48, 49, 50, 35, 52, 51, 36, 37, 38, 39, 56, 63, 32, 33, 34, 51, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2160(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 4, 5, 6, 7, 24, 15, 16, 17, 18, 3, 4, 3, 20, 21, 22, 23, 8, 63, 32, 33, 34, 51, 52, 51, 36, 37, 38, 39, 56, 47, 48, 49, 50, 35, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2161(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 4, 5, 6, 7, 24, 31, 16, 17, 18, 3, 4, 3, 20, 21, 22, 23, 8, 47, 32, 33, 34, 51, 52, 51, 36, 37, 38, 39, 56, 63, 48, 49, 50, 35, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2162(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 4, 5, 6, 7, 24, 15, 0, 17, 18, 3, 4, 3, 20, 21, 22, 23, 8, 63, 48, 33, 34, 51, 52, 51, 36, 37, 38, 39, 56, 47, 32, 49, 50, 35, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2163(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 4, 5, 6, 7, 24, 31, 0, 17, 18, 3, 4, 3, 20, 21, 22, 23, 8, 47, 48, 33, 34, 51, 52, 51, 36, 37, 38, 39, 56, 63, 32, 49, 50, 35, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2164(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 4, 5, 6, 7, 24, 15, 16, 1, 18, 3, 4, 3, 20, 21, 22, 23, 8, 63, 32, 49, 34, 51, 52, 51, 36, 37, 38, 39, 56, 47, 48, 33, 50, 35, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2165(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 4, 5, 6, 7, 24, 31, 16, 1, 18, 3, 4, 3, 20, 21, 22, 23, 8, 47, 32, 49, 34, 51, 52, 51, 36, 37, 38, 39, 56, 63, 48, 33, 50, 35, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2166(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 4, 5, 6, 7, 24, 15, 0, 1, 18, 3, 4, 3, 20, 21, 22, 23, 8, 63, 48, 49, 34, 51, 52, 51, 36, 37, 38, 39, 56, 47, 32, 33, 50, 35, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2167(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 4, 5, 6, 7, 24, 31, 0, 1, 18, 3, 4, 3, 20, 21, 22, 23, 8, 47, 48, 49, 34, 51, 52, 51, 36, 37, 38, 39, 56, 63, 32, 33, 50, 35, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2168(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 4, 5, 6, 7, 24, 15, 16, 17, 2, 3, 4, 3, 20, 21, 22, 23, 8, 63, 32, 33, 50, 51, 52, 51, 36, 37, 38, 39, 56, 47, 48, 49, 34, 35, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2169(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 4, 5, 6, 7, 24, 31, 16, 17, 2, 3, 4, 3, 20, 21, 22, 23, 8, 47, 32, 33, 50, 51, 52, 51, 36, 37, 38, 39, 56, 63, 48, 49, 34, 35, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2170(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 4, 5, 6, 7, 24, 15, 0, 17, 2, 3, 4, 3, 20, 21, 22, 23, 8, 63, 48, 33, 50, 51, 52, 51, 36, 37, 38, 39, 56, 47, 32, 49, 34, 35, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2171(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 4, 5, 6, 7, 24, 31, 0, 17, 2, 3, 4, 3, 20, 21, 22, 23, 8, 47, 48, 33, 50, 51, 52, 51, 36, 37, 38, 39, 56, 63, 32, 49, 34, 35, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2172(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 4, 5, 6, 7, 24, 15, 16, 1, 2, 3, 4, 3, 20, 21, 22, 23, 8, 63, 32, 49, 50, 51, 52, 51, 36, 37, 38, 39, 56, 47, 48, 33, 34, 35, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2173(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 4, 5, 6, 7, 24, 31, 16, 1, 2, 3, 4, 3, 20, 21, 22, 23, 8, 47, 32, 49, 50, 51, 52, 51, 36, 37, 38, 39, 56, 63, 48, 33, 34, 35, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2174(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 4, 5, 6, 7, 24, 15, 0, 1, 2, 3, 4, 3, 20, 21, 22, 23, 8, 63, 48, 49, 50, 51, 52, 51, 36, 37, 38, 39, 56, 47, 32, 33, 34, 35, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2175(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 4, 5, 6, 7, 24, 31, 0, 1, 2, 3, 4, 3, 20, 21, 22, 23, 8, 47, 48, 49, 50, 51, 52, 51, 36, 37, 38, 39, 56, 63, 32, 33, 34, 35, 36, 35, 52, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

#endif // DESGPU_COMPILE_ALL_SALTS


