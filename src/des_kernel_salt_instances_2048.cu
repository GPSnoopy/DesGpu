#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt2048(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 4, 5, 6, 7, 24, 15, 16, 17, 18, 19, 20, 19, 20, 21, 22, 23, 8, 63, 32, 33, 34, 35, 36, 35, 36, 37, 38, 39, 56, 47, 48, 49, 50, 51, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2049(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 4, 5, 6, 7, 24, 31, 16, 17, 18, 19, 20, 19, 20, 21, 22, 23, 8, 47, 32, 33, 34, 35, 36, 35, 36, 37, 38, 39, 56, 63, 48, 49, 50, 51, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2050(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 4, 5, 6, 7, 24, 15, 0, 17, 18, 19, 20, 19, 20, 21, 22, 23, 8, 63, 48, 33, 34, 35, 36, 35, 36, 37, 38, 39, 56, 47, 32, 49, 50, 51, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2051(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 4, 5, 6, 7, 24, 31, 0, 17, 18, 19, 20, 19, 20, 21, 22, 23, 8, 47, 48, 33, 34, 35, 36, 35, 36, 37, 38, 39, 56, 63, 32, 49, 50, 51, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2052(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 4, 5, 6, 7, 24, 15, 16, 1, 18, 19, 20, 19, 20, 21, 22, 23, 8, 63, 32, 49, 34, 35, 36, 35, 36, 37, 38, 39, 56, 47, 48, 33, 50, 51, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2053(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 4, 5, 6, 7, 24, 31, 16, 1, 18, 19, 20, 19, 20, 21, 22, 23, 8, 47, 32, 49, 34, 35, 36, 35, 36, 37, 38, 39, 56, 63, 48, 33, 50, 51, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2054(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 4, 5, 6, 7, 24, 15, 0, 1, 18, 19, 20, 19, 20, 21, 22, 23, 8, 63, 48, 49, 34, 35, 36, 35, 36, 37, 38, 39, 56, 47, 32, 33, 50, 51, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2055(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 4, 5, 6, 7, 24, 31, 0, 1, 18, 19, 20, 19, 20, 21, 22, 23, 8, 47, 48, 49, 34, 35, 36, 35, 36, 37, 38, 39, 56, 63, 32, 33, 50, 51, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2056(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 4, 5, 6, 7, 24, 15, 16, 17, 2, 19, 20, 19, 20, 21, 22, 23, 8, 63, 32, 33, 50, 35, 36, 35, 36, 37, 38, 39, 56, 47, 48, 49, 34, 51, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2057(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 4, 5, 6, 7, 24, 31, 16, 17, 2, 19, 20, 19, 20, 21, 22, 23, 8, 47, 32, 33, 50, 35, 36, 35, 36, 37, 38, 39, 56, 63, 48, 49, 34, 51, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2058(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 4, 5, 6, 7, 24, 15, 0, 17, 2, 19, 20, 19, 20, 21, 22, 23, 8, 63, 48, 33, 50, 35, 36, 35, 36, 37, 38, 39, 56, 47, 32, 49, 34, 51, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2059(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 4, 5, 6, 7, 24, 31, 0, 17, 2, 19, 20, 19, 20, 21, 22, 23, 8, 47, 48, 33, 50, 35, 36, 35, 36, 37, 38, 39, 56, 63, 32, 49, 34, 51, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2060(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 4, 5, 6, 7, 24, 15, 16, 1, 2, 19, 20, 19, 20, 21, 22, 23, 8, 63, 32, 49, 50, 35, 36, 35, 36, 37, 38, 39, 56, 47, 48, 33, 34, 51, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2061(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 4, 5, 6, 7, 24, 31, 16, 1, 2, 19, 20, 19, 20, 21, 22, 23, 8, 47, 32, 49, 50, 35, 36, 35, 36, 37, 38, 39, 56, 63, 48, 33, 34, 51, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2062(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 4, 5, 6, 7, 24, 15, 0, 1, 2, 19, 20, 19, 20, 21, 22, 23, 8, 63, 48, 49, 50, 35, 36, 35, 36, 37, 38, 39, 56, 47, 32, 33, 34, 51, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2063(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 4, 5, 6, 7, 24, 31, 0, 1, 2, 19, 20, 19, 20, 21, 22, 23, 8, 47, 48, 49, 50, 35, 36, 35, 36, 37, 38, 39, 56, 63, 32, 33, 34, 51, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2064(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 4, 5, 6, 7, 24, 15, 16, 17, 18, 3, 20, 19, 20, 21, 22, 23, 8, 63, 32, 33, 34, 51, 36, 35, 36, 37, 38, 39, 56, 47, 48, 49, 50, 35, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2065(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 4, 5, 6, 7, 24, 31, 16, 17, 18, 3, 20, 19, 20, 21, 22, 23, 8, 47, 32, 33, 34, 51, 36, 35, 36, 37, 38, 39, 56, 63, 48, 49, 50, 35, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2066(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 4, 5, 6, 7, 24, 15, 0, 17, 18, 3, 20, 19, 20, 21, 22, 23, 8, 63, 48, 33, 34, 51, 36, 35, 36, 37, 38, 39, 56, 47, 32, 49, 50, 35, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2067(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 4, 5, 6, 7, 24, 31, 0, 17, 18, 3, 20, 19, 20, 21, 22, 23, 8, 47, 48, 33, 34, 51, 36, 35, 36, 37, 38, 39, 56, 63, 32, 49, 50, 35, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2068(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 4, 5, 6, 7, 24, 15, 16, 1, 18, 3, 20, 19, 20, 21, 22, 23, 8, 63, 32, 49, 34, 51, 36, 35, 36, 37, 38, 39, 56, 47, 48, 33, 50, 35, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2069(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 4, 5, 6, 7, 24, 31, 16, 1, 18, 3, 20, 19, 20, 21, 22, 23, 8, 47, 32, 49, 34, 51, 36, 35, 36, 37, 38, 39, 56, 63, 48, 33, 50, 35, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2070(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 4, 5, 6, 7, 24, 15, 0, 1, 18, 3, 20, 19, 20, 21, 22, 23, 8, 63, 48, 49, 34, 51, 36, 35, 36, 37, 38, 39, 56, 47, 32, 33, 50, 35, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2071(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 4, 5, 6, 7, 24, 31, 0, 1, 18, 3, 20, 19, 20, 21, 22, 23, 8, 47, 48, 49, 34, 51, 36, 35, 36, 37, 38, 39, 56, 63, 32, 33, 50, 35, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2072(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 4, 5, 6, 7, 24, 15, 16, 17, 2, 3, 20, 19, 20, 21, 22, 23, 8, 63, 32, 33, 50, 51, 36, 35, 36, 37, 38, 39, 56, 47, 48, 49, 34, 35, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2073(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 4, 5, 6, 7, 24, 31, 16, 17, 2, 3, 20, 19, 20, 21, 22, 23, 8, 47, 32, 33, 50, 51, 36, 35, 36, 37, 38, 39, 56, 63, 48, 49, 34, 35, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2074(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 4, 5, 6, 7, 24, 15, 0, 17, 2, 3, 20, 19, 20, 21, 22, 23, 8, 63, 48, 33, 50, 51, 36, 35, 36, 37, 38, 39, 56, 47, 32, 49, 34, 35, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2075(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 4, 5, 6, 7, 24, 31, 0, 17, 2, 3, 20, 19, 20, 21, 22, 23, 8, 47, 48, 33, 50, 51, 36, 35, 36, 37, 38, 39, 56, 63, 32, 49, 34, 35, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2076(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 4, 5, 6, 7, 24, 15, 16, 1, 2, 3, 20, 19, 20, 21, 22, 23, 8, 63, 32, 49, 50, 51, 36, 35, 36, 37, 38, 39, 56, 47, 48, 33, 34, 35, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2077(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 4, 5, 6, 7, 24, 31, 16, 1, 2, 3, 20, 19, 20, 21, 22, 23, 8, 47, 32, 49, 50, 51, 36, 35, 36, 37, 38, 39, 56, 63, 48, 33, 34, 35, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2078(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 4, 5, 6, 7, 24, 15, 0, 1, 2, 3, 20, 19, 20, 21, 22, 23, 8, 63, 48, 49, 50, 51, 36, 35, 36, 37, 38, 39, 56, 47, 32, 33, 34, 35, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2079(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 4, 5, 6, 7, 24, 31, 0, 1, 2, 3, 20, 19, 20, 21, 22, 23, 8, 47, 48, 49, 50, 51, 36, 35, 36, 37, 38, 39, 56, 63, 32, 33, 34, 35, 52, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2080(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 4, 5, 6, 7, 24, 15, 16, 17, 18, 19, 4, 19, 20, 21, 22, 23, 8, 63, 32, 33, 34, 35, 52, 35, 36, 37, 38, 39, 56, 47, 48, 49, 50, 51, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2081(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 4, 5, 6, 7, 24, 31, 16, 17, 18, 19, 4, 19, 20, 21, 22, 23, 8, 47, 32, 33, 34, 35, 52, 35, 36, 37, 38, 39, 56, 63, 48, 49, 50, 51, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2082(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 4, 5, 6, 7, 24, 15, 0, 17, 18, 19, 4, 19, 20, 21, 22, 23, 8, 63, 48, 33, 34, 35, 52, 35, 36, 37, 38, 39, 56, 47, 32, 49, 50, 51, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2083(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 4, 5, 6, 7, 24, 31, 0, 17, 18, 19, 4, 19, 20, 21, 22, 23, 8, 47, 48, 33, 34, 35, 52, 35, 36, 37, 38, 39, 56, 63, 32, 49, 50, 51, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2084(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 4, 5, 6, 7, 24, 15, 16, 1, 18, 19, 4, 19, 20, 21, 22, 23, 8, 63, 32, 49, 34, 35, 52, 35, 36, 37, 38, 39, 56, 47, 48, 33, 50, 51, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2085(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 4, 5, 6, 7, 24, 31, 16, 1, 18, 19, 4, 19, 20, 21, 22, 23, 8, 47, 32, 49, 34, 35, 52, 35, 36, 37, 38, 39, 56, 63, 48, 33, 50, 51, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2086(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 4, 5, 6, 7, 24, 15, 0, 1, 18, 19, 4, 19, 20, 21, 22, 23, 8, 63, 48, 49, 34, 35, 52, 35, 36, 37, 38, 39, 56, 47, 32, 33, 50, 51, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2087(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 4, 5, 6, 7, 24, 31, 0, 1, 18, 19, 4, 19, 20, 21, 22, 23, 8, 47, 48, 49, 34, 35, 52, 35, 36, 37, 38, 39, 56, 63, 32, 33, 50, 51, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2088(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 4, 5, 6, 7, 24, 15, 16, 17, 2, 19, 4, 19, 20, 21, 22, 23, 8, 63, 32, 33, 50, 35, 52, 35, 36, 37, 38, 39, 56, 47, 48, 49, 34, 51, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2089(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 4, 5, 6, 7, 24, 31, 16, 17, 2, 19, 4, 19, 20, 21, 22, 23, 8, 47, 32, 33, 50, 35, 52, 35, 36, 37, 38, 39, 56, 63, 48, 49, 34, 51, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2090(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 4, 5, 6, 7, 24, 15, 0, 17, 2, 19, 4, 19, 20, 21, 22, 23, 8, 63, 48, 33, 50, 35, 52, 35, 36, 37, 38, 39, 56, 47, 32, 49, 34, 51, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2091(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 4, 5, 6, 7, 24, 31, 0, 17, 2, 19, 4, 19, 20, 21, 22, 23, 8, 47, 48, 33, 50, 35, 52, 35, 36, 37, 38, 39, 56, 63, 32, 49, 34, 51, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2092(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 4, 5, 6, 7, 24, 15, 16, 1, 2, 19, 4, 19, 20, 21, 22, 23, 8, 63, 32, 49, 50, 35, 52, 35, 36, 37, 38, 39, 56, 47, 48, 33, 34, 51, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2093(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 4, 5, 6, 7, 24, 31, 16, 1, 2, 19, 4, 19, 20, 21, 22, 23, 8, 47, 32, 49, 50, 35, 52, 35, 36, 37, 38, 39, 56, 63, 48, 33, 34, 51, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2094(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 4, 5, 6, 7, 24, 15, 0, 1, 2, 19, 4, 19, 20, 21, 22, 23, 8, 63, 48, 49, 50, 35, 52, 35, 36, 37, 38, 39, 56, 47, 32, 33, 34, 51, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2095(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 4, 5, 6, 7, 24, 31, 0, 1, 2, 19, 4, 19, 20, 21, 22, 23, 8, 47, 48, 49, 50, 35, 52, 35, 36, 37, 38, 39, 56, 63, 32, 33, 34, 51, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2096(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 4, 5, 6, 7, 24, 15, 16, 17, 18, 3, 4, 19, 20, 21, 22, 23, 8, 63, 32, 33, 34, 51, 52, 35, 36, 37, 38, 39, 56, 47, 48, 49, 50, 35, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2097(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 4, 5, 6, 7, 24, 31, 16, 17, 18, 3, 4, 19, 20, 21, 22, 23, 8, 47, 32, 33, 34, 51, 52, 35, 36, 37, 38, 39, 56, 63, 48, 49, 50, 35, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2098(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 4, 5, 6, 7, 24, 15, 0, 17, 18, 3, 4, 19, 20, 21, 22, 23, 8, 63, 48, 33, 34, 51, 52, 35, 36, 37, 38, 39, 56, 47, 32, 49, 50, 35, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2099(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 4, 5, 6, 7, 24, 31, 0, 17, 18, 3, 4, 19, 20, 21, 22, 23, 8, 47, 48, 33, 34, 51, 52, 35, 36, 37, 38, 39, 56, 63, 32, 49, 50, 35, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2100(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 4, 5, 6, 7, 24, 15, 16, 1, 18, 3, 4, 19, 20, 21, 22, 23, 8, 63, 32, 49, 34, 51, 52, 35, 36, 37, 38, 39, 56, 47, 48, 33, 50, 35, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2101(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 4, 5, 6, 7, 24, 31, 16, 1, 18, 3, 4, 19, 20, 21, 22, 23, 8, 47, 32, 49, 34, 51, 52, 35, 36, 37, 38, 39, 56, 63, 48, 33, 50, 35, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2102(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 4, 5, 6, 7, 24, 15, 0, 1, 18, 3, 4, 19, 20, 21, 22, 23, 8, 63, 48, 49, 34, 51, 52, 35, 36, 37, 38, 39, 56, 47, 32, 33, 50, 35, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2103(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 4, 5, 6, 7, 24, 31, 0, 1, 18, 3, 4, 19, 20, 21, 22, 23, 8, 47, 48, 49, 34, 51, 52, 35, 36, 37, 38, 39, 56, 63, 32, 33, 50, 35, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2104(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 4, 5, 6, 7, 24, 15, 16, 17, 2, 3, 4, 19, 20, 21, 22, 23, 8, 63, 32, 33, 50, 51, 52, 35, 36, 37, 38, 39, 56, 47, 48, 49, 34, 35, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2105(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 4, 5, 6, 7, 24, 31, 16, 17, 2, 3, 4, 19, 20, 21, 22, 23, 8, 47, 32, 33, 50, 51, 52, 35, 36, 37, 38, 39, 56, 63, 48, 49, 34, 35, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2106(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 4, 5, 6, 7, 24, 15, 0, 17, 2, 3, 4, 19, 20, 21, 22, 23, 8, 63, 48, 33, 50, 51, 52, 35, 36, 37, 38, 39, 56, 47, 32, 49, 34, 35, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2107(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 4, 5, 6, 7, 24, 31, 0, 17, 2, 3, 4, 19, 20, 21, 22, 23, 8, 47, 48, 33, 50, 51, 52, 35, 36, 37, 38, 39, 56, 63, 32, 49, 34, 35, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2108(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 4, 5, 6, 7, 24, 15, 16, 1, 2, 3, 4, 19, 20, 21, 22, 23, 8, 63, 32, 49, 50, 51, 52, 35, 36, 37, 38, 39, 56, 47, 48, 33, 34, 35, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2109(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 4, 5, 6, 7, 24, 31, 16, 1, 2, 3, 4, 19, 20, 21, 22, 23, 8, 47, 32, 49, 50, 51, 52, 35, 36, 37, 38, 39, 56, 63, 48, 33, 34, 35, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2110(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 4, 5, 6, 7, 24, 15, 0, 1, 2, 3, 4, 19, 20, 21, 22, 23, 8, 63, 48, 49, 50, 51, 52, 35, 36, 37, 38, 39, 56, 47, 32, 33, 34, 35, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2111(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 4, 5, 6, 7, 24, 31, 0, 1, 2, 3, 4, 19, 20, 21, 22, 23, 8, 47, 48, 49, 50, 51, 52, 35, 36, 37, 38, 39, 56, 63, 32, 33, 34, 35, 36, 51, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2112(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 4, 5, 6, 7, 24, 15, 16, 17, 18, 19, 20, 3, 20, 21, 22, 23, 8, 63, 32, 33, 34, 35, 36, 51, 36, 37, 38, 39, 56, 47, 48, 49, 50, 51, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2113(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 4, 5, 6, 7, 24, 31, 16, 17, 18, 19, 20, 3, 20, 21, 22, 23, 8, 47, 32, 33, 34, 35, 36, 51, 36, 37, 38, 39, 56, 63, 48, 49, 50, 51, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2114(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 4, 5, 6, 7, 24, 15, 0, 17, 18, 19, 20, 3, 20, 21, 22, 23, 8, 63, 48, 33, 34, 35, 36, 51, 36, 37, 38, 39, 56, 47, 32, 49, 50, 51, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2115(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 4, 5, 6, 7, 24, 31, 0, 17, 18, 19, 20, 3, 20, 21, 22, 23, 8, 47, 48, 33, 34, 35, 36, 51, 36, 37, 38, 39, 56, 63, 32, 49, 50, 51, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2116(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 4, 5, 6, 7, 24, 15, 16, 1, 18, 19, 20, 3, 20, 21, 22, 23, 8, 63, 32, 49, 34, 35, 36, 51, 36, 37, 38, 39, 56, 47, 48, 33, 50, 51, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2117(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 4, 5, 6, 7, 24, 31, 16, 1, 18, 19, 20, 3, 20, 21, 22, 23, 8, 47, 32, 49, 34, 35, 36, 51, 36, 37, 38, 39, 56, 63, 48, 33, 50, 51, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2118(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 4, 5, 6, 7, 24, 15, 0, 1, 18, 19, 20, 3, 20, 21, 22, 23, 8, 63, 48, 49, 34, 35, 36, 51, 36, 37, 38, 39, 56, 47, 32, 33, 50, 51, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2119(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 4, 5, 6, 7, 24, 31, 0, 1, 18, 19, 20, 3, 20, 21, 22, 23, 8, 47, 48, 49, 34, 35, 36, 51, 36, 37, 38, 39, 56, 63, 32, 33, 50, 51, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2120(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 4, 5, 6, 7, 24, 15, 16, 17, 2, 19, 20, 3, 20, 21, 22, 23, 8, 63, 32, 33, 50, 35, 36, 51, 36, 37, 38, 39, 56, 47, 48, 49, 34, 51, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2121(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 4, 5, 6, 7, 24, 31, 16, 17, 2, 19, 20, 3, 20, 21, 22, 23, 8, 47, 32, 33, 50, 35, 36, 51, 36, 37, 38, 39, 56, 63, 48, 49, 34, 51, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2122(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 4, 5, 6, 7, 24, 15, 0, 17, 2, 19, 20, 3, 20, 21, 22, 23, 8, 63, 48, 33, 50, 35, 36, 51, 36, 37, 38, 39, 56, 47, 32, 49, 34, 51, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2123(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 4, 5, 6, 7, 24, 31, 0, 17, 2, 19, 20, 3, 20, 21, 22, 23, 8, 47, 48, 33, 50, 35, 36, 51, 36, 37, 38, 39, 56, 63, 32, 49, 34, 51, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2124(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 4, 5, 6, 7, 24, 15, 16, 1, 2, 19, 20, 3, 20, 21, 22, 23, 8, 63, 32, 49, 50, 35, 36, 51, 36, 37, 38, 39, 56, 47, 48, 33, 34, 51, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2125(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 4, 5, 6, 7, 24, 31, 16, 1, 2, 19, 20, 3, 20, 21, 22, 23, 8, 47, 32, 49, 50, 35, 36, 51, 36, 37, 38, 39, 56, 63, 48, 33, 34, 51, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2126(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 4, 5, 6, 7, 24, 15, 0, 1, 2, 19, 20, 3, 20, 21, 22, 23, 8, 63, 48, 49, 50, 35, 36, 51, 36, 37, 38, 39, 56, 47, 32, 33, 34, 51, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2127(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 4, 5, 6, 7, 24, 31, 0, 1, 2, 19, 20, 3, 20, 21, 22, 23, 8, 47, 48, 49, 50, 35, 36, 51, 36, 37, 38, 39, 56, 63, 32, 33, 34, 51, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2128(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 4, 5, 6, 7, 24, 15, 16, 17, 18, 3, 20, 3, 20, 21, 22, 23, 8, 63, 32, 33, 34, 51, 36, 51, 36, 37, 38, 39, 56, 47, 48, 49, 50, 35, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2129(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 4, 5, 6, 7, 24, 31, 16, 17, 18, 3, 20, 3, 20, 21, 22, 23, 8, 47, 32, 33, 34, 51, 36, 51, 36, 37, 38, 39, 56, 63, 48, 49, 50, 35, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2130(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 4, 5, 6, 7, 24, 15, 0, 17, 18, 3, 20, 3, 20, 21, 22, 23, 8, 63, 48, 33, 34, 51, 36, 51, 36, 37, 38, 39, 56, 47, 32, 49, 50, 35, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2131(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 4, 5, 6, 7, 24, 31, 0, 17, 18, 3, 20, 3, 20, 21, 22, 23, 8, 47, 48, 33, 34, 51, 36, 51, 36, 37, 38, 39, 56, 63, 32, 49, 50, 35, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2132(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 4, 5, 6, 7, 24, 15, 16, 1, 18, 3, 20, 3, 20, 21, 22, 23, 8, 63, 32, 49, 34, 51, 36, 51, 36, 37, 38, 39, 56, 47, 48, 33, 50, 35, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2133(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 4, 5, 6, 7, 24, 31, 16, 1, 18, 3, 20, 3, 20, 21, 22, 23, 8, 47, 32, 49, 34, 51, 36, 51, 36, 37, 38, 39, 56, 63, 48, 33, 50, 35, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2134(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 4, 5, 6, 7, 24, 15, 0, 1, 18, 3, 20, 3, 20, 21, 22, 23, 8, 63, 48, 49, 34, 51, 36, 51, 36, 37, 38, 39, 56, 47, 32, 33, 50, 35, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2135(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 4, 5, 6, 7, 24, 31, 0, 1, 18, 3, 20, 3, 20, 21, 22, 23, 8, 47, 48, 49, 34, 51, 36, 51, 36, 37, 38, 39, 56, 63, 32, 33, 50, 35, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2136(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 4, 5, 6, 7, 24, 15, 16, 17, 2, 3, 20, 3, 20, 21, 22, 23, 8, 63, 32, 33, 50, 51, 36, 51, 36, 37, 38, 39, 56, 47, 48, 49, 34, 35, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2137(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 4, 5, 6, 7, 24, 31, 16, 17, 2, 3, 20, 3, 20, 21, 22, 23, 8, 47, 32, 33, 50, 51, 36, 51, 36, 37, 38, 39, 56, 63, 48, 49, 34, 35, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2138(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 4, 5, 6, 7, 24, 15, 0, 17, 2, 3, 20, 3, 20, 21, 22, 23, 8, 63, 48, 33, 50, 51, 36, 51, 36, 37, 38, 39, 56, 47, 32, 49, 34, 35, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2139(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 4, 5, 6, 7, 24, 31, 0, 17, 2, 3, 20, 3, 20, 21, 22, 23, 8, 47, 48, 33, 50, 51, 36, 51, 36, 37, 38, 39, 56, 63, 32, 49, 34, 35, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2140(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 4, 5, 6, 7, 24, 15, 16, 1, 2, 3, 20, 3, 20, 21, 22, 23, 8, 63, 32, 49, 50, 51, 36, 51, 36, 37, 38, 39, 56, 47, 48, 33, 34, 35, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2141(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 4, 5, 6, 7, 24, 31, 16, 1, 2, 3, 20, 3, 20, 21, 22, 23, 8, 47, 32, 49, 50, 51, 36, 51, 36, 37, 38, 39, 56, 63, 48, 33, 34, 35, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2142(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 4, 5, 6, 7, 24, 15, 0, 1, 2, 3, 20, 3, 20, 21, 22, 23, 8, 63, 48, 49, 50, 51, 36, 51, 36, 37, 38, 39, 56, 47, 32, 33, 34, 35, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2143(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 4, 5, 6, 7, 24, 31, 0, 1, 2, 3, 20, 3, 20, 21, 22, 23, 8, 47, 48, 49, 50, 51, 36, 51, 36, 37, 38, 39, 56, 63, 32, 33, 34, 35, 52, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2144(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 4, 5, 6, 7, 24, 15, 16, 17, 18, 19, 4, 3, 20, 21, 22, 23, 8, 63, 32, 33, 34, 35, 52, 51, 36, 37, 38, 39, 56, 47, 48, 49, 50, 51, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2145(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 4, 5, 6, 7, 24, 31, 16, 17, 18, 19, 4, 3, 20, 21, 22, 23, 8, 47, 32, 33, 34, 35, 52, 51, 36, 37, 38, 39, 56, 63, 48, 49, 50, 51, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2146(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 4, 5, 6, 7, 24, 15, 0, 17, 18, 19, 4, 3, 20, 21, 22, 23, 8, 63, 48, 33, 34, 35, 52, 51, 36, 37, 38, 39, 56, 47, 32, 49, 50, 51, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2147(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 4, 5, 6, 7, 24, 31, 0, 17, 18, 19, 4, 3, 20, 21, 22, 23, 8, 47, 48, 33, 34, 35, 52, 51, 36, 37, 38, 39, 56, 63, 32, 49, 50, 51, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2148(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 4, 5, 6, 7, 24, 15, 16, 1, 18, 19, 4, 3, 20, 21, 22, 23, 8, 63, 32, 49, 34, 35, 52, 51, 36, 37, 38, 39, 56, 47, 48, 33, 50, 51, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2149(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 4, 5, 6, 7, 24, 31, 16, 1, 18, 19, 4, 3, 20, 21, 22, 23, 8, 47, 32, 49, 34, 35, 52, 51, 36, 37, 38, 39, 56, 63, 48, 33, 50, 51, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2150(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 4, 5, 6, 7, 24, 15, 0, 1, 18, 19, 4, 3, 20, 21, 22, 23, 8, 63, 48, 49, 34, 35, 52, 51, 36, 37, 38, 39, 56, 47, 32, 33, 50, 51, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2151(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 4, 5, 6, 7, 24, 31, 0, 1, 18, 19, 4, 3, 20, 21, 22, 23, 8, 47, 48, 49, 34, 35, 52, 51, 36, 37, 38, 39, 56, 63, 32, 33, 50, 51, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2152(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 4, 5, 6, 7, 24, 15, 16, 17, 2, 19, 4, 3, 20, 21, 22, 23, 8, 63, 32, 33, 50, 35, 52, 51, 36, 37, 38, 39, 56, 47, 48, 49, 34, 51, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2153(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 4, 5, 6, 7, 24, 31, 16, 17, 2, 19, 4, 3, 20, 21, 22, 23, 8, 47, 32, 33, 50, 35, 52, 51, 36, 37, 38, 39, 56, 63, 48, 49, 34, 51, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2154(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 4, 5, 6, 7, 24, 15, 0, 17, 2, 19, 4, 3, 20, 21, 22, 23, 8, 63, 48, 33, 50, 35, 52, 51, 36, 37, 38, 39, 56, 47, 32, 49, 34, 51, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2155(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 4, 5, 6, 7, 24, 31, 0, 17, 2, 19, 4, 3, 20, 21, 22, 23, 8, 47, 48, 33, 50, 35, 52, 51, 36, 37, 38, 39, 56, 63, 32, 49, 34, 51, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2156(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 4, 5, 6, 7, 24, 15, 16, 1, 2, 19, 4, 3, 20, 21, 22, 23, 8, 63, 32, 49, 50, 35, 52, 51, 36, 37, 38, 39, 56, 47, 48, 33, 34, 51, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2157(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 4, 5, 6, 7, 24, 31, 16, 1, 2, 19, 4, 3, 20, 21, 22, 23, 8, 47, 32, 49, 50, 35, 52, 51, 36, 37, 38, 39, 56, 63, 48, 33, 34, 51, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2158(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 4, 5, 6, 7, 24, 15, 0, 1, 2, 19, 4, 3, 20, 21, 22, 23, 8, 63, 48, 49, 50, 35, 52, 51, 36, 37, 38, 39, 56, 47, 32, 33, 34, 51, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2159(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 4, 5, 6, 7, 24, 31, 0, 1, 2, 19, 4, 3, 20, 21, 22, 23, 8, 47, 48, 49, 50, 35, 52, 51, 36, 37, 38, 39, 56, 63, 32, 33, 34, 51, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2160(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 4, 5, 6, 7, 24, 15, 16, 17, 18, 3, 4, 3, 20, 21, 22, 23, 8, 63, 32, 33, 34, 51, 52, 51, 36, 37, 38, 39, 56, 47, 48, 49, 50, 35, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2161(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 4, 5, 6, 7, 24, 31, 16, 17, 18, 3, 4, 3, 20, 21, 22, 23, 8, 47, 32, 33, 34, 51, 52, 51, 36, 37, 38, 39, 56, 63, 48, 49, 50, 35, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2162(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 4, 5, 6, 7, 24, 15, 0, 17, 18, 3, 4, 3, 20, 21, 22, 23, 8, 63, 48, 33, 34, 51, 52, 51, 36, 37, 38, 39, 56, 47, 32, 49, 50, 35, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2163(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 4, 5, 6, 7, 24, 31, 0, 17, 18, 3, 4, 3, 20, 21, 22, 23, 8, 47, 48, 33, 34, 51, 52, 51, 36, 37, 38, 39, 56, 63, 32, 49, 50, 35, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2164(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 4, 5, 6, 7, 24, 15, 16, 1, 18, 3, 4, 3, 20, 21, 22, 23, 8, 63, 32, 49, 34, 51, 52, 51, 36, 37, 38, 39, 56, 47, 48, 33, 50, 35, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2165(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 4, 5, 6, 7, 24, 31, 16, 1, 18, 3, 4, 3, 20, 21, 22, 23, 8, 47, 32, 49, 34, 51, 52, 51, 36, 37, 38, 39, 56, 63, 48, 33, 50, 35, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2166(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 4, 5, 6, 7, 24, 15, 0, 1, 18, 3, 4, 3, 20, 21, 22, 23, 8, 63, 48, 49, 34, 51, 52, 51, 36, 37, 38, 39, 56, 47, 32, 33, 50, 35, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2167(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 4, 5, 6, 7, 24, 31, 0, 1, 18, 3, 4, 3, 20, 21, 22, 23, 8, 47, 48, 49, 34, 51, 52, 51, 36, 37, 38, 39, 56, 63, 32, 33, 50, 35, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2168(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 4, 5, 6, 7, 24, 15, 16, 17, 2, 3, 4, 3, 20, 21, 22, 23, 8, 63, 32, 33, 50, 51, 52, 51, 36, 37, 38, 39, 56, 47, 48, 49, 34, 35, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2169(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 4, 5, 6, 7, 24, 31, 16, 17, 2, 3, 4, 3, 20, 21, 22, 23, 8, 47, 32, 33, 50, 51, 52, 51, 36, 37, 38, 39, 56, 63, 48, 49, 34, 35, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2170(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 4, 5, 6, 7, 24, 15, 0, 17, 2, 3, 4, 3, 20, 21, 22, 23, 8, 63, 48, 33, 50, 51, 52, 51, 36, 37, 38, 39, 56, 47, 32, 49, 34, 35, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2171(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 4, 5, 6, 7, 24, 31, 0, 17, 2, 3, 4, 3, 20, 21, 22, 23, 8, 47, 48, 33, 50, 51, 52, 51, 36, 37, 38, 39, 56, 63, 32, 49, 34, 35, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2172(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 4, 5, 6, 7, 24, 15, 16, 1, 2, 3, 4, 3, 20, 21, 22, 23, 8, 63, 32, 49, 50, 51, 52, 51, 36, 37, 38, 39, 56, 47, 48, 33, 34, 35, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2173(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 4, 5, 6, 7, 24, 31, 16, 1, 2, 3, 4, 3, 20, 21, 22, 23, 8, 47, 32, 49, 50, 51, 52, 51, 36, 37, 38, 39, 56, 63, 48, 33, 34, 35, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2174(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 4, 5, 6, 7, 24, 15, 0, 1, 2, 3, 4, 3, 20, 21, 22, 23, 8, 63, 48, 49, 50, 51, 52, 51, 36, 37, 38, 39, 56, 47, 32, 33, 34, 35, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2175(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 4, 5, 6, 7, 24, 31, 0, 1, 2, 3, 4, 3, 20, 21, 22, 23, 8, 47, 48, 49, 50, 51, 52, 51, 36, 37, 38, 39, 56, 63, 32, 33, 34, 35, 36, 35, 52, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}


