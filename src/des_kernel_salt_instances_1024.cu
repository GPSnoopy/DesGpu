#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt1024(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 4, 5, 6, 23, 8, 15, 16, 17, 18, 19, 20, 19, 20, 21, 22, 7, 24, 63, 32, 33, 34, 35, 36, 35, 36, 37, 38, 55, 40, 47, 48, 49, 50, 51, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1025(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 4, 5, 6, 23, 8, 31, 16, 17, 18, 19, 20, 19, 20, 21, 22, 7, 24, 47, 32, 33, 34, 35, 36, 35, 36, 37, 38, 55, 40, 63, 48, 49, 50, 51, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1026(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 4, 5, 6, 23, 8, 15, 0, 17, 18, 19, 20, 19, 20, 21, 22, 7, 24, 63, 48, 33, 34, 35, 36, 35, 36, 37, 38, 55, 40, 47, 32, 49, 50, 51, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1027(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 4, 5, 6, 23, 8, 31, 0, 17, 18, 19, 20, 19, 20, 21, 22, 7, 24, 47, 48, 33, 34, 35, 36, 35, 36, 37, 38, 55, 40, 63, 32, 49, 50, 51, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1028(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 4, 5, 6, 23, 8, 15, 16, 1, 18, 19, 20, 19, 20, 21, 22, 7, 24, 63, 32, 49, 34, 35, 36, 35, 36, 37, 38, 55, 40, 47, 48, 33, 50, 51, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1029(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 4, 5, 6, 23, 8, 31, 16, 1, 18, 19, 20, 19, 20, 21, 22, 7, 24, 47, 32, 49, 34, 35, 36, 35, 36, 37, 38, 55, 40, 63, 48, 33, 50, 51, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1030(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 4, 5, 6, 23, 8, 15, 0, 1, 18, 19, 20, 19, 20, 21, 22, 7, 24, 63, 48, 49, 34, 35, 36, 35, 36, 37, 38, 55, 40, 47, 32, 33, 50, 51, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1031(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 4, 5, 6, 23, 8, 31, 0, 1, 18, 19, 20, 19, 20, 21, 22, 7, 24, 47, 48, 49, 34, 35, 36, 35, 36, 37, 38, 55, 40, 63, 32, 33, 50, 51, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1032(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 4, 5, 6, 23, 8, 15, 16, 17, 2, 19, 20, 19, 20, 21, 22, 7, 24, 63, 32, 33, 50, 35, 36, 35, 36, 37, 38, 55, 40, 47, 48, 49, 34, 51, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1033(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 4, 5, 6, 23, 8, 31, 16, 17, 2, 19, 20, 19, 20, 21, 22, 7, 24, 47, 32, 33, 50, 35, 36, 35, 36, 37, 38, 55, 40, 63, 48, 49, 34, 51, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1034(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 4, 5, 6, 23, 8, 15, 0, 17, 2, 19, 20, 19, 20, 21, 22, 7, 24, 63, 48, 33, 50, 35, 36, 35, 36, 37, 38, 55, 40, 47, 32, 49, 34, 51, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1035(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 4, 5, 6, 23, 8, 31, 0, 17, 2, 19, 20, 19, 20, 21, 22, 7, 24, 47, 48, 33, 50, 35, 36, 35, 36, 37, 38, 55, 40, 63, 32, 49, 34, 51, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1036(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 4, 5, 6, 23, 8, 15, 16, 1, 2, 19, 20, 19, 20, 21, 22, 7, 24, 63, 32, 49, 50, 35, 36, 35, 36, 37, 38, 55, 40, 47, 48, 33, 34, 51, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1037(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 4, 5, 6, 23, 8, 31, 16, 1, 2, 19, 20, 19, 20, 21, 22, 7, 24, 47, 32, 49, 50, 35, 36, 35, 36, 37, 38, 55, 40, 63, 48, 33, 34, 51, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1038(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 4, 5, 6, 23, 8, 15, 0, 1, 2, 19, 20, 19, 20, 21, 22, 7, 24, 63, 48, 49, 50, 35, 36, 35, 36, 37, 38, 55, 40, 47, 32, 33, 34, 51, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1039(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 4, 5, 6, 23, 8, 31, 0, 1, 2, 19, 20, 19, 20, 21, 22, 7, 24, 47, 48, 49, 50, 35, 36, 35, 36, 37, 38, 55, 40, 63, 32, 33, 34, 51, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1040(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 4, 5, 6, 23, 8, 15, 16, 17, 18, 3, 20, 19, 20, 21, 22, 7, 24, 63, 32, 33, 34, 51, 36, 35, 36, 37, 38, 55, 40, 47, 48, 49, 50, 35, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1041(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 4, 5, 6, 23, 8, 31, 16, 17, 18, 3, 20, 19, 20, 21, 22, 7, 24, 47, 32, 33, 34, 51, 36, 35, 36, 37, 38, 55, 40, 63, 48, 49, 50, 35, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1042(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 4, 5, 6, 23, 8, 15, 0, 17, 18, 3, 20, 19, 20, 21, 22, 7, 24, 63, 48, 33, 34, 51, 36, 35, 36, 37, 38, 55, 40, 47, 32, 49, 50, 35, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1043(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 4, 5, 6, 23, 8, 31, 0, 17, 18, 3, 20, 19, 20, 21, 22, 7, 24, 47, 48, 33, 34, 51, 36, 35, 36, 37, 38, 55, 40, 63, 32, 49, 50, 35, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1044(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 4, 5, 6, 23, 8, 15, 16, 1, 18, 3, 20, 19, 20, 21, 22, 7, 24, 63, 32, 49, 34, 51, 36, 35, 36, 37, 38, 55, 40, 47, 48, 33, 50, 35, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1045(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 4, 5, 6, 23, 8, 31, 16, 1, 18, 3, 20, 19, 20, 21, 22, 7, 24, 47, 32, 49, 34, 51, 36, 35, 36, 37, 38, 55, 40, 63, 48, 33, 50, 35, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1046(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 4, 5, 6, 23, 8, 15, 0, 1, 18, 3, 20, 19, 20, 21, 22, 7, 24, 63, 48, 49, 34, 51, 36, 35, 36, 37, 38, 55, 40, 47, 32, 33, 50, 35, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1047(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 4, 5, 6, 23, 8, 31, 0, 1, 18, 3, 20, 19, 20, 21, 22, 7, 24, 47, 48, 49, 34, 51, 36, 35, 36, 37, 38, 55, 40, 63, 32, 33, 50, 35, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1048(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 4, 5, 6, 23, 8, 15, 16, 17, 2, 3, 20, 19, 20, 21, 22, 7, 24, 63, 32, 33, 50, 51, 36, 35, 36, 37, 38, 55, 40, 47, 48, 49, 34, 35, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1049(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 4, 5, 6, 23, 8, 31, 16, 17, 2, 3, 20, 19, 20, 21, 22, 7, 24, 47, 32, 33, 50, 51, 36, 35, 36, 37, 38, 55, 40, 63, 48, 49, 34, 35, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1050(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 4, 5, 6, 23, 8, 15, 0, 17, 2, 3, 20, 19, 20, 21, 22, 7, 24, 63, 48, 33, 50, 51, 36, 35, 36, 37, 38, 55, 40, 47, 32, 49, 34, 35, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1051(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 4, 5, 6, 23, 8, 31, 0, 17, 2, 3, 20, 19, 20, 21, 22, 7, 24, 47, 48, 33, 50, 51, 36, 35, 36, 37, 38, 55, 40, 63, 32, 49, 34, 35, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1052(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 4, 5, 6, 23, 8, 15, 16, 1, 2, 3, 20, 19, 20, 21, 22, 7, 24, 63, 32, 49, 50, 51, 36, 35, 36, 37, 38, 55, 40, 47, 48, 33, 34, 35, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1053(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 4, 5, 6, 23, 8, 31, 16, 1, 2, 3, 20, 19, 20, 21, 22, 7, 24, 47, 32, 49, 50, 51, 36, 35, 36, 37, 38, 55, 40, 63, 48, 33, 34, 35, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1054(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 4, 5, 6, 23, 8, 15, 0, 1, 2, 3, 20, 19, 20, 21, 22, 7, 24, 63, 48, 49, 50, 51, 36, 35, 36, 37, 38, 55, 40, 47, 32, 33, 34, 35, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1055(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 4, 5, 6, 23, 8, 31, 0, 1, 2, 3, 20, 19, 20, 21, 22, 7, 24, 47, 48, 49, 50, 51, 36, 35, 36, 37, 38, 55, 40, 63, 32, 33, 34, 35, 52, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1056(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 4, 5, 6, 23, 8, 15, 16, 17, 18, 19, 4, 19, 20, 21, 22, 7, 24, 63, 32, 33, 34, 35, 52, 35, 36, 37, 38, 55, 40, 47, 48, 49, 50, 51, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1057(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 4, 5, 6, 23, 8, 31, 16, 17, 18, 19, 4, 19, 20, 21, 22, 7, 24, 47, 32, 33, 34, 35, 52, 35, 36, 37, 38, 55, 40, 63, 48, 49, 50, 51, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1058(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 4, 5, 6, 23, 8, 15, 0, 17, 18, 19, 4, 19, 20, 21, 22, 7, 24, 63, 48, 33, 34, 35, 52, 35, 36, 37, 38, 55, 40, 47, 32, 49, 50, 51, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1059(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 4, 5, 6, 23, 8, 31, 0, 17, 18, 19, 4, 19, 20, 21, 22, 7, 24, 47, 48, 33, 34, 35, 52, 35, 36, 37, 38, 55, 40, 63, 32, 49, 50, 51, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1060(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 4, 5, 6, 23, 8, 15, 16, 1, 18, 19, 4, 19, 20, 21, 22, 7, 24, 63, 32, 49, 34, 35, 52, 35, 36, 37, 38, 55, 40, 47, 48, 33, 50, 51, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1061(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 4, 5, 6, 23, 8, 31, 16, 1, 18, 19, 4, 19, 20, 21, 22, 7, 24, 47, 32, 49, 34, 35, 52, 35, 36, 37, 38, 55, 40, 63, 48, 33, 50, 51, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1062(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 4, 5, 6, 23, 8, 15, 0, 1, 18, 19, 4, 19, 20, 21, 22, 7, 24, 63, 48, 49, 34, 35, 52, 35, 36, 37, 38, 55, 40, 47, 32, 33, 50, 51, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1063(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 4, 5, 6, 23, 8, 31, 0, 1, 18, 19, 4, 19, 20, 21, 22, 7, 24, 47, 48, 49, 34, 35, 52, 35, 36, 37, 38, 55, 40, 63, 32, 33, 50, 51, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1064(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 4, 5, 6, 23, 8, 15, 16, 17, 2, 19, 4, 19, 20, 21, 22, 7, 24, 63, 32, 33, 50, 35, 52, 35, 36, 37, 38, 55, 40, 47, 48, 49, 34, 51, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1065(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 4, 5, 6, 23, 8, 31, 16, 17, 2, 19, 4, 19, 20, 21, 22, 7, 24, 47, 32, 33, 50, 35, 52, 35, 36, 37, 38, 55, 40, 63, 48, 49, 34, 51, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1066(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 4, 5, 6, 23, 8, 15, 0, 17, 2, 19, 4, 19, 20, 21, 22, 7, 24, 63, 48, 33, 50, 35, 52, 35, 36, 37, 38, 55, 40, 47, 32, 49, 34, 51, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1067(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 4, 5, 6, 23, 8, 31, 0, 17, 2, 19, 4, 19, 20, 21, 22, 7, 24, 47, 48, 33, 50, 35, 52, 35, 36, 37, 38, 55, 40, 63, 32, 49, 34, 51, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1068(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 4, 5, 6, 23, 8, 15, 16, 1, 2, 19, 4, 19, 20, 21, 22, 7, 24, 63, 32, 49, 50, 35, 52, 35, 36, 37, 38, 55, 40, 47, 48, 33, 34, 51, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1069(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 4, 5, 6, 23, 8, 31, 16, 1, 2, 19, 4, 19, 20, 21, 22, 7, 24, 47, 32, 49, 50, 35, 52, 35, 36, 37, 38, 55, 40, 63, 48, 33, 34, 51, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1070(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 4, 5, 6, 23, 8, 15, 0, 1, 2, 19, 4, 19, 20, 21, 22, 7, 24, 63, 48, 49, 50, 35, 52, 35, 36, 37, 38, 55, 40, 47, 32, 33, 34, 51, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1071(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 4, 5, 6, 23, 8, 31, 0, 1, 2, 19, 4, 19, 20, 21, 22, 7, 24, 47, 48, 49, 50, 35, 52, 35, 36, 37, 38, 55, 40, 63, 32, 33, 34, 51, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1072(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 4, 5, 6, 23, 8, 15, 16, 17, 18, 3, 4, 19, 20, 21, 22, 7, 24, 63, 32, 33, 34, 51, 52, 35, 36, 37, 38, 55, 40, 47, 48, 49, 50, 35, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1073(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 4, 5, 6, 23, 8, 31, 16, 17, 18, 3, 4, 19, 20, 21, 22, 7, 24, 47, 32, 33, 34, 51, 52, 35, 36, 37, 38, 55, 40, 63, 48, 49, 50, 35, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1074(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 4, 5, 6, 23, 8, 15, 0, 17, 18, 3, 4, 19, 20, 21, 22, 7, 24, 63, 48, 33, 34, 51, 52, 35, 36, 37, 38, 55, 40, 47, 32, 49, 50, 35, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1075(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 4, 5, 6, 23, 8, 31, 0, 17, 18, 3, 4, 19, 20, 21, 22, 7, 24, 47, 48, 33, 34, 51, 52, 35, 36, 37, 38, 55, 40, 63, 32, 49, 50, 35, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1076(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 4, 5, 6, 23, 8, 15, 16, 1, 18, 3, 4, 19, 20, 21, 22, 7, 24, 63, 32, 49, 34, 51, 52, 35, 36, 37, 38, 55, 40, 47, 48, 33, 50, 35, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1077(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 4, 5, 6, 23, 8, 31, 16, 1, 18, 3, 4, 19, 20, 21, 22, 7, 24, 47, 32, 49, 34, 51, 52, 35, 36, 37, 38, 55, 40, 63, 48, 33, 50, 35, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1078(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 4, 5, 6, 23, 8, 15, 0, 1, 18, 3, 4, 19, 20, 21, 22, 7, 24, 63, 48, 49, 34, 51, 52, 35, 36, 37, 38, 55, 40, 47, 32, 33, 50, 35, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1079(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 4, 5, 6, 23, 8, 31, 0, 1, 18, 3, 4, 19, 20, 21, 22, 7, 24, 47, 48, 49, 34, 51, 52, 35, 36, 37, 38, 55, 40, 63, 32, 33, 50, 35, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1080(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 4, 5, 6, 23, 8, 15, 16, 17, 2, 3, 4, 19, 20, 21, 22, 7, 24, 63, 32, 33, 50, 51, 52, 35, 36, 37, 38, 55, 40, 47, 48, 49, 34, 35, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1081(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 4, 5, 6, 23, 8, 31, 16, 17, 2, 3, 4, 19, 20, 21, 22, 7, 24, 47, 32, 33, 50, 51, 52, 35, 36, 37, 38, 55, 40, 63, 48, 49, 34, 35, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1082(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 4, 5, 6, 23, 8, 15, 0, 17, 2, 3, 4, 19, 20, 21, 22, 7, 24, 63, 48, 33, 50, 51, 52, 35, 36, 37, 38, 55, 40, 47, 32, 49, 34, 35, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1083(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 4, 5, 6, 23, 8, 31, 0, 17, 2, 3, 4, 19, 20, 21, 22, 7, 24, 47, 48, 33, 50, 51, 52, 35, 36, 37, 38, 55, 40, 63, 32, 49, 34, 35, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1084(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 4, 5, 6, 23, 8, 15, 16, 1, 2, 3, 4, 19, 20, 21, 22, 7, 24, 63, 32, 49, 50, 51, 52, 35, 36, 37, 38, 55, 40, 47, 48, 33, 34, 35, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1085(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 4, 5, 6, 23, 8, 31, 16, 1, 2, 3, 4, 19, 20, 21, 22, 7, 24, 47, 32, 49, 50, 51, 52, 35, 36, 37, 38, 55, 40, 63, 48, 33, 34, 35, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1086(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 4, 5, 6, 23, 8, 15, 0, 1, 2, 3, 4, 19, 20, 21, 22, 7, 24, 63, 48, 49, 50, 51, 52, 35, 36, 37, 38, 55, 40, 47, 32, 33, 34, 35, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1087(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 4, 5, 6, 23, 8, 31, 0, 1, 2, 3, 4, 19, 20, 21, 22, 7, 24, 47, 48, 49, 50, 51, 52, 35, 36, 37, 38, 55, 40, 63, 32, 33, 34, 35, 36, 51, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1088(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 4, 5, 6, 23, 8, 15, 16, 17, 18, 19, 20, 3, 20, 21, 22, 7, 24, 63, 32, 33, 34, 35, 36, 51, 36, 37, 38, 55, 40, 47, 48, 49, 50, 51, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1089(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 4, 5, 6, 23, 8, 31, 16, 17, 18, 19, 20, 3, 20, 21, 22, 7, 24, 47, 32, 33, 34, 35, 36, 51, 36, 37, 38, 55, 40, 63, 48, 49, 50, 51, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1090(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 4, 5, 6, 23, 8, 15, 0, 17, 18, 19, 20, 3, 20, 21, 22, 7, 24, 63, 48, 33, 34, 35, 36, 51, 36, 37, 38, 55, 40, 47, 32, 49, 50, 51, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1091(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 4, 5, 6, 23, 8, 31, 0, 17, 18, 19, 20, 3, 20, 21, 22, 7, 24, 47, 48, 33, 34, 35, 36, 51, 36, 37, 38, 55, 40, 63, 32, 49, 50, 51, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1092(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 4, 5, 6, 23, 8, 15, 16, 1, 18, 19, 20, 3, 20, 21, 22, 7, 24, 63, 32, 49, 34, 35, 36, 51, 36, 37, 38, 55, 40, 47, 48, 33, 50, 51, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1093(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 4, 5, 6, 23, 8, 31, 16, 1, 18, 19, 20, 3, 20, 21, 22, 7, 24, 47, 32, 49, 34, 35, 36, 51, 36, 37, 38, 55, 40, 63, 48, 33, 50, 51, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1094(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 4, 5, 6, 23, 8, 15, 0, 1, 18, 19, 20, 3, 20, 21, 22, 7, 24, 63, 48, 49, 34, 35, 36, 51, 36, 37, 38, 55, 40, 47, 32, 33, 50, 51, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1095(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 4, 5, 6, 23, 8, 31, 0, 1, 18, 19, 20, 3, 20, 21, 22, 7, 24, 47, 48, 49, 34, 35, 36, 51, 36, 37, 38, 55, 40, 63, 32, 33, 50, 51, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1096(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 4, 5, 6, 23, 8, 15, 16, 17, 2, 19, 20, 3, 20, 21, 22, 7, 24, 63, 32, 33, 50, 35, 36, 51, 36, 37, 38, 55, 40, 47, 48, 49, 34, 51, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1097(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 4, 5, 6, 23, 8, 31, 16, 17, 2, 19, 20, 3, 20, 21, 22, 7, 24, 47, 32, 33, 50, 35, 36, 51, 36, 37, 38, 55, 40, 63, 48, 49, 34, 51, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1098(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 4, 5, 6, 23, 8, 15, 0, 17, 2, 19, 20, 3, 20, 21, 22, 7, 24, 63, 48, 33, 50, 35, 36, 51, 36, 37, 38, 55, 40, 47, 32, 49, 34, 51, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1099(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 4, 5, 6, 23, 8, 31, 0, 17, 2, 19, 20, 3, 20, 21, 22, 7, 24, 47, 48, 33, 50, 35, 36, 51, 36, 37, 38, 55, 40, 63, 32, 49, 34, 51, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1100(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 4, 5, 6, 23, 8, 15, 16, 1, 2, 19, 20, 3, 20, 21, 22, 7, 24, 63, 32, 49, 50, 35, 36, 51, 36, 37, 38, 55, 40, 47, 48, 33, 34, 51, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1101(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 4, 5, 6, 23, 8, 31, 16, 1, 2, 19, 20, 3, 20, 21, 22, 7, 24, 47, 32, 49, 50, 35, 36, 51, 36, 37, 38, 55, 40, 63, 48, 33, 34, 51, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1102(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 4, 5, 6, 23, 8, 15, 0, 1, 2, 19, 20, 3, 20, 21, 22, 7, 24, 63, 48, 49, 50, 35, 36, 51, 36, 37, 38, 55, 40, 47, 32, 33, 34, 51, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1103(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 4, 5, 6, 23, 8, 31, 0, 1, 2, 19, 20, 3, 20, 21, 22, 7, 24, 47, 48, 49, 50, 35, 36, 51, 36, 37, 38, 55, 40, 63, 32, 33, 34, 51, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1104(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 4, 5, 6, 23, 8, 15, 16, 17, 18, 3, 20, 3, 20, 21, 22, 7, 24, 63, 32, 33, 34, 51, 36, 51, 36, 37, 38, 55, 40, 47, 48, 49, 50, 35, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1105(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 4, 5, 6, 23, 8, 31, 16, 17, 18, 3, 20, 3, 20, 21, 22, 7, 24, 47, 32, 33, 34, 51, 36, 51, 36, 37, 38, 55, 40, 63, 48, 49, 50, 35, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1106(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 4, 5, 6, 23, 8, 15, 0, 17, 18, 3, 20, 3, 20, 21, 22, 7, 24, 63, 48, 33, 34, 51, 36, 51, 36, 37, 38, 55, 40, 47, 32, 49, 50, 35, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1107(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 4, 5, 6, 23, 8, 31, 0, 17, 18, 3, 20, 3, 20, 21, 22, 7, 24, 47, 48, 33, 34, 51, 36, 51, 36, 37, 38, 55, 40, 63, 32, 49, 50, 35, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1108(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 4, 5, 6, 23, 8, 15, 16, 1, 18, 3, 20, 3, 20, 21, 22, 7, 24, 63, 32, 49, 34, 51, 36, 51, 36, 37, 38, 55, 40, 47, 48, 33, 50, 35, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1109(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 4, 5, 6, 23, 8, 31, 16, 1, 18, 3, 20, 3, 20, 21, 22, 7, 24, 47, 32, 49, 34, 51, 36, 51, 36, 37, 38, 55, 40, 63, 48, 33, 50, 35, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1110(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 4, 5, 6, 23, 8, 15, 0, 1, 18, 3, 20, 3, 20, 21, 22, 7, 24, 63, 48, 49, 34, 51, 36, 51, 36, 37, 38, 55, 40, 47, 32, 33, 50, 35, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1111(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 4, 5, 6, 23, 8, 31, 0, 1, 18, 3, 20, 3, 20, 21, 22, 7, 24, 47, 48, 49, 34, 51, 36, 51, 36, 37, 38, 55, 40, 63, 32, 33, 50, 35, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1112(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 4, 5, 6, 23, 8, 15, 16, 17, 2, 3, 20, 3, 20, 21, 22, 7, 24, 63, 32, 33, 50, 51, 36, 51, 36, 37, 38, 55, 40, 47, 48, 49, 34, 35, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1113(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 4, 5, 6, 23, 8, 31, 16, 17, 2, 3, 20, 3, 20, 21, 22, 7, 24, 47, 32, 33, 50, 51, 36, 51, 36, 37, 38, 55, 40, 63, 48, 49, 34, 35, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1114(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 4, 5, 6, 23, 8, 15, 0, 17, 2, 3, 20, 3, 20, 21, 22, 7, 24, 63, 48, 33, 50, 51, 36, 51, 36, 37, 38, 55, 40, 47, 32, 49, 34, 35, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1115(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 4, 5, 6, 23, 8, 31, 0, 17, 2, 3, 20, 3, 20, 21, 22, 7, 24, 47, 48, 33, 50, 51, 36, 51, 36, 37, 38, 55, 40, 63, 32, 49, 34, 35, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1116(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 4, 5, 6, 23, 8, 15, 16, 1, 2, 3, 20, 3, 20, 21, 22, 7, 24, 63, 32, 49, 50, 51, 36, 51, 36, 37, 38, 55, 40, 47, 48, 33, 34, 35, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1117(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 4, 5, 6, 23, 8, 31, 16, 1, 2, 3, 20, 3, 20, 21, 22, 7, 24, 47, 32, 49, 50, 51, 36, 51, 36, 37, 38, 55, 40, 63, 48, 33, 34, 35, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1118(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 4, 5, 6, 23, 8, 15, 0, 1, 2, 3, 20, 3, 20, 21, 22, 7, 24, 63, 48, 49, 50, 51, 36, 51, 36, 37, 38, 55, 40, 47, 32, 33, 34, 35, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1119(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 4, 5, 6, 23, 8, 31, 0, 1, 2, 3, 20, 3, 20, 21, 22, 7, 24, 47, 48, 49, 50, 51, 36, 51, 36, 37, 38, 55, 40, 63, 32, 33, 34, 35, 52, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1120(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 4, 5, 6, 23, 8, 15, 16, 17, 18, 19, 4, 3, 20, 21, 22, 7, 24, 63, 32, 33, 34, 35, 52, 51, 36, 37, 38, 55, 40, 47, 48, 49, 50, 51, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1121(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 4, 5, 6, 23, 8, 31, 16, 17, 18, 19, 4, 3, 20, 21, 22, 7, 24, 47, 32, 33, 34, 35, 52, 51, 36, 37, 38, 55, 40, 63, 48, 49, 50, 51, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1122(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 4, 5, 6, 23, 8, 15, 0, 17, 18, 19, 4, 3, 20, 21, 22, 7, 24, 63, 48, 33, 34, 35, 52, 51, 36, 37, 38, 55, 40, 47, 32, 49, 50, 51, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1123(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 4, 5, 6, 23, 8, 31, 0, 17, 18, 19, 4, 3, 20, 21, 22, 7, 24, 47, 48, 33, 34, 35, 52, 51, 36, 37, 38, 55, 40, 63, 32, 49, 50, 51, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1124(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 4, 5, 6, 23, 8, 15, 16, 1, 18, 19, 4, 3, 20, 21, 22, 7, 24, 63, 32, 49, 34, 35, 52, 51, 36, 37, 38, 55, 40, 47, 48, 33, 50, 51, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1125(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 4, 5, 6, 23, 8, 31, 16, 1, 18, 19, 4, 3, 20, 21, 22, 7, 24, 47, 32, 49, 34, 35, 52, 51, 36, 37, 38, 55, 40, 63, 48, 33, 50, 51, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1126(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 4, 5, 6, 23, 8, 15, 0, 1, 18, 19, 4, 3, 20, 21, 22, 7, 24, 63, 48, 49, 34, 35, 52, 51, 36, 37, 38, 55, 40, 47, 32, 33, 50, 51, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1127(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 4, 5, 6, 23, 8, 31, 0, 1, 18, 19, 4, 3, 20, 21, 22, 7, 24, 47, 48, 49, 34, 35, 52, 51, 36, 37, 38, 55, 40, 63, 32, 33, 50, 51, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1128(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 4, 5, 6, 23, 8, 15, 16, 17, 2, 19, 4, 3, 20, 21, 22, 7, 24, 63, 32, 33, 50, 35, 52, 51, 36, 37, 38, 55, 40, 47, 48, 49, 34, 51, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1129(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 4, 5, 6, 23, 8, 31, 16, 17, 2, 19, 4, 3, 20, 21, 22, 7, 24, 47, 32, 33, 50, 35, 52, 51, 36, 37, 38, 55, 40, 63, 48, 49, 34, 51, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1130(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 4, 5, 6, 23, 8, 15, 0, 17, 2, 19, 4, 3, 20, 21, 22, 7, 24, 63, 48, 33, 50, 35, 52, 51, 36, 37, 38, 55, 40, 47, 32, 49, 34, 51, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1131(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 4, 5, 6, 23, 8, 31, 0, 17, 2, 19, 4, 3, 20, 21, 22, 7, 24, 47, 48, 33, 50, 35, 52, 51, 36, 37, 38, 55, 40, 63, 32, 49, 34, 51, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1132(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 4, 5, 6, 23, 8, 15, 16, 1, 2, 19, 4, 3, 20, 21, 22, 7, 24, 63, 32, 49, 50, 35, 52, 51, 36, 37, 38, 55, 40, 47, 48, 33, 34, 51, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1133(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 4, 5, 6, 23, 8, 31, 16, 1, 2, 19, 4, 3, 20, 21, 22, 7, 24, 47, 32, 49, 50, 35, 52, 51, 36, 37, 38, 55, 40, 63, 48, 33, 34, 51, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1134(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 4, 5, 6, 23, 8, 15, 0, 1, 2, 19, 4, 3, 20, 21, 22, 7, 24, 63, 48, 49, 50, 35, 52, 51, 36, 37, 38, 55, 40, 47, 32, 33, 34, 51, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1135(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 4, 5, 6, 23, 8, 31, 0, 1, 2, 19, 4, 3, 20, 21, 22, 7, 24, 47, 48, 49, 50, 35, 52, 51, 36, 37, 38, 55, 40, 63, 32, 33, 34, 51, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1136(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 4, 5, 6, 23, 8, 15, 16, 17, 18, 3, 4, 3, 20, 21, 22, 7, 24, 63, 32, 33, 34, 51, 52, 51, 36, 37, 38, 55, 40, 47, 48, 49, 50, 35, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1137(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 4, 5, 6, 23, 8, 31, 16, 17, 18, 3, 4, 3, 20, 21, 22, 7, 24, 47, 32, 33, 34, 51, 52, 51, 36, 37, 38, 55, 40, 63, 48, 49, 50, 35, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1138(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 4, 5, 6, 23, 8, 15, 0, 17, 18, 3, 4, 3, 20, 21, 22, 7, 24, 63, 48, 33, 34, 51, 52, 51, 36, 37, 38, 55, 40, 47, 32, 49, 50, 35, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1139(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 4, 5, 6, 23, 8, 31, 0, 17, 18, 3, 4, 3, 20, 21, 22, 7, 24, 47, 48, 33, 34, 51, 52, 51, 36, 37, 38, 55, 40, 63, 32, 49, 50, 35, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1140(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 4, 5, 6, 23, 8, 15, 16, 1, 18, 3, 4, 3, 20, 21, 22, 7, 24, 63, 32, 49, 34, 51, 52, 51, 36, 37, 38, 55, 40, 47, 48, 33, 50, 35, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1141(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 4, 5, 6, 23, 8, 31, 16, 1, 18, 3, 4, 3, 20, 21, 22, 7, 24, 47, 32, 49, 34, 51, 52, 51, 36, 37, 38, 55, 40, 63, 48, 33, 50, 35, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1142(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 4, 5, 6, 23, 8, 15, 0, 1, 18, 3, 4, 3, 20, 21, 22, 7, 24, 63, 48, 49, 34, 51, 52, 51, 36, 37, 38, 55, 40, 47, 32, 33, 50, 35, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1143(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 4, 5, 6, 23, 8, 31, 0, 1, 18, 3, 4, 3, 20, 21, 22, 7, 24, 47, 48, 49, 34, 51, 52, 51, 36, 37, 38, 55, 40, 63, 32, 33, 50, 35, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1144(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 4, 5, 6, 23, 8, 15, 16, 17, 2, 3, 4, 3, 20, 21, 22, 7, 24, 63, 32, 33, 50, 51, 52, 51, 36, 37, 38, 55, 40, 47, 48, 49, 34, 35, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1145(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 4, 5, 6, 23, 8, 31, 16, 17, 2, 3, 4, 3, 20, 21, 22, 7, 24, 47, 32, 33, 50, 51, 52, 51, 36, 37, 38, 55, 40, 63, 48, 49, 34, 35, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1146(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 4, 5, 6, 23, 8, 15, 0, 17, 2, 3, 4, 3, 20, 21, 22, 7, 24, 63, 48, 33, 50, 51, 52, 51, 36, 37, 38, 55, 40, 47, 32, 49, 34, 35, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1147(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 4, 5, 6, 23, 8, 31, 0, 17, 2, 3, 4, 3, 20, 21, 22, 7, 24, 47, 48, 33, 50, 51, 52, 51, 36, 37, 38, 55, 40, 63, 32, 49, 34, 35, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1148(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 4, 5, 6, 23, 8, 15, 16, 1, 2, 3, 4, 3, 20, 21, 22, 7, 24, 63, 32, 49, 50, 51, 52, 51, 36, 37, 38, 55, 40, 47, 48, 33, 34, 35, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1149(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 4, 5, 6, 23, 8, 31, 16, 1, 2, 3, 4, 3, 20, 21, 22, 7, 24, 47, 32, 49, 50, 51, 52, 51, 36, 37, 38, 55, 40, 63, 48, 33, 34, 35, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1150(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 4, 5, 6, 23, 8, 15, 0, 1, 2, 3, 4, 3, 20, 21, 22, 7, 24, 63, 48, 49, 50, 51, 52, 51, 36, 37, 38, 55, 40, 47, 32, 33, 34, 35, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1151(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 4, 5, 6, 23, 8, 31, 0, 1, 2, 3, 4, 3, 20, 21, 22, 7, 24, 47, 48, 49, 50, 51, 52, 51, 36, 37, 38, 55, 40, 63, 32, 33, 34, 35, 36, 35, 52, 53, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}


