#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

#ifdef DESGPU_COMPILE_ALL_SALTS

void des_25_encrypt_salt768(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 4, 21, 22, 7, 8, 15, 16, 17, 18, 19, 20, 19, 20, 5, 6, 23, 24, 63, 32, 33, 34, 35, 36, 35, 36, 53, 54, 39, 40, 47, 48, 49, 50, 51, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt769(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 4, 21, 22, 7, 8, 31, 16, 17, 18, 19, 20, 19, 20, 5, 6, 23, 24, 47, 32, 33, 34, 35, 36, 35, 36, 53, 54, 39, 40, 63, 48, 49, 50, 51, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt770(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 4, 21, 22, 7, 8, 15, 0, 17, 18, 19, 20, 19, 20, 5, 6, 23, 24, 63, 48, 33, 34, 35, 36, 35, 36, 53, 54, 39, 40, 47, 32, 49, 50, 51, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt771(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 4, 21, 22, 7, 8, 31, 0, 17, 18, 19, 20, 19, 20, 5, 6, 23, 24, 47, 48, 33, 34, 35, 36, 35, 36, 53, 54, 39, 40, 63, 32, 49, 50, 51, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt772(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 4, 21, 22, 7, 8, 15, 16, 1, 18, 19, 20, 19, 20, 5, 6, 23, 24, 63, 32, 49, 34, 35, 36, 35, 36, 53, 54, 39, 40, 47, 48, 33, 50, 51, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt773(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 4, 21, 22, 7, 8, 31, 16, 1, 18, 19, 20, 19, 20, 5, 6, 23, 24, 47, 32, 49, 34, 35, 36, 35, 36, 53, 54, 39, 40, 63, 48, 33, 50, 51, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt774(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 4, 21, 22, 7, 8, 15, 0, 1, 18, 19, 20, 19, 20, 5, 6, 23, 24, 63, 48, 49, 34, 35, 36, 35, 36, 53, 54, 39, 40, 47, 32, 33, 50, 51, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt775(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 4, 21, 22, 7, 8, 31, 0, 1, 18, 19, 20, 19, 20, 5, 6, 23, 24, 47, 48, 49, 34, 35, 36, 35, 36, 53, 54, 39, 40, 63, 32, 33, 50, 51, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt776(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 4, 21, 22, 7, 8, 15, 16, 17, 2, 19, 20, 19, 20, 5, 6, 23, 24, 63, 32, 33, 50, 35, 36, 35, 36, 53, 54, 39, 40, 47, 48, 49, 34, 51, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt777(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 4, 21, 22, 7, 8, 31, 16, 17, 2, 19, 20, 19, 20, 5, 6, 23, 24, 47, 32, 33, 50, 35, 36, 35, 36, 53, 54, 39, 40, 63, 48, 49, 34, 51, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt778(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 4, 21, 22, 7, 8, 15, 0, 17, 2, 19, 20, 19, 20, 5, 6, 23, 24, 63, 48, 33, 50, 35, 36, 35, 36, 53, 54, 39, 40, 47, 32, 49, 34, 51, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt779(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 4, 21, 22, 7, 8, 31, 0, 17, 2, 19, 20, 19, 20, 5, 6, 23, 24, 47, 48, 33, 50, 35, 36, 35, 36, 53, 54, 39, 40, 63, 32, 49, 34, 51, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt780(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 4, 21, 22, 7, 8, 15, 16, 1, 2, 19, 20, 19, 20, 5, 6, 23, 24, 63, 32, 49, 50, 35, 36, 35, 36, 53, 54, 39, 40, 47, 48, 33, 34, 51, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt781(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 4, 21, 22, 7, 8, 31, 16, 1, 2, 19, 20, 19, 20, 5, 6, 23, 24, 47, 32, 49, 50, 35, 36, 35, 36, 53, 54, 39, 40, 63, 48, 33, 34, 51, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt782(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 4, 21, 22, 7, 8, 15, 0, 1, 2, 19, 20, 19, 20, 5, 6, 23, 24, 63, 48, 49, 50, 35, 36, 35, 36, 53, 54, 39, 40, 47, 32, 33, 34, 51, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt783(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 4, 21, 22, 7, 8, 31, 0, 1, 2, 19, 20, 19, 20, 5, 6, 23, 24, 47, 48, 49, 50, 35, 36, 35, 36, 53, 54, 39, 40, 63, 32, 33, 34, 51, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt784(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 4, 21, 22, 7, 8, 15, 16, 17, 18, 3, 20, 19, 20, 5, 6, 23, 24, 63, 32, 33, 34, 51, 36, 35, 36, 53, 54, 39, 40, 47, 48, 49, 50, 35, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt785(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 4, 21, 22, 7, 8, 31, 16, 17, 18, 3, 20, 19, 20, 5, 6, 23, 24, 47, 32, 33, 34, 51, 36, 35, 36, 53, 54, 39, 40, 63, 48, 49, 50, 35, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt786(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 4, 21, 22, 7, 8, 15, 0, 17, 18, 3, 20, 19, 20, 5, 6, 23, 24, 63, 48, 33, 34, 51, 36, 35, 36, 53, 54, 39, 40, 47, 32, 49, 50, 35, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt787(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 4, 21, 22, 7, 8, 31, 0, 17, 18, 3, 20, 19, 20, 5, 6, 23, 24, 47, 48, 33, 34, 51, 36, 35, 36, 53, 54, 39, 40, 63, 32, 49, 50, 35, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt788(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 4, 21, 22, 7, 8, 15, 16, 1, 18, 3, 20, 19, 20, 5, 6, 23, 24, 63, 32, 49, 34, 51, 36, 35, 36, 53, 54, 39, 40, 47, 48, 33, 50, 35, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt789(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 4, 21, 22, 7, 8, 31, 16, 1, 18, 3, 20, 19, 20, 5, 6, 23, 24, 47, 32, 49, 34, 51, 36, 35, 36, 53, 54, 39, 40, 63, 48, 33, 50, 35, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt790(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 4, 21, 22, 7, 8, 15, 0, 1, 18, 3, 20, 19, 20, 5, 6, 23, 24, 63, 48, 49, 34, 51, 36, 35, 36, 53, 54, 39, 40, 47, 32, 33, 50, 35, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt791(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 4, 21, 22, 7, 8, 31, 0, 1, 18, 3, 20, 19, 20, 5, 6, 23, 24, 47, 48, 49, 34, 51, 36, 35, 36, 53, 54, 39, 40, 63, 32, 33, 50, 35, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt792(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 4, 21, 22, 7, 8, 15, 16, 17, 2, 3, 20, 19, 20, 5, 6, 23, 24, 63, 32, 33, 50, 51, 36, 35, 36, 53, 54, 39, 40, 47, 48, 49, 34, 35, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt793(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 4, 21, 22, 7, 8, 31, 16, 17, 2, 3, 20, 19, 20, 5, 6, 23, 24, 47, 32, 33, 50, 51, 36, 35, 36, 53, 54, 39, 40, 63, 48, 49, 34, 35, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt794(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 4, 21, 22, 7, 8, 15, 0, 17, 2, 3, 20, 19, 20, 5, 6, 23, 24, 63, 48, 33, 50, 51, 36, 35, 36, 53, 54, 39, 40, 47, 32, 49, 34, 35, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt795(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 4, 21, 22, 7, 8, 31, 0, 17, 2, 3, 20, 19, 20, 5, 6, 23, 24, 47, 48, 33, 50, 51, 36, 35, 36, 53, 54, 39, 40, 63, 32, 49, 34, 35, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt796(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 4, 21, 22, 7, 8, 15, 16, 1, 2, 3, 20, 19, 20, 5, 6, 23, 24, 63, 32, 49, 50, 51, 36, 35, 36, 53, 54, 39, 40, 47, 48, 33, 34, 35, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt797(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 4, 21, 22, 7, 8, 31, 16, 1, 2, 3, 20, 19, 20, 5, 6, 23, 24, 47, 32, 49, 50, 51, 36, 35, 36, 53, 54, 39, 40, 63, 48, 33, 34, 35, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt798(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 4, 21, 22, 7, 8, 15, 0, 1, 2, 3, 20, 19, 20, 5, 6, 23, 24, 63, 48, 49, 50, 51, 36, 35, 36, 53, 54, 39, 40, 47, 32, 33, 34, 35, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt799(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 4, 21, 22, 7, 8, 31, 0, 1, 2, 3, 20, 19, 20, 5, 6, 23, 24, 47, 48, 49, 50, 51, 36, 35, 36, 53, 54, 39, 40, 63, 32, 33, 34, 35, 52, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt800(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 4, 21, 22, 7, 8, 15, 16, 17, 18, 19, 4, 19, 20, 5, 6, 23, 24, 63, 32, 33, 34, 35, 52, 35, 36, 53, 54, 39, 40, 47, 48, 49, 50, 51, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt801(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 4, 21, 22, 7, 8, 31, 16, 17, 18, 19, 4, 19, 20, 5, 6, 23, 24, 47, 32, 33, 34, 35, 52, 35, 36, 53, 54, 39, 40, 63, 48, 49, 50, 51, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt802(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 4, 21, 22, 7, 8, 15, 0, 17, 18, 19, 4, 19, 20, 5, 6, 23, 24, 63, 48, 33, 34, 35, 52, 35, 36, 53, 54, 39, 40, 47, 32, 49, 50, 51, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt803(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 4, 21, 22, 7, 8, 31, 0, 17, 18, 19, 4, 19, 20, 5, 6, 23, 24, 47, 48, 33, 34, 35, 52, 35, 36, 53, 54, 39, 40, 63, 32, 49, 50, 51, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt804(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 4, 21, 22, 7, 8, 15, 16, 1, 18, 19, 4, 19, 20, 5, 6, 23, 24, 63, 32, 49, 34, 35, 52, 35, 36, 53, 54, 39, 40, 47, 48, 33, 50, 51, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt805(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 4, 21, 22, 7, 8, 31, 16, 1, 18, 19, 4, 19, 20, 5, 6, 23, 24, 47, 32, 49, 34, 35, 52, 35, 36, 53, 54, 39, 40, 63, 48, 33, 50, 51, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt806(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 4, 21, 22, 7, 8, 15, 0, 1, 18, 19, 4, 19, 20, 5, 6, 23, 24, 63, 48, 49, 34, 35, 52, 35, 36, 53, 54, 39, 40, 47, 32, 33, 50, 51, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt807(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 4, 21, 22, 7, 8, 31, 0, 1, 18, 19, 4, 19, 20, 5, 6, 23, 24, 47, 48, 49, 34, 35, 52, 35, 36, 53, 54, 39, 40, 63, 32, 33, 50, 51, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt808(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 4, 21, 22, 7, 8, 15, 16, 17, 2, 19, 4, 19, 20, 5, 6, 23, 24, 63, 32, 33, 50, 35, 52, 35, 36, 53, 54, 39, 40, 47, 48, 49, 34, 51, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt809(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 4, 21, 22, 7, 8, 31, 16, 17, 2, 19, 4, 19, 20, 5, 6, 23, 24, 47, 32, 33, 50, 35, 52, 35, 36, 53, 54, 39, 40, 63, 48, 49, 34, 51, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt810(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 4, 21, 22, 7, 8, 15, 0, 17, 2, 19, 4, 19, 20, 5, 6, 23, 24, 63, 48, 33, 50, 35, 52, 35, 36, 53, 54, 39, 40, 47, 32, 49, 34, 51, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt811(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 4, 21, 22, 7, 8, 31, 0, 17, 2, 19, 4, 19, 20, 5, 6, 23, 24, 47, 48, 33, 50, 35, 52, 35, 36, 53, 54, 39, 40, 63, 32, 49, 34, 51, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt812(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 4, 21, 22, 7, 8, 15, 16, 1, 2, 19, 4, 19, 20, 5, 6, 23, 24, 63, 32, 49, 50, 35, 52, 35, 36, 53, 54, 39, 40, 47, 48, 33, 34, 51, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt813(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 4, 21, 22, 7, 8, 31, 16, 1, 2, 19, 4, 19, 20, 5, 6, 23, 24, 47, 32, 49, 50, 35, 52, 35, 36, 53, 54, 39, 40, 63, 48, 33, 34, 51, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt814(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 4, 21, 22, 7, 8, 15, 0, 1, 2, 19, 4, 19, 20, 5, 6, 23, 24, 63, 48, 49, 50, 35, 52, 35, 36, 53, 54, 39, 40, 47, 32, 33, 34, 51, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt815(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 4, 21, 22, 7, 8, 31, 0, 1, 2, 19, 4, 19, 20, 5, 6, 23, 24, 47, 48, 49, 50, 35, 52, 35, 36, 53, 54, 39, 40, 63, 32, 33, 34, 51, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt816(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 4, 21, 22, 7, 8, 15, 16, 17, 18, 3, 4, 19, 20, 5, 6, 23, 24, 63, 32, 33, 34, 51, 52, 35, 36, 53, 54, 39, 40, 47, 48, 49, 50, 35, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt817(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 4, 21, 22, 7, 8, 31, 16, 17, 18, 3, 4, 19, 20, 5, 6, 23, 24, 47, 32, 33, 34, 51, 52, 35, 36, 53, 54, 39, 40, 63, 48, 49, 50, 35, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt818(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 4, 21, 22, 7, 8, 15, 0, 17, 18, 3, 4, 19, 20, 5, 6, 23, 24, 63, 48, 33, 34, 51, 52, 35, 36, 53, 54, 39, 40, 47, 32, 49, 50, 35, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt819(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 4, 21, 22, 7, 8, 31, 0, 17, 18, 3, 4, 19, 20, 5, 6, 23, 24, 47, 48, 33, 34, 51, 52, 35, 36, 53, 54, 39, 40, 63, 32, 49, 50, 35, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt820(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 4, 21, 22, 7, 8, 15, 16, 1, 18, 3, 4, 19, 20, 5, 6, 23, 24, 63, 32, 49, 34, 51, 52, 35, 36, 53, 54, 39, 40, 47, 48, 33, 50, 35, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt821(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 4, 21, 22, 7, 8, 31, 16, 1, 18, 3, 4, 19, 20, 5, 6, 23, 24, 47, 32, 49, 34, 51, 52, 35, 36, 53, 54, 39, 40, 63, 48, 33, 50, 35, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt822(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 4, 21, 22, 7, 8, 15, 0, 1, 18, 3, 4, 19, 20, 5, 6, 23, 24, 63, 48, 49, 34, 51, 52, 35, 36, 53, 54, 39, 40, 47, 32, 33, 50, 35, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt823(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 4, 21, 22, 7, 8, 31, 0, 1, 18, 3, 4, 19, 20, 5, 6, 23, 24, 47, 48, 49, 34, 51, 52, 35, 36, 53, 54, 39, 40, 63, 32, 33, 50, 35, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt824(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 4, 21, 22, 7, 8, 15, 16, 17, 2, 3, 4, 19, 20, 5, 6, 23, 24, 63, 32, 33, 50, 51, 52, 35, 36, 53, 54, 39, 40, 47, 48, 49, 34, 35, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt825(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 4, 21, 22, 7, 8, 31, 16, 17, 2, 3, 4, 19, 20, 5, 6, 23, 24, 47, 32, 33, 50, 51, 52, 35, 36, 53, 54, 39, 40, 63, 48, 49, 34, 35, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt826(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 4, 21, 22, 7, 8, 15, 0, 17, 2, 3, 4, 19, 20, 5, 6, 23, 24, 63, 48, 33, 50, 51, 52, 35, 36, 53, 54, 39, 40, 47, 32, 49, 34, 35, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt827(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 4, 21, 22, 7, 8, 31, 0, 17, 2, 3, 4, 19, 20, 5, 6, 23, 24, 47, 48, 33, 50, 51, 52, 35, 36, 53, 54, 39, 40, 63, 32, 49, 34, 35, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt828(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 4, 21, 22, 7, 8, 15, 16, 1, 2, 3, 4, 19, 20, 5, 6, 23, 24, 63, 32, 49, 50, 51, 52, 35, 36, 53, 54, 39, 40, 47, 48, 33, 34, 35, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt829(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 4, 21, 22, 7, 8, 31, 16, 1, 2, 3, 4, 19, 20, 5, 6, 23, 24, 47, 32, 49, 50, 51, 52, 35, 36, 53, 54, 39, 40, 63, 48, 33, 34, 35, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt830(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 4, 21, 22, 7, 8, 15, 0, 1, 2, 3, 4, 19, 20, 5, 6, 23, 24, 63, 48, 49, 50, 51, 52, 35, 36, 53, 54, 39, 40, 47, 32, 33, 34, 35, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt831(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 4, 21, 22, 7, 8, 31, 0, 1, 2, 3, 4, 19, 20, 5, 6, 23, 24, 47, 48, 49, 50, 51, 52, 35, 36, 53, 54, 39, 40, 63, 32, 33, 34, 35, 36, 51, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt832(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 4, 21, 22, 7, 8, 15, 16, 17, 18, 19, 20, 3, 20, 5, 6, 23, 24, 63, 32, 33, 34, 35, 36, 51, 36, 53, 54, 39, 40, 47, 48, 49, 50, 51, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt833(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 4, 21, 22, 7, 8, 31, 16, 17, 18, 19, 20, 3, 20, 5, 6, 23, 24, 47, 32, 33, 34, 35, 36, 51, 36, 53, 54, 39, 40, 63, 48, 49, 50, 51, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt834(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 4, 21, 22, 7, 8, 15, 0, 17, 18, 19, 20, 3, 20, 5, 6, 23, 24, 63, 48, 33, 34, 35, 36, 51, 36, 53, 54, 39, 40, 47, 32, 49, 50, 51, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt835(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 4, 21, 22, 7, 8, 31, 0, 17, 18, 19, 20, 3, 20, 5, 6, 23, 24, 47, 48, 33, 34, 35, 36, 51, 36, 53, 54, 39, 40, 63, 32, 49, 50, 51, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt836(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 4, 21, 22, 7, 8, 15, 16, 1, 18, 19, 20, 3, 20, 5, 6, 23, 24, 63, 32, 49, 34, 35, 36, 51, 36, 53, 54, 39, 40, 47, 48, 33, 50, 51, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt837(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 4, 21, 22, 7, 8, 31, 16, 1, 18, 19, 20, 3, 20, 5, 6, 23, 24, 47, 32, 49, 34, 35, 36, 51, 36, 53, 54, 39, 40, 63, 48, 33, 50, 51, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt838(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 4, 21, 22, 7, 8, 15, 0, 1, 18, 19, 20, 3, 20, 5, 6, 23, 24, 63, 48, 49, 34, 35, 36, 51, 36, 53, 54, 39, 40, 47, 32, 33, 50, 51, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt839(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 4, 21, 22, 7, 8, 31, 0, 1, 18, 19, 20, 3, 20, 5, 6, 23, 24, 47, 48, 49, 34, 35, 36, 51, 36, 53, 54, 39, 40, 63, 32, 33, 50, 51, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt840(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 4, 21, 22, 7, 8, 15, 16, 17, 2, 19, 20, 3, 20, 5, 6, 23, 24, 63, 32, 33, 50, 35, 36, 51, 36, 53, 54, 39, 40, 47, 48, 49, 34, 51, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt841(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 4, 21, 22, 7, 8, 31, 16, 17, 2, 19, 20, 3, 20, 5, 6, 23, 24, 47, 32, 33, 50, 35, 36, 51, 36, 53, 54, 39, 40, 63, 48, 49, 34, 51, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt842(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 4, 21, 22, 7, 8, 15, 0, 17, 2, 19, 20, 3, 20, 5, 6, 23, 24, 63, 48, 33, 50, 35, 36, 51, 36, 53, 54, 39, 40, 47, 32, 49, 34, 51, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt843(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 4, 21, 22, 7, 8, 31, 0, 17, 2, 19, 20, 3, 20, 5, 6, 23, 24, 47, 48, 33, 50, 35, 36, 51, 36, 53, 54, 39, 40, 63, 32, 49, 34, 51, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt844(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 4, 21, 22, 7, 8, 15, 16, 1, 2, 19, 20, 3, 20, 5, 6, 23, 24, 63, 32, 49, 50, 35, 36, 51, 36, 53, 54, 39, 40, 47, 48, 33, 34, 51, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt845(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 4, 21, 22, 7, 8, 31, 16, 1, 2, 19, 20, 3, 20, 5, 6, 23, 24, 47, 32, 49, 50, 35, 36, 51, 36, 53, 54, 39, 40, 63, 48, 33, 34, 51, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt846(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 4, 21, 22, 7, 8, 15, 0, 1, 2, 19, 20, 3, 20, 5, 6, 23, 24, 63, 48, 49, 50, 35, 36, 51, 36, 53, 54, 39, 40, 47, 32, 33, 34, 51, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt847(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 4, 21, 22, 7, 8, 31, 0, 1, 2, 19, 20, 3, 20, 5, 6, 23, 24, 47, 48, 49, 50, 35, 36, 51, 36, 53, 54, 39, 40, 63, 32, 33, 34, 51, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt848(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 4, 21, 22, 7, 8, 15, 16, 17, 18, 3, 20, 3, 20, 5, 6, 23, 24, 63, 32, 33, 34, 51, 36, 51, 36, 53, 54, 39, 40, 47, 48, 49, 50, 35, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt849(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 4, 21, 22, 7, 8, 31, 16, 17, 18, 3, 20, 3, 20, 5, 6, 23, 24, 47, 32, 33, 34, 51, 36, 51, 36, 53, 54, 39, 40, 63, 48, 49, 50, 35, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt850(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 4, 21, 22, 7, 8, 15, 0, 17, 18, 3, 20, 3, 20, 5, 6, 23, 24, 63, 48, 33, 34, 51, 36, 51, 36, 53, 54, 39, 40, 47, 32, 49, 50, 35, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt851(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 4, 21, 22, 7, 8, 31, 0, 17, 18, 3, 20, 3, 20, 5, 6, 23, 24, 47, 48, 33, 34, 51, 36, 51, 36, 53, 54, 39, 40, 63, 32, 49, 50, 35, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt852(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 4, 21, 22, 7, 8, 15, 16, 1, 18, 3, 20, 3, 20, 5, 6, 23, 24, 63, 32, 49, 34, 51, 36, 51, 36, 53, 54, 39, 40, 47, 48, 33, 50, 35, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt853(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 4, 21, 22, 7, 8, 31, 16, 1, 18, 3, 20, 3, 20, 5, 6, 23, 24, 47, 32, 49, 34, 51, 36, 51, 36, 53, 54, 39, 40, 63, 48, 33, 50, 35, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt854(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 4, 21, 22, 7, 8, 15, 0, 1, 18, 3, 20, 3, 20, 5, 6, 23, 24, 63, 48, 49, 34, 51, 36, 51, 36, 53, 54, 39, 40, 47, 32, 33, 50, 35, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt855(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 4, 21, 22, 7, 8, 31, 0, 1, 18, 3, 20, 3, 20, 5, 6, 23, 24, 47, 48, 49, 34, 51, 36, 51, 36, 53, 54, 39, 40, 63, 32, 33, 50, 35, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt856(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 4, 21, 22, 7, 8, 15, 16, 17, 2, 3, 20, 3, 20, 5, 6, 23, 24, 63, 32, 33, 50, 51, 36, 51, 36, 53, 54, 39, 40, 47, 48, 49, 34, 35, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt857(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 4, 21, 22, 7, 8, 31, 16, 17, 2, 3, 20, 3, 20, 5, 6, 23, 24, 47, 32, 33, 50, 51, 36, 51, 36, 53, 54, 39, 40, 63, 48, 49, 34, 35, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt858(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 4, 21, 22, 7, 8, 15, 0, 17, 2, 3, 20, 3, 20, 5, 6, 23, 24, 63, 48, 33, 50, 51, 36, 51, 36, 53, 54, 39, 40, 47, 32, 49, 34, 35, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt859(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 4, 21, 22, 7, 8, 31, 0, 17, 2, 3, 20, 3, 20, 5, 6, 23, 24, 47, 48, 33, 50, 51, 36, 51, 36, 53, 54, 39, 40, 63, 32, 49, 34, 35, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt860(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 4, 21, 22, 7, 8, 15, 16, 1, 2, 3, 20, 3, 20, 5, 6, 23, 24, 63, 32, 49, 50, 51, 36, 51, 36, 53, 54, 39, 40, 47, 48, 33, 34, 35, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt861(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 4, 21, 22, 7, 8, 31, 16, 1, 2, 3, 20, 3, 20, 5, 6, 23, 24, 47, 32, 49, 50, 51, 36, 51, 36, 53, 54, 39, 40, 63, 48, 33, 34, 35, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt862(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 4, 21, 22, 7, 8, 15, 0, 1, 2, 3, 20, 3, 20, 5, 6, 23, 24, 63, 48, 49, 50, 51, 36, 51, 36, 53, 54, 39, 40, 47, 32, 33, 34, 35, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt863(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 4, 21, 22, 7, 8, 31, 0, 1, 2, 3, 20, 3, 20, 5, 6, 23, 24, 47, 48, 49, 50, 51, 36, 51, 36, 53, 54, 39, 40, 63, 32, 33, 34, 35, 52, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt864(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 4, 21, 22, 7, 8, 15, 16, 17, 18, 19, 4, 3, 20, 5, 6, 23, 24, 63, 32, 33, 34, 35, 52, 51, 36, 53, 54, 39, 40, 47, 48, 49, 50, 51, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt865(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 4, 21, 22, 7, 8, 31, 16, 17, 18, 19, 4, 3, 20, 5, 6, 23, 24, 47, 32, 33, 34, 35, 52, 51, 36, 53, 54, 39, 40, 63, 48, 49, 50, 51, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt866(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 4, 21, 22, 7, 8, 15, 0, 17, 18, 19, 4, 3, 20, 5, 6, 23, 24, 63, 48, 33, 34, 35, 52, 51, 36, 53, 54, 39, 40, 47, 32, 49, 50, 51, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt867(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 4, 21, 22, 7, 8, 31, 0, 17, 18, 19, 4, 3, 20, 5, 6, 23, 24, 47, 48, 33, 34, 35, 52, 51, 36, 53, 54, 39, 40, 63, 32, 49, 50, 51, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt868(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 4, 21, 22, 7, 8, 15, 16, 1, 18, 19, 4, 3, 20, 5, 6, 23, 24, 63, 32, 49, 34, 35, 52, 51, 36, 53, 54, 39, 40, 47, 48, 33, 50, 51, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt869(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 4, 21, 22, 7, 8, 31, 16, 1, 18, 19, 4, 3, 20, 5, 6, 23, 24, 47, 32, 49, 34, 35, 52, 51, 36, 53, 54, 39, 40, 63, 48, 33, 50, 51, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt870(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 4, 21, 22, 7, 8, 15, 0, 1, 18, 19, 4, 3, 20, 5, 6, 23, 24, 63, 48, 49, 34, 35, 52, 51, 36, 53, 54, 39, 40, 47, 32, 33, 50, 51, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt871(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 4, 21, 22, 7, 8, 31, 0, 1, 18, 19, 4, 3, 20, 5, 6, 23, 24, 47, 48, 49, 34, 35, 52, 51, 36, 53, 54, 39, 40, 63, 32, 33, 50, 51, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt872(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 4, 21, 22, 7, 8, 15, 16, 17, 2, 19, 4, 3, 20, 5, 6, 23, 24, 63, 32, 33, 50, 35, 52, 51, 36, 53, 54, 39, 40, 47, 48, 49, 34, 51, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt873(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 4, 21, 22, 7, 8, 31, 16, 17, 2, 19, 4, 3, 20, 5, 6, 23, 24, 47, 32, 33, 50, 35, 52, 51, 36, 53, 54, 39, 40, 63, 48, 49, 34, 51, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt874(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 4, 21, 22, 7, 8, 15, 0, 17, 2, 19, 4, 3, 20, 5, 6, 23, 24, 63, 48, 33, 50, 35, 52, 51, 36, 53, 54, 39, 40, 47, 32, 49, 34, 51, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt875(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 4, 21, 22, 7, 8, 31, 0, 17, 2, 19, 4, 3, 20, 5, 6, 23, 24, 47, 48, 33, 50, 35, 52, 51, 36, 53, 54, 39, 40, 63, 32, 49, 34, 51, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt876(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 4, 21, 22, 7, 8, 15, 16, 1, 2, 19, 4, 3, 20, 5, 6, 23, 24, 63, 32, 49, 50, 35, 52, 51, 36, 53, 54, 39, 40, 47, 48, 33, 34, 51, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt877(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 4, 21, 22, 7, 8, 31, 16, 1, 2, 19, 4, 3, 20, 5, 6, 23, 24, 47, 32, 49, 50, 35, 52, 51, 36, 53, 54, 39, 40, 63, 48, 33, 34, 51, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt878(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 4, 21, 22, 7, 8, 15, 0, 1, 2, 19, 4, 3, 20, 5, 6, 23, 24, 63, 48, 49, 50, 35, 52, 51, 36, 53, 54, 39, 40, 47, 32, 33, 34, 51, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt879(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 4, 21, 22, 7, 8, 31, 0, 1, 2, 19, 4, 3, 20, 5, 6, 23, 24, 47, 48, 49, 50, 35, 52, 51, 36, 53, 54, 39, 40, 63, 32, 33, 34, 51, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt880(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 4, 21, 22, 7, 8, 15, 16, 17, 18, 3, 4, 3, 20, 5, 6, 23, 24, 63, 32, 33, 34, 51, 52, 51, 36, 53, 54, 39, 40, 47, 48, 49, 50, 35, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt881(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 4, 21, 22, 7, 8, 31, 16, 17, 18, 3, 4, 3, 20, 5, 6, 23, 24, 47, 32, 33, 34, 51, 52, 51, 36, 53, 54, 39, 40, 63, 48, 49, 50, 35, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt882(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 4, 21, 22, 7, 8, 15, 0, 17, 18, 3, 4, 3, 20, 5, 6, 23, 24, 63, 48, 33, 34, 51, 52, 51, 36, 53, 54, 39, 40, 47, 32, 49, 50, 35, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt883(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 4, 21, 22, 7, 8, 31, 0, 17, 18, 3, 4, 3, 20, 5, 6, 23, 24, 47, 48, 33, 34, 51, 52, 51, 36, 53, 54, 39, 40, 63, 32, 49, 50, 35, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt884(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 4, 21, 22, 7, 8, 15, 16, 1, 18, 3, 4, 3, 20, 5, 6, 23, 24, 63, 32, 49, 34, 51, 52, 51, 36, 53, 54, 39, 40, 47, 48, 33, 50, 35, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt885(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 4, 21, 22, 7, 8, 31, 16, 1, 18, 3, 4, 3, 20, 5, 6, 23, 24, 47, 32, 49, 34, 51, 52, 51, 36, 53, 54, 39, 40, 63, 48, 33, 50, 35, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt886(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 4, 21, 22, 7, 8, 15, 0, 1, 18, 3, 4, 3, 20, 5, 6, 23, 24, 63, 48, 49, 34, 51, 52, 51, 36, 53, 54, 39, 40, 47, 32, 33, 50, 35, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt887(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 4, 21, 22, 7, 8, 31, 0, 1, 18, 3, 4, 3, 20, 5, 6, 23, 24, 47, 48, 49, 34, 51, 52, 51, 36, 53, 54, 39, 40, 63, 32, 33, 50, 35, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt888(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 4, 21, 22, 7, 8, 15, 16, 17, 2, 3, 4, 3, 20, 5, 6, 23, 24, 63, 32, 33, 50, 51, 52, 51, 36, 53, 54, 39, 40, 47, 48, 49, 34, 35, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt889(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 4, 21, 22, 7, 8, 31, 16, 17, 2, 3, 4, 3, 20, 5, 6, 23, 24, 47, 32, 33, 50, 51, 52, 51, 36, 53, 54, 39, 40, 63, 48, 49, 34, 35, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt890(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 4, 21, 22, 7, 8, 15, 0, 17, 2, 3, 4, 3, 20, 5, 6, 23, 24, 63, 48, 33, 50, 51, 52, 51, 36, 53, 54, 39, 40, 47, 32, 49, 34, 35, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt891(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 4, 21, 22, 7, 8, 31, 0, 17, 2, 3, 4, 3, 20, 5, 6, 23, 24, 47, 48, 33, 50, 51, 52, 51, 36, 53, 54, 39, 40, 63, 32, 49, 34, 35, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt892(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 4, 21, 22, 7, 8, 15, 16, 1, 2, 3, 4, 3, 20, 5, 6, 23, 24, 63, 32, 49, 50, 51, 52, 51, 36, 53, 54, 39, 40, 47, 48, 33, 34, 35, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt893(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 4, 21, 22, 7, 8, 31, 16, 1, 2, 3, 4, 3, 20, 5, 6, 23, 24, 47, 32, 49, 50, 51, 52, 51, 36, 53, 54, 39, 40, 63, 48, 33, 34, 35, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt894(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 4, 21, 22, 7, 8, 15, 0, 1, 2, 3, 4, 3, 20, 5, 6, 23, 24, 63, 48, 49, 50, 51, 52, 51, 36, 53, 54, 39, 40, 47, 32, 33, 34, 35, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt895(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 4, 21, 22, 7, 8, 31, 0, 1, 2, 3, 4, 3, 20, 5, 6, 23, 24, 47, 48, 49, 50, 51, 52, 51, 36, 53, 54, 39, 40, 63, 32, 33, 34, 35, 36, 35, 52, 37, 38, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

#endif // DESGPU_COMPILE_ALL_SALTS


