#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt768(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 4, 21, 22, 7, 8, 15, 16, 17, 18, 19, 20, 19, 20, 5, 6, 23, 24, 63, 32, 33, 34, 35, 36, 35, 36, 53, 54, 39, 40, 47, 48, 49, 50, 51, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt769(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 4, 21, 22, 7, 8, 31, 16, 17, 18, 19, 20, 19, 20, 5, 6, 23, 24, 47, 32, 33, 34, 35, 36, 35, 36, 53, 54, 39, 40, 63, 48, 49, 50, 51, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt770(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 4, 21, 22, 7, 8, 15, 0, 17, 18, 19, 20, 19, 20, 5, 6, 23, 24, 63, 48, 33, 34, 35, 36, 35, 36, 53, 54, 39, 40, 47, 32, 49, 50, 51, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt771(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 4, 21, 22, 7, 8, 31, 0, 17, 18, 19, 20, 19, 20, 5, 6, 23, 24, 47, 48, 33, 34, 35, 36, 35, 36, 53, 54, 39, 40, 63, 32, 49, 50, 51, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt772(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 4, 21, 22, 7, 8, 15, 16, 1, 18, 19, 20, 19, 20, 5, 6, 23, 24, 63, 32, 49, 34, 35, 36, 35, 36, 53, 54, 39, 40, 47, 48, 33, 50, 51, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt773(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 4, 21, 22, 7, 8, 31, 16, 1, 18, 19, 20, 19, 20, 5, 6, 23, 24, 47, 32, 49, 34, 35, 36, 35, 36, 53, 54, 39, 40, 63, 48, 33, 50, 51, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt774(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 4, 21, 22, 7, 8, 15, 0, 1, 18, 19, 20, 19, 20, 5, 6, 23, 24, 63, 48, 49, 34, 35, 36, 35, 36, 53, 54, 39, 40, 47, 32, 33, 50, 51, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt775(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 4, 21, 22, 7, 8, 31, 0, 1, 18, 19, 20, 19, 20, 5, 6, 23, 24, 47, 48, 49, 34, 35, 36, 35, 36, 53, 54, 39, 40, 63, 32, 33, 50, 51, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt776(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 4, 21, 22, 7, 8, 15, 16, 17, 2, 19, 20, 19, 20, 5, 6, 23, 24, 63, 32, 33, 50, 35, 36, 35, 36, 53, 54, 39, 40, 47, 48, 49, 34, 51, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt777(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 4, 21, 22, 7, 8, 31, 16, 17, 2, 19, 20, 19, 20, 5, 6, 23, 24, 47, 32, 33, 50, 35, 36, 35, 36, 53, 54, 39, 40, 63, 48, 49, 34, 51, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt778(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 4, 21, 22, 7, 8, 15, 0, 17, 2, 19, 20, 19, 20, 5, 6, 23, 24, 63, 48, 33, 50, 35, 36, 35, 36, 53, 54, 39, 40, 47, 32, 49, 34, 51, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt779(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 4, 21, 22, 7, 8, 31, 0, 17, 2, 19, 20, 19, 20, 5, 6, 23, 24, 47, 48, 33, 50, 35, 36, 35, 36, 53, 54, 39, 40, 63, 32, 49, 34, 51, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt780(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 4, 21, 22, 7, 8, 15, 16, 1, 2, 19, 20, 19, 20, 5, 6, 23, 24, 63, 32, 49, 50, 35, 36, 35, 36, 53, 54, 39, 40, 47, 48, 33, 34, 51, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt781(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 4, 21, 22, 7, 8, 31, 16, 1, 2, 19, 20, 19, 20, 5, 6, 23, 24, 47, 32, 49, 50, 35, 36, 35, 36, 53, 54, 39, 40, 63, 48, 33, 34, 51, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt782(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 4, 21, 22, 7, 8, 15, 0, 1, 2, 19, 20, 19, 20, 5, 6, 23, 24, 63, 48, 49, 50, 35, 36, 35, 36, 53, 54, 39, 40, 47, 32, 33, 34, 51, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt783(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 4, 21, 22, 7, 8, 31, 0, 1, 2, 19, 20, 19, 20, 5, 6, 23, 24, 47, 48, 49, 50, 35, 36, 35, 36, 53, 54, 39, 40, 63, 32, 33, 34, 51, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt784(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 4, 21, 22, 7, 8, 15, 16, 17, 18, 3, 20, 19, 20, 5, 6, 23, 24, 63, 32, 33, 34, 51, 36, 35, 36, 53, 54, 39, 40, 47, 48, 49, 50, 35, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt785(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 4, 21, 22, 7, 8, 31, 16, 17, 18, 3, 20, 19, 20, 5, 6, 23, 24, 47, 32, 33, 34, 51, 36, 35, 36, 53, 54, 39, 40, 63, 48, 49, 50, 35, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt786(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 4, 21, 22, 7, 8, 15, 0, 17, 18, 3, 20, 19, 20, 5, 6, 23, 24, 63, 48, 33, 34, 51, 36, 35, 36, 53, 54, 39, 40, 47, 32, 49, 50, 35, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt787(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 4, 21, 22, 7, 8, 31, 0, 17, 18, 3, 20, 19, 20, 5, 6, 23, 24, 47, 48, 33, 34, 51, 36, 35, 36, 53, 54, 39, 40, 63, 32, 49, 50, 35, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt788(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 4, 21, 22, 7, 8, 15, 16, 1, 18, 3, 20, 19, 20, 5, 6, 23, 24, 63, 32, 49, 34, 51, 36, 35, 36, 53, 54, 39, 40, 47, 48, 33, 50, 35, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt789(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 4, 21, 22, 7, 8, 31, 16, 1, 18, 3, 20, 19, 20, 5, 6, 23, 24, 47, 32, 49, 34, 51, 36, 35, 36, 53, 54, 39, 40, 63, 48, 33, 50, 35, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt790(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 4, 21, 22, 7, 8, 15, 0, 1, 18, 3, 20, 19, 20, 5, 6, 23, 24, 63, 48, 49, 34, 51, 36, 35, 36, 53, 54, 39, 40, 47, 32, 33, 50, 35, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt791(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 4, 21, 22, 7, 8, 31, 0, 1, 18, 3, 20, 19, 20, 5, 6, 23, 24, 47, 48, 49, 34, 51, 36, 35, 36, 53, 54, 39, 40, 63, 32, 33, 50, 35, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt792(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 4, 21, 22, 7, 8, 15, 16, 17, 2, 3, 20, 19, 20, 5, 6, 23, 24, 63, 32, 33, 50, 51, 36, 35, 36, 53, 54, 39, 40, 47, 48, 49, 34, 35, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt793(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 4, 21, 22, 7, 8, 31, 16, 17, 2, 3, 20, 19, 20, 5, 6, 23, 24, 47, 32, 33, 50, 51, 36, 35, 36, 53, 54, 39, 40, 63, 48, 49, 34, 35, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt794(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 4, 21, 22, 7, 8, 15, 0, 17, 2, 3, 20, 19, 20, 5, 6, 23, 24, 63, 48, 33, 50, 51, 36, 35, 36, 53, 54, 39, 40, 47, 32, 49, 34, 35, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt795(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 4, 21, 22, 7, 8, 31, 0, 17, 2, 3, 20, 19, 20, 5, 6, 23, 24, 47, 48, 33, 50, 51, 36, 35, 36, 53, 54, 39, 40, 63, 32, 49, 34, 35, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt796(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 4, 21, 22, 7, 8, 15, 16, 1, 2, 3, 20, 19, 20, 5, 6, 23, 24, 63, 32, 49, 50, 51, 36, 35, 36, 53, 54, 39, 40, 47, 48, 33, 34, 35, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt797(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 4, 21, 22, 7, 8, 31, 16, 1, 2, 3, 20, 19, 20, 5, 6, 23, 24, 47, 32, 49, 50, 51, 36, 35, 36, 53, 54, 39, 40, 63, 48, 33, 34, 35, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt798(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 4, 21, 22, 7, 8, 15, 0, 1, 2, 3, 20, 19, 20, 5, 6, 23, 24, 63, 48, 49, 50, 51, 36, 35, 36, 53, 54, 39, 40, 47, 32, 33, 34, 35, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt799(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 4, 21, 22, 7, 8, 31, 0, 1, 2, 3, 20, 19, 20, 5, 6, 23, 24, 47, 48, 49, 50, 51, 36, 35, 36, 53, 54, 39, 40, 63, 32, 33, 34, 35, 52, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt800(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 4, 21, 22, 7, 8, 15, 16, 17, 18, 19, 4, 19, 20, 5, 6, 23, 24, 63, 32, 33, 34, 35, 52, 35, 36, 53, 54, 39, 40, 47, 48, 49, 50, 51, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt801(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 4, 21, 22, 7, 8, 31, 16, 17, 18, 19, 4, 19, 20, 5, 6, 23, 24, 47, 32, 33, 34, 35, 52, 35, 36, 53, 54, 39, 40, 63, 48, 49, 50, 51, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt802(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 4, 21, 22, 7, 8, 15, 0, 17, 18, 19, 4, 19, 20, 5, 6, 23, 24, 63, 48, 33, 34, 35, 52, 35, 36, 53, 54, 39, 40, 47, 32, 49, 50, 51, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt803(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 4, 21, 22, 7, 8, 31, 0, 17, 18, 19, 4, 19, 20, 5, 6, 23, 24, 47, 48, 33, 34, 35, 52, 35, 36, 53, 54, 39, 40, 63, 32, 49, 50, 51, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt804(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 4, 21, 22, 7, 8, 15, 16, 1, 18, 19, 4, 19, 20, 5, 6, 23, 24, 63, 32, 49, 34, 35, 52, 35, 36, 53, 54, 39, 40, 47, 48, 33, 50, 51, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt805(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 4, 21, 22, 7, 8, 31, 16, 1, 18, 19, 4, 19, 20, 5, 6, 23, 24, 47, 32, 49, 34, 35, 52, 35, 36, 53, 54, 39, 40, 63, 48, 33, 50, 51, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt806(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 4, 21, 22, 7, 8, 15, 0, 1, 18, 19, 4, 19, 20, 5, 6, 23, 24, 63, 48, 49, 34, 35, 52, 35, 36, 53, 54, 39, 40, 47, 32, 33, 50, 51, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt807(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 4, 21, 22, 7, 8, 31, 0, 1, 18, 19, 4, 19, 20, 5, 6, 23, 24, 47, 48, 49, 34, 35, 52, 35, 36, 53, 54, 39, 40, 63, 32, 33, 50, 51, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt808(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 4, 21, 22, 7, 8, 15, 16, 17, 2, 19, 4, 19, 20, 5, 6, 23, 24, 63, 32, 33, 50, 35, 52, 35, 36, 53, 54, 39, 40, 47, 48, 49, 34, 51, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt809(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 4, 21, 22, 7, 8, 31, 16, 17, 2, 19, 4, 19, 20, 5, 6, 23, 24, 47, 32, 33, 50, 35, 52, 35, 36, 53, 54, 39, 40, 63, 48, 49, 34, 51, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt810(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 4, 21, 22, 7, 8, 15, 0, 17, 2, 19, 4, 19, 20, 5, 6, 23, 24, 63, 48, 33, 50, 35, 52, 35, 36, 53, 54, 39, 40, 47, 32, 49, 34, 51, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt811(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 4, 21, 22, 7, 8, 31, 0, 17, 2, 19, 4, 19, 20, 5, 6, 23, 24, 47, 48, 33, 50, 35, 52, 35, 36, 53, 54, 39, 40, 63, 32, 49, 34, 51, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt812(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 4, 21, 22, 7, 8, 15, 16, 1, 2, 19, 4, 19, 20, 5, 6, 23, 24, 63, 32, 49, 50, 35, 52, 35, 36, 53, 54, 39, 40, 47, 48, 33, 34, 51, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt813(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 4, 21, 22, 7, 8, 31, 16, 1, 2, 19, 4, 19, 20, 5, 6, 23, 24, 47, 32, 49, 50, 35, 52, 35, 36, 53, 54, 39, 40, 63, 48, 33, 34, 51, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt814(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 4, 21, 22, 7, 8, 15, 0, 1, 2, 19, 4, 19, 20, 5, 6, 23, 24, 63, 48, 49, 50, 35, 52, 35, 36, 53, 54, 39, 40, 47, 32, 33, 34, 51, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt815(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 4, 21, 22, 7, 8, 31, 0, 1, 2, 19, 4, 19, 20, 5, 6, 23, 24, 47, 48, 49, 50, 35, 52, 35, 36, 53, 54, 39, 40, 63, 32, 33, 34, 51, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt816(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 4, 21, 22, 7, 8, 15, 16, 17, 18, 3, 4, 19, 20, 5, 6, 23, 24, 63, 32, 33, 34, 51, 52, 35, 36, 53, 54, 39, 40, 47, 48, 49, 50, 35, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt817(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 4, 21, 22, 7, 8, 31, 16, 17, 18, 3, 4, 19, 20, 5, 6, 23, 24, 47, 32, 33, 34, 51, 52, 35, 36, 53, 54, 39, 40, 63, 48, 49, 50, 35, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt818(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 4, 21, 22, 7, 8, 15, 0, 17, 18, 3, 4, 19, 20, 5, 6, 23, 24, 63, 48, 33, 34, 51, 52, 35, 36, 53, 54, 39, 40, 47, 32, 49, 50, 35, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt819(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 4, 21, 22, 7, 8, 31, 0, 17, 18, 3, 4, 19, 20, 5, 6, 23, 24, 47, 48, 33, 34, 51, 52, 35, 36, 53, 54, 39, 40, 63, 32, 49, 50, 35, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt820(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 4, 21, 22, 7, 8, 15, 16, 1, 18, 3, 4, 19, 20, 5, 6, 23, 24, 63, 32, 49, 34, 51, 52, 35, 36, 53, 54, 39, 40, 47, 48, 33, 50, 35, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt821(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 4, 21, 22, 7, 8, 31, 16, 1, 18, 3, 4, 19, 20, 5, 6, 23, 24, 47, 32, 49, 34, 51, 52, 35, 36, 53, 54, 39, 40, 63, 48, 33, 50, 35, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt822(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 4, 21, 22, 7, 8, 15, 0, 1, 18, 3, 4, 19, 20, 5, 6, 23, 24, 63, 48, 49, 34, 51, 52, 35, 36, 53, 54, 39, 40, 47, 32, 33, 50, 35, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt823(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 4, 21, 22, 7, 8, 31, 0, 1, 18, 3, 4, 19, 20, 5, 6, 23, 24, 47, 48, 49, 34, 51, 52, 35, 36, 53, 54, 39, 40, 63, 32, 33, 50, 35, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt824(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 4, 21, 22, 7, 8, 15, 16, 17, 2, 3, 4, 19, 20, 5, 6, 23, 24, 63, 32, 33, 50, 51, 52, 35, 36, 53, 54, 39, 40, 47, 48, 49, 34, 35, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt825(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 4, 21, 22, 7, 8, 31, 16, 17, 2, 3, 4, 19, 20, 5, 6, 23, 24, 47, 32, 33, 50, 51, 52, 35, 36, 53, 54, 39, 40, 63, 48, 49, 34, 35, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt826(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 4, 21, 22, 7, 8, 15, 0, 17, 2, 3, 4, 19, 20, 5, 6, 23, 24, 63, 48, 33, 50, 51, 52, 35, 36, 53, 54, 39, 40, 47, 32, 49, 34, 35, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt827(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 4, 21, 22, 7, 8, 31, 0, 17, 2, 3, 4, 19, 20, 5, 6, 23, 24, 47, 48, 33, 50, 51, 52, 35, 36, 53, 54, 39, 40, 63, 32, 49, 34, 35, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt828(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 4, 21, 22, 7, 8, 15, 16, 1, 2, 3, 4, 19, 20, 5, 6, 23, 24, 63, 32, 49, 50, 51, 52, 35, 36, 53, 54, 39, 40, 47, 48, 33, 34, 35, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt829(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 4, 21, 22, 7, 8, 31, 16, 1, 2, 3, 4, 19, 20, 5, 6, 23, 24, 47, 32, 49, 50, 51, 52, 35, 36, 53, 54, 39, 40, 63, 48, 33, 34, 35, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt830(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 4, 21, 22, 7, 8, 15, 0, 1, 2, 3, 4, 19, 20, 5, 6, 23, 24, 63, 48, 49, 50, 51, 52, 35, 36, 53, 54, 39, 40, 47, 32, 33, 34, 35, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt831(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 4, 21, 22, 7, 8, 31, 0, 1, 2, 3, 4, 19, 20, 5, 6, 23, 24, 47, 48, 49, 50, 51, 52, 35, 36, 53, 54, 39, 40, 63, 32, 33, 34, 35, 36, 51, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt832(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 4, 21, 22, 7, 8, 15, 16, 17, 18, 19, 20, 3, 20, 5, 6, 23, 24, 63, 32, 33, 34, 35, 36, 51, 36, 53, 54, 39, 40, 47, 48, 49, 50, 51, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt833(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 4, 21, 22, 7, 8, 31, 16, 17, 18, 19, 20, 3, 20, 5, 6, 23, 24, 47, 32, 33, 34, 35, 36, 51, 36, 53, 54, 39, 40, 63, 48, 49, 50, 51, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt834(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 4, 21, 22, 7, 8, 15, 0, 17, 18, 19, 20, 3, 20, 5, 6, 23, 24, 63, 48, 33, 34, 35, 36, 51, 36, 53, 54, 39, 40, 47, 32, 49, 50, 51, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt835(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 4, 21, 22, 7, 8, 31, 0, 17, 18, 19, 20, 3, 20, 5, 6, 23, 24, 47, 48, 33, 34, 35, 36, 51, 36, 53, 54, 39, 40, 63, 32, 49, 50, 51, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt836(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 4, 21, 22, 7, 8, 15, 16, 1, 18, 19, 20, 3, 20, 5, 6, 23, 24, 63, 32, 49, 34, 35, 36, 51, 36, 53, 54, 39, 40, 47, 48, 33, 50, 51, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt837(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 4, 21, 22, 7, 8, 31, 16, 1, 18, 19, 20, 3, 20, 5, 6, 23, 24, 47, 32, 49, 34, 35, 36, 51, 36, 53, 54, 39, 40, 63, 48, 33, 50, 51, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt838(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 4, 21, 22, 7, 8, 15, 0, 1, 18, 19, 20, 3, 20, 5, 6, 23, 24, 63, 48, 49, 34, 35, 36, 51, 36, 53, 54, 39, 40, 47, 32, 33, 50, 51, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt839(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 4, 21, 22, 7, 8, 31, 0, 1, 18, 19, 20, 3, 20, 5, 6, 23, 24, 47, 48, 49, 34, 35, 36, 51, 36, 53, 54, 39, 40, 63, 32, 33, 50, 51, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt840(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 4, 21, 22, 7, 8, 15, 16, 17, 2, 19, 20, 3, 20, 5, 6, 23, 24, 63, 32, 33, 50, 35, 36, 51, 36, 53, 54, 39, 40, 47, 48, 49, 34, 51, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt841(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 4, 21, 22, 7, 8, 31, 16, 17, 2, 19, 20, 3, 20, 5, 6, 23, 24, 47, 32, 33, 50, 35, 36, 51, 36, 53, 54, 39, 40, 63, 48, 49, 34, 51, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt842(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 4, 21, 22, 7, 8, 15, 0, 17, 2, 19, 20, 3, 20, 5, 6, 23, 24, 63, 48, 33, 50, 35, 36, 51, 36, 53, 54, 39, 40, 47, 32, 49, 34, 51, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt843(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 4, 21, 22, 7, 8, 31, 0, 17, 2, 19, 20, 3, 20, 5, 6, 23, 24, 47, 48, 33, 50, 35, 36, 51, 36, 53, 54, 39, 40, 63, 32, 49, 34, 51, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt844(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 4, 21, 22, 7, 8, 15, 16, 1, 2, 19, 20, 3, 20, 5, 6, 23, 24, 63, 32, 49, 50, 35, 36, 51, 36, 53, 54, 39, 40, 47, 48, 33, 34, 51, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt845(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 4, 21, 22, 7, 8, 31, 16, 1, 2, 19, 20, 3, 20, 5, 6, 23, 24, 47, 32, 49, 50, 35, 36, 51, 36, 53, 54, 39, 40, 63, 48, 33, 34, 51, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt846(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 4, 21, 22, 7, 8, 15, 0, 1, 2, 19, 20, 3, 20, 5, 6, 23, 24, 63, 48, 49, 50, 35, 36, 51, 36, 53, 54, 39, 40, 47, 32, 33, 34, 51, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt847(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 4, 21, 22, 7, 8, 31, 0, 1, 2, 19, 20, 3, 20, 5, 6, 23, 24, 47, 48, 49, 50, 35, 36, 51, 36, 53, 54, 39, 40, 63, 32, 33, 34, 51, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt848(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 4, 21, 22, 7, 8, 15, 16, 17, 18, 3, 20, 3, 20, 5, 6, 23, 24, 63, 32, 33, 34, 51, 36, 51, 36, 53, 54, 39, 40, 47, 48, 49, 50, 35, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt849(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 4, 21, 22, 7, 8, 31, 16, 17, 18, 3, 20, 3, 20, 5, 6, 23, 24, 47, 32, 33, 34, 51, 36, 51, 36, 53, 54, 39, 40, 63, 48, 49, 50, 35, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt850(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 4, 21, 22, 7, 8, 15, 0, 17, 18, 3, 20, 3, 20, 5, 6, 23, 24, 63, 48, 33, 34, 51, 36, 51, 36, 53, 54, 39, 40, 47, 32, 49, 50, 35, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt851(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 4, 21, 22, 7, 8, 31, 0, 17, 18, 3, 20, 3, 20, 5, 6, 23, 24, 47, 48, 33, 34, 51, 36, 51, 36, 53, 54, 39, 40, 63, 32, 49, 50, 35, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt852(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 4, 21, 22, 7, 8, 15, 16, 1, 18, 3, 20, 3, 20, 5, 6, 23, 24, 63, 32, 49, 34, 51, 36, 51, 36, 53, 54, 39, 40, 47, 48, 33, 50, 35, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt853(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 4, 21, 22, 7, 8, 31, 16, 1, 18, 3, 20, 3, 20, 5, 6, 23, 24, 47, 32, 49, 34, 51, 36, 51, 36, 53, 54, 39, 40, 63, 48, 33, 50, 35, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt854(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 4, 21, 22, 7, 8, 15, 0, 1, 18, 3, 20, 3, 20, 5, 6, 23, 24, 63, 48, 49, 34, 51, 36, 51, 36, 53, 54, 39, 40, 47, 32, 33, 50, 35, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt855(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 4, 21, 22, 7, 8, 31, 0, 1, 18, 3, 20, 3, 20, 5, 6, 23, 24, 47, 48, 49, 34, 51, 36, 51, 36, 53, 54, 39, 40, 63, 32, 33, 50, 35, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt856(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 4, 21, 22, 7, 8, 15, 16, 17, 2, 3, 20, 3, 20, 5, 6, 23, 24, 63, 32, 33, 50, 51, 36, 51, 36, 53, 54, 39, 40, 47, 48, 49, 34, 35, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt857(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 4, 21, 22, 7, 8, 31, 16, 17, 2, 3, 20, 3, 20, 5, 6, 23, 24, 47, 32, 33, 50, 51, 36, 51, 36, 53, 54, 39, 40, 63, 48, 49, 34, 35, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt858(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 4, 21, 22, 7, 8, 15, 0, 17, 2, 3, 20, 3, 20, 5, 6, 23, 24, 63, 48, 33, 50, 51, 36, 51, 36, 53, 54, 39, 40, 47, 32, 49, 34, 35, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt859(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 4, 21, 22, 7, 8, 31, 0, 17, 2, 3, 20, 3, 20, 5, 6, 23, 24, 47, 48, 33, 50, 51, 36, 51, 36, 53, 54, 39, 40, 63, 32, 49, 34, 35, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt860(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 4, 21, 22, 7, 8, 15, 16, 1, 2, 3, 20, 3, 20, 5, 6, 23, 24, 63, 32, 49, 50, 51, 36, 51, 36, 53, 54, 39, 40, 47, 48, 33, 34, 35, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt861(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 4, 21, 22, 7, 8, 31, 16, 1, 2, 3, 20, 3, 20, 5, 6, 23, 24, 47, 32, 49, 50, 51, 36, 51, 36, 53, 54, 39, 40, 63, 48, 33, 34, 35, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt862(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 4, 21, 22, 7, 8, 15, 0, 1, 2, 3, 20, 3, 20, 5, 6, 23, 24, 63, 48, 49, 50, 51, 36, 51, 36, 53, 54, 39, 40, 47, 32, 33, 34, 35, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt863(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 4, 21, 22, 7, 8, 31, 0, 1, 2, 3, 20, 3, 20, 5, 6, 23, 24, 47, 48, 49, 50, 51, 36, 51, 36, 53, 54, 39, 40, 63, 32, 33, 34, 35, 52, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt864(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 4, 21, 22, 7, 8, 15, 16, 17, 18, 19, 4, 3, 20, 5, 6, 23, 24, 63, 32, 33, 34, 35, 52, 51, 36, 53, 54, 39, 40, 47, 48, 49, 50, 51, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt865(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 4, 21, 22, 7, 8, 31, 16, 17, 18, 19, 4, 3, 20, 5, 6, 23, 24, 47, 32, 33, 34, 35, 52, 51, 36, 53, 54, 39, 40, 63, 48, 49, 50, 51, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt866(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 4, 21, 22, 7, 8, 15, 0, 17, 18, 19, 4, 3, 20, 5, 6, 23, 24, 63, 48, 33, 34, 35, 52, 51, 36, 53, 54, 39, 40, 47, 32, 49, 50, 51, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt867(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 4, 21, 22, 7, 8, 31, 0, 17, 18, 19, 4, 3, 20, 5, 6, 23, 24, 47, 48, 33, 34, 35, 52, 51, 36, 53, 54, 39, 40, 63, 32, 49, 50, 51, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt868(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 4, 21, 22, 7, 8, 15, 16, 1, 18, 19, 4, 3, 20, 5, 6, 23, 24, 63, 32, 49, 34, 35, 52, 51, 36, 53, 54, 39, 40, 47, 48, 33, 50, 51, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt869(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 4, 21, 22, 7, 8, 31, 16, 1, 18, 19, 4, 3, 20, 5, 6, 23, 24, 47, 32, 49, 34, 35, 52, 51, 36, 53, 54, 39, 40, 63, 48, 33, 50, 51, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt870(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 4, 21, 22, 7, 8, 15, 0, 1, 18, 19, 4, 3, 20, 5, 6, 23, 24, 63, 48, 49, 34, 35, 52, 51, 36, 53, 54, 39, 40, 47, 32, 33, 50, 51, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt871(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 4, 21, 22, 7, 8, 31, 0, 1, 18, 19, 4, 3, 20, 5, 6, 23, 24, 47, 48, 49, 34, 35, 52, 51, 36, 53, 54, 39, 40, 63, 32, 33, 50, 51, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt872(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 4, 21, 22, 7, 8, 15, 16, 17, 2, 19, 4, 3, 20, 5, 6, 23, 24, 63, 32, 33, 50, 35, 52, 51, 36, 53, 54, 39, 40, 47, 48, 49, 34, 51, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt873(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 4, 21, 22, 7, 8, 31, 16, 17, 2, 19, 4, 3, 20, 5, 6, 23, 24, 47, 32, 33, 50, 35, 52, 51, 36, 53, 54, 39, 40, 63, 48, 49, 34, 51, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt874(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 4, 21, 22, 7, 8, 15, 0, 17, 2, 19, 4, 3, 20, 5, 6, 23, 24, 63, 48, 33, 50, 35, 52, 51, 36, 53, 54, 39, 40, 47, 32, 49, 34, 51, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt875(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 4, 21, 22, 7, 8, 31, 0, 17, 2, 19, 4, 3, 20, 5, 6, 23, 24, 47, 48, 33, 50, 35, 52, 51, 36, 53, 54, 39, 40, 63, 32, 49, 34, 51, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt876(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 4, 21, 22, 7, 8, 15, 16, 1, 2, 19, 4, 3, 20, 5, 6, 23, 24, 63, 32, 49, 50, 35, 52, 51, 36, 53, 54, 39, 40, 47, 48, 33, 34, 51, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt877(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 4, 21, 22, 7, 8, 31, 16, 1, 2, 19, 4, 3, 20, 5, 6, 23, 24, 47, 32, 49, 50, 35, 52, 51, 36, 53, 54, 39, 40, 63, 48, 33, 34, 51, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt878(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 4, 21, 22, 7, 8, 15, 0, 1, 2, 19, 4, 3, 20, 5, 6, 23, 24, 63, 48, 49, 50, 35, 52, 51, 36, 53, 54, 39, 40, 47, 32, 33, 34, 51, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt879(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 4, 21, 22, 7, 8, 31, 0, 1, 2, 19, 4, 3, 20, 5, 6, 23, 24, 47, 48, 49, 50, 35, 52, 51, 36, 53, 54, 39, 40, 63, 32, 33, 34, 51, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt880(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 4, 21, 22, 7, 8, 15, 16, 17, 18, 3, 4, 3, 20, 5, 6, 23, 24, 63, 32, 33, 34, 51, 52, 51, 36, 53, 54, 39, 40, 47, 48, 49, 50, 35, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt881(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 4, 21, 22, 7, 8, 31, 16, 17, 18, 3, 4, 3, 20, 5, 6, 23, 24, 47, 32, 33, 34, 51, 52, 51, 36, 53, 54, 39, 40, 63, 48, 49, 50, 35, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt882(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 4, 21, 22, 7, 8, 15, 0, 17, 18, 3, 4, 3, 20, 5, 6, 23, 24, 63, 48, 33, 34, 51, 52, 51, 36, 53, 54, 39, 40, 47, 32, 49, 50, 35, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt883(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 4, 21, 22, 7, 8, 31, 0, 17, 18, 3, 4, 3, 20, 5, 6, 23, 24, 47, 48, 33, 34, 51, 52, 51, 36, 53, 54, 39, 40, 63, 32, 49, 50, 35, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt884(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 4, 21, 22, 7, 8, 15, 16, 1, 18, 3, 4, 3, 20, 5, 6, 23, 24, 63, 32, 49, 34, 51, 52, 51, 36, 53, 54, 39, 40, 47, 48, 33, 50, 35, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt885(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 4, 21, 22, 7, 8, 31, 16, 1, 18, 3, 4, 3, 20, 5, 6, 23, 24, 47, 32, 49, 34, 51, 52, 51, 36, 53, 54, 39, 40, 63, 48, 33, 50, 35, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt886(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 4, 21, 22, 7, 8, 15, 0, 1, 18, 3, 4, 3, 20, 5, 6, 23, 24, 63, 48, 49, 34, 51, 52, 51, 36, 53, 54, 39, 40, 47, 32, 33, 50, 35, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt887(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 4, 21, 22, 7, 8, 31, 0, 1, 18, 3, 4, 3, 20, 5, 6, 23, 24, 47, 48, 49, 34, 51, 52, 51, 36, 53, 54, 39, 40, 63, 32, 33, 50, 35, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt888(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 4, 21, 22, 7, 8, 15, 16, 17, 2, 3, 4, 3, 20, 5, 6, 23, 24, 63, 32, 33, 50, 51, 52, 51, 36, 53, 54, 39, 40, 47, 48, 49, 34, 35, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt889(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 4, 21, 22, 7, 8, 31, 16, 17, 2, 3, 4, 3, 20, 5, 6, 23, 24, 47, 32, 33, 50, 51, 52, 51, 36, 53, 54, 39, 40, 63, 48, 49, 34, 35, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt890(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 4, 21, 22, 7, 8, 15, 0, 17, 2, 3, 4, 3, 20, 5, 6, 23, 24, 63, 48, 33, 50, 51, 52, 51, 36, 53, 54, 39, 40, 47, 32, 49, 34, 35, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt891(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 4, 21, 22, 7, 8, 31, 0, 17, 2, 3, 4, 3, 20, 5, 6, 23, 24, 47, 48, 33, 50, 51, 52, 51, 36, 53, 54, 39, 40, 63, 32, 49, 34, 35, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt892(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 4, 21, 22, 7, 8, 15, 16, 1, 2, 3, 4, 3, 20, 5, 6, 23, 24, 63, 32, 49, 50, 51, 52, 51, 36, 53, 54, 39, 40, 47, 48, 33, 34, 35, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt893(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 4, 21, 22, 7, 8, 31, 16, 1, 2, 3, 4, 3, 20, 5, 6, 23, 24, 47, 32, 49, 50, 51, 52, 51, 36, 53, 54, 39, 40, 63, 48, 33, 34, 35, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt894(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 4, 21, 22, 7, 8, 15, 0, 1, 2, 3, 4, 3, 20, 5, 6, 23, 24, 63, 48, 49, 50, 51, 52, 51, 36, 53, 54, 39, 40, 47, 32, 33, 34, 35, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt895(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 4, 21, 22, 7, 8, 31, 0, 1, 2, 3, 4, 3, 20, 5, 6, 23, 24, 47, 48, 49, 50, 51, 52, 51, 36, 53, 54, 39, 40, 63, 32, 33, 34, 35, 36, 35, 52, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}


