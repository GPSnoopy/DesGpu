#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt3328(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 4, 21, 6, 23, 24, 15, 16, 17, 18, 19, 20, 19, 20, 5, 22, 7, 8, 63, 32, 33, 34, 35, 36, 35, 36, 53, 38, 55, 56, 47, 48, 49, 50, 51, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3329(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 4, 21, 6, 23, 24, 31, 16, 17, 18, 19, 20, 19, 20, 5, 22, 7, 8, 47, 32, 33, 34, 35, 36, 35, 36, 53, 38, 55, 56, 63, 48, 49, 50, 51, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3330(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 4, 21, 6, 23, 24, 15, 0, 17, 18, 19, 20, 19, 20, 5, 22, 7, 8, 63, 48, 33, 34, 35, 36, 35, 36, 53, 38, 55, 56, 47, 32, 49, 50, 51, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3331(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 4, 21, 6, 23, 24, 31, 0, 17, 18, 19, 20, 19, 20, 5, 22, 7, 8, 47, 48, 33, 34, 35, 36, 35, 36, 53, 38, 55, 56, 63, 32, 49, 50, 51, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3332(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 4, 21, 6, 23, 24, 15, 16, 1, 18, 19, 20, 19, 20, 5, 22, 7, 8, 63, 32, 49, 34, 35, 36, 35, 36, 53, 38, 55, 56, 47, 48, 33, 50, 51, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3333(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 4, 21, 6, 23, 24, 31, 16, 1, 18, 19, 20, 19, 20, 5, 22, 7, 8, 47, 32, 49, 34, 35, 36, 35, 36, 53, 38, 55, 56, 63, 48, 33, 50, 51, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3334(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 4, 21, 6, 23, 24, 15, 0, 1, 18, 19, 20, 19, 20, 5, 22, 7, 8, 63, 48, 49, 34, 35, 36, 35, 36, 53, 38, 55, 56, 47, 32, 33, 50, 51, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3335(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 4, 21, 6, 23, 24, 31, 0, 1, 18, 19, 20, 19, 20, 5, 22, 7, 8, 47, 48, 49, 34, 35, 36, 35, 36, 53, 38, 55, 56, 63, 32, 33, 50, 51, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3336(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 4, 21, 6, 23, 24, 15, 16, 17, 2, 19, 20, 19, 20, 5, 22, 7, 8, 63, 32, 33, 50, 35, 36, 35, 36, 53, 38, 55, 56, 47, 48, 49, 34, 51, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3337(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 4, 21, 6, 23, 24, 31, 16, 17, 2, 19, 20, 19, 20, 5, 22, 7, 8, 47, 32, 33, 50, 35, 36, 35, 36, 53, 38, 55, 56, 63, 48, 49, 34, 51, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3338(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 4, 21, 6, 23, 24, 15, 0, 17, 2, 19, 20, 19, 20, 5, 22, 7, 8, 63, 48, 33, 50, 35, 36, 35, 36, 53, 38, 55, 56, 47, 32, 49, 34, 51, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3339(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 4, 21, 6, 23, 24, 31, 0, 17, 2, 19, 20, 19, 20, 5, 22, 7, 8, 47, 48, 33, 50, 35, 36, 35, 36, 53, 38, 55, 56, 63, 32, 49, 34, 51, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3340(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 4, 21, 6, 23, 24, 15, 16, 1, 2, 19, 20, 19, 20, 5, 22, 7, 8, 63, 32, 49, 50, 35, 36, 35, 36, 53, 38, 55, 56, 47, 48, 33, 34, 51, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3341(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 4, 21, 6, 23, 24, 31, 16, 1, 2, 19, 20, 19, 20, 5, 22, 7, 8, 47, 32, 49, 50, 35, 36, 35, 36, 53, 38, 55, 56, 63, 48, 33, 34, 51, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3342(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 4, 21, 6, 23, 24, 15, 0, 1, 2, 19, 20, 19, 20, 5, 22, 7, 8, 63, 48, 49, 50, 35, 36, 35, 36, 53, 38, 55, 56, 47, 32, 33, 34, 51, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3343(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 4, 21, 6, 23, 24, 31, 0, 1, 2, 19, 20, 19, 20, 5, 22, 7, 8, 47, 48, 49, 50, 35, 36, 35, 36, 53, 38, 55, 56, 63, 32, 33, 34, 51, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3344(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 4, 21, 6, 23, 24, 15, 16, 17, 18, 3, 20, 19, 20, 5, 22, 7, 8, 63, 32, 33, 34, 51, 36, 35, 36, 53, 38, 55, 56, 47, 48, 49, 50, 35, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3345(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 4, 21, 6, 23, 24, 31, 16, 17, 18, 3, 20, 19, 20, 5, 22, 7, 8, 47, 32, 33, 34, 51, 36, 35, 36, 53, 38, 55, 56, 63, 48, 49, 50, 35, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3346(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 4, 21, 6, 23, 24, 15, 0, 17, 18, 3, 20, 19, 20, 5, 22, 7, 8, 63, 48, 33, 34, 51, 36, 35, 36, 53, 38, 55, 56, 47, 32, 49, 50, 35, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3347(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 4, 21, 6, 23, 24, 31, 0, 17, 18, 3, 20, 19, 20, 5, 22, 7, 8, 47, 48, 33, 34, 51, 36, 35, 36, 53, 38, 55, 56, 63, 32, 49, 50, 35, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3348(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 4, 21, 6, 23, 24, 15, 16, 1, 18, 3, 20, 19, 20, 5, 22, 7, 8, 63, 32, 49, 34, 51, 36, 35, 36, 53, 38, 55, 56, 47, 48, 33, 50, 35, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3349(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 4, 21, 6, 23, 24, 31, 16, 1, 18, 3, 20, 19, 20, 5, 22, 7, 8, 47, 32, 49, 34, 51, 36, 35, 36, 53, 38, 55, 56, 63, 48, 33, 50, 35, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3350(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 4, 21, 6, 23, 24, 15, 0, 1, 18, 3, 20, 19, 20, 5, 22, 7, 8, 63, 48, 49, 34, 51, 36, 35, 36, 53, 38, 55, 56, 47, 32, 33, 50, 35, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3351(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 4, 21, 6, 23, 24, 31, 0, 1, 18, 3, 20, 19, 20, 5, 22, 7, 8, 47, 48, 49, 34, 51, 36, 35, 36, 53, 38, 55, 56, 63, 32, 33, 50, 35, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3352(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 4, 21, 6, 23, 24, 15, 16, 17, 2, 3, 20, 19, 20, 5, 22, 7, 8, 63, 32, 33, 50, 51, 36, 35, 36, 53, 38, 55, 56, 47, 48, 49, 34, 35, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3353(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 4, 21, 6, 23, 24, 31, 16, 17, 2, 3, 20, 19, 20, 5, 22, 7, 8, 47, 32, 33, 50, 51, 36, 35, 36, 53, 38, 55, 56, 63, 48, 49, 34, 35, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3354(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 4, 21, 6, 23, 24, 15, 0, 17, 2, 3, 20, 19, 20, 5, 22, 7, 8, 63, 48, 33, 50, 51, 36, 35, 36, 53, 38, 55, 56, 47, 32, 49, 34, 35, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3355(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 4, 21, 6, 23, 24, 31, 0, 17, 2, 3, 20, 19, 20, 5, 22, 7, 8, 47, 48, 33, 50, 51, 36, 35, 36, 53, 38, 55, 56, 63, 32, 49, 34, 35, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3356(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 4, 21, 6, 23, 24, 15, 16, 1, 2, 3, 20, 19, 20, 5, 22, 7, 8, 63, 32, 49, 50, 51, 36, 35, 36, 53, 38, 55, 56, 47, 48, 33, 34, 35, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3357(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 4, 21, 6, 23, 24, 31, 16, 1, 2, 3, 20, 19, 20, 5, 22, 7, 8, 47, 32, 49, 50, 51, 36, 35, 36, 53, 38, 55, 56, 63, 48, 33, 34, 35, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3358(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 4, 21, 6, 23, 24, 15, 0, 1, 2, 3, 20, 19, 20, 5, 22, 7, 8, 63, 48, 49, 50, 51, 36, 35, 36, 53, 38, 55, 56, 47, 32, 33, 34, 35, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3359(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 4, 21, 6, 23, 24, 31, 0, 1, 2, 3, 20, 19, 20, 5, 22, 7, 8, 47, 48, 49, 50, 51, 36, 35, 36, 53, 38, 55, 56, 63, 32, 33, 34, 35, 52, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3360(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 4, 21, 6, 23, 24, 15, 16, 17, 18, 19, 4, 19, 20, 5, 22, 7, 8, 63, 32, 33, 34, 35, 52, 35, 36, 53, 38, 55, 56, 47, 48, 49, 50, 51, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3361(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 4, 21, 6, 23, 24, 31, 16, 17, 18, 19, 4, 19, 20, 5, 22, 7, 8, 47, 32, 33, 34, 35, 52, 35, 36, 53, 38, 55, 56, 63, 48, 49, 50, 51, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3362(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 4, 21, 6, 23, 24, 15, 0, 17, 18, 19, 4, 19, 20, 5, 22, 7, 8, 63, 48, 33, 34, 35, 52, 35, 36, 53, 38, 55, 56, 47, 32, 49, 50, 51, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3363(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 4, 21, 6, 23, 24, 31, 0, 17, 18, 19, 4, 19, 20, 5, 22, 7, 8, 47, 48, 33, 34, 35, 52, 35, 36, 53, 38, 55, 56, 63, 32, 49, 50, 51, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3364(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 4, 21, 6, 23, 24, 15, 16, 1, 18, 19, 4, 19, 20, 5, 22, 7, 8, 63, 32, 49, 34, 35, 52, 35, 36, 53, 38, 55, 56, 47, 48, 33, 50, 51, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3365(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 4, 21, 6, 23, 24, 31, 16, 1, 18, 19, 4, 19, 20, 5, 22, 7, 8, 47, 32, 49, 34, 35, 52, 35, 36, 53, 38, 55, 56, 63, 48, 33, 50, 51, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3366(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 4, 21, 6, 23, 24, 15, 0, 1, 18, 19, 4, 19, 20, 5, 22, 7, 8, 63, 48, 49, 34, 35, 52, 35, 36, 53, 38, 55, 56, 47, 32, 33, 50, 51, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3367(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 4, 21, 6, 23, 24, 31, 0, 1, 18, 19, 4, 19, 20, 5, 22, 7, 8, 47, 48, 49, 34, 35, 52, 35, 36, 53, 38, 55, 56, 63, 32, 33, 50, 51, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3368(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 4, 21, 6, 23, 24, 15, 16, 17, 2, 19, 4, 19, 20, 5, 22, 7, 8, 63, 32, 33, 50, 35, 52, 35, 36, 53, 38, 55, 56, 47, 48, 49, 34, 51, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3369(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 4, 21, 6, 23, 24, 31, 16, 17, 2, 19, 4, 19, 20, 5, 22, 7, 8, 47, 32, 33, 50, 35, 52, 35, 36, 53, 38, 55, 56, 63, 48, 49, 34, 51, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3370(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 4, 21, 6, 23, 24, 15, 0, 17, 2, 19, 4, 19, 20, 5, 22, 7, 8, 63, 48, 33, 50, 35, 52, 35, 36, 53, 38, 55, 56, 47, 32, 49, 34, 51, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3371(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 4, 21, 6, 23, 24, 31, 0, 17, 2, 19, 4, 19, 20, 5, 22, 7, 8, 47, 48, 33, 50, 35, 52, 35, 36, 53, 38, 55, 56, 63, 32, 49, 34, 51, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3372(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 4, 21, 6, 23, 24, 15, 16, 1, 2, 19, 4, 19, 20, 5, 22, 7, 8, 63, 32, 49, 50, 35, 52, 35, 36, 53, 38, 55, 56, 47, 48, 33, 34, 51, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3373(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 4, 21, 6, 23, 24, 31, 16, 1, 2, 19, 4, 19, 20, 5, 22, 7, 8, 47, 32, 49, 50, 35, 52, 35, 36, 53, 38, 55, 56, 63, 48, 33, 34, 51, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3374(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 4, 21, 6, 23, 24, 15, 0, 1, 2, 19, 4, 19, 20, 5, 22, 7, 8, 63, 48, 49, 50, 35, 52, 35, 36, 53, 38, 55, 56, 47, 32, 33, 34, 51, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3375(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 4, 21, 6, 23, 24, 31, 0, 1, 2, 19, 4, 19, 20, 5, 22, 7, 8, 47, 48, 49, 50, 35, 52, 35, 36, 53, 38, 55, 56, 63, 32, 33, 34, 51, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3376(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 4, 21, 6, 23, 24, 15, 16, 17, 18, 3, 4, 19, 20, 5, 22, 7, 8, 63, 32, 33, 34, 51, 52, 35, 36, 53, 38, 55, 56, 47, 48, 49, 50, 35, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3377(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 4, 21, 6, 23, 24, 31, 16, 17, 18, 3, 4, 19, 20, 5, 22, 7, 8, 47, 32, 33, 34, 51, 52, 35, 36, 53, 38, 55, 56, 63, 48, 49, 50, 35, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3378(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 4, 21, 6, 23, 24, 15, 0, 17, 18, 3, 4, 19, 20, 5, 22, 7, 8, 63, 48, 33, 34, 51, 52, 35, 36, 53, 38, 55, 56, 47, 32, 49, 50, 35, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3379(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 4, 21, 6, 23, 24, 31, 0, 17, 18, 3, 4, 19, 20, 5, 22, 7, 8, 47, 48, 33, 34, 51, 52, 35, 36, 53, 38, 55, 56, 63, 32, 49, 50, 35, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3380(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 4, 21, 6, 23, 24, 15, 16, 1, 18, 3, 4, 19, 20, 5, 22, 7, 8, 63, 32, 49, 34, 51, 52, 35, 36, 53, 38, 55, 56, 47, 48, 33, 50, 35, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3381(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 4, 21, 6, 23, 24, 31, 16, 1, 18, 3, 4, 19, 20, 5, 22, 7, 8, 47, 32, 49, 34, 51, 52, 35, 36, 53, 38, 55, 56, 63, 48, 33, 50, 35, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3382(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 4, 21, 6, 23, 24, 15, 0, 1, 18, 3, 4, 19, 20, 5, 22, 7, 8, 63, 48, 49, 34, 51, 52, 35, 36, 53, 38, 55, 56, 47, 32, 33, 50, 35, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3383(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 4, 21, 6, 23, 24, 31, 0, 1, 18, 3, 4, 19, 20, 5, 22, 7, 8, 47, 48, 49, 34, 51, 52, 35, 36, 53, 38, 55, 56, 63, 32, 33, 50, 35, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3384(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 4, 21, 6, 23, 24, 15, 16, 17, 2, 3, 4, 19, 20, 5, 22, 7, 8, 63, 32, 33, 50, 51, 52, 35, 36, 53, 38, 55, 56, 47, 48, 49, 34, 35, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3385(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 4, 21, 6, 23, 24, 31, 16, 17, 2, 3, 4, 19, 20, 5, 22, 7, 8, 47, 32, 33, 50, 51, 52, 35, 36, 53, 38, 55, 56, 63, 48, 49, 34, 35, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3386(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 4, 21, 6, 23, 24, 15, 0, 17, 2, 3, 4, 19, 20, 5, 22, 7, 8, 63, 48, 33, 50, 51, 52, 35, 36, 53, 38, 55, 56, 47, 32, 49, 34, 35, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3387(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 4, 21, 6, 23, 24, 31, 0, 17, 2, 3, 4, 19, 20, 5, 22, 7, 8, 47, 48, 33, 50, 51, 52, 35, 36, 53, 38, 55, 56, 63, 32, 49, 34, 35, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3388(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 4, 21, 6, 23, 24, 15, 16, 1, 2, 3, 4, 19, 20, 5, 22, 7, 8, 63, 32, 49, 50, 51, 52, 35, 36, 53, 38, 55, 56, 47, 48, 33, 34, 35, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3389(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 4, 21, 6, 23, 24, 31, 16, 1, 2, 3, 4, 19, 20, 5, 22, 7, 8, 47, 32, 49, 50, 51, 52, 35, 36, 53, 38, 55, 56, 63, 48, 33, 34, 35, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3390(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 4, 21, 6, 23, 24, 15, 0, 1, 2, 3, 4, 19, 20, 5, 22, 7, 8, 63, 48, 49, 50, 51, 52, 35, 36, 53, 38, 55, 56, 47, 32, 33, 34, 35, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3391(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 4, 21, 6, 23, 24, 31, 0, 1, 2, 3, 4, 19, 20, 5, 22, 7, 8, 47, 48, 49, 50, 51, 52, 35, 36, 53, 38, 55, 56, 63, 32, 33, 34, 35, 36, 51, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3392(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 4, 21, 6, 23, 24, 15, 16, 17, 18, 19, 20, 3, 20, 5, 22, 7, 8, 63, 32, 33, 34, 35, 36, 51, 36, 53, 38, 55, 56, 47, 48, 49, 50, 51, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3393(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 4, 21, 6, 23, 24, 31, 16, 17, 18, 19, 20, 3, 20, 5, 22, 7, 8, 47, 32, 33, 34, 35, 36, 51, 36, 53, 38, 55, 56, 63, 48, 49, 50, 51, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3394(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 4, 21, 6, 23, 24, 15, 0, 17, 18, 19, 20, 3, 20, 5, 22, 7, 8, 63, 48, 33, 34, 35, 36, 51, 36, 53, 38, 55, 56, 47, 32, 49, 50, 51, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3395(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 4, 21, 6, 23, 24, 31, 0, 17, 18, 19, 20, 3, 20, 5, 22, 7, 8, 47, 48, 33, 34, 35, 36, 51, 36, 53, 38, 55, 56, 63, 32, 49, 50, 51, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3396(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 4, 21, 6, 23, 24, 15, 16, 1, 18, 19, 20, 3, 20, 5, 22, 7, 8, 63, 32, 49, 34, 35, 36, 51, 36, 53, 38, 55, 56, 47, 48, 33, 50, 51, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3397(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 4, 21, 6, 23, 24, 31, 16, 1, 18, 19, 20, 3, 20, 5, 22, 7, 8, 47, 32, 49, 34, 35, 36, 51, 36, 53, 38, 55, 56, 63, 48, 33, 50, 51, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3398(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 4, 21, 6, 23, 24, 15, 0, 1, 18, 19, 20, 3, 20, 5, 22, 7, 8, 63, 48, 49, 34, 35, 36, 51, 36, 53, 38, 55, 56, 47, 32, 33, 50, 51, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3399(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 4, 21, 6, 23, 24, 31, 0, 1, 18, 19, 20, 3, 20, 5, 22, 7, 8, 47, 48, 49, 34, 35, 36, 51, 36, 53, 38, 55, 56, 63, 32, 33, 50, 51, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3400(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 4, 21, 6, 23, 24, 15, 16, 17, 2, 19, 20, 3, 20, 5, 22, 7, 8, 63, 32, 33, 50, 35, 36, 51, 36, 53, 38, 55, 56, 47, 48, 49, 34, 51, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3401(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 4, 21, 6, 23, 24, 31, 16, 17, 2, 19, 20, 3, 20, 5, 22, 7, 8, 47, 32, 33, 50, 35, 36, 51, 36, 53, 38, 55, 56, 63, 48, 49, 34, 51, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3402(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 4, 21, 6, 23, 24, 15, 0, 17, 2, 19, 20, 3, 20, 5, 22, 7, 8, 63, 48, 33, 50, 35, 36, 51, 36, 53, 38, 55, 56, 47, 32, 49, 34, 51, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3403(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 4, 21, 6, 23, 24, 31, 0, 17, 2, 19, 20, 3, 20, 5, 22, 7, 8, 47, 48, 33, 50, 35, 36, 51, 36, 53, 38, 55, 56, 63, 32, 49, 34, 51, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3404(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 4, 21, 6, 23, 24, 15, 16, 1, 2, 19, 20, 3, 20, 5, 22, 7, 8, 63, 32, 49, 50, 35, 36, 51, 36, 53, 38, 55, 56, 47, 48, 33, 34, 51, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3405(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 4, 21, 6, 23, 24, 31, 16, 1, 2, 19, 20, 3, 20, 5, 22, 7, 8, 47, 32, 49, 50, 35, 36, 51, 36, 53, 38, 55, 56, 63, 48, 33, 34, 51, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3406(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 4, 21, 6, 23, 24, 15, 0, 1, 2, 19, 20, 3, 20, 5, 22, 7, 8, 63, 48, 49, 50, 35, 36, 51, 36, 53, 38, 55, 56, 47, 32, 33, 34, 51, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3407(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 4, 21, 6, 23, 24, 31, 0, 1, 2, 19, 20, 3, 20, 5, 22, 7, 8, 47, 48, 49, 50, 35, 36, 51, 36, 53, 38, 55, 56, 63, 32, 33, 34, 51, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3408(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 4, 21, 6, 23, 24, 15, 16, 17, 18, 3, 20, 3, 20, 5, 22, 7, 8, 63, 32, 33, 34, 51, 36, 51, 36, 53, 38, 55, 56, 47, 48, 49, 50, 35, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3409(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 4, 21, 6, 23, 24, 31, 16, 17, 18, 3, 20, 3, 20, 5, 22, 7, 8, 47, 32, 33, 34, 51, 36, 51, 36, 53, 38, 55, 56, 63, 48, 49, 50, 35, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3410(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 4, 21, 6, 23, 24, 15, 0, 17, 18, 3, 20, 3, 20, 5, 22, 7, 8, 63, 48, 33, 34, 51, 36, 51, 36, 53, 38, 55, 56, 47, 32, 49, 50, 35, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3411(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 4, 21, 6, 23, 24, 31, 0, 17, 18, 3, 20, 3, 20, 5, 22, 7, 8, 47, 48, 33, 34, 51, 36, 51, 36, 53, 38, 55, 56, 63, 32, 49, 50, 35, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3412(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 4, 21, 6, 23, 24, 15, 16, 1, 18, 3, 20, 3, 20, 5, 22, 7, 8, 63, 32, 49, 34, 51, 36, 51, 36, 53, 38, 55, 56, 47, 48, 33, 50, 35, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3413(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 4, 21, 6, 23, 24, 31, 16, 1, 18, 3, 20, 3, 20, 5, 22, 7, 8, 47, 32, 49, 34, 51, 36, 51, 36, 53, 38, 55, 56, 63, 48, 33, 50, 35, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3414(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 4, 21, 6, 23, 24, 15, 0, 1, 18, 3, 20, 3, 20, 5, 22, 7, 8, 63, 48, 49, 34, 51, 36, 51, 36, 53, 38, 55, 56, 47, 32, 33, 50, 35, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3415(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 4, 21, 6, 23, 24, 31, 0, 1, 18, 3, 20, 3, 20, 5, 22, 7, 8, 47, 48, 49, 34, 51, 36, 51, 36, 53, 38, 55, 56, 63, 32, 33, 50, 35, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3416(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 4, 21, 6, 23, 24, 15, 16, 17, 2, 3, 20, 3, 20, 5, 22, 7, 8, 63, 32, 33, 50, 51, 36, 51, 36, 53, 38, 55, 56, 47, 48, 49, 34, 35, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3417(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 4, 21, 6, 23, 24, 31, 16, 17, 2, 3, 20, 3, 20, 5, 22, 7, 8, 47, 32, 33, 50, 51, 36, 51, 36, 53, 38, 55, 56, 63, 48, 49, 34, 35, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3418(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 4, 21, 6, 23, 24, 15, 0, 17, 2, 3, 20, 3, 20, 5, 22, 7, 8, 63, 48, 33, 50, 51, 36, 51, 36, 53, 38, 55, 56, 47, 32, 49, 34, 35, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3419(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 4, 21, 6, 23, 24, 31, 0, 17, 2, 3, 20, 3, 20, 5, 22, 7, 8, 47, 48, 33, 50, 51, 36, 51, 36, 53, 38, 55, 56, 63, 32, 49, 34, 35, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3420(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 4, 21, 6, 23, 24, 15, 16, 1, 2, 3, 20, 3, 20, 5, 22, 7, 8, 63, 32, 49, 50, 51, 36, 51, 36, 53, 38, 55, 56, 47, 48, 33, 34, 35, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3421(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 4, 21, 6, 23, 24, 31, 16, 1, 2, 3, 20, 3, 20, 5, 22, 7, 8, 47, 32, 49, 50, 51, 36, 51, 36, 53, 38, 55, 56, 63, 48, 33, 34, 35, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3422(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 4, 21, 6, 23, 24, 15, 0, 1, 2, 3, 20, 3, 20, 5, 22, 7, 8, 63, 48, 49, 50, 51, 36, 51, 36, 53, 38, 55, 56, 47, 32, 33, 34, 35, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3423(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 4, 21, 6, 23, 24, 31, 0, 1, 2, 3, 20, 3, 20, 5, 22, 7, 8, 47, 48, 49, 50, 51, 36, 51, 36, 53, 38, 55, 56, 63, 32, 33, 34, 35, 52, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3424(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 4, 21, 6, 23, 24, 15, 16, 17, 18, 19, 4, 3, 20, 5, 22, 7, 8, 63, 32, 33, 34, 35, 52, 51, 36, 53, 38, 55, 56, 47, 48, 49, 50, 51, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3425(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 4, 21, 6, 23, 24, 31, 16, 17, 18, 19, 4, 3, 20, 5, 22, 7, 8, 47, 32, 33, 34, 35, 52, 51, 36, 53, 38, 55, 56, 63, 48, 49, 50, 51, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3426(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 4, 21, 6, 23, 24, 15, 0, 17, 18, 19, 4, 3, 20, 5, 22, 7, 8, 63, 48, 33, 34, 35, 52, 51, 36, 53, 38, 55, 56, 47, 32, 49, 50, 51, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3427(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 4, 21, 6, 23, 24, 31, 0, 17, 18, 19, 4, 3, 20, 5, 22, 7, 8, 47, 48, 33, 34, 35, 52, 51, 36, 53, 38, 55, 56, 63, 32, 49, 50, 51, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3428(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 4, 21, 6, 23, 24, 15, 16, 1, 18, 19, 4, 3, 20, 5, 22, 7, 8, 63, 32, 49, 34, 35, 52, 51, 36, 53, 38, 55, 56, 47, 48, 33, 50, 51, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3429(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 4, 21, 6, 23, 24, 31, 16, 1, 18, 19, 4, 3, 20, 5, 22, 7, 8, 47, 32, 49, 34, 35, 52, 51, 36, 53, 38, 55, 56, 63, 48, 33, 50, 51, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3430(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 4, 21, 6, 23, 24, 15, 0, 1, 18, 19, 4, 3, 20, 5, 22, 7, 8, 63, 48, 49, 34, 35, 52, 51, 36, 53, 38, 55, 56, 47, 32, 33, 50, 51, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3431(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 4, 21, 6, 23, 24, 31, 0, 1, 18, 19, 4, 3, 20, 5, 22, 7, 8, 47, 48, 49, 34, 35, 52, 51, 36, 53, 38, 55, 56, 63, 32, 33, 50, 51, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3432(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 4, 21, 6, 23, 24, 15, 16, 17, 2, 19, 4, 3, 20, 5, 22, 7, 8, 63, 32, 33, 50, 35, 52, 51, 36, 53, 38, 55, 56, 47, 48, 49, 34, 51, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3433(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 4, 21, 6, 23, 24, 31, 16, 17, 2, 19, 4, 3, 20, 5, 22, 7, 8, 47, 32, 33, 50, 35, 52, 51, 36, 53, 38, 55, 56, 63, 48, 49, 34, 51, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3434(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 4, 21, 6, 23, 24, 15, 0, 17, 2, 19, 4, 3, 20, 5, 22, 7, 8, 63, 48, 33, 50, 35, 52, 51, 36, 53, 38, 55, 56, 47, 32, 49, 34, 51, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3435(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 4, 21, 6, 23, 24, 31, 0, 17, 2, 19, 4, 3, 20, 5, 22, 7, 8, 47, 48, 33, 50, 35, 52, 51, 36, 53, 38, 55, 56, 63, 32, 49, 34, 51, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3436(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 4, 21, 6, 23, 24, 15, 16, 1, 2, 19, 4, 3, 20, 5, 22, 7, 8, 63, 32, 49, 50, 35, 52, 51, 36, 53, 38, 55, 56, 47, 48, 33, 34, 51, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3437(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 4, 21, 6, 23, 24, 31, 16, 1, 2, 19, 4, 3, 20, 5, 22, 7, 8, 47, 32, 49, 50, 35, 52, 51, 36, 53, 38, 55, 56, 63, 48, 33, 34, 51, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3438(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 4, 21, 6, 23, 24, 15, 0, 1, 2, 19, 4, 3, 20, 5, 22, 7, 8, 63, 48, 49, 50, 35, 52, 51, 36, 53, 38, 55, 56, 47, 32, 33, 34, 51, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3439(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 4, 21, 6, 23, 24, 31, 0, 1, 2, 19, 4, 3, 20, 5, 22, 7, 8, 47, 48, 49, 50, 35, 52, 51, 36, 53, 38, 55, 56, 63, 32, 33, 34, 51, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3440(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 4, 21, 6, 23, 24, 15, 16, 17, 18, 3, 4, 3, 20, 5, 22, 7, 8, 63, 32, 33, 34, 51, 52, 51, 36, 53, 38, 55, 56, 47, 48, 49, 50, 35, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3441(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 4, 21, 6, 23, 24, 31, 16, 17, 18, 3, 4, 3, 20, 5, 22, 7, 8, 47, 32, 33, 34, 51, 52, 51, 36, 53, 38, 55, 56, 63, 48, 49, 50, 35, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3442(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 4, 21, 6, 23, 24, 15, 0, 17, 18, 3, 4, 3, 20, 5, 22, 7, 8, 63, 48, 33, 34, 51, 52, 51, 36, 53, 38, 55, 56, 47, 32, 49, 50, 35, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3443(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 4, 21, 6, 23, 24, 31, 0, 17, 18, 3, 4, 3, 20, 5, 22, 7, 8, 47, 48, 33, 34, 51, 52, 51, 36, 53, 38, 55, 56, 63, 32, 49, 50, 35, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3444(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 4, 21, 6, 23, 24, 15, 16, 1, 18, 3, 4, 3, 20, 5, 22, 7, 8, 63, 32, 49, 34, 51, 52, 51, 36, 53, 38, 55, 56, 47, 48, 33, 50, 35, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3445(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 4, 21, 6, 23, 24, 31, 16, 1, 18, 3, 4, 3, 20, 5, 22, 7, 8, 47, 32, 49, 34, 51, 52, 51, 36, 53, 38, 55, 56, 63, 48, 33, 50, 35, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3446(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 4, 21, 6, 23, 24, 15, 0, 1, 18, 3, 4, 3, 20, 5, 22, 7, 8, 63, 48, 49, 34, 51, 52, 51, 36, 53, 38, 55, 56, 47, 32, 33, 50, 35, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3447(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 4, 21, 6, 23, 24, 31, 0, 1, 18, 3, 4, 3, 20, 5, 22, 7, 8, 47, 48, 49, 34, 51, 52, 51, 36, 53, 38, 55, 56, 63, 32, 33, 50, 35, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3448(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 4, 21, 6, 23, 24, 15, 16, 17, 2, 3, 4, 3, 20, 5, 22, 7, 8, 63, 32, 33, 50, 51, 52, 51, 36, 53, 38, 55, 56, 47, 48, 49, 34, 35, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3449(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 4, 21, 6, 23, 24, 31, 16, 17, 2, 3, 4, 3, 20, 5, 22, 7, 8, 47, 32, 33, 50, 51, 52, 51, 36, 53, 38, 55, 56, 63, 48, 49, 34, 35, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3450(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 4, 21, 6, 23, 24, 15, 0, 17, 2, 3, 4, 3, 20, 5, 22, 7, 8, 63, 48, 33, 50, 51, 52, 51, 36, 53, 38, 55, 56, 47, 32, 49, 34, 35, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3451(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 4, 21, 6, 23, 24, 31, 0, 17, 2, 3, 4, 3, 20, 5, 22, 7, 8, 47, 48, 33, 50, 51, 52, 51, 36, 53, 38, 55, 56, 63, 32, 49, 34, 35, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3452(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 4, 21, 6, 23, 24, 15, 16, 1, 2, 3, 4, 3, 20, 5, 22, 7, 8, 63, 32, 49, 50, 51, 52, 51, 36, 53, 38, 55, 56, 47, 48, 33, 34, 35, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3453(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 4, 21, 6, 23, 24, 31, 16, 1, 2, 3, 4, 3, 20, 5, 22, 7, 8, 47, 32, 49, 50, 51, 52, 51, 36, 53, 38, 55, 56, 63, 48, 33, 34, 35, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3454(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 4, 21, 6, 23, 24, 15, 0, 1, 2, 3, 4, 3, 20, 5, 22, 7, 8, 63, 48, 49, 50, 51, 52, 51, 36, 53, 38, 55, 56, 47, 32, 33, 34, 35, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3455(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 4, 21, 6, 23, 24, 31, 0, 1, 2, 3, 4, 3, 20, 5, 22, 7, 8, 47, 48, 49, 50, 51, 52, 51, 36, 53, 38, 55, 56, 63, 32, 33, 34, 35, 36, 35, 52, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}


