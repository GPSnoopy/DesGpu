#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt2304(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 4, 21, 6, 7, 24, 15, 16, 17, 18, 19, 20, 19, 20, 5, 22, 23, 8, 63, 32, 33, 34, 35, 36, 35, 36, 53, 38, 39, 56, 47, 48, 49, 50, 51, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2305(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 4, 21, 6, 7, 24, 31, 16, 17, 18, 19, 20, 19, 20, 5, 22, 23, 8, 47, 32, 33, 34, 35, 36, 35, 36, 53, 38, 39, 56, 63, 48, 49, 50, 51, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2306(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 4, 21, 6, 7, 24, 15, 0, 17, 18, 19, 20, 19, 20, 5, 22, 23, 8, 63, 48, 33, 34, 35, 36, 35, 36, 53, 38, 39, 56, 47, 32, 49, 50, 51, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2307(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 4, 21, 6, 7, 24, 31, 0, 17, 18, 19, 20, 19, 20, 5, 22, 23, 8, 47, 48, 33, 34, 35, 36, 35, 36, 53, 38, 39, 56, 63, 32, 49, 50, 51, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2308(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 4, 21, 6, 7, 24, 15, 16, 1, 18, 19, 20, 19, 20, 5, 22, 23, 8, 63, 32, 49, 34, 35, 36, 35, 36, 53, 38, 39, 56, 47, 48, 33, 50, 51, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2309(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 4, 21, 6, 7, 24, 31, 16, 1, 18, 19, 20, 19, 20, 5, 22, 23, 8, 47, 32, 49, 34, 35, 36, 35, 36, 53, 38, 39, 56, 63, 48, 33, 50, 51, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2310(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 4, 21, 6, 7, 24, 15, 0, 1, 18, 19, 20, 19, 20, 5, 22, 23, 8, 63, 48, 49, 34, 35, 36, 35, 36, 53, 38, 39, 56, 47, 32, 33, 50, 51, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2311(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 4, 21, 6, 7, 24, 31, 0, 1, 18, 19, 20, 19, 20, 5, 22, 23, 8, 47, 48, 49, 34, 35, 36, 35, 36, 53, 38, 39, 56, 63, 32, 33, 50, 51, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2312(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 4, 21, 6, 7, 24, 15, 16, 17, 2, 19, 20, 19, 20, 5, 22, 23, 8, 63, 32, 33, 50, 35, 36, 35, 36, 53, 38, 39, 56, 47, 48, 49, 34, 51, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2313(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 4, 21, 6, 7, 24, 31, 16, 17, 2, 19, 20, 19, 20, 5, 22, 23, 8, 47, 32, 33, 50, 35, 36, 35, 36, 53, 38, 39, 56, 63, 48, 49, 34, 51, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2314(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 4, 21, 6, 7, 24, 15, 0, 17, 2, 19, 20, 19, 20, 5, 22, 23, 8, 63, 48, 33, 50, 35, 36, 35, 36, 53, 38, 39, 56, 47, 32, 49, 34, 51, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2315(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 4, 21, 6, 7, 24, 31, 0, 17, 2, 19, 20, 19, 20, 5, 22, 23, 8, 47, 48, 33, 50, 35, 36, 35, 36, 53, 38, 39, 56, 63, 32, 49, 34, 51, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2316(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 4, 21, 6, 7, 24, 15, 16, 1, 2, 19, 20, 19, 20, 5, 22, 23, 8, 63, 32, 49, 50, 35, 36, 35, 36, 53, 38, 39, 56, 47, 48, 33, 34, 51, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2317(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 4, 21, 6, 7, 24, 31, 16, 1, 2, 19, 20, 19, 20, 5, 22, 23, 8, 47, 32, 49, 50, 35, 36, 35, 36, 53, 38, 39, 56, 63, 48, 33, 34, 51, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2318(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 4, 21, 6, 7, 24, 15, 0, 1, 2, 19, 20, 19, 20, 5, 22, 23, 8, 63, 48, 49, 50, 35, 36, 35, 36, 53, 38, 39, 56, 47, 32, 33, 34, 51, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2319(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 4, 21, 6, 7, 24, 31, 0, 1, 2, 19, 20, 19, 20, 5, 22, 23, 8, 47, 48, 49, 50, 35, 36, 35, 36, 53, 38, 39, 56, 63, 32, 33, 34, 51, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2320(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 4, 21, 6, 7, 24, 15, 16, 17, 18, 3, 20, 19, 20, 5, 22, 23, 8, 63, 32, 33, 34, 51, 36, 35, 36, 53, 38, 39, 56, 47, 48, 49, 50, 35, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2321(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 4, 21, 6, 7, 24, 31, 16, 17, 18, 3, 20, 19, 20, 5, 22, 23, 8, 47, 32, 33, 34, 51, 36, 35, 36, 53, 38, 39, 56, 63, 48, 49, 50, 35, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2322(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 4, 21, 6, 7, 24, 15, 0, 17, 18, 3, 20, 19, 20, 5, 22, 23, 8, 63, 48, 33, 34, 51, 36, 35, 36, 53, 38, 39, 56, 47, 32, 49, 50, 35, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2323(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 4, 21, 6, 7, 24, 31, 0, 17, 18, 3, 20, 19, 20, 5, 22, 23, 8, 47, 48, 33, 34, 51, 36, 35, 36, 53, 38, 39, 56, 63, 32, 49, 50, 35, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2324(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 4, 21, 6, 7, 24, 15, 16, 1, 18, 3, 20, 19, 20, 5, 22, 23, 8, 63, 32, 49, 34, 51, 36, 35, 36, 53, 38, 39, 56, 47, 48, 33, 50, 35, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2325(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 4, 21, 6, 7, 24, 31, 16, 1, 18, 3, 20, 19, 20, 5, 22, 23, 8, 47, 32, 49, 34, 51, 36, 35, 36, 53, 38, 39, 56, 63, 48, 33, 50, 35, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2326(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 4, 21, 6, 7, 24, 15, 0, 1, 18, 3, 20, 19, 20, 5, 22, 23, 8, 63, 48, 49, 34, 51, 36, 35, 36, 53, 38, 39, 56, 47, 32, 33, 50, 35, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2327(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 4, 21, 6, 7, 24, 31, 0, 1, 18, 3, 20, 19, 20, 5, 22, 23, 8, 47, 48, 49, 34, 51, 36, 35, 36, 53, 38, 39, 56, 63, 32, 33, 50, 35, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2328(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 4, 21, 6, 7, 24, 15, 16, 17, 2, 3, 20, 19, 20, 5, 22, 23, 8, 63, 32, 33, 50, 51, 36, 35, 36, 53, 38, 39, 56, 47, 48, 49, 34, 35, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2329(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 4, 21, 6, 7, 24, 31, 16, 17, 2, 3, 20, 19, 20, 5, 22, 23, 8, 47, 32, 33, 50, 51, 36, 35, 36, 53, 38, 39, 56, 63, 48, 49, 34, 35, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2330(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 4, 21, 6, 7, 24, 15, 0, 17, 2, 3, 20, 19, 20, 5, 22, 23, 8, 63, 48, 33, 50, 51, 36, 35, 36, 53, 38, 39, 56, 47, 32, 49, 34, 35, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2331(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 4, 21, 6, 7, 24, 31, 0, 17, 2, 3, 20, 19, 20, 5, 22, 23, 8, 47, 48, 33, 50, 51, 36, 35, 36, 53, 38, 39, 56, 63, 32, 49, 34, 35, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2332(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 4, 21, 6, 7, 24, 15, 16, 1, 2, 3, 20, 19, 20, 5, 22, 23, 8, 63, 32, 49, 50, 51, 36, 35, 36, 53, 38, 39, 56, 47, 48, 33, 34, 35, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2333(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 4, 21, 6, 7, 24, 31, 16, 1, 2, 3, 20, 19, 20, 5, 22, 23, 8, 47, 32, 49, 50, 51, 36, 35, 36, 53, 38, 39, 56, 63, 48, 33, 34, 35, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2334(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 4, 21, 6, 7, 24, 15, 0, 1, 2, 3, 20, 19, 20, 5, 22, 23, 8, 63, 48, 49, 50, 51, 36, 35, 36, 53, 38, 39, 56, 47, 32, 33, 34, 35, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2335(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 4, 21, 6, 7, 24, 31, 0, 1, 2, 3, 20, 19, 20, 5, 22, 23, 8, 47, 48, 49, 50, 51, 36, 35, 36, 53, 38, 39, 56, 63, 32, 33, 34, 35, 52, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2336(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 4, 21, 6, 7, 24, 15, 16, 17, 18, 19, 4, 19, 20, 5, 22, 23, 8, 63, 32, 33, 34, 35, 52, 35, 36, 53, 38, 39, 56, 47, 48, 49, 50, 51, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2337(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 4, 21, 6, 7, 24, 31, 16, 17, 18, 19, 4, 19, 20, 5, 22, 23, 8, 47, 32, 33, 34, 35, 52, 35, 36, 53, 38, 39, 56, 63, 48, 49, 50, 51, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2338(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 4, 21, 6, 7, 24, 15, 0, 17, 18, 19, 4, 19, 20, 5, 22, 23, 8, 63, 48, 33, 34, 35, 52, 35, 36, 53, 38, 39, 56, 47, 32, 49, 50, 51, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2339(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 4, 21, 6, 7, 24, 31, 0, 17, 18, 19, 4, 19, 20, 5, 22, 23, 8, 47, 48, 33, 34, 35, 52, 35, 36, 53, 38, 39, 56, 63, 32, 49, 50, 51, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2340(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 4, 21, 6, 7, 24, 15, 16, 1, 18, 19, 4, 19, 20, 5, 22, 23, 8, 63, 32, 49, 34, 35, 52, 35, 36, 53, 38, 39, 56, 47, 48, 33, 50, 51, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2341(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 4, 21, 6, 7, 24, 31, 16, 1, 18, 19, 4, 19, 20, 5, 22, 23, 8, 47, 32, 49, 34, 35, 52, 35, 36, 53, 38, 39, 56, 63, 48, 33, 50, 51, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2342(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 4, 21, 6, 7, 24, 15, 0, 1, 18, 19, 4, 19, 20, 5, 22, 23, 8, 63, 48, 49, 34, 35, 52, 35, 36, 53, 38, 39, 56, 47, 32, 33, 50, 51, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2343(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 4, 21, 6, 7, 24, 31, 0, 1, 18, 19, 4, 19, 20, 5, 22, 23, 8, 47, 48, 49, 34, 35, 52, 35, 36, 53, 38, 39, 56, 63, 32, 33, 50, 51, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2344(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 4, 21, 6, 7, 24, 15, 16, 17, 2, 19, 4, 19, 20, 5, 22, 23, 8, 63, 32, 33, 50, 35, 52, 35, 36, 53, 38, 39, 56, 47, 48, 49, 34, 51, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2345(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 4, 21, 6, 7, 24, 31, 16, 17, 2, 19, 4, 19, 20, 5, 22, 23, 8, 47, 32, 33, 50, 35, 52, 35, 36, 53, 38, 39, 56, 63, 48, 49, 34, 51, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2346(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 4, 21, 6, 7, 24, 15, 0, 17, 2, 19, 4, 19, 20, 5, 22, 23, 8, 63, 48, 33, 50, 35, 52, 35, 36, 53, 38, 39, 56, 47, 32, 49, 34, 51, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2347(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 4, 21, 6, 7, 24, 31, 0, 17, 2, 19, 4, 19, 20, 5, 22, 23, 8, 47, 48, 33, 50, 35, 52, 35, 36, 53, 38, 39, 56, 63, 32, 49, 34, 51, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2348(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 4, 21, 6, 7, 24, 15, 16, 1, 2, 19, 4, 19, 20, 5, 22, 23, 8, 63, 32, 49, 50, 35, 52, 35, 36, 53, 38, 39, 56, 47, 48, 33, 34, 51, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2349(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 4, 21, 6, 7, 24, 31, 16, 1, 2, 19, 4, 19, 20, 5, 22, 23, 8, 47, 32, 49, 50, 35, 52, 35, 36, 53, 38, 39, 56, 63, 48, 33, 34, 51, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2350(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 4, 21, 6, 7, 24, 15, 0, 1, 2, 19, 4, 19, 20, 5, 22, 23, 8, 63, 48, 49, 50, 35, 52, 35, 36, 53, 38, 39, 56, 47, 32, 33, 34, 51, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2351(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 4, 21, 6, 7, 24, 31, 0, 1, 2, 19, 4, 19, 20, 5, 22, 23, 8, 47, 48, 49, 50, 35, 52, 35, 36, 53, 38, 39, 56, 63, 32, 33, 34, 51, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2352(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 4, 21, 6, 7, 24, 15, 16, 17, 18, 3, 4, 19, 20, 5, 22, 23, 8, 63, 32, 33, 34, 51, 52, 35, 36, 53, 38, 39, 56, 47, 48, 49, 50, 35, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2353(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 4, 21, 6, 7, 24, 31, 16, 17, 18, 3, 4, 19, 20, 5, 22, 23, 8, 47, 32, 33, 34, 51, 52, 35, 36, 53, 38, 39, 56, 63, 48, 49, 50, 35, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2354(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 4, 21, 6, 7, 24, 15, 0, 17, 18, 3, 4, 19, 20, 5, 22, 23, 8, 63, 48, 33, 34, 51, 52, 35, 36, 53, 38, 39, 56, 47, 32, 49, 50, 35, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2355(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 4, 21, 6, 7, 24, 31, 0, 17, 18, 3, 4, 19, 20, 5, 22, 23, 8, 47, 48, 33, 34, 51, 52, 35, 36, 53, 38, 39, 56, 63, 32, 49, 50, 35, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2356(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 4, 21, 6, 7, 24, 15, 16, 1, 18, 3, 4, 19, 20, 5, 22, 23, 8, 63, 32, 49, 34, 51, 52, 35, 36, 53, 38, 39, 56, 47, 48, 33, 50, 35, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2357(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 4, 21, 6, 7, 24, 31, 16, 1, 18, 3, 4, 19, 20, 5, 22, 23, 8, 47, 32, 49, 34, 51, 52, 35, 36, 53, 38, 39, 56, 63, 48, 33, 50, 35, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2358(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 4, 21, 6, 7, 24, 15, 0, 1, 18, 3, 4, 19, 20, 5, 22, 23, 8, 63, 48, 49, 34, 51, 52, 35, 36, 53, 38, 39, 56, 47, 32, 33, 50, 35, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2359(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 4, 21, 6, 7, 24, 31, 0, 1, 18, 3, 4, 19, 20, 5, 22, 23, 8, 47, 48, 49, 34, 51, 52, 35, 36, 53, 38, 39, 56, 63, 32, 33, 50, 35, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2360(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 4, 21, 6, 7, 24, 15, 16, 17, 2, 3, 4, 19, 20, 5, 22, 23, 8, 63, 32, 33, 50, 51, 52, 35, 36, 53, 38, 39, 56, 47, 48, 49, 34, 35, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2361(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 4, 21, 6, 7, 24, 31, 16, 17, 2, 3, 4, 19, 20, 5, 22, 23, 8, 47, 32, 33, 50, 51, 52, 35, 36, 53, 38, 39, 56, 63, 48, 49, 34, 35, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2362(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 4, 21, 6, 7, 24, 15, 0, 17, 2, 3, 4, 19, 20, 5, 22, 23, 8, 63, 48, 33, 50, 51, 52, 35, 36, 53, 38, 39, 56, 47, 32, 49, 34, 35, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2363(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 4, 21, 6, 7, 24, 31, 0, 17, 2, 3, 4, 19, 20, 5, 22, 23, 8, 47, 48, 33, 50, 51, 52, 35, 36, 53, 38, 39, 56, 63, 32, 49, 34, 35, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2364(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 4, 21, 6, 7, 24, 15, 16, 1, 2, 3, 4, 19, 20, 5, 22, 23, 8, 63, 32, 49, 50, 51, 52, 35, 36, 53, 38, 39, 56, 47, 48, 33, 34, 35, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2365(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 4, 21, 6, 7, 24, 31, 16, 1, 2, 3, 4, 19, 20, 5, 22, 23, 8, 47, 32, 49, 50, 51, 52, 35, 36, 53, 38, 39, 56, 63, 48, 33, 34, 35, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2366(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 4, 21, 6, 7, 24, 15, 0, 1, 2, 3, 4, 19, 20, 5, 22, 23, 8, 63, 48, 49, 50, 51, 52, 35, 36, 53, 38, 39, 56, 47, 32, 33, 34, 35, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2367(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 4, 21, 6, 7, 24, 31, 0, 1, 2, 3, 4, 19, 20, 5, 22, 23, 8, 47, 48, 49, 50, 51, 52, 35, 36, 53, 38, 39, 56, 63, 32, 33, 34, 35, 36, 51, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2368(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 4, 21, 6, 7, 24, 15, 16, 17, 18, 19, 20, 3, 20, 5, 22, 23, 8, 63, 32, 33, 34, 35, 36, 51, 36, 53, 38, 39, 56, 47, 48, 49, 50, 51, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2369(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 4, 21, 6, 7, 24, 31, 16, 17, 18, 19, 20, 3, 20, 5, 22, 23, 8, 47, 32, 33, 34, 35, 36, 51, 36, 53, 38, 39, 56, 63, 48, 49, 50, 51, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2370(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 4, 21, 6, 7, 24, 15, 0, 17, 18, 19, 20, 3, 20, 5, 22, 23, 8, 63, 48, 33, 34, 35, 36, 51, 36, 53, 38, 39, 56, 47, 32, 49, 50, 51, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2371(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 4, 21, 6, 7, 24, 31, 0, 17, 18, 19, 20, 3, 20, 5, 22, 23, 8, 47, 48, 33, 34, 35, 36, 51, 36, 53, 38, 39, 56, 63, 32, 49, 50, 51, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2372(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 4, 21, 6, 7, 24, 15, 16, 1, 18, 19, 20, 3, 20, 5, 22, 23, 8, 63, 32, 49, 34, 35, 36, 51, 36, 53, 38, 39, 56, 47, 48, 33, 50, 51, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2373(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 4, 21, 6, 7, 24, 31, 16, 1, 18, 19, 20, 3, 20, 5, 22, 23, 8, 47, 32, 49, 34, 35, 36, 51, 36, 53, 38, 39, 56, 63, 48, 33, 50, 51, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2374(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 4, 21, 6, 7, 24, 15, 0, 1, 18, 19, 20, 3, 20, 5, 22, 23, 8, 63, 48, 49, 34, 35, 36, 51, 36, 53, 38, 39, 56, 47, 32, 33, 50, 51, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2375(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 4, 21, 6, 7, 24, 31, 0, 1, 18, 19, 20, 3, 20, 5, 22, 23, 8, 47, 48, 49, 34, 35, 36, 51, 36, 53, 38, 39, 56, 63, 32, 33, 50, 51, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2376(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 4, 21, 6, 7, 24, 15, 16, 17, 2, 19, 20, 3, 20, 5, 22, 23, 8, 63, 32, 33, 50, 35, 36, 51, 36, 53, 38, 39, 56, 47, 48, 49, 34, 51, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2377(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 4, 21, 6, 7, 24, 31, 16, 17, 2, 19, 20, 3, 20, 5, 22, 23, 8, 47, 32, 33, 50, 35, 36, 51, 36, 53, 38, 39, 56, 63, 48, 49, 34, 51, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2378(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 4, 21, 6, 7, 24, 15, 0, 17, 2, 19, 20, 3, 20, 5, 22, 23, 8, 63, 48, 33, 50, 35, 36, 51, 36, 53, 38, 39, 56, 47, 32, 49, 34, 51, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2379(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 4, 21, 6, 7, 24, 31, 0, 17, 2, 19, 20, 3, 20, 5, 22, 23, 8, 47, 48, 33, 50, 35, 36, 51, 36, 53, 38, 39, 56, 63, 32, 49, 34, 51, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2380(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 4, 21, 6, 7, 24, 15, 16, 1, 2, 19, 20, 3, 20, 5, 22, 23, 8, 63, 32, 49, 50, 35, 36, 51, 36, 53, 38, 39, 56, 47, 48, 33, 34, 51, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2381(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 4, 21, 6, 7, 24, 31, 16, 1, 2, 19, 20, 3, 20, 5, 22, 23, 8, 47, 32, 49, 50, 35, 36, 51, 36, 53, 38, 39, 56, 63, 48, 33, 34, 51, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2382(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 4, 21, 6, 7, 24, 15, 0, 1, 2, 19, 20, 3, 20, 5, 22, 23, 8, 63, 48, 49, 50, 35, 36, 51, 36, 53, 38, 39, 56, 47, 32, 33, 34, 51, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2383(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 4, 21, 6, 7, 24, 31, 0, 1, 2, 19, 20, 3, 20, 5, 22, 23, 8, 47, 48, 49, 50, 35, 36, 51, 36, 53, 38, 39, 56, 63, 32, 33, 34, 51, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2384(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 4, 21, 6, 7, 24, 15, 16, 17, 18, 3, 20, 3, 20, 5, 22, 23, 8, 63, 32, 33, 34, 51, 36, 51, 36, 53, 38, 39, 56, 47, 48, 49, 50, 35, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2385(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 4, 21, 6, 7, 24, 31, 16, 17, 18, 3, 20, 3, 20, 5, 22, 23, 8, 47, 32, 33, 34, 51, 36, 51, 36, 53, 38, 39, 56, 63, 48, 49, 50, 35, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2386(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 4, 21, 6, 7, 24, 15, 0, 17, 18, 3, 20, 3, 20, 5, 22, 23, 8, 63, 48, 33, 34, 51, 36, 51, 36, 53, 38, 39, 56, 47, 32, 49, 50, 35, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2387(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 4, 21, 6, 7, 24, 31, 0, 17, 18, 3, 20, 3, 20, 5, 22, 23, 8, 47, 48, 33, 34, 51, 36, 51, 36, 53, 38, 39, 56, 63, 32, 49, 50, 35, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2388(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 4, 21, 6, 7, 24, 15, 16, 1, 18, 3, 20, 3, 20, 5, 22, 23, 8, 63, 32, 49, 34, 51, 36, 51, 36, 53, 38, 39, 56, 47, 48, 33, 50, 35, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2389(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 4, 21, 6, 7, 24, 31, 16, 1, 18, 3, 20, 3, 20, 5, 22, 23, 8, 47, 32, 49, 34, 51, 36, 51, 36, 53, 38, 39, 56, 63, 48, 33, 50, 35, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2390(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 4, 21, 6, 7, 24, 15, 0, 1, 18, 3, 20, 3, 20, 5, 22, 23, 8, 63, 48, 49, 34, 51, 36, 51, 36, 53, 38, 39, 56, 47, 32, 33, 50, 35, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2391(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 4, 21, 6, 7, 24, 31, 0, 1, 18, 3, 20, 3, 20, 5, 22, 23, 8, 47, 48, 49, 34, 51, 36, 51, 36, 53, 38, 39, 56, 63, 32, 33, 50, 35, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2392(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 4, 21, 6, 7, 24, 15, 16, 17, 2, 3, 20, 3, 20, 5, 22, 23, 8, 63, 32, 33, 50, 51, 36, 51, 36, 53, 38, 39, 56, 47, 48, 49, 34, 35, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2393(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 4, 21, 6, 7, 24, 31, 16, 17, 2, 3, 20, 3, 20, 5, 22, 23, 8, 47, 32, 33, 50, 51, 36, 51, 36, 53, 38, 39, 56, 63, 48, 49, 34, 35, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2394(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 4, 21, 6, 7, 24, 15, 0, 17, 2, 3, 20, 3, 20, 5, 22, 23, 8, 63, 48, 33, 50, 51, 36, 51, 36, 53, 38, 39, 56, 47, 32, 49, 34, 35, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2395(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 4, 21, 6, 7, 24, 31, 0, 17, 2, 3, 20, 3, 20, 5, 22, 23, 8, 47, 48, 33, 50, 51, 36, 51, 36, 53, 38, 39, 56, 63, 32, 49, 34, 35, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2396(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 4, 21, 6, 7, 24, 15, 16, 1, 2, 3, 20, 3, 20, 5, 22, 23, 8, 63, 32, 49, 50, 51, 36, 51, 36, 53, 38, 39, 56, 47, 48, 33, 34, 35, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2397(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 4, 21, 6, 7, 24, 31, 16, 1, 2, 3, 20, 3, 20, 5, 22, 23, 8, 47, 32, 49, 50, 51, 36, 51, 36, 53, 38, 39, 56, 63, 48, 33, 34, 35, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2398(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 4, 21, 6, 7, 24, 15, 0, 1, 2, 3, 20, 3, 20, 5, 22, 23, 8, 63, 48, 49, 50, 51, 36, 51, 36, 53, 38, 39, 56, 47, 32, 33, 34, 35, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2399(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 4, 21, 6, 7, 24, 31, 0, 1, 2, 3, 20, 3, 20, 5, 22, 23, 8, 47, 48, 49, 50, 51, 36, 51, 36, 53, 38, 39, 56, 63, 32, 33, 34, 35, 52, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2400(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 4, 21, 6, 7, 24, 15, 16, 17, 18, 19, 4, 3, 20, 5, 22, 23, 8, 63, 32, 33, 34, 35, 52, 51, 36, 53, 38, 39, 56, 47, 48, 49, 50, 51, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2401(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 4, 21, 6, 7, 24, 31, 16, 17, 18, 19, 4, 3, 20, 5, 22, 23, 8, 47, 32, 33, 34, 35, 52, 51, 36, 53, 38, 39, 56, 63, 48, 49, 50, 51, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2402(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 4, 21, 6, 7, 24, 15, 0, 17, 18, 19, 4, 3, 20, 5, 22, 23, 8, 63, 48, 33, 34, 35, 52, 51, 36, 53, 38, 39, 56, 47, 32, 49, 50, 51, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2403(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 4, 21, 6, 7, 24, 31, 0, 17, 18, 19, 4, 3, 20, 5, 22, 23, 8, 47, 48, 33, 34, 35, 52, 51, 36, 53, 38, 39, 56, 63, 32, 49, 50, 51, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2404(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 4, 21, 6, 7, 24, 15, 16, 1, 18, 19, 4, 3, 20, 5, 22, 23, 8, 63, 32, 49, 34, 35, 52, 51, 36, 53, 38, 39, 56, 47, 48, 33, 50, 51, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2405(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 4, 21, 6, 7, 24, 31, 16, 1, 18, 19, 4, 3, 20, 5, 22, 23, 8, 47, 32, 49, 34, 35, 52, 51, 36, 53, 38, 39, 56, 63, 48, 33, 50, 51, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2406(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 4, 21, 6, 7, 24, 15, 0, 1, 18, 19, 4, 3, 20, 5, 22, 23, 8, 63, 48, 49, 34, 35, 52, 51, 36, 53, 38, 39, 56, 47, 32, 33, 50, 51, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2407(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 4, 21, 6, 7, 24, 31, 0, 1, 18, 19, 4, 3, 20, 5, 22, 23, 8, 47, 48, 49, 34, 35, 52, 51, 36, 53, 38, 39, 56, 63, 32, 33, 50, 51, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2408(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 4, 21, 6, 7, 24, 15, 16, 17, 2, 19, 4, 3, 20, 5, 22, 23, 8, 63, 32, 33, 50, 35, 52, 51, 36, 53, 38, 39, 56, 47, 48, 49, 34, 51, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2409(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 4, 21, 6, 7, 24, 31, 16, 17, 2, 19, 4, 3, 20, 5, 22, 23, 8, 47, 32, 33, 50, 35, 52, 51, 36, 53, 38, 39, 56, 63, 48, 49, 34, 51, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2410(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 4, 21, 6, 7, 24, 15, 0, 17, 2, 19, 4, 3, 20, 5, 22, 23, 8, 63, 48, 33, 50, 35, 52, 51, 36, 53, 38, 39, 56, 47, 32, 49, 34, 51, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2411(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 4, 21, 6, 7, 24, 31, 0, 17, 2, 19, 4, 3, 20, 5, 22, 23, 8, 47, 48, 33, 50, 35, 52, 51, 36, 53, 38, 39, 56, 63, 32, 49, 34, 51, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2412(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 4, 21, 6, 7, 24, 15, 16, 1, 2, 19, 4, 3, 20, 5, 22, 23, 8, 63, 32, 49, 50, 35, 52, 51, 36, 53, 38, 39, 56, 47, 48, 33, 34, 51, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2413(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 4, 21, 6, 7, 24, 31, 16, 1, 2, 19, 4, 3, 20, 5, 22, 23, 8, 47, 32, 49, 50, 35, 52, 51, 36, 53, 38, 39, 56, 63, 48, 33, 34, 51, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2414(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 4, 21, 6, 7, 24, 15, 0, 1, 2, 19, 4, 3, 20, 5, 22, 23, 8, 63, 48, 49, 50, 35, 52, 51, 36, 53, 38, 39, 56, 47, 32, 33, 34, 51, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2415(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 4, 21, 6, 7, 24, 31, 0, 1, 2, 19, 4, 3, 20, 5, 22, 23, 8, 47, 48, 49, 50, 35, 52, 51, 36, 53, 38, 39, 56, 63, 32, 33, 34, 51, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2416(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 4, 21, 6, 7, 24, 15, 16, 17, 18, 3, 4, 3, 20, 5, 22, 23, 8, 63, 32, 33, 34, 51, 52, 51, 36, 53, 38, 39, 56, 47, 48, 49, 50, 35, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2417(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 4, 21, 6, 7, 24, 31, 16, 17, 18, 3, 4, 3, 20, 5, 22, 23, 8, 47, 32, 33, 34, 51, 52, 51, 36, 53, 38, 39, 56, 63, 48, 49, 50, 35, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2418(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 4, 21, 6, 7, 24, 15, 0, 17, 18, 3, 4, 3, 20, 5, 22, 23, 8, 63, 48, 33, 34, 51, 52, 51, 36, 53, 38, 39, 56, 47, 32, 49, 50, 35, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2419(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 4, 21, 6, 7, 24, 31, 0, 17, 18, 3, 4, 3, 20, 5, 22, 23, 8, 47, 48, 33, 34, 51, 52, 51, 36, 53, 38, 39, 56, 63, 32, 49, 50, 35, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2420(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 4, 21, 6, 7, 24, 15, 16, 1, 18, 3, 4, 3, 20, 5, 22, 23, 8, 63, 32, 49, 34, 51, 52, 51, 36, 53, 38, 39, 56, 47, 48, 33, 50, 35, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2421(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 4, 21, 6, 7, 24, 31, 16, 1, 18, 3, 4, 3, 20, 5, 22, 23, 8, 47, 32, 49, 34, 51, 52, 51, 36, 53, 38, 39, 56, 63, 48, 33, 50, 35, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2422(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 4, 21, 6, 7, 24, 15, 0, 1, 18, 3, 4, 3, 20, 5, 22, 23, 8, 63, 48, 49, 34, 51, 52, 51, 36, 53, 38, 39, 56, 47, 32, 33, 50, 35, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2423(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 4, 21, 6, 7, 24, 31, 0, 1, 18, 3, 4, 3, 20, 5, 22, 23, 8, 47, 48, 49, 34, 51, 52, 51, 36, 53, 38, 39, 56, 63, 32, 33, 50, 35, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2424(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 4, 21, 6, 7, 24, 15, 16, 17, 2, 3, 4, 3, 20, 5, 22, 23, 8, 63, 32, 33, 50, 51, 52, 51, 36, 53, 38, 39, 56, 47, 48, 49, 34, 35, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2425(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 4, 21, 6, 7, 24, 31, 16, 17, 2, 3, 4, 3, 20, 5, 22, 23, 8, 47, 32, 33, 50, 51, 52, 51, 36, 53, 38, 39, 56, 63, 48, 49, 34, 35, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2426(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 4, 21, 6, 7, 24, 15, 0, 17, 2, 3, 4, 3, 20, 5, 22, 23, 8, 63, 48, 33, 50, 51, 52, 51, 36, 53, 38, 39, 56, 47, 32, 49, 34, 35, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2427(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 4, 21, 6, 7, 24, 31, 0, 17, 2, 3, 4, 3, 20, 5, 22, 23, 8, 47, 48, 33, 50, 51, 52, 51, 36, 53, 38, 39, 56, 63, 32, 49, 34, 35, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2428(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 4, 21, 6, 7, 24, 15, 16, 1, 2, 3, 4, 3, 20, 5, 22, 23, 8, 63, 32, 49, 50, 51, 52, 51, 36, 53, 38, 39, 56, 47, 48, 33, 34, 35, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2429(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 4, 21, 6, 7, 24, 31, 16, 1, 2, 3, 4, 3, 20, 5, 22, 23, 8, 47, 32, 49, 50, 51, 52, 51, 36, 53, 38, 39, 56, 63, 48, 33, 34, 35, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2430(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 4, 21, 6, 7, 24, 15, 0, 1, 2, 3, 4, 3, 20, 5, 22, 23, 8, 63, 48, 49, 50, 51, 52, 51, 36, 53, 38, 39, 56, 47, 32, 33, 34, 35, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2431(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 4, 21, 6, 7, 24, 31, 0, 1, 2, 3, 4, 3, 20, 5, 22, 23, 8, 47, 48, 49, 50, 51, 52, 51, 36, 53, 38, 39, 56, 63, 32, 33, 34, 35, 36, 35, 52, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}


