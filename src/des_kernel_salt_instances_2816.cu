#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt2816(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 4, 21, 22, 7, 24, 15, 16, 17, 18, 19, 20, 19, 20, 5, 6, 23, 8, 63, 32, 33, 34, 35, 36, 35, 36, 53, 54, 39, 56, 47, 48, 49, 50, 51, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2817(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 4, 21, 22, 7, 24, 31, 16, 17, 18, 19, 20, 19, 20, 5, 6, 23, 8, 47, 32, 33, 34, 35, 36, 35, 36, 53, 54, 39, 56, 63, 48, 49, 50, 51, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2818(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 4, 21, 22, 7, 24, 15, 0, 17, 18, 19, 20, 19, 20, 5, 6, 23, 8, 63, 48, 33, 34, 35, 36, 35, 36, 53, 54, 39, 56, 47, 32, 49, 50, 51, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2819(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 4, 21, 22, 7, 24, 31, 0, 17, 18, 19, 20, 19, 20, 5, 6, 23, 8, 47, 48, 33, 34, 35, 36, 35, 36, 53, 54, 39, 56, 63, 32, 49, 50, 51, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2820(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 4, 21, 22, 7, 24, 15, 16, 1, 18, 19, 20, 19, 20, 5, 6, 23, 8, 63, 32, 49, 34, 35, 36, 35, 36, 53, 54, 39, 56, 47, 48, 33, 50, 51, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2821(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 4, 21, 22, 7, 24, 31, 16, 1, 18, 19, 20, 19, 20, 5, 6, 23, 8, 47, 32, 49, 34, 35, 36, 35, 36, 53, 54, 39, 56, 63, 48, 33, 50, 51, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2822(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 4, 21, 22, 7, 24, 15, 0, 1, 18, 19, 20, 19, 20, 5, 6, 23, 8, 63, 48, 49, 34, 35, 36, 35, 36, 53, 54, 39, 56, 47, 32, 33, 50, 51, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2823(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 4, 21, 22, 7, 24, 31, 0, 1, 18, 19, 20, 19, 20, 5, 6, 23, 8, 47, 48, 49, 34, 35, 36, 35, 36, 53, 54, 39, 56, 63, 32, 33, 50, 51, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2824(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 4, 21, 22, 7, 24, 15, 16, 17, 2, 19, 20, 19, 20, 5, 6, 23, 8, 63, 32, 33, 50, 35, 36, 35, 36, 53, 54, 39, 56, 47, 48, 49, 34, 51, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2825(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 4, 21, 22, 7, 24, 31, 16, 17, 2, 19, 20, 19, 20, 5, 6, 23, 8, 47, 32, 33, 50, 35, 36, 35, 36, 53, 54, 39, 56, 63, 48, 49, 34, 51, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2826(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 4, 21, 22, 7, 24, 15, 0, 17, 2, 19, 20, 19, 20, 5, 6, 23, 8, 63, 48, 33, 50, 35, 36, 35, 36, 53, 54, 39, 56, 47, 32, 49, 34, 51, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2827(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 4, 21, 22, 7, 24, 31, 0, 17, 2, 19, 20, 19, 20, 5, 6, 23, 8, 47, 48, 33, 50, 35, 36, 35, 36, 53, 54, 39, 56, 63, 32, 49, 34, 51, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2828(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 4, 21, 22, 7, 24, 15, 16, 1, 2, 19, 20, 19, 20, 5, 6, 23, 8, 63, 32, 49, 50, 35, 36, 35, 36, 53, 54, 39, 56, 47, 48, 33, 34, 51, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2829(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 4, 21, 22, 7, 24, 31, 16, 1, 2, 19, 20, 19, 20, 5, 6, 23, 8, 47, 32, 49, 50, 35, 36, 35, 36, 53, 54, 39, 56, 63, 48, 33, 34, 51, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2830(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 4, 21, 22, 7, 24, 15, 0, 1, 2, 19, 20, 19, 20, 5, 6, 23, 8, 63, 48, 49, 50, 35, 36, 35, 36, 53, 54, 39, 56, 47, 32, 33, 34, 51, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2831(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 4, 21, 22, 7, 24, 31, 0, 1, 2, 19, 20, 19, 20, 5, 6, 23, 8, 47, 48, 49, 50, 35, 36, 35, 36, 53, 54, 39, 56, 63, 32, 33, 34, 51, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2832(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 4, 21, 22, 7, 24, 15, 16, 17, 18, 3, 20, 19, 20, 5, 6, 23, 8, 63, 32, 33, 34, 51, 36, 35, 36, 53, 54, 39, 56, 47, 48, 49, 50, 35, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2833(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 4, 21, 22, 7, 24, 31, 16, 17, 18, 3, 20, 19, 20, 5, 6, 23, 8, 47, 32, 33, 34, 51, 36, 35, 36, 53, 54, 39, 56, 63, 48, 49, 50, 35, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2834(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 4, 21, 22, 7, 24, 15, 0, 17, 18, 3, 20, 19, 20, 5, 6, 23, 8, 63, 48, 33, 34, 51, 36, 35, 36, 53, 54, 39, 56, 47, 32, 49, 50, 35, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2835(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 4, 21, 22, 7, 24, 31, 0, 17, 18, 3, 20, 19, 20, 5, 6, 23, 8, 47, 48, 33, 34, 51, 36, 35, 36, 53, 54, 39, 56, 63, 32, 49, 50, 35, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2836(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 4, 21, 22, 7, 24, 15, 16, 1, 18, 3, 20, 19, 20, 5, 6, 23, 8, 63, 32, 49, 34, 51, 36, 35, 36, 53, 54, 39, 56, 47, 48, 33, 50, 35, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2837(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 4, 21, 22, 7, 24, 31, 16, 1, 18, 3, 20, 19, 20, 5, 6, 23, 8, 47, 32, 49, 34, 51, 36, 35, 36, 53, 54, 39, 56, 63, 48, 33, 50, 35, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2838(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 4, 21, 22, 7, 24, 15, 0, 1, 18, 3, 20, 19, 20, 5, 6, 23, 8, 63, 48, 49, 34, 51, 36, 35, 36, 53, 54, 39, 56, 47, 32, 33, 50, 35, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2839(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 4, 21, 22, 7, 24, 31, 0, 1, 18, 3, 20, 19, 20, 5, 6, 23, 8, 47, 48, 49, 34, 51, 36, 35, 36, 53, 54, 39, 56, 63, 32, 33, 50, 35, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2840(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 4, 21, 22, 7, 24, 15, 16, 17, 2, 3, 20, 19, 20, 5, 6, 23, 8, 63, 32, 33, 50, 51, 36, 35, 36, 53, 54, 39, 56, 47, 48, 49, 34, 35, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2841(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 4, 21, 22, 7, 24, 31, 16, 17, 2, 3, 20, 19, 20, 5, 6, 23, 8, 47, 32, 33, 50, 51, 36, 35, 36, 53, 54, 39, 56, 63, 48, 49, 34, 35, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2842(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 4, 21, 22, 7, 24, 15, 0, 17, 2, 3, 20, 19, 20, 5, 6, 23, 8, 63, 48, 33, 50, 51, 36, 35, 36, 53, 54, 39, 56, 47, 32, 49, 34, 35, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2843(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 4, 21, 22, 7, 24, 31, 0, 17, 2, 3, 20, 19, 20, 5, 6, 23, 8, 47, 48, 33, 50, 51, 36, 35, 36, 53, 54, 39, 56, 63, 32, 49, 34, 35, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2844(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 4, 21, 22, 7, 24, 15, 16, 1, 2, 3, 20, 19, 20, 5, 6, 23, 8, 63, 32, 49, 50, 51, 36, 35, 36, 53, 54, 39, 56, 47, 48, 33, 34, 35, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2845(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 4, 21, 22, 7, 24, 31, 16, 1, 2, 3, 20, 19, 20, 5, 6, 23, 8, 47, 32, 49, 50, 51, 36, 35, 36, 53, 54, 39, 56, 63, 48, 33, 34, 35, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2846(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 4, 21, 22, 7, 24, 15, 0, 1, 2, 3, 20, 19, 20, 5, 6, 23, 8, 63, 48, 49, 50, 51, 36, 35, 36, 53, 54, 39, 56, 47, 32, 33, 34, 35, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2847(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 4, 21, 22, 7, 24, 31, 0, 1, 2, 3, 20, 19, 20, 5, 6, 23, 8, 47, 48, 49, 50, 51, 36, 35, 36, 53, 54, 39, 56, 63, 32, 33, 34, 35, 52, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2848(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 4, 21, 22, 7, 24, 15, 16, 17, 18, 19, 4, 19, 20, 5, 6, 23, 8, 63, 32, 33, 34, 35, 52, 35, 36, 53, 54, 39, 56, 47, 48, 49, 50, 51, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2849(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 4, 21, 22, 7, 24, 31, 16, 17, 18, 19, 4, 19, 20, 5, 6, 23, 8, 47, 32, 33, 34, 35, 52, 35, 36, 53, 54, 39, 56, 63, 48, 49, 50, 51, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2850(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 4, 21, 22, 7, 24, 15, 0, 17, 18, 19, 4, 19, 20, 5, 6, 23, 8, 63, 48, 33, 34, 35, 52, 35, 36, 53, 54, 39, 56, 47, 32, 49, 50, 51, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2851(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 4, 21, 22, 7, 24, 31, 0, 17, 18, 19, 4, 19, 20, 5, 6, 23, 8, 47, 48, 33, 34, 35, 52, 35, 36, 53, 54, 39, 56, 63, 32, 49, 50, 51, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2852(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 4, 21, 22, 7, 24, 15, 16, 1, 18, 19, 4, 19, 20, 5, 6, 23, 8, 63, 32, 49, 34, 35, 52, 35, 36, 53, 54, 39, 56, 47, 48, 33, 50, 51, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2853(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 4, 21, 22, 7, 24, 31, 16, 1, 18, 19, 4, 19, 20, 5, 6, 23, 8, 47, 32, 49, 34, 35, 52, 35, 36, 53, 54, 39, 56, 63, 48, 33, 50, 51, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2854(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 4, 21, 22, 7, 24, 15, 0, 1, 18, 19, 4, 19, 20, 5, 6, 23, 8, 63, 48, 49, 34, 35, 52, 35, 36, 53, 54, 39, 56, 47, 32, 33, 50, 51, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2855(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 4, 21, 22, 7, 24, 31, 0, 1, 18, 19, 4, 19, 20, 5, 6, 23, 8, 47, 48, 49, 34, 35, 52, 35, 36, 53, 54, 39, 56, 63, 32, 33, 50, 51, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2856(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 4, 21, 22, 7, 24, 15, 16, 17, 2, 19, 4, 19, 20, 5, 6, 23, 8, 63, 32, 33, 50, 35, 52, 35, 36, 53, 54, 39, 56, 47, 48, 49, 34, 51, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2857(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 4, 21, 22, 7, 24, 31, 16, 17, 2, 19, 4, 19, 20, 5, 6, 23, 8, 47, 32, 33, 50, 35, 52, 35, 36, 53, 54, 39, 56, 63, 48, 49, 34, 51, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2858(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 4, 21, 22, 7, 24, 15, 0, 17, 2, 19, 4, 19, 20, 5, 6, 23, 8, 63, 48, 33, 50, 35, 52, 35, 36, 53, 54, 39, 56, 47, 32, 49, 34, 51, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2859(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 4, 21, 22, 7, 24, 31, 0, 17, 2, 19, 4, 19, 20, 5, 6, 23, 8, 47, 48, 33, 50, 35, 52, 35, 36, 53, 54, 39, 56, 63, 32, 49, 34, 51, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2860(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 4, 21, 22, 7, 24, 15, 16, 1, 2, 19, 4, 19, 20, 5, 6, 23, 8, 63, 32, 49, 50, 35, 52, 35, 36, 53, 54, 39, 56, 47, 48, 33, 34, 51, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2861(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 4, 21, 22, 7, 24, 31, 16, 1, 2, 19, 4, 19, 20, 5, 6, 23, 8, 47, 32, 49, 50, 35, 52, 35, 36, 53, 54, 39, 56, 63, 48, 33, 34, 51, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2862(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 4, 21, 22, 7, 24, 15, 0, 1, 2, 19, 4, 19, 20, 5, 6, 23, 8, 63, 48, 49, 50, 35, 52, 35, 36, 53, 54, 39, 56, 47, 32, 33, 34, 51, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2863(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 4, 21, 22, 7, 24, 31, 0, 1, 2, 19, 4, 19, 20, 5, 6, 23, 8, 47, 48, 49, 50, 35, 52, 35, 36, 53, 54, 39, 56, 63, 32, 33, 34, 51, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2864(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 4, 21, 22, 7, 24, 15, 16, 17, 18, 3, 4, 19, 20, 5, 6, 23, 8, 63, 32, 33, 34, 51, 52, 35, 36, 53, 54, 39, 56, 47, 48, 49, 50, 35, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2865(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 4, 21, 22, 7, 24, 31, 16, 17, 18, 3, 4, 19, 20, 5, 6, 23, 8, 47, 32, 33, 34, 51, 52, 35, 36, 53, 54, 39, 56, 63, 48, 49, 50, 35, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2866(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 4, 21, 22, 7, 24, 15, 0, 17, 18, 3, 4, 19, 20, 5, 6, 23, 8, 63, 48, 33, 34, 51, 52, 35, 36, 53, 54, 39, 56, 47, 32, 49, 50, 35, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2867(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 4, 21, 22, 7, 24, 31, 0, 17, 18, 3, 4, 19, 20, 5, 6, 23, 8, 47, 48, 33, 34, 51, 52, 35, 36, 53, 54, 39, 56, 63, 32, 49, 50, 35, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2868(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 4, 21, 22, 7, 24, 15, 16, 1, 18, 3, 4, 19, 20, 5, 6, 23, 8, 63, 32, 49, 34, 51, 52, 35, 36, 53, 54, 39, 56, 47, 48, 33, 50, 35, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2869(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 4, 21, 22, 7, 24, 31, 16, 1, 18, 3, 4, 19, 20, 5, 6, 23, 8, 47, 32, 49, 34, 51, 52, 35, 36, 53, 54, 39, 56, 63, 48, 33, 50, 35, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2870(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 4, 21, 22, 7, 24, 15, 0, 1, 18, 3, 4, 19, 20, 5, 6, 23, 8, 63, 48, 49, 34, 51, 52, 35, 36, 53, 54, 39, 56, 47, 32, 33, 50, 35, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2871(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 4, 21, 22, 7, 24, 31, 0, 1, 18, 3, 4, 19, 20, 5, 6, 23, 8, 47, 48, 49, 34, 51, 52, 35, 36, 53, 54, 39, 56, 63, 32, 33, 50, 35, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2872(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 4, 21, 22, 7, 24, 15, 16, 17, 2, 3, 4, 19, 20, 5, 6, 23, 8, 63, 32, 33, 50, 51, 52, 35, 36, 53, 54, 39, 56, 47, 48, 49, 34, 35, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2873(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 4, 21, 22, 7, 24, 31, 16, 17, 2, 3, 4, 19, 20, 5, 6, 23, 8, 47, 32, 33, 50, 51, 52, 35, 36, 53, 54, 39, 56, 63, 48, 49, 34, 35, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2874(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 4, 21, 22, 7, 24, 15, 0, 17, 2, 3, 4, 19, 20, 5, 6, 23, 8, 63, 48, 33, 50, 51, 52, 35, 36, 53, 54, 39, 56, 47, 32, 49, 34, 35, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2875(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 4, 21, 22, 7, 24, 31, 0, 17, 2, 3, 4, 19, 20, 5, 6, 23, 8, 47, 48, 33, 50, 51, 52, 35, 36, 53, 54, 39, 56, 63, 32, 49, 34, 35, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2876(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 4, 21, 22, 7, 24, 15, 16, 1, 2, 3, 4, 19, 20, 5, 6, 23, 8, 63, 32, 49, 50, 51, 52, 35, 36, 53, 54, 39, 56, 47, 48, 33, 34, 35, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2877(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 4, 21, 22, 7, 24, 31, 16, 1, 2, 3, 4, 19, 20, 5, 6, 23, 8, 47, 32, 49, 50, 51, 52, 35, 36, 53, 54, 39, 56, 63, 48, 33, 34, 35, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2878(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 4, 21, 22, 7, 24, 15, 0, 1, 2, 3, 4, 19, 20, 5, 6, 23, 8, 63, 48, 49, 50, 51, 52, 35, 36, 53, 54, 39, 56, 47, 32, 33, 34, 35, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2879(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 4, 21, 22, 7, 24, 31, 0, 1, 2, 3, 4, 19, 20, 5, 6, 23, 8, 47, 48, 49, 50, 51, 52, 35, 36, 53, 54, 39, 56, 63, 32, 33, 34, 35, 36, 51, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2880(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 4, 21, 22, 7, 24, 15, 16, 17, 18, 19, 20, 3, 20, 5, 6, 23, 8, 63, 32, 33, 34, 35, 36, 51, 36, 53, 54, 39, 56, 47, 48, 49, 50, 51, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2881(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 4, 21, 22, 7, 24, 31, 16, 17, 18, 19, 20, 3, 20, 5, 6, 23, 8, 47, 32, 33, 34, 35, 36, 51, 36, 53, 54, 39, 56, 63, 48, 49, 50, 51, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2882(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 4, 21, 22, 7, 24, 15, 0, 17, 18, 19, 20, 3, 20, 5, 6, 23, 8, 63, 48, 33, 34, 35, 36, 51, 36, 53, 54, 39, 56, 47, 32, 49, 50, 51, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2883(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 4, 21, 22, 7, 24, 31, 0, 17, 18, 19, 20, 3, 20, 5, 6, 23, 8, 47, 48, 33, 34, 35, 36, 51, 36, 53, 54, 39, 56, 63, 32, 49, 50, 51, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2884(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 4, 21, 22, 7, 24, 15, 16, 1, 18, 19, 20, 3, 20, 5, 6, 23, 8, 63, 32, 49, 34, 35, 36, 51, 36, 53, 54, 39, 56, 47, 48, 33, 50, 51, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2885(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 4, 21, 22, 7, 24, 31, 16, 1, 18, 19, 20, 3, 20, 5, 6, 23, 8, 47, 32, 49, 34, 35, 36, 51, 36, 53, 54, 39, 56, 63, 48, 33, 50, 51, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2886(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 4, 21, 22, 7, 24, 15, 0, 1, 18, 19, 20, 3, 20, 5, 6, 23, 8, 63, 48, 49, 34, 35, 36, 51, 36, 53, 54, 39, 56, 47, 32, 33, 50, 51, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2887(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 4, 21, 22, 7, 24, 31, 0, 1, 18, 19, 20, 3, 20, 5, 6, 23, 8, 47, 48, 49, 34, 35, 36, 51, 36, 53, 54, 39, 56, 63, 32, 33, 50, 51, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2888(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 4, 21, 22, 7, 24, 15, 16, 17, 2, 19, 20, 3, 20, 5, 6, 23, 8, 63, 32, 33, 50, 35, 36, 51, 36, 53, 54, 39, 56, 47, 48, 49, 34, 51, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2889(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 4, 21, 22, 7, 24, 31, 16, 17, 2, 19, 20, 3, 20, 5, 6, 23, 8, 47, 32, 33, 50, 35, 36, 51, 36, 53, 54, 39, 56, 63, 48, 49, 34, 51, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2890(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 4, 21, 22, 7, 24, 15, 0, 17, 2, 19, 20, 3, 20, 5, 6, 23, 8, 63, 48, 33, 50, 35, 36, 51, 36, 53, 54, 39, 56, 47, 32, 49, 34, 51, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2891(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 4, 21, 22, 7, 24, 31, 0, 17, 2, 19, 20, 3, 20, 5, 6, 23, 8, 47, 48, 33, 50, 35, 36, 51, 36, 53, 54, 39, 56, 63, 32, 49, 34, 51, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2892(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 4, 21, 22, 7, 24, 15, 16, 1, 2, 19, 20, 3, 20, 5, 6, 23, 8, 63, 32, 49, 50, 35, 36, 51, 36, 53, 54, 39, 56, 47, 48, 33, 34, 51, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2893(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 4, 21, 22, 7, 24, 31, 16, 1, 2, 19, 20, 3, 20, 5, 6, 23, 8, 47, 32, 49, 50, 35, 36, 51, 36, 53, 54, 39, 56, 63, 48, 33, 34, 51, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2894(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 4, 21, 22, 7, 24, 15, 0, 1, 2, 19, 20, 3, 20, 5, 6, 23, 8, 63, 48, 49, 50, 35, 36, 51, 36, 53, 54, 39, 56, 47, 32, 33, 34, 51, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2895(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 4, 21, 22, 7, 24, 31, 0, 1, 2, 19, 20, 3, 20, 5, 6, 23, 8, 47, 48, 49, 50, 35, 36, 51, 36, 53, 54, 39, 56, 63, 32, 33, 34, 51, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2896(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 4, 21, 22, 7, 24, 15, 16, 17, 18, 3, 20, 3, 20, 5, 6, 23, 8, 63, 32, 33, 34, 51, 36, 51, 36, 53, 54, 39, 56, 47, 48, 49, 50, 35, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2897(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 4, 21, 22, 7, 24, 31, 16, 17, 18, 3, 20, 3, 20, 5, 6, 23, 8, 47, 32, 33, 34, 51, 36, 51, 36, 53, 54, 39, 56, 63, 48, 49, 50, 35, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2898(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 4, 21, 22, 7, 24, 15, 0, 17, 18, 3, 20, 3, 20, 5, 6, 23, 8, 63, 48, 33, 34, 51, 36, 51, 36, 53, 54, 39, 56, 47, 32, 49, 50, 35, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2899(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 4, 21, 22, 7, 24, 31, 0, 17, 18, 3, 20, 3, 20, 5, 6, 23, 8, 47, 48, 33, 34, 51, 36, 51, 36, 53, 54, 39, 56, 63, 32, 49, 50, 35, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2900(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 4, 21, 22, 7, 24, 15, 16, 1, 18, 3, 20, 3, 20, 5, 6, 23, 8, 63, 32, 49, 34, 51, 36, 51, 36, 53, 54, 39, 56, 47, 48, 33, 50, 35, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2901(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 4, 21, 22, 7, 24, 31, 16, 1, 18, 3, 20, 3, 20, 5, 6, 23, 8, 47, 32, 49, 34, 51, 36, 51, 36, 53, 54, 39, 56, 63, 48, 33, 50, 35, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2902(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 4, 21, 22, 7, 24, 15, 0, 1, 18, 3, 20, 3, 20, 5, 6, 23, 8, 63, 48, 49, 34, 51, 36, 51, 36, 53, 54, 39, 56, 47, 32, 33, 50, 35, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2903(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 4, 21, 22, 7, 24, 31, 0, 1, 18, 3, 20, 3, 20, 5, 6, 23, 8, 47, 48, 49, 34, 51, 36, 51, 36, 53, 54, 39, 56, 63, 32, 33, 50, 35, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2904(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 4, 21, 22, 7, 24, 15, 16, 17, 2, 3, 20, 3, 20, 5, 6, 23, 8, 63, 32, 33, 50, 51, 36, 51, 36, 53, 54, 39, 56, 47, 48, 49, 34, 35, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2905(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 4, 21, 22, 7, 24, 31, 16, 17, 2, 3, 20, 3, 20, 5, 6, 23, 8, 47, 32, 33, 50, 51, 36, 51, 36, 53, 54, 39, 56, 63, 48, 49, 34, 35, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2906(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 4, 21, 22, 7, 24, 15, 0, 17, 2, 3, 20, 3, 20, 5, 6, 23, 8, 63, 48, 33, 50, 51, 36, 51, 36, 53, 54, 39, 56, 47, 32, 49, 34, 35, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2907(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 4, 21, 22, 7, 24, 31, 0, 17, 2, 3, 20, 3, 20, 5, 6, 23, 8, 47, 48, 33, 50, 51, 36, 51, 36, 53, 54, 39, 56, 63, 32, 49, 34, 35, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2908(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 4, 21, 22, 7, 24, 15, 16, 1, 2, 3, 20, 3, 20, 5, 6, 23, 8, 63, 32, 49, 50, 51, 36, 51, 36, 53, 54, 39, 56, 47, 48, 33, 34, 35, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2909(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 4, 21, 22, 7, 24, 31, 16, 1, 2, 3, 20, 3, 20, 5, 6, 23, 8, 47, 32, 49, 50, 51, 36, 51, 36, 53, 54, 39, 56, 63, 48, 33, 34, 35, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2910(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 4, 21, 22, 7, 24, 15, 0, 1, 2, 3, 20, 3, 20, 5, 6, 23, 8, 63, 48, 49, 50, 51, 36, 51, 36, 53, 54, 39, 56, 47, 32, 33, 34, 35, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2911(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 4, 21, 22, 7, 24, 31, 0, 1, 2, 3, 20, 3, 20, 5, 6, 23, 8, 47, 48, 49, 50, 51, 36, 51, 36, 53, 54, 39, 56, 63, 32, 33, 34, 35, 52, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2912(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 4, 21, 22, 7, 24, 15, 16, 17, 18, 19, 4, 3, 20, 5, 6, 23, 8, 63, 32, 33, 34, 35, 52, 51, 36, 53, 54, 39, 56, 47, 48, 49, 50, 51, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2913(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 4, 21, 22, 7, 24, 31, 16, 17, 18, 19, 4, 3, 20, 5, 6, 23, 8, 47, 32, 33, 34, 35, 52, 51, 36, 53, 54, 39, 56, 63, 48, 49, 50, 51, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2914(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 4, 21, 22, 7, 24, 15, 0, 17, 18, 19, 4, 3, 20, 5, 6, 23, 8, 63, 48, 33, 34, 35, 52, 51, 36, 53, 54, 39, 56, 47, 32, 49, 50, 51, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2915(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 4, 21, 22, 7, 24, 31, 0, 17, 18, 19, 4, 3, 20, 5, 6, 23, 8, 47, 48, 33, 34, 35, 52, 51, 36, 53, 54, 39, 56, 63, 32, 49, 50, 51, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2916(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 4, 21, 22, 7, 24, 15, 16, 1, 18, 19, 4, 3, 20, 5, 6, 23, 8, 63, 32, 49, 34, 35, 52, 51, 36, 53, 54, 39, 56, 47, 48, 33, 50, 51, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2917(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 4, 21, 22, 7, 24, 31, 16, 1, 18, 19, 4, 3, 20, 5, 6, 23, 8, 47, 32, 49, 34, 35, 52, 51, 36, 53, 54, 39, 56, 63, 48, 33, 50, 51, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2918(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 4, 21, 22, 7, 24, 15, 0, 1, 18, 19, 4, 3, 20, 5, 6, 23, 8, 63, 48, 49, 34, 35, 52, 51, 36, 53, 54, 39, 56, 47, 32, 33, 50, 51, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2919(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 4, 21, 22, 7, 24, 31, 0, 1, 18, 19, 4, 3, 20, 5, 6, 23, 8, 47, 48, 49, 34, 35, 52, 51, 36, 53, 54, 39, 56, 63, 32, 33, 50, 51, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2920(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 4, 21, 22, 7, 24, 15, 16, 17, 2, 19, 4, 3, 20, 5, 6, 23, 8, 63, 32, 33, 50, 35, 52, 51, 36, 53, 54, 39, 56, 47, 48, 49, 34, 51, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2921(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 4, 21, 22, 7, 24, 31, 16, 17, 2, 19, 4, 3, 20, 5, 6, 23, 8, 47, 32, 33, 50, 35, 52, 51, 36, 53, 54, 39, 56, 63, 48, 49, 34, 51, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2922(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 4, 21, 22, 7, 24, 15, 0, 17, 2, 19, 4, 3, 20, 5, 6, 23, 8, 63, 48, 33, 50, 35, 52, 51, 36, 53, 54, 39, 56, 47, 32, 49, 34, 51, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2923(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 4, 21, 22, 7, 24, 31, 0, 17, 2, 19, 4, 3, 20, 5, 6, 23, 8, 47, 48, 33, 50, 35, 52, 51, 36, 53, 54, 39, 56, 63, 32, 49, 34, 51, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2924(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 4, 21, 22, 7, 24, 15, 16, 1, 2, 19, 4, 3, 20, 5, 6, 23, 8, 63, 32, 49, 50, 35, 52, 51, 36, 53, 54, 39, 56, 47, 48, 33, 34, 51, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2925(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 4, 21, 22, 7, 24, 31, 16, 1, 2, 19, 4, 3, 20, 5, 6, 23, 8, 47, 32, 49, 50, 35, 52, 51, 36, 53, 54, 39, 56, 63, 48, 33, 34, 51, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2926(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 4, 21, 22, 7, 24, 15, 0, 1, 2, 19, 4, 3, 20, 5, 6, 23, 8, 63, 48, 49, 50, 35, 52, 51, 36, 53, 54, 39, 56, 47, 32, 33, 34, 51, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2927(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 4, 21, 22, 7, 24, 31, 0, 1, 2, 19, 4, 3, 20, 5, 6, 23, 8, 47, 48, 49, 50, 35, 52, 51, 36, 53, 54, 39, 56, 63, 32, 33, 34, 51, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2928(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 4, 21, 22, 7, 24, 15, 16, 17, 18, 3, 4, 3, 20, 5, 6, 23, 8, 63, 32, 33, 34, 51, 52, 51, 36, 53, 54, 39, 56, 47, 48, 49, 50, 35, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2929(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 4, 21, 22, 7, 24, 31, 16, 17, 18, 3, 4, 3, 20, 5, 6, 23, 8, 47, 32, 33, 34, 51, 52, 51, 36, 53, 54, 39, 56, 63, 48, 49, 50, 35, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2930(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 4, 21, 22, 7, 24, 15, 0, 17, 18, 3, 4, 3, 20, 5, 6, 23, 8, 63, 48, 33, 34, 51, 52, 51, 36, 53, 54, 39, 56, 47, 32, 49, 50, 35, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2931(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 4, 21, 22, 7, 24, 31, 0, 17, 18, 3, 4, 3, 20, 5, 6, 23, 8, 47, 48, 33, 34, 51, 52, 51, 36, 53, 54, 39, 56, 63, 32, 49, 50, 35, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2932(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 4, 21, 22, 7, 24, 15, 16, 1, 18, 3, 4, 3, 20, 5, 6, 23, 8, 63, 32, 49, 34, 51, 52, 51, 36, 53, 54, 39, 56, 47, 48, 33, 50, 35, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2933(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 4, 21, 22, 7, 24, 31, 16, 1, 18, 3, 4, 3, 20, 5, 6, 23, 8, 47, 32, 49, 34, 51, 52, 51, 36, 53, 54, 39, 56, 63, 48, 33, 50, 35, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2934(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 4, 21, 22, 7, 24, 15, 0, 1, 18, 3, 4, 3, 20, 5, 6, 23, 8, 63, 48, 49, 34, 51, 52, 51, 36, 53, 54, 39, 56, 47, 32, 33, 50, 35, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2935(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 4, 21, 22, 7, 24, 31, 0, 1, 18, 3, 4, 3, 20, 5, 6, 23, 8, 47, 48, 49, 34, 51, 52, 51, 36, 53, 54, 39, 56, 63, 32, 33, 50, 35, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2936(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 4, 21, 22, 7, 24, 15, 16, 17, 2, 3, 4, 3, 20, 5, 6, 23, 8, 63, 32, 33, 50, 51, 52, 51, 36, 53, 54, 39, 56, 47, 48, 49, 34, 35, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2937(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 4, 21, 22, 7, 24, 31, 16, 17, 2, 3, 4, 3, 20, 5, 6, 23, 8, 47, 32, 33, 50, 51, 52, 51, 36, 53, 54, 39, 56, 63, 48, 49, 34, 35, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2938(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 4, 21, 22, 7, 24, 15, 0, 17, 2, 3, 4, 3, 20, 5, 6, 23, 8, 63, 48, 33, 50, 51, 52, 51, 36, 53, 54, 39, 56, 47, 32, 49, 34, 35, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2939(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 4, 21, 22, 7, 24, 31, 0, 17, 2, 3, 4, 3, 20, 5, 6, 23, 8, 47, 48, 33, 50, 51, 52, 51, 36, 53, 54, 39, 56, 63, 32, 49, 34, 35, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2940(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 4, 21, 22, 7, 24, 15, 16, 1, 2, 3, 4, 3, 20, 5, 6, 23, 8, 63, 32, 49, 50, 51, 52, 51, 36, 53, 54, 39, 56, 47, 48, 33, 34, 35, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2941(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 4, 21, 22, 7, 24, 31, 16, 1, 2, 3, 4, 3, 20, 5, 6, 23, 8, 47, 32, 49, 50, 51, 52, 51, 36, 53, 54, 39, 56, 63, 48, 33, 34, 35, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2942(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 4, 21, 22, 7, 24, 15, 0, 1, 2, 3, 4, 3, 20, 5, 6, 23, 8, 63, 48, 49, 50, 51, 52, 51, 36, 53, 54, 39, 56, 47, 32, 33, 34, 35, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2943(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 4, 21, 22, 7, 24, 31, 0, 1, 2, 3, 4, 3, 20, 5, 6, 23, 8, 47, 48, 49, 50, 51, 52, 51, 36, 53, 54, 39, 56, 63, 32, 33, 34, 35, 36, 35, 52, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}


