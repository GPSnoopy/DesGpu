#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt3456(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 21, 6, 23, 24, 15, 16, 17, 18, 19, 20, 19, 4, 5, 22, 7, 8, 63, 32, 33, 34, 35, 36, 35, 52, 53, 38, 55, 56, 47, 48, 49, 50, 51, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3457(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 21, 6, 23, 24, 31, 16, 17, 18, 19, 20, 19, 4, 5, 22, 7, 8, 47, 32, 33, 34, 35, 36, 35, 52, 53, 38, 55, 56, 63, 48, 49, 50, 51, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3458(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 21, 6, 23, 24, 15, 0, 17, 18, 19, 20, 19, 4, 5, 22, 7, 8, 63, 48, 33, 34, 35, 36, 35, 52, 53, 38, 55, 56, 47, 32, 49, 50, 51, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3459(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 21, 6, 23, 24, 31, 0, 17, 18, 19, 20, 19, 4, 5, 22, 7, 8, 47, 48, 33, 34, 35, 36, 35, 52, 53, 38, 55, 56, 63, 32, 49, 50, 51, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3460(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 21, 6, 23, 24, 15, 16, 1, 18, 19, 20, 19, 4, 5, 22, 7, 8, 63, 32, 49, 34, 35, 36, 35, 52, 53, 38, 55, 56, 47, 48, 33, 50, 51, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3461(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 21, 6, 23, 24, 31, 16, 1, 18, 19, 20, 19, 4, 5, 22, 7, 8, 47, 32, 49, 34, 35, 36, 35, 52, 53, 38, 55, 56, 63, 48, 33, 50, 51, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3462(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 21, 6, 23, 24, 15, 0, 1, 18, 19, 20, 19, 4, 5, 22, 7, 8, 63, 48, 49, 34, 35, 36, 35, 52, 53, 38, 55, 56, 47, 32, 33, 50, 51, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3463(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 21, 6, 23, 24, 31, 0, 1, 18, 19, 20, 19, 4, 5, 22, 7, 8, 47, 48, 49, 34, 35, 36, 35, 52, 53, 38, 55, 56, 63, 32, 33, 50, 51, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3464(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 21, 6, 23, 24, 15, 16, 17, 2, 19, 20, 19, 4, 5, 22, 7, 8, 63, 32, 33, 50, 35, 36, 35, 52, 53, 38, 55, 56, 47, 48, 49, 34, 51, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3465(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 21, 6, 23, 24, 31, 16, 17, 2, 19, 20, 19, 4, 5, 22, 7, 8, 47, 32, 33, 50, 35, 36, 35, 52, 53, 38, 55, 56, 63, 48, 49, 34, 51, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3466(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 21, 6, 23, 24, 15, 0, 17, 2, 19, 20, 19, 4, 5, 22, 7, 8, 63, 48, 33, 50, 35, 36, 35, 52, 53, 38, 55, 56, 47, 32, 49, 34, 51, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3467(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 21, 6, 23, 24, 31, 0, 17, 2, 19, 20, 19, 4, 5, 22, 7, 8, 47, 48, 33, 50, 35, 36, 35, 52, 53, 38, 55, 56, 63, 32, 49, 34, 51, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3468(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 21, 6, 23, 24, 15, 16, 1, 2, 19, 20, 19, 4, 5, 22, 7, 8, 63, 32, 49, 50, 35, 36, 35, 52, 53, 38, 55, 56, 47, 48, 33, 34, 51, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3469(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 21, 6, 23, 24, 31, 16, 1, 2, 19, 20, 19, 4, 5, 22, 7, 8, 47, 32, 49, 50, 35, 36, 35, 52, 53, 38, 55, 56, 63, 48, 33, 34, 51, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3470(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 21, 6, 23, 24, 15, 0, 1, 2, 19, 20, 19, 4, 5, 22, 7, 8, 63, 48, 49, 50, 35, 36, 35, 52, 53, 38, 55, 56, 47, 32, 33, 34, 51, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3471(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 21, 6, 23, 24, 31, 0, 1, 2, 19, 20, 19, 4, 5, 22, 7, 8, 47, 48, 49, 50, 35, 36, 35, 52, 53, 38, 55, 56, 63, 32, 33, 34, 51, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3472(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 21, 6, 23, 24, 15, 16, 17, 18, 3, 20, 19, 4, 5, 22, 7, 8, 63, 32, 33, 34, 51, 36, 35, 52, 53, 38, 55, 56, 47, 48, 49, 50, 35, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3473(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 21, 6, 23, 24, 31, 16, 17, 18, 3, 20, 19, 4, 5, 22, 7, 8, 47, 32, 33, 34, 51, 36, 35, 52, 53, 38, 55, 56, 63, 48, 49, 50, 35, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3474(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 21, 6, 23, 24, 15, 0, 17, 18, 3, 20, 19, 4, 5, 22, 7, 8, 63, 48, 33, 34, 51, 36, 35, 52, 53, 38, 55, 56, 47, 32, 49, 50, 35, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3475(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 21, 6, 23, 24, 31, 0, 17, 18, 3, 20, 19, 4, 5, 22, 7, 8, 47, 48, 33, 34, 51, 36, 35, 52, 53, 38, 55, 56, 63, 32, 49, 50, 35, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3476(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 21, 6, 23, 24, 15, 16, 1, 18, 3, 20, 19, 4, 5, 22, 7, 8, 63, 32, 49, 34, 51, 36, 35, 52, 53, 38, 55, 56, 47, 48, 33, 50, 35, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3477(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 21, 6, 23, 24, 31, 16, 1, 18, 3, 20, 19, 4, 5, 22, 7, 8, 47, 32, 49, 34, 51, 36, 35, 52, 53, 38, 55, 56, 63, 48, 33, 50, 35, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3478(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 21, 6, 23, 24, 15, 0, 1, 18, 3, 20, 19, 4, 5, 22, 7, 8, 63, 48, 49, 34, 51, 36, 35, 52, 53, 38, 55, 56, 47, 32, 33, 50, 35, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3479(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 21, 6, 23, 24, 31, 0, 1, 18, 3, 20, 19, 4, 5, 22, 7, 8, 47, 48, 49, 34, 51, 36, 35, 52, 53, 38, 55, 56, 63, 32, 33, 50, 35, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3480(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 21, 6, 23, 24, 15, 16, 17, 2, 3, 20, 19, 4, 5, 22, 7, 8, 63, 32, 33, 50, 51, 36, 35, 52, 53, 38, 55, 56, 47, 48, 49, 34, 35, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3481(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 21, 6, 23, 24, 31, 16, 17, 2, 3, 20, 19, 4, 5, 22, 7, 8, 47, 32, 33, 50, 51, 36, 35, 52, 53, 38, 55, 56, 63, 48, 49, 34, 35, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3482(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 21, 6, 23, 24, 15, 0, 17, 2, 3, 20, 19, 4, 5, 22, 7, 8, 63, 48, 33, 50, 51, 36, 35, 52, 53, 38, 55, 56, 47, 32, 49, 34, 35, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3483(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 21, 6, 23, 24, 31, 0, 17, 2, 3, 20, 19, 4, 5, 22, 7, 8, 47, 48, 33, 50, 51, 36, 35, 52, 53, 38, 55, 56, 63, 32, 49, 34, 35, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3484(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 21, 6, 23, 24, 15, 16, 1, 2, 3, 20, 19, 4, 5, 22, 7, 8, 63, 32, 49, 50, 51, 36, 35, 52, 53, 38, 55, 56, 47, 48, 33, 34, 35, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3485(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 21, 6, 23, 24, 31, 16, 1, 2, 3, 20, 19, 4, 5, 22, 7, 8, 47, 32, 49, 50, 51, 36, 35, 52, 53, 38, 55, 56, 63, 48, 33, 34, 35, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3486(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 21, 6, 23, 24, 15, 0, 1, 2, 3, 20, 19, 4, 5, 22, 7, 8, 63, 48, 49, 50, 51, 36, 35, 52, 53, 38, 55, 56, 47, 32, 33, 34, 35, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3487(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 21, 6, 23, 24, 31, 0, 1, 2, 3, 20, 19, 4, 5, 22, 7, 8, 47, 48, 49, 50, 51, 36, 35, 52, 53, 38, 55, 56, 63, 32, 33, 34, 35, 52, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3488(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 21, 6, 23, 24, 15, 16, 17, 18, 19, 4, 19, 4, 5, 22, 7, 8, 63, 32, 33, 34, 35, 52, 35, 52, 53, 38, 55, 56, 47, 48, 49, 50, 51, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3489(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 21, 6, 23, 24, 31, 16, 17, 18, 19, 4, 19, 4, 5, 22, 7, 8, 47, 32, 33, 34, 35, 52, 35, 52, 53, 38, 55, 56, 63, 48, 49, 50, 51, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3490(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 21, 6, 23, 24, 15, 0, 17, 18, 19, 4, 19, 4, 5, 22, 7, 8, 63, 48, 33, 34, 35, 52, 35, 52, 53, 38, 55, 56, 47, 32, 49, 50, 51, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3491(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 21, 6, 23, 24, 31, 0, 17, 18, 19, 4, 19, 4, 5, 22, 7, 8, 47, 48, 33, 34, 35, 52, 35, 52, 53, 38, 55, 56, 63, 32, 49, 50, 51, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3492(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 21, 6, 23, 24, 15, 16, 1, 18, 19, 4, 19, 4, 5, 22, 7, 8, 63, 32, 49, 34, 35, 52, 35, 52, 53, 38, 55, 56, 47, 48, 33, 50, 51, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3493(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 21, 6, 23, 24, 31, 16, 1, 18, 19, 4, 19, 4, 5, 22, 7, 8, 47, 32, 49, 34, 35, 52, 35, 52, 53, 38, 55, 56, 63, 48, 33, 50, 51, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3494(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 21, 6, 23, 24, 15, 0, 1, 18, 19, 4, 19, 4, 5, 22, 7, 8, 63, 48, 49, 34, 35, 52, 35, 52, 53, 38, 55, 56, 47, 32, 33, 50, 51, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3495(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 21, 6, 23, 24, 31, 0, 1, 18, 19, 4, 19, 4, 5, 22, 7, 8, 47, 48, 49, 34, 35, 52, 35, 52, 53, 38, 55, 56, 63, 32, 33, 50, 51, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3496(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 21, 6, 23, 24, 15, 16, 17, 2, 19, 4, 19, 4, 5, 22, 7, 8, 63, 32, 33, 50, 35, 52, 35, 52, 53, 38, 55, 56, 47, 48, 49, 34, 51, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3497(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 21, 6, 23, 24, 31, 16, 17, 2, 19, 4, 19, 4, 5, 22, 7, 8, 47, 32, 33, 50, 35, 52, 35, 52, 53, 38, 55, 56, 63, 48, 49, 34, 51, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3498(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 21, 6, 23, 24, 15, 0, 17, 2, 19, 4, 19, 4, 5, 22, 7, 8, 63, 48, 33, 50, 35, 52, 35, 52, 53, 38, 55, 56, 47, 32, 49, 34, 51, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3499(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 21, 6, 23, 24, 31, 0, 17, 2, 19, 4, 19, 4, 5, 22, 7, 8, 47, 48, 33, 50, 35, 52, 35, 52, 53, 38, 55, 56, 63, 32, 49, 34, 51, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3500(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 21, 6, 23, 24, 15, 16, 1, 2, 19, 4, 19, 4, 5, 22, 7, 8, 63, 32, 49, 50, 35, 52, 35, 52, 53, 38, 55, 56, 47, 48, 33, 34, 51, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3501(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 21, 6, 23, 24, 31, 16, 1, 2, 19, 4, 19, 4, 5, 22, 7, 8, 47, 32, 49, 50, 35, 52, 35, 52, 53, 38, 55, 56, 63, 48, 33, 34, 51, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3502(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 21, 6, 23, 24, 15, 0, 1, 2, 19, 4, 19, 4, 5, 22, 7, 8, 63, 48, 49, 50, 35, 52, 35, 52, 53, 38, 55, 56, 47, 32, 33, 34, 51, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3503(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 21, 6, 23, 24, 31, 0, 1, 2, 19, 4, 19, 4, 5, 22, 7, 8, 47, 48, 49, 50, 35, 52, 35, 52, 53, 38, 55, 56, 63, 32, 33, 34, 51, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3504(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 21, 6, 23, 24, 15, 16, 17, 18, 3, 4, 19, 4, 5, 22, 7, 8, 63, 32, 33, 34, 51, 52, 35, 52, 53, 38, 55, 56, 47, 48, 49, 50, 35, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3505(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 21, 6, 23, 24, 31, 16, 17, 18, 3, 4, 19, 4, 5, 22, 7, 8, 47, 32, 33, 34, 51, 52, 35, 52, 53, 38, 55, 56, 63, 48, 49, 50, 35, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3506(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 21, 6, 23, 24, 15, 0, 17, 18, 3, 4, 19, 4, 5, 22, 7, 8, 63, 48, 33, 34, 51, 52, 35, 52, 53, 38, 55, 56, 47, 32, 49, 50, 35, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3507(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 21, 6, 23, 24, 31, 0, 17, 18, 3, 4, 19, 4, 5, 22, 7, 8, 47, 48, 33, 34, 51, 52, 35, 52, 53, 38, 55, 56, 63, 32, 49, 50, 35, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3508(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 21, 6, 23, 24, 15, 16, 1, 18, 3, 4, 19, 4, 5, 22, 7, 8, 63, 32, 49, 34, 51, 52, 35, 52, 53, 38, 55, 56, 47, 48, 33, 50, 35, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3509(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 21, 6, 23, 24, 31, 16, 1, 18, 3, 4, 19, 4, 5, 22, 7, 8, 47, 32, 49, 34, 51, 52, 35, 52, 53, 38, 55, 56, 63, 48, 33, 50, 35, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3510(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 21, 6, 23, 24, 15, 0, 1, 18, 3, 4, 19, 4, 5, 22, 7, 8, 63, 48, 49, 34, 51, 52, 35, 52, 53, 38, 55, 56, 47, 32, 33, 50, 35, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3511(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 21, 6, 23, 24, 31, 0, 1, 18, 3, 4, 19, 4, 5, 22, 7, 8, 47, 48, 49, 34, 51, 52, 35, 52, 53, 38, 55, 56, 63, 32, 33, 50, 35, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3512(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 21, 6, 23, 24, 15, 16, 17, 2, 3, 4, 19, 4, 5, 22, 7, 8, 63, 32, 33, 50, 51, 52, 35, 52, 53, 38, 55, 56, 47, 48, 49, 34, 35, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3513(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 21, 6, 23, 24, 31, 16, 17, 2, 3, 4, 19, 4, 5, 22, 7, 8, 47, 32, 33, 50, 51, 52, 35, 52, 53, 38, 55, 56, 63, 48, 49, 34, 35, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3514(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 21, 6, 23, 24, 15, 0, 17, 2, 3, 4, 19, 4, 5, 22, 7, 8, 63, 48, 33, 50, 51, 52, 35, 52, 53, 38, 55, 56, 47, 32, 49, 34, 35, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3515(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 21, 6, 23, 24, 31, 0, 17, 2, 3, 4, 19, 4, 5, 22, 7, 8, 47, 48, 33, 50, 51, 52, 35, 52, 53, 38, 55, 56, 63, 32, 49, 34, 35, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3516(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 21, 6, 23, 24, 15, 16, 1, 2, 3, 4, 19, 4, 5, 22, 7, 8, 63, 32, 49, 50, 51, 52, 35, 52, 53, 38, 55, 56, 47, 48, 33, 34, 35, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3517(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 21, 6, 23, 24, 31, 16, 1, 2, 3, 4, 19, 4, 5, 22, 7, 8, 47, 32, 49, 50, 51, 52, 35, 52, 53, 38, 55, 56, 63, 48, 33, 34, 35, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3518(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 21, 6, 23, 24, 15, 0, 1, 2, 3, 4, 19, 4, 5, 22, 7, 8, 63, 48, 49, 50, 51, 52, 35, 52, 53, 38, 55, 56, 47, 32, 33, 34, 35, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3519(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 21, 6, 23, 24, 31, 0, 1, 2, 3, 4, 19, 4, 5, 22, 7, 8, 47, 48, 49, 50, 51, 52, 35, 52, 53, 38, 55, 56, 63, 32, 33, 34, 35, 36, 51, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3520(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 21, 6, 23, 24, 15, 16, 17, 18, 19, 20, 3, 4, 5, 22, 7, 8, 63, 32, 33, 34, 35, 36, 51, 52, 53, 38, 55, 56, 47, 48, 49, 50, 51, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3521(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 21, 6, 23, 24, 31, 16, 17, 18, 19, 20, 3, 4, 5, 22, 7, 8, 47, 32, 33, 34, 35, 36, 51, 52, 53, 38, 55, 56, 63, 48, 49, 50, 51, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3522(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 21, 6, 23, 24, 15, 0, 17, 18, 19, 20, 3, 4, 5, 22, 7, 8, 63, 48, 33, 34, 35, 36, 51, 52, 53, 38, 55, 56, 47, 32, 49, 50, 51, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3523(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 21, 6, 23, 24, 31, 0, 17, 18, 19, 20, 3, 4, 5, 22, 7, 8, 47, 48, 33, 34, 35, 36, 51, 52, 53, 38, 55, 56, 63, 32, 49, 50, 51, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3524(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 21, 6, 23, 24, 15, 16, 1, 18, 19, 20, 3, 4, 5, 22, 7, 8, 63, 32, 49, 34, 35, 36, 51, 52, 53, 38, 55, 56, 47, 48, 33, 50, 51, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3525(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 21, 6, 23, 24, 31, 16, 1, 18, 19, 20, 3, 4, 5, 22, 7, 8, 47, 32, 49, 34, 35, 36, 51, 52, 53, 38, 55, 56, 63, 48, 33, 50, 51, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3526(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 21, 6, 23, 24, 15, 0, 1, 18, 19, 20, 3, 4, 5, 22, 7, 8, 63, 48, 49, 34, 35, 36, 51, 52, 53, 38, 55, 56, 47, 32, 33, 50, 51, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3527(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 21, 6, 23, 24, 31, 0, 1, 18, 19, 20, 3, 4, 5, 22, 7, 8, 47, 48, 49, 34, 35, 36, 51, 52, 53, 38, 55, 56, 63, 32, 33, 50, 51, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3528(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 21, 6, 23, 24, 15, 16, 17, 2, 19, 20, 3, 4, 5, 22, 7, 8, 63, 32, 33, 50, 35, 36, 51, 52, 53, 38, 55, 56, 47, 48, 49, 34, 51, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3529(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 21, 6, 23, 24, 31, 16, 17, 2, 19, 20, 3, 4, 5, 22, 7, 8, 47, 32, 33, 50, 35, 36, 51, 52, 53, 38, 55, 56, 63, 48, 49, 34, 51, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3530(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 21, 6, 23, 24, 15, 0, 17, 2, 19, 20, 3, 4, 5, 22, 7, 8, 63, 48, 33, 50, 35, 36, 51, 52, 53, 38, 55, 56, 47, 32, 49, 34, 51, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3531(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 21, 6, 23, 24, 31, 0, 17, 2, 19, 20, 3, 4, 5, 22, 7, 8, 47, 48, 33, 50, 35, 36, 51, 52, 53, 38, 55, 56, 63, 32, 49, 34, 51, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3532(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 21, 6, 23, 24, 15, 16, 1, 2, 19, 20, 3, 4, 5, 22, 7, 8, 63, 32, 49, 50, 35, 36, 51, 52, 53, 38, 55, 56, 47, 48, 33, 34, 51, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3533(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 21, 6, 23, 24, 31, 16, 1, 2, 19, 20, 3, 4, 5, 22, 7, 8, 47, 32, 49, 50, 35, 36, 51, 52, 53, 38, 55, 56, 63, 48, 33, 34, 51, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3534(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 21, 6, 23, 24, 15, 0, 1, 2, 19, 20, 3, 4, 5, 22, 7, 8, 63, 48, 49, 50, 35, 36, 51, 52, 53, 38, 55, 56, 47, 32, 33, 34, 51, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3535(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 21, 6, 23, 24, 31, 0, 1, 2, 19, 20, 3, 4, 5, 22, 7, 8, 47, 48, 49, 50, 35, 36, 51, 52, 53, 38, 55, 56, 63, 32, 33, 34, 51, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3536(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 21, 6, 23, 24, 15, 16, 17, 18, 3, 20, 3, 4, 5, 22, 7, 8, 63, 32, 33, 34, 51, 36, 51, 52, 53, 38, 55, 56, 47, 48, 49, 50, 35, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3537(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 21, 6, 23, 24, 31, 16, 17, 18, 3, 20, 3, 4, 5, 22, 7, 8, 47, 32, 33, 34, 51, 36, 51, 52, 53, 38, 55, 56, 63, 48, 49, 50, 35, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3538(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 21, 6, 23, 24, 15, 0, 17, 18, 3, 20, 3, 4, 5, 22, 7, 8, 63, 48, 33, 34, 51, 36, 51, 52, 53, 38, 55, 56, 47, 32, 49, 50, 35, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3539(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 21, 6, 23, 24, 31, 0, 17, 18, 3, 20, 3, 4, 5, 22, 7, 8, 47, 48, 33, 34, 51, 36, 51, 52, 53, 38, 55, 56, 63, 32, 49, 50, 35, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3540(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 21, 6, 23, 24, 15, 16, 1, 18, 3, 20, 3, 4, 5, 22, 7, 8, 63, 32, 49, 34, 51, 36, 51, 52, 53, 38, 55, 56, 47, 48, 33, 50, 35, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3541(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 21, 6, 23, 24, 31, 16, 1, 18, 3, 20, 3, 4, 5, 22, 7, 8, 47, 32, 49, 34, 51, 36, 51, 52, 53, 38, 55, 56, 63, 48, 33, 50, 35, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3542(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 21, 6, 23, 24, 15, 0, 1, 18, 3, 20, 3, 4, 5, 22, 7, 8, 63, 48, 49, 34, 51, 36, 51, 52, 53, 38, 55, 56, 47, 32, 33, 50, 35, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3543(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 21, 6, 23, 24, 31, 0, 1, 18, 3, 20, 3, 4, 5, 22, 7, 8, 47, 48, 49, 34, 51, 36, 51, 52, 53, 38, 55, 56, 63, 32, 33, 50, 35, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3544(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 21, 6, 23, 24, 15, 16, 17, 2, 3, 20, 3, 4, 5, 22, 7, 8, 63, 32, 33, 50, 51, 36, 51, 52, 53, 38, 55, 56, 47, 48, 49, 34, 35, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3545(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 21, 6, 23, 24, 31, 16, 17, 2, 3, 20, 3, 4, 5, 22, 7, 8, 47, 32, 33, 50, 51, 36, 51, 52, 53, 38, 55, 56, 63, 48, 49, 34, 35, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3546(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 21, 6, 23, 24, 15, 0, 17, 2, 3, 20, 3, 4, 5, 22, 7, 8, 63, 48, 33, 50, 51, 36, 51, 52, 53, 38, 55, 56, 47, 32, 49, 34, 35, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3547(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 21, 6, 23, 24, 31, 0, 17, 2, 3, 20, 3, 4, 5, 22, 7, 8, 47, 48, 33, 50, 51, 36, 51, 52, 53, 38, 55, 56, 63, 32, 49, 34, 35, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3548(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 21, 6, 23, 24, 15, 16, 1, 2, 3, 20, 3, 4, 5, 22, 7, 8, 63, 32, 49, 50, 51, 36, 51, 52, 53, 38, 55, 56, 47, 48, 33, 34, 35, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3549(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 21, 6, 23, 24, 31, 16, 1, 2, 3, 20, 3, 4, 5, 22, 7, 8, 47, 32, 49, 50, 51, 36, 51, 52, 53, 38, 55, 56, 63, 48, 33, 34, 35, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3550(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 21, 6, 23, 24, 15, 0, 1, 2, 3, 20, 3, 4, 5, 22, 7, 8, 63, 48, 49, 50, 51, 36, 51, 52, 53, 38, 55, 56, 47, 32, 33, 34, 35, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3551(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 21, 6, 23, 24, 31, 0, 1, 2, 3, 20, 3, 4, 5, 22, 7, 8, 47, 48, 49, 50, 51, 36, 51, 52, 53, 38, 55, 56, 63, 32, 33, 34, 35, 52, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3552(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 21, 6, 23, 24, 15, 16, 17, 18, 19, 4, 3, 4, 5, 22, 7, 8, 63, 32, 33, 34, 35, 52, 51, 52, 53, 38, 55, 56, 47, 48, 49, 50, 51, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3553(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 21, 6, 23, 24, 31, 16, 17, 18, 19, 4, 3, 4, 5, 22, 7, 8, 47, 32, 33, 34, 35, 52, 51, 52, 53, 38, 55, 56, 63, 48, 49, 50, 51, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3554(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 21, 6, 23, 24, 15, 0, 17, 18, 19, 4, 3, 4, 5, 22, 7, 8, 63, 48, 33, 34, 35, 52, 51, 52, 53, 38, 55, 56, 47, 32, 49, 50, 51, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3555(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 21, 6, 23, 24, 31, 0, 17, 18, 19, 4, 3, 4, 5, 22, 7, 8, 47, 48, 33, 34, 35, 52, 51, 52, 53, 38, 55, 56, 63, 32, 49, 50, 51, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3556(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 21, 6, 23, 24, 15, 16, 1, 18, 19, 4, 3, 4, 5, 22, 7, 8, 63, 32, 49, 34, 35, 52, 51, 52, 53, 38, 55, 56, 47, 48, 33, 50, 51, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3557(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 21, 6, 23, 24, 31, 16, 1, 18, 19, 4, 3, 4, 5, 22, 7, 8, 47, 32, 49, 34, 35, 52, 51, 52, 53, 38, 55, 56, 63, 48, 33, 50, 51, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3558(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 21, 6, 23, 24, 15, 0, 1, 18, 19, 4, 3, 4, 5, 22, 7, 8, 63, 48, 49, 34, 35, 52, 51, 52, 53, 38, 55, 56, 47, 32, 33, 50, 51, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3559(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 21, 6, 23, 24, 31, 0, 1, 18, 19, 4, 3, 4, 5, 22, 7, 8, 47, 48, 49, 34, 35, 52, 51, 52, 53, 38, 55, 56, 63, 32, 33, 50, 51, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3560(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 21, 6, 23, 24, 15, 16, 17, 2, 19, 4, 3, 4, 5, 22, 7, 8, 63, 32, 33, 50, 35, 52, 51, 52, 53, 38, 55, 56, 47, 48, 49, 34, 51, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3561(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 21, 6, 23, 24, 31, 16, 17, 2, 19, 4, 3, 4, 5, 22, 7, 8, 47, 32, 33, 50, 35, 52, 51, 52, 53, 38, 55, 56, 63, 48, 49, 34, 51, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3562(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 21, 6, 23, 24, 15, 0, 17, 2, 19, 4, 3, 4, 5, 22, 7, 8, 63, 48, 33, 50, 35, 52, 51, 52, 53, 38, 55, 56, 47, 32, 49, 34, 51, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3563(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 21, 6, 23, 24, 31, 0, 17, 2, 19, 4, 3, 4, 5, 22, 7, 8, 47, 48, 33, 50, 35, 52, 51, 52, 53, 38, 55, 56, 63, 32, 49, 34, 51, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3564(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 21, 6, 23, 24, 15, 16, 1, 2, 19, 4, 3, 4, 5, 22, 7, 8, 63, 32, 49, 50, 35, 52, 51, 52, 53, 38, 55, 56, 47, 48, 33, 34, 51, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3565(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 21, 6, 23, 24, 31, 16, 1, 2, 19, 4, 3, 4, 5, 22, 7, 8, 47, 32, 49, 50, 35, 52, 51, 52, 53, 38, 55, 56, 63, 48, 33, 34, 51, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3566(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 21, 6, 23, 24, 15, 0, 1, 2, 19, 4, 3, 4, 5, 22, 7, 8, 63, 48, 49, 50, 35, 52, 51, 52, 53, 38, 55, 56, 47, 32, 33, 34, 51, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3567(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 21, 6, 23, 24, 31, 0, 1, 2, 19, 4, 3, 4, 5, 22, 7, 8, 47, 48, 49, 50, 35, 52, 51, 52, 53, 38, 55, 56, 63, 32, 33, 34, 51, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3568(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 21, 6, 23, 24, 15, 16, 17, 18, 3, 4, 3, 4, 5, 22, 7, 8, 63, 32, 33, 34, 51, 52, 51, 52, 53, 38, 55, 56, 47, 48, 49, 50, 35, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3569(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 21, 6, 23, 24, 31, 16, 17, 18, 3, 4, 3, 4, 5, 22, 7, 8, 47, 32, 33, 34, 51, 52, 51, 52, 53, 38, 55, 56, 63, 48, 49, 50, 35, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3570(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 21, 6, 23, 24, 15, 0, 17, 18, 3, 4, 3, 4, 5, 22, 7, 8, 63, 48, 33, 34, 51, 52, 51, 52, 53, 38, 55, 56, 47, 32, 49, 50, 35, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3571(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 21, 6, 23, 24, 31, 0, 17, 18, 3, 4, 3, 4, 5, 22, 7, 8, 47, 48, 33, 34, 51, 52, 51, 52, 53, 38, 55, 56, 63, 32, 49, 50, 35, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3572(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 21, 6, 23, 24, 15, 16, 1, 18, 3, 4, 3, 4, 5, 22, 7, 8, 63, 32, 49, 34, 51, 52, 51, 52, 53, 38, 55, 56, 47, 48, 33, 50, 35, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3573(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 21, 6, 23, 24, 31, 16, 1, 18, 3, 4, 3, 4, 5, 22, 7, 8, 47, 32, 49, 34, 51, 52, 51, 52, 53, 38, 55, 56, 63, 48, 33, 50, 35, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3574(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 21, 6, 23, 24, 15, 0, 1, 18, 3, 4, 3, 4, 5, 22, 7, 8, 63, 48, 49, 34, 51, 52, 51, 52, 53, 38, 55, 56, 47, 32, 33, 50, 35, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3575(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 21, 6, 23, 24, 31, 0, 1, 18, 3, 4, 3, 4, 5, 22, 7, 8, 47, 48, 49, 34, 51, 52, 51, 52, 53, 38, 55, 56, 63, 32, 33, 50, 35, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3576(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 21, 6, 23, 24, 15, 16, 17, 2, 3, 4, 3, 4, 5, 22, 7, 8, 63, 32, 33, 50, 51, 52, 51, 52, 53, 38, 55, 56, 47, 48, 49, 34, 35, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3577(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 21, 6, 23, 24, 31, 16, 17, 2, 3, 4, 3, 4, 5, 22, 7, 8, 47, 32, 33, 50, 51, 52, 51, 52, 53, 38, 55, 56, 63, 48, 49, 34, 35, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3578(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 21, 6, 23, 24, 15, 0, 17, 2, 3, 4, 3, 4, 5, 22, 7, 8, 63, 48, 33, 50, 51, 52, 51, 52, 53, 38, 55, 56, 47, 32, 49, 34, 35, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3579(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 21, 6, 23, 24, 31, 0, 17, 2, 3, 4, 3, 4, 5, 22, 7, 8, 47, 48, 33, 50, 51, 52, 51, 52, 53, 38, 55, 56, 63, 32, 49, 34, 35, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3580(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 21, 6, 23, 24, 15, 16, 1, 2, 3, 4, 3, 4, 5, 22, 7, 8, 63, 32, 49, 50, 51, 52, 51, 52, 53, 38, 55, 56, 47, 48, 33, 34, 35, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3581(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 21, 6, 23, 24, 31, 16, 1, 2, 3, 4, 3, 4, 5, 22, 7, 8, 47, 32, 49, 50, 51, 52, 51, 52, 53, 38, 55, 56, 63, 48, 33, 34, 35, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3582(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 21, 6, 23, 24, 15, 0, 1, 2, 3, 4, 3, 4, 5, 22, 7, 8, 63, 48, 49, 50, 51, 52, 51, 52, 53, 38, 55, 56, 47, 32, 33, 34, 35, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3583(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 21, 6, 23, 24, 31, 0, 1, 2, 3, 4, 3, 4, 5, 22, 7, 8, 47, 48, 49, 50, 51, 52, 51, 52, 53, 38, 55, 56, 63, 32, 33, 34, 35, 36, 35, 36, 37, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}


