#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

#ifdef DESGPU_COMPILE_ALL_SALTS

void des_25_encrypt_salt3456(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 21, 6, 23, 24, 15, 16, 17, 18, 19, 20, 19, 4, 5, 22, 7, 8, 63, 32, 33, 34, 35, 36, 35, 52, 53, 38, 55, 56, 47, 48, 49, 50, 51, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3457(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 21, 6, 23, 24, 31, 16, 17, 18, 19, 20, 19, 4, 5, 22, 7, 8, 47, 32, 33, 34, 35, 36, 35, 52, 53, 38, 55, 56, 63, 48, 49, 50, 51, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3458(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 21, 6, 23, 24, 15, 0, 17, 18, 19, 20, 19, 4, 5, 22, 7, 8, 63, 48, 33, 34, 35, 36, 35, 52, 53, 38, 55, 56, 47, 32, 49, 50, 51, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3459(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 21, 6, 23, 24, 31, 0, 17, 18, 19, 20, 19, 4, 5, 22, 7, 8, 47, 48, 33, 34, 35, 36, 35, 52, 53, 38, 55, 56, 63, 32, 49, 50, 51, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3460(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 21, 6, 23, 24, 15, 16, 1, 18, 19, 20, 19, 4, 5, 22, 7, 8, 63, 32, 49, 34, 35, 36, 35, 52, 53, 38, 55, 56, 47, 48, 33, 50, 51, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3461(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 21, 6, 23, 24, 31, 16, 1, 18, 19, 20, 19, 4, 5, 22, 7, 8, 47, 32, 49, 34, 35, 36, 35, 52, 53, 38, 55, 56, 63, 48, 33, 50, 51, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3462(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 21, 6, 23, 24, 15, 0, 1, 18, 19, 20, 19, 4, 5, 22, 7, 8, 63, 48, 49, 34, 35, 36, 35, 52, 53, 38, 55, 56, 47, 32, 33, 50, 51, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3463(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 21, 6, 23, 24, 31, 0, 1, 18, 19, 20, 19, 4, 5, 22, 7, 8, 47, 48, 49, 34, 35, 36, 35, 52, 53, 38, 55, 56, 63, 32, 33, 50, 51, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3464(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 21, 6, 23, 24, 15, 16, 17, 2, 19, 20, 19, 4, 5, 22, 7, 8, 63, 32, 33, 50, 35, 36, 35, 52, 53, 38, 55, 56, 47, 48, 49, 34, 51, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3465(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 21, 6, 23, 24, 31, 16, 17, 2, 19, 20, 19, 4, 5, 22, 7, 8, 47, 32, 33, 50, 35, 36, 35, 52, 53, 38, 55, 56, 63, 48, 49, 34, 51, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3466(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 21, 6, 23, 24, 15, 0, 17, 2, 19, 20, 19, 4, 5, 22, 7, 8, 63, 48, 33, 50, 35, 36, 35, 52, 53, 38, 55, 56, 47, 32, 49, 34, 51, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3467(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 21, 6, 23, 24, 31, 0, 17, 2, 19, 20, 19, 4, 5, 22, 7, 8, 47, 48, 33, 50, 35, 36, 35, 52, 53, 38, 55, 56, 63, 32, 49, 34, 51, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3468(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 21, 6, 23, 24, 15, 16, 1, 2, 19, 20, 19, 4, 5, 22, 7, 8, 63, 32, 49, 50, 35, 36, 35, 52, 53, 38, 55, 56, 47, 48, 33, 34, 51, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3469(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 21, 6, 23, 24, 31, 16, 1, 2, 19, 20, 19, 4, 5, 22, 7, 8, 47, 32, 49, 50, 35, 36, 35, 52, 53, 38, 55, 56, 63, 48, 33, 34, 51, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3470(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 21, 6, 23, 24, 15, 0, 1, 2, 19, 20, 19, 4, 5, 22, 7, 8, 63, 48, 49, 50, 35, 36, 35, 52, 53, 38, 55, 56, 47, 32, 33, 34, 51, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3471(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 21, 6, 23, 24, 31, 0, 1, 2, 19, 20, 19, 4, 5, 22, 7, 8, 47, 48, 49, 50, 35, 36, 35, 52, 53, 38, 55, 56, 63, 32, 33, 34, 51, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3472(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 21, 6, 23, 24, 15, 16, 17, 18, 3, 20, 19, 4, 5, 22, 7, 8, 63, 32, 33, 34, 51, 36, 35, 52, 53, 38, 55, 56, 47, 48, 49, 50, 35, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3473(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 21, 6, 23, 24, 31, 16, 17, 18, 3, 20, 19, 4, 5, 22, 7, 8, 47, 32, 33, 34, 51, 36, 35, 52, 53, 38, 55, 56, 63, 48, 49, 50, 35, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3474(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 21, 6, 23, 24, 15, 0, 17, 18, 3, 20, 19, 4, 5, 22, 7, 8, 63, 48, 33, 34, 51, 36, 35, 52, 53, 38, 55, 56, 47, 32, 49, 50, 35, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3475(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 21, 6, 23, 24, 31, 0, 17, 18, 3, 20, 19, 4, 5, 22, 7, 8, 47, 48, 33, 34, 51, 36, 35, 52, 53, 38, 55, 56, 63, 32, 49, 50, 35, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3476(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 21, 6, 23, 24, 15, 16, 1, 18, 3, 20, 19, 4, 5, 22, 7, 8, 63, 32, 49, 34, 51, 36, 35, 52, 53, 38, 55, 56, 47, 48, 33, 50, 35, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3477(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 21, 6, 23, 24, 31, 16, 1, 18, 3, 20, 19, 4, 5, 22, 7, 8, 47, 32, 49, 34, 51, 36, 35, 52, 53, 38, 55, 56, 63, 48, 33, 50, 35, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3478(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 21, 6, 23, 24, 15, 0, 1, 18, 3, 20, 19, 4, 5, 22, 7, 8, 63, 48, 49, 34, 51, 36, 35, 52, 53, 38, 55, 56, 47, 32, 33, 50, 35, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3479(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 21, 6, 23, 24, 31, 0, 1, 18, 3, 20, 19, 4, 5, 22, 7, 8, 47, 48, 49, 34, 51, 36, 35, 52, 53, 38, 55, 56, 63, 32, 33, 50, 35, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3480(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 21, 6, 23, 24, 15, 16, 17, 2, 3, 20, 19, 4, 5, 22, 7, 8, 63, 32, 33, 50, 51, 36, 35, 52, 53, 38, 55, 56, 47, 48, 49, 34, 35, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3481(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 21, 6, 23, 24, 31, 16, 17, 2, 3, 20, 19, 4, 5, 22, 7, 8, 47, 32, 33, 50, 51, 36, 35, 52, 53, 38, 55, 56, 63, 48, 49, 34, 35, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3482(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 21, 6, 23, 24, 15, 0, 17, 2, 3, 20, 19, 4, 5, 22, 7, 8, 63, 48, 33, 50, 51, 36, 35, 52, 53, 38, 55, 56, 47, 32, 49, 34, 35, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3483(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 21, 6, 23, 24, 31, 0, 17, 2, 3, 20, 19, 4, 5, 22, 7, 8, 47, 48, 33, 50, 51, 36, 35, 52, 53, 38, 55, 56, 63, 32, 49, 34, 35, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3484(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 21, 6, 23, 24, 15, 16, 1, 2, 3, 20, 19, 4, 5, 22, 7, 8, 63, 32, 49, 50, 51, 36, 35, 52, 53, 38, 55, 56, 47, 48, 33, 34, 35, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3485(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 21, 6, 23, 24, 31, 16, 1, 2, 3, 20, 19, 4, 5, 22, 7, 8, 47, 32, 49, 50, 51, 36, 35, 52, 53, 38, 55, 56, 63, 48, 33, 34, 35, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3486(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 21, 6, 23, 24, 15, 0, 1, 2, 3, 20, 19, 4, 5, 22, 7, 8, 63, 48, 49, 50, 51, 36, 35, 52, 53, 38, 55, 56, 47, 32, 33, 34, 35, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3487(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 21, 6, 23, 24, 31, 0, 1, 2, 3, 20, 19, 4, 5, 22, 7, 8, 47, 48, 49, 50, 51, 36, 35, 52, 53, 38, 55, 56, 63, 32, 33, 34, 35, 52, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3488(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 21, 6, 23, 24, 15, 16, 17, 18, 19, 4, 19, 4, 5, 22, 7, 8, 63, 32, 33, 34, 35, 52, 35, 52, 53, 38, 55, 56, 47, 48, 49, 50, 51, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3489(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 21, 6, 23, 24, 31, 16, 17, 18, 19, 4, 19, 4, 5, 22, 7, 8, 47, 32, 33, 34, 35, 52, 35, 52, 53, 38, 55, 56, 63, 48, 49, 50, 51, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3490(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 21, 6, 23, 24, 15, 0, 17, 18, 19, 4, 19, 4, 5, 22, 7, 8, 63, 48, 33, 34, 35, 52, 35, 52, 53, 38, 55, 56, 47, 32, 49, 50, 51, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3491(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 21, 6, 23, 24, 31, 0, 17, 18, 19, 4, 19, 4, 5, 22, 7, 8, 47, 48, 33, 34, 35, 52, 35, 52, 53, 38, 55, 56, 63, 32, 49, 50, 51, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3492(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 21, 6, 23, 24, 15, 16, 1, 18, 19, 4, 19, 4, 5, 22, 7, 8, 63, 32, 49, 34, 35, 52, 35, 52, 53, 38, 55, 56, 47, 48, 33, 50, 51, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3493(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 21, 6, 23, 24, 31, 16, 1, 18, 19, 4, 19, 4, 5, 22, 7, 8, 47, 32, 49, 34, 35, 52, 35, 52, 53, 38, 55, 56, 63, 48, 33, 50, 51, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3494(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 21, 6, 23, 24, 15, 0, 1, 18, 19, 4, 19, 4, 5, 22, 7, 8, 63, 48, 49, 34, 35, 52, 35, 52, 53, 38, 55, 56, 47, 32, 33, 50, 51, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3495(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 21, 6, 23, 24, 31, 0, 1, 18, 19, 4, 19, 4, 5, 22, 7, 8, 47, 48, 49, 34, 35, 52, 35, 52, 53, 38, 55, 56, 63, 32, 33, 50, 51, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3496(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 21, 6, 23, 24, 15, 16, 17, 2, 19, 4, 19, 4, 5, 22, 7, 8, 63, 32, 33, 50, 35, 52, 35, 52, 53, 38, 55, 56, 47, 48, 49, 34, 51, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3497(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 21, 6, 23, 24, 31, 16, 17, 2, 19, 4, 19, 4, 5, 22, 7, 8, 47, 32, 33, 50, 35, 52, 35, 52, 53, 38, 55, 56, 63, 48, 49, 34, 51, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3498(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 21, 6, 23, 24, 15, 0, 17, 2, 19, 4, 19, 4, 5, 22, 7, 8, 63, 48, 33, 50, 35, 52, 35, 52, 53, 38, 55, 56, 47, 32, 49, 34, 51, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3499(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 21, 6, 23, 24, 31, 0, 17, 2, 19, 4, 19, 4, 5, 22, 7, 8, 47, 48, 33, 50, 35, 52, 35, 52, 53, 38, 55, 56, 63, 32, 49, 34, 51, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3500(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 21, 6, 23, 24, 15, 16, 1, 2, 19, 4, 19, 4, 5, 22, 7, 8, 63, 32, 49, 50, 35, 52, 35, 52, 53, 38, 55, 56, 47, 48, 33, 34, 51, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3501(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 21, 6, 23, 24, 31, 16, 1, 2, 19, 4, 19, 4, 5, 22, 7, 8, 47, 32, 49, 50, 35, 52, 35, 52, 53, 38, 55, 56, 63, 48, 33, 34, 51, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3502(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 21, 6, 23, 24, 15, 0, 1, 2, 19, 4, 19, 4, 5, 22, 7, 8, 63, 48, 49, 50, 35, 52, 35, 52, 53, 38, 55, 56, 47, 32, 33, 34, 51, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3503(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 21, 6, 23, 24, 31, 0, 1, 2, 19, 4, 19, 4, 5, 22, 7, 8, 47, 48, 49, 50, 35, 52, 35, 52, 53, 38, 55, 56, 63, 32, 33, 34, 51, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3504(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 21, 6, 23, 24, 15, 16, 17, 18, 3, 4, 19, 4, 5, 22, 7, 8, 63, 32, 33, 34, 51, 52, 35, 52, 53, 38, 55, 56, 47, 48, 49, 50, 35, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3505(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 21, 6, 23, 24, 31, 16, 17, 18, 3, 4, 19, 4, 5, 22, 7, 8, 47, 32, 33, 34, 51, 52, 35, 52, 53, 38, 55, 56, 63, 48, 49, 50, 35, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3506(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 21, 6, 23, 24, 15, 0, 17, 18, 3, 4, 19, 4, 5, 22, 7, 8, 63, 48, 33, 34, 51, 52, 35, 52, 53, 38, 55, 56, 47, 32, 49, 50, 35, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3507(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 21, 6, 23, 24, 31, 0, 17, 18, 3, 4, 19, 4, 5, 22, 7, 8, 47, 48, 33, 34, 51, 52, 35, 52, 53, 38, 55, 56, 63, 32, 49, 50, 35, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3508(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 21, 6, 23, 24, 15, 16, 1, 18, 3, 4, 19, 4, 5, 22, 7, 8, 63, 32, 49, 34, 51, 52, 35, 52, 53, 38, 55, 56, 47, 48, 33, 50, 35, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3509(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 21, 6, 23, 24, 31, 16, 1, 18, 3, 4, 19, 4, 5, 22, 7, 8, 47, 32, 49, 34, 51, 52, 35, 52, 53, 38, 55, 56, 63, 48, 33, 50, 35, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3510(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 21, 6, 23, 24, 15, 0, 1, 18, 3, 4, 19, 4, 5, 22, 7, 8, 63, 48, 49, 34, 51, 52, 35, 52, 53, 38, 55, 56, 47, 32, 33, 50, 35, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3511(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 21, 6, 23, 24, 31, 0, 1, 18, 3, 4, 19, 4, 5, 22, 7, 8, 47, 48, 49, 34, 51, 52, 35, 52, 53, 38, 55, 56, 63, 32, 33, 50, 35, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3512(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 21, 6, 23, 24, 15, 16, 17, 2, 3, 4, 19, 4, 5, 22, 7, 8, 63, 32, 33, 50, 51, 52, 35, 52, 53, 38, 55, 56, 47, 48, 49, 34, 35, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3513(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 21, 6, 23, 24, 31, 16, 17, 2, 3, 4, 19, 4, 5, 22, 7, 8, 47, 32, 33, 50, 51, 52, 35, 52, 53, 38, 55, 56, 63, 48, 49, 34, 35, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3514(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 21, 6, 23, 24, 15, 0, 17, 2, 3, 4, 19, 4, 5, 22, 7, 8, 63, 48, 33, 50, 51, 52, 35, 52, 53, 38, 55, 56, 47, 32, 49, 34, 35, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3515(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 21, 6, 23, 24, 31, 0, 17, 2, 3, 4, 19, 4, 5, 22, 7, 8, 47, 48, 33, 50, 51, 52, 35, 52, 53, 38, 55, 56, 63, 32, 49, 34, 35, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3516(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 21, 6, 23, 24, 15, 16, 1, 2, 3, 4, 19, 4, 5, 22, 7, 8, 63, 32, 49, 50, 51, 52, 35, 52, 53, 38, 55, 56, 47, 48, 33, 34, 35, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3517(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 21, 6, 23, 24, 31, 16, 1, 2, 3, 4, 19, 4, 5, 22, 7, 8, 47, 32, 49, 50, 51, 52, 35, 52, 53, 38, 55, 56, 63, 48, 33, 34, 35, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3518(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 21, 6, 23, 24, 15, 0, 1, 2, 3, 4, 19, 4, 5, 22, 7, 8, 63, 48, 49, 50, 51, 52, 35, 52, 53, 38, 55, 56, 47, 32, 33, 34, 35, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3519(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 21, 6, 23, 24, 31, 0, 1, 2, 3, 4, 19, 4, 5, 22, 7, 8, 47, 48, 49, 50, 51, 52, 35, 52, 53, 38, 55, 56, 63, 32, 33, 34, 35, 36, 51, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3520(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 21, 6, 23, 24, 15, 16, 17, 18, 19, 20, 3, 4, 5, 22, 7, 8, 63, 32, 33, 34, 35, 36, 51, 52, 53, 38, 55, 56, 47, 48, 49, 50, 51, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3521(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 21, 6, 23, 24, 31, 16, 17, 18, 19, 20, 3, 4, 5, 22, 7, 8, 47, 32, 33, 34, 35, 36, 51, 52, 53, 38, 55, 56, 63, 48, 49, 50, 51, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3522(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 21, 6, 23, 24, 15, 0, 17, 18, 19, 20, 3, 4, 5, 22, 7, 8, 63, 48, 33, 34, 35, 36, 51, 52, 53, 38, 55, 56, 47, 32, 49, 50, 51, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3523(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 21, 6, 23, 24, 31, 0, 17, 18, 19, 20, 3, 4, 5, 22, 7, 8, 47, 48, 33, 34, 35, 36, 51, 52, 53, 38, 55, 56, 63, 32, 49, 50, 51, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3524(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 21, 6, 23, 24, 15, 16, 1, 18, 19, 20, 3, 4, 5, 22, 7, 8, 63, 32, 49, 34, 35, 36, 51, 52, 53, 38, 55, 56, 47, 48, 33, 50, 51, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3525(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 21, 6, 23, 24, 31, 16, 1, 18, 19, 20, 3, 4, 5, 22, 7, 8, 47, 32, 49, 34, 35, 36, 51, 52, 53, 38, 55, 56, 63, 48, 33, 50, 51, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3526(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 21, 6, 23, 24, 15, 0, 1, 18, 19, 20, 3, 4, 5, 22, 7, 8, 63, 48, 49, 34, 35, 36, 51, 52, 53, 38, 55, 56, 47, 32, 33, 50, 51, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3527(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 21, 6, 23, 24, 31, 0, 1, 18, 19, 20, 3, 4, 5, 22, 7, 8, 47, 48, 49, 34, 35, 36, 51, 52, 53, 38, 55, 56, 63, 32, 33, 50, 51, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3528(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 21, 6, 23, 24, 15, 16, 17, 2, 19, 20, 3, 4, 5, 22, 7, 8, 63, 32, 33, 50, 35, 36, 51, 52, 53, 38, 55, 56, 47, 48, 49, 34, 51, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3529(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 21, 6, 23, 24, 31, 16, 17, 2, 19, 20, 3, 4, 5, 22, 7, 8, 47, 32, 33, 50, 35, 36, 51, 52, 53, 38, 55, 56, 63, 48, 49, 34, 51, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3530(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 21, 6, 23, 24, 15, 0, 17, 2, 19, 20, 3, 4, 5, 22, 7, 8, 63, 48, 33, 50, 35, 36, 51, 52, 53, 38, 55, 56, 47, 32, 49, 34, 51, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3531(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 21, 6, 23, 24, 31, 0, 17, 2, 19, 20, 3, 4, 5, 22, 7, 8, 47, 48, 33, 50, 35, 36, 51, 52, 53, 38, 55, 56, 63, 32, 49, 34, 51, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3532(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 21, 6, 23, 24, 15, 16, 1, 2, 19, 20, 3, 4, 5, 22, 7, 8, 63, 32, 49, 50, 35, 36, 51, 52, 53, 38, 55, 56, 47, 48, 33, 34, 51, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3533(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 21, 6, 23, 24, 31, 16, 1, 2, 19, 20, 3, 4, 5, 22, 7, 8, 47, 32, 49, 50, 35, 36, 51, 52, 53, 38, 55, 56, 63, 48, 33, 34, 51, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3534(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 21, 6, 23, 24, 15, 0, 1, 2, 19, 20, 3, 4, 5, 22, 7, 8, 63, 48, 49, 50, 35, 36, 51, 52, 53, 38, 55, 56, 47, 32, 33, 34, 51, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3535(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 21, 6, 23, 24, 31, 0, 1, 2, 19, 20, 3, 4, 5, 22, 7, 8, 47, 48, 49, 50, 35, 36, 51, 52, 53, 38, 55, 56, 63, 32, 33, 34, 51, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3536(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 21, 6, 23, 24, 15, 16, 17, 18, 3, 20, 3, 4, 5, 22, 7, 8, 63, 32, 33, 34, 51, 36, 51, 52, 53, 38, 55, 56, 47, 48, 49, 50, 35, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3537(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 21, 6, 23, 24, 31, 16, 17, 18, 3, 20, 3, 4, 5, 22, 7, 8, 47, 32, 33, 34, 51, 36, 51, 52, 53, 38, 55, 56, 63, 48, 49, 50, 35, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3538(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 21, 6, 23, 24, 15, 0, 17, 18, 3, 20, 3, 4, 5, 22, 7, 8, 63, 48, 33, 34, 51, 36, 51, 52, 53, 38, 55, 56, 47, 32, 49, 50, 35, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3539(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 21, 6, 23, 24, 31, 0, 17, 18, 3, 20, 3, 4, 5, 22, 7, 8, 47, 48, 33, 34, 51, 36, 51, 52, 53, 38, 55, 56, 63, 32, 49, 50, 35, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3540(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 21, 6, 23, 24, 15, 16, 1, 18, 3, 20, 3, 4, 5, 22, 7, 8, 63, 32, 49, 34, 51, 36, 51, 52, 53, 38, 55, 56, 47, 48, 33, 50, 35, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3541(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 21, 6, 23, 24, 31, 16, 1, 18, 3, 20, 3, 4, 5, 22, 7, 8, 47, 32, 49, 34, 51, 36, 51, 52, 53, 38, 55, 56, 63, 48, 33, 50, 35, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3542(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 21, 6, 23, 24, 15, 0, 1, 18, 3, 20, 3, 4, 5, 22, 7, 8, 63, 48, 49, 34, 51, 36, 51, 52, 53, 38, 55, 56, 47, 32, 33, 50, 35, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3543(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 21, 6, 23, 24, 31, 0, 1, 18, 3, 20, 3, 4, 5, 22, 7, 8, 47, 48, 49, 34, 51, 36, 51, 52, 53, 38, 55, 56, 63, 32, 33, 50, 35, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3544(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 21, 6, 23, 24, 15, 16, 17, 2, 3, 20, 3, 4, 5, 22, 7, 8, 63, 32, 33, 50, 51, 36, 51, 52, 53, 38, 55, 56, 47, 48, 49, 34, 35, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3545(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 21, 6, 23, 24, 31, 16, 17, 2, 3, 20, 3, 4, 5, 22, 7, 8, 47, 32, 33, 50, 51, 36, 51, 52, 53, 38, 55, 56, 63, 48, 49, 34, 35, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3546(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 21, 6, 23, 24, 15, 0, 17, 2, 3, 20, 3, 4, 5, 22, 7, 8, 63, 48, 33, 50, 51, 36, 51, 52, 53, 38, 55, 56, 47, 32, 49, 34, 35, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3547(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 21, 6, 23, 24, 31, 0, 17, 2, 3, 20, 3, 4, 5, 22, 7, 8, 47, 48, 33, 50, 51, 36, 51, 52, 53, 38, 55, 56, 63, 32, 49, 34, 35, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3548(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 21, 6, 23, 24, 15, 16, 1, 2, 3, 20, 3, 4, 5, 22, 7, 8, 63, 32, 49, 50, 51, 36, 51, 52, 53, 38, 55, 56, 47, 48, 33, 34, 35, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3549(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 21, 6, 23, 24, 31, 16, 1, 2, 3, 20, 3, 4, 5, 22, 7, 8, 47, 32, 49, 50, 51, 36, 51, 52, 53, 38, 55, 56, 63, 48, 33, 34, 35, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3550(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 21, 6, 23, 24, 15, 0, 1, 2, 3, 20, 3, 4, 5, 22, 7, 8, 63, 48, 49, 50, 51, 36, 51, 52, 53, 38, 55, 56, 47, 32, 33, 34, 35, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3551(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 21, 6, 23, 24, 31, 0, 1, 2, 3, 20, 3, 4, 5, 22, 7, 8, 47, 48, 49, 50, 51, 36, 51, 52, 53, 38, 55, 56, 63, 32, 33, 34, 35, 52, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3552(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 21, 6, 23, 24, 15, 16, 17, 18, 19, 4, 3, 4, 5, 22, 7, 8, 63, 32, 33, 34, 35, 52, 51, 52, 53, 38, 55, 56, 47, 48, 49, 50, 51, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3553(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 21, 6, 23, 24, 31, 16, 17, 18, 19, 4, 3, 4, 5, 22, 7, 8, 47, 32, 33, 34, 35, 52, 51, 52, 53, 38, 55, 56, 63, 48, 49, 50, 51, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3554(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 21, 6, 23, 24, 15, 0, 17, 18, 19, 4, 3, 4, 5, 22, 7, 8, 63, 48, 33, 34, 35, 52, 51, 52, 53, 38, 55, 56, 47, 32, 49, 50, 51, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3555(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 21, 6, 23, 24, 31, 0, 17, 18, 19, 4, 3, 4, 5, 22, 7, 8, 47, 48, 33, 34, 35, 52, 51, 52, 53, 38, 55, 56, 63, 32, 49, 50, 51, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3556(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 21, 6, 23, 24, 15, 16, 1, 18, 19, 4, 3, 4, 5, 22, 7, 8, 63, 32, 49, 34, 35, 52, 51, 52, 53, 38, 55, 56, 47, 48, 33, 50, 51, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3557(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 21, 6, 23, 24, 31, 16, 1, 18, 19, 4, 3, 4, 5, 22, 7, 8, 47, 32, 49, 34, 35, 52, 51, 52, 53, 38, 55, 56, 63, 48, 33, 50, 51, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3558(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 21, 6, 23, 24, 15, 0, 1, 18, 19, 4, 3, 4, 5, 22, 7, 8, 63, 48, 49, 34, 35, 52, 51, 52, 53, 38, 55, 56, 47, 32, 33, 50, 51, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3559(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 21, 6, 23, 24, 31, 0, 1, 18, 19, 4, 3, 4, 5, 22, 7, 8, 47, 48, 49, 34, 35, 52, 51, 52, 53, 38, 55, 56, 63, 32, 33, 50, 51, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3560(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 21, 6, 23, 24, 15, 16, 17, 2, 19, 4, 3, 4, 5, 22, 7, 8, 63, 32, 33, 50, 35, 52, 51, 52, 53, 38, 55, 56, 47, 48, 49, 34, 51, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3561(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 21, 6, 23, 24, 31, 16, 17, 2, 19, 4, 3, 4, 5, 22, 7, 8, 47, 32, 33, 50, 35, 52, 51, 52, 53, 38, 55, 56, 63, 48, 49, 34, 51, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3562(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 21, 6, 23, 24, 15, 0, 17, 2, 19, 4, 3, 4, 5, 22, 7, 8, 63, 48, 33, 50, 35, 52, 51, 52, 53, 38, 55, 56, 47, 32, 49, 34, 51, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3563(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 21, 6, 23, 24, 31, 0, 17, 2, 19, 4, 3, 4, 5, 22, 7, 8, 47, 48, 33, 50, 35, 52, 51, 52, 53, 38, 55, 56, 63, 32, 49, 34, 51, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3564(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 21, 6, 23, 24, 15, 16, 1, 2, 19, 4, 3, 4, 5, 22, 7, 8, 63, 32, 49, 50, 35, 52, 51, 52, 53, 38, 55, 56, 47, 48, 33, 34, 51, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3565(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 21, 6, 23, 24, 31, 16, 1, 2, 19, 4, 3, 4, 5, 22, 7, 8, 47, 32, 49, 50, 35, 52, 51, 52, 53, 38, 55, 56, 63, 48, 33, 34, 51, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3566(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 21, 6, 23, 24, 15, 0, 1, 2, 19, 4, 3, 4, 5, 22, 7, 8, 63, 48, 49, 50, 35, 52, 51, 52, 53, 38, 55, 56, 47, 32, 33, 34, 51, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3567(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 21, 6, 23, 24, 31, 0, 1, 2, 19, 4, 3, 4, 5, 22, 7, 8, 47, 48, 49, 50, 35, 52, 51, 52, 53, 38, 55, 56, 63, 32, 33, 34, 51, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3568(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 21, 6, 23, 24, 15, 16, 17, 18, 3, 4, 3, 4, 5, 22, 7, 8, 63, 32, 33, 34, 51, 52, 51, 52, 53, 38, 55, 56, 47, 48, 49, 50, 35, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3569(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 21, 6, 23, 24, 31, 16, 17, 18, 3, 4, 3, 4, 5, 22, 7, 8, 47, 32, 33, 34, 51, 52, 51, 52, 53, 38, 55, 56, 63, 48, 49, 50, 35, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3570(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 21, 6, 23, 24, 15, 0, 17, 18, 3, 4, 3, 4, 5, 22, 7, 8, 63, 48, 33, 34, 51, 52, 51, 52, 53, 38, 55, 56, 47, 32, 49, 50, 35, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3571(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 21, 6, 23, 24, 31, 0, 17, 18, 3, 4, 3, 4, 5, 22, 7, 8, 47, 48, 33, 34, 51, 52, 51, 52, 53, 38, 55, 56, 63, 32, 49, 50, 35, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3572(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 21, 6, 23, 24, 15, 16, 1, 18, 3, 4, 3, 4, 5, 22, 7, 8, 63, 32, 49, 34, 51, 52, 51, 52, 53, 38, 55, 56, 47, 48, 33, 50, 35, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3573(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 21, 6, 23, 24, 31, 16, 1, 18, 3, 4, 3, 4, 5, 22, 7, 8, 47, 32, 49, 34, 51, 52, 51, 52, 53, 38, 55, 56, 63, 48, 33, 50, 35, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3574(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 21, 6, 23, 24, 15, 0, 1, 18, 3, 4, 3, 4, 5, 22, 7, 8, 63, 48, 49, 34, 51, 52, 51, 52, 53, 38, 55, 56, 47, 32, 33, 50, 35, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3575(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 21, 6, 23, 24, 31, 0, 1, 18, 3, 4, 3, 4, 5, 22, 7, 8, 47, 48, 49, 34, 51, 52, 51, 52, 53, 38, 55, 56, 63, 32, 33, 50, 35, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3576(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 21, 6, 23, 24, 15, 16, 17, 2, 3, 4, 3, 4, 5, 22, 7, 8, 63, 32, 33, 50, 51, 52, 51, 52, 53, 38, 55, 56, 47, 48, 49, 34, 35, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3577(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 21, 6, 23, 24, 31, 16, 17, 2, 3, 4, 3, 4, 5, 22, 7, 8, 47, 32, 33, 50, 51, 52, 51, 52, 53, 38, 55, 56, 63, 48, 49, 34, 35, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3578(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 21, 6, 23, 24, 15, 0, 17, 2, 3, 4, 3, 4, 5, 22, 7, 8, 63, 48, 33, 50, 51, 52, 51, 52, 53, 38, 55, 56, 47, 32, 49, 34, 35, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3579(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 21, 6, 23, 24, 31, 0, 17, 2, 3, 4, 3, 4, 5, 22, 7, 8, 47, 48, 33, 50, 51, 52, 51, 52, 53, 38, 55, 56, 63, 32, 49, 34, 35, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3580(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 21, 6, 23, 24, 15, 16, 1, 2, 3, 4, 3, 4, 5, 22, 7, 8, 63, 32, 49, 50, 51, 52, 51, 52, 53, 38, 55, 56, 47, 48, 33, 34, 35, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3581(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 21, 6, 23, 24, 31, 16, 1, 2, 3, 4, 3, 4, 5, 22, 7, 8, 47, 32, 49, 50, 51, 52, 51, 52, 53, 38, 55, 56, 63, 48, 33, 34, 35, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3582(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 21, 6, 23, 24, 15, 0, 1, 2, 3, 4, 3, 4, 5, 22, 7, 8, 63, 48, 49, 50, 51, 52, 51, 52, 53, 38, 55, 56, 47, 32, 33, 34, 35, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3583(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 21, 6, 23, 24, 31, 0, 1, 2, 3, 4, 3, 4, 5, 22, 7, 8, 47, 48, 49, 50, 51, 52, 51, 52, 53, 38, 55, 56, 63, 32, 33, 34, 35, 36, 35, 36, 37, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

#endif // DESGPU_COMPILE_ALL_SALTS


