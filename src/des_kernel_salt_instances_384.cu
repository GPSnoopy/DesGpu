#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

#ifdef DESGPU_COMPILE_ALL_SALTS

void des_25_encrypt_salt384(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 21, 6, 7, 8, 15, 16, 17, 18, 19, 20, 19, 4, 5, 22, 23, 24, 63, 32, 33, 34, 35, 36, 35, 52, 53, 38, 39, 40, 47, 48, 49, 50, 51, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt385(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 21, 6, 7, 8, 31, 16, 17, 18, 19, 20, 19, 4, 5, 22, 23, 24, 47, 32, 33, 34, 35, 36, 35, 52, 53, 38, 39, 40, 63, 48, 49, 50, 51, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt386(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 21, 6, 7, 8, 15, 0, 17, 18, 19, 20, 19, 4, 5, 22, 23, 24, 63, 48, 33, 34, 35, 36, 35, 52, 53, 38, 39, 40, 47, 32, 49, 50, 51, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt387(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 21, 6, 7, 8, 31, 0, 17, 18, 19, 20, 19, 4, 5, 22, 23, 24, 47, 48, 33, 34, 35, 36, 35, 52, 53, 38, 39, 40, 63, 32, 49, 50, 51, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt388(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 21, 6, 7, 8, 15, 16, 1, 18, 19, 20, 19, 4, 5, 22, 23, 24, 63, 32, 49, 34, 35, 36, 35, 52, 53, 38, 39, 40, 47, 48, 33, 50, 51, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt389(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 21, 6, 7, 8, 31, 16, 1, 18, 19, 20, 19, 4, 5, 22, 23, 24, 47, 32, 49, 34, 35, 36, 35, 52, 53, 38, 39, 40, 63, 48, 33, 50, 51, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt390(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 21, 6, 7, 8, 15, 0, 1, 18, 19, 20, 19, 4, 5, 22, 23, 24, 63, 48, 49, 34, 35, 36, 35, 52, 53, 38, 39, 40, 47, 32, 33, 50, 51, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt391(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 21, 6, 7, 8, 31, 0, 1, 18, 19, 20, 19, 4, 5, 22, 23, 24, 47, 48, 49, 34, 35, 36, 35, 52, 53, 38, 39, 40, 63, 32, 33, 50, 51, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt392(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 21, 6, 7, 8, 15, 16, 17, 2, 19, 20, 19, 4, 5, 22, 23, 24, 63, 32, 33, 50, 35, 36, 35, 52, 53, 38, 39, 40, 47, 48, 49, 34, 51, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt393(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 21, 6, 7, 8, 31, 16, 17, 2, 19, 20, 19, 4, 5, 22, 23, 24, 47, 32, 33, 50, 35, 36, 35, 52, 53, 38, 39, 40, 63, 48, 49, 34, 51, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt394(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 21, 6, 7, 8, 15, 0, 17, 2, 19, 20, 19, 4, 5, 22, 23, 24, 63, 48, 33, 50, 35, 36, 35, 52, 53, 38, 39, 40, 47, 32, 49, 34, 51, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt395(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 21, 6, 7, 8, 31, 0, 17, 2, 19, 20, 19, 4, 5, 22, 23, 24, 47, 48, 33, 50, 35, 36, 35, 52, 53, 38, 39, 40, 63, 32, 49, 34, 51, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt396(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 21, 6, 7, 8, 15, 16, 1, 2, 19, 20, 19, 4, 5, 22, 23, 24, 63, 32, 49, 50, 35, 36, 35, 52, 53, 38, 39, 40, 47, 48, 33, 34, 51, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt397(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 21, 6, 7, 8, 31, 16, 1, 2, 19, 20, 19, 4, 5, 22, 23, 24, 47, 32, 49, 50, 35, 36, 35, 52, 53, 38, 39, 40, 63, 48, 33, 34, 51, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt398(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 21, 6, 7, 8, 15, 0, 1, 2, 19, 20, 19, 4, 5, 22, 23, 24, 63, 48, 49, 50, 35, 36, 35, 52, 53, 38, 39, 40, 47, 32, 33, 34, 51, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt399(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 21, 6, 7, 8, 31, 0, 1, 2, 19, 20, 19, 4, 5, 22, 23, 24, 47, 48, 49, 50, 35, 36, 35, 52, 53, 38, 39, 40, 63, 32, 33, 34, 51, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt400(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 21, 6, 7, 8, 15, 16, 17, 18, 3, 20, 19, 4, 5, 22, 23, 24, 63, 32, 33, 34, 51, 36, 35, 52, 53, 38, 39, 40, 47, 48, 49, 50, 35, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt401(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 21, 6, 7, 8, 31, 16, 17, 18, 3, 20, 19, 4, 5, 22, 23, 24, 47, 32, 33, 34, 51, 36, 35, 52, 53, 38, 39, 40, 63, 48, 49, 50, 35, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt402(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 21, 6, 7, 8, 15, 0, 17, 18, 3, 20, 19, 4, 5, 22, 23, 24, 63, 48, 33, 34, 51, 36, 35, 52, 53, 38, 39, 40, 47, 32, 49, 50, 35, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt403(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 21, 6, 7, 8, 31, 0, 17, 18, 3, 20, 19, 4, 5, 22, 23, 24, 47, 48, 33, 34, 51, 36, 35, 52, 53, 38, 39, 40, 63, 32, 49, 50, 35, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt404(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 21, 6, 7, 8, 15, 16, 1, 18, 3, 20, 19, 4, 5, 22, 23, 24, 63, 32, 49, 34, 51, 36, 35, 52, 53, 38, 39, 40, 47, 48, 33, 50, 35, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt405(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 21, 6, 7, 8, 31, 16, 1, 18, 3, 20, 19, 4, 5, 22, 23, 24, 47, 32, 49, 34, 51, 36, 35, 52, 53, 38, 39, 40, 63, 48, 33, 50, 35, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt406(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 21, 6, 7, 8, 15, 0, 1, 18, 3, 20, 19, 4, 5, 22, 23, 24, 63, 48, 49, 34, 51, 36, 35, 52, 53, 38, 39, 40, 47, 32, 33, 50, 35, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt407(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 21, 6, 7, 8, 31, 0, 1, 18, 3, 20, 19, 4, 5, 22, 23, 24, 47, 48, 49, 34, 51, 36, 35, 52, 53, 38, 39, 40, 63, 32, 33, 50, 35, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt408(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 21, 6, 7, 8, 15, 16, 17, 2, 3, 20, 19, 4, 5, 22, 23, 24, 63, 32, 33, 50, 51, 36, 35, 52, 53, 38, 39, 40, 47, 48, 49, 34, 35, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt409(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 21, 6, 7, 8, 31, 16, 17, 2, 3, 20, 19, 4, 5, 22, 23, 24, 47, 32, 33, 50, 51, 36, 35, 52, 53, 38, 39, 40, 63, 48, 49, 34, 35, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt410(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 21, 6, 7, 8, 15, 0, 17, 2, 3, 20, 19, 4, 5, 22, 23, 24, 63, 48, 33, 50, 51, 36, 35, 52, 53, 38, 39, 40, 47, 32, 49, 34, 35, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt411(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 21, 6, 7, 8, 31, 0, 17, 2, 3, 20, 19, 4, 5, 22, 23, 24, 47, 48, 33, 50, 51, 36, 35, 52, 53, 38, 39, 40, 63, 32, 49, 34, 35, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt412(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 21, 6, 7, 8, 15, 16, 1, 2, 3, 20, 19, 4, 5, 22, 23, 24, 63, 32, 49, 50, 51, 36, 35, 52, 53, 38, 39, 40, 47, 48, 33, 34, 35, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt413(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 21, 6, 7, 8, 31, 16, 1, 2, 3, 20, 19, 4, 5, 22, 23, 24, 47, 32, 49, 50, 51, 36, 35, 52, 53, 38, 39, 40, 63, 48, 33, 34, 35, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt414(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 21, 6, 7, 8, 15, 0, 1, 2, 3, 20, 19, 4, 5, 22, 23, 24, 63, 48, 49, 50, 51, 36, 35, 52, 53, 38, 39, 40, 47, 32, 33, 34, 35, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt415(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 21, 6, 7, 8, 31, 0, 1, 2, 3, 20, 19, 4, 5, 22, 23, 24, 47, 48, 49, 50, 51, 36, 35, 52, 53, 38, 39, 40, 63, 32, 33, 34, 35, 52, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt416(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 21, 6, 7, 8, 15, 16, 17, 18, 19, 4, 19, 4, 5, 22, 23, 24, 63, 32, 33, 34, 35, 52, 35, 52, 53, 38, 39, 40, 47, 48, 49, 50, 51, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt417(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 21, 6, 7, 8, 31, 16, 17, 18, 19, 4, 19, 4, 5, 22, 23, 24, 47, 32, 33, 34, 35, 52, 35, 52, 53, 38, 39, 40, 63, 48, 49, 50, 51, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt418(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 21, 6, 7, 8, 15, 0, 17, 18, 19, 4, 19, 4, 5, 22, 23, 24, 63, 48, 33, 34, 35, 52, 35, 52, 53, 38, 39, 40, 47, 32, 49, 50, 51, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt419(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 21, 6, 7, 8, 31, 0, 17, 18, 19, 4, 19, 4, 5, 22, 23, 24, 47, 48, 33, 34, 35, 52, 35, 52, 53, 38, 39, 40, 63, 32, 49, 50, 51, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt420(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 21, 6, 7, 8, 15, 16, 1, 18, 19, 4, 19, 4, 5, 22, 23, 24, 63, 32, 49, 34, 35, 52, 35, 52, 53, 38, 39, 40, 47, 48, 33, 50, 51, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt421(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 21, 6, 7, 8, 31, 16, 1, 18, 19, 4, 19, 4, 5, 22, 23, 24, 47, 32, 49, 34, 35, 52, 35, 52, 53, 38, 39, 40, 63, 48, 33, 50, 51, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt422(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 21, 6, 7, 8, 15, 0, 1, 18, 19, 4, 19, 4, 5, 22, 23, 24, 63, 48, 49, 34, 35, 52, 35, 52, 53, 38, 39, 40, 47, 32, 33, 50, 51, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt423(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 21, 6, 7, 8, 31, 0, 1, 18, 19, 4, 19, 4, 5, 22, 23, 24, 47, 48, 49, 34, 35, 52, 35, 52, 53, 38, 39, 40, 63, 32, 33, 50, 51, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt424(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 21, 6, 7, 8, 15, 16, 17, 2, 19, 4, 19, 4, 5, 22, 23, 24, 63, 32, 33, 50, 35, 52, 35, 52, 53, 38, 39, 40, 47, 48, 49, 34, 51, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt425(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 21, 6, 7, 8, 31, 16, 17, 2, 19, 4, 19, 4, 5, 22, 23, 24, 47, 32, 33, 50, 35, 52, 35, 52, 53, 38, 39, 40, 63, 48, 49, 34, 51, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt426(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 21, 6, 7, 8, 15, 0, 17, 2, 19, 4, 19, 4, 5, 22, 23, 24, 63, 48, 33, 50, 35, 52, 35, 52, 53, 38, 39, 40, 47, 32, 49, 34, 51, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt427(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 21, 6, 7, 8, 31, 0, 17, 2, 19, 4, 19, 4, 5, 22, 23, 24, 47, 48, 33, 50, 35, 52, 35, 52, 53, 38, 39, 40, 63, 32, 49, 34, 51, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt428(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 21, 6, 7, 8, 15, 16, 1, 2, 19, 4, 19, 4, 5, 22, 23, 24, 63, 32, 49, 50, 35, 52, 35, 52, 53, 38, 39, 40, 47, 48, 33, 34, 51, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt429(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 21, 6, 7, 8, 31, 16, 1, 2, 19, 4, 19, 4, 5, 22, 23, 24, 47, 32, 49, 50, 35, 52, 35, 52, 53, 38, 39, 40, 63, 48, 33, 34, 51, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt430(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 21, 6, 7, 8, 15, 0, 1, 2, 19, 4, 19, 4, 5, 22, 23, 24, 63, 48, 49, 50, 35, 52, 35, 52, 53, 38, 39, 40, 47, 32, 33, 34, 51, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt431(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 21, 6, 7, 8, 31, 0, 1, 2, 19, 4, 19, 4, 5, 22, 23, 24, 47, 48, 49, 50, 35, 52, 35, 52, 53, 38, 39, 40, 63, 32, 33, 34, 51, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt432(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 21, 6, 7, 8, 15, 16, 17, 18, 3, 4, 19, 4, 5, 22, 23, 24, 63, 32, 33, 34, 51, 52, 35, 52, 53, 38, 39, 40, 47, 48, 49, 50, 35, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt433(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 21, 6, 7, 8, 31, 16, 17, 18, 3, 4, 19, 4, 5, 22, 23, 24, 47, 32, 33, 34, 51, 52, 35, 52, 53, 38, 39, 40, 63, 48, 49, 50, 35, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt434(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 21, 6, 7, 8, 15, 0, 17, 18, 3, 4, 19, 4, 5, 22, 23, 24, 63, 48, 33, 34, 51, 52, 35, 52, 53, 38, 39, 40, 47, 32, 49, 50, 35, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt435(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 21, 6, 7, 8, 31, 0, 17, 18, 3, 4, 19, 4, 5, 22, 23, 24, 47, 48, 33, 34, 51, 52, 35, 52, 53, 38, 39, 40, 63, 32, 49, 50, 35, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt436(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 21, 6, 7, 8, 15, 16, 1, 18, 3, 4, 19, 4, 5, 22, 23, 24, 63, 32, 49, 34, 51, 52, 35, 52, 53, 38, 39, 40, 47, 48, 33, 50, 35, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt437(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 21, 6, 7, 8, 31, 16, 1, 18, 3, 4, 19, 4, 5, 22, 23, 24, 47, 32, 49, 34, 51, 52, 35, 52, 53, 38, 39, 40, 63, 48, 33, 50, 35, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt438(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 21, 6, 7, 8, 15, 0, 1, 18, 3, 4, 19, 4, 5, 22, 23, 24, 63, 48, 49, 34, 51, 52, 35, 52, 53, 38, 39, 40, 47, 32, 33, 50, 35, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt439(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 21, 6, 7, 8, 31, 0, 1, 18, 3, 4, 19, 4, 5, 22, 23, 24, 47, 48, 49, 34, 51, 52, 35, 52, 53, 38, 39, 40, 63, 32, 33, 50, 35, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt440(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 21, 6, 7, 8, 15, 16, 17, 2, 3, 4, 19, 4, 5, 22, 23, 24, 63, 32, 33, 50, 51, 52, 35, 52, 53, 38, 39, 40, 47, 48, 49, 34, 35, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt441(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 21, 6, 7, 8, 31, 16, 17, 2, 3, 4, 19, 4, 5, 22, 23, 24, 47, 32, 33, 50, 51, 52, 35, 52, 53, 38, 39, 40, 63, 48, 49, 34, 35, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt442(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 21, 6, 7, 8, 15, 0, 17, 2, 3, 4, 19, 4, 5, 22, 23, 24, 63, 48, 33, 50, 51, 52, 35, 52, 53, 38, 39, 40, 47, 32, 49, 34, 35, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt443(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 21, 6, 7, 8, 31, 0, 17, 2, 3, 4, 19, 4, 5, 22, 23, 24, 47, 48, 33, 50, 51, 52, 35, 52, 53, 38, 39, 40, 63, 32, 49, 34, 35, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt444(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 21, 6, 7, 8, 15, 16, 1, 2, 3, 4, 19, 4, 5, 22, 23, 24, 63, 32, 49, 50, 51, 52, 35, 52, 53, 38, 39, 40, 47, 48, 33, 34, 35, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt445(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 21, 6, 7, 8, 31, 16, 1, 2, 3, 4, 19, 4, 5, 22, 23, 24, 47, 32, 49, 50, 51, 52, 35, 52, 53, 38, 39, 40, 63, 48, 33, 34, 35, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt446(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 21, 6, 7, 8, 15, 0, 1, 2, 3, 4, 19, 4, 5, 22, 23, 24, 63, 48, 49, 50, 51, 52, 35, 52, 53, 38, 39, 40, 47, 32, 33, 34, 35, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt447(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 21, 6, 7, 8, 31, 0, 1, 2, 3, 4, 19, 4, 5, 22, 23, 24, 47, 48, 49, 50, 51, 52, 35, 52, 53, 38, 39, 40, 63, 32, 33, 34, 35, 36, 51, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt448(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 21, 6, 7, 8, 15, 16, 17, 18, 19, 20, 3, 4, 5, 22, 23, 24, 63, 32, 33, 34, 35, 36, 51, 52, 53, 38, 39, 40, 47, 48, 49, 50, 51, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt449(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 21, 6, 7, 8, 31, 16, 17, 18, 19, 20, 3, 4, 5, 22, 23, 24, 47, 32, 33, 34, 35, 36, 51, 52, 53, 38, 39, 40, 63, 48, 49, 50, 51, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt450(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 21, 6, 7, 8, 15, 0, 17, 18, 19, 20, 3, 4, 5, 22, 23, 24, 63, 48, 33, 34, 35, 36, 51, 52, 53, 38, 39, 40, 47, 32, 49, 50, 51, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt451(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 21, 6, 7, 8, 31, 0, 17, 18, 19, 20, 3, 4, 5, 22, 23, 24, 47, 48, 33, 34, 35, 36, 51, 52, 53, 38, 39, 40, 63, 32, 49, 50, 51, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt452(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 21, 6, 7, 8, 15, 16, 1, 18, 19, 20, 3, 4, 5, 22, 23, 24, 63, 32, 49, 34, 35, 36, 51, 52, 53, 38, 39, 40, 47, 48, 33, 50, 51, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt453(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 21, 6, 7, 8, 31, 16, 1, 18, 19, 20, 3, 4, 5, 22, 23, 24, 47, 32, 49, 34, 35, 36, 51, 52, 53, 38, 39, 40, 63, 48, 33, 50, 51, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt454(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 21, 6, 7, 8, 15, 0, 1, 18, 19, 20, 3, 4, 5, 22, 23, 24, 63, 48, 49, 34, 35, 36, 51, 52, 53, 38, 39, 40, 47, 32, 33, 50, 51, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt455(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 21, 6, 7, 8, 31, 0, 1, 18, 19, 20, 3, 4, 5, 22, 23, 24, 47, 48, 49, 34, 35, 36, 51, 52, 53, 38, 39, 40, 63, 32, 33, 50, 51, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt456(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 21, 6, 7, 8, 15, 16, 17, 2, 19, 20, 3, 4, 5, 22, 23, 24, 63, 32, 33, 50, 35, 36, 51, 52, 53, 38, 39, 40, 47, 48, 49, 34, 51, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt457(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 21, 6, 7, 8, 31, 16, 17, 2, 19, 20, 3, 4, 5, 22, 23, 24, 47, 32, 33, 50, 35, 36, 51, 52, 53, 38, 39, 40, 63, 48, 49, 34, 51, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt458(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 21, 6, 7, 8, 15, 0, 17, 2, 19, 20, 3, 4, 5, 22, 23, 24, 63, 48, 33, 50, 35, 36, 51, 52, 53, 38, 39, 40, 47, 32, 49, 34, 51, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt459(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 21, 6, 7, 8, 31, 0, 17, 2, 19, 20, 3, 4, 5, 22, 23, 24, 47, 48, 33, 50, 35, 36, 51, 52, 53, 38, 39, 40, 63, 32, 49, 34, 51, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt460(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 21, 6, 7, 8, 15, 16, 1, 2, 19, 20, 3, 4, 5, 22, 23, 24, 63, 32, 49, 50, 35, 36, 51, 52, 53, 38, 39, 40, 47, 48, 33, 34, 51, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt461(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 21, 6, 7, 8, 31, 16, 1, 2, 19, 20, 3, 4, 5, 22, 23, 24, 47, 32, 49, 50, 35, 36, 51, 52, 53, 38, 39, 40, 63, 48, 33, 34, 51, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt462(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 21, 6, 7, 8, 15, 0, 1, 2, 19, 20, 3, 4, 5, 22, 23, 24, 63, 48, 49, 50, 35, 36, 51, 52, 53, 38, 39, 40, 47, 32, 33, 34, 51, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt463(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 21, 6, 7, 8, 31, 0, 1, 2, 19, 20, 3, 4, 5, 22, 23, 24, 47, 48, 49, 50, 35, 36, 51, 52, 53, 38, 39, 40, 63, 32, 33, 34, 51, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt464(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 21, 6, 7, 8, 15, 16, 17, 18, 3, 20, 3, 4, 5, 22, 23, 24, 63, 32, 33, 34, 51, 36, 51, 52, 53, 38, 39, 40, 47, 48, 49, 50, 35, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt465(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 21, 6, 7, 8, 31, 16, 17, 18, 3, 20, 3, 4, 5, 22, 23, 24, 47, 32, 33, 34, 51, 36, 51, 52, 53, 38, 39, 40, 63, 48, 49, 50, 35, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt466(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 21, 6, 7, 8, 15, 0, 17, 18, 3, 20, 3, 4, 5, 22, 23, 24, 63, 48, 33, 34, 51, 36, 51, 52, 53, 38, 39, 40, 47, 32, 49, 50, 35, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt467(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 21, 6, 7, 8, 31, 0, 17, 18, 3, 20, 3, 4, 5, 22, 23, 24, 47, 48, 33, 34, 51, 36, 51, 52, 53, 38, 39, 40, 63, 32, 49, 50, 35, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt468(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 21, 6, 7, 8, 15, 16, 1, 18, 3, 20, 3, 4, 5, 22, 23, 24, 63, 32, 49, 34, 51, 36, 51, 52, 53, 38, 39, 40, 47, 48, 33, 50, 35, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt469(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 21, 6, 7, 8, 31, 16, 1, 18, 3, 20, 3, 4, 5, 22, 23, 24, 47, 32, 49, 34, 51, 36, 51, 52, 53, 38, 39, 40, 63, 48, 33, 50, 35, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt470(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 21, 6, 7, 8, 15, 0, 1, 18, 3, 20, 3, 4, 5, 22, 23, 24, 63, 48, 49, 34, 51, 36, 51, 52, 53, 38, 39, 40, 47, 32, 33, 50, 35, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt471(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 21, 6, 7, 8, 31, 0, 1, 18, 3, 20, 3, 4, 5, 22, 23, 24, 47, 48, 49, 34, 51, 36, 51, 52, 53, 38, 39, 40, 63, 32, 33, 50, 35, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt472(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 21, 6, 7, 8, 15, 16, 17, 2, 3, 20, 3, 4, 5, 22, 23, 24, 63, 32, 33, 50, 51, 36, 51, 52, 53, 38, 39, 40, 47, 48, 49, 34, 35, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt473(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 21, 6, 7, 8, 31, 16, 17, 2, 3, 20, 3, 4, 5, 22, 23, 24, 47, 32, 33, 50, 51, 36, 51, 52, 53, 38, 39, 40, 63, 48, 49, 34, 35, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt474(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 21, 6, 7, 8, 15, 0, 17, 2, 3, 20, 3, 4, 5, 22, 23, 24, 63, 48, 33, 50, 51, 36, 51, 52, 53, 38, 39, 40, 47, 32, 49, 34, 35, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt475(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 21, 6, 7, 8, 31, 0, 17, 2, 3, 20, 3, 4, 5, 22, 23, 24, 47, 48, 33, 50, 51, 36, 51, 52, 53, 38, 39, 40, 63, 32, 49, 34, 35, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt476(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 21, 6, 7, 8, 15, 16, 1, 2, 3, 20, 3, 4, 5, 22, 23, 24, 63, 32, 49, 50, 51, 36, 51, 52, 53, 38, 39, 40, 47, 48, 33, 34, 35, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt477(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 21, 6, 7, 8, 31, 16, 1, 2, 3, 20, 3, 4, 5, 22, 23, 24, 47, 32, 49, 50, 51, 36, 51, 52, 53, 38, 39, 40, 63, 48, 33, 34, 35, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt478(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 21, 6, 7, 8, 15, 0, 1, 2, 3, 20, 3, 4, 5, 22, 23, 24, 63, 48, 49, 50, 51, 36, 51, 52, 53, 38, 39, 40, 47, 32, 33, 34, 35, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt479(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 21, 6, 7, 8, 31, 0, 1, 2, 3, 20, 3, 4, 5, 22, 23, 24, 47, 48, 49, 50, 51, 36, 51, 52, 53, 38, 39, 40, 63, 32, 33, 34, 35, 52, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt480(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 21, 6, 7, 8, 15, 16, 17, 18, 19, 4, 3, 4, 5, 22, 23, 24, 63, 32, 33, 34, 35, 52, 51, 52, 53, 38, 39, 40, 47, 48, 49, 50, 51, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt481(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 21, 6, 7, 8, 31, 16, 17, 18, 19, 4, 3, 4, 5, 22, 23, 24, 47, 32, 33, 34, 35, 52, 51, 52, 53, 38, 39, 40, 63, 48, 49, 50, 51, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt482(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 21, 6, 7, 8, 15, 0, 17, 18, 19, 4, 3, 4, 5, 22, 23, 24, 63, 48, 33, 34, 35, 52, 51, 52, 53, 38, 39, 40, 47, 32, 49, 50, 51, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt483(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 21, 6, 7, 8, 31, 0, 17, 18, 19, 4, 3, 4, 5, 22, 23, 24, 47, 48, 33, 34, 35, 52, 51, 52, 53, 38, 39, 40, 63, 32, 49, 50, 51, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt484(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 21, 6, 7, 8, 15, 16, 1, 18, 19, 4, 3, 4, 5, 22, 23, 24, 63, 32, 49, 34, 35, 52, 51, 52, 53, 38, 39, 40, 47, 48, 33, 50, 51, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt485(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 21, 6, 7, 8, 31, 16, 1, 18, 19, 4, 3, 4, 5, 22, 23, 24, 47, 32, 49, 34, 35, 52, 51, 52, 53, 38, 39, 40, 63, 48, 33, 50, 51, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt486(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 21, 6, 7, 8, 15, 0, 1, 18, 19, 4, 3, 4, 5, 22, 23, 24, 63, 48, 49, 34, 35, 52, 51, 52, 53, 38, 39, 40, 47, 32, 33, 50, 51, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt487(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 21, 6, 7, 8, 31, 0, 1, 18, 19, 4, 3, 4, 5, 22, 23, 24, 47, 48, 49, 34, 35, 52, 51, 52, 53, 38, 39, 40, 63, 32, 33, 50, 51, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt488(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 21, 6, 7, 8, 15, 16, 17, 2, 19, 4, 3, 4, 5, 22, 23, 24, 63, 32, 33, 50, 35, 52, 51, 52, 53, 38, 39, 40, 47, 48, 49, 34, 51, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt489(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 21, 6, 7, 8, 31, 16, 17, 2, 19, 4, 3, 4, 5, 22, 23, 24, 47, 32, 33, 50, 35, 52, 51, 52, 53, 38, 39, 40, 63, 48, 49, 34, 51, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt490(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 21, 6, 7, 8, 15, 0, 17, 2, 19, 4, 3, 4, 5, 22, 23, 24, 63, 48, 33, 50, 35, 52, 51, 52, 53, 38, 39, 40, 47, 32, 49, 34, 51, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt491(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 21, 6, 7, 8, 31, 0, 17, 2, 19, 4, 3, 4, 5, 22, 23, 24, 47, 48, 33, 50, 35, 52, 51, 52, 53, 38, 39, 40, 63, 32, 49, 34, 51, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt492(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 21, 6, 7, 8, 15, 16, 1, 2, 19, 4, 3, 4, 5, 22, 23, 24, 63, 32, 49, 50, 35, 52, 51, 52, 53, 38, 39, 40, 47, 48, 33, 34, 51, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt493(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 21, 6, 7, 8, 31, 16, 1, 2, 19, 4, 3, 4, 5, 22, 23, 24, 47, 32, 49, 50, 35, 52, 51, 52, 53, 38, 39, 40, 63, 48, 33, 34, 51, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt494(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 21, 6, 7, 8, 15, 0, 1, 2, 19, 4, 3, 4, 5, 22, 23, 24, 63, 48, 49, 50, 35, 52, 51, 52, 53, 38, 39, 40, 47, 32, 33, 34, 51, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt495(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 21, 6, 7, 8, 31, 0, 1, 2, 19, 4, 3, 4, 5, 22, 23, 24, 47, 48, 49, 50, 35, 52, 51, 52, 53, 38, 39, 40, 63, 32, 33, 34, 51, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt496(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 21, 6, 7, 8, 15, 16, 17, 18, 3, 4, 3, 4, 5, 22, 23, 24, 63, 32, 33, 34, 51, 52, 51, 52, 53, 38, 39, 40, 47, 48, 49, 50, 35, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt497(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 21, 6, 7, 8, 31, 16, 17, 18, 3, 4, 3, 4, 5, 22, 23, 24, 47, 32, 33, 34, 51, 52, 51, 52, 53, 38, 39, 40, 63, 48, 49, 50, 35, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt498(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 21, 6, 7, 8, 15, 0, 17, 18, 3, 4, 3, 4, 5, 22, 23, 24, 63, 48, 33, 34, 51, 52, 51, 52, 53, 38, 39, 40, 47, 32, 49, 50, 35, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt499(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 21, 6, 7, 8, 31, 0, 17, 18, 3, 4, 3, 4, 5, 22, 23, 24, 47, 48, 33, 34, 51, 52, 51, 52, 53, 38, 39, 40, 63, 32, 49, 50, 35, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt500(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 21, 6, 7, 8, 15, 16, 1, 18, 3, 4, 3, 4, 5, 22, 23, 24, 63, 32, 49, 34, 51, 52, 51, 52, 53, 38, 39, 40, 47, 48, 33, 50, 35, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt501(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 21, 6, 7, 8, 31, 16, 1, 18, 3, 4, 3, 4, 5, 22, 23, 24, 47, 32, 49, 34, 51, 52, 51, 52, 53, 38, 39, 40, 63, 48, 33, 50, 35, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt502(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 21, 6, 7, 8, 15, 0, 1, 18, 3, 4, 3, 4, 5, 22, 23, 24, 63, 48, 49, 34, 51, 52, 51, 52, 53, 38, 39, 40, 47, 32, 33, 50, 35, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt503(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 21, 6, 7, 8, 31, 0, 1, 18, 3, 4, 3, 4, 5, 22, 23, 24, 47, 48, 49, 34, 51, 52, 51, 52, 53, 38, 39, 40, 63, 32, 33, 50, 35, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt504(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 21, 6, 7, 8, 15, 16, 17, 2, 3, 4, 3, 4, 5, 22, 23, 24, 63, 32, 33, 50, 51, 52, 51, 52, 53, 38, 39, 40, 47, 48, 49, 34, 35, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt505(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 21, 6, 7, 8, 31, 16, 17, 2, 3, 4, 3, 4, 5, 22, 23, 24, 47, 32, 33, 50, 51, 52, 51, 52, 53, 38, 39, 40, 63, 48, 49, 34, 35, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt506(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 21, 6, 7, 8, 15, 0, 17, 2, 3, 4, 3, 4, 5, 22, 23, 24, 63, 48, 33, 50, 51, 52, 51, 52, 53, 38, 39, 40, 47, 32, 49, 34, 35, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt507(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 21, 6, 7, 8, 31, 0, 17, 2, 3, 4, 3, 4, 5, 22, 23, 24, 47, 48, 33, 50, 51, 52, 51, 52, 53, 38, 39, 40, 63, 32, 49, 34, 35, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt508(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 21, 6, 7, 8, 15, 16, 1, 2, 3, 4, 3, 4, 5, 22, 23, 24, 63, 32, 49, 50, 51, 52, 51, 52, 53, 38, 39, 40, 47, 48, 33, 34, 35, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt509(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 21, 6, 7, 8, 31, 16, 1, 2, 3, 4, 3, 4, 5, 22, 23, 24, 47, 32, 49, 50, 51, 52, 51, 52, 53, 38, 39, 40, 63, 48, 33, 34, 35, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt510(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 21, 6, 7, 8, 15, 0, 1, 2, 3, 4, 3, 4, 5, 22, 23, 24, 63, 48, 49, 50, 51, 52, 51, 52, 53, 38, 39, 40, 47, 32, 33, 34, 35, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt511(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 21, 6, 7, 8, 31, 0, 1, 2, 3, 4, 3, 4, 5, 22, 23, 24, 47, 48, 49, 50, 51, 52, 51, 52, 53, 38, 39, 40, 63, 32, 33, 34, 35, 36, 35, 36, 37, 54, 55, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

#endif // DESGPU_COMPILE_ALL_SALTS


