#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt384(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 21, 6, 7, 8, 15, 16, 17, 18, 19, 20, 19, 4, 5, 22, 23, 24, 63, 32, 33, 34, 35, 36, 35, 52, 53, 38, 39, 40, 47, 48, 49, 50, 51, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt385(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 21, 6, 7, 8, 31, 16, 17, 18, 19, 20, 19, 4, 5, 22, 23, 24, 47, 32, 33, 34, 35, 36, 35, 52, 53, 38, 39, 40, 63, 48, 49, 50, 51, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt386(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 21, 6, 7, 8, 15, 0, 17, 18, 19, 20, 19, 4, 5, 22, 23, 24, 63, 48, 33, 34, 35, 36, 35, 52, 53, 38, 39, 40, 47, 32, 49, 50, 51, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt387(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 21, 6, 7, 8, 31, 0, 17, 18, 19, 20, 19, 4, 5, 22, 23, 24, 47, 48, 33, 34, 35, 36, 35, 52, 53, 38, 39, 40, 63, 32, 49, 50, 51, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt388(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 21, 6, 7, 8, 15, 16, 1, 18, 19, 20, 19, 4, 5, 22, 23, 24, 63, 32, 49, 34, 35, 36, 35, 52, 53, 38, 39, 40, 47, 48, 33, 50, 51, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt389(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 21, 6, 7, 8, 31, 16, 1, 18, 19, 20, 19, 4, 5, 22, 23, 24, 47, 32, 49, 34, 35, 36, 35, 52, 53, 38, 39, 40, 63, 48, 33, 50, 51, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt390(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 21, 6, 7, 8, 15, 0, 1, 18, 19, 20, 19, 4, 5, 22, 23, 24, 63, 48, 49, 34, 35, 36, 35, 52, 53, 38, 39, 40, 47, 32, 33, 50, 51, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt391(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 21, 6, 7, 8, 31, 0, 1, 18, 19, 20, 19, 4, 5, 22, 23, 24, 47, 48, 49, 34, 35, 36, 35, 52, 53, 38, 39, 40, 63, 32, 33, 50, 51, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt392(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 21, 6, 7, 8, 15, 16, 17, 2, 19, 20, 19, 4, 5, 22, 23, 24, 63, 32, 33, 50, 35, 36, 35, 52, 53, 38, 39, 40, 47, 48, 49, 34, 51, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt393(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 21, 6, 7, 8, 31, 16, 17, 2, 19, 20, 19, 4, 5, 22, 23, 24, 47, 32, 33, 50, 35, 36, 35, 52, 53, 38, 39, 40, 63, 48, 49, 34, 51, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt394(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 21, 6, 7, 8, 15, 0, 17, 2, 19, 20, 19, 4, 5, 22, 23, 24, 63, 48, 33, 50, 35, 36, 35, 52, 53, 38, 39, 40, 47, 32, 49, 34, 51, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt395(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 21, 6, 7, 8, 31, 0, 17, 2, 19, 20, 19, 4, 5, 22, 23, 24, 47, 48, 33, 50, 35, 36, 35, 52, 53, 38, 39, 40, 63, 32, 49, 34, 51, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt396(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 21, 6, 7, 8, 15, 16, 1, 2, 19, 20, 19, 4, 5, 22, 23, 24, 63, 32, 49, 50, 35, 36, 35, 52, 53, 38, 39, 40, 47, 48, 33, 34, 51, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt397(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 21, 6, 7, 8, 31, 16, 1, 2, 19, 20, 19, 4, 5, 22, 23, 24, 47, 32, 49, 50, 35, 36, 35, 52, 53, 38, 39, 40, 63, 48, 33, 34, 51, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt398(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 21, 6, 7, 8, 15, 0, 1, 2, 19, 20, 19, 4, 5, 22, 23, 24, 63, 48, 49, 50, 35, 36, 35, 52, 53, 38, 39, 40, 47, 32, 33, 34, 51, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt399(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 21, 6, 7, 8, 31, 0, 1, 2, 19, 20, 19, 4, 5, 22, 23, 24, 47, 48, 49, 50, 35, 36, 35, 52, 53, 38, 39, 40, 63, 32, 33, 34, 51, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt400(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 21, 6, 7, 8, 15, 16, 17, 18, 3, 20, 19, 4, 5, 22, 23, 24, 63, 32, 33, 34, 51, 36, 35, 52, 53, 38, 39, 40, 47, 48, 49, 50, 35, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt401(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 21, 6, 7, 8, 31, 16, 17, 18, 3, 20, 19, 4, 5, 22, 23, 24, 47, 32, 33, 34, 51, 36, 35, 52, 53, 38, 39, 40, 63, 48, 49, 50, 35, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt402(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 21, 6, 7, 8, 15, 0, 17, 18, 3, 20, 19, 4, 5, 22, 23, 24, 63, 48, 33, 34, 51, 36, 35, 52, 53, 38, 39, 40, 47, 32, 49, 50, 35, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt403(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 21, 6, 7, 8, 31, 0, 17, 18, 3, 20, 19, 4, 5, 22, 23, 24, 47, 48, 33, 34, 51, 36, 35, 52, 53, 38, 39, 40, 63, 32, 49, 50, 35, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt404(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 21, 6, 7, 8, 15, 16, 1, 18, 3, 20, 19, 4, 5, 22, 23, 24, 63, 32, 49, 34, 51, 36, 35, 52, 53, 38, 39, 40, 47, 48, 33, 50, 35, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt405(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 21, 6, 7, 8, 31, 16, 1, 18, 3, 20, 19, 4, 5, 22, 23, 24, 47, 32, 49, 34, 51, 36, 35, 52, 53, 38, 39, 40, 63, 48, 33, 50, 35, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt406(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 21, 6, 7, 8, 15, 0, 1, 18, 3, 20, 19, 4, 5, 22, 23, 24, 63, 48, 49, 34, 51, 36, 35, 52, 53, 38, 39, 40, 47, 32, 33, 50, 35, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt407(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 21, 6, 7, 8, 31, 0, 1, 18, 3, 20, 19, 4, 5, 22, 23, 24, 47, 48, 49, 34, 51, 36, 35, 52, 53, 38, 39, 40, 63, 32, 33, 50, 35, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt408(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 21, 6, 7, 8, 15, 16, 17, 2, 3, 20, 19, 4, 5, 22, 23, 24, 63, 32, 33, 50, 51, 36, 35, 52, 53, 38, 39, 40, 47, 48, 49, 34, 35, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt409(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 21, 6, 7, 8, 31, 16, 17, 2, 3, 20, 19, 4, 5, 22, 23, 24, 47, 32, 33, 50, 51, 36, 35, 52, 53, 38, 39, 40, 63, 48, 49, 34, 35, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt410(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 21, 6, 7, 8, 15, 0, 17, 2, 3, 20, 19, 4, 5, 22, 23, 24, 63, 48, 33, 50, 51, 36, 35, 52, 53, 38, 39, 40, 47, 32, 49, 34, 35, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt411(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 21, 6, 7, 8, 31, 0, 17, 2, 3, 20, 19, 4, 5, 22, 23, 24, 47, 48, 33, 50, 51, 36, 35, 52, 53, 38, 39, 40, 63, 32, 49, 34, 35, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt412(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 21, 6, 7, 8, 15, 16, 1, 2, 3, 20, 19, 4, 5, 22, 23, 24, 63, 32, 49, 50, 51, 36, 35, 52, 53, 38, 39, 40, 47, 48, 33, 34, 35, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt413(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 21, 6, 7, 8, 31, 16, 1, 2, 3, 20, 19, 4, 5, 22, 23, 24, 47, 32, 49, 50, 51, 36, 35, 52, 53, 38, 39, 40, 63, 48, 33, 34, 35, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt414(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 21, 6, 7, 8, 15, 0, 1, 2, 3, 20, 19, 4, 5, 22, 23, 24, 63, 48, 49, 50, 51, 36, 35, 52, 53, 38, 39, 40, 47, 32, 33, 34, 35, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt415(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 21, 6, 7, 8, 31, 0, 1, 2, 3, 20, 19, 4, 5, 22, 23, 24, 47, 48, 49, 50, 51, 36, 35, 52, 53, 38, 39, 40, 63, 32, 33, 34, 35, 52, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt416(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 21, 6, 7, 8, 15, 16, 17, 18, 19, 4, 19, 4, 5, 22, 23, 24, 63, 32, 33, 34, 35, 52, 35, 52, 53, 38, 39, 40, 47, 48, 49, 50, 51, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt417(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 21, 6, 7, 8, 31, 16, 17, 18, 19, 4, 19, 4, 5, 22, 23, 24, 47, 32, 33, 34, 35, 52, 35, 52, 53, 38, 39, 40, 63, 48, 49, 50, 51, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt418(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 21, 6, 7, 8, 15, 0, 17, 18, 19, 4, 19, 4, 5, 22, 23, 24, 63, 48, 33, 34, 35, 52, 35, 52, 53, 38, 39, 40, 47, 32, 49, 50, 51, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt419(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 21, 6, 7, 8, 31, 0, 17, 18, 19, 4, 19, 4, 5, 22, 23, 24, 47, 48, 33, 34, 35, 52, 35, 52, 53, 38, 39, 40, 63, 32, 49, 50, 51, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt420(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 21, 6, 7, 8, 15, 16, 1, 18, 19, 4, 19, 4, 5, 22, 23, 24, 63, 32, 49, 34, 35, 52, 35, 52, 53, 38, 39, 40, 47, 48, 33, 50, 51, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt421(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 21, 6, 7, 8, 31, 16, 1, 18, 19, 4, 19, 4, 5, 22, 23, 24, 47, 32, 49, 34, 35, 52, 35, 52, 53, 38, 39, 40, 63, 48, 33, 50, 51, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt422(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 21, 6, 7, 8, 15, 0, 1, 18, 19, 4, 19, 4, 5, 22, 23, 24, 63, 48, 49, 34, 35, 52, 35, 52, 53, 38, 39, 40, 47, 32, 33, 50, 51, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt423(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 21, 6, 7, 8, 31, 0, 1, 18, 19, 4, 19, 4, 5, 22, 23, 24, 47, 48, 49, 34, 35, 52, 35, 52, 53, 38, 39, 40, 63, 32, 33, 50, 51, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt424(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 21, 6, 7, 8, 15, 16, 17, 2, 19, 4, 19, 4, 5, 22, 23, 24, 63, 32, 33, 50, 35, 52, 35, 52, 53, 38, 39, 40, 47, 48, 49, 34, 51, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt425(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 21, 6, 7, 8, 31, 16, 17, 2, 19, 4, 19, 4, 5, 22, 23, 24, 47, 32, 33, 50, 35, 52, 35, 52, 53, 38, 39, 40, 63, 48, 49, 34, 51, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt426(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 21, 6, 7, 8, 15, 0, 17, 2, 19, 4, 19, 4, 5, 22, 23, 24, 63, 48, 33, 50, 35, 52, 35, 52, 53, 38, 39, 40, 47, 32, 49, 34, 51, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt427(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 21, 6, 7, 8, 31, 0, 17, 2, 19, 4, 19, 4, 5, 22, 23, 24, 47, 48, 33, 50, 35, 52, 35, 52, 53, 38, 39, 40, 63, 32, 49, 34, 51, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt428(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 21, 6, 7, 8, 15, 16, 1, 2, 19, 4, 19, 4, 5, 22, 23, 24, 63, 32, 49, 50, 35, 52, 35, 52, 53, 38, 39, 40, 47, 48, 33, 34, 51, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt429(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 21, 6, 7, 8, 31, 16, 1, 2, 19, 4, 19, 4, 5, 22, 23, 24, 47, 32, 49, 50, 35, 52, 35, 52, 53, 38, 39, 40, 63, 48, 33, 34, 51, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt430(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 21, 6, 7, 8, 15, 0, 1, 2, 19, 4, 19, 4, 5, 22, 23, 24, 63, 48, 49, 50, 35, 52, 35, 52, 53, 38, 39, 40, 47, 32, 33, 34, 51, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt431(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 21, 6, 7, 8, 31, 0, 1, 2, 19, 4, 19, 4, 5, 22, 23, 24, 47, 48, 49, 50, 35, 52, 35, 52, 53, 38, 39, 40, 63, 32, 33, 34, 51, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt432(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 21, 6, 7, 8, 15, 16, 17, 18, 3, 4, 19, 4, 5, 22, 23, 24, 63, 32, 33, 34, 51, 52, 35, 52, 53, 38, 39, 40, 47, 48, 49, 50, 35, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt433(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 21, 6, 7, 8, 31, 16, 17, 18, 3, 4, 19, 4, 5, 22, 23, 24, 47, 32, 33, 34, 51, 52, 35, 52, 53, 38, 39, 40, 63, 48, 49, 50, 35, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt434(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 21, 6, 7, 8, 15, 0, 17, 18, 3, 4, 19, 4, 5, 22, 23, 24, 63, 48, 33, 34, 51, 52, 35, 52, 53, 38, 39, 40, 47, 32, 49, 50, 35, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt435(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 21, 6, 7, 8, 31, 0, 17, 18, 3, 4, 19, 4, 5, 22, 23, 24, 47, 48, 33, 34, 51, 52, 35, 52, 53, 38, 39, 40, 63, 32, 49, 50, 35, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt436(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 21, 6, 7, 8, 15, 16, 1, 18, 3, 4, 19, 4, 5, 22, 23, 24, 63, 32, 49, 34, 51, 52, 35, 52, 53, 38, 39, 40, 47, 48, 33, 50, 35, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt437(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 21, 6, 7, 8, 31, 16, 1, 18, 3, 4, 19, 4, 5, 22, 23, 24, 47, 32, 49, 34, 51, 52, 35, 52, 53, 38, 39, 40, 63, 48, 33, 50, 35, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt438(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 21, 6, 7, 8, 15, 0, 1, 18, 3, 4, 19, 4, 5, 22, 23, 24, 63, 48, 49, 34, 51, 52, 35, 52, 53, 38, 39, 40, 47, 32, 33, 50, 35, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt439(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 21, 6, 7, 8, 31, 0, 1, 18, 3, 4, 19, 4, 5, 22, 23, 24, 47, 48, 49, 34, 51, 52, 35, 52, 53, 38, 39, 40, 63, 32, 33, 50, 35, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt440(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 21, 6, 7, 8, 15, 16, 17, 2, 3, 4, 19, 4, 5, 22, 23, 24, 63, 32, 33, 50, 51, 52, 35, 52, 53, 38, 39, 40, 47, 48, 49, 34, 35, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt441(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 21, 6, 7, 8, 31, 16, 17, 2, 3, 4, 19, 4, 5, 22, 23, 24, 47, 32, 33, 50, 51, 52, 35, 52, 53, 38, 39, 40, 63, 48, 49, 34, 35, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt442(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 21, 6, 7, 8, 15, 0, 17, 2, 3, 4, 19, 4, 5, 22, 23, 24, 63, 48, 33, 50, 51, 52, 35, 52, 53, 38, 39, 40, 47, 32, 49, 34, 35, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt443(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 21, 6, 7, 8, 31, 0, 17, 2, 3, 4, 19, 4, 5, 22, 23, 24, 47, 48, 33, 50, 51, 52, 35, 52, 53, 38, 39, 40, 63, 32, 49, 34, 35, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt444(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 21, 6, 7, 8, 15, 16, 1, 2, 3, 4, 19, 4, 5, 22, 23, 24, 63, 32, 49, 50, 51, 52, 35, 52, 53, 38, 39, 40, 47, 48, 33, 34, 35, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt445(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 21, 6, 7, 8, 31, 16, 1, 2, 3, 4, 19, 4, 5, 22, 23, 24, 47, 32, 49, 50, 51, 52, 35, 52, 53, 38, 39, 40, 63, 48, 33, 34, 35, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt446(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 21, 6, 7, 8, 15, 0, 1, 2, 3, 4, 19, 4, 5, 22, 23, 24, 63, 48, 49, 50, 51, 52, 35, 52, 53, 38, 39, 40, 47, 32, 33, 34, 35, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt447(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 21, 6, 7, 8, 31, 0, 1, 2, 3, 4, 19, 4, 5, 22, 23, 24, 47, 48, 49, 50, 51, 52, 35, 52, 53, 38, 39, 40, 63, 32, 33, 34, 35, 36, 51, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt448(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 21, 6, 7, 8, 15, 16, 17, 18, 19, 20, 3, 4, 5, 22, 23, 24, 63, 32, 33, 34, 35, 36, 51, 52, 53, 38, 39, 40, 47, 48, 49, 50, 51, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt449(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 21, 6, 7, 8, 31, 16, 17, 18, 19, 20, 3, 4, 5, 22, 23, 24, 47, 32, 33, 34, 35, 36, 51, 52, 53, 38, 39, 40, 63, 48, 49, 50, 51, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt450(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 21, 6, 7, 8, 15, 0, 17, 18, 19, 20, 3, 4, 5, 22, 23, 24, 63, 48, 33, 34, 35, 36, 51, 52, 53, 38, 39, 40, 47, 32, 49, 50, 51, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt451(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 21, 6, 7, 8, 31, 0, 17, 18, 19, 20, 3, 4, 5, 22, 23, 24, 47, 48, 33, 34, 35, 36, 51, 52, 53, 38, 39, 40, 63, 32, 49, 50, 51, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt452(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 21, 6, 7, 8, 15, 16, 1, 18, 19, 20, 3, 4, 5, 22, 23, 24, 63, 32, 49, 34, 35, 36, 51, 52, 53, 38, 39, 40, 47, 48, 33, 50, 51, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt453(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 21, 6, 7, 8, 31, 16, 1, 18, 19, 20, 3, 4, 5, 22, 23, 24, 47, 32, 49, 34, 35, 36, 51, 52, 53, 38, 39, 40, 63, 48, 33, 50, 51, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt454(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 21, 6, 7, 8, 15, 0, 1, 18, 19, 20, 3, 4, 5, 22, 23, 24, 63, 48, 49, 34, 35, 36, 51, 52, 53, 38, 39, 40, 47, 32, 33, 50, 51, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt455(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 21, 6, 7, 8, 31, 0, 1, 18, 19, 20, 3, 4, 5, 22, 23, 24, 47, 48, 49, 34, 35, 36, 51, 52, 53, 38, 39, 40, 63, 32, 33, 50, 51, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt456(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 21, 6, 7, 8, 15, 16, 17, 2, 19, 20, 3, 4, 5, 22, 23, 24, 63, 32, 33, 50, 35, 36, 51, 52, 53, 38, 39, 40, 47, 48, 49, 34, 51, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt457(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 21, 6, 7, 8, 31, 16, 17, 2, 19, 20, 3, 4, 5, 22, 23, 24, 47, 32, 33, 50, 35, 36, 51, 52, 53, 38, 39, 40, 63, 48, 49, 34, 51, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt458(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 21, 6, 7, 8, 15, 0, 17, 2, 19, 20, 3, 4, 5, 22, 23, 24, 63, 48, 33, 50, 35, 36, 51, 52, 53, 38, 39, 40, 47, 32, 49, 34, 51, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt459(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 21, 6, 7, 8, 31, 0, 17, 2, 19, 20, 3, 4, 5, 22, 23, 24, 47, 48, 33, 50, 35, 36, 51, 52, 53, 38, 39, 40, 63, 32, 49, 34, 51, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt460(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 21, 6, 7, 8, 15, 16, 1, 2, 19, 20, 3, 4, 5, 22, 23, 24, 63, 32, 49, 50, 35, 36, 51, 52, 53, 38, 39, 40, 47, 48, 33, 34, 51, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt461(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 21, 6, 7, 8, 31, 16, 1, 2, 19, 20, 3, 4, 5, 22, 23, 24, 47, 32, 49, 50, 35, 36, 51, 52, 53, 38, 39, 40, 63, 48, 33, 34, 51, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt462(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 21, 6, 7, 8, 15, 0, 1, 2, 19, 20, 3, 4, 5, 22, 23, 24, 63, 48, 49, 50, 35, 36, 51, 52, 53, 38, 39, 40, 47, 32, 33, 34, 51, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt463(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 21, 6, 7, 8, 31, 0, 1, 2, 19, 20, 3, 4, 5, 22, 23, 24, 47, 48, 49, 50, 35, 36, 51, 52, 53, 38, 39, 40, 63, 32, 33, 34, 51, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt464(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 21, 6, 7, 8, 15, 16, 17, 18, 3, 20, 3, 4, 5, 22, 23, 24, 63, 32, 33, 34, 51, 36, 51, 52, 53, 38, 39, 40, 47, 48, 49, 50, 35, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt465(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 21, 6, 7, 8, 31, 16, 17, 18, 3, 20, 3, 4, 5, 22, 23, 24, 47, 32, 33, 34, 51, 36, 51, 52, 53, 38, 39, 40, 63, 48, 49, 50, 35, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt466(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 21, 6, 7, 8, 15, 0, 17, 18, 3, 20, 3, 4, 5, 22, 23, 24, 63, 48, 33, 34, 51, 36, 51, 52, 53, 38, 39, 40, 47, 32, 49, 50, 35, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt467(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 21, 6, 7, 8, 31, 0, 17, 18, 3, 20, 3, 4, 5, 22, 23, 24, 47, 48, 33, 34, 51, 36, 51, 52, 53, 38, 39, 40, 63, 32, 49, 50, 35, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt468(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 21, 6, 7, 8, 15, 16, 1, 18, 3, 20, 3, 4, 5, 22, 23, 24, 63, 32, 49, 34, 51, 36, 51, 52, 53, 38, 39, 40, 47, 48, 33, 50, 35, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt469(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 21, 6, 7, 8, 31, 16, 1, 18, 3, 20, 3, 4, 5, 22, 23, 24, 47, 32, 49, 34, 51, 36, 51, 52, 53, 38, 39, 40, 63, 48, 33, 50, 35, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt470(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 21, 6, 7, 8, 15, 0, 1, 18, 3, 20, 3, 4, 5, 22, 23, 24, 63, 48, 49, 34, 51, 36, 51, 52, 53, 38, 39, 40, 47, 32, 33, 50, 35, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt471(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 21, 6, 7, 8, 31, 0, 1, 18, 3, 20, 3, 4, 5, 22, 23, 24, 47, 48, 49, 34, 51, 36, 51, 52, 53, 38, 39, 40, 63, 32, 33, 50, 35, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt472(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 21, 6, 7, 8, 15, 16, 17, 2, 3, 20, 3, 4, 5, 22, 23, 24, 63, 32, 33, 50, 51, 36, 51, 52, 53, 38, 39, 40, 47, 48, 49, 34, 35, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt473(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 21, 6, 7, 8, 31, 16, 17, 2, 3, 20, 3, 4, 5, 22, 23, 24, 47, 32, 33, 50, 51, 36, 51, 52, 53, 38, 39, 40, 63, 48, 49, 34, 35, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt474(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 21, 6, 7, 8, 15, 0, 17, 2, 3, 20, 3, 4, 5, 22, 23, 24, 63, 48, 33, 50, 51, 36, 51, 52, 53, 38, 39, 40, 47, 32, 49, 34, 35, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt475(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 21, 6, 7, 8, 31, 0, 17, 2, 3, 20, 3, 4, 5, 22, 23, 24, 47, 48, 33, 50, 51, 36, 51, 52, 53, 38, 39, 40, 63, 32, 49, 34, 35, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt476(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 21, 6, 7, 8, 15, 16, 1, 2, 3, 20, 3, 4, 5, 22, 23, 24, 63, 32, 49, 50, 51, 36, 51, 52, 53, 38, 39, 40, 47, 48, 33, 34, 35, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt477(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 21, 6, 7, 8, 31, 16, 1, 2, 3, 20, 3, 4, 5, 22, 23, 24, 47, 32, 49, 50, 51, 36, 51, 52, 53, 38, 39, 40, 63, 48, 33, 34, 35, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt478(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 21, 6, 7, 8, 15, 0, 1, 2, 3, 20, 3, 4, 5, 22, 23, 24, 63, 48, 49, 50, 51, 36, 51, 52, 53, 38, 39, 40, 47, 32, 33, 34, 35, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt479(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 21, 6, 7, 8, 31, 0, 1, 2, 3, 20, 3, 4, 5, 22, 23, 24, 47, 48, 49, 50, 51, 36, 51, 52, 53, 38, 39, 40, 63, 32, 33, 34, 35, 52, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt480(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 21, 6, 7, 8, 15, 16, 17, 18, 19, 4, 3, 4, 5, 22, 23, 24, 63, 32, 33, 34, 35, 52, 51, 52, 53, 38, 39, 40, 47, 48, 49, 50, 51, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt481(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 21, 6, 7, 8, 31, 16, 17, 18, 19, 4, 3, 4, 5, 22, 23, 24, 47, 32, 33, 34, 35, 52, 51, 52, 53, 38, 39, 40, 63, 48, 49, 50, 51, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt482(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 21, 6, 7, 8, 15, 0, 17, 18, 19, 4, 3, 4, 5, 22, 23, 24, 63, 48, 33, 34, 35, 52, 51, 52, 53, 38, 39, 40, 47, 32, 49, 50, 51, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt483(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 21, 6, 7, 8, 31, 0, 17, 18, 19, 4, 3, 4, 5, 22, 23, 24, 47, 48, 33, 34, 35, 52, 51, 52, 53, 38, 39, 40, 63, 32, 49, 50, 51, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt484(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 21, 6, 7, 8, 15, 16, 1, 18, 19, 4, 3, 4, 5, 22, 23, 24, 63, 32, 49, 34, 35, 52, 51, 52, 53, 38, 39, 40, 47, 48, 33, 50, 51, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt485(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 21, 6, 7, 8, 31, 16, 1, 18, 19, 4, 3, 4, 5, 22, 23, 24, 47, 32, 49, 34, 35, 52, 51, 52, 53, 38, 39, 40, 63, 48, 33, 50, 51, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt486(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 21, 6, 7, 8, 15, 0, 1, 18, 19, 4, 3, 4, 5, 22, 23, 24, 63, 48, 49, 34, 35, 52, 51, 52, 53, 38, 39, 40, 47, 32, 33, 50, 51, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt487(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 21, 6, 7, 8, 31, 0, 1, 18, 19, 4, 3, 4, 5, 22, 23, 24, 47, 48, 49, 34, 35, 52, 51, 52, 53, 38, 39, 40, 63, 32, 33, 50, 51, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt488(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 21, 6, 7, 8, 15, 16, 17, 2, 19, 4, 3, 4, 5, 22, 23, 24, 63, 32, 33, 50, 35, 52, 51, 52, 53, 38, 39, 40, 47, 48, 49, 34, 51, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt489(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 21, 6, 7, 8, 31, 16, 17, 2, 19, 4, 3, 4, 5, 22, 23, 24, 47, 32, 33, 50, 35, 52, 51, 52, 53, 38, 39, 40, 63, 48, 49, 34, 51, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt490(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 21, 6, 7, 8, 15, 0, 17, 2, 19, 4, 3, 4, 5, 22, 23, 24, 63, 48, 33, 50, 35, 52, 51, 52, 53, 38, 39, 40, 47, 32, 49, 34, 51, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt491(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 21, 6, 7, 8, 31, 0, 17, 2, 19, 4, 3, 4, 5, 22, 23, 24, 47, 48, 33, 50, 35, 52, 51, 52, 53, 38, 39, 40, 63, 32, 49, 34, 51, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt492(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 21, 6, 7, 8, 15, 16, 1, 2, 19, 4, 3, 4, 5, 22, 23, 24, 63, 32, 49, 50, 35, 52, 51, 52, 53, 38, 39, 40, 47, 48, 33, 34, 51, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt493(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 21, 6, 7, 8, 31, 16, 1, 2, 19, 4, 3, 4, 5, 22, 23, 24, 47, 32, 49, 50, 35, 52, 51, 52, 53, 38, 39, 40, 63, 48, 33, 34, 51, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt494(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 21, 6, 7, 8, 15, 0, 1, 2, 19, 4, 3, 4, 5, 22, 23, 24, 63, 48, 49, 50, 35, 52, 51, 52, 53, 38, 39, 40, 47, 32, 33, 34, 51, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt495(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 21, 6, 7, 8, 31, 0, 1, 2, 19, 4, 3, 4, 5, 22, 23, 24, 47, 48, 49, 50, 35, 52, 51, 52, 53, 38, 39, 40, 63, 32, 33, 34, 51, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt496(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 21, 6, 7, 8, 15, 16, 17, 18, 3, 4, 3, 4, 5, 22, 23, 24, 63, 32, 33, 34, 51, 52, 51, 52, 53, 38, 39, 40, 47, 48, 49, 50, 35, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt497(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 21, 6, 7, 8, 31, 16, 17, 18, 3, 4, 3, 4, 5, 22, 23, 24, 47, 32, 33, 34, 51, 52, 51, 52, 53, 38, 39, 40, 63, 48, 49, 50, 35, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt498(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 21, 6, 7, 8, 15, 0, 17, 18, 3, 4, 3, 4, 5, 22, 23, 24, 63, 48, 33, 34, 51, 52, 51, 52, 53, 38, 39, 40, 47, 32, 49, 50, 35, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt499(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 21, 6, 7, 8, 31, 0, 17, 18, 3, 4, 3, 4, 5, 22, 23, 24, 47, 48, 33, 34, 51, 52, 51, 52, 53, 38, 39, 40, 63, 32, 49, 50, 35, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt500(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 21, 6, 7, 8, 15, 16, 1, 18, 3, 4, 3, 4, 5, 22, 23, 24, 63, 32, 49, 34, 51, 52, 51, 52, 53, 38, 39, 40, 47, 48, 33, 50, 35, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt501(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 21, 6, 7, 8, 31, 16, 1, 18, 3, 4, 3, 4, 5, 22, 23, 24, 47, 32, 49, 34, 51, 52, 51, 52, 53, 38, 39, 40, 63, 48, 33, 50, 35, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt502(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 21, 6, 7, 8, 15, 0, 1, 18, 3, 4, 3, 4, 5, 22, 23, 24, 63, 48, 49, 34, 51, 52, 51, 52, 53, 38, 39, 40, 47, 32, 33, 50, 35, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt503(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 21, 6, 7, 8, 31, 0, 1, 18, 3, 4, 3, 4, 5, 22, 23, 24, 47, 48, 49, 34, 51, 52, 51, 52, 53, 38, 39, 40, 63, 32, 33, 50, 35, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt504(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 21, 6, 7, 8, 15, 16, 17, 2, 3, 4, 3, 4, 5, 22, 23, 24, 63, 32, 33, 50, 51, 52, 51, 52, 53, 38, 39, 40, 47, 48, 49, 34, 35, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt505(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 21, 6, 7, 8, 31, 16, 17, 2, 3, 4, 3, 4, 5, 22, 23, 24, 47, 32, 33, 50, 51, 52, 51, 52, 53, 38, 39, 40, 63, 48, 49, 34, 35, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt506(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 21, 6, 7, 8, 15, 0, 17, 2, 3, 4, 3, 4, 5, 22, 23, 24, 63, 48, 33, 50, 51, 52, 51, 52, 53, 38, 39, 40, 47, 32, 49, 34, 35, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt507(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 21, 6, 7, 8, 31, 0, 17, 2, 3, 4, 3, 4, 5, 22, 23, 24, 47, 48, 33, 50, 51, 52, 51, 52, 53, 38, 39, 40, 63, 32, 49, 34, 35, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt508(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 21, 6, 7, 8, 15, 16, 1, 2, 3, 4, 3, 4, 5, 22, 23, 24, 63, 32, 49, 50, 51, 52, 51, 52, 53, 38, 39, 40, 47, 48, 33, 34, 35, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt509(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 21, 6, 7, 8, 31, 16, 1, 2, 3, 4, 3, 4, 5, 22, 23, 24, 47, 32, 49, 50, 51, 52, 51, 52, 53, 38, 39, 40, 63, 48, 33, 34, 35, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt510(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 21, 6, 7, 8, 15, 0, 1, 2, 3, 4, 3, 4, 5, 22, 23, 24, 63, 48, 49, 50, 51, 52, 51, 52, 53, 38, 39, 40, 47, 32, 33, 34, 35, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt511(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 21, 6, 7, 8, 31, 0, 1, 2, 3, 4, 3, 4, 5, 22, 23, 24, 47, 48, 49, 50, 51, 52, 51, 52, 53, 38, 39, 40, 63, 32, 33, 34, 35, 36, 35, 36, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}


