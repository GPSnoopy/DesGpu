#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt1792(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 4, 21, 22, 23, 8, 15, 16, 17, 18, 19, 20, 19, 20, 5, 6, 7, 24, 63, 32, 33, 34, 35, 36, 35, 36, 53, 54, 55, 40, 47, 48, 49, 50, 51, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1793(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 4, 21, 22, 23, 8, 31, 16, 17, 18, 19, 20, 19, 20, 5, 6, 7, 24, 47, 32, 33, 34, 35, 36, 35, 36, 53, 54, 55, 40, 63, 48, 49, 50, 51, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1794(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 4, 21, 22, 23, 8, 15, 0, 17, 18, 19, 20, 19, 20, 5, 6, 7, 24, 63, 48, 33, 34, 35, 36, 35, 36, 53, 54, 55, 40, 47, 32, 49, 50, 51, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1795(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 4, 21, 22, 23, 8, 31, 0, 17, 18, 19, 20, 19, 20, 5, 6, 7, 24, 47, 48, 33, 34, 35, 36, 35, 36, 53, 54, 55, 40, 63, 32, 49, 50, 51, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1796(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 4, 21, 22, 23, 8, 15, 16, 1, 18, 19, 20, 19, 20, 5, 6, 7, 24, 63, 32, 49, 34, 35, 36, 35, 36, 53, 54, 55, 40, 47, 48, 33, 50, 51, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1797(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 4, 21, 22, 23, 8, 31, 16, 1, 18, 19, 20, 19, 20, 5, 6, 7, 24, 47, 32, 49, 34, 35, 36, 35, 36, 53, 54, 55, 40, 63, 48, 33, 50, 51, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1798(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 4, 21, 22, 23, 8, 15, 0, 1, 18, 19, 20, 19, 20, 5, 6, 7, 24, 63, 48, 49, 34, 35, 36, 35, 36, 53, 54, 55, 40, 47, 32, 33, 50, 51, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1799(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 4, 21, 22, 23, 8, 31, 0, 1, 18, 19, 20, 19, 20, 5, 6, 7, 24, 47, 48, 49, 34, 35, 36, 35, 36, 53, 54, 55, 40, 63, 32, 33, 50, 51, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1800(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 4, 21, 22, 23, 8, 15, 16, 17, 2, 19, 20, 19, 20, 5, 6, 7, 24, 63, 32, 33, 50, 35, 36, 35, 36, 53, 54, 55, 40, 47, 48, 49, 34, 51, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1801(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 4, 21, 22, 23, 8, 31, 16, 17, 2, 19, 20, 19, 20, 5, 6, 7, 24, 47, 32, 33, 50, 35, 36, 35, 36, 53, 54, 55, 40, 63, 48, 49, 34, 51, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1802(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 4, 21, 22, 23, 8, 15, 0, 17, 2, 19, 20, 19, 20, 5, 6, 7, 24, 63, 48, 33, 50, 35, 36, 35, 36, 53, 54, 55, 40, 47, 32, 49, 34, 51, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1803(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 4, 21, 22, 23, 8, 31, 0, 17, 2, 19, 20, 19, 20, 5, 6, 7, 24, 47, 48, 33, 50, 35, 36, 35, 36, 53, 54, 55, 40, 63, 32, 49, 34, 51, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1804(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 4, 21, 22, 23, 8, 15, 16, 1, 2, 19, 20, 19, 20, 5, 6, 7, 24, 63, 32, 49, 50, 35, 36, 35, 36, 53, 54, 55, 40, 47, 48, 33, 34, 51, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1805(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 4, 21, 22, 23, 8, 31, 16, 1, 2, 19, 20, 19, 20, 5, 6, 7, 24, 47, 32, 49, 50, 35, 36, 35, 36, 53, 54, 55, 40, 63, 48, 33, 34, 51, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1806(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 4, 21, 22, 23, 8, 15, 0, 1, 2, 19, 20, 19, 20, 5, 6, 7, 24, 63, 48, 49, 50, 35, 36, 35, 36, 53, 54, 55, 40, 47, 32, 33, 34, 51, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1807(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 4, 21, 22, 23, 8, 31, 0, 1, 2, 19, 20, 19, 20, 5, 6, 7, 24, 47, 48, 49, 50, 35, 36, 35, 36, 53, 54, 55, 40, 63, 32, 33, 34, 51, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1808(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 4, 21, 22, 23, 8, 15, 16, 17, 18, 3, 20, 19, 20, 5, 6, 7, 24, 63, 32, 33, 34, 51, 36, 35, 36, 53, 54, 55, 40, 47, 48, 49, 50, 35, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1809(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 4, 21, 22, 23, 8, 31, 16, 17, 18, 3, 20, 19, 20, 5, 6, 7, 24, 47, 32, 33, 34, 51, 36, 35, 36, 53, 54, 55, 40, 63, 48, 49, 50, 35, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1810(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 4, 21, 22, 23, 8, 15, 0, 17, 18, 3, 20, 19, 20, 5, 6, 7, 24, 63, 48, 33, 34, 51, 36, 35, 36, 53, 54, 55, 40, 47, 32, 49, 50, 35, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1811(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 4, 21, 22, 23, 8, 31, 0, 17, 18, 3, 20, 19, 20, 5, 6, 7, 24, 47, 48, 33, 34, 51, 36, 35, 36, 53, 54, 55, 40, 63, 32, 49, 50, 35, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1812(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 4, 21, 22, 23, 8, 15, 16, 1, 18, 3, 20, 19, 20, 5, 6, 7, 24, 63, 32, 49, 34, 51, 36, 35, 36, 53, 54, 55, 40, 47, 48, 33, 50, 35, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1813(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 4, 21, 22, 23, 8, 31, 16, 1, 18, 3, 20, 19, 20, 5, 6, 7, 24, 47, 32, 49, 34, 51, 36, 35, 36, 53, 54, 55, 40, 63, 48, 33, 50, 35, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1814(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 4, 21, 22, 23, 8, 15, 0, 1, 18, 3, 20, 19, 20, 5, 6, 7, 24, 63, 48, 49, 34, 51, 36, 35, 36, 53, 54, 55, 40, 47, 32, 33, 50, 35, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1815(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 4, 21, 22, 23, 8, 31, 0, 1, 18, 3, 20, 19, 20, 5, 6, 7, 24, 47, 48, 49, 34, 51, 36, 35, 36, 53, 54, 55, 40, 63, 32, 33, 50, 35, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1816(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 4, 21, 22, 23, 8, 15, 16, 17, 2, 3, 20, 19, 20, 5, 6, 7, 24, 63, 32, 33, 50, 51, 36, 35, 36, 53, 54, 55, 40, 47, 48, 49, 34, 35, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1817(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 4, 21, 22, 23, 8, 31, 16, 17, 2, 3, 20, 19, 20, 5, 6, 7, 24, 47, 32, 33, 50, 51, 36, 35, 36, 53, 54, 55, 40, 63, 48, 49, 34, 35, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1818(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 4, 21, 22, 23, 8, 15, 0, 17, 2, 3, 20, 19, 20, 5, 6, 7, 24, 63, 48, 33, 50, 51, 36, 35, 36, 53, 54, 55, 40, 47, 32, 49, 34, 35, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1819(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 4, 21, 22, 23, 8, 31, 0, 17, 2, 3, 20, 19, 20, 5, 6, 7, 24, 47, 48, 33, 50, 51, 36, 35, 36, 53, 54, 55, 40, 63, 32, 49, 34, 35, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1820(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 4, 21, 22, 23, 8, 15, 16, 1, 2, 3, 20, 19, 20, 5, 6, 7, 24, 63, 32, 49, 50, 51, 36, 35, 36, 53, 54, 55, 40, 47, 48, 33, 34, 35, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1821(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 4, 21, 22, 23, 8, 31, 16, 1, 2, 3, 20, 19, 20, 5, 6, 7, 24, 47, 32, 49, 50, 51, 36, 35, 36, 53, 54, 55, 40, 63, 48, 33, 34, 35, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1822(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 4, 21, 22, 23, 8, 15, 0, 1, 2, 3, 20, 19, 20, 5, 6, 7, 24, 63, 48, 49, 50, 51, 36, 35, 36, 53, 54, 55, 40, 47, 32, 33, 34, 35, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1823(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 4, 21, 22, 23, 8, 31, 0, 1, 2, 3, 20, 19, 20, 5, 6, 7, 24, 47, 48, 49, 50, 51, 36, 35, 36, 53, 54, 55, 40, 63, 32, 33, 34, 35, 52, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1824(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 4, 21, 22, 23, 8, 15, 16, 17, 18, 19, 4, 19, 20, 5, 6, 7, 24, 63, 32, 33, 34, 35, 52, 35, 36, 53, 54, 55, 40, 47, 48, 49, 50, 51, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1825(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 4, 21, 22, 23, 8, 31, 16, 17, 18, 19, 4, 19, 20, 5, 6, 7, 24, 47, 32, 33, 34, 35, 52, 35, 36, 53, 54, 55, 40, 63, 48, 49, 50, 51, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1826(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 4, 21, 22, 23, 8, 15, 0, 17, 18, 19, 4, 19, 20, 5, 6, 7, 24, 63, 48, 33, 34, 35, 52, 35, 36, 53, 54, 55, 40, 47, 32, 49, 50, 51, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1827(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 4, 21, 22, 23, 8, 31, 0, 17, 18, 19, 4, 19, 20, 5, 6, 7, 24, 47, 48, 33, 34, 35, 52, 35, 36, 53, 54, 55, 40, 63, 32, 49, 50, 51, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1828(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 4, 21, 22, 23, 8, 15, 16, 1, 18, 19, 4, 19, 20, 5, 6, 7, 24, 63, 32, 49, 34, 35, 52, 35, 36, 53, 54, 55, 40, 47, 48, 33, 50, 51, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1829(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 4, 21, 22, 23, 8, 31, 16, 1, 18, 19, 4, 19, 20, 5, 6, 7, 24, 47, 32, 49, 34, 35, 52, 35, 36, 53, 54, 55, 40, 63, 48, 33, 50, 51, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1830(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 4, 21, 22, 23, 8, 15, 0, 1, 18, 19, 4, 19, 20, 5, 6, 7, 24, 63, 48, 49, 34, 35, 52, 35, 36, 53, 54, 55, 40, 47, 32, 33, 50, 51, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1831(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 4, 21, 22, 23, 8, 31, 0, 1, 18, 19, 4, 19, 20, 5, 6, 7, 24, 47, 48, 49, 34, 35, 52, 35, 36, 53, 54, 55, 40, 63, 32, 33, 50, 51, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1832(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 4, 21, 22, 23, 8, 15, 16, 17, 2, 19, 4, 19, 20, 5, 6, 7, 24, 63, 32, 33, 50, 35, 52, 35, 36, 53, 54, 55, 40, 47, 48, 49, 34, 51, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1833(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 4, 21, 22, 23, 8, 31, 16, 17, 2, 19, 4, 19, 20, 5, 6, 7, 24, 47, 32, 33, 50, 35, 52, 35, 36, 53, 54, 55, 40, 63, 48, 49, 34, 51, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1834(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 4, 21, 22, 23, 8, 15, 0, 17, 2, 19, 4, 19, 20, 5, 6, 7, 24, 63, 48, 33, 50, 35, 52, 35, 36, 53, 54, 55, 40, 47, 32, 49, 34, 51, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1835(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 4, 21, 22, 23, 8, 31, 0, 17, 2, 19, 4, 19, 20, 5, 6, 7, 24, 47, 48, 33, 50, 35, 52, 35, 36, 53, 54, 55, 40, 63, 32, 49, 34, 51, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1836(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 4, 21, 22, 23, 8, 15, 16, 1, 2, 19, 4, 19, 20, 5, 6, 7, 24, 63, 32, 49, 50, 35, 52, 35, 36, 53, 54, 55, 40, 47, 48, 33, 34, 51, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1837(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 4, 21, 22, 23, 8, 31, 16, 1, 2, 19, 4, 19, 20, 5, 6, 7, 24, 47, 32, 49, 50, 35, 52, 35, 36, 53, 54, 55, 40, 63, 48, 33, 34, 51, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1838(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 4, 21, 22, 23, 8, 15, 0, 1, 2, 19, 4, 19, 20, 5, 6, 7, 24, 63, 48, 49, 50, 35, 52, 35, 36, 53, 54, 55, 40, 47, 32, 33, 34, 51, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1839(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 4, 21, 22, 23, 8, 31, 0, 1, 2, 19, 4, 19, 20, 5, 6, 7, 24, 47, 48, 49, 50, 35, 52, 35, 36, 53, 54, 55, 40, 63, 32, 33, 34, 51, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1840(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 4, 21, 22, 23, 8, 15, 16, 17, 18, 3, 4, 19, 20, 5, 6, 7, 24, 63, 32, 33, 34, 51, 52, 35, 36, 53, 54, 55, 40, 47, 48, 49, 50, 35, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1841(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 4, 21, 22, 23, 8, 31, 16, 17, 18, 3, 4, 19, 20, 5, 6, 7, 24, 47, 32, 33, 34, 51, 52, 35, 36, 53, 54, 55, 40, 63, 48, 49, 50, 35, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1842(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 4, 21, 22, 23, 8, 15, 0, 17, 18, 3, 4, 19, 20, 5, 6, 7, 24, 63, 48, 33, 34, 51, 52, 35, 36, 53, 54, 55, 40, 47, 32, 49, 50, 35, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1843(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 4, 21, 22, 23, 8, 31, 0, 17, 18, 3, 4, 19, 20, 5, 6, 7, 24, 47, 48, 33, 34, 51, 52, 35, 36, 53, 54, 55, 40, 63, 32, 49, 50, 35, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1844(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 4, 21, 22, 23, 8, 15, 16, 1, 18, 3, 4, 19, 20, 5, 6, 7, 24, 63, 32, 49, 34, 51, 52, 35, 36, 53, 54, 55, 40, 47, 48, 33, 50, 35, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1845(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 4, 21, 22, 23, 8, 31, 16, 1, 18, 3, 4, 19, 20, 5, 6, 7, 24, 47, 32, 49, 34, 51, 52, 35, 36, 53, 54, 55, 40, 63, 48, 33, 50, 35, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1846(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 4, 21, 22, 23, 8, 15, 0, 1, 18, 3, 4, 19, 20, 5, 6, 7, 24, 63, 48, 49, 34, 51, 52, 35, 36, 53, 54, 55, 40, 47, 32, 33, 50, 35, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1847(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 4, 21, 22, 23, 8, 31, 0, 1, 18, 3, 4, 19, 20, 5, 6, 7, 24, 47, 48, 49, 34, 51, 52, 35, 36, 53, 54, 55, 40, 63, 32, 33, 50, 35, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1848(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 4, 21, 22, 23, 8, 15, 16, 17, 2, 3, 4, 19, 20, 5, 6, 7, 24, 63, 32, 33, 50, 51, 52, 35, 36, 53, 54, 55, 40, 47, 48, 49, 34, 35, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1849(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 4, 21, 22, 23, 8, 31, 16, 17, 2, 3, 4, 19, 20, 5, 6, 7, 24, 47, 32, 33, 50, 51, 52, 35, 36, 53, 54, 55, 40, 63, 48, 49, 34, 35, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1850(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 4, 21, 22, 23, 8, 15, 0, 17, 2, 3, 4, 19, 20, 5, 6, 7, 24, 63, 48, 33, 50, 51, 52, 35, 36, 53, 54, 55, 40, 47, 32, 49, 34, 35, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1851(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 4, 21, 22, 23, 8, 31, 0, 17, 2, 3, 4, 19, 20, 5, 6, 7, 24, 47, 48, 33, 50, 51, 52, 35, 36, 53, 54, 55, 40, 63, 32, 49, 34, 35, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1852(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 4, 21, 22, 23, 8, 15, 16, 1, 2, 3, 4, 19, 20, 5, 6, 7, 24, 63, 32, 49, 50, 51, 52, 35, 36, 53, 54, 55, 40, 47, 48, 33, 34, 35, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1853(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 4, 21, 22, 23, 8, 31, 16, 1, 2, 3, 4, 19, 20, 5, 6, 7, 24, 47, 32, 49, 50, 51, 52, 35, 36, 53, 54, 55, 40, 63, 48, 33, 34, 35, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1854(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 4, 21, 22, 23, 8, 15, 0, 1, 2, 3, 4, 19, 20, 5, 6, 7, 24, 63, 48, 49, 50, 51, 52, 35, 36, 53, 54, 55, 40, 47, 32, 33, 34, 35, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1855(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 4, 21, 22, 23, 8, 31, 0, 1, 2, 3, 4, 19, 20, 5, 6, 7, 24, 47, 48, 49, 50, 51, 52, 35, 36, 53, 54, 55, 40, 63, 32, 33, 34, 35, 36, 51, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1856(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 4, 21, 22, 23, 8, 15, 16, 17, 18, 19, 20, 3, 20, 5, 6, 7, 24, 63, 32, 33, 34, 35, 36, 51, 36, 53, 54, 55, 40, 47, 48, 49, 50, 51, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1857(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 4, 21, 22, 23, 8, 31, 16, 17, 18, 19, 20, 3, 20, 5, 6, 7, 24, 47, 32, 33, 34, 35, 36, 51, 36, 53, 54, 55, 40, 63, 48, 49, 50, 51, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1858(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 4, 21, 22, 23, 8, 15, 0, 17, 18, 19, 20, 3, 20, 5, 6, 7, 24, 63, 48, 33, 34, 35, 36, 51, 36, 53, 54, 55, 40, 47, 32, 49, 50, 51, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1859(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 4, 21, 22, 23, 8, 31, 0, 17, 18, 19, 20, 3, 20, 5, 6, 7, 24, 47, 48, 33, 34, 35, 36, 51, 36, 53, 54, 55, 40, 63, 32, 49, 50, 51, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1860(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 4, 21, 22, 23, 8, 15, 16, 1, 18, 19, 20, 3, 20, 5, 6, 7, 24, 63, 32, 49, 34, 35, 36, 51, 36, 53, 54, 55, 40, 47, 48, 33, 50, 51, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1861(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 4, 21, 22, 23, 8, 31, 16, 1, 18, 19, 20, 3, 20, 5, 6, 7, 24, 47, 32, 49, 34, 35, 36, 51, 36, 53, 54, 55, 40, 63, 48, 33, 50, 51, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1862(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 4, 21, 22, 23, 8, 15, 0, 1, 18, 19, 20, 3, 20, 5, 6, 7, 24, 63, 48, 49, 34, 35, 36, 51, 36, 53, 54, 55, 40, 47, 32, 33, 50, 51, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1863(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 4, 21, 22, 23, 8, 31, 0, 1, 18, 19, 20, 3, 20, 5, 6, 7, 24, 47, 48, 49, 34, 35, 36, 51, 36, 53, 54, 55, 40, 63, 32, 33, 50, 51, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1864(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 4, 21, 22, 23, 8, 15, 16, 17, 2, 19, 20, 3, 20, 5, 6, 7, 24, 63, 32, 33, 50, 35, 36, 51, 36, 53, 54, 55, 40, 47, 48, 49, 34, 51, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1865(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 4, 21, 22, 23, 8, 31, 16, 17, 2, 19, 20, 3, 20, 5, 6, 7, 24, 47, 32, 33, 50, 35, 36, 51, 36, 53, 54, 55, 40, 63, 48, 49, 34, 51, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1866(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 4, 21, 22, 23, 8, 15, 0, 17, 2, 19, 20, 3, 20, 5, 6, 7, 24, 63, 48, 33, 50, 35, 36, 51, 36, 53, 54, 55, 40, 47, 32, 49, 34, 51, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1867(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 4, 21, 22, 23, 8, 31, 0, 17, 2, 19, 20, 3, 20, 5, 6, 7, 24, 47, 48, 33, 50, 35, 36, 51, 36, 53, 54, 55, 40, 63, 32, 49, 34, 51, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1868(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 4, 21, 22, 23, 8, 15, 16, 1, 2, 19, 20, 3, 20, 5, 6, 7, 24, 63, 32, 49, 50, 35, 36, 51, 36, 53, 54, 55, 40, 47, 48, 33, 34, 51, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1869(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 4, 21, 22, 23, 8, 31, 16, 1, 2, 19, 20, 3, 20, 5, 6, 7, 24, 47, 32, 49, 50, 35, 36, 51, 36, 53, 54, 55, 40, 63, 48, 33, 34, 51, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1870(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 4, 21, 22, 23, 8, 15, 0, 1, 2, 19, 20, 3, 20, 5, 6, 7, 24, 63, 48, 49, 50, 35, 36, 51, 36, 53, 54, 55, 40, 47, 32, 33, 34, 51, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1871(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 4, 21, 22, 23, 8, 31, 0, 1, 2, 19, 20, 3, 20, 5, 6, 7, 24, 47, 48, 49, 50, 35, 36, 51, 36, 53, 54, 55, 40, 63, 32, 33, 34, 51, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1872(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 4, 21, 22, 23, 8, 15, 16, 17, 18, 3, 20, 3, 20, 5, 6, 7, 24, 63, 32, 33, 34, 51, 36, 51, 36, 53, 54, 55, 40, 47, 48, 49, 50, 35, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1873(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 4, 21, 22, 23, 8, 31, 16, 17, 18, 3, 20, 3, 20, 5, 6, 7, 24, 47, 32, 33, 34, 51, 36, 51, 36, 53, 54, 55, 40, 63, 48, 49, 50, 35, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1874(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 4, 21, 22, 23, 8, 15, 0, 17, 18, 3, 20, 3, 20, 5, 6, 7, 24, 63, 48, 33, 34, 51, 36, 51, 36, 53, 54, 55, 40, 47, 32, 49, 50, 35, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1875(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 4, 21, 22, 23, 8, 31, 0, 17, 18, 3, 20, 3, 20, 5, 6, 7, 24, 47, 48, 33, 34, 51, 36, 51, 36, 53, 54, 55, 40, 63, 32, 49, 50, 35, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1876(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 4, 21, 22, 23, 8, 15, 16, 1, 18, 3, 20, 3, 20, 5, 6, 7, 24, 63, 32, 49, 34, 51, 36, 51, 36, 53, 54, 55, 40, 47, 48, 33, 50, 35, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1877(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 4, 21, 22, 23, 8, 31, 16, 1, 18, 3, 20, 3, 20, 5, 6, 7, 24, 47, 32, 49, 34, 51, 36, 51, 36, 53, 54, 55, 40, 63, 48, 33, 50, 35, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1878(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 4, 21, 22, 23, 8, 15, 0, 1, 18, 3, 20, 3, 20, 5, 6, 7, 24, 63, 48, 49, 34, 51, 36, 51, 36, 53, 54, 55, 40, 47, 32, 33, 50, 35, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1879(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 4, 21, 22, 23, 8, 31, 0, 1, 18, 3, 20, 3, 20, 5, 6, 7, 24, 47, 48, 49, 34, 51, 36, 51, 36, 53, 54, 55, 40, 63, 32, 33, 50, 35, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1880(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 4, 21, 22, 23, 8, 15, 16, 17, 2, 3, 20, 3, 20, 5, 6, 7, 24, 63, 32, 33, 50, 51, 36, 51, 36, 53, 54, 55, 40, 47, 48, 49, 34, 35, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1881(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 4, 21, 22, 23, 8, 31, 16, 17, 2, 3, 20, 3, 20, 5, 6, 7, 24, 47, 32, 33, 50, 51, 36, 51, 36, 53, 54, 55, 40, 63, 48, 49, 34, 35, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1882(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 4, 21, 22, 23, 8, 15, 0, 17, 2, 3, 20, 3, 20, 5, 6, 7, 24, 63, 48, 33, 50, 51, 36, 51, 36, 53, 54, 55, 40, 47, 32, 49, 34, 35, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1883(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 4, 21, 22, 23, 8, 31, 0, 17, 2, 3, 20, 3, 20, 5, 6, 7, 24, 47, 48, 33, 50, 51, 36, 51, 36, 53, 54, 55, 40, 63, 32, 49, 34, 35, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1884(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 4, 21, 22, 23, 8, 15, 16, 1, 2, 3, 20, 3, 20, 5, 6, 7, 24, 63, 32, 49, 50, 51, 36, 51, 36, 53, 54, 55, 40, 47, 48, 33, 34, 35, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1885(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 4, 21, 22, 23, 8, 31, 16, 1, 2, 3, 20, 3, 20, 5, 6, 7, 24, 47, 32, 49, 50, 51, 36, 51, 36, 53, 54, 55, 40, 63, 48, 33, 34, 35, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1886(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 4, 21, 22, 23, 8, 15, 0, 1, 2, 3, 20, 3, 20, 5, 6, 7, 24, 63, 48, 49, 50, 51, 36, 51, 36, 53, 54, 55, 40, 47, 32, 33, 34, 35, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1887(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 4, 21, 22, 23, 8, 31, 0, 1, 2, 3, 20, 3, 20, 5, 6, 7, 24, 47, 48, 49, 50, 51, 36, 51, 36, 53, 54, 55, 40, 63, 32, 33, 34, 35, 52, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1888(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 4, 21, 22, 23, 8, 15, 16, 17, 18, 19, 4, 3, 20, 5, 6, 7, 24, 63, 32, 33, 34, 35, 52, 51, 36, 53, 54, 55, 40, 47, 48, 49, 50, 51, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1889(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 4, 21, 22, 23, 8, 31, 16, 17, 18, 19, 4, 3, 20, 5, 6, 7, 24, 47, 32, 33, 34, 35, 52, 51, 36, 53, 54, 55, 40, 63, 48, 49, 50, 51, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1890(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 4, 21, 22, 23, 8, 15, 0, 17, 18, 19, 4, 3, 20, 5, 6, 7, 24, 63, 48, 33, 34, 35, 52, 51, 36, 53, 54, 55, 40, 47, 32, 49, 50, 51, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1891(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 4, 21, 22, 23, 8, 31, 0, 17, 18, 19, 4, 3, 20, 5, 6, 7, 24, 47, 48, 33, 34, 35, 52, 51, 36, 53, 54, 55, 40, 63, 32, 49, 50, 51, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1892(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 4, 21, 22, 23, 8, 15, 16, 1, 18, 19, 4, 3, 20, 5, 6, 7, 24, 63, 32, 49, 34, 35, 52, 51, 36, 53, 54, 55, 40, 47, 48, 33, 50, 51, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1893(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 4, 21, 22, 23, 8, 31, 16, 1, 18, 19, 4, 3, 20, 5, 6, 7, 24, 47, 32, 49, 34, 35, 52, 51, 36, 53, 54, 55, 40, 63, 48, 33, 50, 51, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1894(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 4, 21, 22, 23, 8, 15, 0, 1, 18, 19, 4, 3, 20, 5, 6, 7, 24, 63, 48, 49, 34, 35, 52, 51, 36, 53, 54, 55, 40, 47, 32, 33, 50, 51, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1895(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 4, 21, 22, 23, 8, 31, 0, 1, 18, 19, 4, 3, 20, 5, 6, 7, 24, 47, 48, 49, 34, 35, 52, 51, 36, 53, 54, 55, 40, 63, 32, 33, 50, 51, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1896(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 4, 21, 22, 23, 8, 15, 16, 17, 2, 19, 4, 3, 20, 5, 6, 7, 24, 63, 32, 33, 50, 35, 52, 51, 36, 53, 54, 55, 40, 47, 48, 49, 34, 51, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1897(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 4, 21, 22, 23, 8, 31, 16, 17, 2, 19, 4, 3, 20, 5, 6, 7, 24, 47, 32, 33, 50, 35, 52, 51, 36, 53, 54, 55, 40, 63, 48, 49, 34, 51, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1898(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 4, 21, 22, 23, 8, 15, 0, 17, 2, 19, 4, 3, 20, 5, 6, 7, 24, 63, 48, 33, 50, 35, 52, 51, 36, 53, 54, 55, 40, 47, 32, 49, 34, 51, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1899(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 4, 21, 22, 23, 8, 31, 0, 17, 2, 19, 4, 3, 20, 5, 6, 7, 24, 47, 48, 33, 50, 35, 52, 51, 36, 53, 54, 55, 40, 63, 32, 49, 34, 51, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1900(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 4, 21, 22, 23, 8, 15, 16, 1, 2, 19, 4, 3, 20, 5, 6, 7, 24, 63, 32, 49, 50, 35, 52, 51, 36, 53, 54, 55, 40, 47, 48, 33, 34, 51, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1901(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 4, 21, 22, 23, 8, 31, 16, 1, 2, 19, 4, 3, 20, 5, 6, 7, 24, 47, 32, 49, 50, 35, 52, 51, 36, 53, 54, 55, 40, 63, 48, 33, 34, 51, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1902(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 4, 21, 22, 23, 8, 15, 0, 1, 2, 19, 4, 3, 20, 5, 6, 7, 24, 63, 48, 49, 50, 35, 52, 51, 36, 53, 54, 55, 40, 47, 32, 33, 34, 51, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1903(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 4, 21, 22, 23, 8, 31, 0, 1, 2, 19, 4, 3, 20, 5, 6, 7, 24, 47, 48, 49, 50, 35, 52, 51, 36, 53, 54, 55, 40, 63, 32, 33, 34, 51, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1904(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 4, 21, 22, 23, 8, 15, 16, 17, 18, 3, 4, 3, 20, 5, 6, 7, 24, 63, 32, 33, 34, 51, 52, 51, 36, 53, 54, 55, 40, 47, 48, 49, 50, 35, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1905(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 4, 21, 22, 23, 8, 31, 16, 17, 18, 3, 4, 3, 20, 5, 6, 7, 24, 47, 32, 33, 34, 51, 52, 51, 36, 53, 54, 55, 40, 63, 48, 49, 50, 35, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1906(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 4, 21, 22, 23, 8, 15, 0, 17, 18, 3, 4, 3, 20, 5, 6, 7, 24, 63, 48, 33, 34, 51, 52, 51, 36, 53, 54, 55, 40, 47, 32, 49, 50, 35, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1907(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 4, 21, 22, 23, 8, 31, 0, 17, 18, 3, 4, 3, 20, 5, 6, 7, 24, 47, 48, 33, 34, 51, 52, 51, 36, 53, 54, 55, 40, 63, 32, 49, 50, 35, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1908(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 4, 21, 22, 23, 8, 15, 16, 1, 18, 3, 4, 3, 20, 5, 6, 7, 24, 63, 32, 49, 34, 51, 52, 51, 36, 53, 54, 55, 40, 47, 48, 33, 50, 35, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1909(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 4, 21, 22, 23, 8, 31, 16, 1, 18, 3, 4, 3, 20, 5, 6, 7, 24, 47, 32, 49, 34, 51, 52, 51, 36, 53, 54, 55, 40, 63, 48, 33, 50, 35, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1910(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 4, 21, 22, 23, 8, 15, 0, 1, 18, 3, 4, 3, 20, 5, 6, 7, 24, 63, 48, 49, 34, 51, 52, 51, 36, 53, 54, 55, 40, 47, 32, 33, 50, 35, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1911(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 4, 21, 22, 23, 8, 31, 0, 1, 18, 3, 4, 3, 20, 5, 6, 7, 24, 47, 48, 49, 34, 51, 52, 51, 36, 53, 54, 55, 40, 63, 32, 33, 50, 35, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1912(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 4, 21, 22, 23, 8, 15, 16, 17, 2, 3, 4, 3, 20, 5, 6, 7, 24, 63, 32, 33, 50, 51, 52, 51, 36, 53, 54, 55, 40, 47, 48, 49, 34, 35, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1913(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 4, 21, 22, 23, 8, 31, 16, 17, 2, 3, 4, 3, 20, 5, 6, 7, 24, 47, 32, 33, 50, 51, 52, 51, 36, 53, 54, 55, 40, 63, 48, 49, 34, 35, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1914(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 4, 21, 22, 23, 8, 15, 0, 17, 2, 3, 4, 3, 20, 5, 6, 7, 24, 63, 48, 33, 50, 51, 52, 51, 36, 53, 54, 55, 40, 47, 32, 49, 34, 35, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1915(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 4, 21, 22, 23, 8, 31, 0, 17, 2, 3, 4, 3, 20, 5, 6, 7, 24, 47, 48, 33, 50, 51, 52, 51, 36, 53, 54, 55, 40, 63, 32, 49, 34, 35, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1916(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 4, 21, 22, 23, 8, 15, 16, 1, 2, 3, 4, 3, 20, 5, 6, 7, 24, 63, 32, 49, 50, 51, 52, 51, 36, 53, 54, 55, 40, 47, 48, 33, 34, 35, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1917(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 4, 21, 22, 23, 8, 31, 16, 1, 2, 3, 4, 3, 20, 5, 6, 7, 24, 47, 32, 49, 50, 51, 52, 51, 36, 53, 54, 55, 40, 63, 48, 33, 34, 35, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1918(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 4, 21, 22, 23, 8, 15, 0, 1, 2, 3, 4, 3, 20, 5, 6, 7, 24, 63, 48, 49, 50, 51, 52, 51, 36, 53, 54, 55, 40, 47, 32, 33, 34, 35, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1919(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 4, 21, 22, 23, 8, 31, 0, 1, 2, 3, 4, 3, 20, 5, 6, 7, 24, 47, 48, 49, 50, 51, 52, 51, 36, 53, 54, 55, 40, 63, 32, 33, 34, 35, 36, 35, 52, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}


