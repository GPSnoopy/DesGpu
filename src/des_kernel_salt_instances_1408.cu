#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt1408(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 21, 6, 23, 8, 15, 16, 17, 18, 19, 20, 19, 4, 5, 22, 7, 24, 63, 32, 33, 34, 35, 36, 35, 52, 53, 38, 55, 40, 47, 48, 49, 50, 51, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1409(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 21, 6, 23, 8, 31, 16, 17, 18, 19, 20, 19, 4, 5, 22, 7, 24, 47, 32, 33, 34, 35, 36, 35, 52, 53, 38, 55, 40, 63, 48, 49, 50, 51, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1410(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 21, 6, 23, 8, 15, 0, 17, 18, 19, 20, 19, 4, 5, 22, 7, 24, 63, 48, 33, 34, 35, 36, 35, 52, 53, 38, 55, 40, 47, 32, 49, 50, 51, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1411(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 21, 6, 23, 8, 31, 0, 17, 18, 19, 20, 19, 4, 5, 22, 7, 24, 47, 48, 33, 34, 35, 36, 35, 52, 53, 38, 55, 40, 63, 32, 49, 50, 51, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1412(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 21, 6, 23, 8, 15, 16, 1, 18, 19, 20, 19, 4, 5, 22, 7, 24, 63, 32, 49, 34, 35, 36, 35, 52, 53, 38, 55, 40, 47, 48, 33, 50, 51, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1413(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 21, 6, 23, 8, 31, 16, 1, 18, 19, 20, 19, 4, 5, 22, 7, 24, 47, 32, 49, 34, 35, 36, 35, 52, 53, 38, 55, 40, 63, 48, 33, 50, 51, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1414(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 21, 6, 23, 8, 15, 0, 1, 18, 19, 20, 19, 4, 5, 22, 7, 24, 63, 48, 49, 34, 35, 36, 35, 52, 53, 38, 55, 40, 47, 32, 33, 50, 51, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1415(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 21, 6, 23, 8, 31, 0, 1, 18, 19, 20, 19, 4, 5, 22, 7, 24, 47, 48, 49, 34, 35, 36, 35, 52, 53, 38, 55, 40, 63, 32, 33, 50, 51, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1416(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 21, 6, 23, 8, 15, 16, 17, 2, 19, 20, 19, 4, 5, 22, 7, 24, 63, 32, 33, 50, 35, 36, 35, 52, 53, 38, 55, 40, 47, 48, 49, 34, 51, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1417(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 21, 6, 23, 8, 31, 16, 17, 2, 19, 20, 19, 4, 5, 22, 7, 24, 47, 32, 33, 50, 35, 36, 35, 52, 53, 38, 55, 40, 63, 48, 49, 34, 51, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1418(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 21, 6, 23, 8, 15, 0, 17, 2, 19, 20, 19, 4, 5, 22, 7, 24, 63, 48, 33, 50, 35, 36, 35, 52, 53, 38, 55, 40, 47, 32, 49, 34, 51, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1419(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 21, 6, 23, 8, 31, 0, 17, 2, 19, 20, 19, 4, 5, 22, 7, 24, 47, 48, 33, 50, 35, 36, 35, 52, 53, 38, 55, 40, 63, 32, 49, 34, 51, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1420(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 21, 6, 23, 8, 15, 16, 1, 2, 19, 20, 19, 4, 5, 22, 7, 24, 63, 32, 49, 50, 35, 36, 35, 52, 53, 38, 55, 40, 47, 48, 33, 34, 51, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1421(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 21, 6, 23, 8, 31, 16, 1, 2, 19, 20, 19, 4, 5, 22, 7, 24, 47, 32, 49, 50, 35, 36, 35, 52, 53, 38, 55, 40, 63, 48, 33, 34, 51, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1422(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 21, 6, 23, 8, 15, 0, 1, 2, 19, 20, 19, 4, 5, 22, 7, 24, 63, 48, 49, 50, 35, 36, 35, 52, 53, 38, 55, 40, 47, 32, 33, 34, 51, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1423(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 21, 6, 23, 8, 31, 0, 1, 2, 19, 20, 19, 4, 5, 22, 7, 24, 47, 48, 49, 50, 35, 36, 35, 52, 53, 38, 55, 40, 63, 32, 33, 34, 51, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1424(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 21, 6, 23, 8, 15, 16, 17, 18, 3, 20, 19, 4, 5, 22, 7, 24, 63, 32, 33, 34, 51, 36, 35, 52, 53, 38, 55, 40, 47, 48, 49, 50, 35, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1425(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 21, 6, 23, 8, 31, 16, 17, 18, 3, 20, 19, 4, 5, 22, 7, 24, 47, 32, 33, 34, 51, 36, 35, 52, 53, 38, 55, 40, 63, 48, 49, 50, 35, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1426(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 21, 6, 23, 8, 15, 0, 17, 18, 3, 20, 19, 4, 5, 22, 7, 24, 63, 48, 33, 34, 51, 36, 35, 52, 53, 38, 55, 40, 47, 32, 49, 50, 35, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1427(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 21, 6, 23, 8, 31, 0, 17, 18, 3, 20, 19, 4, 5, 22, 7, 24, 47, 48, 33, 34, 51, 36, 35, 52, 53, 38, 55, 40, 63, 32, 49, 50, 35, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1428(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 21, 6, 23, 8, 15, 16, 1, 18, 3, 20, 19, 4, 5, 22, 7, 24, 63, 32, 49, 34, 51, 36, 35, 52, 53, 38, 55, 40, 47, 48, 33, 50, 35, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1429(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 21, 6, 23, 8, 31, 16, 1, 18, 3, 20, 19, 4, 5, 22, 7, 24, 47, 32, 49, 34, 51, 36, 35, 52, 53, 38, 55, 40, 63, 48, 33, 50, 35, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1430(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 21, 6, 23, 8, 15, 0, 1, 18, 3, 20, 19, 4, 5, 22, 7, 24, 63, 48, 49, 34, 51, 36, 35, 52, 53, 38, 55, 40, 47, 32, 33, 50, 35, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1431(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 21, 6, 23, 8, 31, 0, 1, 18, 3, 20, 19, 4, 5, 22, 7, 24, 47, 48, 49, 34, 51, 36, 35, 52, 53, 38, 55, 40, 63, 32, 33, 50, 35, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1432(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 21, 6, 23, 8, 15, 16, 17, 2, 3, 20, 19, 4, 5, 22, 7, 24, 63, 32, 33, 50, 51, 36, 35, 52, 53, 38, 55, 40, 47, 48, 49, 34, 35, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1433(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 21, 6, 23, 8, 31, 16, 17, 2, 3, 20, 19, 4, 5, 22, 7, 24, 47, 32, 33, 50, 51, 36, 35, 52, 53, 38, 55, 40, 63, 48, 49, 34, 35, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1434(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 21, 6, 23, 8, 15, 0, 17, 2, 3, 20, 19, 4, 5, 22, 7, 24, 63, 48, 33, 50, 51, 36, 35, 52, 53, 38, 55, 40, 47, 32, 49, 34, 35, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1435(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 21, 6, 23, 8, 31, 0, 17, 2, 3, 20, 19, 4, 5, 22, 7, 24, 47, 48, 33, 50, 51, 36, 35, 52, 53, 38, 55, 40, 63, 32, 49, 34, 35, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1436(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 21, 6, 23, 8, 15, 16, 1, 2, 3, 20, 19, 4, 5, 22, 7, 24, 63, 32, 49, 50, 51, 36, 35, 52, 53, 38, 55, 40, 47, 48, 33, 34, 35, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1437(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 21, 6, 23, 8, 31, 16, 1, 2, 3, 20, 19, 4, 5, 22, 7, 24, 47, 32, 49, 50, 51, 36, 35, 52, 53, 38, 55, 40, 63, 48, 33, 34, 35, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1438(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 21, 6, 23, 8, 15, 0, 1, 2, 3, 20, 19, 4, 5, 22, 7, 24, 63, 48, 49, 50, 51, 36, 35, 52, 53, 38, 55, 40, 47, 32, 33, 34, 35, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1439(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 21, 6, 23, 8, 31, 0, 1, 2, 3, 20, 19, 4, 5, 22, 7, 24, 47, 48, 49, 50, 51, 36, 35, 52, 53, 38, 55, 40, 63, 32, 33, 34, 35, 52, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1440(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 21, 6, 23, 8, 15, 16, 17, 18, 19, 4, 19, 4, 5, 22, 7, 24, 63, 32, 33, 34, 35, 52, 35, 52, 53, 38, 55, 40, 47, 48, 49, 50, 51, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1441(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 21, 6, 23, 8, 31, 16, 17, 18, 19, 4, 19, 4, 5, 22, 7, 24, 47, 32, 33, 34, 35, 52, 35, 52, 53, 38, 55, 40, 63, 48, 49, 50, 51, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1442(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 21, 6, 23, 8, 15, 0, 17, 18, 19, 4, 19, 4, 5, 22, 7, 24, 63, 48, 33, 34, 35, 52, 35, 52, 53, 38, 55, 40, 47, 32, 49, 50, 51, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1443(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 21, 6, 23, 8, 31, 0, 17, 18, 19, 4, 19, 4, 5, 22, 7, 24, 47, 48, 33, 34, 35, 52, 35, 52, 53, 38, 55, 40, 63, 32, 49, 50, 51, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1444(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 21, 6, 23, 8, 15, 16, 1, 18, 19, 4, 19, 4, 5, 22, 7, 24, 63, 32, 49, 34, 35, 52, 35, 52, 53, 38, 55, 40, 47, 48, 33, 50, 51, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1445(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 21, 6, 23, 8, 31, 16, 1, 18, 19, 4, 19, 4, 5, 22, 7, 24, 47, 32, 49, 34, 35, 52, 35, 52, 53, 38, 55, 40, 63, 48, 33, 50, 51, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1446(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 21, 6, 23, 8, 15, 0, 1, 18, 19, 4, 19, 4, 5, 22, 7, 24, 63, 48, 49, 34, 35, 52, 35, 52, 53, 38, 55, 40, 47, 32, 33, 50, 51, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1447(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 21, 6, 23, 8, 31, 0, 1, 18, 19, 4, 19, 4, 5, 22, 7, 24, 47, 48, 49, 34, 35, 52, 35, 52, 53, 38, 55, 40, 63, 32, 33, 50, 51, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1448(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 21, 6, 23, 8, 15, 16, 17, 2, 19, 4, 19, 4, 5, 22, 7, 24, 63, 32, 33, 50, 35, 52, 35, 52, 53, 38, 55, 40, 47, 48, 49, 34, 51, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1449(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 21, 6, 23, 8, 31, 16, 17, 2, 19, 4, 19, 4, 5, 22, 7, 24, 47, 32, 33, 50, 35, 52, 35, 52, 53, 38, 55, 40, 63, 48, 49, 34, 51, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1450(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 21, 6, 23, 8, 15, 0, 17, 2, 19, 4, 19, 4, 5, 22, 7, 24, 63, 48, 33, 50, 35, 52, 35, 52, 53, 38, 55, 40, 47, 32, 49, 34, 51, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1451(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 21, 6, 23, 8, 31, 0, 17, 2, 19, 4, 19, 4, 5, 22, 7, 24, 47, 48, 33, 50, 35, 52, 35, 52, 53, 38, 55, 40, 63, 32, 49, 34, 51, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1452(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 21, 6, 23, 8, 15, 16, 1, 2, 19, 4, 19, 4, 5, 22, 7, 24, 63, 32, 49, 50, 35, 52, 35, 52, 53, 38, 55, 40, 47, 48, 33, 34, 51, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1453(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 21, 6, 23, 8, 31, 16, 1, 2, 19, 4, 19, 4, 5, 22, 7, 24, 47, 32, 49, 50, 35, 52, 35, 52, 53, 38, 55, 40, 63, 48, 33, 34, 51, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1454(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 21, 6, 23, 8, 15, 0, 1, 2, 19, 4, 19, 4, 5, 22, 7, 24, 63, 48, 49, 50, 35, 52, 35, 52, 53, 38, 55, 40, 47, 32, 33, 34, 51, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1455(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 21, 6, 23, 8, 31, 0, 1, 2, 19, 4, 19, 4, 5, 22, 7, 24, 47, 48, 49, 50, 35, 52, 35, 52, 53, 38, 55, 40, 63, 32, 33, 34, 51, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1456(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 21, 6, 23, 8, 15, 16, 17, 18, 3, 4, 19, 4, 5, 22, 7, 24, 63, 32, 33, 34, 51, 52, 35, 52, 53, 38, 55, 40, 47, 48, 49, 50, 35, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1457(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 21, 6, 23, 8, 31, 16, 17, 18, 3, 4, 19, 4, 5, 22, 7, 24, 47, 32, 33, 34, 51, 52, 35, 52, 53, 38, 55, 40, 63, 48, 49, 50, 35, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1458(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 21, 6, 23, 8, 15, 0, 17, 18, 3, 4, 19, 4, 5, 22, 7, 24, 63, 48, 33, 34, 51, 52, 35, 52, 53, 38, 55, 40, 47, 32, 49, 50, 35, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1459(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 21, 6, 23, 8, 31, 0, 17, 18, 3, 4, 19, 4, 5, 22, 7, 24, 47, 48, 33, 34, 51, 52, 35, 52, 53, 38, 55, 40, 63, 32, 49, 50, 35, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1460(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 21, 6, 23, 8, 15, 16, 1, 18, 3, 4, 19, 4, 5, 22, 7, 24, 63, 32, 49, 34, 51, 52, 35, 52, 53, 38, 55, 40, 47, 48, 33, 50, 35, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1461(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 21, 6, 23, 8, 31, 16, 1, 18, 3, 4, 19, 4, 5, 22, 7, 24, 47, 32, 49, 34, 51, 52, 35, 52, 53, 38, 55, 40, 63, 48, 33, 50, 35, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1462(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 21, 6, 23, 8, 15, 0, 1, 18, 3, 4, 19, 4, 5, 22, 7, 24, 63, 48, 49, 34, 51, 52, 35, 52, 53, 38, 55, 40, 47, 32, 33, 50, 35, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1463(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 21, 6, 23, 8, 31, 0, 1, 18, 3, 4, 19, 4, 5, 22, 7, 24, 47, 48, 49, 34, 51, 52, 35, 52, 53, 38, 55, 40, 63, 32, 33, 50, 35, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1464(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 21, 6, 23, 8, 15, 16, 17, 2, 3, 4, 19, 4, 5, 22, 7, 24, 63, 32, 33, 50, 51, 52, 35, 52, 53, 38, 55, 40, 47, 48, 49, 34, 35, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1465(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 21, 6, 23, 8, 31, 16, 17, 2, 3, 4, 19, 4, 5, 22, 7, 24, 47, 32, 33, 50, 51, 52, 35, 52, 53, 38, 55, 40, 63, 48, 49, 34, 35, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1466(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 21, 6, 23, 8, 15, 0, 17, 2, 3, 4, 19, 4, 5, 22, 7, 24, 63, 48, 33, 50, 51, 52, 35, 52, 53, 38, 55, 40, 47, 32, 49, 34, 35, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1467(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 21, 6, 23, 8, 31, 0, 17, 2, 3, 4, 19, 4, 5, 22, 7, 24, 47, 48, 33, 50, 51, 52, 35, 52, 53, 38, 55, 40, 63, 32, 49, 34, 35, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1468(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 21, 6, 23, 8, 15, 16, 1, 2, 3, 4, 19, 4, 5, 22, 7, 24, 63, 32, 49, 50, 51, 52, 35, 52, 53, 38, 55, 40, 47, 48, 33, 34, 35, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1469(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 21, 6, 23, 8, 31, 16, 1, 2, 3, 4, 19, 4, 5, 22, 7, 24, 47, 32, 49, 50, 51, 52, 35, 52, 53, 38, 55, 40, 63, 48, 33, 34, 35, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1470(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 21, 6, 23, 8, 15, 0, 1, 2, 3, 4, 19, 4, 5, 22, 7, 24, 63, 48, 49, 50, 51, 52, 35, 52, 53, 38, 55, 40, 47, 32, 33, 34, 35, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1471(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 21, 6, 23, 8, 31, 0, 1, 2, 3, 4, 19, 4, 5, 22, 7, 24, 47, 48, 49, 50, 51, 52, 35, 52, 53, 38, 55, 40, 63, 32, 33, 34, 35, 36, 51, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1472(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 21, 6, 23, 8, 15, 16, 17, 18, 19, 20, 3, 4, 5, 22, 7, 24, 63, 32, 33, 34, 35, 36, 51, 52, 53, 38, 55, 40, 47, 48, 49, 50, 51, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1473(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 21, 6, 23, 8, 31, 16, 17, 18, 19, 20, 3, 4, 5, 22, 7, 24, 47, 32, 33, 34, 35, 36, 51, 52, 53, 38, 55, 40, 63, 48, 49, 50, 51, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1474(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 21, 6, 23, 8, 15, 0, 17, 18, 19, 20, 3, 4, 5, 22, 7, 24, 63, 48, 33, 34, 35, 36, 51, 52, 53, 38, 55, 40, 47, 32, 49, 50, 51, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1475(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 21, 6, 23, 8, 31, 0, 17, 18, 19, 20, 3, 4, 5, 22, 7, 24, 47, 48, 33, 34, 35, 36, 51, 52, 53, 38, 55, 40, 63, 32, 49, 50, 51, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1476(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 21, 6, 23, 8, 15, 16, 1, 18, 19, 20, 3, 4, 5, 22, 7, 24, 63, 32, 49, 34, 35, 36, 51, 52, 53, 38, 55, 40, 47, 48, 33, 50, 51, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1477(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 21, 6, 23, 8, 31, 16, 1, 18, 19, 20, 3, 4, 5, 22, 7, 24, 47, 32, 49, 34, 35, 36, 51, 52, 53, 38, 55, 40, 63, 48, 33, 50, 51, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1478(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 21, 6, 23, 8, 15, 0, 1, 18, 19, 20, 3, 4, 5, 22, 7, 24, 63, 48, 49, 34, 35, 36, 51, 52, 53, 38, 55, 40, 47, 32, 33, 50, 51, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1479(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 21, 6, 23, 8, 31, 0, 1, 18, 19, 20, 3, 4, 5, 22, 7, 24, 47, 48, 49, 34, 35, 36, 51, 52, 53, 38, 55, 40, 63, 32, 33, 50, 51, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1480(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 21, 6, 23, 8, 15, 16, 17, 2, 19, 20, 3, 4, 5, 22, 7, 24, 63, 32, 33, 50, 35, 36, 51, 52, 53, 38, 55, 40, 47, 48, 49, 34, 51, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1481(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 21, 6, 23, 8, 31, 16, 17, 2, 19, 20, 3, 4, 5, 22, 7, 24, 47, 32, 33, 50, 35, 36, 51, 52, 53, 38, 55, 40, 63, 48, 49, 34, 51, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1482(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 21, 6, 23, 8, 15, 0, 17, 2, 19, 20, 3, 4, 5, 22, 7, 24, 63, 48, 33, 50, 35, 36, 51, 52, 53, 38, 55, 40, 47, 32, 49, 34, 51, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1483(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 21, 6, 23, 8, 31, 0, 17, 2, 19, 20, 3, 4, 5, 22, 7, 24, 47, 48, 33, 50, 35, 36, 51, 52, 53, 38, 55, 40, 63, 32, 49, 34, 51, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1484(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 21, 6, 23, 8, 15, 16, 1, 2, 19, 20, 3, 4, 5, 22, 7, 24, 63, 32, 49, 50, 35, 36, 51, 52, 53, 38, 55, 40, 47, 48, 33, 34, 51, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1485(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 21, 6, 23, 8, 31, 16, 1, 2, 19, 20, 3, 4, 5, 22, 7, 24, 47, 32, 49, 50, 35, 36, 51, 52, 53, 38, 55, 40, 63, 48, 33, 34, 51, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1486(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 21, 6, 23, 8, 15, 0, 1, 2, 19, 20, 3, 4, 5, 22, 7, 24, 63, 48, 49, 50, 35, 36, 51, 52, 53, 38, 55, 40, 47, 32, 33, 34, 51, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1487(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 21, 6, 23, 8, 31, 0, 1, 2, 19, 20, 3, 4, 5, 22, 7, 24, 47, 48, 49, 50, 35, 36, 51, 52, 53, 38, 55, 40, 63, 32, 33, 34, 51, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1488(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 21, 6, 23, 8, 15, 16, 17, 18, 3, 20, 3, 4, 5, 22, 7, 24, 63, 32, 33, 34, 51, 36, 51, 52, 53, 38, 55, 40, 47, 48, 49, 50, 35, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1489(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 21, 6, 23, 8, 31, 16, 17, 18, 3, 20, 3, 4, 5, 22, 7, 24, 47, 32, 33, 34, 51, 36, 51, 52, 53, 38, 55, 40, 63, 48, 49, 50, 35, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1490(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 21, 6, 23, 8, 15, 0, 17, 18, 3, 20, 3, 4, 5, 22, 7, 24, 63, 48, 33, 34, 51, 36, 51, 52, 53, 38, 55, 40, 47, 32, 49, 50, 35, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1491(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 21, 6, 23, 8, 31, 0, 17, 18, 3, 20, 3, 4, 5, 22, 7, 24, 47, 48, 33, 34, 51, 36, 51, 52, 53, 38, 55, 40, 63, 32, 49, 50, 35, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1492(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 21, 6, 23, 8, 15, 16, 1, 18, 3, 20, 3, 4, 5, 22, 7, 24, 63, 32, 49, 34, 51, 36, 51, 52, 53, 38, 55, 40, 47, 48, 33, 50, 35, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1493(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 21, 6, 23, 8, 31, 16, 1, 18, 3, 20, 3, 4, 5, 22, 7, 24, 47, 32, 49, 34, 51, 36, 51, 52, 53, 38, 55, 40, 63, 48, 33, 50, 35, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1494(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 21, 6, 23, 8, 15, 0, 1, 18, 3, 20, 3, 4, 5, 22, 7, 24, 63, 48, 49, 34, 51, 36, 51, 52, 53, 38, 55, 40, 47, 32, 33, 50, 35, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1495(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 21, 6, 23, 8, 31, 0, 1, 18, 3, 20, 3, 4, 5, 22, 7, 24, 47, 48, 49, 34, 51, 36, 51, 52, 53, 38, 55, 40, 63, 32, 33, 50, 35, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1496(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 21, 6, 23, 8, 15, 16, 17, 2, 3, 20, 3, 4, 5, 22, 7, 24, 63, 32, 33, 50, 51, 36, 51, 52, 53, 38, 55, 40, 47, 48, 49, 34, 35, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1497(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 21, 6, 23, 8, 31, 16, 17, 2, 3, 20, 3, 4, 5, 22, 7, 24, 47, 32, 33, 50, 51, 36, 51, 52, 53, 38, 55, 40, 63, 48, 49, 34, 35, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1498(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 21, 6, 23, 8, 15, 0, 17, 2, 3, 20, 3, 4, 5, 22, 7, 24, 63, 48, 33, 50, 51, 36, 51, 52, 53, 38, 55, 40, 47, 32, 49, 34, 35, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1499(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 21, 6, 23, 8, 31, 0, 17, 2, 3, 20, 3, 4, 5, 22, 7, 24, 47, 48, 33, 50, 51, 36, 51, 52, 53, 38, 55, 40, 63, 32, 49, 34, 35, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1500(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 21, 6, 23, 8, 15, 16, 1, 2, 3, 20, 3, 4, 5, 22, 7, 24, 63, 32, 49, 50, 51, 36, 51, 52, 53, 38, 55, 40, 47, 48, 33, 34, 35, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1501(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 21, 6, 23, 8, 31, 16, 1, 2, 3, 20, 3, 4, 5, 22, 7, 24, 47, 32, 49, 50, 51, 36, 51, 52, 53, 38, 55, 40, 63, 48, 33, 34, 35, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1502(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 21, 6, 23, 8, 15, 0, 1, 2, 3, 20, 3, 4, 5, 22, 7, 24, 63, 48, 49, 50, 51, 36, 51, 52, 53, 38, 55, 40, 47, 32, 33, 34, 35, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1503(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 21, 6, 23, 8, 31, 0, 1, 2, 3, 20, 3, 4, 5, 22, 7, 24, 47, 48, 49, 50, 51, 36, 51, 52, 53, 38, 55, 40, 63, 32, 33, 34, 35, 52, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1504(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 21, 6, 23, 8, 15, 16, 17, 18, 19, 4, 3, 4, 5, 22, 7, 24, 63, 32, 33, 34, 35, 52, 51, 52, 53, 38, 55, 40, 47, 48, 49, 50, 51, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1505(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 21, 6, 23, 8, 31, 16, 17, 18, 19, 4, 3, 4, 5, 22, 7, 24, 47, 32, 33, 34, 35, 52, 51, 52, 53, 38, 55, 40, 63, 48, 49, 50, 51, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1506(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 21, 6, 23, 8, 15, 0, 17, 18, 19, 4, 3, 4, 5, 22, 7, 24, 63, 48, 33, 34, 35, 52, 51, 52, 53, 38, 55, 40, 47, 32, 49, 50, 51, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1507(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 21, 6, 23, 8, 31, 0, 17, 18, 19, 4, 3, 4, 5, 22, 7, 24, 47, 48, 33, 34, 35, 52, 51, 52, 53, 38, 55, 40, 63, 32, 49, 50, 51, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1508(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 21, 6, 23, 8, 15, 16, 1, 18, 19, 4, 3, 4, 5, 22, 7, 24, 63, 32, 49, 34, 35, 52, 51, 52, 53, 38, 55, 40, 47, 48, 33, 50, 51, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1509(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 21, 6, 23, 8, 31, 16, 1, 18, 19, 4, 3, 4, 5, 22, 7, 24, 47, 32, 49, 34, 35, 52, 51, 52, 53, 38, 55, 40, 63, 48, 33, 50, 51, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1510(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 21, 6, 23, 8, 15, 0, 1, 18, 19, 4, 3, 4, 5, 22, 7, 24, 63, 48, 49, 34, 35, 52, 51, 52, 53, 38, 55, 40, 47, 32, 33, 50, 51, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1511(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 21, 6, 23, 8, 31, 0, 1, 18, 19, 4, 3, 4, 5, 22, 7, 24, 47, 48, 49, 34, 35, 52, 51, 52, 53, 38, 55, 40, 63, 32, 33, 50, 51, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1512(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 21, 6, 23, 8, 15, 16, 17, 2, 19, 4, 3, 4, 5, 22, 7, 24, 63, 32, 33, 50, 35, 52, 51, 52, 53, 38, 55, 40, 47, 48, 49, 34, 51, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1513(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 21, 6, 23, 8, 31, 16, 17, 2, 19, 4, 3, 4, 5, 22, 7, 24, 47, 32, 33, 50, 35, 52, 51, 52, 53, 38, 55, 40, 63, 48, 49, 34, 51, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1514(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 21, 6, 23, 8, 15, 0, 17, 2, 19, 4, 3, 4, 5, 22, 7, 24, 63, 48, 33, 50, 35, 52, 51, 52, 53, 38, 55, 40, 47, 32, 49, 34, 51, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1515(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 21, 6, 23, 8, 31, 0, 17, 2, 19, 4, 3, 4, 5, 22, 7, 24, 47, 48, 33, 50, 35, 52, 51, 52, 53, 38, 55, 40, 63, 32, 49, 34, 51, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1516(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 21, 6, 23, 8, 15, 16, 1, 2, 19, 4, 3, 4, 5, 22, 7, 24, 63, 32, 49, 50, 35, 52, 51, 52, 53, 38, 55, 40, 47, 48, 33, 34, 51, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1517(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 21, 6, 23, 8, 31, 16, 1, 2, 19, 4, 3, 4, 5, 22, 7, 24, 47, 32, 49, 50, 35, 52, 51, 52, 53, 38, 55, 40, 63, 48, 33, 34, 51, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1518(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 21, 6, 23, 8, 15, 0, 1, 2, 19, 4, 3, 4, 5, 22, 7, 24, 63, 48, 49, 50, 35, 52, 51, 52, 53, 38, 55, 40, 47, 32, 33, 34, 51, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1519(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 21, 6, 23, 8, 31, 0, 1, 2, 19, 4, 3, 4, 5, 22, 7, 24, 47, 48, 49, 50, 35, 52, 51, 52, 53, 38, 55, 40, 63, 32, 33, 34, 51, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1520(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 21, 6, 23, 8, 15, 16, 17, 18, 3, 4, 3, 4, 5, 22, 7, 24, 63, 32, 33, 34, 51, 52, 51, 52, 53, 38, 55, 40, 47, 48, 49, 50, 35, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1521(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 21, 6, 23, 8, 31, 16, 17, 18, 3, 4, 3, 4, 5, 22, 7, 24, 47, 32, 33, 34, 51, 52, 51, 52, 53, 38, 55, 40, 63, 48, 49, 50, 35, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1522(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 21, 6, 23, 8, 15, 0, 17, 18, 3, 4, 3, 4, 5, 22, 7, 24, 63, 48, 33, 34, 51, 52, 51, 52, 53, 38, 55, 40, 47, 32, 49, 50, 35, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1523(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 21, 6, 23, 8, 31, 0, 17, 18, 3, 4, 3, 4, 5, 22, 7, 24, 47, 48, 33, 34, 51, 52, 51, 52, 53, 38, 55, 40, 63, 32, 49, 50, 35, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1524(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 21, 6, 23, 8, 15, 16, 1, 18, 3, 4, 3, 4, 5, 22, 7, 24, 63, 32, 49, 34, 51, 52, 51, 52, 53, 38, 55, 40, 47, 48, 33, 50, 35, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1525(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 21, 6, 23, 8, 31, 16, 1, 18, 3, 4, 3, 4, 5, 22, 7, 24, 47, 32, 49, 34, 51, 52, 51, 52, 53, 38, 55, 40, 63, 48, 33, 50, 35, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1526(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 21, 6, 23, 8, 15, 0, 1, 18, 3, 4, 3, 4, 5, 22, 7, 24, 63, 48, 49, 34, 51, 52, 51, 52, 53, 38, 55, 40, 47, 32, 33, 50, 35, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1527(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 21, 6, 23, 8, 31, 0, 1, 18, 3, 4, 3, 4, 5, 22, 7, 24, 47, 48, 49, 34, 51, 52, 51, 52, 53, 38, 55, 40, 63, 32, 33, 50, 35, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1528(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 21, 6, 23, 8, 15, 16, 17, 2, 3, 4, 3, 4, 5, 22, 7, 24, 63, 32, 33, 50, 51, 52, 51, 52, 53, 38, 55, 40, 47, 48, 49, 34, 35, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1529(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 21, 6, 23, 8, 31, 16, 17, 2, 3, 4, 3, 4, 5, 22, 7, 24, 47, 32, 33, 50, 51, 52, 51, 52, 53, 38, 55, 40, 63, 48, 49, 34, 35, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1530(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 21, 6, 23, 8, 15, 0, 17, 2, 3, 4, 3, 4, 5, 22, 7, 24, 63, 48, 33, 50, 51, 52, 51, 52, 53, 38, 55, 40, 47, 32, 49, 34, 35, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1531(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 21, 6, 23, 8, 31, 0, 17, 2, 3, 4, 3, 4, 5, 22, 7, 24, 47, 48, 33, 50, 51, 52, 51, 52, 53, 38, 55, 40, 63, 32, 49, 34, 35, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1532(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 21, 6, 23, 8, 15, 16, 1, 2, 3, 4, 3, 4, 5, 22, 7, 24, 63, 32, 49, 50, 51, 52, 51, 52, 53, 38, 55, 40, 47, 48, 33, 34, 35, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1533(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 21, 6, 23, 8, 31, 16, 1, 2, 3, 4, 3, 4, 5, 22, 7, 24, 47, 32, 49, 50, 51, 52, 51, 52, 53, 38, 55, 40, 63, 48, 33, 34, 35, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1534(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 21, 6, 23, 8, 15, 0, 1, 2, 3, 4, 3, 4, 5, 22, 7, 24, 63, 48, 49, 50, 51, 52, 51, 52, 53, 38, 55, 40, 47, 32, 33, 34, 35, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1535(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 21, 6, 23, 8, 31, 0, 1, 2, 3, 4, 3, 4, 5, 22, 7, 24, 47, 48, 49, 50, 51, 52, 51, 52, 53, 38, 55, 40, 63, 32, 33, 34, 35, 36, 35, 36, 37, 54, 39, 56>(unchecked_hashes, bitsplitted_keys);
}


