#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt896(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 21, 22, 7, 8, 15, 16, 17, 18, 19, 20, 19, 4, 5, 6, 23, 24, 63, 32, 33, 34, 35, 36, 35, 52, 53, 54, 39, 40, 47, 48, 49, 50, 51, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt897(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 21, 22, 7, 8, 31, 16, 17, 18, 19, 20, 19, 4, 5, 6, 23, 24, 47, 32, 33, 34, 35, 36, 35, 52, 53, 54, 39, 40, 63, 48, 49, 50, 51, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt898(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 21, 22, 7, 8, 15, 0, 17, 18, 19, 20, 19, 4, 5, 6, 23, 24, 63, 48, 33, 34, 35, 36, 35, 52, 53, 54, 39, 40, 47, 32, 49, 50, 51, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt899(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 21, 22, 7, 8, 31, 0, 17, 18, 19, 20, 19, 4, 5, 6, 23, 24, 47, 48, 33, 34, 35, 36, 35, 52, 53, 54, 39, 40, 63, 32, 49, 50, 51, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt900(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 21, 22, 7, 8, 15, 16, 1, 18, 19, 20, 19, 4, 5, 6, 23, 24, 63, 32, 49, 34, 35, 36, 35, 52, 53, 54, 39, 40, 47, 48, 33, 50, 51, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt901(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 21, 22, 7, 8, 31, 16, 1, 18, 19, 20, 19, 4, 5, 6, 23, 24, 47, 32, 49, 34, 35, 36, 35, 52, 53, 54, 39, 40, 63, 48, 33, 50, 51, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt902(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 21, 22, 7, 8, 15, 0, 1, 18, 19, 20, 19, 4, 5, 6, 23, 24, 63, 48, 49, 34, 35, 36, 35, 52, 53, 54, 39, 40, 47, 32, 33, 50, 51, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt903(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 21, 22, 7, 8, 31, 0, 1, 18, 19, 20, 19, 4, 5, 6, 23, 24, 47, 48, 49, 34, 35, 36, 35, 52, 53, 54, 39, 40, 63, 32, 33, 50, 51, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt904(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 21, 22, 7, 8, 15, 16, 17, 2, 19, 20, 19, 4, 5, 6, 23, 24, 63, 32, 33, 50, 35, 36, 35, 52, 53, 54, 39, 40, 47, 48, 49, 34, 51, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt905(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 21, 22, 7, 8, 31, 16, 17, 2, 19, 20, 19, 4, 5, 6, 23, 24, 47, 32, 33, 50, 35, 36, 35, 52, 53, 54, 39, 40, 63, 48, 49, 34, 51, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt906(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 21, 22, 7, 8, 15, 0, 17, 2, 19, 20, 19, 4, 5, 6, 23, 24, 63, 48, 33, 50, 35, 36, 35, 52, 53, 54, 39, 40, 47, 32, 49, 34, 51, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt907(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 21, 22, 7, 8, 31, 0, 17, 2, 19, 20, 19, 4, 5, 6, 23, 24, 47, 48, 33, 50, 35, 36, 35, 52, 53, 54, 39, 40, 63, 32, 49, 34, 51, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt908(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 21, 22, 7, 8, 15, 16, 1, 2, 19, 20, 19, 4, 5, 6, 23, 24, 63, 32, 49, 50, 35, 36, 35, 52, 53, 54, 39, 40, 47, 48, 33, 34, 51, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt909(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 21, 22, 7, 8, 31, 16, 1, 2, 19, 20, 19, 4, 5, 6, 23, 24, 47, 32, 49, 50, 35, 36, 35, 52, 53, 54, 39, 40, 63, 48, 33, 34, 51, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt910(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 21, 22, 7, 8, 15, 0, 1, 2, 19, 20, 19, 4, 5, 6, 23, 24, 63, 48, 49, 50, 35, 36, 35, 52, 53, 54, 39, 40, 47, 32, 33, 34, 51, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt911(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 21, 22, 7, 8, 31, 0, 1, 2, 19, 20, 19, 4, 5, 6, 23, 24, 47, 48, 49, 50, 35, 36, 35, 52, 53, 54, 39, 40, 63, 32, 33, 34, 51, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt912(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 21, 22, 7, 8, 15, 16, 17, 18, 3, 20, 19, 4, 5, 6, 23, 24, 63, 32, 33, 34, 51, 36, 35, 52, 53, 54, 39, 40, 47, 48, 49, 50, 35, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt913(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 21, 22, 7, 8, 31, 16, 17, 18, 3, 20, 19, 4, 5, 6, 23, 24, 47, 32, 33, 34, 51, 36, 35, 52, 53, 54, 39, 40, 63, 48, 49, 50, 35, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt914(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 21, 22, 7, 8, 15, 0, 17, 18, 3, 20, 19, 4, 5, 6, 23, 24, 63, 48, 33, 34, 51, 36, 35, 52, 53, 54, 39, 40, 47, 32, 49, 50, 35, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt915(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 21, 22, 7, 8, 31, 0, 17, 18, 3, 20, 19, 4, 5, 6, 23, 24, 47, 48, 33, 34, 51, 36, 35, 52, 53, 54, 39, 40, 63, 32, 49, 50, 35, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt916(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 21, 22, 7, 8, 15, 16, 1, 18, 3, 20, 19, 4, 5, 6, 23, 24, 63, 32, 49, 34, 51, 36, 35, 52, 53, 54, 39, 40, 47, 48, 33, 50, 35, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt917(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 21, 22, 7, 8, 31, 16, 1, 18, 3, 20, 19, 4, 5, 6, 23, 24, 47, 32, 49, 34, 51, 36, 35, 52, 53, 54, 39, 40, 63, 48, 33, 50, 35, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt918(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 21, 22, 7, 8, 15, 0, 1, 18, 3, 20, 19, 4, 5, 6, 23, 24, 63, 48, 49, 34, 51, 36, 35, 52, 53, 54, 39, 40, 47, 32, 33, 50, 35, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt919(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 21, 22, 7, 8, 31, 0, 1, 18, 3, 20, 19, 4, 5, 6, 23, 24, 47, 48, 49, 34, 51, 36, 35, 52, 53, 54, 39, 40, 63, 32, 33, 50, 35, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt920(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 21, 22, 7, 8, 15, 16, 17, 2, 3, 20, 19, 4, 5, 6, 23, 24, 63, 32, 33, 50, 51, 36, 35, 52, 53, 54, 39, 40, 47, 48, 49, 34, 35, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt921(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 21, 22, 7, 8, 31, 16, 17, 2, 3, 20, 19, 4, 5, 6, 23, 24, 47, 32, 33, 50, 51, 36, 35, 52, 53, 54, 39, 40, 63, 48, 49, 34, 35, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt922(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 21, 22, 7, 8, 15, 0, 17, 2, 3, 20, 19, 4, 5, 6, 23, 24, 63, 48, 33, 50, 51, 36, 35, 52, 53, 54, 39, 40, 47, 32, 49, 34, 35, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt923(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 21, 22, 7, 8, 31, 0, 17, 2, 3, 20, 19, 4, 5, 6, 23, 24, 47, 48, 33, 50, 51, 36, 35, 52, 53, 54, 39, 40, 63, 32, 49, 34, 35, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt924(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 21, 22, 7, 8, 15, 16, 1, 2, 3, 20, 19, 4, 5, 6, 23, 24, 63, 32, 49, 50, 51, 36, 35, 52, 53, 54, 39, 40, 47, 48, 33, 34, 35, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt925(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 21, 22, 7, 8, 31, 16, 1, 2, 3, 20, 19, 4, 5, 6, 23, 24, 47, 32, 49, 50, 51, 36, 35, 52, 53, 54, 39, 40, 63, 48, 33, 34, 35, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt926(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 21, 22, 7, 8, 15, 0, 1, 2, 3, 20, 19, 4, 5, 6, 23, 24, 63, 48, 49, 50, 51, 36, 35, 52, 53, 54, 39, 40, 47, 32, 33, 34, 35, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt927(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 21, 22, 7, 8, 31, 0, 1, 2, 3, 20, 19, 4, 5, 6, 23, 24, 47, 48, 49, 50, 51, 36, 35, 52, 53, 54, 39, 40, 63, 32, 33, 34, 35, 52, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt928(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 21, 22, 7, 8, 15, 16, 17, 18, 19, 4, 19, 4, 5, 6, 23, 24, 63, 32, 33, 34, 35, 52, 35, 52, 53, 54, 39, 40, 47, 48, 49, 50, 51, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt929(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 21, 22, 7, 8, 31, 16, 17, 18, 19, 4, 19, 4, 5, 6, 23, 24, 47, 32, 33, 34, 35, 52, 35, 52, 53, 54, 39, 40, 63, 48, 49, 50, 51, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt930(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 21, 22, 7, 8, 15, 0, 17, 18, 19, 4, 19, 4, 5, 6, 23, 24, 63, 48, 33, 34, 35, 52, 35, 52, 53, 54, 39, 40, 47, 32, 49, 50, 51, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt931(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 21, 22, 7, 8, 31, 0, 17, 18, 19, 4, 19, 4, 5, 6, 23, 24, 47, 48, 33, 34, 35, 52, 35, 52, 53, 54, 39, 40, 63, 32, 49, 50, 51, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt932(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 21, 22, 7, 8, 15, 16, 1, 18, 19, 4, 19, 4, 5, 6, 23, 24, 63, 32, 49, 34, 35, 52, 35, 52, 53, 54, 39, 40, 47, 48, 33, 50, 51, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt933(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 21, 22, 7, 8, 31, 16, 1, 18, 19, 4, 19, 4, 5, 6, 23, 24, 47, 32, 49, 34, 35, 52, 35, 52, 53, 54, 39, 40, 63, 48, 33, 50, 51, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt934(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 21, 22, 7, 8, 15, 0, 1, 18, 19, 4, 19, 4, 5, 6, 23, 24, 63, 48, 49, 34, 35, 52, 35, 52, 53, 54, 39, 40, 47, 32, 33, 50, 51, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt935(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 21, 22, 7, 8, 31, 0, 1, 18, 19, 4, 19, 4, 5, 6, 23, 24, 47, 48, 49, 34, 35, 52, 35, 52, 53, 54, 39, 40, 63, 32, 33, 50, 51, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt936(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 21, 22, 7, 8, 15, 16, 17, 2, 19, 4, 19, 4, 5, 6, 23, 24, 63, 32, 33, 50, 35, 52, 35, 52, 53, 54, 39, 40, 47, 48, 49, 34, 51, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt937(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 21, 22, 7, 8, 31, 16, 17, 2, 19, 4, 19, 4, 5, 6, 23, 24, 47, 32, 33, 50, 35, 52, 35, 52, 53, 54, 39, 40, 63, 48, 49, 34, 51, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt938(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 21, 22, 7, 8, 15, 0, 17, 2, 19, 4, 19, 4, 5, 6, 23, 24, 63, 48, 33, 50, 35, 52, 35, 52, 53, 54, 39, 40, 47, 32, 49, 34, 51, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt939(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 21, 22, 7, 8, 31, 0, 17, 2, 19, 4, 19, 4, 5, 6, 23, 24, 47, 48, 33, 50, 35, 52, 35, 52, 53, 54, 39, 40, 63, 32, 49, 34, 51, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt940(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 21, 22, 7, 8, 15, 16, 1, 2, 19, 4, 19, 4, 5, 6, 23, 24, 63, 32, 49, 50, 35, 52, 35, 52, 53, 54, 39, 40, 47, 48, 33, 34, 51, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt941(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 21, 22, 7, 8, 31, 16, 1, 2, 19, 4, 19, 4, 5, 6, 23, 24, 47, 32, 49, 50, 35, 52, 35, 52, 53, 54, 39, 40, 63, 48, 33, 34, 51, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt942(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 21, 22, 7, 8, 15, 0, 1, 2, 19, 4, 19, 4, 5, 6, 23, 24, 63, 48, 49, 50, 35, 52, 35, 52, 53, 54, 39, 40, 47, 32, 33, 34, 51, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt943(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 21, 22, 7, 8, 31, 0, 1, 2, 19, 4, 19, 4, 5, 6, 23, 24, 47, 48, 49, 50, 35, 52, 35, 52, 53, 54, 39, 40, 63, 32, 33, 34, 51, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt944(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 21, 22, 7, 8, 15, 16, 17, 18, 3, 4, 19, 4, 5, 6, 23, 24, 63, 32, 33, 34, 51, 52, 35, 52, 53, 54, 39, 40, 47, 48, 49, 50, 35, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt945(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 21, 22, 7, 8, 31, 16, 17, 18, 3, 4, 19, 4, 5, 6, 23, 24, 47, 32, 33, 34, 51, 52, 35, 52, 53, 54, 39, 40, 63, 48, 49, 50, 35, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt946(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 21, 22, 7, 8, 15, 0, 17, 18, 3, 4, 19, 4, 5, 6, 23, 24, 63, 48, 33, 34, 51, 52, 35, 52, 53, 54, 39, 40, 47, 32, 49, 50, 35, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt947(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 21, 22, 7, 8, 31, 0, 17, 18, 3, 4, 19, 4, 5, 6, 23, 24, 47, 48, 33, 34, 51, 52, 35, 52, 53, 54, 39, 40, 63, 32, 49, 50, 35, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt948(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 21, 22, 7, 8, 15, 16, 1, 18, 3, 4, 19, 4, 5, 6, 23, 24, 63, 32, 49, 34, 51, 52, 35, 52, 53, 54, 39, 40, 47, 48, 33, 50, 35, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt949(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 21, 22, 7, 8, 31, 16, 1, 18, 3, 4, 19, 4, 5, 6, 23, 24, 47, 32, 49, 34, 51, 52, 35, 52, 53, 54, 39, 40, 63, 48, 33, 50, 35, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt950(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 21, 22, 7, 8, 15, 0, 1, 18, 3, 4, 19, 4, 5, 6, 23, 24, 63, 48, 49, 34, 51, 52, 35, 52, 53, 54, 39, 40, 47, 32, 33, 50, 35, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt951(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 21, 22, 7, 8, 31, 0, 1, 18, 3, 4, 19, 4, 5, 6, 23, 24, 47, 48, 49, 34, 51, 52, 35, 52, 53, 54, 39, 40, 63, 32, 33, 50, 35, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt952(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 21, 22, 7, 8, 15, 16, 17, 2, 3, 4, 19, 4, 5, 6, 23, 24, 63, 32, 33, 50, 51, 52, 35, 52, 53, 54, 39, 40, 47, 48, 49, 34, 35, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt953(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 21, 22, 7, 8, 31, 16, 17, 2, 3, 4, 19, 4, 5, 6, 23, 24, 47, 32, 33, 50, 51, 52, 35, 52, 53, 54, 39, 40, 63, 48, 49, 34, 35, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt954(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 21, 22, 7, 8, 15, 0, 17, 2, 3, 4, 19, 4, 5, 6, 23, 24, 63, 48, 33, 50, 51, 52, 35, 52, 53, 54, 39, 40, 47, 32, 49, 34, 35, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt955(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 21, 22, 7, 8, 31, 0, 17, 2, 3, 4, 19, 4, 5, 6, 23, 24, 47, 48, 33, 50, 51, 52, 35, 52, 53, 54, 39, 40, 63, 32, 49, 34, 35, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt956(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 21, 22, 7, 8, 15, 16, 1, 2, 3, 4, 19, 4, 5, 6, 23, 24, 63, 32, 49, 50, 51, 52, 35, 52, 53, 54, 39, 40, 47, 48, 33, 34, 35, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt957(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 21, 22, 7, 8, 31, 16, 1, 2, 3, 4, 19, 4, 5, 6, 23, 24, 47, 32, 49, 50, 51, 52, 35, 52, 53, 54, 39, 40, 63, 48, 33, 34, 35, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt958(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 21, 22, 7, 8, 15, 0, 1, 2, 3, 4, 19, 4, 5, 6, 23, 24, 63, 48, 49, 50, 51, 52, 35, 52, 53, 54, 39, 40, 47, 32, 33, 34, 35, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt959(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 21, 22, 7, 8, 31, 0, 1, 2, 3, 4, 19, 4, 5, 6, 23, 24, 47, 48, 49, 50, 51, 52, 35, 52, 53, 54, 39, 40, 63, 32, 33, 34, 35, 36, 51, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt960(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 21, 22, 7, 8, 15, 16, 17, 18, 19, 20, 3, 4, 5, 6, 23, 24, 63, 32, 33, 34, 35, 36, 51, 52, 53, 54, 39, 40, 47, 48, 49, 50, 51, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt961(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 21, 22, 7, 8, 31, 16, 17, 18, 19, 20, 3, 4, 5, 6, 23, 24, 47, 32, 33, 34, 35, 36, 51, 52, 53, 54, 39, 40, 63, 48, 49, 50, 51, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt962(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 21, 22, 7, 8, 15, 0, 17, 18, 19, 20, 3, 4, 5, 6, 23, 24, 63, 48, 33, 34, 35, 36, 51, 52, 53, 54, 39, 40, 47, 32, 49, 50, 51, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt963(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 21, 22, 7, 8, 31, 0, 17, 18, 19, 20, 3, 4, 5, 6, 23, 24, 47, 48, 33, 34, 35, 36, 51, 52, 53, 54, 39, 40, 63, 32, 49, 50, 51, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt964(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 21, 22, 7, 8, 15, 16, 1, 18, 19, 20, 3, 4, 5, 6, 23, 24, 63, 32, 49, 34, 35, 36, 51, 52, 53, 54, 39, 40, 47, 48, 33, 50, 51, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt965(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 21, 22, 7, 8, 31, 16, 1, 18, 19, 20, 3, 4, 5, 6, 23, 24, 47, 32, 49, 34, 35, 36, 51, 52, 53, 54, 39, 40, 63, 48, 33, 50, 51, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt966(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 21, 22, 7, 8, 15, 0, 1, 18, 19, 20, 3, 4, 5, 6, 23, 24, 63, 48, 49, 34, 35, 36, 51, 52, 53, 54, 39, 40, 47, 32, 33, 50, 51, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt967(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 21, 22, 7, 8, 31, 0, 1, 18, 19, 20, 3, 4, 5, 6, 23, 24, 47, 48, 49, 34, 35, 36, 51, 52, 53, 54, 39, 40, 63, 32, 33, 50, 51, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt968(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 21, 22, 7, 8, 15, 16, 17, 2, 19, 20, 3, 4, 5, 6, 23, 24, 63, 32, 33, 50, 35, 36, 51, 52, 53, 54, 39, 40, 47, 48, 49, 34, 51, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt969(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 21, 22, 7, 8, 31, 16, 17, 2, 19, 20, 3, 4, 5, 6, 23, 24, 47, 32, 33, 50, 35, 36, 51, 52, 53, 54, 39, 40, 63, 48, 49, 34, 51, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt970(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 21, 22, 7, 8, 15, 0, 17, 2, 19, 20, 3, 4, 5, 6, 23, 24, 63, 48, 33, 50, 35, 36, 51, 52, 53, 54, 39, 40, 47, 32, 49, 34, 51, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt971(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 21, 22, 7, 8, 31, 0, 17, 2, 19, 20, 3, 4, 5, 6, 23, 24, 47, 48, 33, 50, 35, 36, 51, 52, 53, 54, 39, 40, 63, 32, 49, 34, 51, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt972(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 21, 22, 7, 8, 15, 16, 1, 2, 19, 20, 3, 4, 5, 6, 23, 24, 63, 32, 49, 50, 35, 36, 51, 52, 53, 54, 39, 40, 47, 48, 33, 34, 51, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt973(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 21, 22, 7, 8, 31, 16, 1, 2, 19, 20, 3, 4, 5, 6, 23, 24, 47, 32, 49, 50, 35, 36, 51, 52, 53, 54, 39, 40, 63, 48, 33, 34, 51, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt974(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 21, 22, 7, 8, 15, 0, 1, 2, 19, 20, 3, 4, 5, 6, 23, 24, 63, 48, 49, 50, 35, 36, 51, 52, 53, 54, 39, 40, 47, 32, 33, 34, 51, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt975(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 21, 22, 7, 8, 31, 0, 1, 2, 19, 20, 3, 4, 5, 6, 23, 24, 47, 48, 49, 50, 35, 36, 51, 52, 53, 54, 39, 40, 63, 32, 33, 34, 51, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt976(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 21, 22, 7, 8, 15, 16, 17, 18, 3, 20, 3, 4, 5, 6, 23, 24, 63, 32, 33, 34, 51, 36, 51, 52, 53, 54, 39, 40, 47, 48, 49, 50, 35, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt977(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 21, 22, 7, 8, 31, 16, 17, 18, 3, 20, 3, 4, 5, 6, 23, 24, 47, 32, 33, 34, 51, 36, 51, 52, 53, 54, 39, 40, 63, 48, 49, 50, 35, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt978(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 21, 22, 7, 8, 15, 0, 17, 18, 3, 20, 3, 4, 5, 6, 23, 24, 63, 48, 33, 34, 51, 36, 51, 52, 53, 54, 39, 40, 47, 32, 49, 50, 35, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt979(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 21, 22, 7, 8, 31, 0, 17, 18, 3, 20, 3, 4, 5, 6, 23, 24, 47, 48, 33, 34, 51, 36, 51, 52, 53, 54, 39, 40, 63, 32, 49, 50, 35, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt980(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 21, 22, 7, 8, 15, 16, 1, 18, 3, 20, 3, 4, 5, 6, 23, 24, 63, 32, 49, 34, 51, 36, 51, 52, 53, 54, 39, 40, 47, 48, 33, 50, 35, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt981(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 21, 22, 7, 8, 31, 16, 1, 18, 3, 20, 3, 4, 5, 6, 23, 24, 47, 32, 49, 34, 51, 36, 51, 52, 53, 54, 39, 40, 63, 48, 33, 50, 35, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt982(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 21, 22, 7, 8, 15, 0, 1, 18, 3, 20, 3, 4, 5, 6, 23, 24, 63, 48, 49, 34, 51, 36, 51, 52, 53, 54, 39, 40, 47, 32, 33, 50, 35, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt983(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 21, 22, 7, 8, 31, 0, 1, 18, 3, 20, 3, 4, 5, 6, 23, 24, 47, 48, 49, 34, 51, 36, 51, 52, 53, 54, 39, 40, 63, 32, 33, 50, 35, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt984(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 21, 22, 7, 8, 15, 16, 17, 2, 3, 20, 3, 4, 5, 6, 23, 24, 63, 32, 33, 50, 51, 36, 51, 52, 53, 54, 39, 40, 47, 48, 49, 34, 35, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt985(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 21, 22, 7, 8, 31, 16, 17, 2, 3, 20, 3, 4, 5, 6, 23, 24, 47, 32, 33, 50, 51, 36, 51, 52, 53, 54, 39, 40, 63, 48, 49, 34, 35, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt986(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 21, 22, 7, 8, 15, 0, 17, 2, 3, 20, 3, 4, 5, 6, 23, 24, 63, 48, 33, 50, 51, 36, 51, 52, 53, 54, 39, 40, 47, 32, 49, 34, 35, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt987(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 21, 22, 7, 8, 31, 0, 17, 2, 3, 20, 3, 4, 5, 6, 23, 24, 47, 48, 33, 50, 51, 36, 51, 52, 53, 54, 39, 40, 63, 32, 49, 34, 35, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt988(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 21, 22, 7, 8, 15, 16, 1, 2, 3, 20, 3, 4, 5, 6, 23, 24, 63, 32, 49, 50, 51, 36, 51, 52, 53, 54, 39, 40, 47, 48, 33, 34, 35, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt989(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 21, 22, 7, 8, 31, 16, 1, 2, 3, 20, 3, 4, 5, 6, 23, 24, 47, 32, 49, 50, 51, 36, 51, 52, 53, 54, 39, 40, 63, 48, 33, 34, 35, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt990(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 21, 22, 7, 8, 15, 0, 1, 2, 3, 20, 3, 4, 5, 6, 23, 24, 63, 48, 49, 50, 51, 36, 51, 52, 53, 54, 39, 40, 47, 32, 33, 34, 35, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt991(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 21, 22, 7, 8, 31, 0, 1, 2, 3, 20, 3, 4, 5, 6, 23, 24, 47, 48, 49, 50, 51, 36, 51, 52, 53, 54, 39, 40, 63, 32, 33, 34, 35, 52, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt992(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 21, 22, 7, 8, 15, 16, 17, 18, 19, 4, 3, 4, 5, 6, 23, 24, 63, 32, 33, 34, 35, 52, 51, 52, 53, 54, 39, 40, 47, 48, 49, 50, 51, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt993(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 21, 22, 7, 8, 31, 16, 17, 18, 19, 4, 3, 4, 5, 6, 23, 24, 47, 32, 33, 34, 35, 52, 51, 52, 53, 54, 39, 40, 63, 48, 49, 50, 51, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt994(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 21, 22, 7, 8, 15, 0, 17, 18, 19, 4, 3, 4, 5, 6, 23, 24, 63, 48, 33, 34, 35, 52, 51, 52, 53, 54, 39, 40, 47, 32, 49, 50, 51, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt995(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 21, 22, 7, 8, 31, 0, 17, 18, 19, 4, 3, 4, 5, 6, 23, 24, 47, 48, 33, 34, 35, 52, 51, 52, 53, 54, 39, 40, 63, 32, 49, 50, 51, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt996(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 21, 22, 7, 8, 15, 16, 1, 18, 19, 4, 3, 4, 5, 6, 23, 24, 63, 32, 49, 34, 35, 52, 51, 52, 53, 54, 39, 40, 47, 48, 33, 50, 51, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt997(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 21, 22, 7, 8, 31, 16, 1, 18, 19, 4, 3, 4, 5, 6, 23, 24, 47, 32, 49, 34, 35, 52, 51, 52, 53, 54, 39, 40, 63, 48, 33, 50, 51, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt998(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 21, 22, 7, 8, 15, 0, 1, 18, 19, 4, 3, 4, 5, 6, 23, 24, 63, 48, 49, 34, 35, 52, 51, 52, 53, 54, 39, 40, 47, 32, 33, 50, 51, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt999(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 21, 22, 7, 8, 31, 0, 1, 18, 19, 4, 3, 4, 5, 6, 23, 24, 47, 48, 49, 34, 35, 52, 51, 52, 53, 54, 39, 40, 63, 32, 33, 50, 51, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1000(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 21, 22, 7, 8, 15, 16, 17, 2, 19, 4, 3, 4, 5, 6, 23, 24, 63, 32, 33, 50, 35, 52, 51, 52, 53, 54, 39, 40, 47, 48, 49, 34, 51, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1001(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 21, 22, 7, 8, 31, 16, 17, 2, 19, 4, 3, 4, 5, 6, 23, 24, 47, 32, 33, 50, 35, 52, 51, 52, 53, 54, 39, 40, 63, 48, 49, 34, 51, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1002(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 21, 22, 7, 8, 15, 0, 17, 2, 19, 4, 3, 4, 5, 6, 23, 24, 63, 48, 33, 50, 35, 52, 51, 52, 53, 54, 39, 40, 47, 32, 49, 34, 51, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1003(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 21, 22, 7, 8, 31, 0, 17, 2, 19, 4, 3, 4, 5, 6, 23, 24, 47, 48, 33, 50, 35, 52, 51, 52, 53, 54, 39, 40, 63, 32, 49, 34, 51, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1004(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 21, 22, 7, 8, 15, 16, 1, 2, 19, 4, 3, 4, 5, 6, 23, 24, 63, 32, 49, 50, 35, 52, 51, 52, 53, 54, 39, 40, 47, 48, 33, 34, 51, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1005(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 21, 22, 7, 8, 31, 16, 1, 2, 19, 4, 3, 4, 5, 6, 23, 24, 47, 32, 49, 50, 35, 52, 51, 52, 53, 54, 39, 40, 63, 48, 33, 34, 51, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1006(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 21, 22, 7, 8, 15, 0, 1, 2, 19, 4, 3, 4, 5, 6, 23, 24, 63, 48, 49, 50, 35, 52, 51, 52, 53, 54, 39, 40, 47, 32, 33, 34, 51, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1007(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 21, 22, 7, 8, 31, 0, 1, 2, 19, 4, 3, 4, 5, 6, 23, 24, 47, 48, 49, 50, 35, 52, 51, 52, 53, 54, 39, 40, 63, 32, 33, 34, 51, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1008(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 21, 22, 7, 8, 15, 16, 17, 18, 3, 4, 3, 4, 5, 6, 23, 24, 63, 32, 33, 34, 51, 52, 51, 52, 53, 54, 39, 40, 47, 48, 49, 50, 35, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1009(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 21, 22, 7, 8, 31, 16, 17, 18, 3, 4, 3, 4, 5, 6, 23, 24, 47, 32, 33, 34, 51, 52, 51, 52, 53, 54, 39, 40, 63, 48, 49, 50, 35, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1010(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 21, 22, 7, 8, 15, 0, 17, 18, 3, 4, 3, 4, 5, 6, 23, 24, 63, 48, 33, 34, 51, 52, 51, 52, 53, 54, 39, 40, 47, 32, 49, 50, 35, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1011(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 21, 22, 7, 8, 31, 0, 17, 18, 3, 4, 3, 4, 5, 6, 23, 24, 47, 48, 33, 34, 51, 52, 51, 52, 53, 54, 39, 40, 63, 32, 49, 50, 35, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1012(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 21, 22, 7, 8, 15, 16, 1, 18, 3, 4, 3, 4, 5, 6, 23, 24, 63, 32, 49, 34, 51, 52, 51, 52, 53, 54, 39, 40, 47, 48, 33, 50, 35, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1013(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 21, 22, 7, 8, 31, 16, 1, 18, 3, 4, 3, 4, 5, 6, 23, 24, 47, 32, 49, 34, 51, 52, 51, 52, 53, 54, 39, 40, 63, 48, 33, 50, 35, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1014(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 21, 22, 7, 8, 15, 0, 1, 18, 3, 4, 3, 4, 5, 6, 23, 24, 63, 48, 49, 34, 51, 52, 51, 52, 53, 54, 39, 40, 47, 32, 33, 50, 35, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1015(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 21, 22, 7, 8, 31, 0, 1, 18, 3, 4, 3, 4, 5, 6, 23, 24, 47, 48, 49, 34, 51, 52, 51, 52, 53, 54, 39, 40, 63, 32, 33, 50, 35, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1016(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 21, 22, 7, 8, 15, 16, 17, 2, 3, 4, 3, 4, 5, 6, 23, 24, 63, 32, 33, 50, 51, 52, 51, 52, 53, 54, 39, 40, 47, 48, 49, 34, 35, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1017(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 21, 22, 7, 8, 31, 16, 17, 2, 3, 4, 3, 4, 5, 6, 23, 24, 47, 32, 33, 50, 51, 52, 51, 52, 53, 54, 39, 40, 63, 48, 49, 34, 35, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1018(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 21, 22, 7, 8, 15, 0, 17, 2, 3, 4, 3, 4, 5, 6, 23, 24, 63, 48, 33, 50, 51, 52, 51, 52, 53, 54, 39, 40, 47, 32, 49, 34, 35, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1019(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 21, 22, 7, 8, 31, 0, 17, 2, 3, 4, 3, 4, 5, 6, 23, 24, 47, 48, 33, 50, 51, 52, 51, 52, 53, 54, 39, 40, 63, 32, 49, 34, 35, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1020(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 21, 22, 7, 8, 15, 16, 1, 2, 3, 4, 3, 4, 5, 6, 23, 24, 63, 32, 49, 50, 51, 52, 51, 52, 53, 54, 39, 40, 47, 48, 33, 34, 35, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1021(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 21, 22, 7, 8, 31, 16, 1, 2, 3, 4, 3, 4, 5, 6, 23, 24, 47, 32, 49, 50, 51, 52, 51, 52, 53, 54, 39, 40, 63, 48, 33, 34, 35, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1022(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 21, 22, 7, 8, 15, 0, 1, 2, 3, 4, 3, 4, 5, 6, 23, 24, 63, 48, 49, 50, 51, 52, 51, 52, 53, 54, 39, 40, 47, 32, 33, 34, 35, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1023(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 21, 22, 7, 8, 31, 0, 1, 2, 3, 4, 3, 4, 5, 6, 23, 24, 47, 48, 49, 50, 51, 52, 51, 52, 53, 54, 39, 40, 63, 32, 33, 34, 35, 36, 35, 36, 37, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}


