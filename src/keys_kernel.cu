#include "hip/hip_runtime.h"
/*
 * This software is Copyright (c) 2015 Sayantan Datta <std2048 at gmail dot com>
 * and it is hereby released to the general public under the following terms:
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted.
 * Based on Solar Designer implementation of DES_bs_b.c in jtr-v1.7.9
 */

#include "keys_kernel.h"
#include "logical_ops.h"

#define mask01 0x01010101
#define mask02 0x02020202
#define mask04 0x04040404
#define mask08 0x08080808
#define mask10 0x10101010
#define mask20 0x20202020
#define mask40 0x40404040
#define mask80 0x80808080

#define vand_shl1_or(dst, src, mask)		\
	vand(tmp, src, mask); 					\
	vshl1(tmp, tmp);						\
	vor(dst, dst, tmp)

#define vand_shl_or(dst, src, mask, shift)	\
	vand(tmp, src, mask); 					\
	vshl(tmp, tmp, shift); 					\
	vor(dst, dst, tmp)

#define vand_shl1(dst, src, mask) 			\
	vand(tmp, src, mask) ;					\
	vshl1(dst, tmp)

#define vand_or(dst, src, mask) 			\
	vand(tmp, src, mask); 					\
	vor(dst, dst, tmp)

#define vand_shr_or(dst, src, mask, shift)	\
	vand(tmp, src, mask); 					\
	vshr(tmp, tmp, shift); 					\
	vor(dst, dst, tmp)

#define vand_shr(dst, src, mask, shift) 	\
	vand(tmp, src, mask); 					\
	vshr(dst, tmp, shift)

#define LOAD_V 							\
	vtype v0 = *(vtype*) &vp[0]; 		\
	vtype v1 = *(vtype*) &vp[1]; 		\
	vtype v2 = *(vtype*) &vp[2]; 		\
	vtype v3 = *(vtype*) &vp[3]; 		\
	vtype v4 = *(vtype*) &vp[4]; 		\
	vtype v5 = *(vtype*) &vp[5];		\
	vtype v6 = *(vtype*) &vp[6]; 		\
	vtype v7 = *(vtype*) &vp[7];

#define FINALIZE_NEXT_KEY_BIT_0g { 		\
	vtype m = mask01, va, vb, tmp; 		\
	vand(va, v0, m); 					\
	vand_shl1(vb, v1, m); 				\
	vand_shl_or(va, v2, m, 2); 			\
	vand_shl_or(vb, v3, m, 3); 			\
	vand_shl_or(va, v4, m, 4); 			\
	vand_shl_or(vb, v5, m, 5); 			\
	vand_shl_or(va, v6, m, 6); 			\
	vand_shl_or(vb, v7, m, 7); 			\
	vor(kp[0], va, vb);					\
	kp += gws;							\
}

#define FINALIZE_NEXT_KEY_BIT_1g { 		\
	vtype m = mask02, va, vb, tmp; 		\
	vand_shr(va, v0, m, 1); 			\
	vand(vb, v1, m); 					\
	vand_shl1_or(va, v2, m); 			\
	vand_shl_or(vb, v3, m, 2); 			\
	vand_shl_or(va, v4, m, 3); 			\
	vand_shl_or(vb, v5, m, 4); 			\
	vand_shl_or(va, v6, m, 5); 			\
	vand_shl_or(vb, v7, m, 6); 			\
	vor(kp[0], va, vb); 				\
	kp += gws;							\
}

#define FINALIZE_NEXT_KEY_BIT_2g { 		\
	vtype m = mask04, va, vb, tmp; 		\
	vand_shr(va, v0, m, 2); 			\
	vand_shr(vb, v1, m, 1); 			\
	vand_or(va, v2, m); 				\
	vand_shl1_or(vb, v3, m); 			\
	vand_shl_or(va, v4, m, 2); 			\
	vand_shl_or(vb, v5, m, 3); 			\
	vand_shl_or(va, v6, m, 4); 			\
	vand_shl_or(vb, v7, m, 5); 			\
	vor(kp[0], va, vb); 				\
	kp += gws;							\
}

#define FINALIZE_NEXT_KEY_BIT_3g { 		\
	vtype m = mask08, va, vb, tmp; 		\
	vand_shr(va, v0, m, 3); 			\
	vand_shr(vb, v1, m, 2); 			\
	vand_shr_or(va, v2, m, 1); 			\
	vand_or(vb, v3, m); 				\
	vand_shl1_or(va, v4, m); 			\
	vand_shl_or(vb, v5, m, 2); 			\
	vand_shl_or(va, v6, m, 3); 			\
	vand_shl_or(vb, v7, m, 4); 			\
	vor(kp[0], va, vb); 				\
	kp += gws;							\
}

#define FINALIZE_NEXT_KEY_BIT_4g { 		\
	vtype m = mask10, va, vb, tmp; 		\
	vand_shr(va, v0, m, 4); 			\
	vand_shr(vb, v1, m, 3); 			\
	vand_shr_or(va, v2, m, 2); 			\
	vand_shr_or(vb, v3, m, 1); 			\
	vand_or(va, v4, m); 				\
	vand_shl1_or(vb, v5, m); 			\
	vand_shl_or(va, v6, m, 2); 			\
	vand_shl_or(vb, v7, m, 3); 			\
	vor(kp[0], va, vb); 				\
	kp += gws;							\
}

#define FINALIZE_NEXT_KEY_BIT_5g { 		\
	vtype m = mask20, va, vb, tmp; 		\
	vand_shr(va, v0, m, 5); 			\
	vand_shr(vb, v1, m, 4); 			\
	vand_shr_or(va, v2, m, 3); 			\
	vand_shr_or(vb, v3, m, 2); 			\
	vand_shr_or(va, v4, m, 1); 			\
	vand_or(vb, v5, m); 				\
	vand_shl1_or(va, v6, m); 			\
	vand_shl_or(vb, v7, m, 2); 			\
	vor(kp[0], va, vb); 				\
	kp += gws;							\
}

#define FINALIZE_NEXT_KEY_BIT_6g { 		\
	vtype m = mask40, va, vb, tmp; 		\
	vand_shr(va, v0, m, 6); 			\
	vand_shr(vb, v1, m, 5); 			\
	vand_shr_or(va, v2, m, 4); 			\
	vand_shr_or(vb, v3, m, 3); 			\
	vand_shr_or(va, v4, m, 2); 			\
	vand_shr_or(vb, v5, m, 1); 			\
	vand_or(va, v6, m); 				\
	vand_shl1_or(vb, v7, m); 			\
	vor(kp[0], va, vb); 				\
	kp += gws; 							\
}

#define FINALIZE_NEXT_KEY_BIT_7g { 		\
	vtype m = mask80, va, vb, tmp; 		\
	vand_shr(va, v0, m, 7); 			\
	vand_shr(vb, v1, m, 6); 			\
	vand_shr_or(va, v2, m, 5); 			\
	vand_shr_or(vb, v3, m, 4); 			\
	vand_shr_or(va, v4, m, 3); 			\
	vand_shr_or(vb, v5, m, 2); 			\
	vand_shr_or(va, v6, m, 1); 			\
	vand_or(vb, v7, m); 				\
	vor(kp[0], va, vb); 				\
	kp += gws;							\
}

// The bit-splitting CUDA kernel takes 64-bit keys from keys_transfer and turns them into 56-bit keys (i.e. ASCII printable subset).
// The keys are split 1-bit per column (56 columns in total).
__global__ void bitsplit_keys(
	vtype* bitsplitted_keys,
	const keys_transfer* keys_transfers)
{

	const int section = blockIdx.x * blockDim.x + threadIdx.x;
	const int gws = gridDim.x * blockDim.x;
	
	vtype *kp = &bitsplitted_keys[section];

	for (int ic = 0; ic < 8; ++ic) 
	{
		const vtype *vp = &keys_transfers[section].v[ic][0];
		
		LOAD_V
		FINALIZE_NEXT_KEY_BIT_0g
		FINALIZE_NEXT_KEY_BIT_1g
		FINALIZE_NEXT_KEY_BIT_2g
		FINALIZE_NEXT_KEY_BIT_3g
		FINALIZE_NEXT_KEY_BIT_4g
		FINALIZE_NEXT_KEY_BIT_5g
		FINALIZE_NEXT_KEY_BIT_6g
	}
}

void bitsplit_keys(size_t num_blocks, size_t threads_per_block, vtype* bitsplitted_keys, const keys_transfer* keys_transfers)
{
	bitsplit_keys<<<num_blocks, threads_per_block>>>(bitsplitted_keys, keys_transfers);
}
