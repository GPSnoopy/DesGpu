#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

#ifdef DESGPU_COMPILE_ALL_SALTS

void des_25_encrypt_salt3712(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 5, 22, 23, 24, 15, 16, 17, 18, 19, 20, 19, 4, 21, 6, 7, 8, 63, 32, 33, 34, 35, 36, 35, 52, 37, 54, 55, 56, 47, 48, 49, 50, 51, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3713(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 5, 22, 23, 24, 31, 16, 17, 18, 19, 20, 19, 4, 21, 6, 7, 8, 47, 32, 33, 34, 35, 36, 35, 52, 37, 54, 55, 56, 63, 48, 49, 50, 51, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3714(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 5, 22, 23, 24, 15, 0, 17, 18, 19, 20, 19, 4, 21, 6, 7, 8, 63, 48, 33, 34, 35, 36, 35, 52, 37, 54, 55, 56, 47, 32, 49, 50, 51, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3715(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 5, 22, 23, 24, 31, 0, 17, 18, 19, 20, 19, 4, 21, 6, 7, 8, 47, 48, 33, 34, 35, 36, 35, 52, 37, 54, 55, 56, 63, 32, 49, 50, 51, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3716(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 5, 22, 23, 24, 15, 16, 1, 18, 19, 20, 19, 4, 21, 6, 7, 8, 63, 32, 49, 34, 35, 36, 35, 52, 37, 54, 55, 56, 47, 48, 33, 50, 51, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3717(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 5, 22, 23, 24, 31, 16, 1, 18, 19, 20, 19, 4, 21, 6, 7, 8, 47, 32, 49, 34, 35, 36, 35, 52, 37, 54, 55, 56, 63, 48, 33, 50, 51, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3718(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 5, 22, 23, 24, 15, 0, 1, 18, 19, 20, 19, 4, 21, 6, 7, 8, 63, 48, 49, 34, 35, 36, 35, 52, 37, 54, 55, 56, 47, 32, 33, 50, 51, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3719(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 5, 22, 23, 24, 31, 0, 1, 18, 19, 20, 19, 4, 21, 6, 7, 8, 47, 48, 49, 34, 35, 36, 35, 52, 37, 54, 55, 56, 63, 32, 33, 50, 51, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3720(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 5, 22, 23, 24, 15, 16, 17, 2, 19, 20, 19, 4, 21, 6, 7, 8, 63, 32, 33, 50, 35, 36, 35, 52, 37, 54, 55, 56, 47, 48, 49, 34, 51, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3721(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 5, 22, 23, 24, 31, 16, 17, 2, 19, 20, 19, 4, 21, 6, 7, 8, 47, 32, 33, 50, 35, 36, 35, 52, 37, 54, 55, 56, 63, 48, 49, 34, 51, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3722(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 5, 22, 23, 24, 15, 0, 17, 2, 19, 20, 19, 4, 21, 6, 7, 8, 63, 48, 33, 50, 35, 36, 35, 52, 37, 54, 55, 56, 47, 32, 49, 34, 51, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3723(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 5, 22, 23, 24, 31, 0, 17, 2, 19, 20, 19, 4, 21, 6, 7, 8, 47, 48, 33, 50, 35, 36, 35, 52, 37, 54, 55, 56, 63, 32, 49, 34, 51, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3724(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 5, 22, 23, 24, 15, 16, 1, 2, 19, 20, 19, 4, 21, 6, 7, 8, 63, 32, 49, 50, 35, 36, 35, 52, 37, 54, 55, 56, 47, 48, 33, 34, 51, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3725(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 5, 22, 23, 24, 31, 16, 1, 2, 19, 20, 19, 4, 21, 6, 7, 8, 47, 32, 49, 50, 35, 36, 35, 52, 37, 54, 55, 56, 63, 48, 33, 34, 51, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3726(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 5, 22, 23, 24, 15, 0, 1, 2, 19, 20, 19, 4, 21, 6, 7, 8, 63, 48, 49, 50, 35, 36, 35, 52, 37, 54, 55, 56, 47, 32, 33, 34, 51, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3727(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 5, 22, 23, 24, 31, 0, 1, 2, 19, 20, 19, 4, 21, 6, 7, 8, 47, 48, 49, 50, 35, 36, 35, 52, 37, 54, 55, 56, 63, 32, 33, 34, 51, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3728(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 5, 22, 23, 24, 15, 16, 17, 18, 3, 20, 19, 4, 21, 6, 7, 8, 63, 32, 33, 34, 51, 36, 35, 52, 37, 54, 55, 56, 47, 48, 49, 50, 35, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3729(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 5, 22, 23, 24, 31, 16, 17, 18, 3, 20, 19, 4, 21, 6, 7, 8, 47, 32, 33, 34, 51, 36, 35, 52, 37, 54, 55, 56, 63, 48, 49, 50, 35, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3730(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 5, 22, 23, 24, 15, 0, 17, 18, 3, 20, 19, 4, 21, 6, 7, 8, 63, 48, 33, 34, 51, 36, 35, 52, 37, 54, 55, 56, 47, 32, 49, 50, 35, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3731(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 5, 22, 23, 24, 31, 0, 17, 18, 3, 20, 19, 4, 21, 6, 7, 8, 47, 48, 33, 34, 51, 36, 35, 52, 37, 54, 55, 56, 63, 32, 49, 50, 35, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3732(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 5, 22, 23, 24, 15, 16, 1, 18, 3, 20, 19, 4, 21, 6, 7, 8, 63, 32, 49, 34, 51, 36, 35, 52, 37, 54, 55, 56, 47, 48, 33, 50, 35, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3733(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 5, 22, 23, 24, 31, 16, 1, 18, 3, 20, 19, 4, 21, 6, 7, 8, 47, 32, 49, 34, 51, 36, 35, 52, 37, 54, 55, 56, 63, 48, 33, 50, 35, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3734(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 5, 22, 23, 24, 15, 0, 1, 18, 3, 20, 19, 4, 21, 6, 7, 8, 63, 48, 49, 34, 51, 36, 35, 52, 37, 54, 55, 56, 47, 32, 33, 50, 35, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3735(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 5, 22, 23, 24, 31, 0, 1, 18, 3, 20, 19, 4, 21, 6, 7, 8, 47, 48, 49, 34, 51, 36, 35, 52, 37, 54, 55, 56, 63, 32, 33, 50, 35, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3736(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 5, 22, 23, 24, 15, 16, 17, 2, 3, 20, 19, 4, 21, 6, 7, 8, 63, 32, 33, 50, 51, 36, 35, 52, 37, 54, 55, 56, 47, 48, 49, 34, 35, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3737(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 5, 22, 23, 24, 31, 16, 17, 2, 3, 20, 19, 4, 21, 6, 7, 8, 47, 32, 33, 50, 51, 36, 35, 52, 37, 54, 55, 56, 63, 48, 49, 34, 35, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3738(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 5, 22, 23, 24, 15, 0, 17, 2, 3, 20, 19, 4, 21, 6, 7, 8, 63, 48, 33, 50, 51, 36, 35, 52, 37, 54, 55, 56, 47, 32, 49, 34, 35, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3739(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 5, 22, 23, 24, 31, 0, 17, 2, 3, 20, 19, 4, 21, 6, 7, 8, 47, 48, 33, 50, 51, 36, 35, 52, 37, 54, 55, 56, 63, 32, 49, 34, 35, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3740(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 5, 22, 23, 24, 15, 16, 1, 2, 3, 20, 19, 4, 21, 6, 7, 8, 63, 32, 49, 50, 51, 36, 35, 52, 37, 54, 55, 56, 47, 48, 33, 34, 35, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3741(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 5, 22, 23, 24, 31, 16, 1, 2, 3, 20, 19, 4, 21, 6, 7, 8, 47, 32, 49, 50, 51, 36, 35, 52, 37, 54, 55, 56, 63, 48, 33, 34, 35, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3742(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 5, 22, 23, 24, 15, 0, 1, 2, 3, 20, 19, 4, 21, 6, 7, 8, 63, 48, 49, 50, 51, 36, 35, 52, 37, 54, 55, 56, 47, 32, 33, 34, 35, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3743(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 5, 22, 23, 24, 31, 0, 1, 2, 3, 20, 19, 4, 21, 6, 7, 8, 47, 48, 49, 50, 51, 36, 35, 52, 37, 54, 55, 56, 63, 32, 33, 34, 35, 52, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3744(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 5, 22, 23, 24, 15, 16, 17, 18, 19, 4, 19, 4, 21, 6, 7, 8, 63, 32, 33, 34, 35, 52, 35, 52, 37, 54, 55, 56, 47, 48, 49, 50, 51, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3745(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 5, 22, 23, 24, 31, 16, 17, 18, 19, 4, 19, 4, 21, 6, 7, 8, 47, 32, 33, 34, 35, 52, 35, 52, 37, 54, 55, 56, 63, 48, 49, 50, 51, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3746(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 5, 22, 23, 24, 15, 0, 17, 18, 19, 4, 19, 4, 21, 6, 7, 8, 63, 48, 33, 34, 35, 52, 35, 52, 37, 54, 55, 56, 47, 32, 49, 50, 51, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3747(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 5, 22, 23, 24, 31, 0, 17, 18, 19, 4, 19, 4, 21, 6, 7, 8, 47, 48, 33, 34, 35, 52, 35, 52, 37, 54, 55, 56, 63, 32, 49, 50, 51, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3748(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 5, 22, 23, 24, 15, 16, 1, 18, 19, 4, 19, 4, 21, 6, 7, 8, 63, 32, 49, 34, 35, 52, 35, 52, 37, 54, 55, 56, 47, 48, 33, 50, 51, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3749(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 5, 22, 23, 24, 31, 16, 1, 18, 19, 4, 19, 4, 21, 6, 7, 8, 47, 32, 49, 34, 35, 52, 35, 52, 37, 54, 55, 56, 63, 48, 33, 50, 51, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3750(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 5, 22, 23, 24, 15, 0, 1, 18, 19, 4, 19, 4, 21, 6, 7, 8, 63, 48, 49, 34, 35, 52, 35, 52, 37, 54, 55, 56, 47, 32, 33, 50, 51, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3751(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 5, 22, 23, 24, 31, 0, 1, 18, 19, 4, 19, 4, 21, 6, 7, 8, 47, 48, 49, 34, 35, 52, 35, 52, 37, 54, 55, 56, 63, 32, 33, 50, 51, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3752(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 5, 22, 23, 24, 15, 16, 17, 2, 19, 4, 19, 4, 21, 6, 7, 8, 63, 32, 33, 50, 35, 52, 35, 52, 37, 54, 55, 56, 47, 48, 49, 34, 51, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3753(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 5, 22, 23, 24, 31, 16, 17, 2, 19, 4, 19, 4, 21, 6, 7, 8, 47, 32, 33, 50, 35, 52, 35, 52, 37, 54, 55, 56, 63, 48, 49, 34, 51, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3754(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 5, 22, 23, 24, 15, 0, 17, 2, 19, 4, 19, 4, 21, 6, 7, 8, 63, 48, 33, 50, 35, 52, 35, 52, 37, 54, 55, 56, 47, 32, 49, 34, 51, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3755(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 5, 22, 23, 24, 31, 0, 17, 2, 19, 4, 19, 4, 21, 6, 7, 8, 47, 48, 33, 50, 35, 52, 35, 52, 37, 54, 55, 56, 63, 32, 49, 34, 51, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3756(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 5, 22, 23, 24, 15, 16, 1, 2, 19, 4, 19, 4, 21, 6, 7, 8, 63, 32, 49, 50, 35, 52, 35, 52, 37, 54, 55, 56, 47, 48, 33, 34, 51, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3757(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 5, 22, 23, 24, 31, 16, 1, 2, 19, 4, 19, 4, 21, 6, 7, 8, 47, 32, 49, 50, 35, 52, 35, 52, 37, 54, 55, 56, 63, 48, 33, 34, 51, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3758(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 5, 22, 23, 24, 15, 0, 1, 2, 19, 4, 19, 4, 21, 6, 7, 8, 63, 48, 49, 50, 35, 52, 35, 52, 37, 54, 55, 56, 47, 32, 33, 34, 51, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3759(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 5, 22, 23, 24, 31, 0, 1, 2, 19, 4, 19, 4, 21, 6, 7, 8, 47, 48, 49, 50, 35, 52, 35, 52, 37, 54, 55, 56, 63, 32, 33, 34, 51, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3760(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 5, 22, 23, 24, 15, 16, 17, 18, 3, 4, 19, 4, 21, 6, 7, 8, 63, 32, 33, 34, 51, 52, 35, 52, 37, 54, 55, 56, 47, 48, 49, 50, 35, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3761(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 5, 22, 23, 24, 31, 16, 17, 18, 3, 4, 19, 4, 21, 6, 7, 8, 47, 32, 33, 34, 51, 52, 35, 52, 37, 54, 55, 56, 63, 48, 49, 50, 35, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3762(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 5, 22, 23, 24, 15, 0, 17, 18, 3, 4, 19, 4, 21, 6, 7, 8, 63, 48, 33, 34, 51, 52, 35, 52, 37, 54, 55, 56, 47, 32, 49, 50, 35, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3763(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 5, 22, 23, 24, 31, 0, 17, 18, 3, 4, 19, 4, 21, 6, 7, 8, 47, 48, 33, 34, 51, 52, 35, 52, 37, 54, 55, 56, 63, 32, 49, 50, 35, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3764(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 5, 22, 23, 24, 15, 16, 1, 18, 3, 4, 19, 4, 21, 6, 7, 8, 63, 32, 49, 34, 51, 52, 35, 52, 37, 54, 55, 56, 47, 48, 33, 50, 35, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3765(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 5, 22, 23, 24, 31, 16, 1, 18, 3, 4, 19, 4, 21, 6, 7, 8, 47, 32, 49, 34, 51, 52, 35, 52, 37, 54, 55, 56, 63, 48, 33, 50, 35, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3766(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 5, 22, 23, 24, 15, 0, 1, 18, 3, 4, 19, 4, 21, 6, 7, 8, 63, 48, 49, 34, 51, 52, 35, 52, 37, 54, 55, 56, 47, 32, 33, 50, 35, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3767(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 5, 22, 23, 24, 31, 0, 1, 18, 3, 4, 19, 4, 21, 6, 7, 8, 47, 48, 49, 34, 51, 52, 35, 52, 37, 54, 55, 56, 63, 32, 33, 50, 35, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3768(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 5, 22, 23, 24, 15, 16, 17, 2, 3, 4, 19, 4, 21, 6, 7, 8, 63, 32, 33, 50, 51, 52, 35, 52, 37, 54, 55, 56, 47, 48, 49, 34, 35, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3769(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 5, 22, 23, 24, 31, 16, 17, 2, 3, 4, 19, 4, 21, 6, 7, 8, 47, 32, 33, 50, 51, 52, 35, 52, 37, 54, 55, 56, 63, 48, 49, 34, 35, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3770(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 5, 22, 23, 24, 15, 0, 17, 2, 3, 4, 19, 4, 21, 6, 7, 8, 63, 48, 33, 50, 51, 52, 35, 52, 37, 54, 55, 56, 47, 32, 49, 34, 35, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3771(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 5, 22, 23, 24, 31, 0, 17, 2, 3, 4, 19, 4, 21, 6, 7, 8, 47, 48, 33, 50, 51, 52, 35, 52, 37, 54, 55, 56, 63, 32, 49, 34, 35, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3772(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 5, 22, 23, 24, 15, 16, 1, 2, 3, 4, 19, 4, 21, 6, 7, 8, 63, 32, 49, 50, 51, 52, 35, 52, 37, 54, 55, 56, 47, 48, 33, 34, 35, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3773(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 5, 22, 23, 24, 31, 16, 1, 2, 3, 4, 19, 4, 21, 6, 7, 8, 47, 32, 49, 50, 51, 52, 35, 52, 37, 54, 55, 56, 63, 48, 33, 34, 35, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3774(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 5, 22, 23, 24, 15, 0, 1, 2, 3, 4, 19, 4, 21, 6, 7, 8, 63, 48, 49, 50, 51, 52, 35, 52, 37, 54, 55, 56, 47, 32, 33, 34, 35, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3775(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 5, 22, 23, 24, 31, 0, 1, 2, 3, 4, 19, 4, 21, 6, 7, 8, 47, 48, 49, 50, 51, 52, 35, 52, 37, 54, 55, 56, 63, 32, 33, 34, 35, 36, 51, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3776(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 5, 22, 23, 24, 15, 16, 17, 18, 19, 20, 3, 4, 21, 6, 7, 8, 63, 32, 33, 34, 35, 36, 51, 52, 37, 54, 55, 56, 47, 48, 49, 50, 51, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3777(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 5, 22, 23, 24, 31, 16, 17, 18, 19, 20, 3, 4, 21, 6, 7, 8, 47, 32, 33, 34, 35, 36, 51, 52, 37, 54, 55, 56, 63, 48, 49, 50, 51, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3778(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 5, 22, 23, 24, 15, 0, 17, 18, 19, 20, 3, 4, 21, 6, 7, 8, 63, 48, 33, 34, 35, 36, 51, 52, 37, 54, 55, 56, 47, 32, 49, 50, 51, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3779(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 5, 22, 23, 24, 31, 0, 17, 18, 19, 20, 3, 4, 21, 6, 7, 8, 47, 48, 33, 34, 35, 36, 51, 52, 37, 54, 55, 56, 63, 32, 49, 50, 51, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3780(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 5, 22, 23, 24, 15, 16, 1, 18, 19, 20, 3, 4, 21, 6, 7, 8, 63, 32, 49, 34, 35, 36, 51, 52, 37, 54, 55, 56, 47, 48, 33, 50, 51, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3781(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 5, 22, 23, 24, 31, 16, 1, 18, 19, 20, 3, 4, 21, 6, 7, 8, 47, 32, 49, 34, 35, 36, 51, 52, 37, 54, 55, 56, 63, 48, 33, 50, 51, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3782(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 5, 22, 23, 24, 15, 0, 1, 18, 19, 20, 3, 4, 21, 6, 7, 8, 63, 48, 49, 34, 35, 36, 51, 52, 37, 54, 55, 56, 47, 32, 33, 50, 51, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3783(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 5, 22, 23, 24, 31, 0, 1, 18, 19, 20, 3, 4, 21, 6, 7, 8, 47, 48, 49, 34, 35, 36, 51, 52, 37, 54, 55, 56, 63, 32, 33, 50, 51, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3784(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 5, 22, 23, 24, 15, 16, 17, 2, 19, 20, 3, 4, 21, 6, 7, 8, 63, 32, 33, 50, 35, 36, 51, 52, 37, 54, 55, 56, 47, 48, 49, 34, 51, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3785(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 5, 22, 23, 24, 31, 16, 17, 2, 19, 20, 3, 4, 21, 6, 7, 8, 47, 32, 33, 50, 35, 36, 51, 52, 37, 54, 55, 56, 63, 48, 49, 34, 51, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3786(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 5, 22, 23, 24, 15, 0, 17, 2, 19, 20, 3, 4, 21, 6, 7, 8, 63, 48, 33, 50, 35, 36, 51, 52, 37, 54, 55, 56, 47, 32, 49, 34, 51, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3787(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 5, 22, 23, 24, 31, 0, 17, 2, 19, 20, 3, 4, 21, 6, 7, 8, 47, 48, 33, 50, 35, 36, 51, 52, 37, 54, 55, 56, 63, 32, 49, 34, 51, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3788(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 5, 22, 23, 24, 15, 16, 1, 2, 19, 20, 3, 4, 21, 6, 7, 8, 63, 32, 49, 50, 35, 36, 51, 52, 37, 54, 55, 56, 47, 48, 33, 34, 51, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3789(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 5, 22, 23, 24, 31, 16, 1, 2, 19, 20, 3, 4, 21, 6, 7, 8, 47, 32, 49, 50, 35, 36, 51, 52, 37, 54, 55, 56, 63, 48, 33, 34, 51, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3790(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 5, 22, 23, 24, 15, 0, 1, 2, 19, 20, 3, 4, 21, 6, 7, 8, 63, 48, 49, 50, 35, 36, 51, 52, 37, 54, 55, 56, 47, 32, 33, 34, 51, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3791(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 5, 22, 23, 24, 31, 0, 1, 2, 19, 20, 3, 4, 21, 6, 7, 8, 47, 48, 49, 50, 35, 36, 51, 52, 37, 54, 55, 56, 63, 32, 33, 34, 51, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3792(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 5, 22, 23, 24, 15, 16, 17, 18, 3, 20, 3, 4, 21, 6, 7, 8, 63, 32, 33, 34, 51, 36, 51, 52, 37, 54, 55, 56, 47, 48, 49, 50, 35, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3793(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 5, 22, 23, 24, 31, 16, 17, 18, 3, 20, 3, 4, 21, 6, 7, 8, 47, 32, 33, 34, 51, 36, 51, 52, 37, 54, 55, 56, 63, 48, 49, 50, 35, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3794(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 5, 22, 23, 24, 15, 0, 17, 18, 3, 20, 3, 4, 21, 6, 7, 8, 63, 48, 33, 34, 51, 36, 51, 52, 37, 54, 55, 56, 47, 32, 49, 50, 35, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3795(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 5, 22, 23, 24, 31, 0, 17, 18, 3, 20, 3, 4, 21, 6, 7, 8, 47, 48, 33, 34, 51, 36, 51, 52, 37, 54, 55, 56, 63, 32, 49, 50, 35, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3796(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 5, 22, 23, 24, 15, 16, 1, 18, 3, 20, 3, 4, 21, 6, 7, 8, 63, 32, 49, 34, 51, 36, 51, 52, 37, 54, 55, 56, 47, 48, 33, 50, 35, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3797(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 5, 22, 23, 24, 31, 16, 1, 18, 3, 20, 3, 4, 21, 6, 7, 8, 47, 32, 49, 34, 51, 36, 51, 52, 37, 54, 55, 56, 63, 48, 33, 50, 35, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3798(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 5, 22, 23, 24, 15, 0, 1, 18, 3, 20, 3, 4, 21, 6, 7, 8, 63, 48, 49, 34, 51, 36, 51, 52, 37, 54, 55, 56, 47, 32, 33, 50, 35, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3799(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 5, 22, 23, 24, 31, 0, 1, 18, 3, 20, 3, 4, 21, 6, 7, 8, 47, 48, 49, 34, 51, 36, 51, 52, 37, 54, 55, 56, 63, 32, 33, 50, 35, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3800(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 5, 22, 23, 24, 15, 16, 17, 2, 3, 20, 3, 4, 21, 6, 7, 8, 63, 32, 33, 50, 51, 36, 51, 52, 37, 54, 55, 56, 47, 48, 49, 34, 35, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3801(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 5, 22, 23, 24, 31, 16, 17, 2, 3, 20, 3, 4, 21, 6, 7, 8, 47, 32, 33, 50, 51, 36, 51, 52, 37, 54, 55, 56, 63, 48, 49, 34, 35, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3802(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 5, 22, 23, 24, 15, 0, 17, 2, 3, 20, 3, 4, 21, 6, 7, 8, 63, 48, 33, 50, 51, 36, 51, 52, 37, 54, 55, 56, 47, 32, 49, 34, 35, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3803(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 5, 22, 23, 24, 31, 0, 17, 2, 3, 20, 3, 4, 21, 6, 7, 8, 47, 48, 33, 50, 51, 36, 51, 52, 37, 54, 55, 56, 63, 32, 49, 34, 35, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3804(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 5, 22, 23, 24, 15, 16, 1, 2, 3, 20, 3, 4, 21, 6, 7, 8, 63, 32, 49, 50, 51, 36, 51, 52, 37, 54, 55, 56, 47, 48, 33, 34, 35, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3805(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 5, 22, 23, 24, 31, 16, 1, 2, 3, 20, 3, 4, 21, 6, 7, 8, 47, 32, 49, 50, 51, 36, 51, 52, 37, 54, 55, 56, 63, 48, 33, 34, 35, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3806(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 5, 22, 23, 24, 15, 0, 1, 2, 3, 20, 3, 4, 21, 6, 7, 8, 63, 48, 49, 50, 51, 36, 51, 52, 37, 54, 55, 56, 47, 32, 33, 34, 35, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3807(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 5, 22, 23, 24, 31, 0, 1, 2, 3, 20, 3, 4, 21, 6, 7, 8, 47, 48, 49, 50, 51, 36, 51, 52, 37, 54, 55, 56, 63, 32, 33, 34, 35, 52, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3808(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 5, 22, 23, 24, 15, 16, 17, 18, 19, 4, 3, 4, 21, 6, 7, 8, 63, 32, 33, 34, 35, 52, 51, 52, 37, 54, 55, 56, 47, 48, 49, 50, 51, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3809(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 5, 22, 23, 24, 31, 16, 17, 18, 19, 4, 3, 4, 21, 6, 7, 8, 47, 32, 33, 34, 35, 52, 51, 52, 37, 54, 55, 56, 63, 48, 49, 50, 51, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3810(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 5, 22, 23, 24, 15, 0, 17, 18, 19, 4, 3, 4, 21, 6, 7, 8, 63, 48, 33, 34, 35, 52, 51, 52, 37, 54, 55, 56, 47, 32, 49, 50, 51, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3811(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 5, 22, 23, 24, 31, 0, 17, 18, 19, 4, 3, 4, 21, 6, 7, 8, 47, 48, 33, 34, 35, 52, 51, 52, 37, 54, 55, 56, 63, 32, 49, 50, 51, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3812(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 5, 22, 23, 24, 15, 16, 1, 18, 19, 4, 3, 4, 21, 6, 7, 8, 63, 32, 49, 34, 35, 52, 51, 52, 37, 54, 55, 56, 47, 48, 33, 50, 51, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3813(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 5, 22, 23, 24, 31, 16, 1, 18, 19, 4, 3, 4, 21, 6, 7, 8, 47, 32, 49, 34, 35, 52, 51, 52, 37, 54, 55, 56, 63, 48, 33, 50, 51, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3814(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 5, 22, 23, 24, 15, 0, 1, 18, 19, 4, 3, 4, 21, 6, 7, 8, 63, 48, 49, 34, 35, 52, 51, 52, 37, 54, 55, 56, 47, 32, 33, 50, 51, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3815(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 5, 22, 23, 24, 31, 0, 1, 18, 19, 4, 3, 4, 21, 6, 7, 8, 47, 48, 49, 34, 35, 52, 51, 52, 37, 54, 55, 56, 63, 32, 33, 50, 51, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3816(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 5, 22, 23, 24, 15, 16, 17, 2, 19, 4, 3, 4, 21, 6, 7, 8, 63, 32, 33, 50, 35, 52, 51, 52, 37, 54, 55, 56, 47, 48, 49, 34, 51, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3817(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 5, 22, 23, 24, 31, 16, 17, 2, 19, 4, 3, 4, 21, 6, 7, 8, 47, 32, 33, 50, 35, 52, 51, 52, 37, 54, 55, 56, 63, 48, 49, 34, 51, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3818(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 5, 22, 23, 24, 15, 0, 17, 2, 19, 4, 3, 4, 21, 6, 7, 8, 63, 48, 33, 50, 35, 52, 51, 52, 37, 54, 55, 56, 47, 32, 49, 34, 51, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3819(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 5, 22, 23, 24, 31, 0, 17, 2, 19, 4, 3, 4, 21, 6, 7, 8, 47, 48, 33, 50, 35, 52, 51, 52, 37, 54, 55, 56, 63, 32, 49, 34, 51, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3820(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 5, 22, 23, 24, 15, 16, 1, 2, 19, 4, 3, 4, 21, 6, 7, 8, 63, 32, 49, 50, 35, 52, 51, 52, 37, 54, 55, 56, 47, 48, 33, 34, 51, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3821(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 5, 22, 23, 24, 31, 16, 1, 2, 19, 4, 3, 4, 21, 6, 7, 8, 47, 32, 49, 50, 35, 52, 51, 52, 37, 54, 55, 56, 63, 48, 33, 34, 51, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3822(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 5, 22, 23, 24, 15, 0, 1, 2, 19, 4, 3, 4, 21, 6, 7, 8, 63, 48, 49, 50, 35, 52, 51, 52, 37, 54, 55, 56, 47, 32, 33, 34, 51, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3823(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 5, 22, 23, 24, 31, 0, 1, 2, 19, 4, 3, 4, 21, 6, 7, 8, 47, 48, 49, 50, 35, 52, 51, 52, 37, 54, 55, 56, 63, 32, 33, 34, 51, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3824(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 5, 22, 23, 24, 15, 16, 17, 18, 3, 4, 3, 4, 21, 6, 7, 8, 63, 32, 33, 34, 51, 52, 51, 52, 37, 54, 55, 56, 47, 48, 49, 50, 35, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3825(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 5, 22, 23, 24, 31, 16, 17, 18, 3, 4, 3, 4, 21, 6, 7, 8, 47, 32, 33, 34, 51, 52, 51, 52, 37, 54, 55, 56, 63, 48, 49, 50, 35, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3826(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 5, 22, 23, 24, 15, 0, 17, 18, 3, 4, 3, 4, 21, 6, 7, 8, 63, 48, 33, 34, 51, 52, 51, 52, 37, 54, 55, 56, 47, 32, 49, 50, 35, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3827(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 5, 22, 23, 24, 31, 0, 17, 18, 3, 4, 3, 4, 21, 6, 7, 8, 47, 48, 33, 34, 51, 52, 51, 52, 37, 54, 55, 56, 63, 32, 49, 50, 35, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3828(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 5, 22, 23, 24, 15, 16, 1, 18, 3, 4, 3, 4, 21, 6, 7, 8, 63, 32, 49, 34, 51, 52, 51, 52, 37, 54, 55, 56, 47, 48, 33, 50, 35, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3829(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 5, 22, 23, 24, 31, 16, 1, 18, 3, 4, 3, 4, 21, 6, 7, 8, 47, 32, 49, 34, 51, 52, 51, 52, 37, 54, 55, 56, 63, 48, 33, 50, 35, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3830(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 5, 22, 23, 24, 15, 0, 1, 18, 3, 4, 3, 4, 21, 6, 7, 8, 63, 48, 49, 34, 51, 52, 51, 52, 37, 54, 55, 56, 47, 32, 33, 50, 35, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3831(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 5, 22, 23, 24, 31, 0, 1, 18, 3, 4, 3, 4, 21, 6, 7, 8, 47, 48, 49, 34, 51, 52, 51, 52, 37, 54, 55, 56, 63, 32, 33, 50, 35, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3832(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 5, 22, 23, 24, 15, 16, 17, 2, 3, 4, 3, 4, 21, 6, 7, 8, 63, 32, 33, 50, 51, 52, 51, 52, 37, 54, 55, 56, 47, 48, 49, 34, 35, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3833(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 5, 22, 23, 24, 31, 16, 17, 2, 3, 4, 3, 4, 21, 6, 7, 8, 47, 32, 33, 50, 51, 52, 51, 52, 37, 54, 55, 56, 63, 48, 49, 34, 35, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3834(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 5, 22, 23, 24, 15, 0, 17, 2, 3, 4, 3, 4, 21, 6, 7, 8, 63, 48, 33, 50, 51, 52, 51, 52, 37, 54, 55, 56, 47, 32, 49, 34, 35, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3835(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 5, 22, 23, 24, 31, 0, 17, 2, 3, 4, 3, 4, 21, 6, 7, 8, 47, 48, 33, 50, 51, 52, 51, 52, 37, 54, 55, 56, 63, 32, 49, 34, 35, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3836(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 5, 22, 23, 24, 15, 16, 1, 2, 3, 4, 3, 4, 21, 6, 7, 8, 63, 32, 49, 50, 51, 52, 51, 52, 37, 54, 55, 56, 47, 48, 33, 34, 35, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3837(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 5, 22, 23, 24, 31, 16, 1, 2, 3, 4, 3, 4, 21, 6, 7, 8, 47, 32, 49, 50, 51, 52, 51, 52, 37, 54, 55, 56, 63, 48, 33, 34, 35, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3838(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 5, 22, 23, 24, 15, 0, 1, 2, 3, 4, 3, 4, 21, 6, 7, 8, 63, 48, 49, 50, 51, 52, 51, 52, 37, 54, 55, 56, 47, 32, 33, 34, 35, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3839(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 5, 22, 23, 24, 31, 0, 1, 2, 3, 4, 3, 4, 21, 6, 7, 8, 47, 48, 49, 50, 51, 52, 51, 52, 37, 54, 55, 56, 63, 32, 33, 34, 35, 36, 35, 36, 53, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

#endif // DESGPU_COMPILE_ALL_SALTS


