#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt3712(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 5, 22, 23, 24, 15, 16, 17, 18, 19, 20, 19, 4, 21, 6, 7, 8, 63, 32, 33, 34, 35, 36, 35, 52, 37, 54, 55, 56, 47, 48, 49, 50, 51, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3713(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 5, 22, 23, 24, 31, 16, 17, 18, 19, 20, 19, 4, 21, 6, 7, 8, 47, 32, 33, 34, 35, 36, 35, 52, 37, 54, 55, 56, 63, 48, 49, 50, 51, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3714(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 5, 22, 23, 24, 15, 0, 17, 18, 19, 20, 19, 4, 21, 6, 7, 8, 63, 48, 33, 34, 35, 36, 35, 52, 37, 54, 55, 56, 47, 32, 49, 50, 51, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3715(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 5, 22, 23, 24, 31, 0, 17, 18, 19, 20, 19, 4, 21, 6, 7, 8, 47, 48, 33, 34, 35, 36, 35, 52, 37, 54, 55, 56, 63, 32, 49, 50, 51, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3716(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 5, 22, 23, 24, 15, 16, 1, 18, 19, 20, 19, 4, 21, 6, 7, 8, 63, 32, 49, 34, 35, 36, 35, 52, 37, 54, 55, 56, 47, 48, 33, 50, 51, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3717(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 5, 22, 23, 24, 31, 16, 1, 18, 19, 20, 19, 4, 21, 6, 7, 8, 47, 32, 49, 34, 35, 36, 35, 52, 37, 54, 55, 56, 63, 48, 33, 50, 51, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3718(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 5, 22, 23, 24, 15, 0, 1, 18, 19, 20, 19, 4, 21, 6, 7, 8, 63, 48, 49, 34, 35, 36, 35, 52, 37, 54, 55, 56, 47, 32, 33, 50, 51, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3719(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 5, 22, 23, 24, 31, 0, 1, 18, 19, 20, 19, 4, 21, 6, 7, 8, 47, 48, 49, 34, 35, 36, 35, 52, 37, 54, 55, 56, 63, 32, 33, 50, 51, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3720(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 5, 22, 23, 24, 15, 16, 17, 2, 19, 20, 19, 4, 21, 6, 7, 8, 63, 32, 33, 50, 35, 36, 35, 52, 37, 54, 55, 56, 47, 48, 49, 34, 51, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3721(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 5, 22, 23, 24, 31, 16, 17, 2, 19, 20, 19, 4, 21, 6, 7, 8, 47, 32, 33, 50, 35, 36, 35, 52, 37, 54, 55, 56, 63, 48, 49, 34, 51, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3722(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 5, 22, 23, 24, 15, 0, 17, 2, 19, 20, 19, 4, 21, 6, 7, 8, 63, 48, 33, 50, 35, 36, 35, 52, 37, 54, 55, 56, 47, 32, 49, 34, 51, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3723(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 5, 22, 23, 24, 31, 0, 17, 2, 19, 20, 19, 4, 21, 6, 7, 8, 47, 48, 33, 50, 35, 36, 35, 52, 37, 54, 55, 56, 63, 32, 49, 34, 51, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3724(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 5, 22, 23, 24, 15, 16, 1, 2, 19, 20, 19, 4, 21, 6, 7, 8, 63, 32, 49, 50, 35, 36, 35, 52, 37, 54, 55, 56, 47, 48, 33, 34, 51, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3725(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 5, 22, 23, 24, 31, 16, 1, 2, 19, 20, 19, 4, 21, 6, 7, 8, 47, 32, 49, 50, 35, 36, 35, 52, 37, 54, 55, 56, 63, 48, 33, 34, 51, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3726(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 5, 22, 23, 24, 15, 0, 1, 2, 19, 20, 19, 4, 21, 6, 7, 8, 63, 48, 49, 50, 35, 36, 35, 52, 37, 54, 55, 56, 47, 32, 33, 34, 51, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3727(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 5, 22, 23, 24, 31, 0, 1, 2, 19, 20, 19, 4, 21, 6, 7, 8, 47, 48, 49, 50, 35, 36, 35, 52, 37, 54, 55, 56, 63, 32, 33, 34, 51, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3728(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 5, 22, 23, 24, 15, 16, 17, 18, 3, 20, 19, 4, 21, 6, 7, 8, 63, 32, 33, 34, 51, 36, 35, 52, 37, 54, 55, 56, 47, 48, 49, 50, 35, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3729(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 5, 22, 23, 24, 31, 16, 17, 18, 3, 20, 19, 4, 21, 6, 7, 8, 47, 32, 33, 34, 51, 36, 35, 52, 37, 54, 55, 56, 63, 48, 49, 50, 35, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3730(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 5, 22, 23, 24, 15, 0, 17, 18, 3, 20, 19, 4, 21, 6, 7, 8, 63, 48, 33, 34, 51, 36, 35, 52, 37, 54, 55, 56, 47, 32, 49, 50, 35, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3731(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 5, 22, 23, 24, 31, 0, 17, 18, 3, 20, 19, 4, 21, 6, 7, 8, 47, 48, 33, 34, 51, 36, 35, 52, 37, 54, 55, 56, 63, 32, 49, 50, 35, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3732(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 5, 22, 23, 24, 15, 16, 1, 18, 3, 20, 19, 4, 21, 6, 7, 8, 63, 32, 49, 34, 51, 36, 35, 52, 37, 54, 55, 56, 47, 48, 33, 50, 35, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3733(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 5, 22, 23, 24, 31, 16, 1, 18, 3, 20, 19, 4, 21, 6, 7, 8, 47, 32, 49, 34, 51, 36, 35, 52, 37, 54, 55, 56, 63, 48, 33, 50, 35, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3734(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 5, 22, 23, 24, 15, 0, 1, 18, 3, 20, 19, 4, 21, 6, 7, 8, 63, 48, 49, 34, 51, 36, 35, 52, 37, 54, 55, 56, 47, 32, 33, 50, 35, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3735(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 5, 22, 23, 24, 31, 0, 1, 18, 3, 20, 19, 4, 21, 6, 7, 8, 47, 48, 49, 34, 51, 36, 35, 52, 37, 54, 55, 56, 63, 32, 33, 50, 35, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3736(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 5, 22, 23, 24, 15, 16, 17, 2, 3, 20, 19, 4, 21, 6, 7, 8, 63, 32, 33, 50, 51, 36, 35, 52, 37, 54, 55, 56, 47, 48, 49, 34, 35, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3737(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 5, 22, 23, 24, 31, 16, 17, 2, 3, 20, 19, 4, 21, 6, 7, 8, 47, 32, 33, 50, 51, 36, 35, 52, 37, 54, 55, 56, 63, 48, 49, 34, 35, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3738(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 5, 22, 23, 24, 15, 0, 17, 2, 3, 20, 19, 4, 21, 6, 7, 8, 63, 48, 33, 50, 51, 36, 35, 52, 37, 54, 55, 56, 47, 32, 49, 34, 35, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3739(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 5, 22, 23, 24, 31, 0, 17, 2, 3, 20, 19, 4, 21, 6, 7, 8, 47, 48, 33, 50, 51, 36, 35, 52, 37, 54, 55, 56, 63, 32, 49, 34, 35, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3740(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 5, 22, 23, 24, 15, 16, 1, 2, 3, 20, 19, 4, 21, 6, 7, 8, 63, 32, 49, 50, 51, 36, 35, 52, 37, 54, 55, 56, 47, 48, 33, 34, 35, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3741(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 5, 22, 23, 24, 31, 16, 1, 2, 3, 20, 19, 4, 21, 6, 7, 8, 47, 32, 49, 50, 51, 36, 35, 52, 37, 54, 55, 56, 63, 48, 33, 34, 35, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3742(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 5, 22, 23, 24, 15, 0, 1, 2, 3, 20, 19, 4, 21, 6, 7, 8, 63, 48, 49, 50, 51, 36, 35, 52, 37, 54, 55, 56, 47, 32, 33, 34, 35, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3743(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 5, 22, 23, 24, 31, 0, 1, 2, 3, 20, 19, 4, 21, 6, 7, 8, 47, 48, 49, 50, 51, 36, 35, 52, 37, 54, 55, 56, 63, 32, 33, 34, 35, 52, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3744(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 5, 22, 23, 24, 15, 16, 17, 18, 19, 4, 19, 4, 21, 6, 7, 8, 63, 32, 33, 34, 35, 52, 35, 52, 37, 54, 55, 56, 47, 48, 49, 50, 51, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3745(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 5, 22, 23, 24, 31, 16, 17, 18, 19, 4, 19, 4, 21, 6, 7, 8, 47, 32, 33, 34, 35, 52, 35, 52, 37, 54, 55, 56, 63, 48, 49, 50, 51, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3746(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 5, 22, 23, 24, 15, 0, 17, 18, 19, 4, 19, 4, 21, 6, 7, 8, 63, 48, 33, 34, 35, 52, 35, 52, 37, 54, 55, 56, 47, 32, 49, 50, 51, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3747(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 5, 22, 23, 24, 31, 0, 17, 18, 19, 4, 19, 4, 21, 6, 7, 8, 47, 48, 33, 34, 35, 52, 35, 52, 37, 54, 55, 56, 63, 32, 49, 50, 51, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3748(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 5, 22, 23, 24, 15, 16, 1, 18, 19, 4, 19, 4, 21, 6, 7, 8, 63, 32, 49, 34, 35, 52, 35, 52, 37, 54, 55, 56, 47, 48, 33, 50, 51, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3749(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 5, 22, 23, 24, 31, 16, 1, 18, 19, 4, 19, 4, 21, 6, 7, 8, 47, 32, 49, 34, 35, 52, 35, 52, 37, 54, 55, 56, 63, 48, 33, 50, 51, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3750(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 5, 22, 23, 24, 15, 0, 1, 18, 19, 4, 19, 4, 21, 6, 7, 8, 63, 48, 49, 34, 35, 52, 35, 52, 37, 54, 55, 56, 47, 32, 33, 50, 51, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3751(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 5, 22, 23, 24, 31, 0, 1, 18, 19, 4, 19, 4, 21, 6, 7, 8, 47, 48, 49, 34, 35, 52, 35, 52, 37, 54, 55, 56, 63, 32, 33, 50, 51, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3752(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 5, 22, 23, 24, 15, 16, 17, 2, 19, 4, 19, 4, 21, 6, 7, 8, 63, 32, 33, 50, 35, 52, 35, 52, 37, 54, 55, 56, 47, 48, 49, 34, 51, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3753(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 5, 22, 23, 24, 31, 16, 17, 2, 19, 4, 19, 4, 21, 6, 7, 8, 47, 32, 33, 50, 35, 52, 35, 52, 37, 54, 55, 56, 63, 48, 49, 34, 51, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3754(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 5, 22, 23, 24, 15, 0, 17, 2, 19, 4, 19, 4, 21, 6, 7, 8, 63, 48, 33, 50, 35, 52, 35, 52, 37, 54, 55, 56, 47, 32, 49, 34, 51, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3755(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 5, 22, 23, 24, 31, 0, 17, 2, 19, 4, 19, 4, 21, 6, 7, 8, 47, 48, 33, 50, 35, 52, 35, 52, 37, 54, 55, 56, 63, 32, 49, 34, 51, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3756(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 5, 22, 23, 24, 15, 16, 1, 2, 19, 4, 19, 4, 21, 6, 7, 8, 63, 32, 49, 50, 35, 52, 35, 52, 37, 54, 55, 56, 47, 48, 33, 34, 51, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3757(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 5, 22, 23, 24, 31, 16, 1, 2, 19, 4, 19, 4, 21, 6, 7, 8, 47, 32, 49, 50, 35, 52, 35, 52, 37, 54, 55, 56, 63, 48, 33, 34, 51, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3758(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 5, 22, 23, 24, 15, 0, 1, 2, 19, 4, 19, 4, 21, 6, 7, 8, 63, 48, 49, 50, 35, 52, 35, 52, 37, 54, 55, 56, 47, 32, 33, 34, 51, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3759(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 5, 22, 23, 24, 31, 0, 1, 2, 19, 4, 19, 4, 21, 6, 7, 8, 47, 48, 49, 50, 35, 52, 35, 52, 37, 54, 55, 56, 63, 32, 33, 34, 51, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3760(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 5, 22, 23, 24, 15, 16, 17, 18, 3, 4, 19, 4, 21, 6, 7, 8, 63, 32, 33, 34, 51, 52, 35, 52, 37, 54, 55, 56, 47, 48, 49, 50, 35, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3761(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 5, 22, 23, 24, 31, 16, 17, 18, 3, 4, 19, 4, 21, 6, 7, 8, 47, 32, 33, 34, 51, 52, 35, 52, 37, 54, 55, 56, 63, 48, 49, 50, 35, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3762(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 5, 22, 23, 24, 15, 0, 17, 18, 3, 4, 19, 4, 21, 6, 7, 8, 63, 48, 33, 34, 51, 52, 35, 52, 37, 54, 55, 56, 47, 32, 49, 50, 35, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3763(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 5, 22, 23, 24, 31, 0, 17, 18, 3, 4, 19, 4, 21, 6, 7, 8, 47, 48, 33, 34, 51, 52, 35, 52, 37, 54, 55, 56, 63, 32, 49, 50, 35, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3764(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 5, 22, 23, 24, 15, 16, 1, 18, 3, 4, 19, 4, 21, 6, 7, 8, 63, 32, 49, 34, 51, 52, 35, 52, 37, 54, 55, 56, 47, 48, 33, 50, 35, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3765(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 5, 22, 23, 24, 31, 16, 1, 18, 3, 4, 19, 4, 21, 6, 7, 8, 47, 32, 49, 34, 51, 52, 35, 52, 37, 54, 55, 56, 63, 48, 33, 50, 35, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3766(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 5, 22, 23, 24, 15, 0, 1, 18, 3, 4, 19, 4, 21, 6, 7, 8, 63, 48, 49, 34, 51, 52, 35, 52, 37, 54, 55, 56, 47, 32, 33, 50, 35, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3767(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 5, 22, 23, 24, 31, 0, 1, 18, 3, 4, 19, 4, 21, 6, 7, 8, 47, 48, 49, 34, 51, 52, 35, 52, 37, 54, 55, 56, 63, 32, 33, 50, 35, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3768(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 5, 22, 23, 24, 15, 16, 17, 2, 3, 4, 19, 4, 21, 6, 7, 8, 63, 32, 33, 50, 51, 52, 35, 52, 37, 54, 55, 56, 47, 48, 49, 34, 35, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3769(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 5, 22, 23, 24, 31, 16, 17, 2, 3, 4, 19, 4, 21, 6, 7, 8, 47, 32, 33, 50, 51, 52, 35, 52, 37, 54, 55, 56, 63, 48, 49, 34, 35, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3770(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 5, 22, 23, 24, 15, 0, 17, 2, 3, 4, 19, 4, 21, 6, 7, 8, 63, 48, 33, 50, 51, 52, 35, 52, 37, 54, 55, 56, 47, 32, 49, 34, 35, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3771(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 5, 22, 23, 24, 31, 0, 17, 2, 3, 4, 19, 4, 21, 6, 7, 8, 47, 48, 33, 50, 51, 52, 35, 52, 37, 54, 55, 56, 63, 32, 49, 34, 35, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3772(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 5, 22, 23, 24, 15, 16, 1, 2, 3, 4, 19, 4, 21, 6, 7, 8, 63, 32, 49, 50, 51, 52, 35, 52, 37, 54, 55, 56, 47, 48, 33, 34, 35, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3773(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 5, 22, 23, 24, 31, 16, 1, 2, 3, 4, 19, 4, 21, 6, 7, 8, 47, 32, 49, 50, 51, 52, 35, 52, 37, 54, 55, 56, 63, 48, 33, 34, 35, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3774(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 5, 22, 23, 24, 15, 0, 1, 2, 3, 4, 19, 4, 21, 6, 7, 8, 63, 48, 49, 50, 51, 52, 35, 52, 37, 54, 55, 56, 47, 32, 33, 34, 35, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3775(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 5, 22, 23, 24, 31, 0, 1, 2, 3, 4, 19, 4, 21, 6, 7, 8, 47, 48, 49, 50, 51, 52, 35, 52, 37, 54, 55, 56, 63, 32, 33, 34, 35, 36, 51, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3776(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 5, 22, 23, 24, 15, 16, 17, 18, 19, 20, 3, 4, 21, 6, 7, 8, 63, 32, 33, 34, 35, 36, 51, 52, 37, 54, 55, 56, 47, 48, 49, 50, 51, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3777(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 5, 22, 23, 24, 31, 16, 17, 18, 19, 20, 3, 4, 21, 6, 7, 8, 47, 32, 33, 34, 35, 36, 51, 52, 37, 54, 55, 56, 63, 48, 49, 50, 51, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3778(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 5, 22, 23, 24, 15, 0, 17, 18, 19, 20, 3, 4, 21, 6, 7, 8, 63, 48, 33, 34, 35, 36, 51, 52, 37, 54, 55, 56, 47, 32, 49, 50, 51, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3779(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 5, 22, 23, 24, 31, 0, 17, 18, 19, 20, 3, 4, 21, 6, 7, 8, 47, 48, 33, 34, 35, 36, 51, 52, 37, 54, 55, 56, 63, 32, 49, 50, 51, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3780(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 5, 22, 23, 24, 15, 16, 1, 18, 19, 20, 3, 4, 21, 6, 7, 8, 63, 32, 49, 34, 35, 36, 51, 52, 37, 54, 55, 56, 47, 48, 33, 50, 51, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3781(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 5, 22, 23, 24, 31, 16, 1, 18, 19, 20, 3, 4, 21, 6, 7, 8, 47, 32, 49, 34, 35, 36, 51, 52, 37, 54, 55, 56, 63, 48, 33, 50, 51, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3782(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 5, 22, 23, 24, 15, 0, 1, 18, 19, 20, 3, 4, 21, 6, 7, 8, 63, 48, 49, 34, 35, 36, 51, 52, 37, 54, 55, 56, 47, 32, 33, 50, 51, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3783(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 5, 22, 23, 24, 31, 0, 1, 18, 19, 20, 3, 4, 21, 6, 7, 8, 47, 48, 49, 34, 35, 36, 51, 52, 37, 54, 55, 56, 63, 32, 33, 50, 51, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3784(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 5, 22, 23, 24, 15, 16, 17, 2, 19, 20, 3, 4, 21, 6, 7, 8, 63, 32, 33, 50, 35, 36, 51, 52, 37, 54, 55, 56, 47, 48, 49, 34, 51, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3785(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 5, 22, 23, 24, 31, 16, 17, 2, 19, 20, 3, 4, 21, 6, 7, 8, 47, 32, 33, 50, 35, 36, 51, 52, 37, 54, 55, 56, 63, 48, 49, 34, 51, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3786(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 5, 22, 23, 24, 15, 0, 17, 2, 19, 20, 3, 4, 21, 6, 7, 8, 63, 48, 33, 50, 35, 36, 51, 52, 37, 54, 55, 56, 47, 32, 49, 34, 51, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3787(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 5, 22, 23, 24, 31, 0, 17, 2, 19, 20, 3, 4, 21, 6, 7, 8, 47, 48, 33, 50, 35, 36, 51, 52, 37, 54, 55, 56, 63, 32, 49, 34, 51, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3788(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 5, 22, 23, 24, 15, 16, 1, 2, 19, 20, 3, 4, 21, 6, 7, 8, 63, 32, 49, 50, 35, 36, 51, 52, 37, 54, 55, 56, 47, 48, 33, 34, 51, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3789(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 5, 22, 23, 24, 31, 16, 1, 2, 19, 20, 3, 4, 21, 6, 7, 8, 47, 32, 49, 50, 35, 36, 51, 52, 37, 54, 55, 56, 63, 48, 33, 34, 51, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3790(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 5, 22, 23, 24, 15, 0, 1, 2, 19, 20, 3, 4, 21, 6, 7, 8, 63, 48, 49, 50, 35, 36, 51, 52, 37, 54, 55, 56, 47, 32, 33, 34, 51, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3791(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 5, 22, 23, 24, 31, 0, 1, 2, 19, 20, 3, 4, 21, 6, 7, 8, 47, 48, 49, 50, 35, 36, 51, 52, 37, 54, 55, 56, 63, 32, 33, 34, 51, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3792(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 5, 22, 23, 24, 15, 16, 17, 18, 3, 20, 3, 4, 21, 6, 7, 8, 63, 32, 33, 34, 51, 36, 51, 52, 37, 54, 55, 56, 47, 48, 49, 50, 35, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3793(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 5, 22, 23, 24, 31, 16, 17, 18, 3, 20, 3, 4, 21, 6, 7, 8, 47, 32, 33, 34, 51, 36, 51, 52, 37, 54, 55, 56, 63, 48, 49, 50, 35, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3794(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 5, 22, 23, 24, 15, 0, 17, 18, 3, 20, 3, 4, 21, 6, 7, 8, 63, 48, 33, 34, 51, 36, 51, 52, 37, 54, 55, 56, 47, 32, 49, 50, 35, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3795(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 5, 22, 23, 24, 31, 0, 17, 18, 3, 20, 3, 4, 21, 6, 7, 8, 47, 48, 33, 34, 51, 36, 51, 52, 37, 54, 55, 56, 63, 32, 49, 50, 35, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3796(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 5, 22, 23, 24, 15, 16, 1, 18, 3, 20, 3, 4, 21, 6, 7, 8, 63, 32, 49, 34, 51, 36, 51, 52, 37, 54, 55, 56, 47, 48, 33, 50, 35, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3797(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 5, 22, 23, 24, 31, 16, 1, 18, 3, 20, 3, 4, 21, 6, 7, 8, 47, 32, 49, 34, 51, 36, 51, 52, 37, 54, 55, 56, 63, 48, 33, 50, 35, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3798(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 5, 22, 23, 24, 15, 0, 1, 18, 3, 20, 3, 4, 21, 6, 7, 8, 63, 48, 49, 34, 51, 36, 51, 52, 37, 54, 55, 56, 47, 32, 33, 50, 35, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3799(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 5, 22, 23, 24, 31, 0, 1, 18, 3, 20, 3, 4, 21, 6, 7, 8, 47, 48, 49, 34, 51, 36, 51, 52, 37, 54, 55, 56, 63, 32, 33, 50, 35, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3800(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 5, 22, 23, 24, 15, 16, 17, 2, 3, 20, 3, 4, 21, 6, 7, 8, 63, 32, 33, 50, 51, 36, 51, 52, 37, 54, 55, 56, 47, 48, 49, 34, 35, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3801(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 5, 22, 23, 24, 31, 16, 17, 2, 3, 20, 3, 4, 21, 6, 7, 8, 47, 32, 33, 50, 51, 36, 51, 52, 37, 54, 55, 56, 63, 48, 49, 34, 35, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3802(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 5, 22, 23, 24, 15, 0, 17, 2, 3, 20, 3, 4, 21, 6, 7, 8, 63, 48, 33, 50, 51, 36, 51, 52, 37, 54, 55, 56, 47, 32, 49, 34, 35, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3803(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 5, 22, 23, 24, 31, 0, 17, 2, 3, 20, 3, 4, 21, 6, 7, 8, 47, 48, 33, 50, 51, 36, 51, 52, 37, 54, 55, 56, 63, 32, 49, 34, 35, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3804(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 5, 22, 23, 24, 15, 16, 1, 2, 3, 20, 3, 4, 21, 6, 7, 8, 63, 32, 49, 50, 51, 36, 51, 52, 37, 54, 55, 56, 47, 48, 33, 34, 35, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3805(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 5, 22, 23, 24, 31, 16, 1, 2, 3, 20, 3, 4, 21, 6, 7, 8, 47, 32, 49, 50, 51, 36, 51, 52, 37, 54, 55, 56, 63, 48, 33, 34, 35, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3806(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 5, 22, 23, 24, 15, 0, 1, 2, 3, 20, 3, 4, 21, 6, 7, 8, 63, 48, 49, 50, 51, 36, 51, 52, 37, 54, 55, 56, 47, 32, 33, 34, 35, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3807(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 5, 22, 23, 24, 31, 0, 1, 2, 3, 20, 3, 4, 21, 6, 7, 8, 47, 48, 49, 50, 51, 36, 51, 52, 37, 54, 55, 56, 63, 32, 33, 34, 35, 52, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3808(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 5, 22, 23, 24, 15, 16, 17, 18, 19, 4, 3, 4, 21, 6, 7, 8, 63, 32, 33, 34, 35, 52, 51, 52, 37, 54, 55, 56, 47, 48, 49, 50, 51, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3809(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 5, 22, 23, 24, 31, 16, 17, 18, 19, 4, 3, 4, 21, 6, 7, 8, 47, 32, 33, 34, 35, 52, 51, 52, 37, 54, 55, 56, 63, 48, 49, 50, 51, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3810(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 5, 22, 23, 24, 15, 0, 17, 18, 19, 4, 3, 4, 21, 6, 7, 8, 63, 48, 33, 34, 35, 52, 51, 52, 37, 54, 55, 56, 47, 32, 49, 50, 51, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3811(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 5, 22, 23, 24, 31, 0, 17, 18, 19, 4, 3, 4, 21, 6, 7, 8, 47, 48, 33, 34, 35, 52, 51, 52, 37, 54, 55, 56, 63, 32, 49, 50, 51, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3812(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 5, 22, 23, 24, 15, 16, 1, 18, 19, 4, 3, 4, 21, 6, 7, 8, 63, 32, 49, 34, 35, 52, 51, 52, 37, 54, 55, 56, 47, 48, 33, 50, 51, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3813(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 5, 22, 23, 24, 31, 16, 1, 18, 19, 4, 3, 4, 21, 6, 7, 8, 47, 32, 49, 34, 35, 52, 51, 52, 37, 54, 55, 56, 63, 48, 33, 50, 51, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3814(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 5, 22, 23, 24, 15, 0, 1, 18, 19, 4, 3, 4, 21, 6, 7, 8, 63, 48, 49, 34, 35, 52, 51, 52, 37, 54, 55, 56, 47, 32, 33, 50, 51, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3815(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 5, 22, 23, 24, 31, 0, 1, 18, 19, 4, 3, 4, 21, 6, 7, 8, 47, 48, 49, 34, 35, 52, 51, 52, 37, 54, 55, 56, 63, 32, 33, 50, 51, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3816(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 5, 22, 23, 24, 15, 16, 17, 2, 19, 4, 3, 4, 21, 6, 7, 8, 63, 32, 33, 50, 35, 52, 51, 52, 37, 54, 55, 56, 47, 48, 49, 34, 51, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3817(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 5, 22, 23, 24, 31, 16, 17, 2, 19, 4, 3, 4, 21, 6, 7, 8, 47, 32, 33, 50, 35, 52, 51, 52, 37, 54, 55, 56, 63, 48, 49, 34, 51, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3818(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 5, 22, 23, 24, 15, 0, 17, 2, 19, 4, 3, 4, 21, 6, 7, 8, 63, 48, 33, 50, 35, 52, 51, 52, 37, 54, 55, 56, 47, 32, 49, 34, 51, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3819(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 5, 22, 23, 24, 31, 0, 17, 2, 19, 4, 3, 4, 21, 6, 7, 8, 47, 48, 33, 50, 35, 52, 51, 52, 37, 54, 55, 56, 63, 32, 49, 34, 51, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3820(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 5, 22, 23, 24, 15, 16, 1, 2, 19, 4, 3, 4, 21, 6, 7, 8, 63, 32, 49, 50, 35, 52, 51, 52, 37, 54, 55, 56, 47, 48, 33, 34, 51, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3821(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 5, 22, 23, 24, 31, 16, 1, 2, 19, 4, 3, 4, 21, 6, 7, 8, 47, 32, 49, 50, 35, 52, 51, 52, 37, 54, 55, 56, 63, 48, 33, 34, 51, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3822(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 5, 22, 23, 24, 15, 0, 1, 2, 19, 4, 3, 4, 21, 6, 7, 8, 63, 48, 49, 50, 35, 52, 51, 52, 37, 54, 55, 56, 47, 32, 33, 34, 51, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3823(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 5, 22, 23, 24, 31, 0, 1, 2, 19, 4, 3, 4, 21, 6, 7, 8, 47, 48, 49, 50, 35, 52, 51, 52, 37, 54, 55, 56, 63, 32, 33, 34, 51, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3824(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 5, 22, 23, 24, 15, 16, 17, 18, 3, 4, 3, 4, 21, 6, 7, 8, 63, 32, 33, 34, 51, 52, 51, 52, 37, 54, 55, 56, 47, 48, 49, 50, 35, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3825(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 5, 22, 23, 24, 31, 16, 17, 18, 3, 4, 3, 4, 21, 6, 7, 8, 47, 32, 33, 34, 51, 52, 51, 52, 37, 54, 55, 56, 63, 48, 49, 50, 35, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3826(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 5, 22, 23, 24, 15, 0, 17, 18, 3, 4, 3, 4, 21, 6, 7, 8, 63, 48, 33, 34, 51, 52, 51, 52, 37, 54, 55, 56, 47, 32, 49, 50, 35, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3827(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 5, 22, 23, 24, 31, 0, 17, 18, 3, 4, 3, 4, 21, 6, 7, 8, 47, 48, 33, 34, 51, 52, 51, 52, 37, 54, 55, 56, 63, 32, 49, 50, 35, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3828(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 5, 22, 23, 24, 15, 16, 1, 18, 3, 4, 3, 4, 21, 6, 7, 8, 63, 32, 49, 34, 51, 52, 51, 52, 37, 54, 55, 56, 47, 48, 33, 50, 35, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3829(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 5, 22, 23, 24, 31, 16, 1, 18, 3, 4, 3, 4, 21, 6, 7, 8, 47, 32, 49, 34, 51, 52, 51, 52, 37, 54, 55, 56, 63, 48, 33, 50, 35, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3830(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 5, 22, 23, 24, 15, 0, 1, 18, 3, 4, 3, 4, 21, 6, 7, 8, 63, 48, 49, 34, 51, 52, 51, 52, 37, 54, 55, 56, 47, 32, 33, 50, 35, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3831(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 5, 22, 23, 24, 31, 0, 1, 18, 3, 4, 3, 4, 21, 6, 7, 8, 47, 48, 49, 34, 51, 52, 51, 52, 37, 54, 55, 56, 63, 32, 33, 50, 35, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3832(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 5, 22, 23, 24, 15, 16, 17, 2, 3, 4, 3, 4, 21, 6, 7, 8, 63, 32, 33, 50, 51, 52, 51, 52, 37, 54, 55, 56, 47, 48, 49, 34, 35, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3833(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 5, 22, 23, 24, 31, 16, 17, 2, 3, 4, 3, 4, 21, 6, 7, 8, 47, 32, 33, 50, 51, 52, 51, 52, 37, 54, 55, 56, 63, 48, 49, 34, 35, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3834(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 5, 22, 23, 24, 15, 0, 17, 2, 3, 4, 3, 4, 21, 6, 7, 8, 63, 48, 33, 50, 51, 52, 51, 52, 37, 54, 55, 56, 47, 32, 49, 34, 35, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3835(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 5, 22, 23, 24, 31, 0, 17, 2, 3, 4, 3, 4, 21, 6, 7, 8, 47, 48, 33, 50, 51, 52, 51, 52, 37, 54, 55, 56, 63, 32, 49, 34, 35, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3836(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 5, 22, 23, 24, 15, 16, 1, 2, 3, 4, 3, 4, 21, 6, 7, 8, 63, 32, 49, 50, 51, 52, 51, 52, 37, 54, 55, 56, 47, 48, 33, 34, 35, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3837(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 5, 22, 23, 24, 31, 16, 1, 2, 3, 4, 3, 4, 21, 6, 7, 8, 47, 32, 49, 50, 51, 52, 51, 52, 37, 54, 55, 56, 63, 48, 33, 34, 35, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3838(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 5, 22, 23, 24, 15, 0, 1, 2, 3, 4, 3, 4, 21, 6, 7, 8, 63, 48, 49, 50, 51, 52, 51, 52, 37, 54, 55, 56, 47, 32, 33, 34, 35, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3839(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 5, 22, 23, 24, 31, 0, 1, 2, 3, 4, 3, 4, 21, 6, 7, 8, 47, 48, 49, 50, 51, 52, 51, 52, 37, 54, 55, 56, 63, 32, 33, 34, 35, 36, 35, 36, 53, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}


