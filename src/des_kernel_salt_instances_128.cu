#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt128(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 5, 6, 7, 8, 15, 16, 17, 18, 19, 20, 19, 4, 21, 22, 23, 24, 63, 32, 33, 34, 35, 36, 35, 52, 37, 38, 39, 40, 47, 48, 49, 50, 51, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt129(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 5, 6, 7, 8, 31, 16, 17, 18, 19, 20, 19, 4, 21, 22, 23, 24, 47, 32, 33, 34, 35, 36, 35, 52, 37, 38, 39, 40, 63, 48, 49, 50, 51, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt130(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 5, 6, 7, 8, 15, 0, 17, 18, 19, 20, 19, 4, 21, 22, 23, 24, 63, 48, 33, 34, 35, 36, 35, 52, 37, 38, 39, 40, 47, 32, 49, 50, 51, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt131(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 5, 6, 7, 8, 31, 0, 17, 18, 19, 20, 19, 4, 21, 22, 23, 24, 47, 48, 33, 34, 35, 36, 35, 52, 37, 38, 39, 40, 63, 32, 49, 50, 51, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt132(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 5, 6, 7, 8, 15, 16, 1, 18, 19, 20, 19, 4, 21, 22, 23, 24, 63, 32, 49, 34, 35, 36, 35, 52, 37, 38, 39, 40, 47, 48, 33, 50, 51, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt133(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 5, 6, 7, 8, 31, 16, 1, 18, 19, 20, 19, 4, 21, 22, 23, 24, 47, 32, 49, 34, 35, 36, 35, 52, 37, 38, 39, 40, 63, 48, 33, 50, 51, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt134(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 5, 6, 7, 8, 15, 0, 1, 18, 19, 20, 19, 4, 21, 22, 23, 24, 63, 48, 49, 34, 35, 36, 35, 52, 37, 38, 39, 40, 47, 32, 33, 50, 51, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt135(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 5, 6, 7, 8, 31, 0, 1, 18, 19, 20, 19, 4, 21, 22, 23, 24, 47, 48, 49, 34, 35, 36, 35, 52, 37, 38, 39, 40, 63, 32, 33, 50, 51, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt136(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 5, 6, 7, 8, 15, 16, 17, 2, 19, 20, 19, 4, 21, 22, 23, 24, 63, 32, 33, 50, 35, 36, 35, 52, 37, 38, 39, 40, 47, 48, 49, 34, 51, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt137(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 5, 6, 7, 8, 31, 16, 17, 2, 19, 20, 19, 4, 21, 22, 23, 24, 47, 32, 33, 50, 35, 36, 35, 52, 37, 38, 39, 40, 63, 48, 49, 34, 51, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt138(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 5, 6, 7, 8, 15, 0, 17, 2, 19, 20, 19, 4, 21, 22, 23, 24, 63, 48, 33, 50, 35, 36, 35, 52, 37, 38, 39, 40, 47, 32, 49, 34, 51, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt139(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 5, 6, 7, 8, 31, 0, 17, 2, 19, 20, 19, 4, 21, 22, 23, 24, 47, 48, 33, 50, 35, 36, 35, 52, 37, 38, 39, 40, 63, 32, 49, 34, 51, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt140(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 5, 6, 7, 8, 15, 16, 1, 2, 19, 20, 19, 4, 21, 22, 23, 24, 63, 32, 49, 50, 35, 36, 35, 52, 37, 38, 39, 40, 47, 48, 33, 34, 51, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt141(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 5, 6, 7, 8, 31, 16, 1, 2, 19, 20, 19, 4, 21, 22, 23, 24, 47, 32, 49, 50, 35, 36, 35, 52, 37, 38, 39, 40, 63, 48, 33, 34, 51, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt142(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 5, 6, 7, 8, 15, 0, 1, 2, 19, 20, 19, 4, 21, 22, 23, 24, 63, 48, 49, 50, 35, 36, 35, 52, 37, 38, 39, 40, 47, 32, 33, 34, 51, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt143(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 5, 6, 7, 8, 31, 0, 1, 2, 19, 20, 19, 4, 21, 22, 23, 24, 47, 48, 49, 50, 35, 36, 35, 52, 37, 38, 39, 40, 63, 32, 33, 34, 51, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt144(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 5, 6, 7, 8, 15, 16, 17, 18, 3, 20, 19, 4, 21, 22, 23, 24, 63, 32, 33, 34, 51, 36, 35, 52, 37, 38, 39, 40, 47, 48, 49, 50, 35, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt145(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 5, 6, 7, 8, 31, 16, 17, 18, 3, 20, 19, 4, 21, 22, 23, 24, 47, 32, 33, 34, 51, 36, 35, 52, 37, 38, 39, 40, 63, 48, 49, 50, 35, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt146(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 5, 6, 7, 8, 15, 0, 17, 18, 3, 20, 19, 4, 21, 22, 23, 24, 63, 48, 33, 34, 51, 36, 35, 52, 37, 38, 39, 40, 47, 32, 49, 50, 35, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt147(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 5, 6, 7, 8, 31, 0, 17, 18, 3, 20, 19, 4, 21, 22, 23, 24, 47, 48, 33, 34, 51, 36, 35, 52, 37, 38, 39, 40, 63, 32, 49, 50, 35, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt148(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 5, 6, 7, 8, 15, 16, 1, 18, 3, 20, 19, 4, 21, 22, 23, 24, 63, 32, 49, 34, 51, 36, 35, 52, 37, 38, 39, 40, 47, 48, 33, 50, 35, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt149(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 5, 6, 7, 8, 31, 16, 1, 18, 3, 20, 19, 4, 21, 22, 23, 24, 47, 32, 49, 34, 51, 36, 35, 52, 37, 38, 39, 40, 63, 48, 33, 50, 35, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt150(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 5, 6, 7, 8, 15, 0, 1, 18, 3, 20, 19, 4, 21, 22, 23, 24, 63, 48, 49, 34, 51, 36, 35, 52, 37, 38, 39, 40, 47, 32, 33, 50, 35, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt151(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 5, 6, 7, 8, 31, 0, 1, 18, 3, 20, 19, 4, 21, 22, 23, 24, 47, 48, 49, 34, 51, 36, 35, 52, 37, 38, 39, 40, 63, 32, 33, 50, 35, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt152(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 5, 6, 7, 8, 15, 16, 17, 2, 3, 20, 19, 4, 21, 22, 23, 24, 63, 32, 33, 50, 51, 36, 35, 52, 37, 38, 39, 40, 47, 48, 49, 34, 35, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt153(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 5, 6, 7, 8, 31, 16, 17, 2, 3, 20, 19, 4, 21, 22, 23, 24, 47, 32, 33, 50, 51, 36, 35, 52, 37, 38, 39, 40, 63, 48, 49, 34, 35, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt154(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 5, 6, 7, 8, 15, 0, 17, 2, 3, 20, 19, 4, 21, 22, 23, 24, 63, 48, 33, 50, 51, 36, 35, 52, 37, 38, 39, 40, 47, 32, 49, 34, 35, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt155(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 5, 6, 7, 8, 31, 0, 17, 2, 3, 20, 19, 4, 21, 22, 23, 24, 47, 48, 33, 50, 51, 36, 35, 52, 37, 38, 39, 40, 63, 32, 49, 34, 35, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt156(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 5, 6, 7, 8, 15, 16, 1, 2, 3, 20, 19, 4, 21, 22, 23, 24, 63, 32, 49, 50, 51, 36, 35, 52, 37, 38, 39, 40, 47, 48, 33, 34, 35, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt157(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 5, 6, 7, 8, 31, 16, 1, 2, 3, 20, 19, 4, 21, 22, 23, 24, 47, 32, 49, 50, 51, 36, 35, 52, 37, 38, 39, 40, 63, 48, 33, 34, 35, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt158(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 5, 6, 7, 8, 15, 0, 1, 2, 3, 20, 19, 4, 21, 22, 23, 24, 63, 48, 49, 50, 51, 36, 35, 52, 37, 38, 39, 40, 47, 32, 33, 34, 35, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt159(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 5, 6, 7, 8, 31, 0, 1, 2, 3, 20, 19, 4, 21, 22, 23, 24, 47, 48, 49, 50, 51, 36, 35, 52, 37, 38, 39, 40, 63, 32, 33, 34, 35, 52, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt160(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 5, 6, 7, 8, 15, 16, 17, 18, 19, 4, 19, 4, 21, 22, 23, 24, 63, 32, 33, 34, 35, 52, 35, 52, 37, 38, 39, 40, 47, 48, 49, 50, 51, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt161(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 5, 6, 7, 8, 31, 16, 17, 18, 19, 4, 19, 4, 21, 22, 23, 24, 47, 32, 33, 34, 35, 52, 35, 52, 37, 38, 39, 40, 63, 48, 49, 50, 51, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt162(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 5, 6, 7, 8, 15, 0, 17, 18, 19, 4, 19, 4, 21, 22, 23, 24, 63, 48, 33, 34, 35, 52, 35, 52, 37, 38, 39, 40, 47, 32, 49, 50, 51, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt163(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 5, 6, 7, 8, 31, 0, 17, 18, 19, 4, 19, 4, 21, 22, 23, 24, 47, 48, 33, 34, 35, 52, 35, 52, 37, 38, 39, 40, 63, 32, 49, 50, 51, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt164(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 5, 6, 7, 8, 15, 16, 1, 18, 19, 4, 19, 4, 21, 22, 23, 24, 63, 32, 49, 34, 35, 52, 35, 52, 37, 38, 39, 40, 47, 48, 33, 50, 51, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt165(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 5, 6, 7, 8, 31, 16, 1, 18, 19, 4, 19, 4, 21, 22, 23, 24, 47, 32, 49, 34, 35, 52, 35, 52, 37, 38, 39, 40, 63, 48, 33, 50, 51, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt166(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 5, 6, 7, 8, 15, 0, 1, 18, 19, 4, 19, 4, 21, 22, 23, 24, 63, 48, 49, 34, 35, 52, 35, 52, 37, 38, 39, 40, 47, 32, 33, 50, 51, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt167(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 5, 6, 7, 8, 31, 0, 1, 18, 19, 4, 19, 4, 21, 22, 23, 24, 47, 48, 49, 34, 35, 52, 35, 52, 37, 38, 39, 40, 63, 32, 33, 50, 51, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt168(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 5, 6, 7, 8, 15, 16, 17, 2, 19, 4, 19, 4, 21, 22, 23, 24, 63, 32, 33, 50, 35, 52, 35, 52, 37, 38, 39, 40, 47, 48, 49, 34, 51, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt169(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 5, 6, 7, 8, 31, 16, 17, 2, 19, 4, 19, 4, 21, 22, 23, 24, 47, 32, 33, 50, 35, 52, 35, 52, 37, 38, 39, 40, 63, 48, 49, 34, 51, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt170(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 5, 6, 7, 8, 15, 0, 17, 2, 19, 4, 19, 4, 21, 22, 23, 24, 63, 48, 33, 50, 35, 52, 35, 52, 37, 38, 39, 40, 47, 32, 49, 34, 51, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt171(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 5, 6, 7, 8, 31, 0, 17, 2, 19, 4, 19, 4, 21, 22, 23, 24, 47, 48, 33, 50, 35, 52, 35, 52, 37, 38, 39, 40, 63, 32, 49, 34, 51, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt172(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 5, 6, 7, 8, 15, 16, 1, 2, 19, 4, 19, 4, 21, 22, 23, 24, 63, 32, 49, 50, 35, 52, 35, 52, 37, 38, 39, 40, 47, 48, 33, 34, 51, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt173(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 5, 6, 7, 8, 31, 16, 1, 2, 19, 4, 19, 4, 21, 22, 23, 24, 47, 32, 49, 50, 35, 52, 35, 52, 37, 38, 39, 40, 63, 48, 33, 34, 51, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt174(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 5, 6, 7, 8, 15, 0, 1, 2, 19, 4, 19, 4, 21, 22, 23, 24, 63, 48, 49, 50, 35, 52, 35, 52, 37, 38, 39, 40, 47, 32, 33, 34, 51, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt175(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 5, 6, 7, 8, 31, 0, 1, 2, 19, 4, 19, 4, 21, 22, 23, 24, 47, 48, 49, 50, 35, 52, 35, 52, 37, 38, 39, 40, 63, 32, 33, 34, 51, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt176(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 5, 6, 7, 8, 15, 16, 17, 18, 3, 4, 19, 4, 21, 22, 23, 24, 63, 32, 33, 34, 51, 52, 35, 52, 37, 38, 39, 40, 47, 48, 49, 50, 35, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt177(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 5, 6, 7, 8, 31, 16, 17, 18, 3, 4, 19, 4, 21, 22, 23, 24, 47, 32, 33, 34, 51, 52, 35, 52, 37, 38, 39, 40, 63, 48, 49, 50, 35, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt178(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 5, 6, 7, 8, 15, 0, 17, 18, 3, 4, 19, 4, 21, 22, 23, 24, 63, 48, 33, 34, 51, 52, 35, 52, 37, 38, 39, 40, 47, 32, 49, 50, 35, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt179(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 5, 6, 7, 8, 31, 0, 17, 18, 3, 4, 19, 4, 21, 22, 23, 24, 47, 48, 33, 34, 51, 52, 35, 52, 37, 38, 39, 40, 63, 32, 49, 50, 35, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt180(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 5, 6, 7, 8, 15, 16, 1, 18, 3, 4, 19, 4, 21, 22, 23, 24, 63, 32, 49, 34, 51, 52, 35, 52, 37, 38, 39, 40, 47, 48, 33, 50, 35, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt181(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 5, 6, 7, 8, 31, 16, 1, 18, 3, 4, 19, 4, 21, 22, 23, 24, 47, 32, 49, 34, 51, 52, 35, 52, 37, 38, 39, 40, 63, 48, 33, 50, 35, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt182(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 5, 6, 7, 8, 15, 0, 1, 18, 3, 4, 19, 4, 21, 22, 23, 24, 63, 48, 49, 34, 51, 52, 35, 52, 37, 38, 39, 40, 47, 32, 33, 50, 35, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt183(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 5, 6, 7, 8, 31, 0, 1, 18, 3, 4, 19, 4, 21, 22, 23, 24, 47, 48, 49, 34, 51, 52, 35, 52, 37, 38, 39, 40, 63, 32, 33, 50, 35, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt184(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 5, 6, 7, 8, 15, 16, 17, 2, 3, 4, 19, 4, 21, 22, 23, 24, 63, 32, 33, 50, 51, 52, 35, 52, 37, 38, 39, 40, 47, 48, 49, 34, 35, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt185(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 5, 6, 7, 8, 31, 16, 17, 2, 3, 4, 19, 4, 21, 22, 23, 24, 47, 32, 33, 50, 51, 52, 35, 52, 37, 38, 39, 40, 63, 48, 49, 34, 35, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt186(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 5, 6, 7, 8, 15, 0, 17, 2, 3, 4, 19, 4, 21, 22, 23, 24, 63, 48, 33, 50, 51, 52, 35, 52, 37, 38, 39, 40, 47, 32, 49, 34, 35, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt187(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 5, 6, 7, 8, 31, 0, 17, 2, 3, 4, 19, 4, 21, 22, 23, 24, 47, 48, 33, 50, 51, 52, 35, 52, 37, 38, 39, 40, 63, 32, 49, 34, 35, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt188(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 5, 6, 7, 8, 15, 16, 1, 2, 3, 4, 19, 4, 21, 22, 23, 24, 63, 32, 49, 50, 51, 52, 35, 52, 37, 38, 39, 40, 47, 48, 33, 34, 35, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt189(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 5, 6, 7, 8, 31, 16, 1, 2, 3, 4, 19, 4, 21, 22, 23, 24, 47, 32, 49, 50, 51, 52, 35, 52, 37, 38, 39, 40, 63, 48, 33, 34, 35, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt190(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 5, 6, 7, 8, 15, 0, 1, 2, 3, 4, 19, 4, 21, 22, 23, 24, 63, 48, 49, 50, 51, 52, 35, 52, 37, 38, 39, 40, 47, 32, 33, 34, 35, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt191(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 5, 6, 7, 8, 31, 0, 1, 2, 3, 4, 19, 4, 21, 22, 23, 24, 47, 48, 49, 50, 51, 52, 35, 52, 37, 38, 39, 40, 63, 32, 33, 34, 35, 36, 51, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt192(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 5, 6, 7, 8, 15, 16, 17, 18, 19, 20, 3, 4, 21, 22, 23, 24, 63, 32, 33, 34, 35, 36, 51, 52, 37, 38, 39, 40, 47, 48, 49, 50, 51, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt193(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 5, 6, 7, 8, 31, 16, 17, 18, 19, 20, 3, 4, 21, 22, 23, 24, 47, 32, 33, 34, 35, 36, 51, 52, 37, 38, 39, 40, 63, 48, 49, 50, 51, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt194(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 5, 6, 7, 8, 15, 0, 17, 18, 19, 20, 3, 4, 21, 22, 23, 24, 63, 48, 33, 34, 35, 36, 51, 52, 37, 38, 39, 40, 47, 32, 49, 50, 51, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt195(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 5, 6, 7, 8, 31, 0, 17, 18, 19, 20, 3, 4, 21, 22, 23, 24, 47, 48, 33, 34, 35, 36, 51, 52, 37, 38, 39, 40, 63, 32, 49, 50, 51, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt196(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 5, 6, 7, 8, 15, 16, 1, 18, 19, 20, 3, 4, 21, 22, 23, 24, 63, 32, 49, 34, 35, 36, 51, 52, 37, 38, 39, 40, 47, 48, 33, 50, 51, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt197(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 5, 6, 7, 8, 31, 16, 1, 18, 19, 20, 3, 4, 21, 22, 23, 24, 47, 32, 49, 34, 35, 36, 51, 52, 37, 38, 39, 40, 63, 48, 33, 50, 51, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt198(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 5, 6, 7, 8, 15, 0, 1, 18, 19, 20, 3, 4, 21, 22, 23, 24, 63, 48, 49, 34, 35, 36, 51, 52, 37, 38, 39, 40, 47, 32, 33, 50, 51, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt199(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 5, 6, 7, 8, 31, 0, 1, 18, 19, 20, 3, 4, 21, 22, 23, 24, 47, 48, 49, 34, 35, 36, 51, 52, 37, 38, 39, 40, 63, 32, 33, 50, 51, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt200(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 5, 6, 7, 8, 15, 16, 17, 2, 19, 20, 3, 4, 21, 22, 23, 24, 63, 32, 33, 50, 35, 36, 51, 52, 37, 38, 39, 40, 47, 48, 49, 34, 51, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt201(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 5, 6, 7, 8, 31, 16, 17, 2, 19, 20, 3, 4, 21, 22, 23, 24, 47, 32, 33, 50, 35, 36, 51, 52, 37, 38, 39, 40, 63, 48, 49, 34, 51, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt202(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 5, 6, 7, 8, 15, 0, 17, 2, 19, 20, 3, 4, 21, 22, 23, 24, 63, 48, 33, 50, 35, 36, 51, 52, 37, 38, 39, 40, 47, 32, 49, 34, 51, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt203(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 5, 6, 7, 8, 31, 0, 17, 2, 19, 20, 3, 4, 21, 22, 23, 24, 47, 48, 33, 50, 35, 36, 51, 52, 37, 38, 39, 40, 63, 32, 49, 34, 51, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt204(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 5, 6, 7, 8, 15, 16, 1, 2, 19, 20, 3, 4, 21, 22, 23, 24, 63, 32, 49, 50, 35, 36, 51, 52, 37, 38, 39, 40, 47, 48, 33, 34, 51, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt205(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 5, 6, 7, 8, 31, 16, 1, 2, 19, 20, 3, 4, 21, 22, 23, 24, 47, 32, 49, 50, 35, 36, 51, 52, 37, 38, 39, 40, 63, 48, 33, 34, 51, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt206(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 5, 6, 7, 8, 15, 0, 1, 2, 19, 20, 3, 4, 21, 22, 23, 24, 63, 48, 49, 50, 35, 36, 51, 52, 37, 38, 39, 40, 47, 32, 33, 34, 51, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt207(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 5, 6, 7, 8, 31, 0, 1, 2, 19, 20, 3, 4, 21, 22, 23, 24, 47, 48, 49, 50, 35, 36, 51, 52, 37, 38, 39, 40, 63, 32, 33, 34, 51, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt208(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 5, 6, 7, 8, 15, 16, 17, 18, 3, 20, 3, 4, 21, 22, 23, 24, 63, 32, 33, 34, 51, 36, 51, 52, 37, 38, 39, 40, 47, 48, 49, 50, 35, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt209(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 5, 6, 7, 8, 31, 16, 17, 18, 3, 20, 3, 4, 21, 22, 23, 24, 47, 32, 33, 34, 51, 36, 51, 52, 37, 38, 39, 40, 63, 48, 49, 50, 35, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt210(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 5, 6, 7, 8, 15, 0, 17, 18, 3, 20, 3, 4, 21, 22, 23, 24, 63, 48, 33, 34, 51, 36, 51, 52, 37, 38, 39, 40, 47, 32, 49, 50, 35, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt211(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 5, 6, 7, 8, 31, 0, 17, 18, 3, 20, 3, 4, 21, 22, 23, 24, 47, 48, 33, 34, 51, 36, 51, 52, 37, 38, 39, 40, 63, 32, 49, 50, 35, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt212(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 5, 6, 7, 8, 15, 16, 1, 18, 3, 20, 3, 4, 21, 22, 23, 24, 63, 32, 49, 34, 51, 36, 51, 52, 37, 38, 39, 40, 47, 48, 33, 50, 35, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt213(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 5, 6, 7, 8, 31, 16, 1, 18, 3, 20, 3, 4, 21, 22, 23, 24, 47, 32, 49, 34, 51, 36, 51, 52, 37, 38, 39, 40, 63, 48, 33, 50, 35, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt214(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 5, 6, 7, 8, 15, 0, 1, 18, 3, 20, 3, 4, 21, 22, 23, 24, 63, 48, 49, 34, 51, 36, 51, 52, 37, 38, 39, 40, 47, 32, 33, 50, 35, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt215(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 5, 6, 7, 8, 31, 0, 1, 18, 3, 20, 3, 4, 21, 22, 23, 24, 47, 48, 49, 34, 51, 36, 51, 52, 37, 38, 39, 40, 63, 32, 33, 50, 35, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt216(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 5, 6, 7, 8, 15, 16, 17, 2, 3, 20, 3, 4, 21, 22, 23, 24, 63, 32, 33, 50, 51, 36, 51, 52, 37, 38, 39, 40, 47, 48, 49, 34, 35, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt217(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 5, 6, 7, 8, 31, 16, 17, 2, 3, 20, 3, 4, 21, 22, 23, 24, 47, 32, 33, 50, 51, 36, 51, 52, 37, 38, 39, 40, 63, 48, 49, 34, 35, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt218(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 5, 6, 7, 8, 15, 0, 17, 2, 3, 20, 3, 4, 21, 22, 23, 24, 63, 48, 33, 50, 51, 36, 51, 52, 37, 38, 39, 40, 47, 32, 49, 34, 35, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt219(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 5, 6, 7, 8, 31, 0, 17, 2, 3, 20, 3, 4, 21, 22, 23, 24, 47, 48, 33, 50, 51, 36, 51, 52, 37, 38, 39, 40, 63, 32, 49, 34, 35, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt220(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 5, 6, 7, 8, 15, 16, 1, 2, 3, 20, 3, 4, 21, 22, 23, 24, 63, 32, 49, 50, 51, 36, 51, 52, 37, 38, 39, 40, 47, 48, 33, 34, 35, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt221(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 5, 6, 7, 8, 31, 16, 1, 2, 3, 20, 3, 4, 21, 22, 23, 24, 47, 32, 49, 50, 51, 36, 51, 52, 37, 38, 39, 40, 63, 48, 33, 34, 35, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt222(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 5, 6, 7, 8, 15, 0, 1, 2, 3, 20, 3, 4, 21, 22, 23, 24, 63, 48, 49, 50, 51, 36, 51, 52, 37, 38, 39, 40, 47, 32, 33, 34, 35, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt223(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 5, 6, 7, 8, 31, 0, 1, 2, 3, 20, 3, 4, 21, 22, 23, 24, 47, 48, 49, 50, 51, 36, 51, 52, 37, 38, 39, 40, 63, 32, 33, 34, 35, 52, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt224(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 5, 6, 7, 8, 15, 16, 17, 18, 19, 4, 3, 4, 21, 22, 23, 24, 63, 32, 33, 34, 35, 52, 51, 52, 37, 38, 39, 40, 47, 48, 49, 50, 51, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt225(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 5, 6, 7, 8, 31, 16, 17, 18, 19, 4, 3, 4, 21, 22, 23, 24, 47, 32, 33, 34, 35, 52, 51, 52, 37, 38, 39, 40, 63, 48, 49, 50, 51, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt226(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 5, 6, 7, 8, 15, 0, 17, 18, 19, 4, 3, 4, 21, 22, 23, 24, 63, 48, 33, 34, 35, 52, 51, 52, 37, 38, 39, 40, 47, 32, 49, 50, 51, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt227(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 5, 6, 7, 8, 31, 0, 17, 18, 19, 4, 3, 4, 21, 22, 23, 24, 47, 48, 33, 34, 35, 52, 51, 52, 37, 38, 39, 40, 63, 32, 49, 50, 51, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt228(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 5, 6, 7, 8, 15, 16, 1, 18, 19, 4, 3, 4, 21, 22, 23, 24, 63, 32, 49, 34, 35, 52, 51, 52, 37, 38, 39, 40, 47, 48, 33, 50, 51, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt229(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 5, 6, 7, 8, 31, 16, 1, 18, 19, 4, 3, 4, 21, 22, 23, 24, 47, 32, 49, 34, 35, 52, 51, 52, 37, 38, 39, 40, 63, 48, 33, 50, 51, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt230(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 5, 6, 7, 8, 15, 0, 1, 18, 19, 4, 3, 4, 21, 22, 23, 24, 63, 48, 49, 34, 35, 52, 51, 52, 37, 38, 39, 40, 47, 32, 33, 50, 51, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt231(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 5, 6, 7, 8, 31, 0, 1, 18, 19, 4, 3, 4, 21, 22, 23, 24, 47, 48, 49, 34, 35, 52, 51, 52, 37, 38, 39, 40, 63, 32, 33, 50, 51, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt232(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 5, 6, 7, 8, 15, 16, 17, 2, 19, 4, 3, 4, 21, 22, 23, 24, 63, 32, 33, 50, 35, 52, 51, 52, 37, 38, 39, 40, 47, 48, 49, 34, 51, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt233(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 5, 6, 7, 8, 31, 16, 17, 2, 19, 4, 3, 4, 21, 22, 23, 24, 47, 32, 33, 50, 35, 52, 51, 52, 37, 38, 39, 40, 63, 48, 49, 34, 51, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt234(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 5, 6, 7, 8, 15, 0, 17, 2, 19, 4, 3, 4, 21, 22, 23, 24, 63, 48, 33, 50, 35, 52, 51, 52, 37, 38, 39, 40, 47, 32, 49, 34, 51, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt235(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 5, 6, 7, 8, 31, 0, 17, 2, 19, 4, 3, 4, 21, 22, 23, 24, 47, 48, 33, 50, 35, 52, 51, 52, 37, 38, 39, 40, 63, 32, 49, 34, 51, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt236(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 5, 6, 7, 8, 15, 16, 1, 2, 19, 4, 3, 4, 21, 22, 23, 24, 63, 32, 49, 50, 35, 52, 51, 52, 37, 38, 39, 40, 47, 48, 33, 34, 51, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt237(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 5, 6, 7, 8, 31, 16, 1, 2, 19, 4, 3, 4, 21, 22, 23, 24, 47, 32, 49, 50, 35, 52, 51, 52, 37, 38, 39, 40, 63, 48, 33, 34, 51, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt238(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 5, 6, 7, 8, 15, 0, 1, 2, 19, 4, 3, 4, 21, 22, 23, 24, 63, 48, 49, 50, 35, 52, 51, 52, 37, 38, 39, 40, 47, 32, 33, 34, 51, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt239(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 5, 6, 7, 8, 31, 0, 1, 2, 19, 4, 3, 4, 21, 22, 23, 24, 47, 48, 49, 50, 35, 52, 51, 52, 37, 38, 39, 40, 63, 32, 33, 34, 51, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt240(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 5, 6, 7, 8, 15, 16, 17, 18, 3, 4, 3, 4, 21, 22, 23, 24, 63, 32, 33, 34, 51, 52, 51, 52, 37, 38, 39, 40, 47, 48, 49, 50, 35, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt241(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 5, 6, 7, 8, 31, 16, 17, 18, 3, 4, 3, 4, 21, 22, 23, 24, 47, 32, 33, 34, 51, 52, 51, 52, 37, 38, 39, 40, 63, 48, 49, 50, 35, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt242(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 5, 6, 7, 8, 15, 0, 17, 18, 3, 4, 3, 4, 21, 22, 23, 24, 63, 48, 33, 34, 51, 52, 51, 52, 37, 38, 39, 40, 47, 32, 49, 50, 35, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt243(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 5, 6, 7, 8, 31, 0, 17, 18, 3, 4, 3, 4, 21, 22, 23, 24, 47, 48, 33, 34, 51, 52, 51, 52, 37, 38, 39, 40, 63, 32, 49, 50, 35, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt244(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 5, 6, 7, 8, 15, 16, 1, 18, 3, 4, 3, 4, 21, 22, 23, 24, 63, 32, 49, 34, 51, 52, 51, 52, 37, 38, 39, 40, 47, 48, 33, 50, 35, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt245(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 5, 6, 7, 8, 31, 16, 1, 18, 3, 4, 3, 4, 21, 22, 23, 24, 47, 32, 49, 34, 51, 52, 51, 52, 37, 38, 39, 40, 63, 48, 33, 50, 35, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt246(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 5, 6, 7, 8, 15, 0, 1, 18, 3, 4, 3, 4, 21, 22, 23, 24, 63, 48, 49, 34, 51, 52, 51, 52, 37, 38, 39, 40, 47, 32, 33, 50, 35, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt247(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 5, 6, 7, 8, 31, 0, 1, 18, 3, 4, 3, 4, 21, 22, 23, 24, 47, 48, 49, 34, 51, 52, 51, 52, 37, 38, 39, 40, 63, 32, 33, 50, 35, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt248(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 5, 6, 7, 8, 15, 16, 17, 2, 3, 4, 3, 4, 21, 22, 23, 24, 63, 32, 33, 50, 51, 52, 51, 52, 37, 38, 39, 40, 47, 48, 49, 34, 35, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt249(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 5, 6, 7, 8, 31, 16, 17, 2, 3, 4, 3, 4, 21, 22, 23, 24, 47, 32, 33, 50, 51, 52, 51, 52, 37, 38, 39, 40, 63, 48, 49, 34, 35, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt250(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 5, 6, 7, 8, 15, 0, 17, 2, 3, 4, 3, 4, 21, 22, 23, 24, 63, 48, 33, 50, 51, 52, 51, 52, 37, 38, 39, 40, 47, 32, 49, 34, 35, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt251(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 5, 6, 7, 8, 31, 0, 17, 2, 3, 4, 3, 4, 21, 22, 23, 24, 47, 48, 33, 50, 51, 52, 51, 52, 37, 38, 39, 40, 63, 32, 49, 34, 35, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt252(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 5, 6, 7, 8, 15, 16, 1, 2, 3, 4, 3, 4, 21, 22, 23, 24, 63, 32, 49, 50, 51, 52, 51, 52, 37, 38, 39, 40, 47, 48, 33, 34, 35, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt253(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 5, 6, 7, 8, 31, 16, 1, 2, 3, 4, 3, 4, 21, 22, 23, 24, 47, 32, 49, 50, 51, 52, 51, 52, 37, 38, 39, 40, 63, 48, 33, 34, 35, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt254(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 5, 6, 7, 8, 15, 0, 1, 2, 3, 4, 3, 4, 21, 22, 23, 24, 63, 48, 49, 50, 51, 52, 51, 52, 37, 38, 39, 40, 47, 32, 33, 34, 35, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt255(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 5, 6, 7, 8, 31, 0, 1, 2, 3, 4, 3, 4, 21, 22, 23, 24, 47, 48, 49, 50, 51, 52, 51, 52, 37, 38, 39, 40, 63, 32, 33, 34, 35, 36, 35, 36, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}


