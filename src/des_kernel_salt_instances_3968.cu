#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt3968(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 21, 22, 23, 24, 15, 16, 17, 18, 19, 20, 19, 4, 5, 6, 7, 8, 63, 32, 33, 34, 35, 36, 35, 52, 53, 54, 55, 56, 47, 48, 49, 50, 51, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3969(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 21, 22, 23, 24, 31, 16, 17, 18, 19, 20, 19, 4, 5, 6, 7, 8, 47, 32, 33, 34, 35, 36, 35, 52, 53, 54, 55, 56, 63, 48, 49, 50, 51, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3970(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 21, 22, 23, 24, 15, 0, 17, 18, 19, 20, 19, 4, 5, 6, 7, 8, 63, 48, 33, 34, 35, 36, 35, 52, 53, 54, 55, 56, 47, 32, 49, 50, 51, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3971(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 21, 22, 23, 24, 31, 0, 17, 18, 19, 20, 19, 4, 5, 6, 7, 8, 47, 48, 33, 34, 35, 36, 35, 52, 53, 54, 55, 56, 63, 32, 49, 50, 51, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3972(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 21, 22, 23, 24, 15, 16, 1, 18, 19, 20, 19, 4, 5, 6, 7, 8, 63, 32, 49, 34, 35, 36, 35, 52, 53, 54, 55, 56, 47, 48, 33, 50, 51, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3973(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 21, 22, 23, 24, 31, 16, 1, 18, 19, 20, 19, 4, 5, 6, 7, 8, 47, 32, 49, 34, 35, 36, 35, 52, 53, 54, 55, 56, 63, 48, 33, 50, 51, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3974(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 21, 22, 23, 24, 15, 0, 1, 18, 19, 20, 19, 4, 5, 6, 7, 8, 63, 48, 49, 34, 35, 36, 35, 52, 53, 54, 55, 56, 47, 32, 33, 50, 51, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3975(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 21, 22, 23, 24, 31, 0, 1, 18, 19, 20, 19, 4, 5, 6, 7, 8, 47, 48, 49, 34, 35, 36, 35, 52, 53, 54, 55, 56, 63, 32, 33, 50, 51, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3976(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 21, 22, 23, 24, 15, 16, 17, 2, 19, 20, 19, 4, 5, 6, 7, 8, 63, 32, 33, 50, 35, 36, 35, 52, 53, 54, 55, 56, 47, 48, 49, 34, 51, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3977(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 21, 22, 23, 24, 31, 16, 17, 2, 19, 20, 19, 4, 5, 6, 7, 8, 47, 32, 33, 50, 35, 36, 35, 52, 53, 54, 55, 56, 63, 48, 49, 34, 51, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3978(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 21, 22, 23, 24, 15, 0, 17, 2, 19, 20, 19, 4, 5, 6, 7, 8, 63, 48, 33, 50, 35, 36, 35, 52, 53, 54, 55, 56, 47, 32, 49, 34, 51, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3979(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 21, 22, 23, 24, 31, 0, 17, 2, 19, 20, 19, 4, 5, 6, 7, 8, 47, 48, 33, 50, 35, 36, 35, 52, 53, 54, 55, 56, 63, 32, 49, 34, 51, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3980(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 21, 22, 23, 24, 15, 16, 1, 2, 19, 20, 19, 4, 5, 6, 7, 8, 63, 32, 49, 50, 35, 36, 35, 52, 53, 54, 55, 56, 47, 48, 33, 34, 51, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3981(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 21, 22, 23, 24, 31, 16, 1, 2, 19, 20, 19, 4, 5, 6, 7, 8, 47, 32, 49, 50, 35, 36, 35, 52, 53, 54, 55, 56, 63, 48, 33, 34, 51, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3982(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 21, 22, 23, 24, 15, 0, 1, 2, 19, 20, 19, 4, 5, 6, 7, 8, 63, 48, 49, 50, 35, 36, 35, 52, 53, 54, 55, 56, 47, 32, 33, 34, 51, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3983(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 21, 22, 23, 24, 31, 0, 1, 2, 19, 20, 19, 4, 5, 6, 7, 8, 47, 48, 49, 50, 35, 36, 35, 52, 53, 54, 55, 56, 63, 32, 33, 34, 51, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3984(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 21, 22, 23, 24, 15, 16, 17, 18, 3, 20, 19, 4, 5, 6, 7, 8, 63, 32, 33, 34, 51, 36, 35, 52, 53, 54, 55, 56, 47, 48, 49, 50, 35, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3985(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 21, 22, 23, 24, 31, 16, 17, 18, 3, 20, 19, 4, 5, 6, 7, 8, 47, 32, 33, 34, 51, 36, 35, 52, 53, 54, 55, 56, 63, 48, 49, 50, 35, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3986(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 21, 22, 23, 24, 15, 0, 17, 18, 3, 20, 19, 4, 5, 6, 7, 8, 63, 48, 33, 34, 51, 36, 35, 52, 53, 54, 55, 56, 47, 32, 49, 50, 35, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3987(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 21, 22, 23, 24, 31, 0, 17, 18, 3, 20, 19, 4, 5, 6, 7, 8, 47, 48, 33, 34, 51, 36, 35, 52, 53, 54, 55, 56, 63, 32, 49, 50, 35, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3988(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 21, 22, 23, 24, 15, 16, 1, 18, 3, 20, 19, 4, 5, 6, 7, 8, 63, 32, 49, 34, 51, 36, 35, 52, 53, 54, 55, 56, 47, 48, 33, 50, 35, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3989(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 21, 22, 23, 24, 31, 16, 1, 18, 3, 20, 19, 4, 5, 6, 7, 8, 47, 32, 49, 34, 51, 36, 35, 52, 53, 54, 55, 56, 63, 48, 33, 50, 35, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3990(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 21, 22, 23, 24, 15, 0, 1, 18, 3, 20, 19, 4, 5, 6, 7, 8, 63, 48, 49, 34, 51, 36, 35, 52, 53, 54, 55, 56, 47, 32, 33, 50, 35, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3991(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 21, 22, 23, 24, 31, 0, 1, 18, 3, 20, 19, 4, 5, 6, 7, 8, 47, 48, 49, 34, 51, 36, 35, 52, 53, 54, 55, 56, 63, 32, 33, 50, 35, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3992(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 21, 22, 23, 24, 15, 16, 17, 2, 3, 20, 19, 4, 5, 6, 7, 8, 63, 32, 33, 50, 51, 36, 35, 52, 53, 54, 55, 56, 47, 48, 49, 34, 35, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3993(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 21, 22, 23, 24, 31, 16, 17, 2, 3, 20, 19, 4, 5, 6, 7, 8, 47, 32, 33, 50, 51, 36, 35, 52, 53, 54, 55, 56, 63, 48, 49, 34, 35, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3994(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 21, 22, 23, 24, 15, 0, 17, 2, 3, 20, 19, 4, 5, 6, 7, 8, 63, 48, 33, 50, 51, 36, 35, 52, 53, 54, 55, 56, 47, 32, 49, 34, 35, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3995(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 21, 22, 23, 24, 31, 0, 17, 2, 3, 20, 19, 4, 5, 6, 7, 8, 47, 48, 33, 50, 51, 36, 35, 52, 53, 54, 55, 56, 63, 32, 49, 34, 35, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3996(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 21, 22, 23, 24, 15, 16, 1, 2, 3, 20, 19, 4, 5, 6, 7, 8, 63, 32, 49, 50, 51, 36, 35, 52, 53, 54, 55, 56, 47, 48, 33, 34, 35, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3997(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 21, 22, 23, 24, 31, 16, 1, 2, 3, 20, 19, 4, 5, 6, 7, 8, 47, 32, 49, 50, 51, 36, 35, 52, 53, 54, 55, 56, 63, 48, 33, 34, 35, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3998(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 21, 22, 23, 24, 15, 0, 1, 2, 3, 20, 19, 4, 5, 6, 7, 8, 63, 48, 49, 50, 51, 36, 35, 52, 53, 54, 55, 56, 47, 32, 33, 34, 35, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3999(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 21, 22, 23, 24, 31, 0, 1, 2, 3, 20, 19, 4, 5, 6, 7, 8, 47, 48, 49, 50, 51, 36, 35, 52, 53, 54, 55, 56, 63, 32, 33, 34, 35, 52, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4000(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 21, 22, 23, 24, 15, 16, 17, 18, 19, 4, 19, 4, 5, 6, 7, 8, 63, 32, 33, 34, 35, 52, 35, 52, 53, 54, 55, 56, 47, 48, 49, 50, 51, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4001(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 21, 22, 23, 24, 31, 16, 17, 18, 19, 4, 19, 4, 5, 6, 7, 8, 47, 32, 33, 34, 35, 52, 35, 52, 53, 54, 55, 56, 63, 48, 49, 50, 51, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4002(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 21, 22, 23, 24, 15, 0, 17, 18, 19, 4, 19, 4, 5, 6, 7, 8, 63, 48, 33, 34, 35, 52, 35, 52, 53, 54, 55, 56, 47, 32, 49, 50, 51, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4003(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 21, 22, 23, 24, 31, 0, 17, 18, 19, 4, 19, 4, 5, 6, 7, 8, 47, 48, 33, 34, 35, 52, 35, 52, 53, 54, 55, 56, 63, 32, 49, 50, 51, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4004(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 21, 22, 23, 24, 15, 16, 1, 18, 19, 4, 19, 4, 5, 6, 7, 8, 63, 32, 49, 34, 35, 52, 35, 52, 53, 54, 55, 56, 47, 48, 33, 50, 51, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4005(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 21, 22, 23, 24, 31, 16, 1, 18, 19, 4, 19, 4, 5, 6, 7, 8, 47, 32, 49, 34, 35, 52, 35, 52, 53, 54, 55, 56, 63, 48, 33, 50, 51, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4006(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 21, 22, 23, 24, 15, 0, 1, 18, 19, 4, 19, 4, 5, 6, 7, 8, 63, 48, 49, 34, 35, 52, 35, 52, 53, 54, 55, 56, 47, 32, 33, 50, 51, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4007(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 21, 22, 23, 24, 31, 0, 1, 18, 19, 4, 19, 4, 5, 6, 7, 8, 47, 48, 49, 34, 35, 52, 35, 52, 53, 54, 55, 56, 63, 32, 33, 50, 51, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4008(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 21, 22, 23, 24, 15, 16, 17, 2, 19, 4, 19, 4, 5, 6, 7, 8, 63, 32, 33, 50, 35, 52, 35, 52, 53, 54, 55, 56, 47, 48, 49, 34, 51, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4009(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 21, 22, 23, 24, 31, 16, 17, 2, 19, 4, 19, 4, 5, 6, 7, 8, 47, 32, 33, 50, 35, 52, 35, 52, 53, 54, 55, 56, 63, 48, 49, 34, 51, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4010(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 21, 22, 23, 24, 15, 0, 17, 2, 19, 4, 19, 4, 5, 6, 7, 8, 63, 48, 33, 50, 35, 52, 35, 52, 53, 54, 55, 56, 47, 32, 49, 34, 51, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4011(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 21, 22, 23, 24, 31, 0, 17, 2, 19, 4, 19, 4, 5, 6, 7, 8, 47, 48, 33, 50, 35, 52, 35, 52, 53, 54, 55, 56, 63, 32, 49, 34, 51, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4012(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 21, 22, 23, 24, 15, 16, 1, 2, 19, 4, 19, 4, 5, 6, 7, 8, 63, 32, 49, 50, 35, 52, 35, 52, 53, 54, 55, 56, 47, 48, 33, 34, 51, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4013(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 21, 22, 23, 24, 31, 16, 1, 2, 19, 4, 19, 4, 5, 6, 7, 8, 47, 32, 49, 50, 35, 52, 35, 52, 53, 54, 55, 56, 63, 48, 33, 34, 51, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4014(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 21, 22, 23, 24, 15, 0, 1, 2, 19, 4, 19, 4, 5, 6, 7, 8, 63, 48, 49, 50, 35, 52, 35, 52, 53, 54, 55, 56, 47, 32, 33, 34, 51, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4015(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 21, 22, 23, 24, 31, 0, 1, 2, 19, 4, 19, 4, 5, 6, 7, 8, 47, 48, 49, 50, 35, 52, 35, 52, 53, 54, 55, 56, 63, 32, 33, 34, 51, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4016(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 21, 22, 23, 24, 15, 16, 17, 18, 3, 4, 19, 4, 5, 6, 7, 8, 63, 32, 33, 34, 51, 52, 35, 52, 53, 54, 55, 56, 47, 48, 49, 50, 35, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4017(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 21, 22, 23, 24, 31, 16, 17, 18, 3, 4, 19, 4, 5, 6, 7, 8, 47, 32, 33, 34, 51, 52, 35, 52, 53, 54, 55, 56, 63, 48, 49, 50, 35, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4018(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 21, 22, 23, 24, 15, 0, 17, 18, 3, 4, 19, 4, 5, 6, 7, 8, 63, 48, 33, 34, 51, 52, 35, 52, 53, 54, 55, 56, 47, 32, 49, 50, 35, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4019(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 21, 22, 23, 24, 31, 0, 17, 18, 3, 4, 19, 4, 5, 6, 7, 8, 47, 48, 33, 34, 51, 52, 35, 52, 53, 54, 55, 56, 63, 32, 49, 50, 35, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4020(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 21, 22, 23, 24, 15, 16, 1, 18, 3, 4, 19, 4, 5, 6, 7, 8, 63, 32, 49, 34, 51, 52, 35, 52, 53, 54, 55, 56, 47, 48, 33, 50, 35, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4021(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 21, 22, 23, 24, 31, 16, 1, 18, 3, 4, 19, 4, 5, 6, 7, 8, 47, 32, 49, 34, 51, 52, 35, 52, 53, 54, 55, 56, 63, 48, 33, 50, 35, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4022(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 21, 22, 23, 24, 15, 0, 1, 18, 3, 4, 19, 4, 5, 6, 7, 8, 63, 48, 49, 34, 51, 52, 35, 52, 53, 54, 55, 56, 47, 32, 33, 50, 35, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4023(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 21, 22, 23, 24, 31, 0, 1, 18, 3, 4, 19, 4, 5, 6, 7, 8, 47, 48, 49, 34, 51, 52, 35, 52, 53, 54, 55, 56, 63, 32, 33, 50, 35, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4024(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 21, 22, 23, 24, 15, 16, 17, 2, 3, 4, 19, 4, 5, 6, 7, 8, 63, 32, 33, 50, 51, 52, 35, 52, 53, 54, 55, 56, 47, 48, 49, 34, 35, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4025(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 21, 22, 23, 24, 31, 16, 17, 2, 3, 4, 19, 4, 5, 6, 7, 8, 47, 32, 33, 50, 51, 52, 35, 52, 53, 54, 55, 56, 63, 48, 49, 34, 35, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4026(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 21, 22, 23, 24, 15, 0, 17, 2, 3, 4, 19, 4, 5, 6, 7, 8, 63, 48, 33, 50, 51, 52, 35, 52, 53, 54, 55, 56, 47, 32, 49, 34, 35, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4027(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 21, 22, 23, 24, 31, 0, 17, 2, 3, 4, 19, 4, 5, 6, 7, 8, 47, 48, 33, 50, 51, 52, 35, 52, 53, 54, 55, 56, 63, 32, 49, 34, 35, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4028(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 21, 22, 23, 24, 15, 16, 1, 2, 3, 4, 19, 4, 5, 6, 7, 8, 63, 32, 49, 50, 51, 52, 35, 52, 53, 54, 55, 56, 47, 48, 33, 34, 35, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4029(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 21, 22, 23, 24, 31, 16, 1, 2, 3, 4, 19, 4, 5, 6, 7, 8, 47, 32, 49, 50, 51, 52, 35, 52, 53, 54, 55, 56, 63, 48, 33, 34, 35, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4030(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 21, 22, 23, 24, 15, 0, 1, 2, 3, 4, 19, 4, 5, 6, 7, 8, 63, 48, 49, 50, 51, 52, 35, 52, 53, 54, 55, 56, 47, 32, 33, 34, 35, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4031(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 21, 22, 23, 24, 31, 0, 1, 2, 3, 4, 19, 4, 5, 6, 7, 8, 47, 48, 49, 50, 51, 52, 35, 52, 53, 54, 55, 56, 63, 32, 33, 34, 35, 36, 51, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4032(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 21, 22, 23, 24, 15, 16, 17, 18, 19, 20, 3, 4, 5, 6, 7, 8, 63, 32, 33, 34, 35, 36, 51, 52, 53, 54, 55, 56, 47, 48, 49, 50, 51, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4033(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 21, 22, 23, 24, 31, 16, 17, 18, 19, 20, 3, 4, 5, 6, 7, 8, 47, 32, 33, 34, 35, 36, 51, 52, 53, 54, 55, 56, 63, 48, 49, 50, 51, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4034(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 21, 22, 23, 24, 15, 0, 17, 18, 19, 20, 3, 4, 5, 6, 7, 8, 63, 48, 33, 34, 35, 36, 51, 52, 53, 54, 55, 56, 47, 32, 49, 50, 51, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4035(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 21, 22, 23, 24, 31, 0, 17, 18, 19, 20, 3, 4, 5, 6, 7, 8, 47, 48, 33, 34, 35, 36, 51, 52, 53, 54, 55, 56, 63, 32, 49, 50, 51, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4036(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 21, 22, 23, 24, 15, 16, 1, 18, 19, 20, 3, 4, 5, 6, 7, 8, 63, 32, 49, 34, 35, 36, 51, 52, 53, 54, 55, 56, 47, 48, 33, 50, 51, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4037(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 21, 22, 23, 24, 31, 16, 1, 18, 19, 20, 3, 4, 5, 6, 7, 8, 47, 32, 49, 34, 35, 36, 51, 52, 53, 54, 55, 56, 63, 48, 33, 50, 51, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4038(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 21, 22, 23, 24, 15, 0, 1, 18, 19, 20, 3, 4, 5, 6, 7, 8, 63, 48, 49, 34, 35, 36, 51, 52, 53, 54, 55, 56, 47, 32, 33, 50, 51, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4039(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 21, 22, 23, 24, 31, 0, 1, 18, 19, 20, 3, 4, 5, 6, 7, 8, 47, 48, 49, 34, 35, 36, 51, 52, 53, 54, 55, 56, 63, 32, 33, 50, 51, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4040(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 21, 22, 23, 24, 15, 16, 17, 2, 19, 20, 3, 4, 5, 6, 7, 8, 63, 32, 33, 50, 35, 36, 51, 52, 53, 54, 55, 56, 47, 48, 49, 34, 51, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4041(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 21, 22, 23, 24, 31, 16, 17, 2, 19, 20, 3, 4, 5, 6, 7, 8, 47, 32, 33, 50, 35, 36, 51, 52, 53, 54, 55, 56, 63, 48, 49, 34, 51, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4042(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 21, 22, 23, 24, 15, 0, 17, 2, 19, 20, 3, 4, 5, 6, 7, 8, 63, 48, 33, 50, 35, 36, 51, 52, 53, 54, 55, 56, 47, 32, 49, 34, 51, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4043(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 21, 22, 23, 24, 31, 0, 17, 2, 19, 20, 3, 4, 5, 6, 7, 8, 47, 48, 33, 50, 35, 36, 51, 52, 53, 54, 55, 56, 63, 32, 49, 34, 51, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4044(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 21, 22, 23, 24, 15, 16, 1, 2, 19, 20, 3, 4, 5, 6, 7, 8, 63, 32, 49, 50, 35, 36, 51, 52, 53, 54, 55, 56, 47, 48, 33, 34, 51, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4045(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 21, 22, 23, 24, 31, 16, 1, 2, 19, 20, 3, 4, 5, 6, 7, 8, 47, 32, 49, 50, 35, 36, 51, 52, 53, 54, 55, 56, 63, 48, 33, 34, 51, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4046(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 21, 22, 23, 24, 15, 0, 1, 2, 19, 20, 3, 4, 5, 6, 7, 8, 63, 48, 49, 50, 35, 36, 51, 52, 53, 54, 55, 56, 47, 32, 33, 34, 51, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4047(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 21, 22, 23, 24, 31, 0, 1, 2, 19, 20, 3, 4, 5, 6, 7, 8, 47, 48, 49, 50, 35, 36, 51, 52, 53, 54, 55, 56, 63, 32, 33, 34, 51, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4048(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 21, 22, 23, 24, 15, 16, 17, 18, 3, 20, 3, 4, 5, 6, 7, 8, 63, 32, 33, 34, 51, 36, 51, 52, 53, 54, 55, 56, 47, 48, 49, 50, 35, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4049(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 21, 22, 23, 24, 31, 16, 17, 18, 3, 20, 3, 4, 5, 6, 7, 8, 47, 32, 33, 34, 51, 36, 51, 52, 53, 54, 55, 56, 63, 48, 49, 50, 35, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4050(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 21, 22, 23, 24, 15, 0, 17, 18, 3, 20, 3, 4, 5, 6, 7, 8, 63, 48, 33, 34, 51, 36, 51, 52, 53, 54, 55, 56, 47, 32, 49, 50, 35, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4051(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 21, 22, 23, 24, 31, 0, 17, 18, 3, 20, 3, 4, 5, 6, 7, 8, 47, 48, 33, 34, 51, 36, 51, 52, 53, 54, 55, 56, 63, 32, 49, 50, 35, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4052(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 21, 22, 23, 24, 15, 16, 1, 18, 3, 20, 3, 4, 5, 6, 7, 8, 63, 32, 49, 34, 51, 36, 51, 52, 53, 54, 55, 56, 47, 48, 33, 50, 35, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4053(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 21, 22, 23, 24, 31, 16, 1, 18, 3, 20, 3, 4, 5, 6, 7, 8, 47, 32, 49, 34, 51, 36, 51, 52, 53, 54, 55, 56, 63, 48, 33, 50, 35, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4054(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 21, 22, 23, 24, 15, 0, 1, 18, 3, 20, 3, 4, 5, 6, 7, 8, 63, 48, 49, 34, 51, 36, 51, 52, 53, 54, 55, 56, 47, 32, 33, 50, 35, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4055(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 21, 22, 23, 24, 31, 0, 1, 18, 3, 20, 3, 4, 5, 6, 7, 8, 47, 48, 49, 34, 51, 36, 51, 52, 53, 54, 55, 56, 63, 32, 33, 50, 35, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4056(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 21, 22, 23, 24, 15, 16, 17, 2, 3, 20, 3, 4, 5, 6, 7, 8, 63, 32, 33, 50, 51, 36, 51, 52, 53, 54, 55, 56, 47, 48, 49, 34, 35, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4057(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 21, 22, 23, 24, 31, 16, 17, 2, 3, 20, 3, 4, 5, 6, 7, 8, 47, 32, 33, 50, 51, 36, 51, 52, 53, 54, 55, 56, 63, 48, 49, 34, 35, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4058(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 21, 22, 23, 24, 15, 0, 17, 2, 3, 20, 3, 4, 5, 6, 7, 8, 63, 48, 33, 50, 51, 36, 51, 52, 53, 54, 55, 56, 47, 32, 49, 34, 35, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4059(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 21, 22, 23, 24, 31, 0, 17, 2, 3, 20, 3, 4, 5, 6, 7, 8, 47, 48, 33, 50, 51, 36, 51, 52, 53, 54, 55, 56, 63, 32, 49, 34, 35, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4060(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 21, 22, 23, 24, 15, 16, 1, 2, 3, 20, 3, 4, 5, 6, 7, 8, 63, 32, 49, 50, 51, 36, 51, 52, 53, 54, 55, 56, 47, 48, 33, 34, 35, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4061(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 21, 22, 23, 24, 31, 16, 1, 2, 3, 20, 3, 4, 5, 6, 7, 8, 47, 32, 49, 50, 51, 36, 51, 52, 53, 54, 55, 56, 63, 48, 33, 34, 35, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4062(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 21, 22, 23, 24, 15, 0, 1, 2, 3, 20, 3, 4, 5, 6, 7, 8, 63, 48, 49, 50, 51, 36, 51, 52, 53, 54, 55, 56, 47, 32, 33, 34, 35, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4063(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 21, 22, 23, 24, 31, 0, 1, 2, 3, 20, 3, 4, 5, 6, 7, 8, 47, 48, 49, 50, 51, 36, 51, 52, 53, 54, 55, 56, 63, 32, 33, 34, 35, 52, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4064(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 21, 22, 23, 24, 15, 16, 17, 18, 19, 4, 3, 4, 5, 6, 7, 8, 63, 32, 33, 34, 35, 52, 51, 52, 53, 54, 55, 56, 47, 48, 49, 50, 51, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4065(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 21, 22, 23, 24, 31, 16, 17, 18, 19, 4, 3, 4, 5, 6, 7, 8, 47, 32, 33, 34, 35, 52, 51, 52, 53, 54, 55, 56, 63, 48, 49, 50, 51, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4066(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 21, 22, 23, 24, 15, 0, 17, 18, 19, 4, 3, 4, 5, 6, 7, 8, 63, 48, 33, 34, 35, 52, 51, 52, 53, 54, 55, 56, 47, 32, 49, 50, 51, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4067(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 21, 22, 23, 24, 31, 0, 17, 18, 19, 4, 3, 4, 5, 6, 7, 8, 47, 48, 33, 34, 35, 52, 51, 52, 53, 54, 55, 56, 63, 32, 49, 50, 51, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4068(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 21, 22, 23, 24, 15, 16, 1, 18, 19, 4, 3, 4, 5, 6, 7, 8, 63, 32, 49, 34, 35, 52, 51, 52, 53, 54, 55, 56, 47, 48, 33, 50, 51, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4069(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 21, 22, 23, 24, 31, 16, 1, 18, 19, 4, 3, 4, 5, 6, 7, 8, 47, 32, 49, 34, 35, 52, 51, 52, 53, 54, 55, 56, 63, 48, 33, 50, 51, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4070(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 21, 22, 23, 24, 15, 0, 1, 18, 19, 4, 3, 4, 5, 6, 7, 8, 63, 48, 49, 34, 35, 52, 51, 52, 53, 54, 55, 56, 47, 32, 33, 50, 51, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4071(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 21, 22, 23, 24, 31, 0, 1, 18, 19, 4, 3, 4, 5, 6, 7, 8, 47, 48, 49, 34, 35, 52, 51, 52, 53, 54, 55, 56, 63, 32, 33, 50, 51, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4072(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 21, 22, 23, 24, 15, 16, 17, 2, 19, 4, 3, 4, 5, 6, 7, 8, 63, 32, 33, 50, 35, 52, 51, 52, 53, 54, 55, 56, 47, 48, 49, 34, 51, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4073(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 21, 22, 23, 24, 31, 16, 17, 2, 19, 4, 3, 4, 5, 6, 7, 8, 47, 32, 33, 50, 35, 52, 51, 52, 53, 54, 55, 56, 63, 48, 49, 34, 51, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4074(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 21, 22, 23, 24, 15, 0, 17, 2, 19, 4, 3, 4, 5, 6, 7, 8, 63, 48, 33, 50, 35, 52, 51, 52, 53, 54, 55, 56, 47, 32, 49, 34, 51, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4075(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 21, 22, 23, 24, 31, 0, 17, 2, 19, 4, 3, 4, 5, 6, 7, 8, 47, 48, 33, 50, 35, 52, 51, 52, 53, 54, 55, 56, 63, 32, 49, 34, 51, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4076(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 21, 22, 23, 24, 15, 16, 1, 2, 19, 4, 3, 4, 5, 6, 7, 8, 63, 32, 49, 50, 35, 52, 51, 52, 53, 54, 55, 56, 47, 48, 33, 34, 51, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4077(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 21, 22, 23, 24, 31, 16, 1, 2, 19, 4, 3, 4, 5, 6, 7, 8, 47, 32, 49, 50, 35, 52, 51, 52, 53, 54, 55, 56, 63, 48, 33, 34, 51, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4078(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 21, 22, 23, 24, 15, 0, 1, 2, 19, 4, 3, 4, 5, 6, 7, 8, 63, 48, 49, 50, 35, 52, 51, 52, 53, 54, 55, 56, 47, 32, 33, 34, 51, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4079(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 21, 22, 23, 24, 31, 0, 1, 2, 19, 4, 3, 4, 5, 6, 7, 8, 47, 48, 49, 50, 35, 52, 51, 52, 53, 54, 55, 56, 63, 32, 33, 34, 51, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4080(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 21, 22, 23, 24, 15, 16, 17, 18, 3, 4, 3, 4, 5, 6, 7, 8, 63, 32, 33, 34, 51, 52, 51, 52, 53, 54, 55, 56, 47, 48, 49, 50, 35, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4081(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 21, 22, 23, 24, 31, 16, 17, 18, 3, 4, 3, 4, 5, 6, 7, 8, 47, 32, 33, 34, 51, 52, 51, 52, 53, 54, 55, 56, 63, 48, 49, 50, 35, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4082(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 21, 22, 23, 24, 15, 0, 17, 18, 3, 4, 3, 4, 5, 6, 7, 8, 63, 48, 33, 34, 51, 52, 51, 52, 53, 54, 55, 56, 47, 32, 49, 50, 35, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4083(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 21, 22, 23, 24, 31, 0, 17, 18, 3, 4, 3, 4, 5, 6, 7, 8, 47, 48, 33, 34, 51, 52, 51, 52, 53, 54, 55, 56, 63, 32, 49, 50, 35, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4084(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 21, 22, 23, 24, 15, 16, 1, 18, 3, 4, 3, 4, 5, 6, 7, 8, 63, 32, 49, 34, 51, 52, 51, 52, 53, 54, 55, 56, 47, 48, 33, 50, 35, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4085(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 21, 22, 23, 24, 31, 16, 1, 18, 3, 4, 3, 4, 5, 6, 7, 8, 47, 32, 49, 34, 51, 52, 51, 52, 53, 54, 55, 56, 63, 48, 33, 50, 35, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4086(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 21, 22, 23, 24, 15, 0, 1, 18, 3, 4, 3, 4, 5, 6, 7, 8, 63, 48, 49, 34, 51, 52, 51, 52, 53, 54, 55, 56, 47, 32, 33, 50, 35, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4087(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 21, 22, 23, 24, 31, 0, 1, 18, 3, 4, 3, 4, 5, 6, 7, 8, 47, 48, 49, 34, 51, 52, 51, 52, 53, 54, 55, 56, 63, 32, 33, 50, 35, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4088(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 21, 22, 23, 24, 15, 16, 17, 2, 3, 4, 3, 4, 5, 6, 7, 8, 63, 32, 33, 50, 51, 52, 51, 52, 53, 54, 55, 56, 47, 48, 49, 34, 35, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4089(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 21, 22, 23, 24, 31, 16, 17, 2, 3, 4, 3, 4, 5, 6, 7, 8, 47, 32, 33, 50, 51, 52, 51, 52, 53, 54, 55, 56, 63, 48, 49, 34, 35, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4090(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 21, 22, 23, 24, 15, 0, 17, 2, 3, 4, 3, 4, 5, 6, 7, 8, 63, 48, 33, 50, 51, 52, 51, 52, 53, 54, 55, 56, 47, 32, 49, 34, 35, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4091(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 21, 22, 23, 24, 31, 0, 17, 2, 3, 4, 3, 4, 5, 6, 7, 8, 47, 48, 33, 50, 51, 52, 51, 52, 53, 54, 55, 56, 63, 32, 49, 34, 35, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4092(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 21, 22, 23, 24, 15, 16, 1, 2, 3, 4, 3, 4, 5, 6, 7, 8, 63, 32, 49, 50, 51, 52, 51, 52, 53, 54, 55, 56, 47, 48, 33, 34, 35, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4093(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 21, 22, 23, 24, 31, 16, 1, 2, 3, 4, 3, 4, 5, 6, 7, 8, 47, 32, 49, 50, 51, 52, 51, 52, 53, 54, 55, 56, 63, 48, 33, 34, 35, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4094(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 21, 22, 23, 24, 15, 0, 1, 2, 3, 4, 3, 4, 5, 6, 7, 8, 63, 48, 49, 50, 51, 52, 51, 52, 53, 54, 55, 56, 47, 32, 33, 34, 35, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4095(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 21, 22, 23, 24, 31, 0, 1, 2, 3, 4, 3, 4, 5, 6, 7, 8, 47, 48, 49, 50, 51, 52, 51, 52, 53, 54, 55, 56, 63, 32, 33, 34, 35, 36, 35, 36, 37, 38, 39, 40>(unchecked_hashes, bitsplitted_keys);
}


