#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

#ifdef DESGPU_COMPILE_ALL_SALTS

void des_25_encrypt_salt3968(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 21, 22, 23, 24, 15, 16, 17, 18, 19, 20, 19, 4, 5, 6, 7, 8, 63, 32, 33, 34, 35, 36, 35, 52, 53, 54, 55, 56, 47, 48, 49, 50, 51, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3969(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 21, 22, 23, 24, 31, 16, 17, 18, 19, 20, 19, 4, 5, 6, 7, 8, 47, 32, 33, 34, 35, 36, 35, 52, 53, 54, 55, 56, 63, 48, 49, 50, 51, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3970(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 21, 22, 23, 24, 15, 0, 17, 18, 19, 20, 19, 4, 5, 6, 7, 8, 63, 48, 33, 34, 35, 36, 35, 52, 53, 54, 55, 56, 47, 32, 49, 50, 51, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3971(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 21, 22, 23, 24, 31, 0, 17, 18, 19, 20, 19, 4, 5, 6, 7, 8, 47, 48, 33, 34, 35, 36, 35, 52, 53, 54, 55, 56, 63, 32, 49, 50, 51, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3972(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 21, 22, 23, 24, 15, 16, 1, 18, 19, 20, 19, 4, 5, 6, 7, 8, 63, 32, 49, 34, 35, 36, 35, 52, 53, 54, 55, 56, 47, 48, 33, 50, 51, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3973(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 21, 22, 23, 24, 31, 16, 1, 18, 19, 20, 19, 4, 5, 6, 7, 8, 47, 32, 49, 34, 35, 36, 35, 52, 53, 54, 55, 56, 63, 48, 33, 50, 51, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3974(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 21, 22, 23, 24, 15, 0, 1, 18, 19, 20, 19, 4, 5, 6, 7, 8, 63, 48, 49, 34, 35, 36, 35, 52, 53, 54, 55, 56, 47, 32, 33, 50, 51, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3975(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 21, 22, 23, 24, 31, 0, 1, 18, 19, 20, 19, 4, 5, 6, 7, 8, 47, 48, 49, 34, 35, 36, 35, 52, 53, 54, 55, 56, 63, 32, 33, 50, 51, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3976(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 21, 22, 23, 24, 15, 16, 17, 2, 19, 20, 19, 4, 5, 6, 7, 8, 63, 32, 33, 50, 35, 36, 35, 52, 53, 54, 55, 56, 47, 48, 49, 34, 51, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3977(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 21, 22, 23, 24, 31, 16, 17, 2, 19, 20, 19, 4, 5, 6, 7, 8, 47, 32, 33, 50, 35, 36, 35, 52, 53, 54, 55, 56, 63, 48, 49, 34, 51, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3978(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 21, 22, 23, 24, 15, 0, 17, 2, 19, 20, 19, 4, 5, 6, 7, 8, 63, 48, 33, 50, 35, 36, 35, 52, 53, 54, 55, 56, 47, 32, 49, 34, 51, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3979(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 21, 22, 23, 24, 31, 0, 17, 2, 19, 20, 19, 4, 5, 6, 7, 8, 47, 48, 33, 50, 35, 36, 35, 52, 53, 54, 55, 56, 63, 32, 49, 34, 51, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3980(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 21, 22, 23, 24, 15, 16, 1, 2, 19, 20, 19, 4, 5, 6, 7, 8, 63, 32, 49, 50, 35, 36, 35, 52, 53, 54, 55, 56, 47, 48, 33, 34, 51, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3981(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 21, 22, 23, 24, 31, 16, 1, 2, 19, 20, 19, 4, 5, 6, 7, 8, 47, 32, 49, 50, 35, 36, 35, 52, 53, 54, 55, 56, 63, 48, 33, 34, 51, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3982(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 21, 22, 23, 24, 15, 0, 1, 2, 19, 20, 19, 4, 5, 6, 7, 8, 63, 48, 49, 50, 35, 36, 35, 52, 53, 54, 55, 56, 47, 32, 33, 34, 51, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3983(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 21, 22, 23, 24, 31, 0, 1, 2, 19, 20, 19, 4, 5, 6, 7, 8, 47, 48, 49, 50, 35, 36, 35, 52, 53, 54, 55, 56, 63, 32, 33, 34, 51, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3984(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 21, 22, 23, 24, 15, 16, 17, 18, 3, 20, 19, 4, 5, 6, 7, 8, 63, 32, 33, 34, 51, 36, 35, 52, 53, 54, 55, 56, 47, 48, 49, 50, 35, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3985(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 21, 22, 23, 24, 31, 16, 17, 18, 3, 20, 19, 4, 5, 6, 7, 8, 47, 32, 33, 34, 51, 36, 35, 52, 53, 54, 55, 56, 63, 48, 49, 50, 35, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3986(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 21, 22, 23, 24, 15, 0, 17, 18, 3, 20, 19, 4, 5, 6, 7, 8, 63, 48, 33, 34, 51, 36, 35, 52, 53, 54, 55, 56, 47, 32, 49, 50, 35, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3987(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 21, 22, 23, 24, 31, 0, 17, 18, 3, 20, 19, 4, 5, 6, 7, 8, 47, 48, 33, 34, 51, 36, 35, 52, 53, 54, 55, 56, 63, 32, 49, 50, 35, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3988(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 21, 22, 23, 24, 15, 16, 1, 18, 3, 20, 19, 4, 5, 6, 7, 8, 63, 32, 49, 34, 51, 36, 35, 52, 53, 54, 55, 56, 47, 48, 33, 50, 35, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3989(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 21, 22, 23, 24, 31, 16, 1, 18, 3, 20, 19, 4, 5, 6, 7, 8, 47, 32, 49, 34, 51, 36, 35, 52, 53, 54, 55, 56, 63, 48, 33, 50, 35, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3990(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 21, 22, 23, 24, 15, 0, 1, 18, 3, 20, 19, 4, 5, 6, 7, 8, 63, 48, 49, 34, 51, 36, 35, 52, 53, 54, 55, 56, 47, 32, 33, 50, 35, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3991(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 21, 22, 23, 24, 31, 0, 1, 18, 3, 20, 19, 4, 5, 6, 7, 8, 47, 48, 49, 34, 51, 36, 35, 52, 53, 54, 55, 56, 63, 32, 33, 50, 35, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3992(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 21, 22, 23, 24, 15, 16, 17, 2, 3, 20, 19, 4, 5, 6, 7, 8, 63, 32, 33, 50, 51, 36, 35, 52, 53, 54, 55, 56, 47, 48, 49, 34, 35, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3993(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 21, 22, 23, 24, 31, 16, 17, 2, 3, 20, 19, 4, 5, 6, 7, 8, 47, 32, 33, 50, 51, 36, 35, 52, 53, 54, 55, 56, 63, 48, 49, 34, 35, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3994(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 21, 22, 23, 24, 15, 0, 17, 2, 3, 20, 19, 4, 5, 6, 7, 8, 63, 48, 33, 50, 51, 36, 35, 52, 53, 54, 55, 56, 47, 32, 49, 34, 35, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3995(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 21, 22, 23, 24, 31, 0, 17, 2, 3, 20, 19, 4, 5, 6, 7, 8, 47, 48, 33, 50, 51, 36, 35, 52, 53, 54, 55, 56, 63, 32, 49, 34, 35, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3996(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 21, 22, 23, 24, 15, 16, 1, 2, 3, 20, 19, 4, 5, 6, 7, 8, 63, 32, 49, 50, 51, 36, 35, 52, 53, 54, 55, 56, 47, 48, 33, 34, 35, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3997(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 21, 22, 23, 24, 31, 16, 1, 2, 3, 20, 19, 4, 5, 6, 7, 8, 47, 32, 49, 50, 51, 36, 35, 52, 53, 54, 55, 56, 63, 48, 33, 34, 35, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3998(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 21, 22, 23, 24, 15, 0, 1, 2, 3, 20, 19, 4, 5, 6, 7, 8, 63, 48, 49, 50, 51, 36, 35, 52, 53, 54, 55, 56, 47, 32, 33, 34, 35, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3999(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 21, 22, 23, 24, 31, 0, 1, 2, 3, 20, 19, 4, 5, 6, 7, 8, 47, 48, 49, 50, 51, 36, 35, 52, 53, 54, 55, 56, 63, 32, 33, 34, 35, 52, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4000(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 21, 22, 23, 24, 15, 16, 17, 18, 19, 4, 19, 4, 5, 6, 7, 8, 63, 32, 33, 34, 35, 52, 35, 52, 53, 54, 55, 56, 47, 48, 49, 50, 51, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4001(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 21, 22, 23, 24, 31, 16, 17, 18, 19, 4, 19, 4, 5, 6, 7, 8, 47, 32, 33, 34, 35, 52, 35, 52, 53, 54, 55, 56, 63, 48, 49, 50, 51, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4002(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 21, 22, 23, 24, 15, 0, 17, 18, 19, 4, 19, 4, 5, 6, 7, 8, 63, 48, 33, 34, 35, 52, 35, 52, 53, 54, 55, 56, 47, 32, 49, 50, 51, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4003(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 21, 22, 23, 24, 31, 0, 17, 18, 19, 4, 19, 4, 5, 6, 7, 8, 47, 48, 33, 34, 35, 52, 35, 52, 53, 54, 55, 56, 63, 32, 49, 50, 51, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4004(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 21, 22, 23, 24, 15, 16, 1, 18, 19, 4, 19, 4, 5, 6, 7, 8, 63, 32, 49, 34, 35, 52, 35, 52, 53, 54, 55, 56, 47, 48, 33, 50, 51, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4005(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 21, 22, 23, 24, 31, 16, 1, 18, 19, 4, 19, 4, 5, 6, 7, 8, 47, 32, 49, 34, 35, 52, 35, 52, 53, 54, 55, 56, 63, 48, 33, 50, 51, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4006(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 21, 22, 23, 24, 15, 0, 1, 18, 19, 4, 19, 4, 5, 6, 7, 8, 63, 48, 49, 34, 35, 52, 35, 52, 53, 54, 55, 56, 47, 32, 33, 50, 51, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4007(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 21, 22, 23, 24, 31, 0, 1, 18, 19, 4, 19, 4, 5, 6, 7, 8, 47, 48, 49, 34, 35, 52, 35, 52, 53, 54, 55, 56, 63, 32, 33, 50, 51, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4008(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 21, 22, 23, 24, 15, 16, 17, 2, 19, 4, 19, 4, 5, 6, 7, 8, 63, 32, 33, 50, 35, 52, 35, 52, 53, 54, 55, 56, 47, 48, 49, 34, 51, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4009(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 21, 22, 23, 24, 31, 16, 17, 2, 19, 4, 19, 4, 5, 6, 7, 8, 47, 32, 33, 50, 35, 52, 35, 52, 53, 54, 55, 56, 63, 48, 49, 34, 51, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4010(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 21, 22, 23, 24, 15, 0, 17, 2, 19, 4, 19, 4, 5, 6, 7, 8, 63, 48, 33, 50, 35, 52, 35, 52, 53, 54, 55, 56, 47, 32, 49, 34, 51, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4011(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 21, 22, 23, 24, 31, 0, 17, 2, 19, 4, 19, 4, 5, 6, 7, 8, 47, 48, 33, 50, 35, 52, 35, 52, 53, 54, 55, 56, 63, 32, 49, 34, 51, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4012(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 21, 22, 23, 24, 15, 16, 1, 2, 19, 4, 19, 4, 5, 6, 7, 8, 63, 32, 49, 50, 35, 52, 35, 52, 53, 54, 55, 56, 47, 48, 33, 34, 51, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4013(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 21, 22, 23, 24, 31, 16, 1, 2, 19, 4, 19, 4, 5, 6, 7, 8, 47, 32, 49, 50, 35, 52, 35, 52, 53, 54, 55, 56, 63, 48, 33, 34, 51, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4014(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 21, 22, 23, 24, 15, 0, 1, 2, 19, 4, 19, 4, 5, 6, 7, 8, 63, 48, 49, 50, 35, 52, 35, 52, 53, 54, 55, 56, 47, 32, 33, 34, 51, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4015(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 21, 22, 23, 24, 31, 0, 1, 2, 19, 4, 19, 4, 5, 6, 7, 8, 47, 48, 49, 50, 35, 52, 35, 52, 53, 54, 55, 56, 63, 32, 33, 34, 51, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4016(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 21, 22, 23, 24, 15, 16, 17, 18, 3, 4, 19, 4, 5, 6, 7, 8, 63, 32, 33, 34, 51, 52, 35, 52, 53, 54, 55, 56, 47, 48, 49, 50, 35, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4017(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 21, 22, 23, 24, 31, 16, 17, 18, 3, 4, 19, 4, 5, 6, 7, 8, 47, 32, 33, 34, 51, 52, 35, 52, 53, 54, 55, 56, 63, 48, 49, 50, 35, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4018(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 21, 22, 23, 24, 15, 0, 17, 18, 3, 4, 19, 4, 5, 6, 7, 8, 63, 48, 33, 34, 51, 52, 35, 52, 53, 54, 55, 56, 47, 32, 49, 50, 35, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4019(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 21, 22, 23, 24, 31, 0, 17, 18, 3, 4, 19, 4, 5, 6, 7, 8, 47, 48, 33, 34, 51, 52, 35, 52, 53, 54, 55, 56, 63, 32, 49, 50, 35, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4020(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 21, 22, 23, 24, 15, 16, 1, 18, 3, 4, 19, 4, 5, 6, 7, 8, 63, 32, 49, 34, 51, 52, 35, 52, 53, 54, 55, 56, 47, 48, 33, 50, 35, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4021(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 21, 22, 23, 24, 31, 16, 1, 18, 3, 4, 19, 4, 5, 6, 7, 8, 47, 32, 49, 34, 51, 52, 35, 52, 53, 54, 55, 56, 63, 48, 33, 50, 35, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4022(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 21, 22, 23, 24, 15, 0, 1, 18, 3, 4, 19, 4, 5, 6, 7, 8, 63, 48, 49, 34, 51, 52, 35, 52, 53, 54, 55, 56, 47, 32, 33, 50, 35, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4023(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 21, 22, 23, 24, 31, 0, 1, 18, 3, 4, 19, 4, 5, 6, 7, 8, 47, 48, 49, 34, 51, 52, 35, 52, 53, 54, 55, 56, 63, 32, 33, 50, 35, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4024(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 21, 22, 23, 24, 15, 16, 17, 2, 3, 4, 19, 4, 5, 6, 7, 8, 63, 32, 33, 50, 51, 52, 35, 52, 53, 54, 55, 56, 47, 48, 49, 34, 35, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4025(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 21, 22, 23, 24, 31, 16, 17, 2, 3, 4, 19, 4, 5, 6, 7, 8, 47, 32, 33, 50, 51, 52, 35, 52, 53, 54, 55, 56, 63, 48, 49, 34, 35, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4026(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 21, 22, 23, 24, 15, 0, 17, 2, 3, 4, 19, 4, 5, 6, 7, 8, 63, 48, 33, 50, 51, 52, 35, 52, 53, 54, 55, 56, 47, 32, 49, 34, 35, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4027(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 21, 22, 23, 24, 31, 0, 17, 2, 3, 4, 19, 4, 5, 6, 7, 8, 47, 48, 33, 50, 51, 52, 35, 52, 53, 54, 55, 56, 63, 32, 49, 34, 35, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4028(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 21, 22, 23, 24, 15, 16, 1, 2, 3, 4, 19, 4, 5, 6, 7, 8, 63, 32, 49, 50, 51, 52, 35, 52, 53, 54, 55, 56, 47, 48, 33, 34, 35, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4029(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 21, 22, 23, 24, 31, 16, 1, 2, 3, 4, 19, 4, 5, 6, 7, 8, 47, 32, 49, 50, 51, 52, 35, 52, 53, 54, 55, 56, 63, 48, 33, 34, 35, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4030(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 21, 22, 23, 24, 15, 0, 1, 2, 3, 4, 19, 4, 5, 6, 7, 8, 63, 48, 49, 50, 51, 52, 35, 52, 53, 54, 55, 56, 47, 32, 33, 34, 35, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4031(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 21, 22, 23, 24, 31, 0, 1, 2, 3, 4, 19, 4, 5, 6, 7, 8, 47, 48, 49, 50, 51, 52, 35, 52, 53, 54, 55, 56, 63, 32, 33, 34, 35, 36, 51, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4032(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 21, 22, 23, 24, 15, 16, 17, 18, 19, 20, 3, 4, 5, 6, 7, 8, 63, 32, 33, 34, 35, 36, 51, 52, 53, 54, 55, 56, 47, 48, 49, 50, 51, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4033(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 21, 22, 23, 24, 31, 16, 17, 18, 19, 20, 3, 4, 5, 6, 7, 8, 47, 32, 33, 34, 35, 36, 51, 52, 53, 54, 55, 56, 63, 48, 49, 50, 51, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4034(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 21, 22, 23, 24, 15, 0, 17, 18, 19, 20, 3, 4, 5, 6, 7, 8, 63, 48, 33, 34, 35, 36, 51, 52, 53, 54, 55, 56, 47, 32, 49, 50, 51, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4035(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 21, 22, 23, 24, 31, 0, 17, 18, 19, 20, 3, 4, 5, 6, 7, 8, 47, 48, 33, 34, 35, 36, 51, 52, 53, 54, 55, 56, 63, 32, 49, 50, 51, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4036(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 21, 22, 23, 24, 15, 16, 1, 18, 19, 20, 3, 4, 5, 6, 7, 8, 63, 32, 49, 34, 35, 36, 51, 52, 53, 54, 55, 56, 47, 48, 33, 50, 51, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4037(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 21, 22, 23, 24, 31, 16, 1, 18, 19, 20, 3, 4, 5, 6, 7, 8, 47, 32, 49, 34, 35, 36, 51, 52, 53, 54, 55, 56, 63, 48, 33, 50, 51, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4038(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 21, 22, 23, 24, 15, 0, 1, 18, 19, 20, 3, 4, 5, 6, 7, 8, 63, 48, 49, 34, 35, 36, 51, 52, 53, 54, 55, 56, 47, 32, 33, 50, 51, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4039(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 21, 22, 23, 24, 31, 0, 1, 18, 19, 20, 3, 4, 5, 6, 7, 8, 47, 48, 49, 34, 35, 36, 51, 52, 53, 54, 55, 56, 63, 32, 33, 50, 51, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4040(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 21, 22, 23, 24, 15, 16, 17, 2, 19, 20, 3, 4, 5, 6, 7, 8, 63, 32, 33, 50, 35, 36, 51, 52, 53, 54, 55, 56, 47, 48, 49, 34, 51, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4041(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 21, 22, 23, 24, 31, 16, 17, 2, 19, 20, 3, 4, 5, 6, 7, 8, 47, 32, 33, 50, 35, 36, 51, 52, 53, 54, 55, 56, 63, 48, 49, 34, 51, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4042(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 21, 22, 23, 24, 15, 0, 17, 2, 19, 20, 3, 4, 5, 6, 7, 8, 63, 48, 33, 50, 35, 36, 51, 52, 53, 54, 55, 56, 47, 32, 49, 34, 51, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4043(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 21, 22, 23, 24, 31, 0, 17, 2, 19, 20, 3, 4, 5, 6, 7, 8, 47, 48, 33, 50, 35, 36, 51, 52, 53, 54, 55, 56, 63, 32, 49, 34, 51, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4044(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 21, 22, 23, 24, 15, 16, 1, 2, 19, 20, 3, 4, 5, 6, 7, 8, 63, 32, 49, 50, 35, 36, 51, 52, 53, 54, 55, 56, 47, 48, 33, 34, 51, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4045(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 21, 22, 23, 24, 31, 16, 1, 2, 19, 20, 3, 4, 5, 6, 7, 8, 47, 32, 49, 50, 35, 36, 51, 52, 53, 54, 55, 56, 63, 48, 33, 34, 51, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4046(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 21, 22, 23, 24, 15, 0, 1, 2, 19, 20, 3, 4, 5, 6, 7, 8, 63, 48, 49, 50, 35, 36, 51, 52, 53, 54, 55, 56, 47, 32, 33, 34, 51, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4047(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 21, 22, 23, 24, 31, 0, 1, 2, 19, 20, 3, 4, 5, 6, 7, 8, 47, 48, 49, 50, 35, 36, 51, 52, 53, 54, 55, 56, 63, 32, 33, 34, 51, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4048(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 21, 22, 23, 24, 15, 16, 17, 18, 3, 20, 3, 4, 5, 6, 7, 8, 63, 32, 33, 34, 51, 36, 51, 52, 53, 54, 55, 56, 47, 48, 49, 50, 35, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4049(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 21, 22, 23, 24, 31, 16, 17, 18, 3, 20, 3, 4, 5, 6, 7, 8, 47, 32, 33, 34, 51, 36, 51, 52, 53, 54, 55, 56, 63, 48, 49, 50, 35, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4050(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 21, 22, 23, 24, 15, 0, 17, 18, 3, 20, 3, 4, 5, 6, 7, 8, 63, 48, 33, 34, 51, 36, 51, 52, 53, 54, 55, 56, 47, 32, 49, 50, 35, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4051(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 21, 22, 23, 24, 31, 0, 17, 18, 3, 20, 3, 4, 5, 6, 7, 8, 47, 48, 33, 34, 51, 36, 51, 52, 53, 54, 55, 56, 63, 32, 49, 50, 35, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4052(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 21, 22, 23, 24, 15, 16, 1, 18, 3, 20, 3, 4, 5, 6, 7, 8, 63, 32, 49, 34, 51, 36, 51, 52, 53, 54, 55, 56, 47, 48, 33, 50, 35, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4053(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 21, 22, 23, 24, 31, 16, 1, 18, 3, 20, 3, 4, 5, 6, 7, 8, 47, 32, 49, 34, 51, 36, 51, 52, 53, 54, 55, 56, 63, 48, 33, 50, 35, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4054(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 21, 22, 23, 24, 15, 0, 1, 18, 3, 20, 3, 4, 5, 6, 7, 8, 63, 48, 49, 34, 51, 36, 51, 52, 53, 54, 55, 56, 47, 32, 33, 50, 35, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4055(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 21, 22, 23, 24, 31, 0, 1, 18, 3, 20, 3, 4, 5, 6, 7, 8, 47, 48, 49, 34, 51, 36, 51, 52, 53, 54, 55, 56, 63, 32, 33, 50, 35, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4056(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 21, 22, 23, 24, 15, 16, 17, 2, 3, 20, 3, 4, 5, 6, 7, 8, 63, 32, 33, 50, 51, 36, 51, 52, 53, 54, 55, 56, 47, 48, 49, 34, 35, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4057(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 21, 22, 23, 24, 31, 16, 17, 2, 3, 20, 3, 4, 5, 6, 7, 8, 47, 32, 33, 50, 51, 36, 51, 52, 53, 54, 55, 56, 63, 48, 49, 34, 35, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4058(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 21, 22, 23, 24, 15, 0, 17, 2, 3, 20, 3, 4, 5, 6, 7, 8, 63, 48, 33, 50, 51, 36, 51, 52, 53, 54, 55, 56, 47, 32, 49, 34, 35, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4059(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 21, 22, 23, 24, 31, 0, 17, 2, 3, 20, 3, 4, 5, 6, 7, 8, 47, 48, 33, 50, 51, 36, 51, 52, 53, 54, 55, 56, 63, 32, 49, 34, 35, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4060(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 21, 22, 23, 24, 15, 16, 1, 2, 3, 20, 3, 4, 5, 6, 7, 8, 63, 32, 49, 50, 51, 36, 51, 52, 53, 54, 55, 56, 47, 48, 33, 34, 35, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4061(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 21, 22, 23, 24, 31, 16, 1, 2, 3, 20, 3, 4, 5, 6, 7, 8, 47, 32, 49, 50, 51, 36, 51, 52, 53, 54, 55, 56, 63, 48, 33, 34, 35, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4062(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 21, 22, 23, 24, 15, 0, 1, 2, 3, 20, 3, 4, 5, 6, 7, 8, 63, 48, 49, 50, 51, 36, 51, 52, 53, 54, 55, 56, 47, 32, 33, 34, 35, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4063(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 21, 22, 23, 24, 31, 0, 1, 2, 3, 20, 3, 4, 5, 6, 7, 8, 47, 48, 49, 50, 51, 36, 51, 52, 53, 54, 55, 56, 63, 32, 33, 34, 35, 52, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4064(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 21, 22, 23, 24, 15, 16, 17, 18, 19, 4, 3, 4, 5, 6, 7, 8, 63, 32, 33, 34, 35, 52, 51, 52, 53, 54, 55, 56, 47, 48, 49, 50, 51, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4065(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 21, 22, 23, 24, 31, 16, 17, 18, 19, 4, 3, 4, 5, 6, 7, 8, 47, 32, 33, 34, 35, 52, 51, 52, 53, 54, 55, 56, 63, 48, 49, 50, 51, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4066(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 21, 22, 23, 24, 15, 0, 17, 18, 19, 4, 3, 4, 5, 6, 7, 8, 63, 48, 33, 34, 35, 52, 51, 52, 53, 54, 55, 56, 47, 32, 49, 50, 51, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4067(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 21, 22, 23, 24, 31, 0, 17, 18, 19, 4, 3, 4, 5, 6, 7, 8, 47, 48, 33, 34, 35, 52, 51, 52, 53, 54, 55, 56, 63, 32, 49, 50, 51, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4068(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 21, 22, 23, 24, 15, 16, 1, 18, 19, 4, 3, 4, 5, 6, 7, 8, 63, 32, 49, 34, 35, 52, 51, 52, 53, 54, 55, 56, 47, 48, 33, 50, 51, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4069(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 21, 22, 23, 24, 31, 16, 1, 18, 19, 4, 3, 4, 5, 6, 7, 8, 47, 32, 49, 34, 35, 52, 51, 52, 53, 54, 55, 56, 63, 48, 33, 50, 51, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4070(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 21, 22, 23, 24, 15, 0, 1, 18, 19, 4, 3, 4, 5, 6, 7, 8, 63, 48, 49, 34, 35, 52, 51, 52, 53, 54, 55, 56, 47, 32, 33, 50, 51, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4071(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 21, 22, 23, 24, 31, 0, 1, 18, 19, 4, 3, 4, 5, 6, 7, 8, 47, 48, 49, 34, 35, 52, 51, 52, 53, 54, 55, 56, 63, 32, 33, 50, 51, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4072(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 21, 22, 23, 24, 15, 16, 17, 2, 19, 4, 3, 4, 5, 6, 7, 8, 63, 32, 33, 50, 35, 52, 51, 52, 53, 54, 55, 56, 47, 48, 49, 34, 51, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4073(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 21, 22, 23, 24, 31, 16, 17, 2, 19, 4, 3, 4, 5, 6, 7, 8, 47, 32, 33, 50, 35, 52, 51, 52, 53, 54, 55, 56, 63, 48, 49, 34, 51, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4074(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 21, 22, 23, 24, 15, 0, 17, 2, 19, 4, 3, 4, 5, 6, 7, 8, 63, 48, 33, 50, 35, 52, 51, 52, 53, 54, 55, 56, 47, 32, 49, 34, 51, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4075(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 21, 22, 23, 24, 31, 0, 17, 2, 19, 4, 3, 4, 5, 6, 7, 8, 47, 48, 33, 50, 35, 52, 51, 52, 53, 54, 55, 56, 63, 32, 49, 34, 51, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4076(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 21, 22, 23, 24, 15, 16, 1, 2, 19, 4, 3, 4, 5, 6, 7, 8, 63, 32, 49, 50, 35, 52, 51, 52, 53, 54, 55, 56, 47, 48, 33, 34, 51, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4077(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 21, 22, 23, 24, 31, 16, 1, 2, 19, 4, 3, 4, 5, 6, 7, 8, 47, 32, 49, 50, 35, 52, 51, 52, 53, 54, 55, 56, 63, 48, 33, 34, 51, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4078(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 21, 22, 23, 24, 15, 0, 1, 2, 19, 4, 3, 4, 5, 6, 7, 8, 63, 48, 49, 50, 35, 52, 51, 52, 53, 54, 55, 56, 47, 32, 33, 34, 51, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4079(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 21, 22, 23, 24, 31, 0, 1, 2, 19, 4, 3, 4, 5, 6, 7, 8, 47, 48, 49, 50, 35, 52, 51, 52, 53, 54, 55, 56, 63, 32, 33, 34, 51, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4080(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 21, 22, 23, 24, 15, 16, 17, 18, 3, 4, 3, 4, 5, 6, 7, 8, 63, 32, 33, 34, 51, 52, 51, 52, 53, 54, 55, 56, 47, 48, 49, 50, 35, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4081(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 21, 22, 23, 24, 31, 16, 17, 18, 3, 4, 3, 4, 5, 6, 7, 8, 47, 32, 33, 34, 51, 52, 51, 52, 53, 54, 55, 56, 63, 48, 49, 50, 35, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4082(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 21, 22, 23, 24, 15, 0, 17, 18, 3, 4, 3, 4, 5, 6, 7, 8, 63, 48, 33, 34, 51, 52, 51, 52, 53, 54, 55, 56, 47, 32, 49, 50, 35, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4083(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 21, 22, 23, 24, 31, 0, 17, 18, 3, 4, 3, 4, 5, 6, 7, 8, 47, 48, 33, 34, 51, 52, 51, 52, 53, 54, 55, 56, 63, 32, 49, 50, 35, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4084(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 21, 22, 23, 24, 15, 16, 1, 18, 3, 4, 3, 4, 5, 6, 7, 8, 63, 32, 49, 34, 51, 52, 51, 52, 53, 54, 55, 56, 47, 48, 33, 50, 35, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4085(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 21, 22, 23, 24, 31, 16, 1, 18, 3, 4, 3, 4, 5, 6, 7, 8, 47, 32, 49, 34, 51, 52, 51, 52, 53, 54, 55, 56, 63, 48, 33, 50, 35, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4086(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 21, 22, 23, 24, 15, 0, 1, 18, 3, 4, 3, 4, 5, 6, 7, 8, 63, 48, 49, 34, 51, 52, 51, 52, 53, 54, 55, 56, 47, 32, 33, 50, 35, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4087(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 21, 22, 23, 24, 31, 0, 1, 18, 3, 4, 3, 4, 5, 6, 7, 8, 47, 48, 49, 34, 51, 52, 51, 52, 53, 54, 55, 56, 63, 32, 33, 50, 35, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4088(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 21, 22, 23, 24, 15, 16, 17, 2, 3, 4, 3, 4, 5, 6, 7, 8, 63, 32, 33, 50, 51, 52, 51, 52, 53, 54, 55, 56, 47, 48, 49, 34, 35, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4089(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 21, 22, 23, 24, 31, 16, 17, 2, 3, 4, 3, 4, 5, 6, 7, 8, 47, 32, 33, 50, 51, 52, 51, 52, 53, 54, 55, 56, 63, 48, 49, 34, 35, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4090(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 21, 22, 23, 24, 15, 0, 17, 2, 3, 4, 3, 4, 5, 6, 7, 8, 63, 48, 33, 50, 51, 52, 51, 52, 53, 54, 55, 56, 47, 32, 49, 34, 35, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4091(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 21, 22, 23, 24, 31, 0, 17, 2, 3, 4, 3, 4, 5, 6, 7, 8, 47, 48, 33, 50, 51, 52, 51, 52, 53, 54, 55, 56, 63, 32, 49, 34, 35, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4092(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 21, 22, 23, 24, 15, 16, 1, 2, 3, 4, 3, 4, 5, 6, 7, 8, 63, 32, 49, 50, 51, 52, 51, 52, 53, 54, 55, 56, 47, 48, 33, 34, 35, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4093(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 21, 22, 23, 24, 31, 16, 1, 2, 3, 4, 3, 4, 5, 6, 7, 8, 47, 32, 49, 50, 51, 52, 51, 52, 53, 54, 55, 56, 63, 48, 33, 34, 35, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4094(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 21, 22, 23, 24, 15, 0, 1, 2, 3, 4, 3, 4, 5, 6, 7, 8, 63, 48, 49, 50, 51, 52, 51, 52, 53, 54, 55, 56, 47, 32, 33, 34, 35, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt4095(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 21, 22, 23, 24, 31, 0, 1, 2, 3, 4, 3, 4, 5, 6, 7, 8, 47, 48, 49, 50, 51, 52, 51, 52, 53, 54, 55, 56, 63, 32, 33, 34, 35, 36, 35, 36, 37, 38, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

#endif // DESGPU_COMPILE_ALL_SALTS


