#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt640(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 5, 22, 7, 8, 15, 16, 17, 18, 19, 20, 19, 4, 21, 6, 23, 24, 63, 32, 33, 34, 35, 36, 35, 52, 37, 54, 39, 40, 47, 48, 49, 50, 51, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt641(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 5, 22, 7, 8, 31, 16, 17, 18, 19, 20, 19, 4, 21, 6, 23, 24, 47, 32, 33, 34, 35, 36, 35, 52, 37, 54, 39, 40, 63, 48, 49, 50, 51, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt642(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 5, 22, 7, 8, 15, 0, 17, 18, 19, 20, 19, 4, 21, 6, 23, 24, 63, 48, 33, 34, 35, 36, 35, 52, 37, 54, 39, 40, 47, 32, 49, 50, 51, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt643(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 5, 22, 7, 8, 31, 0, 17, 18, 19, 20, 19, 4, 21, 6, 23, 24, 47, 48, 33, 34, 35, 36, 35, 52, 37, 54, 39, 40, 63, 32, 49, 50, 51, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt644(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 5, 22, 7, 8, 15, 16, 1, 18, 19, 20, 19, 4, 21, 6, 23, 24, 63, 32, 49, 34, 35, 36, 35, 52, 37, 54, 39, 40, 47, 48, 33, 50, 51, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt645(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 5, 22, 7, 8, 31, 16, 1, 18, 19, 20, 19, 4, 21, 6, 23, 24, 47, 32, 49, 34, 35, 36, 35, 52, 37, 54, 39, 40, 63, 48, 33, 50, 51, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt646(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 5, 22, 7, 8, 15, 0, 1, 18, 19, 20, 19, 4, 21, 6, 23, 24, 63, 48, 49, 34, 35, 36, 35, 52, 37, 54, 39, 40, 47, 32, 33, 50, 51, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt647(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 5, 22, 7, 8, 31, 0, 1, 18, 19, 20, 19, 4, 21, 6, 23, 24, 47, 48, 49, 34, 35, 36, 35, 52, 37, 54, 39, 40, 63, 32, 33, 50, 51, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt648(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 5, 22, 7, 8, 15, 16, 17, 2, 19, 20, 19, 4, 21, 6, 23, 24, 63, 32, 33, 50, 35, 36, 35, 52, 37, 54, 39, 40, 47, 48, 49, 34, 51, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt649(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 5, 22, 7, 8, 31, 16, 17, 2, 19, 20, 19, 4, 21, 6, 23, 24, 47, 32, 33, 50, 35, 36, 35, 52, 37, 54, 39, 40, 63, 48, 49, 34, 51, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt650(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 5, 22, 7, 8, 15, 0, 17, 2, 19, 20, 19, 4, 21, 6, 23, 24, 63, 48, 33, 50, 35, 36, 35, 52, 37, 54, 39, 40, 47, 32, 49, 34, 51, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt651(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 5, 22, 7, 8, 31, 0, 17, 2, 19, 20, 19, 4, 21, 6, 23, 24, 47, 48, 33, 50, 35, 36, 35, 52, 37, 54, 39, 40, 63, 32, 49, 34, 51, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt652(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 5, 22, 7, 8, 15, 16, 1, 2, 19, 20, 19, 4, 21, 6, 23, 24, 63, 32, 49, 50, 35, 36, 35, 52, 37, 54, 39, 40, 47, 48, 33, 34, 51, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt653(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 5, 22, 7, 8, 31, 16, 1, 2, 19, 20, 19, 4, 21, 6, 23, 24, 47, 32, 49, 50, 35, 36, 35, 52, 37, 54, 39, 40, 63, 48, 33, 34, 51, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt654(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 5, 22, 7, 8, 15, 0, 1, 2, 19, 20, 19, 4, 21, 6, 23, 24, 63, 48, 49, 50, 35, 36, 35, 52, 37, 54, 39, 40, 47, 32, 33, 34, 51, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt655(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 5, 22, 7, 8, 31, 0, 1, 2, 19, 20, 19, 4, 21, 6, 23, 24, 47, 48, 49, 50, 35, 36, 35, 52, 37, 54, 39, 40, 63, 32, 33, 34, 51, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt656(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 5, 22, 7, 8, 15, 16, 17, 18, 3, 20, 19, 4, 21, 6, 23, 24, 63, 32, 33, 34, 51, 36, 35, 52, 37, 54, 39, 40, 47, 48, 49, 50, 35, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt657(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 5, 22, 7, 8, 31, 16, 17, 18, 3, 20, 19, 4, 21, 6, 23, 24, 47, 32, 33, 34, 51, 36, 35, 52, 37, 54, 39, 40, 63, 48, 49, 50, 35, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt658(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 5, 22, 7, 8, 15, 0, 17, 18, 3, 20, 19, 4, 21, 6, 23, 24, 63, 48, 33, 34, 51, 36, 35, 52, 37, 54, 39, 40, 47, 32, 49, 50, 35, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt659(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 5, 22, 7, 8, 31, 0, 17, 18, 3, 20, 19, 4, 21, 6, 23, 24, 47, 48, 33, 34, 51, 36, 35, 52, 37, 54, 39, 40, 63, 32, 49, 50, 35, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt660(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 5, 22, 7, 8, 15, 16, 1, 18, 3, 20, 19, 4, 21, 6, 23, 24, 63, 32, 49, 34, 51, 36, 35, 52, 37, 54, 39, 40, 47, 48, 33, 50, 35, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt661(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 5, 22, 7, 8, 31, 16, 1, 18, 3, 20, 19, 4, 21, 6, 23, 24, 47, 32, 49, 34, 51, 36, 35, 52, 37, 54, 39, 40, 63, 48, 33, 50, 35, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt662(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 5, 22, 7, 8, 15, 0, 1, 18, 3, 20, 19, 4, 21, 6, 23, 24, 63, 48, 49, 34, 51, 36, 35, 52, 37, 54, 39, 40, 47, 32, 33, 50, 35, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt663(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 5, 22, 7, 8, 31, 0, 1, 18, 3, 20, 19, 4, 21, 6, 23, 24, 47, 48, 49, 34, 51, 36, 35, 52, 37, 54, 39, 40, 63, 32, 33, 50, 35, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt664(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 5, 22, 7, 8, 15, 16, 17, 2, 3, 20, 19, 4, 21, 6, 23, 24, 63, 32, 33, 50, 51, 36, 35, 52, 37, 54, 39, 40, 47, 48, 49, 34, 35, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt665(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 5, 22, 7, 8, 31, 16, 17, 2, 3, 20, 19, 4, 21, 6, 23, 24, 47, 32, 33, 50, 51, 36, 35, 52, 37, 54, 39, 40, 63, 48, 49, 34, 35, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt666(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 5, 22, 7, 8, 15, 0, 17, 2, 3, 20, 19, 4, 21, 6, 23, 24, 63, 48, 33, 50, 51, 36, 35, 52, 37, 54, 39, 40, 47, 32, 49, 34, 35, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt667(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 5, 22, 7, 8, 31, 0, 17, 2, 3, 20, 19, 4, 21, 6, 23, 24, 47, 48, 33, 50, 51, 36, 35, 52, 37, 54, 39, 40, 63, 32, 49, 34, 35, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt668(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 5, 22, 7, 8, 15, 16, 1, 2, 3, 20, 19, 4, 21, 6, 23, 24, 63, 32, 49, 50, 51, 36, 35, 52, 37, 54, 39, 40, 47, 48, 33, 34, 35, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt669(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 5, 22, 7, 8, 31, 16, 1, 2, 3, 20, 19, 4, 21, 6, 23, 24, 47, 32, 49, 50, 51, 36, 35, 52, 37, 54, 39, 40, 63, 48, 33, 34, 35, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt670(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 5, 22, 7, 8, 15, 0, 1, 2, 3, 20, 19, 4, 21, 6, 23, 24, 63, 48, 49, 50, 51, 36, 35, 52, 37, 54, 39, 40, 47, 32, 33, 34, 35, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt671(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 5, 22, 7, 8, 31, 0, 1, 2, 3, 20, 19, 4, 21, 6, 23, 24, 47, 48, 49, 50, 51, 36, 35, 52, 37, 54, 39, 40, 63, 32, 33, 34, 35, 52, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt672(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 5, 22, 7, 8, 15, 16, 17, 18, 19, 4, 19, 4, 21, 6, 23, 24, 63, 32, 33, 34, 35, 52, 35, 52, 37, 54, 39, 40, 47, 48, 49, 50, 51, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt673(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 5, 22, 7, 8, 31, 16, 17, 18, 19, 4, 19, 4, 21, 6, 23, 24, 47, 32, 33, 34, 35, 52, 35, 52, 37, 54, 39, 40, 63, 48, 49, 50, 51, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt674(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 5, 22, 7, 8, 15, 0, 17, 18, 19, 4, 19, 4, 21, 6, 23, 24, 63, 48, 33, 34, 35, 52, 35, 52, 37, 54, 39, 40, 47, 32, 49, 50, 51, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt675(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 5, 22, 7, 8, 31, 0, 17, 18, 19, 4, 19, 4, 21, 6, 23, 24, 47, 48, 33, 34, 35, 52, 35, 52, 37, 54, 39, 40, 63, 32, 49, 50, 51, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt676(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 5, 22, 7, 8, 15, 16, 1, 18, 19, 4, 19, 4, 21, 6, 23, 24, 63, 32, 49, 34, 35, 52, 35, 52, 37, 54, 39, 40, 47, 48, 33, 50, 51, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt677(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 5, 22, 7, 8, 31, 16, 1, 18, 19, 4, 19, 4, 21, 6, 23, 24, 47, 32, 49, 34, 35, 52, 35, 52, 37, 54, 39, 40, 63, 48, 33, 50, 51, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt678(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 5, 22, 7, 8, 15, 0, 1, 18, 19, 4, 19, 4, 21, 6, 23, 24, 63, 48, 49, 34, 35, 52, 35, 52, 37, 54, 39, 40, 47, 32, 33, 50, 51, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt679(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 5, 22, 7, 8, 31, 0, 1, 18, 19, 4, 19, 4, 21, 6, 23, 24, 47, 48, 49, 34, 35, 52, 35, 52, 37, 54, 39, 40, 63, 32, 33, 50, 51, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt680(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 5, 22, 7, 8, 15, 16, 17, 2, 19, 4, 19, 4, 21, 6, 23, 24, 63, 32, 33, 50, 35, 52, 35, 52, 37, 54, 39, 40, 47, 48, 49, 34, 51, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt681(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 5, 22, 7, 8, 31, 16, 17, 2, 19, 4, 19, 4, 21, 6, 23, 24, 47, 32, 33, 50, 35, 52, 35, 52, 37, 54, 39, 40, 63, 48, 49, 34, 51, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt682(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 5, 22, 7, 8, 15, 0, 17, 2, 19, 4, 19, 4, 21, 6, 23, 24, 63, 48, 33, 50, 35, 52, 35, 52, 37, 54, 39, 40, 47, 32, 49, 34, 51, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt683(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 5, 22, 7, 8, 31, 0, 17, 2, 19, 4, 19, 4, 21, 6, 23, 24, 47, 48, 33, 50, 35, 52, 35, 52, 37, 54, 39, 40, 63, 32, 49, 34, 51, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt684(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 5, 22, 7, 8, 15, 16, 1, 2, 19, 4, 19, 4, 21, 6, 23, 24, 63, 32, 49, 50, 35, 52, 35, 52, 37, 54, 39, 40, 47, 48, 33, 34, 51, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt685(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 5, 22, 7, 8, 31, 16, 1, 2, 19, 4, 19, 4, 21, 6, 23, 24, 47, 32, 49, 50, 35, 52, 35, 52, 37, 54, 39, 40, 63, 48, 33, 34, 51, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt686(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 5, 22, 7, 8, 15, 0, 1, 2, 19, 4, 19, 4, 21, 6, 23, 24, 63, 48, 49, 50, 35, 52, 35, 52, 37, 54, 39, 40, 47, 32, 33, 34, 51, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt687(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 5, 22, 7, 8, 31, 0, 1, 2, 19, 4, 19, 4, 21, 6, 23, 24, 47, 48, 49, 50, 35, 52, 35, 52, 37, 54, 39, 40, 63, 32, 33, 34, 51, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt688(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 5, 22, 7, 8, 15, 16, 17, 18, 3, 4, 19, 4, 21, 6, 23, 24, 63, 32, 33, 34, 51, 52, 35, 52, 37, 54, 39, 40, 47, 48, 49, 50, 35, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt689(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 5, 22, 7, 8, 31, 16, 17, 18, 3, 4, 19, 4, 21, 6, 23, 24, 47, 32, 33, 34, 51, 52, 35, 52, 37, 54, 39, 40, 63, 48, 49, 50, 35, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt690(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 5, 22, 7, 8, 15, 0, 17, 18, 3, 4, 19, 4, 21, 6, 23, 24, 63, 48, 33, 34, 51, 52, 35, 52, 37, 54, 39, 40, 47, 32, 49, 50, 35, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt691(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 5, 22, 7, 8, 31, 0, 17, 18, 3, 4, 19, 4, 21, 6, 23, 24, 47, 48, 33, 34, 51, 52, 35, 52, 37, 54, 39, 40, 63, 32, 49, 50, 35, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt692(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 5, 22, 7, 8, 15, 16, 1, 18, 3, 4, 19, 4, 21, 6, 23, 24, 63, 32, 49, 34, 51, 52, 35, 52, 37, 54, 39, 40, 47, 48, 33, 50, 35, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt693(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 5, 22, 7, 8, 31, 16, 1, 18, 3, 4, 19, 4, 21, 6, 23, 24, 47, 32, 49, 34, 51, 52, 35, 52, 37, 54, 39, 40, 63, 48, 33, 50, 35, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt694(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 5, 22, 7, 8, 15, 0, 1, 18, 3, 4, 19, 4, 21, 6, 23, 24, 63, 48, 49, 34, 51, 52, 35, 52, 37, 54, 39, 40, 47, 32, 33, 50, 35, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt695(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 5, 22, 7, 8, 31, 0, 1, 18, 3, 4, 19, 4, 21, 6, 23, 24, 47, 48, 49, 34, 51, 52, 35, 52, 37, 54, 39, 40, 63, 32, 33, 50, 35, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt696(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 5, 22, 7, 8, 15, 16, 17, 2, 3, 4, 19, 4, 21, 6, 23, 24, 63, 32, 33, 50, 51, 52, 35, 52, 37, 54, 39, 40, 47, 48, 49, 34, 35, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt697(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 5, 22, 7, 8, 31, 16, 17, 2, 3, 4, 19, 4, 21, 6, 23, 24, 47, 32, 33, 50, 51, 52, 35, 52, 37, 54, 39, 40, 63, 48, 49, 34, 35, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt698(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 5, 22, 7, 8, 15, 0, 17, 2, 3, 4, 19, 4, 21, 6, 23, 24, 63, 48, 33, 50, 51, 52, 35, 52, 37, 54, 39, 40, 47, 32, 49, 34, 35, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt699(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 5, 22, 7, 8, 31, 0, 17, 2, 3, 4, 19, 4, 21, 6, 23, 24, 47, 48, 33, 50, 51, 52, 35, 52, 37, 54, 39, 40, 63, 32, 49, 34, 35, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt700(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 5, 22, 7, 8, 15, 16, 1, 2, 3, 4, 19, 4, 21, 6, 23, 24, 63, 32, 49, 50, 51, 52, 35, 52, 37, 54, 39, 40, 47, 48, 33, 34, 35, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt701(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 5, 22, 7, 8, 31, 16, 1, 2, 3, 4, 19, 4, 21, 6, 23, 24, 47, 32, 49, 50, 51, 52, 35, 52, 37, 54, 39, 40, 63, 48, 33, 34, 35, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt702(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 5, 22, 7, 8, 15, 0, 1, 2, 3, 4, 19, 4, 21, 6, 23, 24, 63, 48, 49, 50, 51, 52, 35, 52, 37, 54, 39, 40, 47, 32, 33, 34, 35, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt703(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 5, 22, 7, 8, 31, 0, 1, 2, 3, 4, 19, 4, 21, 6, 23, 24, 47, 48, 49, 50, 51, 52, 35, 52, 37, 54, 39, 40, 63, 32, 33, 34, 35, 36, 51, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt704(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 5, 22, 7, 8, 15, 16, 17, 18, 19, 20, 3, 4, 21, 6, 23, 24, 63, 32, 33, 34, 35, 36, 51, 52, 37, 54, 39, 40, 47, 48, 49, 50, 51, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt705(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 5, 22, 7, 8, 31, 16, 17, 18, 19, 20, 3, 4, 21, 6, 23, 24, 47, 32, 33, 34, 35, 36, 51, 52, 37, 54, 39, 40, 63, 48, 49, 50, 51, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt706(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 5, 22, 7, 8, 15, 0, 17, 18, 19, 20, 3, 4, 21, 6, 23, 24, 63, 48, 33, 34, 35, 36, 51, 52, 37, 54, 39, 40, 47, 32, 49, 50, 51, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt707(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 5, 22, 7, 8, 31, 0, 17, 18, 19, 20, 3, 4, 21, 6, 23, 24, 47, 48, 33, 34, 35, 36, 51, 52, 37, 54, 39, 40, 63, 32, 49, 50, 51, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt708(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 5, 22, 7, 8, 15, 16, 1, 18, 19, 20, 3, 4, 21, 6, 23, 24, 63, 32, 49, 34, 35, 36, 51, 52, 37, 54, 39, 40, 47, 48, 33, 50, 51, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt709(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 5, 22, 7, 8, 31, 16, 1, 18, 19, 20, 3, 4, 21, 6, 23, 24, 47, 32, 49, 34, 35, 36, 51, 52, 37, 54, 39, 40, 63, 48, 33, 50, 51, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt710(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 5, 22, 7, 8, 15, 0, 1, 18, 19, 20, 3, 4, 21, 6, 23, 24, 63, 48, 49, 34, 35, 36, 51, 52, 37, 54, 39, 40, 47, 32, 33, 50, 51, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt711(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 5, 22, 7, 8, 31, 0, 1, 18, 19, 20, 3, 4, 21, 6, 23, 24, 47, 48, 49, 34, 35, 36, 51, 52, 37, 54, 39, 40, 63, 32, 33, 50, 51, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt712(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 5, 22, 7, 8, 15, 16, 17, 2, 19, 20, 3, 4, 21, 6, 23, 24, 63, 32, 33, 50, 35, 36, 51, 52, 37, 54, 39, 40, 47, 48, 49, 34, 51, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt713(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 5, 22, 7, 8, 31, 16, 17, 2, 19, 20, 3, 4, 21, 6, 23, 24, 47, 32, 33, 50, 35, 36, 51, 52, 37, 54, 39, 40, 63, 48, 49, 34, 51, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt714(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 5, 22, 7, 8, 15, 0, 17, 2, 19, 20, 3, 4, 21, 6, 23, 24, 63, 48, 33, 50, 35, 36, 51, 52, 37, 54, 39, 40, 47, 32, 49, 34, 51, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt715(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 5, 22, 7, 8, 31, 0, 17, 2, 19, 20, 3, 4, 21, 6, 23, 24, 47, 48, 33, 50, 35, 36, 51, 52, 37, 54, 39, 40, 63, 32, 49, 34, 51, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt716(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 5, 22, 7, 8, 15, 16, 1, 2, 19, 20, 3, 4, 21, 6, 23, 24, 63, 32, 49, 50, 35, 36, 51, 52, 37, 54, 39, 40, 47, 48, 33, 34, 51, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt717(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 5, 22, 7, 8, 31, 16, 1, 2, 19, 20, 3, 4, 21, 6, 23, 24, 47, 32, 49, 50, 35, 36, 51, 52, 37, 54, 39, 40, 63, 48, 33, 34, 51, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt718(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 5, 22, 7, 8, 15, 0, 1, 2, 19, 20, 3, 4, 21, 6, 23, 24, 63, 48, 49, 50, 35, 36, 51, 52, 37, 54, 39, 40, 47, 32, 33, 34, 51, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt719(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 5, 22, 7, 8, 31, 0, 1, 2, 19, 20, 3, 4, 21, 6, 23, 24, 47, 48, 49, 50, 35, 36, 51, 52, 37, 54, 39, 40, 63, 32, 33, 34, 51, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt720(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 5, 22, 7, 8, 15, 16, 17, 18, 3, 20, 3, 4, 21, 6, 23, 24, 63, 32, 33, 34, 51, 36, 51, 52, 37, 54, 39, 40, 47, 48, 49, 50, 35, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt721(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 5, 22, 7, 8, 31, 16, 17, 18, 3, 20, 3, 4, 21, 6, 23, 24, 47, 32, 33, 34, 51, 36, 51, 52, 37, 54, 39, 40, 63, 48, 49, 50, 35, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt722(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 5, 22, 7, 8, 15, 0, 17, 18, 3, 20, 3, 4, 21, 6, 23, 24, 63, 48, 33, 34, 51, 36, 51, 52, 37, 54, 39, 40, 47, 32, 49, 50, 35, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt723(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 5, 22, 7, 8, 31, 0, 17, 18, 3, 20, 3, 4, 21, 6, 23, 24, 47, 48, 33, 34, 51, 36, 51, 52, 37, 54, 39, 40, 63, 32, 49, 50, 35, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt724(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 5, 22, 7, 8, 15, 16, 1, 18, 3, 20, 3, 4, 21, 6, 23, 24, 63, 32, 49, 34, 51, 36, 51, 52, 37, 54, 39, 40, 47, 48, 33, 50, 35, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt725(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 5, 22, 7, 8, 31, 16, 1, 18, 3, 20, 3, 4, 21, 6, 23, 24, 47, 32, 49, 34, 51, 36, 51, 52, 37, 54, 39, 40, 63, 48, 33, 50, 35, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt726(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 5, 22, 7, 8, 15, 0, 1, 18, 3, 20, 3, 4, 21, 6, 23, 24, 63, 48, 49, 34, 51, 36, 51, 52, 37, 54, 39, 40, 47, 32, 33, 50, 35, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt727(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 5, 22, 7, 8, 31, 0, 1, 18, 3, 20, 3, 4, 21, 6, 23, 24, 47, 48, 49, 34, 51, 36, 51, 52, 37, 54, 39, 40, 63, 32, 33, 50, 35, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt728(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 5, 22, 7, 8, 15, 16, 17, 2, 3, 20, 3, 4, 21, 6, 23, 24, 63, 32, 33, 50, 51, 36, 51, 52, 37, 54, 39, 40, 47, 48, 49, 34, 35, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt729(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 5, 22, 7, 8, 31, 16, 17, 2, 3, 20, 3, 4, 21, 6, 23, 24, 47, 32, 33, 50, 51, 36, 51, 52, 37, 54, 39, 40, 63, 48, 49, 34, 35, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt730(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 5, 22, 7, 8, 15, 0, 17, 2, 3, 20, 3, 4, 21, 6, 23, 24, 63, 48, 33, 50, 51, 36, 51, 52, 37, 54, 39, 40, 47, 32, 49, 34, 35, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt731(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 5, 22, 7, 8, 31, 0, 17, 2, 3, 20, 3, 4, 21, 6, 23, 24, 47, 48, 33, 50, 51, 36, 51, 52, 37, 54, 39, 40, 63, 32, 49, 34, 35, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt732(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 5, 22, 7, 8, 15, 16, 1, 2, 3, 20, 3, 4, 21, 6, 23, 24, 63, 32, 49, 50, 51, 36, 51, 52, 37, 54, 39, 40, 47, 48, 33, 34, 35, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt733(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 5, 22, 7, 8, 31, 16, 1, 2, 3, 20, 3, 4, 21, 6, 23, 24, 47, 32, 49, 50, 51, 36, 51, 52, 37, 54, 39, 40, 63, 48, 33, 34, 35, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt734(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 5, 22, 7, 8, 15, 0, 1, 2, 3, 20, 3, 4, 21, 6, 23, 24, 63, 48, 49, 50, 51, 36, 51, 52, 37, 54, 39, 40, 47, 32, 33, 34, 35, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt735(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 5, 22, 7, 8, 31, 0, 1, 2, 3, 20, 3, 4, 21, 6, 23, 24, 47, 48, 49, 50, 51, 36, 51, 52, 37, 54, 39, 40, 63, 32, 33, 34, 35, 52, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt736(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 5, 22, 7, 8, 15, 16, 17, 18, 19, 4, 3, 4, 21, 6, 23, 24, 63, 32, 33, 34, 35, 52, 51, 52, 37, 54, 39, 40, 47, 48, 49, 50, 51, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt737(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 5, 22, 7, 8, 31, 16, 17, 18, 19, 4, 3, 4, 21, 6, 23, 24, 47, 32, 33, 34, 35, 52, 51, 52, 37, 54, 39, 40, 63, 48, 49, 50, 51, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt738(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 5, 22, 7, 8, 15, 0, 17, 18, 19, 4, 3, 4, 21, 6, 23, 24, 63, 48, 33, 34, 35, 52, 51, 52, 37, 54, 39, 40, 47, 32, 49, 50, 51, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt739(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 5, 22, 7, 8, 31, 0, 17, 18, 19, 4, 3, 4, 21, 6, 23, 24, 47, 48, 33, 34, 35, 52, 51, 52, 37, 54, 39, 40, 63, 32, 49, 50, 51, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt740(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 5, 22, 7, 8, 15, 16, 1, 18, 19, 4, 3, 4, 21, 6, 23, 24, 63, 32, 49, 34, 35, 52, 51, 52, 37, 54, 39, 40, 47, 48, 33, 50, 51, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt741(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 5, 22, 7, 8, 31, 16, 1, 18, 19, 4, 3, 4, 21, 6, 23, 24, 47, 32, 49, 34, 35, 52, 51, 52, 37, 54, 39, 40, 63, 48, 33, 50, 51, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt742(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 5, 22, 7, 8, 15, 0, 1, 18, 19, 4, 3, 4, 21, 6, 23, 24, 63, 48, 49, 34, 35, 52, 51, 52, 37, 54, 39, 40, 47, 32, 33, 50, 51, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt743(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 5, 22, 7, 8, 31, 0, 1, 18, 19, 4, 3, 4, 21, 6, 23, 24, 47, 48, 49, 34, 35, 52, 51, 52, 37, 54, 39, 40, 63, 32, 33, 50, 51, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt744(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 5, 22, 7, 8, 15, 16, 17, 2, 19, 4, 3, 4, 21, 6, 23, 24, 63, 32, 33, 50, 35, 52, 51, 52, 37, 54, 39, 40, 47, 48, 49, 34, 51, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt745(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 5, 22, 7, 8, 31, 16, 17, 2, 19, 4, 3, 4, 21, 6, 23, 24, 47, 32, 33, 50, 35, 52, 51, 52, 37, 54, 39, 40, 63, 48, 49, 34, 51, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt746(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 5, 22, 7, 8, 15, 0, 17, 2, 19, 4, 3, 4, 21, 6, 23, 24, 63, 48, 33, 50, 35, 52, 51, 52, 37, 54, 39, 40, 47, 32, 49, 34, 51, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt747(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 5, 22, 7, 8, 31, 0, 17, 2, 19, 4, 3, 4, 21, 6, 23, 24, 47, 48, 33, 50, 35, 52, 51, 52, 37, 54, 39, 40, 63, 32, 49, 34, 51, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt748(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 5, 22, 7, 8, 15, 16, 1, 2, 19, 4, 3, 4, 21, 6, 23, 24, 63, 32, 49, 50, 35, 52, 51, 52, 37, 54, 39, 40, 47, 48, 33, 34, 51, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt749(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 5, 22, 7, 8, 31, 16, 1, 2, 19, 4, 3, 4, 21, 6, 23, 24, 47, 32, 49, 50, 35, 52, 51, 52, 37, 54, 39, 40, 63, 48, 33, 34, 51, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt750(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 5, 22, 7, 8, 15, 0, 1, 2, 19, 4, 3, 4, 21, 6, 23, 24, 63, 48, 49, 50, 35, 52, 51, 52, 37, 54, 39, 40, 47, 32, 33, 34, 51, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt751(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 5, 22, 7, 8, 31, 0, 1, 2, 19, 4, 3, 4, 21, 6, 23, 24, 47, 48, 49, 50, 35, 52, 51, 52, 37, 54, 39, 40, 63, 32, 33, 34, 51, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt752(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 5, 22, 7, 8, 15, 16, 17, 18, 3, 4, 3, 4, 21, 6, 23, 24, 63, 32, 33, 34, 51, 52, 51, 52, 37, 54, 39, 40, 47, 48, 49, 50, 35, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt753(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 5, 22, 7, 8, 31, 16, 17, 18, 3, 4, 3, 4, 21, 6, 23, 24, 47, 32, 33, 34, 51, 52, 51, 52, 37, 54, 39, 40, 63, 48, 49, 50, 35, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt754(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 5, 22, 7, 8, 15, 0, 17, 18, 3, 4, 3, 4, 21, 6, 23, 24, 63, 48, 33, 34, 51, 52, 51, 52, 37, 54, 39, 40, 47, 32, 49, 50, 35, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt755(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 5, 22, 7, 8, 31, 0, 17, 18, 3, 4, 3, 4, 21, 6, 23, 24, 47, 48, 33, 34, 51, 52, 51, 52, 37, 54, 39, 40, 63, 32, 49, 50, 35, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt756(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 5, 22, 7, 8, 15, 16, 1, 18, 3, 4, 3, 4, 21, 6, 23, 24, 63, 32, 49, 34, 51, 52, 51, 52, 37, 54, 39, 40, 47, 48, 33, 50, 35, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt757(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 5, 22, 7, 8, 31, 16, 1, 18, 3, 4, 3, 4, 21, 6, 23, 24, 47, 32, 49, 34, 51, 52, 51, 52, 37, 54, 39, 40, 63, 48, 33, 50, 35, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt758(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 5, 22, 7, 8, 15, 0, 1, 18, 3, 4, 3, 4, 21, 6, 23, 24, 63, 48, 49, 34, 51, 52, 51, 52, 37, 54, 39, 40, 47, 32, 33, 50, 35, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt759(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 5, 22, 7, 8, 31, 0, 1, 18, 3, 4, 3, 4, 21, 6, 23, 24, 47, 48, 49, 34, 51, 52, 51, 52, 37, 54, 39, 40, 63, 32, 33, 50, 35, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt760(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 5, 22, 7, 8, 15, 16, 17, 2, 3, 4, 3, 4, 21, 6, 23, 24, 63, 32, 33, 50, 51, 52, 51, 52, 37, 54, 39, 40, 47, 48, 49, 34, 35, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt761(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 5, 22, 7, 8, 31, 16, 17, 2, 3, 4, 3, 4, 21, 6, 23, 24, 47, 32, 33, 50, 51, 52, 51, 52, 37, 54, 39, 40, 63, 48, 49, 34, 35, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt762(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 5, 22, 7, 8, 15, 0, 17, 2, 3, 4, 3, 4, 21, 6, 23, 24, 63, 48, 33, 50, 51, 52, 51, 52, 37, 54, 39, 40, 47, 32, 49, 34, 35, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt763(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 5, 22, 7, 8, 31, 0, 17, 2, 3, 4, 3, 4, 21, 6, 23, 24, 47, 48, 33, 50, 51, 52, 51, 52, 37, 54, 39, 40, 63, 32, 49, 34, 35, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt764(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 5, 22, 7, 8, 15, 16, 1, 2, 3, 4, 3, 4, 21, 6, 23, 24, 63, 32, 49, 50, 51, 52, 51, 52, 37, 54, 39, 40, 47, 48, 33, 34, 35, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt765(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 5, 22, 7, 8, 31, 16, 1, 2, 3, 4, 3, 4, 21, 6, 23, 24, 47, 32, 49, 50, 51, 52, 51, 52, 37, 54, 39, 40, 63, 48, 33, 34, 35, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt766(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 5, 22, 7, 8, 15, 0, 1, 2, 3, 4, 3, 4, 21, 6, 23, 24, 63, 48, 49, 50, 51, 52, 51, 52, 37, 54, 39, 40, 47, 32, 33, 34, 35, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt767(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 5, 22, 7, 8, 31, 0, 1, 2, 3, 4, 3, 4, 21, 6, 23, 24, 47, 48, 49, 50, 51, 52, 51, 52, 37, 54, 39, 40, 63, 32, 33, 34, 35, 36, 35, 36, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}


