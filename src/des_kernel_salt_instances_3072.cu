#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

#ifdef DESGPU_COMPILE_ALL_SALTS

void des_25_encrypt_salt3072(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 4, 5, 6, 23, 24, 15, 16, 17, 18, 19, 20, 19, 20, 21, 22, 7, 8, 63, 32, 33, 34, 35, 36, 35, 36, 37, 38, 55, 56, 47, 48, 49, 50, 51, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3073(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 4, 5, 6, 23, 24, 31, 16, 17, 18, 19, 20, 19, 20, 21, 22, 7, 8, 47, 32, 33, 34, 35, 36, 35, 36, 37, 38, 55, 56, 63, 48, 49, 50, 51, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3074(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 4, 5, 6, 23, 24, 15, 0, 17, 18, 19, 20, 19, 20, 21, 22, 7, 8, 63, 48, 33, 34, 35, 36, 35, 36, 37, 38, 55, 56, 47, 32, 49, 50, 51, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3075(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 4, 5, 6, 23, 24, 31, 0, 17, 18, 19, 20, 19, 20, 21, 22, 7, 8, 47, 48, 33, 34, 35, 36, 35, 36, 37, 38, 55, 56, 63, 32, 49, 50, 51, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3076(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 4, 5, 6, 23, 24, 15, 16, 1, 18, 19, 20, 19, 20, 21, 22, 7, 8, 63, 32, 49, 34, 35, 36, 35, 36, 37, 38, 55, 56, 47, 48, 33, 50, 51, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3077(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 4, 5, 6, 23, 24, 31, 16, 1, 18, 19, 20, 19, 20, 21, 22, 7, 8, 47, 32, 49, 34, 35, 36, 35, 36, 37, 38, 55, 56, 63, 48, 33, 50, 51, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3078(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 4, 5, 6, 23, 24, 15, 0, 1, 18, 19, 20, 19, 20, 21, 22, 7, 8, 63, 48, 49, 34, 35, 36, 35, 36, 37, 38, 55, 56, 47, 32, 33, 50, 51, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3079(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 4, 5, 6, 23, 24, 31, 0, 1, 18, 19, 20, 19, 20, 21, 22, 7, 8, 47, 48, 49, 34, 35, 36, 35, 36, 37, 38, 55, 56, 63, 32, 33, 50, 51, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3080(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 4, 5, 6, 23, 24, 15, 16, 17, 2, 19, 20, 19, 20, 21, 22, 7, 8, 63, 32, 33, 50, 35, 36, 35, 36, 37, 38, 55, 56, 47, 48, 49, 34, 51, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3081(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 4, 5, 6, 23, 24, 31, 16, 17, 2, 19, 20, 19, 20, 21, 22, 7, 8, 47, 32, 33, 50, 35, 36, 35, 36, 37, 38, 55, 56, 63, 48, 49, 34, 51, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3082(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 4, 5, 6, 23, 24, 15, 0, 17, 2, 19, 20, 19, 20, 21, 22, 7, 8, 63, 48, 33, 50, 35, 36, 35, 36, 37, 38, 55, 56, 47, 32, 49, 34, 51, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3083(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 4, 5, 6, 23, 24, 31, 0, 17, 2, 19, 20, 19, 20, 21, 22, 7, 8, 47, 48, 33, 50, 35, 36, 35, 36, 37, 38, 55, 56, 63, 32, 49, 34, 51, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3084(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 4, 5, 6, 23, 24, 15, 16, 1, 2, 19, 20, 19, 20, 21, 22, 7, 8, 63, 32, 49, 50, 35, 36, 35, 36, 37, 38, 55, 56, 47, 48, 33, 34, 51, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3085(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 4, 5, 6, 23, 24, 31, 16, 1, 2, 19, 20, 19, 20, 21, 22, 7, 8, 47, 32, 49, 50, 35, 36, 35, 36, 37, 38, 55, 56, 63, 48, 33, 34, 51, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3086(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 4, 5, 6, 23, 24, 15, 0, 1, 2, 19, 20, 19, 20, 21, 22, 7, 8, 63, 48, 49, 50, 35, 36, 35, 36, 37, 38, 55, 56, 47, 32, 33, 34, 51, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3087(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 4, 5, 6, 23, 24, 31, 0, 1, 2, 19, 20, 19, 20, 21, 22, 7, 8, 47, 48, 49, 50, 35, 36, 35, 36, 37, 38, 55, 56, 63, 32, 33, 34, 51, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3088(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 4, 5, 6, 23, 24, 15, 16, 17, 18, 3, 20, 19, 20, 21, 22, 7, 8, 63, 32, 33, 34, 51, 36, 35, 36, 37, 38, 55, 56, 47, 48, 49, 50, 35, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3089(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 4, 5, 6, 23, 24, 31, 16, 17, 18, 3, 20, 19, 20, 21, 22, 7, 8, 47, 32, 33, 34, 51, 36, 35, 36, 37, 38, 55, 56, 63, 48, 49, 50, 35, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3090(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 4, 5, 6, 23, 24, 15, 0, 17, 18, 3, 20, 19, 20, 21, 22, 7, 8, 63, 48, 33, 34, 51, 36, 35, 36, 37, 38, 55, 56, 47, 32, 49, 50, 35, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3091(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 4, 5, 6, 23, 24, 31, 0, 17, 18, 3, 20, 19, 20, 21, 22, 7, 8, 47, 48, 33, 34, 51, 36, 35, 36, 37, 38, 55, 56, 63, 32, 49, 50, 35, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3092(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 4, 5, 6, 23, 24, 15, 16, 1, 18, 3, 20, 19, 20, 21, 22, 7, 8, 63, 32, 49, 34, 51, 36, 35, 36, 37, 38, 55, 56, 47, 48, 33, 50, 35, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3093(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 4, 5, 6, 23, 24, 31, 16, 1, 18, 3, 20, 19, 20, 21, 22, 7, 8, 47, 32, 49, 34, 51, 36, 35, 36, 37, 38, 55, 56, 63, 48, 33, 50, 35, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3094(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 4, 5, 6, 23, 24, 15, 0, 1, 18, 3, 20, 19, 20, 21, 22, 7, 8, 63, 48, 49, 34, 51, 36, 35, 36, 37, 38, 55, 56, 47, 32, 33, 50, 35, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3095(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 4, 5, 6, 23, 24, 31, 0, 1, 18, 3, 20, 19, 20, 21, 22, 7, 8, 47, 48, 49, 34, 51, 36, 35, 36, 37, 38, 55, 56, 63, 32, 33, 50, 35, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3096(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 4, 5, 6, 23, 24, 15, 16, 17, 2, 3, 20, 19, 20, 21, 22, 7, 8, 63, 32, 33, 50, 51, 36, 35, 36, 37, 38, 55, 56, 47, 48, 49, 34, 35, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3097(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 4, 5, 6, 23, 24, 31, 16, 17, 2, 3, 20, 19, 20, 21, 22, 7, 8, 47, 32, 33, 50, 51, 36, 35, 36, 37, 38, 55, 56, 63, 48, 49, 34, 35, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3098(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 4, 5, 6, 23, 24, 15, 0, 17, 2, 3, 20, 19, 20, 21, 22, 7, 8, 63, 48, 33, 50, 51, 36, 35, 36, 37, 38, 55, 56, 47, 32, 49, 34, 35, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3099(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 4, 5, 6, 23, 24, 31, 0, 17, 2, 3, 20, 19, 20, 21, 22, 7, 8, 47, 48, 33, 50, 51, 36, 35, 36, 37, 38, 55, 56, 63, 32, 49, 34, 35, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3100(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 4, 5, 6, 23, 24, 15, 16, 1, 2, 3, 20, 19, 20, 21, 22, 7, 8, 63, 32, 49, 50, 51, 36, 35, 36, 37, 38, 55, 56, 47, 48, 33, 34, 35, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3101(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 4, 5, 6, 23, 24, 31, 16, 1, 2, 3, 20, 19, 20, 21, 22, 7, 8, 47, 32, 49, 50, 51, 36, 35, 36, 37, 38, 55, 56, 63, 48, 33, 34, 35, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3102(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 4, 5, 6, 23, 24, 15, 0, 1, 2, 3, 20, 19, 20, 21, 22, 7, 8, 63, 48, 49, 50, 51, 36, 35, 36, 37, 38, 55, 56, 47, 32, 33, 34, 35, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3103(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 4, 5, 6, 23, 24, 31, 0, 1, 2, 3, 20, 19, 20, 21, 22, 7, 8, 47, 48, 49, 50, 51, 36, 35, 36, 37, 38, 55, 56, 63, 32, 33, 34, 35, 52, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3104(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 4, 5, 6, 23, 24, 15, 16, 17, 18, 19, 4, 19, 20, 21, 22, 7, 8, 63, 32, 33, 34, 35, 52, 35, 36, 37, 38, 55, 56, 47, 48, 49, 50, 51, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3105(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 4, 5, 6, 23, 24, 31, 16, 17, 18, 19, 4, 19, 20, 21, 22, 7, 8, 47, 32, 33, 34, 35, 52, 35, 36, 37, 38, 55, 56, 63, 48, 49, 50, 51, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3106(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 4, 5, 6, 23, 24, 15, 0, 17, 18, 19, 4, 19, 20, 21, 22, 7, 8, 63, 48, 33, 34, 35, 52, 35, 36, 37, 38, 55, 56, 47, 32, 49, 50, 51, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3107(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 4, 5, 6, 23, 24, 31, 0, 17, 18, 19, 4, 19, 20, 21, 22, 7, 8, 47, 48, 33, 34, 35, 52, 35, 36, 37, 38, 55, 56, 63, 32, 49, 50, 51, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3108(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 4, 5, 6, 23, 24, 15, 16, 1, 18, 19, 4, 19, 20, 21, 22, 7, 8, 63, 32, 49, 34, 35, 52, 35, 36, 37, 38, 55, 56, 47, 48, 33, 50, 51, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3109(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 4, 5, 6, 23, 24, 31, 16, 1, 18, 19, 4, 19, 20, 21, 22, 7, 8, 47, 32, 49, 34, 35, 52, 35, 36, 37, 38, 55, 56, 63, 48, 33, 50, 51, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3110(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 4, 5, 6, 23, 24, 15, 0, 1, 18, 19, 4, 19, 20, 21, 22, 7, 8, 63, 48, 49, 34, 35, 52, 35, 36, 37, 38, 55, 56, 47, 32, 33, 50, 51, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3111(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 4, 5, 6, 23, 24, 31, 0, 1, 18, 19, 4, 19, 20, 21, 22, 7, 8, 47, 48, 49, 34, 35, 52, 35, 36, 37, 38, 55, 56, 63, 32, 33, 50, 51, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3112(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 4, 5, 6, 23, 24, 15, 16, 17, 2, 19, 4, 19, 20, 21, 22, 7, 8, 63, 32, 33, 50, 35, 52, 35, 36, 37, 38, 55, 56, 47, 48, 49, 34, 51, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3113(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 4, 5, 6, 23, 24, 31, 16, 17, 2, 19, 4, 19, 20, 21, 22, 7, 8, 47, 32, 33, 50, 35, 52, 35, 36, 37, 38, 55, 56, 63, 48, 49, 34, 51, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3114(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 4, 5, 6, 23, 24, 15, 0, 17, 2, 19, 4, 19, 20, 21, 22, 7, 8, 63, 48, 33, 50, 35, 52, 35, 36, 37, 38, 55, 56, 47, 32, 49, 34, 51, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3115(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 4, 5, 6, 23, 24, 31, 0, 17, 2, 19, 4, 19, 20, 21, 22, 7, 8, 47, 48, 33, 50, 35, 52, 35, 36, 37, 38, 55, 56, 63, 32, 49, 34, 51, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3116(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 4, 5, 6, 23, 24, 15, 16, 1, 2, 19, 4, 19, 20, 21, 22, 7, 8, 63, 32, 49, 50, 35, 52, 35, 36, 37, 38, 55, 56, 47, 48, 33, 34, 51, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3117(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 4, 5, 6, 23, 24, 31, 16, 1, 2, 19, 4, 19, 20, 21, 22, 7, 8, 47, 32, 49, 50, 35, 52, 35, 36, 37, 38, 55, 56, 63, 48, 33, 34, 51, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3118(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 4, 5, 6, 23, 24, 15, 0, 1, 2, 19, 4, 19, 20, 21, 22, 7, 8, 63, 48, 49, 50, 35, 52, 35, 36, 37, 38, 55, 56, 47, 32, 33, 34, 51, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3119(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 4, 5, 6, 23, 24, 31, 0, 1, 2, 19, 4, 19, 20, 21, 22, 7, 8, 47, 48, 49, 50, 35, 52, 35, 36, 37, 38, 55, 56, 63, 32, 33, 34, 51, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3120(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 4, 5, 6, 23, 24, 15, 16, 17, 18, 3, 4, 19, 20, 21, 22, 7, 8, 63, 32, 33, 34, 51, 52, 35, 36, 37, 38, 55, 56, 47, 48, 49, 50, 35, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3121(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 4, 5, 6, 23, 24, 31, 16, 17, 18, 3, 4, 19, 20, 21, 22, 7, 8, 47, 32, 33, 34, 51, 52, 35, 36, 37, 38, 55, 56, 63, 48, 49, 50, 35, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3122(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 4, 5, 6, 23, 24, 15, 0, 17, 18, 3, 4, 19, 20, 21, 22, 7, 8, 63, 48, 33, 34, 51, 52, 35, 36, 37, 38, 55, 56, 47, 32, 49, 50, 35, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3123(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 4, 5, 6, 23, 24, 31, 0, 17, 18, 3, 4, 19, 20, 21, 22, 7, 8, 47, 48, 33, 34, 51, 52, 35, 36, 37, 38, 55, 56, 63, 32, 49, 50, 35, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3124(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 4, 5, 6, 23, 24, 15, 16, 1, 18, 3, 4, 19, 20, 21, 22, 7, 8, 63, 32, 49, 34, 51, 52, 35, 36, 37, 38, 55, 56, 47, 48, 33, 50, 35, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3125(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 4, 5, 6, 23, 24, 31, 16, 1, 18, 3, 4, 19, 20, 21, 22, 7, 8, 47, 32, 49, 34, 51, 52, 35, 36, 37, 38, 55, 56, 63, 48, 33, 50, 35, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3126(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 4, 5, 6, 23, 24, 15, 0, 1, 18, 3, 4, 19, 20, 21, 22, 7, 8, 63, 48, 49, 34, 51, 52, 35, 36, 37, 38, 55, 56, 47, 32, 33, 50, 35, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3127(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 4, 5, 6, 23, 24, 31, 0, 1, 18, 3, 4, 19, 20, 21, 22, 7, 8, 47, 48, 49, 34, 51, 52, 35, 36, 37, 38, 55, 56, 63, 32, 33, 50, 35, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3128(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 4, 5, 6, 23, 24, 15, 16, 17, 2, 3, 4, 19, 20, 21, 22, 7, 8, 63, 32, 33, 50, 51, 52, 35, 36, 37, 38, 55, 56, 47, 48, 49, 34, 35, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3129(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 4, 5, 6, 23, 24, 31, 16, 17, 2, 3, 4, 19, 20, 21, 22, 7, 8, 47, 32, 33, 50, 51, 52, 35, 36, 37, 38, 55, 56, 63, 48, 49, 34, 35, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3130(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 4, 5, 6, 23, 24, 15, 0, 17, 2, 3, 4, 19, 20, 21, 22, 7, 8, 63, 48, 33, 50, 51, 52, 35, 36, 37, 38, 55, 56, 47, 32, 49, 34, 35, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3131(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 4, 5, 6, 23, 24, 31, 0, 17, 2, 3, 4, 19, 20, 21, 22, 7, 8, 47, 48, 33, 50, 51, 52, 35, 36, 37, 38, 55, 56, 63, 32, 49, 34, 35, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3132(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 4, 5, 6, 23, 24, 15, 16, 1, 2, 3, 4, 19, 20, 21, 22, 7, 8, 63, 32, 49, 50, 51, 52, 35, 36, 37, 38, 55, 56, 47, 48, 33, 34, 35, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3133(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 4, 5, 6, 23, 24, 31, 16, 1, 2, 3, 4, 19, 20, 21, 22, 7, 8, 47, 32, 49, 50, 51, 52, 35, 36, 37, 38, 55, 56, 63, 48, 33, 34, 35, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3134(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 4, 5, 6, 23, 24, 15, 0, 1, 2, 3, 4, 19, 20, 21, 22, 7, 8, 63, 48, 49, 50, 51, 52, 35, 36, 37, 38, 55, 56, 47, 32, 33, 34, 35, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3135(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 4, 5, 6, 23, 24, 31, 0, 1, 2, 3, 4, 19, 20, 21, 22, 7, 8, 47, 48, 49, 50, 51, 52, 35, 36, 37, 38, 55, 56, 63, 32, 33, 34, 35, 36, 51, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3136(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 4, 5, 6, 23, 24, 15, 16, 17, 18, 19, 20, 3, 20, 21, 22, 7, 8, 63, 32, 33, 34, 35, 36, 51, 36, 37, 38, 55, 56, 47, 48, 49, 50, 51, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3137(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 4, 5, 6, 23, 24, 31, 16, 17, 18, 19, 20, 3, 20, 21, 22, 7, 8, 47, 32, 33, 34, 35, 36, 51, 36, 37, 38, 55, 56, 63, 48, 49, 50, 51, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3138(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 4, 5, 6, 23, 24, 15, 0, 17, 18, 19, 20, 3, 20, 21, 22, 7, 8, 63, 48, 33, 34, 35, 36, 51, 36, 37, 38, 55, 56, 47, 32, 49, 50, 51, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3139(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 4, 5, 6, 23, 24, 31, 0, 17, 18, 19, 20, 3, 20, 21, 22, 7, 8, 47, 48, 33, 34, 35, 36, 51, 36, 37, 38, 55, 56, 63, 32, 49, 50, 51, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3140(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 4, 5, 6, 23, 24, 15, 16, 1, 18, 19, 20, 3, 20, 21, 22, 7, 8, 63, 32, 49, 34, 35, 36, 51, 36, 37, 38, 55, 56, 47, 48, 33, 50, 51, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3141(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 4, 5, 6, 23, 24, 31, 16, 1, 18, 19, 20, 3, 20, 21, 22, 7, 8, 47, 32, 49, 34, 35, 36, 51, 36, 37, 38, 55, 56, 63, 48, 33, 50, 51, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3142(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 4, 5, 6, 23, 24, 15, 0, 1, 18, 19, 20, 3, 20, 21, 22, 7, 8, 63, 48, 49, 34, 35, 36, 51, 36, 37, 38, 55, 56, 47, 32, 33, 50, 51, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3143(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 4, 5, 6, 23, 24, 31, 0, 1, 18, 19, 20, 3, 20, 21, 22, 7, 8, 47, 48, 49, 34, 35, 36, 51, 36, 37, 38, 55, 56, 63, 32, 33, 50, 51, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3144(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 4, 5, 6, 23, 24, 15, 16, 17, 2, 19, 20, 3, 20, 21, 22, 7, 8, 63, 32, 33, 50, 35, 36, 51, 36, 37, 38, 55, 56, 47, 48, 49, 34, 51, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3145(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 4, 5, 6, 23, 24, 31, 16, 17, 2, 19, 20, 3, 20, 21, 22, 7, 8, 47, 32, 33, 50, 35, 36, 51, 36, 37, 38, 55, 56, 63, 48, 49, 34, 51, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3146(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 4, 5, 6, 23, 24, 15, 0, 17, 2, 19, 20, 3, 20, 21, 22, 7, 8, 63, 48, 33, 50, 35, 36, 51, 36, 37, 38, 55, 56, 47, 32, 49, 34, 51, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3147(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 4, 5, 6, 23, 24, 31, 0, 17, 2, 19, 20, 3, 20, 21, 22, 7, 8, 47, 48, 33, 50, 35, 36, 51, 36, 37, 38, 55, 56, 63, 32, 49, 34, 51, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3148(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 4, 5, 6, 23, 24, 15, 16, 1, 2, 19, 20, 3, 20, 21, 22, 7, 8, 63, 32, 49, 50, 35, 36, 51, 36, 37, 38, 55, 56, 47, 48, 33, 34, 51, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3149(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 4, 5, 6, 23, 24, 31, 16, 1, 2, 19, 20, 3, 20, 21, 22, 7, 8, 47, 32, 49, 50, 35, 36, 51, 36, 37, 38, 55, 56, 63, 48, 33, 34, 51, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3150(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 4, 5, 6, 23, 24, 15, 0, 1, 2, 19, 20, 3, 20, 21, 22, 7, 8, 63, 48, 49, 50, 35, 36, 51, 36, 37, 38, 55, 56, 47, 32, 33, 34, 51, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3151(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 4, 5, 6, 23, 24, 31, 0, 1, 2, 19, 20, 3, 20, 21, 22, 7, 8, 47, 48, 49, 50, 35, 36, 51, 36, 37, 38, 55, 56, 63, 32, 33, 34, 51, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3152(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 4, 5, 6, 23, 24, 15, 16, 17, 18, 3, 20, 3, 20, 21, 22, 7, 8, 63, 32, 33, 34, 51, 36, 51, 36, 37, 38, 55, 56, 47, 48, 49, 50, 35, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3153(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 4, 5, 6, 23, 24, 31, 16, 17, 18, 3, 20, 3, 20, 21, 22, 7, 8, 47, 32, 33, 34, 51, 36, 51, 36, 37, 38, 55, 56, 63, 48, 49, 50, 35, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3154(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 4, 5, 6, 23, 24, 15, 0, 17, 18, 3, 20, 3, 20, 21, 22, 7, 8, 63, 48, 33, 34, 51, 36, 51, 36, 37, 38, 55, 56, 47, 32, 49, 50, 35, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3155(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 4, 5, 6, 23, 24, 31, 0, 17, 18, 3, 20, 3, 20, 21, 22, 7, 8, 47, 48, 33, 34, 51, 36, 51, 36, 37, 38, 55, 56, 63, 32, 49, 50, 35, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3156(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 4, 5, 6, 23, 24, 15, 16, 1, 18, 3, 20, 3, 20, 21, 22, 7, 8, 63, 32, 49, 34, 51, 36, 51, 36, 37, 38, 55, 56, 47, 48, 33, 50, 35, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3157(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 4, 5, 6, 23, 24, 31, 16, 1, 18, 3, 20, 3, 20, 21, 22, 7, 8, 47, 32, 49, 34, 51, 36, 51, 36, 37, 38, 55, 56, 63, 48, 33, 50, 35, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3158(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 4, 5, 6, 23, 24, 15, 0, 1, 18, 3, 20, 3, 20, 21, 22, 7, 8, 63, 48, 49, 34, 51, 36, 51, 36, 37, 38, 55, 56, 47, 32, 33, 50, 35, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3159(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 4, 5, 6, 23, 24, 31, 0, 1, 18, 3, 20, 3, 20, 21, 22, 7, 8, 47, 48, 49, 34, 51, 36, 51, 36, 37, 38, 55, 56, 63, 32, 33, 50, 35, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3160(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 4, 5, 6, 23, 24, 15, 16, 17, 2, 3, 20, 3, 20, 21, 22, 7, 8, 63, 32, 33, 50, 51, 36, 51, 36, 37, 38, 55, 56, 47, 48, 49, 34, 35, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3161(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 4, 5, 6, 23, 24, 31, 16, 17, 2, 3, 20, 3, 20, 21, 22, 7, 8, 47, 32, 33, 50, 51, 36, 51, 36, 37, 38, 55, 56, 63, 48, 49, 34, 35, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3162(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 4, 5, 6, 23, 24, 15, 0, 17, 2, 3, 20, 3, 20, 21, 22, 7, 8, 63, 48, 33, 50, 51, 36, 51, 36, 37, 38, 55, 56, 47, 32, 49, 34, 35, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3163(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 4, 5, 6, 23, 24, 31, 0, 17, 2, 3, 20, 3, 20, 21, 22, 7, 8, 47, 48, 33, 50, 51, 36, 51, 36, 37, 38, 55, 56, 63, 32, 49, 34, 35, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3164(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 4, 5, 6, 23, 24, 15, 16, 1, 2, 3, 20, 3, 20, 21, 22, 7, 8, 63, 32, 49, 50, 51, 36, 51, 36, 37, 38, 55, 56, 47, 48, 33, 34, 35, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3165(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 4, 5, 6, 23, 24, 31, 16, 1, 2, 3, 20, 3, 20, 21, 22, 7, 8, 47, 32, 49, 50, 51, 36, 51, 36, 37, 38, 55, 56, 63, 48, 33, 34, 35, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3166(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 4, 5, 6, 23, 24, 15, 0, 1, 2, 3, 20, 3, 20, 21, 22, 7, 8, 63, 48, 49, 50, 51, 36, 51, 36, 37, 38, 55, 56, 47, 32, 33, 34, 35, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3167(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 4, 5, 6, 23, 24, 31, 0, 1, 2, 3, 20, 3, 20, 21, 22, 7, 8, 47, 48, 49, 50, 51, 36, 51, 36, 37, 38, 55, 56, 63, 32, 33, 34, 35, 52, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3168(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 4, 5, 6, 23, 24, 15, 16, 17, 18, 19, 4, 3, 20, 21, 22, 7, 8, 63, 32, 33, 34, 35, 52, 51, 36, 37, 38, 55, 56, 47, 48, 49, 50, 51, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3169(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 4, 5, 6, 23, 24, 31, 16, 17, 18, 19, 4, 3, 20, 21, 22, 7, 8, 47, 32, 33, 34, 35, 52, 51, 36, 37, 38, 55, 56, 63, 48, 49, 50, 51, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3170(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 4, 5, 6, 23, 24, 15, 0, 17, 18, 19, 4, 3, 20, 21, 22, 7, 8, 63, 48, 33, 34, 35, 52, 51, 36, 37, 38, 55, 56, 47, 32, 49, 50, 51, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3171(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 4, 5, 6, 23, 24, 31, 0, 17, 18, 19, 4, 3, 20, 21, 22, 7, 8, 47, 48, 33, 34, 35, 52, 51, 36, 37, 38, 55, 56, 63, 32, 49, 50, 51, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3172(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 4, 5, 6, 23, 24, 15, 16, 1, 18, 19, 4, 3, 20, 21, 22, 7, 8, 63, 32, 49, 34, 35, 52, 51, 36, 37, 38, 55, 56, 47, 48, 33, 50, 51, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3173(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 4, 5, 6, 23, 24, 31, 16, 1, 18, 19, 4, 3, 20, 21, 22, 7, 8, 47, 32, 49, 34, 35, 52, 51, 36, 37, 38, 55, 56, 63, 48, 33, 50, 51, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3174(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 4, 5, 6, 23, 24, 15, 0, 1, 18, 19, 4, 3, 20, 21, 22, 7, 8, 63, 48, 49, 34, 35, 52, 51, 36, 37, 38, 55, 56, 47, 32, 33, 50, 51, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3175(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 4, 5, 6, 23, 24, 31, 0, 1, 18, 19, 4, 3, 20, 21, 22, 7, 8, 47, 48, 49, 34, 35, 52, 51, 36, 37, 38, 55, 56, 63, 32, 33, 50, 51, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3176(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 4, 5, 6, 23, 24, 15, 16, 17, 2, 19, 4, 3, 20, 21, 22, 7, 8, 63, 32, 33, 50, 35, 52, 51, 36, 37, 38, 55, 56, 47, 48, 49, 34, 51, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3177(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 4, 5, 6, 23, 24, 31, 16, 17, 2, 19, 4, 3, 20, 21, 22, 7, 8, 47, 32, 33, 50, 35, 52, 51, 36, 37, 38, 55, 56, 63, 48, 49, 34, 51, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3178(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 4, 5, 6, 23, 24, 15, 0, 17, 2, 19, 4, 3, 20, 21, 22, 7, 8, 63, 48, 33, 50, 35, 52, 51, 36, 37, 38, 55, 56, 47, 32, 49, 34, 51, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3179(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 4, 5, 6, 23, 24, 31, 0, 17, 2, 19, 4, 3, 20, 21, 22, 7, 8, 47, 48, 33, 50, 35, 52, 51, 36, 37, 38, 55, 56, 63, 32, 49, 34, 51, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3180(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 4, 5, 6, 23, 24, 15, 16, 1, 2, 19, 4, 3, 20, 21, 22, 7, 8, 63, 32, 49, 50, 35, 52, 51, 36, 37, 38, 55, 56, 47, 48, 33, 34, 51, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3181(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 4, 5, 6, 23, 24, 31, 16, 1, 2, 19, 4, 3, 20, 21, 22, 7, 8, 47, 32, 49, 50, 35, 52, 51, 36, 37, 38, 55, 56, 63, 48, 33, 34, 51, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3182(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 4, 5, 6, 23, 24, 15, 0, 1, 2, 19, 4, 3, 20, 21, 22, 7, 8, 63, 48, 49, 50, 35, 52, 51, 36, 37, 38, 55, 56, 47, 32, 33, 34, 51, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3183(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 4, 5, 6, 23, 24, 31, 0, 1, 2, 19, 4, 3, 20, 21, 22, 7, 8, 47, 48, 49, 50, 35, 52, 51, 36, 37, 38, 55, 56, 63, 32, 33, 34, 51, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3184(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 4, 5, 6, 23, 24, 15, 16, 17, 18, 3, 4, 3, 20, 21, 22, 7, 8, 63, 32, 33, 34, 51, 52, 51, 36, 37, 38, 55, 56, 47, 48, 49, 50, 35, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3185(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 4, 5, 6, 23, 24, 31, 16, 17, 18, 3, 4, 3, 20, 21, 22, 7, 8, 47, 32, 33, 34, 51, 52, 51, 36, 37, 38, 55, 56, 63, 48, 49, 50, 35, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3186(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 4, 5, 6, 23, 24, 15, 0, 17, 18, 3, 4, 3, 20, 21, 22, 7, 8, 63, 48, 33, 34, 51, 52, 51, 36, 37, 38, 55, 56, 47, 32, 49, 50, 35, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3187(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 4, 5, 6, 23, 24, 31, 0, 17, 18, 3, 4, 3, 20, 21, 22, 7, 8, 47, 48, 33, 34, 51, 52, 51, 36, 37, 38, 55, 56, 63, 32, 49, 50, 35, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3188(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 4, 5, 6, 23, 24, 15, 16, 1, 18, 3, 4, 3, 20, 21, 22, 7, 8, 63, 32, 49, 34, 51, 52, 51, 36, 37, 38, 55, 56, 47, 48, 33, 50, 35, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3189(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 4, 5, 6, 23, 24, 31, 16, 1, 18, 3, 4, 3, 20, 21, 22, 7, 8, 47, 32, 49, 34, 51, 52, 51, 36, 37, 38, 55, 56, 63, 48, 33, 50, 35, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3190(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 4, 5, 6, 23, 24, 15, 0, 1, 18, 3, 4, 3, 20, 21, 22, 7, 8, 63, 48, 49, 34, 51, 52, 51, 36, 37, 38, 55, 56, 47, 32, 33, 50, 35, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3191(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 4, 5, 6, 23, 24, 31, 0, 1, 18, 3, 4, 3, 20, 21, 22, 7, 8, 47, 48, 49, 34, 51, 52, 51, 36, 37, 38, 55, 56, 63, 32, 33, 50, 35, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3192(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 4, 5, 6, 23, 24, 15, 16, 17, 2, 3, 4, 3, 20, 21, 22, 7, 8, 63, 32, 33, 50, 51, 52, 51, 36, 37, 38, 55, 56, 47, 48, 49, 34, 35, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3193(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 4, 5, 6, 23, 24, 31, 16, 17, 2, 3, 4, 3, 20, 21, 22, 7, 8, 47, 32, 33, 50, 51, 52, 51, 36, 37, 38, 55, 56, 63, 48, 49, 34, 35, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3194(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 4, 5, 6, 23, 24, 15, 0, 17, 2, 3, 4, 3, 20, 21, 22, 7, 8, 63, 48, 33, 50, 51, 52, 51, 36, 37, 38, 55, 56, 47, 32, 49, 34, 35, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3195(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 4, 5, 6, 23, 24, 31, 0, 17, 2, 3, 4, 3, 20, 21, 22, 7, 8, 47, 48, 33, 50, 51, 52, 51, 36, 37, 38, 55, 56, 63, 32, 49, 34, 35, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3196(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 4, 5, 6, 23, 24, 15, 16, 1, 2, 3, 4, 3, 20, 21, 22, 7, 8, 63, 32, 49, 50, 51, 52, 51, 36, 37, 38, 55, 56, 47, 48, 33, 34, 35, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3197(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 4, 5, 6, 23, 24, 31, 16, 1, 2, 3, 4, 3, 20, 21, 22, 7, 8, 47, 32, 49, 50, 51, 52, 51, 36, 37, 38, 55, 56, 63, 48, 33, 34, 35, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3198(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 4, 5, 6, 23, 24, 15, 0, 1, 2, 3, 4, 3, 20, 21, 22, 7, 8, 63, 48, 49, 50, 51, 52, 51, 36, 37, 38, 55, 56, 47, 32, 33, 34, 35, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3199(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 4, 5, 6, 23, 24, 31, 0, 1, 2, 3, 4, 3, 20, 21, 22, 7, 8, 47, 48, 49, 50, 51, 52, 51, 36, 37, 38, 55, 56, 63, 32, 33, 34, 35, 36, 35, 52, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

#endif // DESGPU_COMPILE_ALL_SALTS


