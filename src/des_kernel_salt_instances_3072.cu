#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt3072(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 4, 5, 6, 23, 24, 15, 16, 17, 18, 19, 20, 19, 20, 21, 22, 7, 8, 63, 32, 33, 34, 35, 36, 35, 36, 37, 38, 55, 56, 47, 48, 49, 50, 51, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3073(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 4, 5, 6, 23, 24, 31, 16, 17, 18, 19, 20, 19, 20, 21, 22, 7, 8, 47, 32, 33, 34, 35, 36, 35, 36, 37, 38, 55, 56, 63, 48, 49, 50, 51, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3074(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 4, 5, 6, 23, 24, 15, 0, 17, 18, 19, 20, 19, 20, 21, 22, 7, 8, 63, 48, 33, 34, 35, 36, 35, 36, 37, 38, 55, 56, 47, 32, 49, 50, 51, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3075(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 4, 5, 6, 23, 24, 31, 0, 17, 18, 19, 20, 19, 20, 21, 22, 7, 8, 47, 48, 33, 34, 35, 36, 35, 36, 37, 38, 55, 56, 63, 32, 49, 50, 51, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3076(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 4, 5, 6, 23, 24, 15, 16, 1, 18, 19, 20, 19, 20, 21, 22, 7, 8, 63, 32, 49, 34, 35, 36, 35, 36, 37, 38, 55, 56, 47, 48, 33, 50, 51, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3077(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 4, 5, 6, 23, 24, 31, 16, 1, 18, 19, 20, 19, 20, 21, 22, 7, 8, 47, 32, 49, 34, 35, 36, 35, 36, 37, 38, 55, 56, 63, 48, 33, 50, 51, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3078(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 4, 5, 6, 23, 24, 15, 0, 1, 18, 19, 20, 19, 20, 21, 22, 7, 8, 63, 48, 49, 34, 35, 36, 35, 36, 37, 38, 55, 56, 47, 32, 33, 50, 51, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3079(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 4, 5, 6, 23, 24, 31, 0, 1, 18, 19, 20, 19, 20, 21, 22, 7, 8, 47, 48, 49, 34, 35, 36, 35, 36, 37, 38, 55, 56, 63, 32, 33, 50, 51, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3080(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 4, 5, 6, 23, 24, 15, 16, 17, 2, 19, 20, 19, 20, 21, 22, 7, 8, 63, 32, 33, 50, 35, 36, 35, 36, 37, 38, 55, 56, 47, 48, 49, 34, 51, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3081(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 4, 5, 6, 23, 24, 31, 16, 17, 2, 19, 20, 19, 20, 21, 22, 7, 8, 47, 32, 33, 50, 35, 36, 35, 36, 37, 38, 55, 56, 63, 48, 49, 34, 51, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3082(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 4, 5, 6, 23, 24, 15, 0, 17, 2, 19, 20, 19, 20, 21, 22, 7, 8, 63, 48, 33, 50, 35, 36, 35, 36, 37, 38, 55, 56, 47, 32, 49, 34, 51, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3083(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 4, 5, 6, 23, 24, 31, 0, 17, 2, 19, 20, 19, 20, 21, 22, 7, 8, 47, 48, 33, 50, 35, 36, 35, 36, 37, 38, 55, 56, 63, 32, 49, 34, 51, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3084(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 4, 5, 6, 23, 24, 15, 16, 1, 2, 19, 20, 19, 20, 21, 22, 7, 8, 63, 32, 49, 50, 35, 36, 35, 36, 37, 38, 55, 56, 47, 48, 33, 34, 51, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3085(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 4, 5, 6, 23, 24, 31, 16, 1, 2, 19, 20, 19, 20, 21, 22, 7, 8, 47, 32, 49, 50, 35, 36, 35, 36, 37, 38, 55, 56, 63, 48, 33, 34, 51, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3086(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 4, 5, 6, 23, 24, 15, 0, 1, 2, 19, 20, 19, 20, 21, 22, 7, 8, 63, 48, 49, 50, 35, 36, 35, 36, 37, 38, 55, 56, 47, 32, 33, 34, 51, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3087(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 4, 5, 6, 23, 24, 31, 0, 1, 2, 19, 20, 19, 20, 21, 22, 7, 8, 47, 48, 49, 50, 35, 36, 35, 36, 37, 38, 55, 56, 63, 32, 33, 34, 51, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3088(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 4, 5, 6, 23, 24, 15, 16, 17, 18, 3, 20, 19, 20, 21, 22, 7, 8, 63, 32, 33, 34, 51, 36, 35, 36, 37, 38, 55, 56, 47, 48, 49, 50, 35, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3089(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 4, 5, 6, 23, 24, 31, 16, 17, 18, 3, 20, 19, 20, 21, 22, 7, 8, 47, 32, 33, 34, 51, 36, 35, 36, 37, 38, 55, 56, 63, 48, 49, 50, 35, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3090(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 4, 5, 6, 23, 24, 15, 0, 17, 18, 3, 20, 19, 20, 21, 22, 7, 8, 63, 48, 33, 34, 51, 36, 35, 36, 37, 38, 55, 56, 47, 32, 49, 50, 35, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3091(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 4, 5, 6, 23, 24, 31, 0, 17, 18, 3, 20, 19, 20, 21, 22, 7, 8, 47, 48, 33, 34, 51, 36, 35, 36, 37, 38, 55, 56, 63, 32, 49, 50, 35, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3092(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 4, 5, 6, 23, 24, 15, 16, 1, 18, 3, 20, 19, 20, 21, 22, 7, 8, 63, 32, 49, 34, 51, 36, 35, 36, 37, 38, 55, 56, 47, 48, 33, 50, 35, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3093(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 4, 5, 6, 23, 24, 31, 16, 1, 18, 3, 20, 19, 20, 21, 22, 7, 8, 47, 32, 49, 34, 51, 36, 35, 36, 37, 38, 55, 56, 63, 48, 33, 50, 35, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3094(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 4, 5, 6, 23, 24, 15, 0, 1, 18, 3, 20, 19, 20, 21, 22, 7, 8, 63, 48, 49, 34, 51, 36, 35, 36, 37, 38, 55, 56, 47, 32, 33, 50, 35, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3095(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 4, 5, 6, 23, 24, 31, 0, 1, 18, 3, 20, 19, 20, 21, 22, 7, 8, 47, 48, 49, 34, 51, 36, 35, 36, 37, 38, 55, 56, 63, 32, 33, 50, 35, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3096(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 4, 5, 6, 23, 24, 15, 16, 17, 2, 3, 20, 19, 20, 21, 22, 7, 8, 63, 32, 33, 50, 51, 36, 35, 36, 37, 38, 55, 56, 47, 48, 49, 34, 35, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3097(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 4, 5, 6, 23, 24, 31, 16, 17, 2, 3, 20, 19, 20, 21, 22, 7, 8, 47, 32, 33, 50, 51, 36, 35, 36, 37, 38, 55, 56, 63, 48, 49, 34, 35, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3098(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 4, 5, 6, 23, 24, 15, 0, 17, 2, 3, 20, 19, 20, 21, 22, 7, 8, 63, 48, 33, 50, 51, 36, 35, 36, 37, 38, 55, 56, 47, 32, 49, 34, 35, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3099(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 4, 5, 6, 23, 24, 31, 0, 17, 2, 3, 20, 19, 20, 21, 22, 7, 8, 47, 48, 33, 50, 51, 36, 35, 36, 37, 38, 55, 56, 63, 32, 49, 34, 35, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3100(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 4, 5, 6, 23, 24, 15, 16, 1, 2, 3, 20, 19, 20, 21, 22, 7, 8, 63, 32, 49, 50, 51, 36, 35, 36, 37, 38, 55, 56, 47, 48, 33, 34, 35, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3101(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 4, 5, 6, 23, 24, 31, 16, 1, 2, 3, 20, 19, 20, 21, 22, 7, 8, 47, 32, 49, 50, 51, 36, 35, 36, 37, 38, 55, 56, 63, 48, 33, 34, 35, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3102(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 4, 5, 6, 23, 24, 15, 0, 1, 2, 3, 20, 19, 20, 21, 22, 7, 8, 63, 48, 49, 50, 51, 36, 35, 36, 37, 38, 55, 56, 47, 32, 33, 34, 35, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3103(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 4, 5, 6, 23, 24, 31, 0, 1, 2, 3, 20, 19, 20, 21, 22, 7, 8, 47, 48, 49, 50, 51, 36, 35, 36, 37, 38, 55, 56, 63, 32, 33, 34, 35, 52, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3104(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 4, 5, 6, 23, 24, 15, 16, 17, 18, 19, 4, 19, 20, 21, 22, 7, 8, 63, 32, 33, 34, 35, 52, 35, 36, 37, 38, 55, 56, 47, 48, 49, 50, 51, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3105(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 4, 5, 6, 23, 24, 31, 16, 17, 18, 19, 4, 19, 20, 21, 22, 7, 8, 47, 32, 33, 34, 35, 52, 35, 36, 37, 38, 55, 56, 63, 48, 49, 50, 51, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3106(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 4, 5, 6, 23, 24, 15, 0, 17, 18, 19, 4, 19, 20, 21, 22, 7, 8, 63, 48, 33, 34, 35, 52, 35, 36, 37, 38, 55, 56, 47, 32, 49, 50, 51, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3107(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 4, 5, 6, 23, 24, 31, 0, 17, 18, 19, 4, 19, 20, 21, 22, 7, 8, 47, 48, 33, 34, 35, 52, 35, 36, 37, 38, 55, 56, 63, 32, 49, 50, 51, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3108(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 4, 5, 6, 23, 24, 15, 16, 1, 18, 19, 4, 19, 20, 21, 22, 7, 8, 63, 32, 49, 34, 35, 52, 35, 36, 37, 38, 55, 56, 47, 48, 33, 50, 51, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3109(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 4, 5, 6, 23, 24, 31, 16, 1, 18, 19, 4, 19, 20, 21, 22, 7, 8, 47, 32, 49, 34, 35, 52, 35, 36, 37, 38, 55, 56, 63, 48, 33, 50, 51, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3110(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 4, 5, 6, 23, 24, 15, 0, 1, 18, 19, 4, 19, 20, 21, 22, 7, 8, 63, 48, 49, 34, 35, 52, 35, 36, 37, 38, 55, 56, 47, 32, 33, 50, 51, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3111(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 4, 5, 6, 23, 24, 31, 0, 1, 18, 19, 4, 19, 20, 21, 22, 7, 8, 47, 48, 49, 34, 35, 52, 35, 36, 37, 38, 55, 56, 63, 32, 33, 50, 51, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3112(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 4, 5, 6, 23, 24, 15, 16, 17, 2, 19, 4, 19, 20, 21, 22, 7, 8, 63, 32, 33, 50, 35, 52, 35, 36, 37, 38, 55, 56, 47, 48, 49, 34, 51, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3113(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 4, 5, 6, 23, 24, 31, 16, 17, 2, 19, 4, 19, 20, 21, 22, 7, 8, 47, 32, 33, 50, 35, 52, 35, 36, 37, 38, 55, 56, 63, 48, 49, 34, 51, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3114(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 4, 5, 6, 23, 24, 15, 0, 17, 2, 19, 4, 19, 20, 21, 22, 7, 8, 63, 48, 33, 50, 35, 52, 35, 36, 37, 38, 55, 56, 47, 32, 49, 34, 51, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3115(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 4, 5, 6, 23, 24, 31, 0, 17, 2, 19, 4, 19, 20, 21, 22, 7, 8, 47, 48, 33, 50, 35, 52, 35, 36, 37, 38, 55, 56, 63, 32, 49, 34, 51, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3116(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 4, 5, 6, 23, 24, 15, 16, 1, 2, 19, 4, 19, 20, 21, 22, 7, 8, 63, 32, 49, 50, 35, 52, 35, 36, 37, 38, 55, 56, 47, 48, 33, 34, 51, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3117(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 4, 5, 6, 23, 24, 31, 16, 1, 2, 19, 4, 19, 20, 21, 22, 7, 8, 47, 32, 49, 50, 35, 52, 35, 36, 37, 38, 55, 56, 63, 48, 33, 34, 51, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3118(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 4, 5, 6, 23, 24, 15, 0, 1, 2, 19, 4, 19, 20, 21, 22, 7, 8, 63, 48, 49, 50, 35, 52, 35, 36, 37, 38, 55, 56, 47, 32, 33, 34, 51, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3119(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 4, 5, 6, 23, 24, 31, 0, 1, 2, 19, 4, 19, 20, 21, 22, 7, 8, 47, 48, 49, 50, 35, 52, 35, 36, 37, 38, 55, 56, 63, 32, 33, 34, 51, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3120(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 4, 5, 6, 23, 24, 15, 16, 17, 18, 3, 4, 19, 20, 21, 22, 7, 8, 63, 32, 33, 34, 51, 52, 35, 36, 37, 38, 55, 56, 47, 48, 49, 50, 35, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3121(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 4, 5, 6, 23, 24, 31, 16, 17, 18, 3, 4, 19, 20, 21, 22, 7, 8, 47, 32, 33, 34, 51, 52, 35, 36, 37, 38, 55, 56, 63, 48, 49, 50, 35, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3122(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 4, 5, 6, 23, 24, 15, 0, 17, 18, 3, 4, 19, 20, 21, 22, 7, 8, 63, 48, 33, 34, 51, 52, 35, 36, 37, 38, 55, 56, 47, 32, 49, 50, 35, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3123(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 4, 5, 6, 23, 24, 31, 0, 17, 18, 3, 4, 19, 20, 21, 22, 7, 8, 47, 48, 33, 34, 51, 52, 35, 36, 37, 38, 55, 56, 63, 32, 49, 50, 35, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3124(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 4, 5, 6, 23, 24, 15, 16, 1, 18, 3, 4, 19, 20, 21, 22, 7, 8, 63, 32, 49, 34, 51, 52, 35, 36, 37, 38, 55, 56, 47, 48, 33, 50, 35, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3125(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 4, 5, 6, 23, 24, 31, 16, 1, 18, 3, 4, 19, 20, 21, 22, 7, 8, 47, 32, 49, 34, 51, 52, 35, 36, 37, 38, 55, 56, 63, 48, 33, 50, 35, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3126(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 4, 5, 6, 23, 24, 15, 0, 1, 18, 3, 4, 19, 20, 21, 22, 7, 8, 63, 48, 49, 34, 51, 52, 35, 36, 37, 38, 55, 56, 47, 32, 33, 50, 35, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3127(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 4, 5, 6, 23, 24, 31, 0, 1, 18, 3, 4, 19, 20, 21, 22, 7, 8, 47, 48, 49, 34, 51, 52, 35, 36, 37, 38, 55, 56, 63, 32, 33, 50, 35, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3128(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 4, 5, 6, 23, 24, 15, 16, 17, 2, 3, 4, 19, 20, 21, 22, 7, 8, 63, 32, 33, 50, 51, 52, 35, 36, 37, 38, 55, 56, 47, 48, 49, 34, 35, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3129(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 4, 5, 6, 23, 24, 31, 16, 17, 2, 3, 4, 19, 20, 21, 22, 7, 8, 47, 32, 33, 50, 51, 52, 35, 36, 37, 38, 55, 56, 63, 48, 49, 34, 35, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3130(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 4, 5, 6, 23, 24, 15, 0, 17, 2, 3, 4, 19, 20, 21, 22, 7, 8, 63, 48, 33, 50, 51, 52, 35, 36, 37, 38, 55, 56, 47, 32, 49, 34, 35, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3131(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 4, 5, 6, 23, 24, 31, 0, 17, 2, 3, 4, 19, 20, 21, 22, 7, 8, 47, 48, 33, 50, 51, 52, 35, 36, 37, 38, 55, 56, 63, 32, 49, 34, 35, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3132(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 4, 5, 6, 23, 24, 15, 16, 1, 2, 3, 4, 19, 20, 21, 22, 7, 8, 63, 32, 49, 50, 51, 52, 35, 36, 37, 38, 55, 56, 47, 48, 33, 34, 35, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3133(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 4, 5, 6, 23, 24, 31, 16, 1, 2, 3, 4, 19, 20, 21, 22, 7, 8, 47, 32, 49, 50, 51, 52, 35, 36, 37, 38, 55, 56, 63, 48, 33, 34, 35, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3134(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 4, 5, 6, 23, 24, 15, 0, 1, 2, 3, 4, 19, 20, 21, 22, 7, 8, 63, 48, 49, 50, 51, 52, 35, 36, 37, 38, 55, 56, 47, 32, 33, 34, 35, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3135(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 4, 5, 6, 23, 24, 31, 0, 1, 2, 3, 4, 19, 20, 21, 22, 7, 8, 47, 48, 49, 50, 51, 52, 35, 36, 37, 38, 55, 56, 63, 32, 33, 34, 35, 36, 51, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3136(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 4, 5, 6, 23, 24, 15, 16, 17, 18, 19, 20, 3, 20, 21, 22, 7, 8, 63, 32, 33, 34, 35, 36, 51, 36, 37, 38, 55, 56, 47, 48, 49, 50, 51, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3137(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 4, 5, 6, 23, 24, 31, 16, 17, 18, 19, 20, 3, 20, 21, 22, 7, 8, 47, 32, 33, 34, 35, 36, 51, 36, 37, 38, 55, 56, 63, 48, 49, 50, 51, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3138(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 4, 5, 6, 23, 24, 15, 0, 17, 18, 19, 20, 3, 20, 21, 22, 7, 8, 63, 48, 33, 34, 35, 36, 51, 36, 37, 38, 55, 56, 47, 32, 49, 50, 51, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3139(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 4, 5, 6, 23, 24, 31, 0, 17, 18, 19, 20, 3, 20, 21, 22, 7, 8, 47, 48, 33, 34, 35, 36, 51, 36, 37, 38, 55, 56, 63, 32, 49, 50, 51, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3140(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 4, 5, 6, 23, 24, 15, 16, 1, 18, 19, 20, 3, 20, 21, 22, 7, 8, 63, 32, 49, 34, 35, 36, 51, 36, 37, 38, 55, 56, 47, 48, 33, 50, 51, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3141(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 4, 5, 6, 23, 24, 31, 16, 1, 18, 19, 20, 3, 20, 21, 22, 7, 8, 47, 32, 49, 34, 35, 36, 51, 36, 37, 38, 55, 56, 63, 48, 33, 50, 51, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3142(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 4, 5, 6, 23, 24, 15, 0, 1, 18, 19, 20, 3, 20, 21, 22, 7, 8, 63, 48, 49, 34, 35, 36, 51, 36, 37, 38, 55, 56, 47, 32, 33, 50, 51, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3143(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 4, 5, 6, 23, 24, 31, 0, 1, 18, 19, 20, 3, 20, 21, 22, 7, 8, 47, 48, 49, 34, 35, 36, 51, 36, 37, 38, 55, 56, 63, 32, 33, 50, 51, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3144(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 4, 5, 6, 23, 24, 15, 16, 17, 2, 19, 20, 3, 20, 21, 22, 7, 8, 63, 32, 33, 50, 35, 36, 51, 36, 37, 38, 55, 56, 47, 48, 49, 34, 51, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3145(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 4, 5, 6, 23, 24, 31, 16, 17, 2, 19, 20, 3, 20, 21, 22, 7, 8, 47, 32, 33, 50, 35, 36, 51, 36, 37, 38, 55, 56, 63, 48, 49, 34, 51, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3146(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 4, 5, 6, 23, 24, 15, 0, 17, 2, 19, 20, 3, 20, 21, 22, 7, 8, 63, 48, 33, 50, 35, 36, 51, 36, 37, 38, 55, 56, 47, 32, 49, 34, 51, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3147(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 4, 5, 6, 23, 24, 31, 0, 17, 2, 19, 20, 3, 20, 21, 22, 7, 8, 47, 48, 33, 50, 35, 36, 51, 36, 37, 38, 55, 56, 63, 32, 49, 34, 51, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3148(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 4, 5, 6, 23, 24, 15, 16, 1, 2, 19, 20, 3, 20, 21, 22, 7, 8, 63, 32, 49, 50, 35, 36, 51, 36, 37, 38, 55, 56, 47, 48, 33, 34, 51, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3149(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 4, 5, 6, 23, 24, 31, 16, 1, 2, 19, 20, 3, 20, 21, 22, 7, 8, 47, 32, 49, 50, 35, 36, 51, 36, 37, 38, 55, 56, 63, 48, 33, 34, 51, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3150(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 4, 5, 6, 23, 24, 15, 0, 1, 2, 19, 20, 3, 20, 21, 22, 7, 8, 63, 48, 49, 50, 35, 36, 51, 36, 37, 38, 55, 56, 47, 32, 33, 34, 51, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3151(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 4, 5, 6, 23, 24, 31, 0, 1, 2, 19, 20, 3, 20, 21, 22, 7, 8, 47, 48, 49, 50, 35, 36, 51, 36, 37, 38, 55, 56, 63, 32, 33, 34, 51, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3152(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 4, 5, 6, 23, 24, 15, 16, 17, 18, 3, 20, 3, 20, 21, 22, 7, 8, 63, 32, 33, 34, 51, 36, 51, 36, 37, 38, 55, 56, 47, 48, 49, 50, 35, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3153(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 4, 5, 6, 23, 24, 31, 16, 17, 18, 3, 20, 3, 20, 21, 22, 7, 8, 47, 32, 33, 34, 51, 36, 51, 36, 37, 38, 55, 56, 63, 48, 49, 50, 35, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3154(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 4, 5, 6, 23, 24, 15, 0, 17, 18, 3, 20, 3, 20, 21, 22, 7, 8, 63, 48, 33, 34, 51, 36, 51, 36, 37, 38, 55, 56, 47, 32, 49, 50, 35, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3155(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 4, 5, 6, 23, 24, 31, 0, 17, 18, 3, 20, 3, 20, 21, 22, 7, 8, 47, 48, 33, 34, 51, 36, 51, 36, 37, 38, 55, 56, 63, 32, 49, 50, 35, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3156(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 4, 5, 6, 23, 24, 15, 16, 1, 18, 3, 20, 3, 20, 21, 22, 7, 8, 63, 32, 49, 34, 51, 36, 51, 36, 37, 38, 55, 56, 47, 48, 33, 50, 35, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3157(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 4, 5, 6, 23, 24, 31, 16, 1, 18, 3, 20, 3, 20, 21, 22, 7, 8, 47, 32, 49, 34, 51, 36, 51, 36, 37, 38, 55, 56, 63, 48, 33, 50, 35, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3158(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 4, 5, 6, 23, 24, 15, 0, 1, 18, 3, 20, 3, 20, 21, 22, 7, 8, 63, 48, 49, 34, 51, 36, 51, 36, 37, 38, 55, 56, 47, 32, 33, 50, 35, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3159(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 4, 5, 6, 23, 24, 31, 0, 1, 18, 3, 20, 3, 20, 21, 22, 7, 8, 47, 48, 49, 34, 51, 36, 51, 36, 37, 38, 55, 56, 63, 32, 33, 50, 35, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3160(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 4, 5, 6, 23, 24, 15, 16, 17, 2, 3, 20, 3, 20, 21, 22, 7, 8, 63, 32, 33, 50, 51, 36, 51, 36, 37, 38, 55, 56, 47, 48, 49, 34, 35, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3161(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 4, 5, 6, 23, 24, 31, 16, 17, 2, 3, 20, 3, 20, 21, 22, 7, 8, 47, 32, 33, 50, 51, 36, 51, 36, 37, 38, 55, 56, 63, 48, 49, 34, 35, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3162(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 4, 5, 6, 23, 24, 15, 0, 17, 2, 3, 20, 3, 20, 21, 22, 7, 8, 63, 48, 33, 50, 51, 36, 51, 36, 37, 38, 55, 56, 47, 32, 49, 34, 35, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3163(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 4, 5, 6, 23, 24, 31, 0, 17, 2, 3, 20, 3, 20, 21, 22, 7, 8, 47, 48, 33, 50, 51, 36, 51, 36, 37, 38, 55, 56, 63, 32, 49, 34, 35, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3164(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 4, 5, 6, 23, 24, 15, 16, 1, 2, 3, 20, 3, 20, 21, 22, 7, 8, 63, 32, 49, 50, 51, 36, 51, 36, 37, 38, 55, 56, 47, 48, 33, 34, 35, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3165(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 4, 5, 6, 23, 24, 31, 16, 1, 2, 3, 20, 3, 20, 21, 22, 7, 8, 47, 32, 49, 50, 51, 36, 51, 36, 37, 38, 55, 56, 63, 48, 33, 34, 35, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3166(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 4, 5, 6, 23, 24, 15, 0, 1, 2, 3, 20, 3, 20, 21, 22, 7, 8, 63, 48, 49, 50, 51, 36, 51, 36, 37, 38, 55, 56, 47, 32, 33, 34, 35, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3167(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 4, 5, 6, 23, 24, 31, 0, 1, 2, 3, 20, 3, 20, 21, 22, 7, 8, 47, 48, 49, 50, 51, 36, 51, 36, 37, 38, 55, 56, 63, 32, 33, 34, 35, 52, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3168(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 4, 5, 6, 23, 24, 15, 16, 17, 18, 19, 4, 3, 20, 21, 22, 7, 8, 63, 32, 33, 34, 35, 52, 51, 36, 37, 38, 55, 56, 47, 48, 49, 50, 51, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3169(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 4, 5, 6, 23, 24, 31, 16, 17, 18, 19, 4, 3, 20, 21, 22, 7, 8, 47, 32, 33, 34, 35, 52, 51, 36, 37, 38, 55, 56, 63, 48, 49, 50, 51, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3170(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 4, 5, 6, 23, 24, 15, 0, 17, 18, 19, 4, 3, 20, 21, 22, 7, 8, 63, 48, 33, 34, 35, 52, 51, 36, 37, 38, 55, 56, 47, 32, 49, 50, 51, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3171(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 4, 5, 6, 23, 24, 31, 0, 17, 18, 19, 4, 3, 20, 21, 22, 7, 8, 47, 48, 33, 34, 35, 52, 51, 36, 37, 38, 55, 56, 63, 32, 49, 50, 51, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3172(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 4, 5, 6, 23, 24, 15, 16, 1, 18, 19, 4, 3, 20, 21, 22, 7, 8, 63, 32, 49, 34, 35, 52, 51, 36, 37, 38, 55, 56, 47, 48, 33, 50, 51, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3173(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 4, 5, 6, 23, 24, 31, 16, 1, 18, 19, 4, 3, 20, 21, 22, 7, 8, 47, 32, 49, 34, 35, 52, 51, 36, 37, 38, 55, 56, 63, 48, 33, 50, 51, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3174(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 4, 5, 6, 23, 24, 15, 0, 1, 18, 19, 4, 3, 20, 21, 22, 7, 8, 63, 48, 49, 34, 35, 52, 51, 36, 37, 38, 55, 56, 47, 32, 33, 50, 51, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3175(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 4, 5, 6, 23, 24, 31, 0, 1, 18, 19, 4, 3, 20, 21, 22, 7, 8, 47, 48, 49, 34, 35, 52, 51, 36, 37, 38, 55, 56, 63, 32, 33, 50, 51, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3176(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 4, 5, 6, 23, 24, 15, 16, 17, 2, 19, 4, 3, 20, 21, 22, 7, 8, 63, 32, 33, 50, 35, 52, 51, 36, 37, 38, 55, 56, 47, 48, 49, 34, 51, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3177(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 4, 5, 6, 23, 24, 31, 16, 17, 2, 19, 4, 3, 20, 21, 22, 7, 8, 47, 32, 33, 50, 35, 52, 51, 36, 37, 38, 55, 56, 63, 48, 49, 34, 51, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3178(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 4, 5, 6, 23, 24, 15, 0, 17, 2, 19, 4, 3, 20, 21, 22, 7, 8, 63, 48, 33, 50, 35, 52, 51, 36, 37, 38, 55, 56, 47, 32, 49, 34, 51, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3179(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 4, 5, 6, 23, 24, 31, 0, 17, 2, 19, 4, 3, 20, 21, 22, 7, 8, 47, 48, 33, 50, 35, 52, 51, 36, 37, 38, 55, 56, 63, 32, 49, 34, 51, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3180(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 4, 5, 6, 23, 24, 15, 16, 1, 2, 19, 4, 3, 20, 21, 22, 7, 8, 63, 32, 49, 50, 35, 52, 51, 36, 37, 38, 55, 56, 47, 48, 33, 34, 51, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3181(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 4, 5, 6, 23, 24, 31, 16, 1, 2, 19, 4, 3, 20, 21, 22, 7, 8, 47, 32, 49, 50, 35, 52, 51, 36, 37, 38, 55, 56, 63, 48, 33, 34, 51, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3182(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 4, 5, 6, 23, 24, 15, 0, 1, 2, 19, 4, 3, 20, 21, 22, 7, 8, 63, 48, 49, 50, 35, 52, 51, 36, 37, 38, 55, 56, 47, 32, 33, 34, 51, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3183(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 4, 5, 6, 23, 24, 31, 0, 1, 2, 19, 4, 3, 20, 21, 22, 7, 8, 47, 48, 49, 50, 35, 52, 51, 36, 37, 38, 55, 56, 63, 32, 33, 34, 51, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3184(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 4, 5, 6, 23, 24, 15, 16, 17, 18, 3, 4, 3, 20, 21, 22, 7, 8, 63, 32, 33, 34, 51, 52, 51, 36, 37, 38, 55, 56, 47, 48, 49, 50, 35, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3185(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 4, 5, 6, 23, 24, 31, 16, 17, 18, 3, 4, 3, 20, 21, 22, 7, 8, 47, 32, 33, 34, 51, 52, 51, 36, 37, 38, 55, 56, 63, 48, 49, 50, 35, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3186(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 4, 5, 6, 23, 24, 15, 0, 17, 18, 3, 4, 3, 20, 21, 22, 7, 8, 63, 48, 33, 34, 51, 52, 51, 36, 37, 38, 55, 56, 47, 32, 49, 50, 35, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3187(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 4, 5, 6, 23, 24, 31, 0, 17, 18, 3, 4, 3, 20, 21, 22, 7, 8, 47, 48, 33, 34, 51, 52, 51, 36, 37, 38, 55, 56, 63, 32, 49, 50, 35, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3188(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 4, 5, 6, 23, 24, 15, 16, 1, 18, 3, 4, 3, 20, 21, 22, 7, 8, 63, 32, 49, 34, 51, 52, 51, 36, 37, 38, 55, 56, 47, 48, 33, 50, 35, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3189(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 4, 5, 6, 23, 24, 31, 16, 1, 18, 3, 4, 3, 20, 21, 22, 7, 8, 47, 32, 49, 34, 51, 52, 51, 36, 37, 38, 55, 56, 63, 48, 33, 50, 35, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3190(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 4, 5, 6, 23, 24, 15, 0, 1, 18, 3, 4, 3, 20, 21, 22, 7, 8, 63, 48, 49, 34, 51, 52, 51, 36, 37, 38, 55, 56, 47, 32, 33, 50, 35, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3191(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 4, 5, 6, 23, 24, 31, 0, 1, 18, 3, 4, 3, 20, 21, 22, 7, 8, 47, 48, 49, 34, 51, 52, 51, 36, 37, 38, 55, 56, 63, 32, 33, 50, 35, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3192(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 4, 5, 6, 23, 24, 15, 16, 17, 2, 3, 4, 3, 20, 21, 22, 7, 8, 63, 32, 33, 50, 51, 52, 51, 36, 37, 38, 55, 56, 47, 48, 49, 34, 35, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3193(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 4, 5, 6, 23, 24, 31, 16, 17, 2, 3, 4, 3, 20, 21, 22, 7, 8, 47, 32, 33, 50, 51, 52, 51, 36, 37, 38, 55, 56, 63, 48, 49, 34, 35, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3194(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 4, 5, 6, 23, 24, 15, 0, 17, 2, 3, 4, 3, 20, 21, 22, 7, 8, 63, 48, 33, 50, 51, 52, 51, 36, 37, 38, 55, 56, 47, 32, 49, 34, 35, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3195(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 4, 5, 6, 23, 24, 31, 0, 17, 2, 3, 4, 3, 20, 21, 22, 7, 8, 47, 48, 33, 50, 51, 52, 51, 36, 37, 38, 55, 56, 63, 32, 49, 34, 35, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3196(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 4, 5, 6, 23, 24, 15, 16, 1, 2, 3, 4, 3, 20, 21, 22, 7, 8, 63, 32, 49, 50, 51, 52, 51, 36, 37, 38, 55, 56, 47, 48, 33, 34, 35, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3197(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 4, 5, 6, 23, 24, 31, 16, 1, 2, 3, 4, 3, 20, 21, 22, 7, 8, 47, 32, 49, 50, 51, 52, 51, 36, 37, 38, 55, 56, 63, 48, 33, 34, 35, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3198(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 4, 5, 6, 23, 24, 15, 0, 1, 2, 3, 4, 3, 20, 21, 22, 7, 8, 63, 48, 49, 50, 51, 52, 51, 36, 37, 38, 55, 56, 47, 32, 33, 34, 35, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3199(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 4, 5, 6, 23, 24, 31, 0, 1, 2, 3, 4, 3, 20, 21, 22, 7, 8, 47, 48, 49, 50, 51, 52, 51, 36, 37, 38, 55, 56, 63, 32, 33, 34, 35, 36, 35, 52, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}


