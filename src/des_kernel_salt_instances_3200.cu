#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt3200(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 5, 6, 23, 24, 15, 16, 17, 18, 19, 20, 19, 4, 21, 22, 7, 8, 63, 32, 33, 34, 35, 36, 35, 52, 37, 38, 55, 56, 47, 48, 49, 50, 51, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3201(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 5, 6, 23, 24, 31, 16, 17, 18, 19, 20, 19, 4, 21, 22, 7, 8, 47, 32, 33, 34, 35, 36, 35, 52, 37, 38, 55, 56, 63, 48, 49, 50, 51, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3202(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 5, 6, 23, 24, 15, 0, 17, 18, 19, 20, 19, 4, 21, 22, 7, 8, 63, 48, 33, 34, 35, 36, 35, 52, 37, 38, 55, 56, 47, 32, 49, 50, 51, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3203(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 5, 6, 23, 24, 31, 0, 17, 18, 19, 20, 19, 4, 21, 22, 7, 8, 47, 48, 33, 34, 35, 36, 35, 52, 37, 38, 55, 56, 63, 32, 49, 50, 51, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3204(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 5, 6, 23, 24, 15, 16, 1, 18, 19, 20, 19, 4, 21, 22, 7, 8, 63, 32, 49, 34, 35, 36, 35, 52, 37, 38, 55, 56, 47, 48, 33, 50, 51, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3205(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 5, 6, 23, 24, 31, 16, 1, 18, 19, 20, 19, 4, 21, 22, 7, 8, 47, 32, 49, 34, 35, 36, 35, 52, 37, 38, 55, 56, 63, 48, 33, 50, 51, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3206(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 5, 6, 23, 24, 15, 0, 1, 18, 19, 20, 19, 4, 21, 22, 7, 8, 63, 48, 49, 34, 35, 36, 35, 52, 37, 38, 55, 56, 47, 32, 33, 50, 51, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3207(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 5, 6, 23, 24, 31, 0, 1, 18, 19, 20, 19, 4, 21, 22, 7, 8, 47, 48, 49, 34, 35, 36, 35, 52, 37, 38, 55, 56, 63, 32, 33, 50, 51, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3208(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 5, 6, 23, 24, 15, 16, 17, 2, 19, 20, 19, 4, 21, 22, 7, 8, 63, 32, 33, 50, 35, 36, 35, 52, 37, 38, 55, 56, 47, 48, 49, 34, 51, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3209(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 5, 6, 23, 24, 31, 16, 17, 2, 19, 20, 19, 4, 21, 22, 7, 8, 47, 32, 33, 50, 35, 36, 35, 52, 37, 38, 55, 56, 63, 48, 49, 34, 51, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3210(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 5, 6, 23, 24, 15, 0, 17, 2, 19, 20, 19, 4, 21, 22, 7, 8, 63, 48, 33, 50, 35, 36, 35, 52, 37, 38, 55, 56, 47, 32, 49, 34, 51, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3211(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 5, 6, 23, 24, 31, 0, 17, 2, 19, 20, 19, 4, 21, 22, 7, 8, 47, 48, 33, 50, 35, 36, 35, 52, 37, 38, 55, 56, 63, 32, 49, 34, 51, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3212(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 5, 6, 23, 24, 15, 16, 1, 2, 19, 20, 19, 4, 21, 22, 7, 8, 63, 32, 49, 50, 35, 36, 35, 52, 37, 38, 55, 56, 47, 48, 33, 34, 51, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3213(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 5, 6, 23, 24, 31, 16, 1, 2, 19, 20, 19, 4, 21, 22, 7, 8, 47, 32, 49, 50, 35, 36, 35, 52, 37, 38, 55, 56, 63, 48, 33, 34, 51, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3214(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 5, 6, 23, 24, 15, 0, 1, 2, 19, 20, 19, 4, 21, 22, 7, 8, 63, 48, 49, 50, 35, 36, 35, 52, 37, 38, 55, 56, 47, 32, 33, 34, 51, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3215(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 5, 6, 23, 24, 31, 0, 1, 2, 19, 20, 19, 4, 21, 22, 7, 8, 47, 48, 49, 50, 35, 36, 35, 52, 37, 38, 55, 56, 63, 32, 33, 34, 51, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3216(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 5, 6, 23, 24, 15, 16, 17, 18, 3, 20, 19, 4, 21, 22, 7, 8, 63, 32, 33, 34, 51, 36, 35, 52, 37, 38, 55, 56, 47, 48, 49, 50, 35, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3217(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 5, 6, 23, 24, 31, 16, 17, 18, 3, 20, 19, 4, 21, 22, 7, 8, 47, 32, 33, 34, 51, 36, 35, 52, 37, 38, 55, 56, 63, 48, 49, 50, 35, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3218(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 5, 6, 23, 24, 15, 0, 17, 18, 3, 20, 19, 4, 21, 22, 7, 8, 63, 48, 33, 34, 51, 36, 35, 52, 37, 38, 55, 56, 47, 32, 49, 50, 35, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3219(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 5, 6, 23, 24, 31, 0, 17, 18, 3, 20, 19, 4, 21, 22, 7, 8, 47, 48, 33, 34, 51, 36, 35, 52, 37, 38, 55, 56, 63, 32, 49, 50, 35, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3220(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 5, 6, 23, 24, 15, 16, 1, 18, 3, 20, 19, 4, 21, 22, 7, 8, 63, 32, 49, 34, 51, 36, 35, 52, 37, 38, 55, 56, 47, 48, 33, 50, 35, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3221(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 5, 6, 23, 24, 31, 16, 1, 18, 3, 20, 19, 4, 21, 22, 7, 8, 47, 32, 49, 34, 51, 36, 35, 52, 37, 38, 55, 56, 63, 48, 33, 50, 35, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3222(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 5, 6, 23, 24, 15, 0, 1, 18, 3, 20, 19, 4, 21, 22, 7, 8, 63, 48, 49, 34, 51, 36, 35, 52, 37, 38, 55, 56, 47, 32, 33, 50, 35, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3223(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 5, 6, 23, 24, 31, 0, 1, 18, 3, 20, 19, 4, 21, 22, 7, 8, 47, 48, 49, 34, 51, 36, 35, 52, 37, 38, 55, 56, 63, 32, 33, 50, 35, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3224(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 5, 6, 23, 24, 15, 16, 17, 2, 3, 20, 19, 4, 21, 22, 7, 8, 63, 32, 33, 50, 51, 36, 35, 52, 37, 38, 55, 56, 47, 48, 49, 34, 35, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3225(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 5, 6, 23, 24, 31, 16, 17, 2, 3, 20, 19, 4, 21, 22, 7, 8, 47, 32, 33, 50, 51, 36, 35, 52, 37, 38, 55, 56, 63, 48, 49, 34, 35, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3226(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 5, 6, 23, 24, 15, 0, 17, 2, 3, 20, 19, 4, 21, 22, 7, 8, 63, 48, 33, 50, 51, 36, 35, 52, 37, 38, 55, 56, 47, 32, 49, 34, 35, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3227(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 5, 6, 23, 24, 31, 0, 17, 2, 3, 20, 19, 4, 21, 22, 7, 8, 47, 48, 33, 50, 51, 36, 35, 52, 37, 38, 55, 56, 63, 32, 49, 34, 35, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3228(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 5, 6, 23, 24, 15, 16, 1, 2, 3, 20, 19, 4, 21, 22, 7, 8, 63, 32, 49, 50, 51, 36, 35, 52, 37, 38, 55, 56, 47, 48, 33, 34, 35, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3229(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 5, 6, 23, 24, 31, 16, 1, 2, 3, 20, 19, 4, 21, 22, 7, 8, 47, 32, 49, 50, 51, 36, 35, 52, 37, 38, 55, 56, 63, 48, 33, 34, 35, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3230(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 5, 6, 23, 24, 15, 0, 1, 2, 3, 20, 19, 4, 21, 22, 7, 8, 63, 48, 49, 50, 51, 36, 35, 52, 37, 38, 55, 56, 47, 32, 33, 34, 35, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3231(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 5, 6, 23, 24, 31, 0, 1, 2, 3, 20, 19, 4, 21, 22, 7, 8, 47, 48, 49, 50, 51, 36, 35, 52, 37, 38, 55, 56, 63, 32, 33, 34, 35, 52, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3232(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 5, 6, 23, 24, 15, 16, 17, 18, 19, 4, 19, 4, 21, 22, 7, 8, 63, 32, 33, 34, 35, 52, 35, 52, 37, 38, 55, 56, 47, 48, 49, 50, 51, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3233(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 5, 6, 23, 24, 31, 16, 17, 18, 19, 4, 19, 4, 21, 22, 7, 8, 47, 32, 33, 34, 35, 52, 35, 52, 37, 38, 55, 56, 63, 48, 49, 50, 51, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3234(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 5, 6, 23, 24, 15, 0, 17, 18, 19, 4, 19, 4, 21, 22, 7, 8, 63, 48, 33, 34, 35, 52, 35, 52, 37, 38, 55, 56, 47, 32, 49, 50, 51, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3235(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 5, 6, 23, 24, 31, 0, 17, 18, 19, 4, 19, 4, 21, 22, 7, 8, 47, 48, 33, 34, 35, 52, 35, 52, 37, 38, 55, 56, 63, 32, 49, 50, 51, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3236(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 5, 6, 23, 24, 15, 16, 1, 18, 19, 4, 19, 4, 21, 22, 7, 8, 63, 32, 49, 34, 35, 52, 35, 52, 37, 38, 55, 56, 47, 48, 33, 50, 51, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3237(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 5, 6, 23, 24, 31, 16, 1, 18, 19, 4, 19, 4, 21, 22, 7, 8, 47, 32, 49, 34, 35, 52, 35, 52, 37, 38, 55, 56, 63, 48, 33, 50, 51, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3238(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 5, 6, 23, 24, 15, 0, 1, 18, 19, 4, 19, 4, 21, 22, 7, 8, 63, 48, 49, 34, 35, 52, 35, 52, 37, 38, 55, 56, 47, 32, 33, 50, 51, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3239(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 5, 6, 23, 24, 31, 0, 1, 18, 19, 4, 19, 4, 21, 22, 7, 8, 47, 48, 49, 34, 35, 52, 35, 52, 37, 38, 55, 56, 63, 32, 33, 50, 51, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3240(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 5, 6, 23, 24, 15, 16, 17, 2, 19, 4, 19, 4, 21, 22, 7, 8, 63, 32, 33, 50, 35, 52, 35, 52, 37, 38, 55, 56, 47, 48, 49, 34, 51, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3241(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 5, 6, 23, 24, 31, 16, 17, 2, 19, 4, 19, 4, 21, 22, 7, 8, 47, 32, 33, 50, 35, 52, 35, 52, 37, 38, 55, 56, 63, 48, 49, 34, 51, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3242(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 5, 6, 23, 24, 15, 0, 17, 2, 19, 4, 19, 4, 21, 22, 7, 8, 63, 48, 33, 50, 35, 52, 35, 52, 37, 38, 55, 56, 47, 32, 49, 34, 51, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3243(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 5, 6, 23, 24, 31, 0, 17, 2, 19, 4, 19, 4, 21, 22, 7, 8, 47, 48, 33, 50, 35, 52, 35, 52, 37, 38, 55, 56, 63, 32, 49, 34, 51, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3244(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 5, 6, 23, 24, 15, 16, 1, 2, 19, 4, 19, 4, 21, 22, 7, 8, 63, 32, 49, 50, 35, 52, 35, 52, 37, 38, 55, 56, 47, 48, 33, 34, 51, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3245(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 5, 6, 23, 24, 31, 16, 1, 2, 19, 4, 19, 4, 21, 22, 7, 8, 47, 32, 49, 50, 35, 52, 35, 52, 37, 38, 55, 56, 63, 48, 33, 34, 51, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3246(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 5, 6, 23, 24, 15, 0, 1, 2, 19, 4, 19, 4, 21, 22, 7, 8, 63, 48, 49, 50, 35, 52, 35, 52, 37, 38, 55, 56, 47, 32, 33, 34, 51, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3247(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 5, 6, 23, 24, 31, 0, 1, 2, 19, 4, 19, 4, 21, 22, 7, 8, 47, 48, 49, 50, 35, 52, 35, 52, 37, 38, 55, 56, 63, 32, 33, 34, 51, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3248(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 5, 6, 23, 24, 15, 16, 17, 18, 3, 4, 19, 4, 21, 22, 7, 8, 63, 32, 33, 34, 51, 52, 35, 52, 37, 38, 55, 56, 47, 48, 49, 50, 35, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3249(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 5, 6, 23, 24, 31, 16, 17, 18, 3, 4, 19, 4, 21, 22, 7, 8, 47, 32, 33, 34, 51, 52, 35, 52, 37, 38, 55, 56, 63, 48, 49, 50, 35, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3250(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 5, 6, 23, 24, 15, 0, 17, 18, 3, 4, 19, 4, 21, 22, 7, 8, 63, 48, 33, 34, 51, 52, 35, 52, 37, 38, 55, 56, 47, 32, 49, 50, 35, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3251(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 5, 6, 23, 24, 31, 0, 17, 18, 3, 4, 19, 4, 21, 22, 7, 8, 47, 48, 33, 34, 51, 52, 35, 52, 37, 38, 55, 56, 63, 32, 49, 50, 35, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3252(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 5, 6, 23, 24, 15, 16, 1, 18, 3, 4, 19, 4, 21, 22, 7, 8, 63, 32, 49, 34, 51, 52, 35, 52, 37, 38, 55, 56, 47, 48, 33, 50, 35, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3253(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 5, 6, 23, 24, 31, 16, 1, 18, 3, 4, 19, 4, 21, 22, 7, 8, 47, 32, 49, 34, 51, 52, 35, 52, 37, 38, 55, 56, 63, 48, 33, 50, 35, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3254(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 5, 6, 23, 24, 15, 0, 1, 18, 3, 4, 19, 4, 21, 22, 7, 8, 63, 48, 49, 34, 51, 52, 35, 52, 37, 38, 55, 56, 47, 32, 33, 50, 35, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3255(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 5, 6, 23, 24, 31, 0, 1, 18, 3, 4, 19, 4, 21, 22, 7, 8, 47, 48, 49, 34, 51, 52, 35, 52, 37, 38, 55, 56, 63, 32, 33, 50, 35, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3256(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 5, 6, 23, 24, 15, 16, 17, 2, 3, 4, 19, 4, 21, 22, 7, 8, 63, 32, 33, 50, 51, 52, 35, 52, 37, 38, 55, 56, 47, 48, 49, 34, 35, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3257(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 5, 6, 23, 24, 31, 16, 17, 2, 3, 4, 19, 4, 21, 22, 7, 8, 47, 32, 33, 50, 51, 52, 35, 52, 37, 38, 55, 56, 63, 48, 49, 34, 35, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3258(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 5, 6, 23, 24, 15, 0, 17, 2, 3, 4, 19, 4, 21, 22, 7, 8, 63, 48, 33, 50, 51, 52, 35, 52, 37, 38, 55, 56, 47, 32, 49, 34, 35, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3259(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 5, 6, 23, 24, 31, 0, 17, 2, 3, 4, 19, 4, 21, 22, 7, 8, 47, 48, 33, 50, 51, 52, 35, 52, 37, 38, 55, 56, 63, 32, 49, 34, 35, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3260(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 5, 6, 23, 24, 15, 16, 1, 2, 3, 4, 19, 4, 21, 22, 7, 8, 63, 32, 49, 50, 51, 52, 35, 52, 37, 38, 55, 56, 47, 48, 33, 34, 35, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3261(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 5, 6, 23, 24, 31, 16, 1, 2, 3, 4, 19, 4, 21, 22, 7, 8, 47, 32, 49, 50, 51, 52, 35, 52, 37, 38, 55, 56, 63, 48, 33, 34, 35, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3262(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 5, 6, 23, 24, 15, 0, 1, 2, 3, 4, 19, 4, 21, 22, 7, 8, 63, 48, 49, 50, 51, 52, 35, 52, 37, 38, 55, 56, 47, 32, 33, 34, 35, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3263(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 5, 6, 23, 24, 31, 0, 1, 2, 3, 4, 19, 4, 21, 22, 7, 8, 47, 48, 49, 50, 51, 52, 35, 52, 37, 38, 55, 56, 63, 32, 33, 34, 35, 36, 51, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3264(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 5, 6, 23, 24, 15, 16, 17, 18, 19, 20, 3, 4, 21, 22, 7, 8, 63, 32, 33, 34, 35, 36, 51, 52, 37, 38, 55, 56, 47, 48, 49, 50, 51, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3265(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 5, 6, 23, 24, 31, 16, 17, 18, 19, 20, 3, 4, 21, 22, 7, 8, 47, 32, 33, 34, 35, 36, 51, 52, 37, 38, 55, 56, 63, 48, 49, 50, 51, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3266(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 5, 6, 23, 24, 15, 0, 17, 18, 19, 20, 3, 4, 21, 22, 7, 8, 63, 48, 33, 34, 35, 36, 51, 52, 37, 38, 55, 56, 47, 32, 49, 50, 51, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3267(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 5, 6, 23, 24, 31, 0, 17, 18, 19, 20, 3, 4, 21, 22, 7, 8, 47, 48, 33, 34, 35, 36, 51, 52, 37, 38, 55, 56, 63, 32, 49, 50, 51, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3268(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 5, 6, 23, 24, 15, 16, 1, 18, 19, 20, 3, 4, 21, 22, 7, 8, 63, 32, 49, 34, 35, 36, 51, 52, 37, 38, 55, 56, 47, 48, 33, 50, 51, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3269(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 5, 6, 23, 24, 31, 16, 1, 18, 19, 20, 3, 4, 21, 22, 7, 8, 47, 32, 49, 34, 35, 36, 51, 52, 37, 38, 55, 56, 63, 48, 33, 50, 51, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3270(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 5, 6, 23, 24, 15, 0, 1, 18, 19, 20, 3, 4, 21, 22, 7, 8, 63, 48, 49, 34, 35, 36, 51, 52, 37, 38, 55, 56, 47, 32, 33, 50, 51, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3271(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 5, 6, 23, 24, 31, 0, 1, 18, 19, 20, 3, 4, 21, 22, 7, 8, 47, 48, 49, 34, 35, 36, 51, 52, 37, 38, 55, 56, 63, 32, 33, 50, 51, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3272(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 5, 6, 23, 24, 15, 16, 17, 2, 19, 20, 3, 4, 21, 22, 7, 8, 63, 32, 33, 50, 35, 36, 51, 52, 37, 38, 55, 56, 47, 48, 49, 34, 51, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3273(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 5, 6, 23, 24, 31, 16, 17, 2, 19, 20, 3, 4, 21, 22, 7, 8, 47, 32, 33, 50, 35, 36, 51, 52, 37, 38, 55, 56, 63, 48, 49, 34, 51, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3274(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 5, 6, 23, 24, 15, 0, 17, 2, 19, 20, 3, 4, 21, 22, 7, 8, 63, 48, 33, 50, 35, 36, 51, 52, 37, 38, 55, 56, 47, 32, 49, 34, 51, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3275(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 5, 6, 23, 24, 31, 0, 17, 2, 19, 20, 3, 4, 21, 22, 7, 8, 47, 48, 33, 50, 35, 36, 51, 52, 37, 38, 55, 56, 63, 32, 49, 34, 51, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3276(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 5, 6, 23, 24, 15, 16, 1, 2, 19, 20, 3, 4, 21, 22, 7, 8, 63, 32, 49, 50, 35, 36, 51, 52, 37, 38, 55, 56, 47, 48, 33, 34, 51, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3277(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 5, 6, 23, 24, 31, 16, 1, 2, 19, 20, 3, 4, 21, 22, 7, 8, 47, 32, 49, 50, 35, 36, 51, 52, 37, 38, 55, 56, 63, 48, 33, 34, 51, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3278(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 5, 6, 23, 24, 15, 0, 1, 2, 19, 20, 3, 4, 21, 22, 7, 8, 63, 48, 49, 50, 35, 36, 51, 52, 37, 38, 55, 56, 47, 32, 33, 34, 51, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3279(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 5, 6, 23, 24, 31, 0, 1, 2, 19, 20, 3, 4, 21, 22, 7, 8, 47, 48, 49, 50, 35, 36, 51, 52, 37, 38, 55, 56, 63, 32, 33, 34, 51, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3280(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 5, 6, 23, 24, 15, 16, 17, 18, 3, 20, 3, 4, 21, 22, 7, 8, 63, 32, 33, 34, 51, 36, 51, 52, 37, 38, 55, 56, 47, 48, 49, 50, 35, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3281(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 5, 6, 23, 24, 31, 16, 17, 18, 3, 20, 3, 4, 21, 22, 7, 8, 47, 32, 33, 34, 51, 36, 51, 52, 37, 38, 55, 56, 63, 48, 49, 50, 35, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3282(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 5, 6, 23, 24, 15, 0, 17, 18, 3, 20, 3, 4, 21, 22, 7, 8, 63, 48, 33, 34, 51, 36, 51, 52, 37, 38, 55, 56, 47, 32, 49, 50, 35, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3283(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 5, 6, 23, 24, 31, 0, 17, 18, 3, 20, 3, 4, 21, 22, 7, 8, 47, 48, 33, 34, 51, 36, 51, 52, 37, 38, 55, 56, 63, 32, 49, 50, 35, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3284(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 5, 6, 23, 24, 15, 16, 1, 18, 3, 20, 3, 4, 21, 22, 7, 8, 63, 32, 49, 34, 51, 36, 51, 52, 37, 38, 55, 56, 47, 48, 33, 50, 35, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3285(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 5, 6, 23, 24, 31, 16, 1, 18, 3, 20, 3, 4, 21, 22, 7, 8, 47, 32, 49, 34, 51, 36, 51, 52, 37, 38, 55, 56, 63, 48, 33, 50, 35, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3286(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 5, 6, 23, 24, 15, 0, 1, 18, 3, 20, 3, 4, 21, 22, 7, 8, 63, 48, 49, 34, 51, 36, 51, 52, 37, 38, 55, 56, 47, 32, 33, 50, 35, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3287(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 5, 6, 23, 24, 31, 0, 1, 18, 3, 20, 3, 4, 21, 22, 7, 8, 47, 48, 49, 34, 51, 36, 51, 52, 37, 38, 55, 56, 63, 32, 33, 50, 35, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3288(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 5, 6, 23, 24, 15, 16, 17, 2, 3, 20, 3, 4, 21, 22, 7, 8, 63, 32, 33, 50, 51, 36, 51, 52, 37, 38, 55, 56, 47, 48, 49, 34, 35, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3289(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 5, 6, 23, 24, 31, 16, 17, 2, 3, 20, 3, 4, 21, 22, 7, 8, 47, 32, 33, 50, 51, 36, 51, 52, 37, 38, 55, 56, 63, 48, 49, 34, 35, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3290(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 5, 6, 23, 24, 15, 0, 17, 2, 3, 20, 3, 4, 21, 22, 7, 8, 63, 48, 33, 50, 51, 36, 51, 52, 37, 38, 55, 56, 47, 32, 49, 34, 35, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3291(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 5, 6, 23, 24, 31, 0, 17, 2, 3, 20, 3, 4, 21, 22, 7, 8, 47, 48, 33, 50, 51, 36, 51, 52, 37, 38, 55, 56, 63, 32, 49, 34, 35, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3292(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 5, 6, 23, 24, 15, 16, 1, 2, 3, 20, 3, 4, 21, 22, 7, 8, 63, 32, 49, 50, 51, 36, 51, 52, 37, 38, 55, 56, 47, 48, 33, 34, 35, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3293(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 5, 6, 23, 24, 31, 16, 1, 2, 3, 20, 3, 4, 21, 22, 7, 8, 47, 32, 49, 50, 51, 36, 51, 52, 37, 38, 55, 56, 63, 48, 33, 34, 35, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3294(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 5, 6, 23, 24, 15, 0, 1, 2, 3, 20, 3, 4, 21, 22, 7, 8, 63, 48, 49, 50, 51, 36, 51, 52, 37, 38, 55, 56, 47, 32, 33, 34, 35, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3295(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 5, 6, 23, 24, 31, 0, 1, 2, 3, 20, 3, 4, 21, 22, 7, 8, 47, 48, 49, 50, 51, 36, 51, 52, 37, 38, 55, 56, 63, 32, 33, 34, 35, 52, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3296(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 5, 6, 23, 24, 15, 16, 17, 18, 19, 4, 3, 4, 21, 22, 7, 8, 63, 32, 33, 34, 35, 52, 51, 52, 37, 38, 55, 56, 47, 48, 49, 50, 51, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3297(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 5, 6, 23, 24, 31, 16, 17, 18, 19, 4, 3, 4, 21, 22, 7, 8, 47, 32, 33, 34, 35, 52, 51, 52, 37, 38, 55, 56, 63, 48, 49, 50, 51, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3298(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 5, 6, 23, 24, 15, 0, 17, 18, 19, 4, 3, 4, 21, 22, 7, 8, 63, 48, 33, 34, 35, 52, 51, 52, 37, 38, 55, 56, 47, 32, 49, 50, 51, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3299(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 5, 6, 23, 24, 31, 0, 17, 18, 19, 4, 3, 4, 21, 22, 7, 8, 47, 48, 33, 34, 35, 52, 51, 52, 37, 38, 55, 56, 63, 32, 49, 50, 51, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3300(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 5, 6, 23, 24, 15, 16, 1, 18, 19, 4, 3, 4, 21, 22, 7, 8, 63, 32, 49, 34, 35, 52, 51, 52, 37, 38, 55, 56, 47, 48, 33, 50, 51, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3301(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 5, 6, 23, 24, 31, 16, 1, 18, 19, 4, 3, 4, 21, 22, 7, 8, 47, 32, 49, 34, 35, 52, 51, 52, 37, 38, 55, 56, 63, 48, 33, 50, 51, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3302(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 5, 6, 23, 24, 15, 0, 1, 18, 19, 4, 3, 4, 21, 22, 7, 8, 63, 48, 49, 34, 35, 52, 51, 52, 37, 38, 55, 56, 47, 32, 33, 50, 51, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3303(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 5, 6, 23, 24, 31, 0, 1, 18, 19, 4, 3, 4, 21, 22, 7, 8, 47, 48, 49, 34, 35, 52, 51, 52, 37, 38, 55, 56, 63, 32, 33, 50, 51, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3304(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 5, 6, 23, 24, 15, 16, 17, 2, 19, 4, 3, 4, 21, 22, 7, 8, 63, 32, 33, 50, 35, 52, 51, 52, 37, 38, 55, 56, 47, 48, 49, 34, 51, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3305(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 5, 6, 23, 24, 31, 16, 17, 2, 19, 4, 3, 4, 21, 22, 7, 8, 47, 32, 33, 50, 35, 52, 51, 52, 37, 38, 55, 56, 63, 48, 49, 34, 51, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3306(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 5, 6, 23, 24, 15, 0, 17, 2, 19, 4, 3, 4, 21, 22, 7, 8, 63, 48, 33, 50, 35, 52, 51, 52, 37, 38, 55, 56, 47, 32, 49, 34, 51, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3307(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 5, 6, 23, 24, 31, 0, 17, 2, 19, 4, 3, 4, 21, 22, 7, 8, 47, 48, 33, 50, 35, 52, 51, 52, 37, 38, 55, 56, 63, 32, 49, 34, 51, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3308(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 5, 6, 23, 24, 15, 16, 1, 2, 19, 4, 3, 4, 21, 22, 7, 8, 63, 32, 49, 50, 35, 52, 51, 52, 37, 38, 55, 56, 47, 48, 33, 34, 51, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3309(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 5, 6, 23, 24, 31, 16, 1, 2, 19, 4, 3, 4, 21, 22, 7, 8, 47, 32, 49, 50, 35, 52, 51, 52, 37, 38, 55, 56, 63, 48, 33, 34, 51, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3310(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 5, 6, 23, 24, 15, 0, 1, 2, 19, 4, 3, 4, 21, 22, 7, 8, 63, 48, 49, 50, 35, 52, 51, 52, 37, 38, 55, 56, 47, 32, 33, 34, 51, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3311(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 5, 6, 23, 24, 31, 0, 1, 2, 19, 4, 3, 4, 21, 22, 7, 8, 47, 48, 49, 50, 35, 52, 51, 52, 37, 38, 55, 56, 63, 32, 33, 34, 51, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3312(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 5, 6, 23, 24, 15, 16, 17, 18, 3, 4, 3, 4, 21, 22, 7, 8, 63, 32, 33, 34, 51, 52, 51, 52, 37, 38, 55, 56, 47, 48, 49, 50, 35, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3313(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 5, 6, 23, 24, 31, 16, 17, 18, 3, 4, 3, 4, 21, 22, 7, 8, 47, 32, 33, 34, 51, 52, 51, 52, 37, 38, 55, 56, 63, 48, 49, 50, 35, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3314(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 5, 6, 23, 24, 15, 0, 17, 18, 3, 4, 3, 4, 21, 22, 7, 8, 63, 48, 33, 34, 51, 52, 51, 52, 37, 38, 55, 56, 47, 32, 49, 50, 35, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3315(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 5, 6, 23, 24, 31, 0, 17, 18, 3, 4, 3, 4, 21, 22, 7, 8, 47, 48, 33, 34, 51, 52, 51, 52, 37, 38, 55, 56, 63, 32, 49, 50, 35, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3316(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 5, 6, 23, 24, 15, 16, 1, 18, 3, 4, 3, 4, 21, 22, 7, 8, 63, 32, 49, 34, 51, 52, 51, 52, 37, 38, 55, 56, 47, 48, 33, 50, 35, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3317(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 5, 6, 23, 24, 31, 16, 1, 18, 3, 4, 3, 4, 21, 22, 7, 8, 47, 32, 49, 34, 51, 52, 51, 52, 37, 38, 55, 56, 63, 48, 33, 50, 35, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3318(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 5, 6, 23, 24, 15, 0, 1, 18, 3, 4, 3, 4, 21, 22, 7, 8, 63, 48, 49, 34, 51, 52, 51, 52, 37, 38, 55, 56, 47, 32, 33, 50, 35, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3319(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 5, 6, 23, 24, 31, 0, 1, 18, 3, 4, 3, 4, 21, 22, 7, 8, 47, 48, 49, 34, 51, 52, 51, 52, 37, 38, 55, 56, 63, 32, 33, 50, 35, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3320(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 5, 6, 23, 24, 15, 16, 17, 2, 3, 4, 3, 4, 21, 22, 7, 8, 63, 32, 33, 50, 51, 52, 51, 52, 37, 38, 55, 56, 47, 48, 49, 34, 35, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3321(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 5, 6, 23, 24, 31, 16, 17, 2, 3, 4, 3, 4, 21, 22, 7, 8, 47, 32, 33, 50, 51, 52, 51, 52, 37, 38, 55, 56, 63, 48, 49, 34, 35, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3322(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 5, 6, 23, 24, 15, 0, 17, 2, 3, 4, 3, 4, 21, 22, 7, 8, 63, 48, 33, 50, 51, 52, 51, 52, 37, 38, 55, 56, 47, 32, 49, 34, 35, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3323(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 5, 6, 23, 24, 31, 0, 17, 2, 3, 4, 3, 4, 21, 22, 7, 8, 47, 48, 33, 50, 51, 52, 51, 52, 37, 38, 55, 56, 63, 32, 49, 34, 35, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3324(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 5, 6, 23, 24, 15, 16, 1, 2, 3, 4, 3, 4, 21, 22, 7, 8, 63, 32, 49, 50, 51, 52, 51, 52, 37, 38, 55, 56, 47, 48, 33, 34, 35, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3325(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 5, 6, 23, 24, 31, 16, 1, 2, 3, 4, 3, 4, 21, 22, 7, 8, 47, 32, 49, 50, 51, 52, 51, 52, 37, 38, 55, 56, 63, 48, 33, 34, 35, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3326(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 5, 6, 23, 24, 15, 0, 1, 2, 3, 4, 3, 4, 21, 22, 7, 8, 63, 48, 49, 50, 51, 52, 51, 52, 37, 38, 55, 56, 47, 32, 33, 34, 35, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3327(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 5, 6, 23, 24, 31, 0, 1, 2, 3, 4, 3, 4, 21, 22, 7, 8, 47, 48, 49, 50, 51, 52, 51, 52, 37, 38, 55, 56, 63, 32, 33, 34, 35, 36, 35, 36, 53, 54, 39, 40>(unchecked_hashes, bitsplitted_keys);
}


