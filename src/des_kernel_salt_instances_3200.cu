#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

#ifdef DESGPU_COMPILE_ALL_SALTS

void des_25_encrypt_salt3200(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 5, 6, 23, 24, 15, 16, 17, 18, 19, 20, 19, 4, 21, 22, 7, 8, 63, 32, 33, 34, 35, 36, 35, 52, 37, 38, 55, 56, 47, 48, 49, 50, 51, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3201(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 5, 6, 23, 24, 31, 16, 17, 18, 19, 20, 19, 4, 21, 22, 7, 8, 47, 32, 33, 34, 35, 36, 35, 52, 37, 38, 55, 56, 63, 48, 49, 50, 51, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3202(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 5, 6, 23, 24, 15, 0, 17, 18, 19, 20, 19, 4, 21, 22, 7, 8, 63, 48, 33, 34, 35, 36, 35, 52, 37, 38, 55, 56, 47, 32, 49, 50, 51, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3203(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 5, 6, 23, 24, 31, 0, 17, 18, 19, 20, 19, 4, 21, 22, 7, 8, 47, 48, 33, 34, 35, 36, 35, 52, 37, 38, 55, 56, 63, 32, 49, 50, 51, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3204(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 5, 6, 23, 24, 15, 16, 1, 18, 19, 20, 19, 4, 21, 22, 7, 8, 63, 32, 49, 34, 35, 36, 35, 52, 37, 38, 55, 56, 47, 48, 33, 50, 51, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3205(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 5, 6, 23, 24, 31, 16, 1, 18, 19, 20, 19, 4, 21, 22, 7, 8, 47, 32, 49, 34, 35, 36, 35, 52, 37, 38, 55, 56, 63, 48, 33, 50, 51, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3206(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 5, 6, 23, 24, 15, 0, 1, 18, 19, 20, 19, 4, 21, 22, 7, 8, 63, 48, 49, 34, 35, 36, 35, 52, 37, 38, 55, 56, 47, 32, 33, 50, 51, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3207(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 5, 6, 23, 24, 31, 0, 1, 18, 19, 20, 19, 4, 21, 22, 7, 8, 47, 48, 49, 34, 35, 36, 35, 52, 37, 38, 55, 56, 63, 32, 33, 50, 51, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3208(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 5, 6, 23, 24, 15, 16, 17, 2, 19, 20, 19, 4, 21, 22, 7, 8, 63, 32, 33, 50, 35, 36, 35, 52, 37, 38, 55, 56, 47, 48, 49, 34, 51, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3209(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 5, 6, 23, 24, 31, 16, 17, 2, 19, 20, 19, 4, 21, 22, 7, 8, 47, 32, 33, 50, 35, 36, 35, 52, 37, 38, 55, 56, 63, 48, 49, 34, 51, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3210(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 5, 6, 23, 24, 15, 0, 17, 2, 19, 20, 19, 4, 21, 22, 7, 8, 63, 48, 33, 50, 35, 36, 35, 52, 37, 38, 55, 56, 47, 32, 49, 34, 51, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3211(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 5, 6, 23, 24, 31, 0, 17, 2, 19, 20, 19, 4, 21, 22, 7, 8, 47, 48, 33, 50, 35, 36, 35, 52, 37, 38, 55, 56, 63, 32, 49, 34, 51, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3212(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 5, 6, 23, 24, 15, 16, 1, 2, 19, 20, 19, 4, 21, 22, 7, 8, 63, 32, 49, 50, 35, 36, 35, 52, 37, 38, 55, 56, 47, 48, 33, 34, 51, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3213(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 5, 6, 23, 24, 31, 16, 1, 2, 19, 20, 19, 4, 21, 22, 7, 8, 47, 32, 49, 50, 35, 36, 35, 52, 37, 38, 55, 56, 63, 48, 33, 34, 51, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3214(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 5, 6, 23, 24, 15, 0, 1, 2, 19, 20, 19, 4, 21, 22, 7, 8, 63, 48, 49, 50, 35, 36, 35, 52, 37, 38, 55, 56, 47, 32, 33, 34, 51, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3215(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 5, 6, 23, 24, 31, 0, 1, 2, 19, 20, 19, 4, 21, 22, 7, 8, 47, 48, 49, 50, 35, 36, 35, 52, 37, 38, 55, 56, 63, 32, 33, 34, 51, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3216(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 5, 6, 23, 24, 15, 16, 17, 18, 3, 20, 19, 4, 21, 22, 7, 8, 63, 32, 33, 34, 51, 36, 35, 52, 37, 38, 55, 56, 47, 48, 49, 50, 35, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3217(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 5, 6, 23, 24, 31, 16, 17, 18, 3, 20, 19, 4, 21, 22, 7, 8, 47, 32, 33, 34, 51, 36, 35, 52, 37, 38, 55, 56, 63, 48, 49, 50, 35, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3218(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 5, 6, 23, 24, 15, 0, 17, 18, 3, 20, 19, 4, 21, 22, 7, 8, 63, 48, 33, 34, 51, 36, 35, 52, 37, 38, 55, 56, 47, 32, 49, 50, 35, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3219(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 5, 6, 23, 24, 31, 0, 17, 18, 3, 20, 19, 4, 21, 22, 7, 8, 47, 48, 33, 34, 51, 36, 35, 52, 37, 38, 55, 56, 63, 32, 49, 50, 35, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3220(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 5, 6, 23, 24, 15, 16, 1, 18, 3, 20, 19, 4, 21, 22, 7, 8, 63, 32, 49, 34, 51, 36, 35, 52, 37, 38, 55, 56, 47, 48, 33, 50, 35, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3221(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 5, 6, 23, 24, 31, 16, 1, 18, 3, 20, 19, 4, 21, 22, 7, 8, 47, 32, 49, 34, 51, 36, 35, 52, 37, 38, 55, 56, 63, 48, 33, 50, 35, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3222(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 5, 6, 23, 24, 15, 0, 1, 18, 3, 20, 19, 4, 21, 22, 7, 8, 63, 48, 49, 34, 51, 36, 35, 52, 37, 38, 55, 56, 47, 32, 33, 50, 35, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3223(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 5, 6, 23, 24, 31, 0, 1, 18, 3, 20, 19, 4, 21, 22, 7, 8, 47, 48, 49, 34, 51, 36, 35, 52, 37, 38, 55, 56, 63, 32, 33, 50, 35, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3224(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 5, 6, 23, 24, 15, 16, 17, 2, 3, 20, 19, 4, 21, 22, 7, 8, 63, 32, 33, 50, 51, 36, 35, 52, 37, 38, 55, 56, 47, 48, 49, 34, 35, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3225(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 5, 6, 23, 24, 31, 16, 17, 2, 3, 20, 19, 4, 21, 22, 7, 8, 47, 32, 33, 50, 51, 36, 35, 52, 37, 38, 55, 56, 63, 48, 49, 34, 35, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3226(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 5, 6, 23, 24, 15, 0, 17, 2, 3, 20, 19, 4, 21, 22, 7, 8, 63, 48, 33, 50, 51, 36, 35, 52, 37, 38, 55, 56, 47, 32, 49, 34, 35, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3227(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 5, 6, 23, 24, 31, 0, 17, 2, 3, 20, 19, 4, 21, 22, 7, 8, 47, 48, 33, 50, 51, 36, 35, 52, 37, 38, 55, 56, 63, 32, 49, 34, 35, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3228(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 5, 6, 23, 24, 15, 16, 1, 2, 3, 20, 19, 4, 21, 22, 7, 8, 63, 32, 49, 50, 51, 36, 35, 52, 37, 38, 55, 56, 47, 48, 33, 34, 35, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3229(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 5, 6, 23, 24, 31, 16, 1, 2, 3, 20, 19, 4, 21, 22, 7, 8, 47, 32, 49, 50, 51, 36, 35, 52, 37, 38, 55, 56, 63, 48, 33, 34, 35, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3230(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 5, 6, 23, 24, 15, 0, 1, 2, 3, 20, 19, 4, 21, 22, 7, 8, 63, 48, 49, 50, 51, 36, 35, 52, 37, 38, 55, 56, 47, 32, 33, 34, 35, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3231(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 5, 6, 23, 24, 31, 0, 1, 2, 3, 20, 19, 4, 21, 22, 7, 8, 47, 48, 49, 50, 51, 36, 35, 52, 37, 38, 55, 56, 63, 32, 33, 34, 35, 52, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3232(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 5, 6, 23, 24, 15, 16, 17, 18, 19, 4, 19, 4, 21, 22, 7, 8, 63, 32, 33, 34, 35, 52, 35, 52, 37, 38, 55, 56, 47, 48, 49, 50, 51, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3233(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 5, 6, 23, 24, 31, 16, 17, 18, 19, 4, 19, 4, 21, 22, 7, 8, 47, 32, 33, 34, 35, 52, 35, 52, 37, 38, 55, 56, 63, 48, 49, 50, 51, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3234(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 5, 6, 23, 24, 15, 0, 17, 18, 19, 4, 19, 4, 21, 22, 7, 8, 63, 48, 33, 34, 35, 52, 35, 52, 37, 38, 55, 56, 47, 32, 49, 50, 51, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3235(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 5, 6, 23, 24, 31, 0, 17, 18, 19, 4, 19, 4, 21, 22, 7, 8, 47, 48, 33, 34, 35, 52, 35, 52, 37, 38, 55, 56, 63, 32, 49, 50, 51, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3236(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 5, 6, 23, 24, 15, 16, 1, 18, 19, 4, 19, 4, 21, 22, 7, 8, 63, 32, 49, 34, 35, 52, 35, 52, 37, 38, 55, 56, 47, 48, 33, 50, 51, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3237(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 5, 6, 23, 24, 31, 16, 1, 18, 19, 4, 19, 4, 21, 22, 7, 8, 47, 32, 49, 34, 35, 52, 35, 52, 37, 38, 55, 56, 63, 48, 33, 50, 51, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3238(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 5, 6, 23, 24, 15, 0, 1, 18, 19, 4, 19, 4, 21, 22, 7, 8, 63, 48, 49, 34, 35, 52, 35, 52, 37, 38, 55, 56, 47, 32, 33, 50, 51, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3239(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 5, 6, 23, 24, 31, 0, 1, 18, 19, 4, 19, 4, 21, 22, 7, 8, 47, 48, 49, 34, 35, 52, 35, 52, 37, 38, 55, 56, 63, 32, 33, 50, 51, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3240(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 5, 6, 23, 24, 15, 16, 17, 2, 19, 4, 19, 4, 21, 22, 7, 8, 63, 32, 33, 50, 35, 52, 35, 52, 37, 38, 55, 56, 47, 48, 49, 34, 51, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3241(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 5, 6, 23, 24, 31, 16, 17, 2, 19, 4, 19, 4, 21, 22, 7, 8, 47, 32, 33, 50, 35, 52, 35, 52, 37, 38, 55, 56, 63, 48, 49, 34, 51, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3242(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 5, 6, 23, 24, 15, 0, 17, 2, 19, 4, 19, 4, 21, 22, 7, 8, 63, 48, 33, 50, 35, 52, 35, 52, 37, 38, 55, 56, 47, 32, 49, 34, 51, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3243(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 5, 6, 23, 24, 31, 0, 17, 2, 19, 4, 19, 4, 21, 22, 7, 8, 47, 48, 33, 50, 35, 52, 35, 52, 37, 38, 55, 56, 63, 32, 49, 34, 51, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3244(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 5, 6, 23, 24, 15, 16, 1, 2, 19, 4, 19, 4, 21, 22, 7, 8, 63, 32, 49, 50, 35, 52, 35, 52, 37, 38, 55, 56, 47, 48, 33, 34, 51, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3245(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 5, 6, 23, 24, 31, 16, 1, 2, 19, 4, 19, 4, 21, 22, 7, 8, 47, 32, 49, 50, 35, 52, 35, 52, 37, 38, 55, 56, 63, 48, 33, 34, 51, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3246(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 5, 6, 23, 24, 15, 0, 1, 2, 19, 4, 19, 4, 21, 22, 7, 8, 63, 48, 49, 50, 35, 52, 35, 52, 37, 38, 55, 56, 47, 32, 33, 34, 51, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3247(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 5, 6, 23, 24, 31, 0, 1, 2, 19, 4, 19, 4, 21, 22, 7, 8, 47, 48, 49, 50, 35, 52, 35, 52, 37, 38, 55, 56, 63, 32, 33, 34, 51, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3248(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 5, 6, 23, 24, 15, 16, 17, 18, 3, 4, 19, 4, 21, 22, 7, 8, 63, 32, 33, 34, 51, 52, 35, 52, 37, 38, 55, 56, 47, 48, 49, 50, 35, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3249(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 5, 6, 23, 24, 31, 16, 17, 18, 3, 4, 19, 4, 21, 22, 7, 8, 47, 32, 33, 34, 51, 52, 35, 52, 37, 38, 55, 56, 63, 48, 49, 50, 35, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3250(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 5, 6, 23, 24, 15, 0, 17, 18, 3, 4, 19, 4, 21, 22, 7, 8, 63, 48, 33, 34, 51, 52, 35, 52, 37, 38, 55, 56, 47, 32, 49, 50, 35, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3251(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 5, 6, 23, 24, 31, 0, 17, 18, 3, 4, 19, 4, 21, 22, 7, 8, 47, 48, 33, 34, 51, 52, 35, 52, 37, 38, 55, 56, 63, 32, 49, 50, 35, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3252(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 5, 6, 23, 24, 15, 16, 1, 18, 3, 4, 19, 4, 21, 22, 7, 8, 63, 32, 49, 34, 51, 52, 35, 52, 37, 38, 55, 56, 47, 48, 33, 50, 35, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3253(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 5, 6, 23, 24, 31, 16, 1, 18, 3, 4, 19, 4, 21, 22, 7, 8, 47, 32, 49, 34, 51, 52, 35, 52, 37, 38, 55, 56, 63, 48, 33, 50, 35, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3254(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 5, 6, 23, 24, 15, 0, 1, 18, 3, 4, 19, 4, 21, 22, 7, 8, 63, 48, 49, 34, 51, 52, 35, 52, 37, 38, 55, 56, 47, 32, 33, 50, 35, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3255(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 5, 6, 23, 24, 31, 0, 1, 18, 3, 4, 19, 4, 21, 22, 7, 8, 47, 48, 49, 34, 51, 52, 35, 52, 37, 38, 55, 56, 63, 32, 33, 50, 35, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3256(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 5, 6, 23, 24, 15, 16, 17, 2, 3, 4, 19, 4, 21, 22, 7, 8, 63, 32, 33, 50, 51, 52, 35, 52, 37, 38, 55, 56, 47, 48, 49, 34, 35, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3257(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 5, 6, 23, 24, 31, 16, 17, 2, 3, 4, 19, 4, 21, 22, 7, 8, 47, 32, 33, 50, 51, 52, 35, 52, 37, 38, 55, 56, 63, 48, 49, 34, 35, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3258(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 5, 6, 23, 24, 15, 0, 17, 2, 3, 4, 19, 4, 21, 22, 7, 8, 63, 48, 33, 50, 51, 52, 35, 52, 37, 38, 55, 56, 47, 32, 49, 34, 35, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3259(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 5, 6, 23, 24, 31, 0, 17, 2, 3, 4, 19, 4, 21, 22, 7, 8, 47, 48, 33, 50, 51, 52, 35, 52, 37, 38, 55, 56, 63, 32, 49, 34, 35, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3260(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 5, 6, 23, 24, 15, 16, 1, 2, 3, 4, 19, 4, 21, 22, 7, 8, 63, 32, 49, 50, 51, 52, 35, 52, 37, 38, 55, 56, 47, 48, 33, 34, 35, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3261(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 5, 6, 23, 24, 31, 16, 1, 2, 3, 4, 19, 4, 21, 22, 7, 8, 47, 32, 49, 50, 51, 52, 35, 52, 37, 38, 55, 56, 63, 48, 33, 34, 35, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3262(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 5, 6, 23, 24, 15, 0, 1, 2, 3, 4, 19, 4, 21, 22, 7, 8, 63, 48, 49, 50, 51, 52, 35, 52, 37, 38, 55, 56, 47, 32, 33, 34, 35, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3263(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 5, 6, 23, 24, 31, 0, 1, 2, 3, 4, 19, 4, 21, 22, 7, 8, 47, 48, 49, 50, 51, 52, 35, 52, 37, 38, 55, 56, 63, 32, 33, 34, 35, 36, 51, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3264(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 5, 6, 23, 24, 15, 16, 17, 18, 19, 20, 3, 4, 21, 22, 7, 8, 63, 32, 33, 34, 35, 36, 51, 52, 37, 38, 55, 56, 47, 48, 49, 50, 51, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3265(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 5, 6, 23, 24, 31, 16, 17, 18, 19, 20, 3, 4, 21, 22, 7, 8, 47, 32, 33, 34, 35, 36, 51, 52, 37, 38, 55, 56, 63, 48, 49, 50, 51, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3266(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 5, 6, 23, 24, 15, 0, 17, 18, 19, 20, 3, 4, 21, 22, 7, 8, 63, 48, 33, 34, 35, 36, 51, 52, 37, 38, 55, 56, 47, 32, 49, 50, 51, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3267(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 5, 6, 23, 24, 31, 0, 17, 18, 19, 20, 3, 4, 21, 22, 7, 8, 47, 48, 33, 34, 35, 36, 51, 52, 37, 38, 55, 56, 63, 32, 49, 50, 51, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3268(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 5, 6, 23, 24, 15, 16, 1, 18, 19, 20, 3, 4, 21, 22, 7, 8, 63, 32, 49, 34, 35, 36, 51, 52, 37, 38, 55, 56, 47, 48, 33, 50, 51, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3269(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 5, 6, 23, 24, 31, 16, 1, 18, 19, 20, 3, 4, 21, 22, 7, 8, 47, 32, 49, 34, 35, 36, 51, 52, 37, 38, 55, 56, 63, 48, 33, 50, 51, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3270(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 5, 6, 23, 24, 15, 0, 1, 18, 19, 20, 3, 4, 21, 22, 7, 8, 63, 48, 49, 34, 35, 36, 51, 52, 37, 38, 55, 56, 47, 32, 33, 50, 51, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3271(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 5, 6, 23, 24, 31, 0, 1, 18, 19, 20, 3, 4, 21, 22, 7, 8, 47, 48, 49, 34, 35, 36, 51, 52, 37, 38, 55, 56, 63, 32, 33, 50, 51, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3272(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 5, 6, 23, 24, 15, 16, 17, 2, 19, 20, 3, 4, 21, 22, 7, 8, 63, 32, 33, 50, 35, 36, 51, 52, 37, 38, 55, 56, 47, 48, 49, 34, 51, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3273(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 5, 6, 23, 24, 31, 16, 17, 2, 19, 20, 3, 4, 21, 22, 7, 8, 47, 32, 33, 50, 35, 36, 51, 52, 37, 38, 55, 56, 63, 48, 49, 34, 51, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3274(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 5, 6, 23, 24, 15, 0, 17, 2, 19, 20, 3, 4, 21, 22, 7, 8, 63, 48, 33, 50, 35, 36, 51, 52, 37, 38, 55, 56, 47, 32, 49, 34, 51, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3275(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 5, 6, 23, 24, 31, 0, 17, 2, 19, 20, 3, 4, 21, 22, 7, 8, 47, 48, 33, 50, 35, 36, 51, 52, 37, 38, 55, 56, 63, 32, 49, 34, 51, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3276(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 5, 6, 23, 24, 15, 16, 1, 2, 19, 20, 3, 4, 21, 22, 7, 8, 63, 32, 49, 50, 35, 36, 51, 52, 37, 38, 55, 56, 47, 48, 33, 34, 51, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3277(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 5, 6, 23, 24, 31, 16, 1, 2, 19, 20, 3, 4, 21, 22, 7, 8, 47, 32, 49, 50, 35, 36, 51, 52, 37, 38, 55, 56, 63, 48, 33, 34, 51, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3278(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 5, 6, 23, 24, 15, 0, 1, 2, 19, 20, 3, 4, 21, 22, 7, 8, 63, 48, 49, 50, 35, 36, 51, 52, 37, 38, 55, 56, 47, 32, 33, 34, 51, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3279(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 5, 6, 23, 24, 31, 0, 1, 2, 19, 20, 3, 4, 21, 22, 7, 8, 47, 48, 49, 50, 35, 36, 51, 52, 37, 38, 55, 56, 63, 32, 33, 34, 51, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3280(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 5, 6, 23, 24, 15, 16, 17, 18, 3, 20, 3, 4, 21, 22, 7, 8, 63, 32, 33, 34, 51, 36, 51, 52, 37, 38, 55, 56, 47, 48, 49, 50, 35, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3281(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 5, 6, 23, 24, 31, 16, 17, 18, 3, 20, 3, 4, 21, 22, 7, 8, 47, 32, 33, 34, 51, 36, 51, 52, 37, 38, 55, 56, 63, 48, 49, 50, 35, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3282(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 5, 6, 23, 24, 15, 0, 17, 18, 3, 20, 3, 4, 21, 22, 7, 8, 63, 48, 33, 34, 51, 36, 51, 52, 37, 38, 55, 56, 47, 32, 49, 50, 35, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3283(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 5, 6, 23, 24, 31, 0, 17, 18, 3, 20, 3, 4, 21, 22, 7, 8, 47, 48, 33, 34, 51, 36, 51, 52, 37, 38, 55, 56, 63, 32, 49, 50, 35, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3284(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 5, 6, 23, 24, 15, 16, 1, 18, 3, 20, 3, 4, 21, 22, 7, 8, 63, 32, 49, 34, 51, 36, 51, 52, 37, 38, 55, 56, 47, 48, 33, 50, 35, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3285(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 5, 6, 23, 24, 31, 16, 1, 18, 3, 20, 3, 4, 21, 22, 7, 8, 47, 32, 49, 34, 51, 36, 51, 52, 37, 38, 55, 56, 63, 48, 33, 50, 35, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3286(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 5, 6, 23, 24, 15, 0, 1, 18, 3, 20, 3, 4, 21, 22, 7, 8, 63, 48, 49, 34, 51, 36, 51, 52, 37, 38, 55, 56, 47, 32, 33, 50, 35, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3287(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 5, 6, 23, 24, 31, 0, 1, 18, 3, 20, 3, 4, 21, 22, 7, 8, 47, 48, 49, 34, 51, 36, 51, 52, 37, 38, 55, 56, 63, 32, 33, 50, 35, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3288(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 5, 6, 23, 24, 15, 16, 17, 2, 3, 20, 3, 4, 21, 22, 7, 8, 63, 32, 33, 50, 51, 36, 51, 52, 37, 38, 55, 56, 47, 48, 49, 34, 35, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3289(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 5, 6, 23, 24, 31, 16, 17, 2, 3, 20, 3, 4, 21, 22, 7, 8, 47, 32, 33, 50, 51, 36, 51, 52, 37, 38, 55, 56, 63, 48, 49, 34, 35, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3290(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 5, 6, 23, 24, 15, 0, 17, 2, 3, 20, 3, 4, 21, 22, 7, 8, 63, 48, 33, 50, 51, 36, 51, 52, 37, 38, 55, 56, 47, 32, 49, 34, 35, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3291(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 5, 6, 23, 24, 31, 0, 17, 2, 3, 20, 3, 4, 21, 22, 7, 8, 47, 48, 33, 50, 51, 36, 51, 52, 37, 38, 55, 56, 63, 32, 49, 34, 35, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3292(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 5, 6, 23, 24, 15, 16, 1, 2, 3, 20, 3, 4, 21, 22, 7, 8, 63, 32, 49, 50, 51, 36, 51, 52, 37, 38, 55, 56, 47, 48, 33, 34, 35, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3293(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 5, 6, 23, 24, 31, 16, 1, 2, 3, 20, 3, 4, 21, 22, 7, 8, 47, 32, 49, 50, 51, 36, 51, 52, 37, 38, 55, 56, 63, 48, 33, 34, 35, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3294(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 5, 6, 23, 24, 15, 0, 1, 2, 3, 20, 3, 4, 21, 22, 7, 8, 63, 48, 49, 50, 51, 36, 51, 52, 37, 38, 55, 56, 47, 32, 33, 34, 35, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3295(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 5, 6, 23, 24, 31, 0, 1, 2, 3, 20, 3, 4, 21, 22, 7, 8, 47, 48, 49, 50, 51, 36, 51, 52, 37, 38, 55, 56, 63, 32, 33, 34, 35, 52, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3296(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 5, 6, 23, 24, 15, 16, 17, 18, 19, 4, 3, 4, 21, 22, 7, 8, 63, 32, 33, 34, 35, 52, 51, 52, 37, 38, 55, 56, 47, 48, 49, 50, 51, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3297(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 5, 6, 23, 24, 31, 16, 17, 18, 19, 4, 3, 4, 21, 22, 7, 8, 47, 32, 33, 34, 35, 52, 51, 52, 37, 38, 55, 56, 63, 48, 49, 50, 51, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3298(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 5, 6, 23, 24, 15, 0, 17, 18, 19, 4, 3, 4, 21, 22, 7, 8, 63, 48, 33, 34, 35, 52, 51, 52, 37, 38, 55, 56, 47, 32, 49, 50, 51, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3299(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 5, 6, 23, 24, 31, 0, 17, 18, 19, 4, 3, 4, 21, 22, 7, 8, 47, 48, 33, 34, 35, 52, 51, 52, 37, 38, 55, 56, 63, 32, 49, 50, 51, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3300(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 5, 6, 23, 24, 15, 16, 1, 18, 19, 4, 3, 4, 21, 22, 7, 8, 63, 32, 49, 34, 35, 52, 51, 52, 37, 38, 55, 56, 47, 48, 33, 50, 51, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3301(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 5, 6, 23, 24, 31, 16, 1, 18, 19, 4, 3, 4, 21, 22, 7, 8, 47, 32, 49, 34, 35, 52, 51, 52, 37, 38, 55, 56, 63, 48, 33, 50, 51, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3302(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 5, 6, 23, 24, 15, 0, 1, 18, 19, 4, 3, 4, 21, 22, 7, 8, 63, 48, 49, 34, 35, 52, 51, 52, 37, 38, 55, 56, 47, 32, 33, 50, 51, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3303(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 5, 6, 23, 24, 31, 0, 1, 18, 19, 4, 3, 4, 21, 22, 7, 8, 47, 48, 49, 34, 35, 52, 51, 52, 37, 38, 55, 56, 63, 32, 33, 50, 51, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3304(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 5, 6, 23, 24, 15, 16, 17, 2, 19, 4, 3, 4, 21, 22, 7, 8, 63, 32, 33, 50, 35, 52, 51, 52, 37, 38, 55, 56, 47, 48, 49, 34, 51, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3305(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 5, 6, 23, 24, 31, 16, 17, 2, 19, 4, 3, 4, 21, 22, 7, 8, 47, 32, 33, 50, 35, 52, 51, 52, 37, 38, 55, 56, 63, 48, 49, 34, 51, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3306(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 5, 6, 23, 24, 15, 0, 17, 2, 19, 4, 3, 4, 21, 22, 7, 8, 63, 48, 33, 50, 35, 52, 51, 52, 37, 38, 55, 56, 47, 32, 49, 34, 51, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3307(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 5, 6, 23, 24, 31, 0, 17, 2, 19, 4, 3, 4, 21, 22, 7, 8, 47, 48, 33, 50, 35, 52, 51, 52, 37, 38, 55, 56, 63, 32, 49, 34, 51, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3308(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 5, 6, 23, 24, 15, 16, 1, 2, 19, 4, 3, 4, 21, 22, 7, 8, 63, 32, 49, 50, 35, 52, 51, 52, 37, 38, 55, 56, 47, 48, 33, 34, 51, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3309(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 5, 6, 23, 24, 31, 16, 1, 2, 19, 4, 3, 4, 21, 22, 7, 8, 47, 32, 49, 50, 35, 52, 51, 52, 37, 38, 55, 56, 63, 48, 33, 34, 51, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3310(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 5, 6, 23, 24, 15, 0, 1, 2, 19, 4, 3, 4, 21, 22, 7, 8, 63, 48, 49, 50, 35, 52, 51, 52, 37, 38, 55, 56, 47, 32, 33, 34, 51, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3311(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 5, 6, 23, 24, 31, 0, 1, 2, 19, 4, 3, 4, 21, 22, 7, 8, 47, 48, 49, 50, 35, 52, 51, 52, 37, 38, 55, 56, 63, 32, 33, 34, 51, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3312(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 5, 6, 23, 24, 15, 16, 17, 18, 3, 4, 3, 4, 21, 22, 7, 8, 63, 32, 33, 34, 51, 52, 51, 52, 37, 38, 55, 56, 47, 48, 49, 50, 35, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3313(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 5, 6, 23, 24, 31, 16, 17, 18, 3, 4, 3, 4, 21, 22, 7, 8, 47, 32, 33, 34, 51, 52, 51, 52, 37, 38, 55, 56, 63, 48, 49, 50, 35, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3314(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 5, 6, 23, 24, 15, 0, 17, 18, 3, 4, 3, 4, 21, 22, 7, 8, 63, 48, 33, 34, 51, 52, 51, 52, 37, 38, 55, 56, 47, 32, 49, 50, 35, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3315(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 5, 6, 23, 24, 31, 0, 17, 18, 3, 4, 3, 4, 21, 22, 7, 8, 47, 48, 33, 34, 51, 52, 51, 52, 37, 38, 55, 56, 63, 32, 49, 50, 35, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3316(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 5, 6, 23, 24, 15, 16, 1, 18, 3, 4, 3, 4, 21, 22, 7, 8, 63, 32, 49, 34, 51, 52, 51, 52, 37, 38, 55, 56, 47, 48, 33, 50, 35, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3317(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 5, 6, 23, 24, 31, 16, 1, 18, 3, 4, 3, 4, 21, 22, 7, 8, 47, 32, 49, 34, 51, 52, 51, 52, 37, 38, 55, 56, 63, 48, 33, 50, 35, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3318(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 5, 6, 23, 24, 15, 0, 1, 18, 3, 4, 3, 4, 21, 22, 7, 8, 63, 48, 49, 34, 51, 52, 51, 52, 37, 38, 55, 56, 47, 32, 33, 50, 35, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3319(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 5, 6, 23, 24, 31, 0, 1, 18, 3, 4, 3, 4, 21, 22, 7, 8, 47, 48, 49, 34, 51, 52, 51, 52, 37, 38, 55, 56, 63, 32, 33, 50, 35, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3320(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 5, 6, 23, 24, 15, 16, 17, 2, 3, 4, 3, 4, 21, 22, 7, 8, 63, 32, 33, 50, 51, 52, 51, 52, 37, 38, 55, 56, 47, 48, 49, 34, 35, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3321(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 5, 6, 23, 24, 31, 16, 17, 2, 3, 4, 3, 4, 21, 22, 7, 8, 47, 32, 33, 50, 51, 52, 51, 52, 37, 38, 55, 56, 63, 48, 49, 34, 35, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3322(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 5, 6, 23, 24, 15, 0, 17, 2, 3, 4, 3, 4, 21, 22, 7, 8, 63, 48, 33, 50, 51, 52, 51, 52, 37, 38, 55, 56, 47, 32, 49, 34, 35, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3323(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 5, 6, 23, 24, 31, 0, 17, 2, 3, 4, 3, 4, 21, 22, 7, 8, 47, 48, 33, 50, 51, 52, 51, 52, 37, 38, 55, 56, 63, 32, 49, 34, 35, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3324(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 5, 6, 23, 24, 15, 16, 1, 2, 3, 4, 3, 4, 21, 22, 7, 8, 63, 32, 49, 50, 51, 52, 51, 52, 37, 38, 55, 56, 47, 48, 33, 34, 35, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3325(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 5, 6, 23, 24, 31, 16, 1, 2, 3, 4, 3, 4, 21, 22, 7, 8, 47, 32, 49, 50, 51, 52, 51, 52, 37, 38, 55, 56, 63, 48, 33, 34, 35, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3326(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 5, 6, 23, 24, 15, 0, 1, 2, 3, 4, 3, 4, 21, 22, 7, 8, 63, 48, 49, 50, 51, 52, 51, 52, 37, 38, 55, 56, 47, 32, 33, 34, 35, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3327(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 5, 6, 23, 24, 31, 0, 1, 2, 3, 4, 3, 4, 21, 22, 7, 8, 47, 48, 49, 50, 51, 52, 51, 52, 37, 38, 55, 56, 63, 32, 33, 34, 35, 36, 35, 36, 53, 54, 39, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

#endif // DESGPU_COMPILE_ALL_SALTS


