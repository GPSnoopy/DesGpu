#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt2688(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 5, 22, 7, 24, 15, 16, 17, 18, 19, 20, 19, 4, 21, 6, 23, 8, 63, 32, 33, 34, 35, 36, 35, 52, 37, 54, 39, 56, 47, 48, 49, 50, 51, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2689(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 5, 22, 7, 24, 31, 16, 17, 18, 19, 20, 19, 4, 21, 6, 23, 8, 47, 32, 33, 34, 35, 36, 35, 52, 37, 54, 39, 56, 63, 48, 49, 50, 51, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2690(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 5, 22, 7, 24, 15, 0, 17, 18, 19, 20, 19, 4, 21, 6, 23, 8, 63, 48, 33, 34, 35, 36, 35, 52, 37, 54, 39, 56, 47, 32, 49, 50, 51, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2691(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 5, 22, 7, 24, 31, 0, 17, 18, 19, 20, 19, 4, 21, 6, 23, 8, 47, 48, 33, 34, 35, 36, 35, 52, 37, 54, 39, 56, 63, 32, 49, 50, 51, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2692(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 5, 22, 7, 24, 15, 16, 1, 18, 19, 20, 19, 4, 21, 6, 23, 8, 63, 32, 49, 34, 35, 36, 35, 52, 37, 54, 39, 56, 47, 48, 33, 50, 51, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2693(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 5, 22, 7, 24, 31, 16, 1, 18, 19, 20, 19, 4, 21, 6, 23, 8, 47, 32, 49, 34, 35, 36, 35, 52, 37, 54, 39, 56, 63, 48, 33, 50, 51, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2694(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 5, 22, 7, 24, 15, 0, 1, 18, 19, 20, 19, 4, 21, 6, 23, 8, 63, 48, 49, 34, 35, 36, 35, 52, 37, 54, 39, 56, 47, 32, 33, 50, 51, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2695(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 5, 22, 7, 24, 31, 0, 1, 18, 19, 20, 19, 4, 21, 6, 23, 8, 47, 48, 49, 34, 35, 36, 35, 52, 37, 54, 39, 56, 63, 32, 33, 50, 51, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2696(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 5, 22, 7, 24, 15, 16, 17, 2, 19, 20, 19, 4, 21, 6, 23, 8, 63, 32, 33, 50, 35, 36, 35, 52, 37, 54, 39, 56, 47, 48, 49, 34, 51, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2697(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 5, 22, 7, 24, 31, 16, 17, 2, 19, 20, 19, 4, 21, 6, 23, 8, 47, 32, 33, 50, 35, 36, 35, 52, 37, 54, 39, 56, 63, 48, 49, 34, 51, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2698(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 5, 22, 7, 24, 15, 0, 17, 2, 19, 20, 19, 4, 21, 6, 23, 8, 63, 48, 33, 50, 35, 36, 35, 52, 37, 54, 39, 56, 47, 32, 49, 34, 51, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2699(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 5, 22, 7, 24, 31, 0, 17, 2, 19, 20, 19, 4, 21, 6, 23, 8, 47, 48, 33, 50, 35, 36, 35, 52, 37, 54, 39, 56, 63, 32, 49, 34, 51, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2700(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 5, 22, 7, 24, 15, 16, 1, 2, 19, 20, 19, 4, 21, 6, 23, 8, 63, 32, 49, 50, 35, 36, 35, 52, 37, 54, 39, 56, 47, 48, 33, 34, 51, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2701(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 5, 22, 7, 24, 31, 16, 1, 2, 19, 20, 19, 4, 21, 6, 23, 8, 47, 32, 49, 50, 35, 36, 35, 52, 37, 54, 39, 56, 63, 48, 33, 34, 51, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2702(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 5, 22, 7, 24, 15, 0, 1, 2, 19, 20, 19, 4, 21, 6, 23, 8, 63, 48, 49, 50, 35, 36, 35, 52, 37, 54, 39, 56, 47, 32, 33, 34, 51, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2703(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 5, 22, 7, 24, 31, 0, 1, 2, 19, 20, 19, 4, 21, 6, 23, 8, 47, 48, 49, 50, 35, 36, 35, 52, 37, 54, 39, 56, 63, 32, 33, 34, 51, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2704(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 5, 22, 7, 24, 15, 16, 17, 18, 3, 20, 19, 4, 21, 6, 23, 8, 63, 32, 33, 34, 51, 36, 35, 52, 37, 54, 39, 56, 47, 48, 49, 50, 35, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2705(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 5, 22, 7, 24, 31, 16, 17, 18, 3, 20, 19, 4, 21, 6, 23, 8, 47, 32, 33, 34, 51, 36, 35, 52, 37, 54, 39, 56, 63, 48, 49, 50, 35, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2706(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 5, 22, 7, 24, 15, 0, 17, 18, 3, 20, 19, 4, 21, 6, 23, 8, 63, 48, 33, 34, 51, 36, 35, 52, 37, 54, 39, 56, 47, 32, 49, 50, 35, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2707(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 5, 22, 7, 24, 31, 0, 17, 18, 3, 20, 19, 4, 21, 6, 23, 8, 47, 48, 33, 34, 51, 36, 35, 52, 37, 54, 39, 56, 63, 32, 49, 50, 35, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2708(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 5, 22, 7, 24, 15, 16, 1, 18, 3, 20, 19, 4, 21, 6, 23, 8, 63, 32, 49, 34, 51, 36, 35, 52, 37, 54, 39, 56, 47, 48, 33, 50, 35, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2709(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 5, 22, 7, 24, 31, 16, 1, 18, 3, 20, 19, 4, 21, 6, 23, 8, 47, 32, 49, 34, 51, 36, 35, 52, 37, 54, 39, 56, 63, 48, 33, 50, 35, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2710(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 5, 22, 7, 24, 15, 0, 1, 18, 3, 20, 19, 4, 21, 6, 23, 8, 63, 48, 49, 34, 51, 36, 35, 52, 37, 54, 39, 56, 47, 32, 33, 50, 35, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2711(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 5, 22, 7, 24, 31, 0, 1, 18, 3, 20, 19, 4, 21, 6, 23, 8, 47, 48, 49, 34, 51, 36, 35, 52, 37, 54, 39, 56, 63, 32, 33, 50, 35, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2712(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 5, 22, 7, 24, 15, 16, 17, 2, 3, 20, 19, 4, 21, 6, 23, 8, 63, 32, 33, 50, 51, 36, 35, 52, 37, 54, 39, 56, 47, 48, 49, 34, 35, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2713(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 5, 22, 7, 24, 31, 16, 17, 2, 3, 20, 19, 4, 21, 6, 23, 8, 47, 32, 33, 50, 51, 36, 35, 52, 37, 54, 39, 56, 63, 48, 49, 34, 35, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2714(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 5, 22, 7, 24, 15, 0, 17, 2, 3, 20, 19, 4, 21, 6, 23, 8, 63, 48, 33, 50, 51, 36, 35, 52, 37, 54, 39, 56, 47, 32, 49, 34, 35, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2715(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 5, 22, 7, 24, 31, 0, 17, 2, 3, 20, 19, 4, 21, 6, 23, 8, 47, 48, 33, 50, 51, 36, 35, 52, 37, 54, 39, 56, 63, 32, 49, 34, 35, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2716(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 5, 22, 7, 24, 15, 16, 1, 2, 3, 20, 19, 4, 21, 6, 23, 8, 63, 32, 49, 50, 51, 36, 35, 52, 37, 54, 39, 56, 47, 48, 33, 34, 35, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2717(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 5, 22, 7, 24, 31, 16, 1, 2, 3, 20, 19, 4, 21, 6, 23, 8, 47, 32, 49, 50, 51, 36, 35, 52, 37, 54, 39, 56, 63, 48, 33, 34, 35, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2718(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 5, 22, 7, 24, 15, 0, 1, 2, 3, 20, 19, 4, 21, 6, 23, 8, 63, 48, 49, 50, 51, 36, 35, 52, 37, 54, 39, 56, 47, 32, 33, 34, 35, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2719(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 5, 22, 7, 24, 31, 0, 1, 2, 3, 20, 19, 4, 21, 6, 23, 8, 47, 48, 49, 50, 51, 36, 35, 52, 37, 54, 39, 56, 63, 32, 33, 34, 35, 52, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2720(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 5, 22, 7, 24, 15, 16, 17, 18, 19, 4, 19, 4, 21, 6, 23, 8, 63, 32, 33, 34, 35, 52, 35, 52, 37, 54, 39, 56, 47, 48, 49, 50, 51, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2721(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 5, 22, 7, 24, 31, 16, 17, 18, 19, 4, 19, 4, 21, 6, 23, 8, 47, 32, 33, 34, 35, 52, 35, 52, 37, 54, 39, 56, 63, 48, 49, 50, 51, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2722(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 5, 22, 7, 24, 15, 0, 17, 18, 19, 4, 19, 4, 21, 6, 23, 8, 63, 48, 33, 34, 35, 52, 35, 52, 37, 54, 39, 56, 47, 32, 49, 50, 51, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2723(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 5, 22, 7, 24, 31, 0, 17, 18, 19, 4, 19, 4, 21, 6, 23, 8, 47, 48, 33, 34, 35, 52, 35, 52, 37, 54, 39, 56, 63, 32, 49, 50, 51, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2724(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 5, 22, 7, 24, 15, 16, 1, 18, 19, 4, 19, 4, 21, 6, 23, 8, 63, 32, 49, 34, 35, 52, 35, 52, 37, 54, 39, 56, 47, 48, 33, 50, 51, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2725(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 5, 22, 7, 24, 31, 16, 1, 18, 19, 4, 19, 4, 21, 6, 23, 8, 47, 32, 49, 34, 35, 52, 35, 52, 37, 54, 39, 56, 63, 48, 33, 50, 51, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2726(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 5, 22, 7, 24, 15, 0, 1, 18, 19, 4, 19, 4, 21, 6, 23, 8, 63, 48, 49, 34, 35, 52, 35, 52, 37, 54, 39, 56, 47, 32, 33, 50, 51, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2727(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 5, 22, 7, 24, 31, 0, 1, 18, 19, 4, 19, 4, 21, 6, 23, 8, 47, 48, 49, 34, 35, 52, 35, 52, 37, 54, 39, 56, 63, 32, 33, 50, 51, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2728(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 5, 22, 7, 24, 15, 16, 17, 2, 19, 4, 19, 4, 21, 6, 23, 8, 63, 32, 33, 50, 35, 52, 35, 52, 37, 54, 39, 56, 47, 48, 49, 34, 51, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2729(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 5, 22, 7, 24, 31, 16, 17, 2, 19, 4, 19, 4, 21, 6, 23, 8, 47, 32, 33, 50, 35, 52, 35, 52, 37, 54, 39, 56, 63, 48, 49, 34, 51, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2730(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 5, 22, 7, 24, 15, 0, 17, 2, 19, 4, 19, 4, 21, 6, 23, 8, 63, 48, 33, 50, 35, 52, 35, 52, 37, 54, 39, 56, 47, 32, 49, 34, 51, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2731(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 5, 22, 7, 24, 31, 0, 17, 2, 19, 4, 19, 4, 21, 6, 23, 8, 47, 48, 33, 50, 35, 52, 35, 52, 37, 54, 39, 56, 63, 32, 49, 34, 51, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2732(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 5, 22, 7, 24, 15, 16, 1, 2, 19, 4, 19, 4, 21, 6, 23, 8, 63, 32, 49, 50, 35, 52, 35, 52, 37, 54, 39, 56, 47, 48, 33, 34, 51, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2733(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 5, 22, 7, 24, 31, 16, 1, 2, 19, 4, 19, 4, 21, 6, 23, 8, 47, 32, 49, 50, 35, 52, 35, 52, 37, 54, 39, 56, 63, 48, 33, 34, 51, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2734(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 5, 22, 7, 24, 15, 0, 1, 2, 19, 4, 19, 4, 21, 6, 23, 8, 63, 48, 49, 50, 35, 52, 35, 52, 37, 54, 39, 56, 47, 32, 33, 34, 51, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2735(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 5, 22, 7, 24, 31, 0, 1, 2, 19, 4, 19, 4, 21, 6, 23, 8, 47, 48, 49, 50, 35, 52, 35, 52, 37, 54, 39, 56, 63, 32, 33, 34, 51, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2736(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 5, 22, 7, 24, 15, 16, 17, 18, 3, 4, 19, 4, 21, 6, 23, 8, 63, 32, 33, 34, 51, 52, 35, 52, 37, 54, 39, 56, 47, 48, 49, 50, 35, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2737(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 5, 22, 7, 24, 31, 16, 17, 18, 3, 4, 19, 4, 21, 6, 23, 8, 47, 32, 33, 34, 51, 52, 35, 52, 37, 54, 39, 56, 63, 48, 49, 50, 35, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2738(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 5, 22, 7, 24, 15, 0, 17, 18, 3, 4, 19, 4, 21, 6, 23, 8, 63, 48, 33, 34, 51, 52, 35, 52, 37, 54, 39, 56, 47, 32, 49, 50, 35, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2739(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 5, 22, 7, 24, 31, 0, 17, 18, 3, 4, 19, 4, 21, 6, 23, 8, 47, 48, 33, 34, 51, 52, 35, 52, 37, 54, 39, 56, 63, 32, 49, 50, 35, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2740(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 5, 22, 7, 24, 15, 16, 1, 18, 3, 4, 19, 4, 21, 6, 23, 8, 63, 32, 49, 34, 51, 52, 35, 52, 37, 54, 39, 56, 47, 48, 33, 50, 35, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2741(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 5, 22, 7, 24, 31, 16, 1, 18, 3, 4, 19, 4, 21, 6, 23, 8, 47, 32, 49, 34, 51, 52, 35, 52, 37, 54, 39, 56, 63, 48, 33, 50, 35, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2742(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 5, 22, 7, 24, 15, 0, 1, 18, 3, 4, 19, 4, 21, 6, 23, 8, 63, 48, 49, 34, 51, 52, 35, 52, 37, 54, 39, 56, 47, 32, 33, 50, 35, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2743(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 5, 22, 7, 24, 31, 0, 1, 18, 3, 4, 19, 4, 21, 6, 23, 8, 47, 48, 49, 34, 51, 52, 35, 52, 37, 54, 39, 56, 63, 32, 33, 50, 35, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2744(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 5, 22, 7, 24, 15, 16, 17, 2, 3, 4, 19, 4, 21, 6, 23, 8, 63, 32, 33, 50, 51, 52, 35, 52, 37, 54, 39, 56, 47, 48, 49, 34, 35, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2745(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 5, 22, 7, 24, 31, 16, 17, 2, 3, 4, 19, 4, 21, 6, 23, 8, 47, 32, 33, 50, 51, 52, 35, 52, 37, 54, 39, 56, 63, 48, 49, 34, 35, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2746(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 5, 22, 7, 24, 15, 0, 17, 2, 3, 4, 19, 4, 21, 6, 23, 8, 63, 48, 33, 50, 51, 52, 35, 52, 37, 54, 39, 56, 47, 32, 49, 34, 35, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2747(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 5, 22, 7, 24, 31, 0, 17, 2, 3, 4, 19, 4, 21, 6, 23, 8, 47, 48, 33, 50, 51, 52, 35, 52, 37, 54, 39, 56, 63, 32, 49, 34, 35, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2748(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 5, 22, 7, 24, 15, 16, 1, 2, 3, 4, 19, 4, 21, 6, 23, 8, 63, 32, 49, 50, 51, 52, 35, 52, 37, 54, 39, 56, 47, 48, 33, 34, 35, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2749(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 5, 22, 7, 24, 31, 16, 1, 2, 3, 4, 19, 4, 21, 6, 23, 8, 47, 32, 49, 50, 51, 52, 35, 52, 37, 54, 39, 56, 63, 48, 33, 34, 35, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2750(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 5, 22, 7, 24, 15, 0, 1, 2, 3, 4, 19, 4, 21, 6, 23, 8, 63, 48, 49, 50, 51, 52, 35, 52, 37, 54, 39, 56, 47, 32, 33, 34, 35, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2751(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 5, 22, 7, 24, 31, 0, 1, 2, 3, 4, 19, 4, 21, 6, 23, 8, 47, 48, 49, 50, 51, 52, 35, 52, 37, 54, 39, 56, 63, 32, 33, 34, 35, 36, 51, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2752(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 5, 22, 7, 24, 15, 16, 17, 18, 19, 20, 3, 4, 21, 6, 23, 8, 63, 32, 33, 34, 35, 36, 51, 52, 37, 54, 39, 56, 47, 48, 49, 50, 51, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2753(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 5, 22, 7, 24, 31, 16, 17, 18, 19, 20, 3, 4, 21, 6, 23, 8, 47, 32, 33, 34, 35, 36, 51, 52, 37, 54, 39, 56, 63, 48, 49, 50, 51, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2754(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 5, 22, 7, 24, 15, 0, 17, 18, 19, 20, 3, 4, 21, 6, 23, 8, 63, 48, 33, 34, 35, 36, 51, 52, 37, 54, 39, 56, 47, 32, 49, 50, 51, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2755(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 5, 22, 7, 24, 31, 0, 17, 18, 19, 20, 3, 4, 21, 6, 23, 8, 47, 48, 33, 34, 35, 36, 51, 52, 37, 54, 39, 56, 63, 32, 49, 50, 51, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2756(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 5, 22, 7, 24, 15, 16, 1, 18, 19, 20, 3, 4, 21, 6, 23, 8, 63, 32, 49, 34, 35, 36, 51, 52, 37, 54, 39, 56, 47, 48, 33, 50, 51, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2757(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 5, 22, 7, 24, 31, 16, 1, 18, 19, 20, 3, 4, 21, 6, 23, 8, 47, 32, 49, 34, 35, 36, 51, 52, 37, 54, 39, 56, 63, 48, 33, 50, 51, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2758(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 5, 22, 7, 24, 15, 0, 1, 18, 19, 20, 3, 4, 21, 6, 23, 8, 63, 48, 49, 34, 35, 36, 51, 52, 37, 54, 39, 56, 47, 32, 33, 50, 51, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2759(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 5, 22, 7, 24, 31, 0, 1, 18, 19, 20, 3, 4, 21, 6, 23, 8, 47, 48, 49, 34, 35, 36, 51, 52, 37, 54, 39, 56, 63, 32, 33, 50, 51, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2760(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 5, 22, 7, 24, 15, 16, 17, 2, 19, 20, 3, 4, 21, 6, 23, 8, 63, 32, 33, 50, 35, 36, 51, 52, 37, 54, 39, 56, 47, 48, 49, 34, 51, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2761(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 5, 22, 7, 24, 31, 16, 17, 2, 19, 20, 3, 4, 21, 6, 23, 8, 47, 32, 33, 50, 35, 36, 51, 52, 37, 54, 39, 56, 63, 48, 49, 34, 51, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2762(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 5, 22, 7, 24, 15, 0, 17, 2, 19, 20, 3, 4, 21, 6, 23, 8, 63, 48, 33, 50, 35, 36, 51, 52, 37, 54, 39, 56, 47, 32, 49, 34, 51, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2763(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 5, 22, 7, 24, 31, 0, 17, 2, 19, 20, 3, 4, 21, 6, 23, 8, 47, 48, 33, 50, 35, 36, 51, 52, 37, 54, 39, 56, 63, 32, 49, 34, 51, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2764(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 5, 22, 7, 24, 15, 16, 1, 2, 19, 20, 3, 4, 21, 6, 23, 8, 63, 32, 49, 50, 35, 36, 51, 52, 37, 54, 39, 56, 47, 48, 33, 34, 51, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2765(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 5, 22, 7, 24, 31, 16, 1, 2, 19, 20, 3, 4, 21, 6, 23, 8, 47, 32, 49, 50, 35, 36, 51, 52, 37, 54, 39, 56, 63, 48, 33, 34, 51, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2766(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 5, 22, 7, 24, 15, 0, 1, 2, 19, 20, 3, 4, 21, 6, 23, 8, 63, 48, 49, 50, 35, 36, 51, 52, 37, 54, 39, 56, 47, 32, 33, 34, 51, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2767(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 5, 22, 7, 24, 31, 0, 1, 2, 19, 20, 3, 4, 21, 6, 23, 8, 47, 48, 49, 50, 35, 36, 51, 52, 37, 54, 39, 56, 63, 32, 33, 34, 51, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2768(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 5, 22, 7, 24, 15, 16, 17, 18, 3, 20, 3, 4, 21, 6, 23, 8, 63, 32, 33, 34, 51, 36, 51, 52, 37, 54, 39, 56, 47, 48, 49, 50, 35, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2769(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 5, 22, 7, 24, 31, 16, 17, 18, 3, 20, 3, 4, 21, 6, 23, 8, 47, 32, 33, 34, 51, 36, 51, 52, 37, 54, 39, 56, 63, 48, 49, 50, 35, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2770(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 5, 22, 7, 24, 15, 0, 17, 18, 3, 20, 3, 4, 21, 6, 23, 8, 63, 48, 33, 34, 51, 36, 51, 52, 37, 54, 39, 56, 47, 32, 49, 50, 35, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2771(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 5, 22, 7, 24, 31, 0, 17, 18, 3, 20, 3, 4, 21, 6, 23, 8, 47, 48, 33, 34, 51, 36, 51, 52, 37, 54, 39, 56, 63, 32, 49, 50, 35, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2772(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 5, 22, 7, 24, 15, 16, 1, 18, 3, 20, 3, 4, 21, 6, 23, 8, 63, 32, 49, 34, 51, 36, 51, 52, 37, 54, 39, 56, 47, 48, 33, 50, 35, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2773(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 5, 22, 7, 24, 31, 16, 1, 18, 3, 20, 3, 4, 21, 6, 23, 8, 47, 32, 49, 34, 51, 36, 51, 52, 37, 54, 39, 56, 63, 48, 33, 50, 35, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2774(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 5, 22, 7, 24, 15, 0, 1, 18, 3, 20, 3, 4, 21, 6, 23, 8, 63, 48, 49, 34, 51, 36, 51, 52, 37, 54, 39, 56, 47, 32, 33, 50, 35, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2775(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 5, 22, 7, 24, 31, 0, 1, 18, 3, 20, 3, 4, 21, 6, 23, 8, 47, 48, 49, 34, 51, 36, 51, 52, 37, 54, 39, 56, 63, 32, 33, 50, 35, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2776(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 5, 22, 7, 24, 15, 16, 17, 2, 3, 20, 3, 4, 21, 6, 23, 8, 63, 32, 33, 50, 51, 36, 51, 52, 37, 54, 39, 56, 47, 48, 49, 34, 35, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2777(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 5, 22, 7, 24, 31, 16, 17, 2, 3, 20, 3, 4, 21, 6, 23, 8, 47, 32, 33, 50, 51, 36, 51, 52, 37, 54, 39, 56, 63, 48, 49, 34, 35, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2778(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 5, 22, 7, 24, 15, 0, 17, 2, 3, 20, 3, 4, 21, 6, 23, 8, 63, 48, 33, 50, 51, 36, 51, 52, 37, 54, 39, 56, 47, 32, 49, 34, 35, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2779(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 5, 22, 7, 24, 31, 0, 17, 2, 3, 20, 3, 4, 21, 6, 23, 8, 47, 48, 33, 50, 51, 36, 51, 52, 37, 54, 39, 56, 63, 32, 49, 34, 35, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2780(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 5, 22, 7, 24, 15, 16, 1, 2, 3, 20, 3, 4, 21, 6, 23, 8, 63, 32, 49, 50, 51, 36, 51, 52, 37, 54, 39, 56, 47, 48, 33, 34, 35, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2781(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 5, 22, 7, 24, 31, 16, 1, 2, 3, 20, 3, 4, 21, 6, 23, 8, 47, 32, 49, 50, 51, 36, 51, 52, 37, 54, 39, 56, 63, 48, 33, 34, 35, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2782(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 5, 22, 7, 24, 15, 0, 1, 2, 3, 20, 3, 4, 21, 6, 23, 8, 63, 48, 49, 50, 51, 36, 51, 52, 37, 54, 39, 56, 47, 32, 33, 34, 35, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2783(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 5, 22, 7, 24, 31, 0, 1, 2, 3, 20, 3, 4, 21, 6, 23, 8, 47, 48, 49, 50, 51, 36, 51, 52, 37, 54, 39, 56, 63, 32, 33, 34, 35, 52, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2784(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 5, 22, 7, 24, 15, 16, 17, 18, 19, 4, 3, 4, 21, 6, 23, 8, 63, 32, 33, 34, 35, 52, 51, 52, 37, 54, 39, 56, 47, 48, 49, 50, 51, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2785(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 5, 22, 7, 24, 31, 16, 17, 18, 19, 4, 3, 4, 21, 6, 23, 8, 47, 32, 33, 34, 35, 52, 51, 52, 37, 54, 39, 56, 63, 48, 49, 50, 51, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2786(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 5, 22, 7, 24, 15, 0, 17, 18, 19, 4, 3, 4, 21, 6, 23, 8, 63, 48, 33, 34, 35, 52, 51, 52, 37, 54, 39, 56, 47, 32, 49, 50, 51, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2787(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 5, 22, 7, 24, 31, 0, 17, 18, 19, 4, 3, 4, 21, 6, 23, 8, 47, 48, 33, 34, 35, 52, 51, 52, 37, 54, 39, 56, 63, 32, 49, 50, 51, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2788(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 5, 22, 7, 24, 15, 16, 1, 18, 19, 4, 3, 4, 21, 6, 23, 8, 63, 32, 49, 34, 35, 52, 51, 52, 37, 54, 39, 56, 47, 48, 33, 50, 51, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2789(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 5, 22, 7, 24, 31, 16, 1, 18, 19, 4, 3, 4, 21, 6, 23, 8, 47, 32, 49, 34, 35, 52, 51, 52, 37, 54, 39, 56, 63, 48, 33, 50, 51, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2790(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 5, 22, 7, 24, 15, 0, 1, 18, 19, 4, 3, 4, 21, 6, 23, 8, 63, 48, 49, 34, 35, 52, 51, 52, 37, 54, 39, 56, 47, 32, 33, 50, 51, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2791(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 5, 22, 7, 24, 31, 0, 1, 18, 19, 4, 3, 4, 21, 6, 23, 8, 47, 48, 49, 34, 35, 52, 51, 52, 37, 54, 39, 56, 63, 32, 33, 50, 51, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2792(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 5, 22, 7, 24, 15, 16, 17, 2, 19, 4, 3, 4, 21, 6, 23, 8, 63, 32, 33, 50, 35, 52, 51, 52, 37, 54, 39, 56, 47, 48, 49, 34, 51, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2793(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 5, 22, 7, 24, 31, 16, 17, 2, 19, 4, 3, 4, 21, 6, 23, 8, 47, 32, 33, 50, 35, 52, 51, 52, 37, 54, 39, 56, 63, 48, 49, 34, 51, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2794(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 5, 22, 7, 24, 15, 0, 17, 2, 19, 4, 3, 4, 21, 6, 23, 8, 63, 48, 33, 50, 35, 52, 51, 52, 37, 54, 39, 56, 47, 32, 49, 34, 51, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2795(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 5, 22, 7, 24, 31, 0, 17, 2, 19, 4, 3, 4, 21, 6, 23, 8, 47, 48, 33, 50, 35, 52, 51, 52, 37, 54, 39, 56, 63, 32, 49, 34, 51, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2796(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 5, 22, 7, 24, 15, 16, 1, 2, 19, 4, 3, 4, 21, 6, 23, 8, 63, 32, 49, 50, 35, 52, 51, 52, 37, 54, 39, 56, 47, 48, 33, 34, 51, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2797(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 5, 22, 7, 24, 31, 16, 1, 2, 19, 4, 3, 4, 21, 6, 23, 8, 47, 32, 49, 50, 35, 52, 51, 52, 37, 54, 39, 56, 63, 48, 33, 34, 51, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2798(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 5, 22, 7, 24, 15, 0, 1, 2, 19, 4, 3, 4, 21, 6, 23, 8, 63, 48, 49, 50, 35, 52, 51, 52, 37, 54, 39, 56, 47, 32, 33, 34, 51, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2799(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 5, 22, 7, 24, 31, 0, 1, 2, 19, 4, 3, 4, 21, 6, 23, 8, 47, 48, 49, 50, 35, 52, 51, 52, 37, 54, 39, 56, 63, 32, 33, 34, 51, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2800(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 5, 22, 7, 24, 15, 16, 17, 18, 3, 4, 3, 4, 21, 6, 23, 8, 63, 32, 33, 34, 51, 52, 51, 52, 37, 54, 39, 56, 47, 48, 49, 50, 35, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2801(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 5, 22, 7, 24, 31, 16, 17, 18, 3, 4, 3, 4, 21, 6, 23, 8, 47, 32, 33, 34, 51, 52, 51, 52, 37, 54, 39, 56, 63, 48, 49, 50, 35, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2802(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 5, 22, 7, 24, 15, 0, 17, 18, 3, 4, 3, 4, 21, 6, 23, 8, 63, 48, 33, 34, 51, 52, 51, 52, 37, 54, 39, 56, 47, 32, 49, 50, 35, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2803(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 5, 22, 7, 24, 31, 0, 17, 18, 3, 4, 3, 4, 21, 6, 23, 8, 47, 48, 33, 34, 51, 52, 51, 52, 37, 54, 39, 56, 63, 32, 49, 50, 35, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2804(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 5, 22, 7, 24, 15, 16, 1, 18, 3, 4, 3, 4, 21, 6, 23, 8, 63, 32, 49, 34, 51, 52, 51, 52, 37, 54, 39, 56, 47, 48, 33, 50, 35, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2805(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 5, 22, 7, 24, 31, 16, 1, 18, 3, 4, 3, 4, 21, 6, 23, 8, 47, 32, 49, 34, 51, 52, 51, 52, 37, 54, 39, 56, 63, 48, 33, 50, 35, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2806(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 5, 22, 7, 24, 15, 0, 1, 18, 3, 4, 3, 4, 21, 6, 23, 8, 63, 48, 49, 34, 51, 52, 51, 52, 37, 54, 39, 56, 47, 32, 33, 50, 35, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2807(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 5, 22, 7, 24, 31, 0, 1, 18, 3, 4, 3, 4, 21, 6, 23, 8, 47, 48, 49, 34, 51, 52, 51, 52, 37, 54, 39, 56, 63, 32, 33, 50, 35, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2808(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 5, 22, 7, 24, 15, 16, 17, 2, 3, 4, 3, 4, 21, 6, 23, 8, 63, 32, 33, 50, 51, 52, 51, 52, 37, 54, 39, 56, 47, 48, 49, 34, 35, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2809(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 5, 22, 7, 24, 31, 16, 17, 2, 3, 4, 3, 4, 21, 6, 23, 8, 47, 32, 33, 50, 51, 52, 51, 52, 37, 54, 39, 56, 63, 48, 49, 34, 35, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2810(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 5, 22, 7, 24, 15, 0, 17, 2, 3, 4, 3, 4, 21, 6, 23, 8, 63, 48, 33, 50, 51, 52, 51, 52, 37, 54, 39, 56, 47, 32, 49, 34, 35, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2811(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 5, 22, 7, 24, 31, 0, 17, 2, 3, 4, 3, 4, 21, 6, 23, 8, 47, 48, 33, 50, 51, 52, 51, 52, 37, 54, 39, 56, 63, 32, 49, 34, 35, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2812(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 5, 22, 7, 24, 15, 16, 1, 2, 3, 4, 3, 4, 21, 6, 23, 8, 63, 32, 49, 50, 51, 52, 51, 52, 37, 54, 39, 56, 47, 48, 33, 34, 35, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2813(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 5, 22, 7, 24, 31, 16, 1, 2, 3, 4, 3, 4, 21, 6, 23, 8, 47, 32, 49, 50, 51, 52, 51, 52, 37, 54, 39, 56, 63, 48, 33, 34, 35, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2814(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 5, 22, 7, 24, 15, 0, 1, 2, 3, 4, 3, 4, 21, 6, 23, 8, 63, 48, 49, 50, 51, 52, 51, 52, 37, 54, 39, 56, 47, 32, 33, 34, 35, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2815(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 5, 22, 7, 24, 31, 0, 1, 2, 3, 4, 3, 4, 21, 6, 23, 8, 47, 48, 49, 50, 51, 52, 51, 52, 37, 54, 39, 56, 63, 32, 33, 34, 35, 36, 35, 36, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}


