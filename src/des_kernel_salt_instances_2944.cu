#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

#ifdef DESGPU_COMPILE_ALL_SALTS

void des_25_encrypt_salt2944(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 21, 22, 7, 24, 15, 16, 17, 18, 19, 20, 19, 4, 5, 6, 23, 8, 63, 32, 33, 34, 35, 36, 35, 52, 53, 54, 39, 56, 47, 48, 49, 50, 51, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2945(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 21, 22, 7, 24, 31, 16, 17, 18, 19, 20, 19, 4, 5, 6, 23, 8, 47, 32, 33, 34, 35, 36, 35, 52, 53, 54, 39, 56, 63, 48, 49, 50, 51, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2946(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 21, 22, 7, 24, 15, 0, 17, 18, 19, 20, 19, 4, 5, 6, 23, 8, 63, 48, 33, 34, 35, 36, 35, 52, 53, 54, 39, 56, 47, 32, 49, 50, 51, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2947(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 21, 22, 7, 24, 31, 0, 17, 18, 19, 20, 19, 4, 5, 6, 23, 8, 47, 48, 33, 34, 35, 36, 35, 52, 53, 54, 39, 56, 63, 32, 49, 50, 51, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2948(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 21, 22, 7, 24, 15, 16, 1, 18, 19, 20, 19, 4, 5, 6, 23, 8, 63, 32, 49, 34, 35, 36, 35, 52, 53, 54, 39, 56, 47, 48, 33, 50, 51, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2949(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 21, 22, 7, 24, 31, 16, 1, 18, 19, 20, 19, 4, 5, 6, 23, 8, 47, 32, 49, 34, 35, 36, 35, 52, 53, 54, 39, 56, 63, 48, 33, 50, 51, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2950(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 21, 22, 7, 24, 15, 0, 1, 18, 19, 20, 19, 4, 5, 6, 23, 8, 63, 48, 49, 34, 35, 36, 35, 52, 53, 54, 39, 56, 47, 32, 33, 50, 51, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2951(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 21, 22, 7, 24, 31, 0, 1, 18, 19, 20, 19, 4, 5, 6, 23, 8, 47, 48, 49, 34, 35, 36, 35, 52, 53, 54, 39, 56, 63, 32, 33, 50, 51, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2952(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 21, 22, 7, 24, 15, 16, 17, 2, 19, 20, 19, 4, 5, 6, 23, 8, 63, 32, 33, 50, 35, 36, 35, 52, 53, 54, 39, 56, 47, 48, 49, 34, 51, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2953(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 21, 22, 7, 24, 31, 16, 17, 2, 19, 20, 19, 4, 5, 6, 23, 8, 47, 32, 33, 50, 35, 36, 35, 52, 53, 54, 39, 56, 63, 48, 49, 34, 51, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2954(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 21, 22, 7, 24, 15, 0, 17, 2, 19, 20, 19, 4, 5, 6, 23, 8, 63, 48, 33, 50, 35, 36, 35, 52, 53, 54, 39, 56, 47, 32, 49, 34, 51, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2955(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 21, 22, 7, 24, 31, 0, 17, 2, 19, 20, 19, 4, 5, 6, 23, 8, 47, 48, 33, 50, 35, 36, 35, 52, 53, 54, 39, 56, 63, 32, 49, 34, 51, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2956(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 21, 22, 7, 24, 15, 16, 1, 2, 19, 20, 19, 4, 5, 6, 23, 8, 63, 32, 49, 50, 35, 36, 35, 52, 53, 54, 39, 56, 47, 48, 33, 34, 51, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2957(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 21, 22, 7, 24, 31, 16, 1, 2, 19, 20, 19, 4, 5, 6, 23, 8, 47, 32, 49, 50, 35, 36, 35, 52, 53, 54, 39, 56, 63, 48, 33, 34, 51, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2958(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 21, 22, 7, 24, 15, 0, 1, 2, 19, 20, 19, 4, 5, 6, 23, 8, 63, 48, 49, 50, 35, 36, 35, 52, 53, 54, 39, 56, 47, 32, 33, 34, 51, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2959(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 21, 22, 7, 24, 31, 0, 1, 2, 19, 20, 19, 4, 5, 6, 23, 8, 47, 48, 49, 50, 35, 36, 35, 52, 53, 54, 39, 56, 63, 32, 33, 34, 51, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2960(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 21, 22, 7, 24, 15, 16, 17, 18, 3, 20, 19, 4, 5, 6, 23, 8, 63, 32, 33, 34, 51, 36, 35, 52, 53, 54, 39, 56, 47, 48, 49, 50, 35, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2961(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 21, 22, 7, 24, 31, 16, 17, 18, 3, 20, 19, 4, 5, 6, 23, 8, 47, 32, 33, 34, 51, 36, 35, 52, 53, 54, 39, 56, 63, 48, 49, 50, 35, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2962(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 21, 22, 7, 24, 15, 0, 17, 18, 3, 20, 19, 4, 5, 6, 23, 8, 63, 48, 33, 34, 51, 36, 35, 52, 53, 54, 39, 56, 47, 32, 49, 50, 35, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2963(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 21, 22, 7, 24, 31, 0, 17, 18, 3, 20, 19, 4, 5, 6, 23, 8, 47, 48, 33, 34, 51, 36, 35, 52, 53, 54, 39, 56, 63, 32, 49, 50, 35, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2964(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 21, 22, 7, 24, 15, 16, 1, 18, 3, 20, 19, 4, 5, 6, 23, 8, 63, 32, 49, 34, 51, 36, 35, 52, 53, 54, 39, 56, 47, 48, 33, 50, 35, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2965(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 21, 22, 7, 24, 31, 16, 1, 18, 3, 20, 19, 4, 5, 6, 23, 8, 47, 32, 49, 34, 51, 36, 35, 52, 53, 54, 39, 56, 63, 48, 33, 50, 35, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2966(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 21, 22, 7, 24, 15, 0, 1, 18, 3, 20, 19, 4, 5, 6, 23, 8, 63, 48, 49, 34, 51, 36, 35, 52, 53, 54, 39, 56, 47, 32, 33, 50, 35, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2967(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 21, 22, 7, 24, 31, 0, 1, 18, 3, 20, 19, 4, 5, 6, 23, 8, 47, 48, 49, 34, 51, 36, 35, 52, 53, 54, 39, 56, 63, 32, 33, 50, 35, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2968(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 21, 22, 7, 24, 15, 16, 17, 2, 3, 20, 19, 4, 5, 6, 23, 8, 63, 32, 33, 50, 51, 36, 35, 52, 53, 54, 39, 56, 47, 48, 49, 34, 35, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2969(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 21, 22, 7, 24, 31, 16, 17, 2, 3, 20, 19, 4, 5, 6, 23, 8, 47, 32, 33, 50, 51, 36, 35, 52, 53, 54, 39, 56, 63, 48, 49, 34, 35, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2970(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 21, 22, 7, 24, 15, 0, 17, 2, 3, 20, 19, 4, 5, 6, 23, 8, 63, 48, 33, 50, 51, 36, 35, 52, 53, 54, 39, 56, 47, 32, 49, 34, 35, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2971(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 21, 22, 7, 24, 31, 0, 17, 2, 3, 20, 19, 4, 5, 6, 23, 8, 47, 48, 33, 50, 51, 36, 35, 52, 53, 54, 39, 56, 63, 32, 49, 34, 35, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2972(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 21, 22, 7, 24, 15, 16, 1, 2, 3, 20, 19, 4, 5, 6, 23, 8, 63, 32, 49, 50, 51, 36, 35, 52, 53, 54, 39, 56, 47, 48, 33, 34, 35, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2973(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 21, 22, 7, 24, 31, 16, 1, 2, 3, 20, 19, 4, 5, 6, 23, 8, 47, 32, 49, 50, 51, 36, 35, 52, 53, 54, 39, 56, 63, 48, 33, 34, 35, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2974(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 21, 22, 7, 24, 15, 0, 1, 2, 3, 20, 19, 4, 5, 6, 23, 8, 63, 48, 49, 50, 51, 36, 35, 52, 53, 54, 39, 56, 47, 32, 33, 34, 35, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2975(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 21, 22, 7, 24, 31, 0, 1, 2, 3, 20, 19, 4, 5, 6, 23, 8, 47, 48, 49, 50, 51, 36, 35, 52, 53, 54, 39, 56, 63, 32, 33, 34, 35, 52, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2976(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 21, 22, 7, 24, 15, 16, 17, 18, 19, 4, 19, 4, 5, 6, 23, 8, 63, 32, 33, 34, 35, 52, 35, 52, 53, 54, 39, 56, 47, 48, 49, 50, 51, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2977(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 21, 22, 7, 24, 31, 16, 17, 18, 19, 4, 19, 4, 5, 6, 23, 8, 47, 32, 33, 34, 35, 52, 35, 52, 53, 54, 39, 56, 63, 48, 49, 50, 51, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2978(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 21, 22, 7, 24, 15, 0, 17, 18, 19, 4, 19, 4, 5, 6, 23, 8, 63, 48, 33, 34, 35, 52, 35, 52, 53, 54, 39, 56, 47, 32, 49, 50, 51, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2979(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 21, 22, 7, 24, 31, 0, 17, 18, 19, 4, 19, 4, 5, 6, 23, 8, 47, 48, 33, 34, 35, 52, 35, 52, 53, 54, 39, 56, 63, 32, 49, 50, 51, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2980(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 21, 22, 7, 24, 15, 16, 1, 18, 19, 4, 19, 4, 5, 6, 23, 8, 63, 32, 49, 34, 35, 52, 35, 52, 53, 54, 39, 56, 47, 48, 33, 50, 51, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2981(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 21, 22, 7, 24, 31, 16, 1, 18, 19, 4, 19, 4, 5, 6, 23, 8, 47, 32, 49, 34, 35, 52, 35, 52, 53, 54, 39, 56, 63, 48, 33, 50, 51, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2982(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 21, 22, 7, 24, 15, 0, 1, 18, 19, 4, 19, 4, 5, 6, 23, 8, 63, 48, 49, 34, 35, 52, 35, 52, 53, 54, 39, 56, 47, 32, 33, 50, 51, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2983(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 21, 22, 7, 24, 31, 0, 1, 18, 19, 4, 19, 4, 5, 6, 23, 8, 47, 48, 49, 34, 35, 52, 35, 52, 53, 54, 39, 56, 63, 32, 33, 50, 51, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2984(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 21, 22, 7, 24, 15, 16, 17, 2, 19, 4, 19, 4, 5, 6, 23, 8, 63, 32, 33, 50, 35, 52, 35, 52, 53, 54, 39, 56, 47, 48, 49, 34, 51, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2985(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 21, 22, 7, 24, 31, 16, 17, 2, 19, 4, 19, 4, 5, 6, 23, 8, 47, 32, 33, 50, 35, 52, 35, 52, 53, 54, 39, 56, 63, 48, 49, 34, 51, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2986(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 21, 22, 7, 24, 15, 0, 17, 2, 19, 4, 19, 4, 5, 6, 23, 8, 63, 48, 33, 50, 35, 52, 35, 52, 53, 54, 39, 56, 47, 32, 49, 34, 51, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2987(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 21, 22, 7, 24, 31, 0, 17, 2, 19, 4, 19, 4, 5, 6, 23, 8, 47, 48, 33, 50, 35, 52, 35, 52, 53, 54, 39, 56, 63, 32, 49, 34, 51, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2988(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 21, 22, 7, 24, 15, 16, 1, 2, 19, 4, 19, 4, 5, 6, 23, 8, 63, 32, 49, 50, 35, 52, 35, 52, 53, 54, 39, 56, 47, 48, 33, 34, 51, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2989(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 21, 22, 7, 24, 31, 16, 1, 2, 19, 4, 19, 4, 5, 6, 23, 8, 47, 32, 49, 50, 35, 52, 35, 52, 53, 54, 39, 56, 63, 48, 33, 34, 51, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2990(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 21, 22, 7, 24, 15, 0, 1, 2, 19, 4, 19, 4, 5, 6, 23, 8, 63, 48, 49, 50, 35, 52, 35, 52, 53, 54, 39, 56, 47, 32, 33, 34, 51, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2991(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 21, 22, 7, 24, 31, 0, 1, 2, 19, 4, 19, 4, 5, 6, 23, 8, 47, 48, 49, 50, 35, 52, 35, 52, 53, 54, 39, 56, 63, 32, 33, 34, 51, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2992(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 21, 22, 7, 24, 15, 16, 17, 18, 3, 4, 19, 4, 5, 6, 23, 8, 63, 32, 33, 34, 51, 52, 35, 52, 53, 54, 39, 56, 47, 48, 49, 50, 35, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2993(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 21, 22, 7, 24, 31, 16, 17, 18, 3, 4, 19, 4, 5, 6, 23, 8, 47, 32, 33, 34, 51, 52, 35, 52, 53, 54, 39, 56, 63, 48, 49, 50, 35, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2994(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 21, 22, 7, 24, 15, 0, 17, 18, 3, 4, 19, 4, 5, 6, 23, 8, 63, 48, 33, 34, 51, 52, 35, 52, 53, 54, 39, 56, 47, 32, 49, 50, 35, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2995(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 21, 22, 7, 24, 31, 0, 17, 18, 3, 4, 19, 4, 5, 6, 23, 8, 47, 48, 33, 34, 51, 52, 35, 52, 53, 54, 39, 56, 63, 32, 49, 50, 35, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2996(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 21, 22, 7, 24, 15, 16, 1, 18, 3, 4, 19, 4, 5, 6, 23, 8, 63, 32, 49, 34, 51, 52, 35, 52, 53, 54, 39, 56, 47, 48, 33, 50, 35, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2997(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 21, 22, 7, 24, 31, 16, 1, 18, 3, 4, 19, 4, 5, 6, 23, 8, 47, 32, 49, 34, 51, 52, 35, 52, 53, 54, 39, 56, 63, 48, 33, 50, 35, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2998(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 21, 22, 7, 24, 15, 0, 1, 18, 3, 4, 19, 4, 5, 6, 23, 8, 63, 48, 49, 34, 51, 52, 35, 52, 53, 54, 39, 56, 47, 32, 33, 50, 35, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2999(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 21, 22, 7, 24, 31, 0, 1, 18, 3, 4, 19, 4, 5, 6, 23, 8, 47, 48, 49, 34, 51, 52, 35, 52, 53, 54, 39, 56, 63, 32, 33, 50, 35, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3000(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 21, 22, 7, 24, 15, 16, 17, 2, 3, 4, 19, 4, 5, 6, 23, 8, 63, 32, 33, 50, 51, 52, 35, 52, 53, 54, 39, 56, 47, 48, 49, 34, 35, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3001(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 21, 22, 7, 24, 31, 16, 17, 2, 3, 4, 19, 4, 5, 6, 23, 8, 47, 32, 33, 50, 51, 52, 35, 52, 53, 54, 39, 56, 63, 48, 49, 34, 35, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3002(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 21, 22, 7, 24, 15, 0, 17, 2, 3, 4, 19, 4, 5, 6, 23, 8, 63, 48, 33, 50, 51, 52, 35, 52, 53, 54, 39, 56, 47, 32, 49, 34, 35, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3003(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 21, 22, 7, 24, 31, 0, 17, 2, 3, 4, 19, 4, 5, 6, 23, 8, 47, 48, 33, 50, 51, 52, 35, 52, 53, 54, 39, 56, 63, 32, 49, 34, 35, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3004(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 21, 22, 7, 24, 15, 16, 1, 2, 3, 4, 19, 4, 5, 6, 23, 8, 63, 32, 49, 50, 51, 52, 35, 52, 53, 54, 39, 56, 47, 48, 33, 34, 35, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3005(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 21, 22, 7, 24, 31, 16, 1, 2, 3, 4, 19, 4, 5, 6, 23, 8, 47, 32, 49, 50, 51, 52, 35, 52, 53, 54, 39, 56, 63, 48, 33, 34, 35, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3006(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 21, 22, 7, 24, 15, 0, 1, 2, 3, 4, 19, 4, 5, 6, 23, 8, 63, 48, 49, 50, 51, 52, 35, 52, 53, 54, 39, 56, 47, 32, 33, 34, 35, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3007(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 21, 22, 7, 24, 31, 0, 1, 2, 3, 4, 19, 4, 5, 6, 23, 8, 47, 48, 49, 50, 51, 52, 35, 52, 53, 54, 39, 56, 63, 32, 33, 34, 35, 36, 51, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3008(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 21, 22, 7, 24, 15, 16, 17, 18, 19, 20, 3, 4, 5, 6, 23, 8, 63, 32, 33, 34, 35, 36, 51, 52, 53, 54, 39, 56, 47, 48, 49, 50, 51, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3009(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 21, 22, 7, 24, 31, 16, 17, 18, 19, 20, 3, 4, 5, 6, 23, 8, 47, 32, 33, 34, 35, 36, 51, 52, 53, 54, 39, 56, 63, 48, 49, 50, 51, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3010(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 21, 22, 7, 24, 15, 0, 17, 18, 19, 20, 3, 4, 5, 6, 23, 8, 63, 48, 33, 34, 35, 36, 51, 52, 53, 54, 39, 56, 47, 32, 49, 50, 51, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3011(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 21, 22, 7, 24, 31, 0, 17, 18, 19, 20, 3, 4, 5, 6, 23, 8, 47, 48, 33, 34, 35, 36, 51, 52, 53, 54, 39, 56, 63, 32, 49, 50, 51, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3012(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 21, 22, 7, 24, 15, 16, 1, 18, 19, 20, 3, 4, 5, 6, 23, 8, 63, 32, 49, 34, 35, 36, 51, 52, 53, 54, 39, 56, 47, 48, 33, 50, 51, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3013(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 21, 22, 7, 24, 31, 16, 1, 18, 19, 20, 3, 4, 5, 6, 23, 8, 47, 32, 49, 34, 35, 36, 51, 52, 53, 54, 39, 56, 63, 48, 33, 50, 51, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3014(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 21, 22, 7, 24, 15, 0, 1, 18, 19, 20, 3, 4, 5, 6, 23, 8, 63, 48, 49, 34, 35, 36, 51, 52, 53, 54, 39, 56, 47, 32, 33, 50, 51, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3015(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 21, 22, 7, 24, 31, 0, 1, 18, 19, 20, 3, 4, 5, 6, 23, 8, 47, 48, 49, 34, 35, 36, 51, 52, 53, 54, 39, 56, 63, 32, 33, 50, 51, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3016(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 21, 22, 7, 24, 15, 16, 17, 2, 19, 20, 3, 4, 5, 6, 23, 8, 63, 32, 33, 50, 35, 36, 51, 52, 53, 54, 39, 56, 47, 48, 49, 34, 51, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3017(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 21, 22, 7, 24, 31, 16, 17, 2, 19, 20, 3, 4, 5, 6, 23, 8, 47, 32, 33, 50, 35, 36, 51, 52, 53, 54, 39, 56, 63, 48, 49, 34, 51, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3018(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 21, 22, 7, 24, 15, 0, 17, 2, 19, 20, 3, 4, 5, 6, 23, 8, 63, 48, 33, 50, 35, 36, 51, 52, 53, 54, 39, 56, 47, 32, 49, 34, 51, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3019(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 21, 22, 7, 24, 31, 0, 17, 2, 19, 20, 3, 4, 5, 6, 23, 8, 47, 48, 33, 50, 35, 36, 51, 52, 53, 54, 39, 56, 63, 32, 49, 34, 51, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3020(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 21, 22, 7, 24, 15, 16, 1, 2, 19, 20, 3, 4, 5, 6, 23, 8, 63, 32, 49, 50, 35, 36, 51, 52, 53, 54, 39, 56, 47, 48, 33, 34, 51, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3021(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 21, 22, 7, 24, 31, 16, 1, 2, 19, 20, 3, 4, 5, 6, 23, 8, 47, 32, 49, 50, 35, 36, 51, 52, 53, 54, 39, 56, 63, 48, 33, 34, 51, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3022(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 21, 22, 7, 24, 15, 0, 1, 2, 19, 20, 3, 4, 5, 6, 23, 8, 63, 48, 49, 50, 35, 36, 51, 52, 53, 54, 39, 56, 47, 32, 33, 34, 51, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3023(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 21, 22, 7, 24, 31, 0, 1, 2, 19, 20, 3, 4, 5, 6, 23, 8, 47, 48, 49, 50, 35, 36, 51, 52, 53, 54, 39, 56, 63, 32, 33, 34, 51, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3024(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 21, 22, 7, 24, 15, 16, 17, 18, 3, 20, 3, 4, 5, 6, 23, 8, 63, 32, 33, 34, 51, 36, 51, 52, 53, 54, 39, 56, 47, 48, 49, 50, 35, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3025(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 21, 22, 7, 24, 31, 16, 17, 18, 3, 20, 3, 4, 5, 6, 23, 8, 47, 32, 33, 34, 51, 36, 51, 52, 53, 54, 39, 56, 63, 48, 49, 50, 35, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3026(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 21, 22, 7, 24, 15, 0, 17, 18, 3, 20, 3, 4, 5, 6, 23, 8, 63, 48, 33, 34, 51, 36, 51, 52, 53, 54, 39, 56, 47, 32, 49, 50, 35, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3027(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 21, 22, 7, 24, 31, 0, 17, 18, 3, 20, 3, 4, 5, 6, 23, 8, 47, 48, 33, 34, 51, 36, 51, 52, 53, 54, 39, 56, 63, 32, 49, 50, 35, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3028(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 21, 22, 7, 24, 15, 16, 1, 18, 3, 20, 3, 4, 5, 6, 23, 8, 63, 32, 49, 34, 51, 36, 51, 52, 53, 54, 39, 56, 47, 48, 33, 50, 35, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3029(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 21, 22, 7, 24, 31, 16, 1, 18, 3, 20, 3, 4, 5, 6, 23, 8, 47, 32, 49, 34, 51, 36, 51, 52, 53, 54, 39, 56, 63, 48, 33, 50, 35, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3030(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 21, 22, 7, 24, 15, 0, 1, 18, 3, 20, 3, 4, 5, 6, 23, 8, 63, 48, 49, 34, 51, 36, 51, 52, 53, 54, 39, 56, 47, 32, 33, 50, 35, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3031(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 21, 22, 7, 24, 31, 0, 1, 18, 3, 20, 3, 4, 5, 6, 23, 8, 47, 48, 49, 34, 51, 36, 51, 52, 53, 54, 39, 56, 63, 32, 33, 50, 35, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3032(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 21, 22, 7, 24, 15, 16, 17, 2, 3, 20, 3, 4, 5, 6, 23, 8, 63, 32, 33, 50, 51, 36, 51, 52, 53, 54, 39, 56, 47, 48, 49, 34, 35, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3033(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 21, 22, 7, 24, 31, 16, 17, 2, 3, 20, 3, 4, 5, 6, 23, 8, 47, 32, 33, 50, 51, 36, 51, 52, 53, 54, 39, 56, 63, 48, 49, 34, 35, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3034(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 21, 22, 7, 24, 15, 0, 17, 2, 3, 20, 3, 4, 5, 6, 23, 8, 63, 48, 33, 50, 51, 36, 51, 52, 53, 54, 39, 56, 47, 32, 49, 34, 35, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3035(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 21, 22, 7, 24, 31, 0, 17, 2, 3, 20, 3, 4, 5, 6, 23, 8, 47, 48, 33, 50, 51, 36, 51, 52, 53, 54, 39, 56, 63, 32, 49, 34, 35, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3036(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 21, 22, 7, 24, 15, 16, 1, 2, 3, 20, 3, 4, 5, 6, 23, 8, 63, 32, 49, 50, 51, 36, 51, 52, 53, 54, 39, 56, 47, 48, 33, 34, 35, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3037(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 21, 22, 7, 24, 31, 16, 1, 2, 3, 20, 3, 4, 5, 6, 23, 8, 47, 32, 49, 50, 51, 36, 51, 52, 53, 54, 39, 56, 63, 48, 33, 34, 35, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3038(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 21, 22, 7, 24, 15, 0, 1, 2, 3, 20, 3, 4, 5, 6, 23, 8, 63, 48, 49, 50, 51, 36, 51, 52, 53, 54, 39, 56, 47, 32, 33, 34, 35, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3039(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 21, 22, 7, 24, 31, 0, 1, 2, 3, 20, 3, 4, 5, 6, 23, 8, 47, 48, 49, 50, 51, 36, 51, 52, 53, 54, 39, 56, 63, 32, 33, 34, 35, 52, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3040(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 21, 22, 7, 24, 15, 16, 17, 18, 19, 4, 3, 4, 5, 6, 23, 8, 63, 32, 33, 34, 35, 52, 51, 52, 53, 54, 39, 56, 47, 48, 49, 50, 51, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3041(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 21, 22, 7, 24, 31, 16, 17, 18, 19, 4, 3, 4, 5, 6, 23, 8, 47, 32, 33, 34, 35, 52, 51, 52, 53, 54, 39, 56, 63, 48, 49, 50, 51, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3042(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 21, 22, 7, 24, 15, 0, 17, 18, 19, 4, 3, 4, 5, 6, 23, 8, 63, 48, 33, 34, 35, 52, 51, 52, 53, 54, 39, 56, 47, 32, 49, 50, 51, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3043(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 21, 22, 7, 24, 31, 0, 17, 18, 19, 4, 3, 4, 5, 6, 23, 8, 47, 48, 33, 34, 35, 52, 51, 52, 53, 54, 39, 56, 63, 32, 49, 50, 51, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3044(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 21, 22, 7, 24, 15, 16, 1, 18, 19, 4, 3, 4, 5, 6, 23, 8, 63, 32, 49, 34, 35, 52, 51, 52, 53, 54, 39, 56, 47, 48, 33, 50, 51, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3045(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 21, 22, 7, 24, 31, 16, 1, 18, 19, 4, 3, 4, 5, 6, 23, 8, 47, 32, 49, 34, 35, 52, 51, 52, 53, 54, 39, 56, 63, 48, 33, 50, 51, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3046(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 21, 22, 7, 24, 15, 0, 1, 18, 19, 4, 3, 4, 5, 6, 23, 8, 63, 48, 49, 34, 35, 52, 51, 52, 53, 54, 39, 56, 47, 32, 33, 50, 51, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3047(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 21, 22, 7, 24, 31, 0, 1, 18, 19, 4, 3, 4, 5, 6, 23, 8, 47, 48, 49, 34, 35, 52, 51, 52, 53, 54, 39, 56, 63, 32, 33, 50, 51, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3048(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 21, 22, 7, 24, 15, 16, 17, 2, 19, 4, 3, 4, 5, 6, 23, 8, 63, 32, 33, 50, 35, 52, 51, 52, 53, 54, 39, 56, 47, 48, 49, 34, 51, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3049(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 21, 22, 7, 24, 31, 16, 17, 2, 19, 4, 3, 4, 5, 6, 23, 8, 47, 32, 33, 50, 35, 52, 51, 52, 53, 54, 39, 56, 63, 48, 49, 34, 51, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3050(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 21, 22, 7, 24, 15, 0, 17, 2, 19, 4, 3, 4, 5, 6, 23, 8, 63, 48, 33, 50, 35, 52, 51, 52, 53, 54, 39, 56, 47, 32, 49, 34, 51, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3051(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 21, 22, 7, 24, 31, 0, 17, 2, 19, 4, 3, 4, 5, 6, 23, 8, 47, 48, 33, 50, 35, 52, 51, 52, 53, 54, 39, 56, 63, 32, 49, 34, 51, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3052(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 21, 22, 7, 24, 15, 16, 1, 2, 19, 4, 3, 4, 5, 6, 23, 8, 63, 32, 49, 50, 35, 52, 51, 52, 53, 54, 39, 56, 47, 48, 33, 34, 51, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3053(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 21, 22, 7, 24, 31, 16, 1, 2, 19, 4, 3, 4, 5, 6, 23, 8, 47, 32, 49, 50, 35, 52, 51, 52, 53, 54, 39, 56, 63, 48, 33, 34, 51, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3054(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 21, 22, 7, 24, 15, 0, 1, 2, 19, 4, 3, 4, 5, 6, 23, 8, 63, 48, 49, 50, 35, 52, 51, 52, 53, 54, 39, 56, 47, 32, 33, 34, 51, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3055(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 21, 22, 7, 24, 31, 0, 1, 2, 19, 4, 3, 4, 5, 6, 23, 8, 47, 48, 49, 50, 35, 52, 51, 52, 53, 54, 39, 56, 63, 32, 33, 34, 51, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3056(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 21, 22, 7, 24, 15, 16, 17, 18, 3, 4, 3, 4, 5, 6, 23, 8, 63, 32, 33, 34, 51, 52, 51, 52, 53, 54, 39, 56, 47, 48, 49, 50, 35, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3057(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 21, 22, 7, 24, 31, 16, 17, 18, 3, 4, 3, 4, 5, 6, 23, 8, 47, 32, 33, 34, 51, 52, 51, 52, 53, 54, 39, 56, 63, 48, 49, 50, 35, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3058(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 21, 22, 7, 24, 15, 0, 17, 18, 3, 4, 3, 4, 5, 6, 23, 8, 63, 48, 33, 34, 51, 52, 51, 52, 53, 54, 39, 56, 47, 32, 49, 50, 35, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3059(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 21, 22, 7, 24, 31, 0, 17, 18, 3, 4, 3, 4, 5, 6, 23, 8, 47, 48, 33, 34, 51, 52, 51, 52, 53, 54, 39, 56, 63, 32, 49, 50, 35, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3060(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 21, 22, 7, 24, 15, 16, 1, 18, 3, 4, 3, 4, 5, 6, 23, 8, 63, 32, 49, 34, 51, 52, 51, 52, 53, 54, 39, 56, 47, 48, 33, 50, 35, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3061(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 21, 22, 7, 24, 31, 16, 1, 18, 3, 4, 3, 4, 5, 6, 23, 8, 47, 32, 49, 34, 51, 52, 51, 52, 53, 54, 39, 56, 63, 48, 33, 50, 35, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3062(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 21, 22, 7, 24, 15, 0, 1, 18, 3, 4, 3, 4, 5, 6, 23, 8, 63, 48, 49, 34, 51, 52, 51, 52, 53, 54, 39, 56, 47, 32, 33, 50, 35, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3063(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 21, 22, 7, 24, 31, 0, 1, 18, 3, 4, 3, 4, 5, 6, 23, 8, 47, 48, 49, 34, 51, 52, 51, 52, 53, 54, 39, 56, 63, 32, 33, 50, 35, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3064(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 21, 22, 7, 24, 15, 16, 17, 2, 3, 4, 3, 4, 5, 6, 23, 8, 63, 32, 33, 50, 51, 52, 51, 52, 53, 54, 39, 56, 47, 48, 49, 34, 35, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3065(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 21, 22, 7, 24, 31, 16, 17, 2, 3, 4, 3, 4, 5, 6, 23, 8, 47, 32, 33, 50, 51, 52, 51, 52, 53, 54, 39, 56, 63, 48, 49, 34, 35, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3066(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 21, 22, 7, 24, 15, 0, 17, 2, 3, 4, 3, 4, 5, 6, 23, 8, 63, 48, 33, 50, 51, 52, 51, 52, 53, 54, 39, 56, 47, 32, 49, 34, 35, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3067(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 21, 22, 7, 24, 31, 0, 17, 2, 3, 4, 3, 4, 5, 6, 23, 8, 47, 48, 33, 50, 51, 52, 51, 52, 53, 54, 39, 56, 63, 32, 49, 34, 35, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3068(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 21, 22, 7, 24, 15, 16, 1, 2, 3, 4, 3, 4, 5, 6, 23, 8, 63, 32, 49, 50, 51, 52, 51, 52, 53, 54, 39, 56, 47, 48, 33, 34, 35, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3069(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 21, 22, 7, 24, 31, 16, 1, 2, 3, 4, 3, 4, 5, 6, 23, 8, 47, 32, 49, 50, 51, 52, 51, 52, 53, 54, 39, 56, 63, 48, 33, 34, 35, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3070(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 21, 22, 7, 24, 15, 0, 1, 2, 3, 4, 3, 4, 5, 6, 23, 8, 63, 48, 49, 50, 51, 52, 51, 52, 53, 54, 39, 56, 47, 32, 33, 34, 35, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt3071(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 21, 22, 7, 24, 31, 0, 1, 2, 3, 4, 3, 4, 5, 6, 23, 8, 47, 48, 49, 50, 51, 52, 51, 52, 53, 54, 39, 56, 63, 32, 33, 34, 35, 36, 35, 36, 37, 38, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

#endif // DESGPU_COMPILE_ALL_SALTS


