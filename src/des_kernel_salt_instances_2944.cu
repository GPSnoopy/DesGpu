#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt2944(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 21, 22, 7, 24, 15, 16, 17, 18, 19, 20, 19, 4, 5, 6, 23, 8, 63, 32, 33, 34, 35, 36, 35, 52, 53, 54, 39, 56, 47, 48, 49, 50, 51, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2945(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 21, 22, 7, 24, 31, 16, 17, 18, 19, 20, 19, 4, 5, 6, 23, 8, 47, 32, 33, 34, 35, 36, 35, 52, 53, 54, 39, 56, 63, 48, 49, 50, 51, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2946(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 21, 22, 7, 24, 15, 0, 17, 18, 19, 20, 19, 4, 5, 6, 23, 8, 63, 48, 33, 34, 35, 36, 35, 52, 53, 54, 39, 56, 47, 32, 49, 50, 51, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2947(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 21, 22, 7, 24, 31, 0, 17, 18, 19, 20, 19, 4, 5, 6, 23, 8, 47, 48, 33, 34, 35, 36, 35, 52, 53, 54, 39, 56, 63, 32, 49, 50, 51, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2948(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 21, 22, 7, 24, 15, 16, 1, 18, 19, 20, 19, 4, 5, 6, 23, 8, 63, 32, 49, 34, 35, 36, 35, 52, 53, 54, 39, 56, 47, 48, 33, 50, 51, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2949(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 21, 22, 7, 24, 31, 16, 1, 18, 19, 20, 19, 4, 5, 6, 23, 8, 47, 32, 49, 34, 35, 36, 35, 52, 53, 54, 39, 56, 63, 48, 33, 50, 51, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2950(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 21, 22, 7, 24, 15, 0, 1, 18, 19, 20, 19, 4, 5, 6, 23, 8, 63, 48, 49, 34, 35, 36, 35, 52, 53, 54, 39, 56, 47, 32, 33, 50, 51, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2951(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 21, 22, 7, 24, 31, 0, 1, 18, 19, 20, 19, 4, 5, 6, 23, 8, 47, 48, 49, 34, 35, 36, 35, 52, 53, 54, 39, 56, 63, 32, 33, 50, 51, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2952(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 21, 22, 7, 24, 15, 16, 17, 2, 19, 20, 19, 4, 5, 6, 23, 8, 63, 32, 33, 50, 35, 36, 35, 52, 53, 54, 39, 56, 47, 48, 49, 34, 51, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2953(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 21, 22, 7, 24, 31, 16, 17, 2, 19, 20, 19, 4, 5, 6, 23, 8, 47, 32, 33, 50, 35, 36, 35, 52, 53, 54, 39, 56, 63, 48, 49, 34, 51, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2954(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 21, 22, 7, 24, 15, 0, 17, 2, 19, 20, 19, 4, 5, 6, 23, 8, 63, 48, 33, 50, 35, 36, 35, 52, 53, 54, 39, 56, 47, 32, 49, 34, 51, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2955(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 21, 22, 7, 24, 31, 0, 17, 2, 19, 20, 19, 4, 5, 6, 23, 8, 47, 48, 33, 50, 35, 36, 35, 52, 53, 54, 39, 56, 63, 32, 49, 34, 51, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2956(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 21, 22, 7, 24, 15, 16, 1, 2, 19, 20, 19, 4, 5, 6, 23, 8, 63, 32, 49, 50, 35, 36, 35, 52, 53, 54, 39, 56, 47, 48, 33, 34, 51, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2957(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 21, 22, 7, 24, 31, 16, 1, 2, 19, 20, 19, 4, 5, 6, 23, 8, 47, 32, 49, 50, 35, 36, 35, 52, 53, 54, 39, 56, 63, 48, 33, 34, 51, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2958(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 21, 22, 7, 24, 15, 0, 1, 2, 19, 20, 19, 4, 5, 6, 23, 8, 63, 48, 49, 50, 35, 36, 35, 52, 53, 54, 39, 56, 47, 32, 33, 34, 51, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2959(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 21, 22, 7, 24, 31, 0, 1, 2, 19, 20, 19, 4, 5, 6, 23, 8, 47, 48, 49, 50, 35, 36, 35, 52, 53, 54, 39, 56, 63, 32, 33, 34, 51, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2960(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 21, 22, 7, 24, 15, 16, 17, 18, 3, 20, 19, 4, 5, 6, 23, 8, 63, 32, 33, 34, 51, 36, 35, 52, 53, 54, 39, 56, 47, 48, 49, 50, 35, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2961(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 21, 22, 7, 24, 31, 16, 17, 18, 3, 20, 19, 4, 5, 6, 23, 8, 47, 32, 33, 34, 51, 36, 35, 52, 53, 54, 39, 56, 63, 48, 49, 50, 35, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2962(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 21, 22, 7, 24, 15, 0, 17, 18, 3, 20, 19, 4, 5, 6, 23, 8, 63, 48, 33, 34, 51, 36, 35, 52, 53, 54, 39, 56, 47, 32, 49, 50, 35, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2963(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 21, 22, 7, 24, 31, 0, 17, 18, 3, 20, 19, 4, 5, 6, 23, 8, 47, 48, 33, 34, 51, 36, 35, 52, 53, 54, 39, 56, 63, 32, 49, 50, 35, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2964(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 21, 22, 7, 24, 15, 16, 1, 18, 3, 20, 19, 4, 5, 6, 23, 8, 63, 32, 49, 34, 51, 36, 35, 52, 53, 54, 39, 56, 47, 48, 33, 50, 35, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2965(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 21, 22, 7, 24, 31, 16, 1, 18, 3, 20, 19, 4, 5, 6, 23, 8, 47, 32, 49, 34, 51, 36, 35, 52, 53, 54, 39, 56, 63, 48, 33, 50, 35, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2966(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 21, 22, 7, 24, 15, 0, 1, 18, 3, 20, 19, 4, 5, 6, 23, 8, 63, 48, 49, 34, 51, 36, 35, 52, 53, 54, 39, 56, 47, 32, 33, 50, 35, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2967(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 21, 22, 7, 24, 31, 0, 1, 18, 3, 20, 19, 4, 5, 6, 23, 8, 47, 48, 49, 34, 51, 36, 35, 52, 53, 54, 39, 56, 63, 32, 33, 50, 35, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2968(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 21, 22, 7, 24, 15, 16, 17, 2, 3, 20, 19, 4, 5, 6, 23, 8, 63, 32, 33, 50, 51, 36, 35, 52, 53, 54, 39, 56, 47, 48, 49, 34, 35, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2969(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 21, 22, 7, 24, 31, 16, 17, 2, 3, 20, 19, 4, 5, 6, 23, 8, 47, 32, 33, 50, 51, 36, 35, 52, 53, 54, 39, 56, 63, 48, 49, 34, 35, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2970(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 21, 22, 7, 24, 15, 0, 17, 2, 3, 20, 19, 4, 5, 6, 23, 8, 63, 48, 33, 50, 51, 36, 35, 52, 53, 54, 39, 56, 47, 32, 49, 34, 35, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2971(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 21, 22, 7, 24, 31, 0, 17, 2, 3, 20, 19, 4, 5, 6, 23, 8, 47, 48, 33, 50, 51, 36, 35, 52, 53, 54, 39, 56, 63, 32, 49, 34, 35, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2972(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 21, 22, 7, 24, 15, 16, 1, 2, 3, 20, 19, 4, 5, 6, 23, 8, 63, 32, 49, 50, 51, 36, 35, 52, 53, 54, 39, 56, 47, 48, 33, 34, 35, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2973(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 21, 22, 7, 24, 31, 16, 1, 2, 3, 20, 19, 4, 5, 6, 23, 8, 47, 32, 49, 50, 51, 36, 35, 52, 53, 54, 39, 56, 63, 48, 33, 34, 35, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2974(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 21, 22, 7, 24, 15, 0, 1, 2, 3, 20, 19, 4, 5, 6, 23, 8, 63, 48, 49, 50, 51, 36, 35, 52, 53, 54, 39, 56, 47, 32, 33, 34, 35, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2975(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 21, 22, 7, 24, 31, 0, 1, 2, 3, 20, 19, 4, 5, 6, 23, 8, 47, 48, 49, 50, 51, 36, 35, 52, 53, 54, 39, 56, 63, 32, 33, 34, 35, 52, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2976(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 21, 22, 7, 24, 15, 16, 17, 18, 19, 4, 19, 4, 5, 6, 23, 8, 63, 32, 33, 34, 35, 52, 35, 52, 53, 54, 39, 56, 47, 48, 49, 50, 51, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2977(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 21, 22, 7, 24, 31, 16, 17, 18, 19, 4, 19, 4, 5, 6, 23, 8, 47, 32, 33, 34, 35, 52, 35, 52, 53, 54, 39, 56, 63, 48, 49, 50, 51, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2978(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 21, 22, 7, 24, 15, 0, 17, 18, 19, 4, 19, 4, 5, 6, 23, 8, 63, 48, 33, 34, 35, 52, 35, 52, 53, 54, 39, 56, 47, 32, 49, 50, 51, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2979(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 21, 22, 7, 24, 31, 0, 17, 18, 19, 4, 19, 4, 5, 6, 23, 8, 47, 48, 33, 34, 35, 52, 35, 52, 53, 54, 39, 56, 63, 32, 49, 50, 51, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2980(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 21, 22, 7, 24, 15, 16, 1, 18, 19, 4, 19, 4, 5, 6, 23, 8, 63, 32, 49, 34, 35, 52, 35, 52, 53, 54, 39, 56, 47, 48, 33, 50, 51, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2981(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 21, 22, 7, 24, 31, 16, 1, 18, 19, 4, 19, 4, 5, 6, 23, 8, 47, 32, 49, 34, 35, 52, 35, 52, 53, 54, 39, 56, 63, 48, 33, 50, 51, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2982(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 21, 22, 7, 24, 15, 0, 1, 18, 19, 4, 19, 4, 5, 6, 23, 8, 63, 48, 49, 34, 35, 52, 35, 52, 53, 54, 39, 56, 47, 32, 33, 50, 51, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2983(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 21, 22, 7, 24, 31, 0, 1, 18, 19, 4, 19, 4, 5, 6, 23, 8, 47, 48, 49, 34, 35, 52, 35, 52, 53, 54, 39, 56, 63, 32, 33, 50, 51, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2984(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 21, 22, 7, 24, 15, 16, 17, 2, 19, 4, 19, 4, 5, 6, 23, 8, 63, 32, 33, 50, 35, 52, 35, 52, 53, 54, 39, 56, 47, 48, 49, 34, 51, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2985(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 21, 22, 7, 24, 31, 16, 17, 2, 19, 4, 19, 4, 5, 6, 23, 8, 47, 32, 33, 50, 35, 52, 35, 52, 53, 54, 39, 56, 63, 48, 49, 34, 51, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2986(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 21, 22, 7, 24, 15, 0, 17, 2, 19, 4, 19, 4, 5, 6, 23, 8, 63, 48, 33, 50, 35, 52, 35, 52, 53, 54, 39, 56, 47, 32, 49, 34, 51, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2987(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 21, 22, 7, 24, 31, 0, 17, 2, 19, 4, 19, 4, 5, 6, 23, 8, 47, 48, 33, 50, 35, 52, 35, 52, 53, 54, 39, 56, 63, 32, 49, 34, 51, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2988(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 21, 22, 7, 24, 15, 16, 1, 2, 19, 4, 19, 4, 5, 6, 23, 8, 63, 32, 49, 50, 35, 52, 35, 52, 53, 54, 39, 56, 47, 48, 33, 34, 51, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2989(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 21, 22, 7, 24, 31, 16, 1, 2, 19, 4, 19, 4, 5, 6, 23, 8, 47, 32, 49, 50, 35, 52, 35, 52, 53, 54, 39, 56, 63, 48, 33, 34, 51, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2990(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 21, 22, 7, 24, 15, 0, 1, 2, 19, 4, 19, 4, 5, 6, 23, 8, 63, 48, 49, 50, 35, 52, 35, 52, 53, 54, 39, 56, 47, 32, 33, 34, 51, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2991(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 21, 22, 7, 24, 31, 0, 1, 2, 19, 4, 19, 4, 5, 6, 23, 8, 47, 48, 49, 50, 35, 52, 35, 52, 53, 54, 39, 56, 63, 32, 33, 34, 51, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2992(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 21, 22, 7, 24, 15, 16, 17, 18, 3, 4, 19, 4, 5, 6, 23, 8, 63, 32, 33, 34, 51, 52, 35, 52, 53, 54, 39, 56, 47, 48, 49, 50, 35, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2993(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 21, 22, 7, 24, 31, 16, 17, 18, 3, 4, 19, 4, 5, 6, 23, 8, 47, 32, 33, 34, 51, 52, 35, 52, 53, 54, 39, 56, 63, 48, 49, 50, 35, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2994(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 21, 22, 7, 24, 15, 0, 17, 18, 3, 4, 19, 4, 5, 6, 23, 8, 63, 48, 33, 34, 51, 52, 35, 52, 53, 54, 39, 56, 47, 32, 49, 50, 35, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2995(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 21, 22, 7, 24, 31, 0, 17, 18, 3, 4, 19, 4, 5, 6, 23, 8, 47, 48, 33, 34, 51, 52, 35, 52, 53, 54, 39, 56, 63, 32, 49, 50, 35, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2996(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 21, 22, 7, 24, 15, 16, 1, 18, 3, 4, 19, 4, 5, 6, 23, 8, 63, 32, 49, 34, 51, 52, 35, 52, 53, 54, 39, 56, 47, 48, 33, 50, 35, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2997(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 21, 22, 7, 24, 31, 16, 1, 18, 3, 4, 19, 4, 5, 6, 23, 8, 47, 32, 49, 34, 51, 52, 35, 52, 53, 54, 39, 56, 63, 48, 33, 50, 35, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2998(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 21, 22, 7, 24, 15, 0, 1, 18, 3, 4, 19, 4, 5, 6, 23, 8, 63, 48, 49, 34, 51, 52, 35, 52, 53, 54, 39, 56, 47, 32, 33, 50, 35, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2999(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 21, 22, 7, 24, 31, 0, 1, 18, 3, 4, 19, 4, 5, 6, 23, 8, 47, 48, 49, 34, 51, 52, 35, 52, 53, 54, 39, 56, 63, 32, 33, 50, 35, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3000(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 21, 22, 7, 24, 15, 16, 17, 2, 3, 4, 19, 4, 5, 6, 23, 8, 63, 32, 33, 50, 51, 52, 35, 52, 53, 54, 39, 56, 47, 48, 49, 34, 35, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3001(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 21, 22, 7, 24, 31, 16, 17, 2, 3, 4, 19, 4, 5, 6, 23, 8, 47, 32, 33, 50, 51, 52, 35, 52, 53, 54, 39, 56, 63, 48, 49, 34, 35, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3002(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 21, 22, 7, 24, 15, 0, 17, 2, 3, 4, 19, 4, 5, 6, 23, 8, 63, 48, 33, 50, 51, 52, 35, 52, 53, 54, 39, 56, 47, 32, 49, 34, 35, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3003(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 21, 22, 7, 24, 31, 0, 17, 2, 3, 4, 19, 4, 5, 6, 23, 8, 47, 48, 33, 50, 51, 52, 35, 52, 53, 54, 39, 56, 63, 32, 49, 34, 35, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3004(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 21, 22, 7, 24, 15, 16, 1, 2, 3, 4, 19, 4, 5, 6, 23, 8, 63, 32, 49, 50, 51, 52, 35, 52, 53, 54, 39, 56, 47, 48, 33, 34, 35, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3005(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 21, 22, 7, 24, 31, 16, 1, 2, 3, 4, 19, 4, 5, 6, 23, 8, 47, 32, 49, 50, 51, 52, 35, 52, 53, 54, 39, 56, 63, 48, 33, 34, 35, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3006(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 21, 22, 7, 24, 15, 0, 1, 2, 3, 4, 19, 4, 5, 6, 23, 8, 63, 48, 49, 50, 51, 52, 35, 52, 53, 54, 39, 56, 47, 32, 33, 34, 35, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3007(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 21, 22, 7, 24, 31, 0, 1, 2, 3, 4, 19, 4, 5, 6, 23, 8, 47, 48, 49, 50, 51, 52, 35, 52, 53, 54, 39, 56, 63, 32, 33, 34, 35, 36, 51, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3008(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 21, 22, 7, 24, 15, 16, 17, 18, 19, 20, 3, 4, 5, 6, 23, 8, 63, 32, 33, 34, 35, 36, 51, 52, 53, 54, 39, 56, 47, 48, 49, 50, 51, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3009(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 21, 22, 7, 24, 31, 16, 17, 18, 19, 20, 3, 4, 5, 6, 23, 8, 47, 32, 33, 34, 35, 36, 51, 52, 53, 54, 39, 56, 63, 48, 49, 50, 51, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3010(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 21, 22, 7, 24, 15, 0, 17, 18, 19, 20, 3, 4, 5, 6, 23, 8, 63, 48, 33, 34, 35, 36, 51, 52, 53, 54, 39, 56, 47, 32, 49, 50, 51, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3011(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 21, 22, 7, 24, 31, 0, 17, 18, 19, 20, 3, 4, 5, 6, 23, 8, 47, 48, 33, 34, 35, 36, 51, 52, 53, 54, 39, 56, 63, 32, 49, 50, 51, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3012(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 21, 22, 7, 24, 15, 16, 1, 18, 19, 20, 3, 4, 5, 6, 23, 8, 63, 32, 49, 34, 35, 36, 51, 52, 53, 54, 39, 56, 47, 48, 33, 50, 51, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3013(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 21, 22, 7, 24, 31, 16, 1, 18, 19, 20, 3, 4, 5, 6, 23, 8, 47, 32, 49, 34, 35, 36, 51, 52, 53, 54, 39, 56, 63, 48, 33, 50, 51, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3014(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 21, 22, 7, 24, 15, 0, 1, 18, 19, 20, 3, 4, 5, 6, 23, 8, 63, 48, 49, 34, 35, 36, 51, 52, 53, 54, 39, 56, 47, 32, 33, 50, 51, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3015(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 21, 22, 7, 24, 31, 0, 1, 18, 19, 20, 3, 4, 5, 6, 23, 8, 47, 48, 49, 34, 35, 36, 51, 52, 53, 54, 39, 56, 63, 32, 33, 50, 51, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3016(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 21, 22, 7, 24, 15, 16, 17, 2, 19, 20, 3, 4, 5, 6, 23, 8, 63, 32, 33, 50, 35, 36, 51, 52, 53, 54, 39, 56, 47, 48, 49, 34, 51, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3017(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 21, 22, 7, 24, 31, 16, 17, 2, 19, 20, 3, 4, 5, 6, 23, 8, 47, 32, 33, 50, 35, 36, 51, 52, 53, 54, 39, 56, 63, 48, 49, 34, 51, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3018(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 21, 22, 7, 24, 15, 0, 17, 2, 19, 20, 3, 4, 5, 6, 23, 8, 63, 48, 33, 50, 35, 36, 51, 52, 53, 54, 39, 56, 47, 32, 49, 34, 51, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3019(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 21, 22, 7, 24, 31, 0, 17, 2, 19, 20, 3, 4, 5, 6, 23, 8, 47, 48, 33, 50, 35, 36, 51, 52, 53, 54, 39, 56, 63, 32, 49, 34, 51, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3020(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 21, 22, 7, 24, 15, 16, 1, 2, 19, 20, 3, 4, 5, 6, 23, 8, 63, 32, 49, 50, 35, 36, 51, 52, 53, 54, 39, 56, 47, 48, 33, 34, 51, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3021(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 21, 22, 7, 24, 31, 16, 1, 2, 19, 20, 3, 4, 5, 6, 23, 8, 47, 32, 49, 50, 35, 36, 51, 52, 53, 54, 39, 56, 63, 48, 33, 34, 51, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3022(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 21, 22, 7, 24, 15, 0, 1, 2, 19, 20, 3, 4, 5, 6, 23, 8, 63, 48, 49, 50, 35, 36, 51, 52, 53, 54, 39, 56, 47, 32, 33, 34, 51, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3023(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 21, 22, 7, 24, 31, 0, 1, 2, 19, 20, 3, 4, 5, 6, 23, 8, 47, 48, 49, 50, 35, 36, 51, 52, 53, 54, 39, 56, 63, 32, 33, 34, 51, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3024(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 21, 22, 7, 24, 15, 16, 17, 18, 3, 20, 3, 4, 5, 6, 23, 8, 63, 32, 33, 34, 51, 36, 51, 52, 53, 54, 39, 56, 47, 48, 49, 50, 35, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3025(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 21, 22, 7, 24, 31, 16, 17, 18, 3, 20, 3, 4, 5, 6, 23, 8, 47, 32, 33, 34, 51, 36, 51, 52, 53, 54, 39, 56, 63, 48, 49, 50, 35, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3026(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 21, 22, 7, 24, 15, 0, 17, 18, 3, 20, 3, 4, 5, 6, 23, 8, 63, 48, 33, 34, 51, 36, 51, 52, 53, 54, 39, 56, 47, 32, 49, 50, 35, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3027(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 21, 22, 7, 24, 31, 0, 17, 18, 3, 20, 3, 4, 5, 6, 23, 8, 47, 48, 33, 34, 51, 36, 51, 52, 53, 54, 39, 56, 63, 32, 49, 50, 35, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3028(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 21, 22, 7, 24, 15, 16, 1, 18, 3, 20, 3, 4, 5, 6, 23, 8, 63, 32, 49, 34, 51, 36, 51, 52, 53, 54, 39, 56, 47, 48, 33, 50, 35, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3029(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 21, 22, 7, 24, 31, 16, 1, 18, 3, 20, 3, 4, 5, 6, 23, 8, 47, 32, 49, 34, 51, 36, 51, 52, 53, 54, 39, 56, 63, 48, 33, 50, 35, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3030(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 21, 22, 7, 24, 15, 0, 1, 18, 3, 20, 3, 4, 5, 6, 23, 8, 63, 48, 49, 34, 51, 36, 51, 52, 53, 54, 39, 56, 47, 32, 33, 50, 35, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3031(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 21, 22, 7, 24, 31, 0, 1, 18, 3, 20, 3, 4, 5, 6, 23, 8, 47, 48, 49, 34, 51, 36, 51, 52, 53, 54, 39, 56, 63, 32, 33, 50, 35, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3032(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 21, 22, 7, 24, 15, 16, 17, 2, 3, 20, 3, 4, 5, 6, 23, 8, 63, 32, 33, 50, 51, 36, 51, 52, 53, 54, 39, 56, 47, 48, 49, 34, 35, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3033(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 21, 22, 7, 24, 31, 16, 17, 2, 3, 20, 3, 4, 5, 6, 23, 8, 47, 32, 33, 50, 51, 36, 51, 52, 53, 54, 39, 56, 63, 48, 49, 34, 35, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3034(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 21, 22, 7, 24, 15, 0, 17, 2, 3, 20, 3, 4, 5, 6, 23, 8, 63, 48, 33, 50, 51, 36, 51, 52, 53, 54, 39, 56, 47, 32, 49, 34, 35, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3035(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 21, 22, 7, 24, 31, 0, 17, 2, 3, 20, 3, 4, 5, 6, 23, 8, 47, 48, 33, 50, 51, 36, 51, 52, 53, 54, 39, 56, 63, 32, 49, 34, 35, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3036(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 21, 22, 7, 24, 15, 16, 1, 2, 3, 20, 3, 4, 5, 6, 23, 8, 63, 32, 49, 50, 51, 36, 51, 52, 53, 54, 39, 56, 47, 48, 33, 34, 35, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3037(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 21, 22, 7, 24, 31, 16, 1, 2, 3, 20, 3, 4, 5, 6, 23, 8, 47, 32, 49, 50, 51, 36, 51, 52, 53, 54, 39, 56, 63, 48, 33, 34, 35, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3038(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 21, 22, 7, 24, 15, 0, 1, 2, 3, 20, 3, 4, 5, 6, 23, 8, 63, 48, 49, 50, 51, 36, 51, 52, 53, 54, 39, 56, 47, 32, 33, 34, 35, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3039(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 21, 22, 7, 24, 31, 0, 1, 2, 3, 20, 3, 4, 5, 6, 23, 8, 47, 48, 49, 50, 51, 36, 51, 52, 53, 54, 39, 56, 63, 32, 33, 34, 35, 52, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3040(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 21, 22, 7, 24, 15, 16, 17, 18, 19, 4, 3, 4, 5, 6, 23, 8, 63, 32, 33, 34, 35, 52, 51, 52, 53, 54, 39, 56, 47, 48, 49, 50, 51, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3041(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 21, 22, 7, 24, 31, 16, 17, 18, 19, 4, 3, 4, 5, 6, 23, 8, 47, 32, 33, 34, 35, 52, 51, 52, 53, 54, 39, 56, 63, 48, 49, 50, 51, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3042(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 21, 22, 7, 24, 15, 0, 17, 18, 19, 4, 3, 4, 5, 6, 23, 8, 63, 48, 33, 34, 35, 52, 51, 52, 53, 54, 39, 56, 47, 32, 49, 50, 51, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3043(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 21, 22, 7, 24, 31, 0, 17, 18, 19, 4, 3, 4, 5, 6, 23, 8, 47, 48, 33, 34, 35, 52, 51, 52, 53, 54, 39, 56, 63, 32, 49, 50, 51, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3044(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 21, 22, 7, 24, 15, 16, 1, 18, 19, 4, 3, 4, 5, 6, 23, 8, 63, 32, 49, 34, 35, 52, 51, 52, 53, 54, 39, 56, 47, 48, 33, 50, 51, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3045(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 21, 22, 7, 24, 31, 16, 1, 18, 19, 4, 3, 4, 5, 6, 23, 8, 47, 32, 49, 34, 35, 52, 51, 52, 53, 54, 39, 56, 63, 48, 33, 50, 51, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3046(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 21, 22, 7, 24, 15, 0, 1, 18, 19, 4, 3, 4, 5, 6, 23, 8, 63, 48, 49, 34, 35, 52, 51, 52, 53, 54, 39, 56, 47, 32, 33, 50, 51, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3047(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 21, 22, 7, 24, 31, 0, 1, 18, 19, 4, 3, 4, 5, 6, 23, 8, 47, 48, 49, 34, 35, 52, 51, 52, 53, 54, 39, 56, 63, 32, 33, 50, 51, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3048(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 21, 22, 7, 24, 15, 16, 17, 2, 19, 4, 3, 4, 5, 6, 23, 8, 63, 32, 33, 50, 35, 52, 51, 52, 53, 54, 39, 56, 47, 48, 49, 34, 51, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3049(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 21, 22, 7, 24, 31, 16, 17, 2, 19, 4, 3, 4, 5, 6, 23, 8, 47, 32, 33, 50, 35, 52, 51, 52, 53, 54, 39, 56, 63, 48, 49, 34, 51, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3050(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 21, 22, 7, 24, 15, 0, 17, 2, 19, 4, 3, 4, 5, 6, 23, 8, 63, 48, 33, 50, 35, 52, 51, 52, 53, 54, 39, 56, 47, 32, 49, 34, 51, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3051(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 21, 22, 7, 24, 31, 0, 17, 2, 19, 4, 3, 4, 5, 6, 23, 8, 47, 48, 33, 50, 35, 52, 51, 52, 53, 54, 39, 56, 63, 32, 49, 34, 51, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3052(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 21, 22, 7, 24, 15, 16, 1, 2, 19, 4, 3, 4, 5, 6, 23, 8, 63, 32, 49, 50, 35, 52, 51, 52, 53, 54, 39, 56, 47, 48, 33, 34, 51, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3053(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 21, 22, 7, 24, 31, 16, 1, 2, 19, 4, 3, 4, 5, 6, 23, 8, 47, 32, 49, 50, 35, 52, 51, 52, 53, 54, 39, 56, 63, 48, 33, 34, 51, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3054(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 21, 22, 7, 24, 15, 0, 1, 2, 19, 4, 3, 4, 5, 6, 23, 8, 63, 48, 49, 50, 35, 52, 51, 52, 53, 54, 39, 56, 47, 32, 33, 34, 51, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3055(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 21, 22, 7, 24, 31, 0, 1, 2, 19, 4, 3, 4, 5, 6, 23, 8, 47, 48, 49, 50, 35, 52, 51, 52, 53, 54, 39, 56, 63, 32, 33, 34, 51, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3056(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 21, 22, 7, 24, 15, 16, 17, 18, 3, 4, 3, 4, 5, 6, 23, 8, 63, 32, 33, 34, 51, 52, 51, 52, 53, 54, 39, 56, 47, 48, 49, 50, 35, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3057(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 21, 22, 7, 24, 31, 16, 17, 18, 3, 4, 3, 4, 5, 6, 23, 8, 47, 32, 33, 34, 51, 52, 51, 52, 53, 54, 39, 56, 63, 48, 49, 50, 35, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3058(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 21, 22, 7, 24, 15, 0, 17, 18, 3, 4, 3, 4, 5, 6, 23, 8, 63, 48, 33, 34, 51, 52, 51, 52, 53, 54, 39, 56, 47, 32, 49, 50, 35, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3059(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 21, 22, 7, 24, 31, 0, 17, 18, 3, 4, 3, 4, 5, 6, 23, 8, 47, 48, 33, 34, 51, 52, 51, 52, 53, 54, 39, 56, 63, 32, 49, 50, 35, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3060(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 21, 22, 7, 24, 15, 16, 1, 18, 3, 4, 3, 4, 5, 6, 23, 8, 63, 32, 49, 34, 51, 52, 51, 52, 53, 54, 39, 56, 47, 48, 33, 50, 35, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3061(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 21, 22, 7, 24, 31, 16, 1, 18, 3, 4, 3, 4, 5, 6, 23, 8, 47, 32, 49, 34, 51, 52, 51, 52, 53, 54, 39, 56, 63, 48, 33, 50, 35, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3062(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 21, 22, 7, 24, 15, 0, 1, 18, 3, 4, 3, 4, 5, 6, 23, 8, 63, 48, 49, 34, 51, 52, 51, 52, 53, 54, 39, 56, 47, 32, 33, 50, 35, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3063(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 21, 22, 7, 24, 31, 0, 1, 18, 3, 4, 3, 4, 5, 6, 23, 8, 47, 48, 49, 34, 51, 52, 51, 52, 53, 54, 39, 56, 63, 32, 33, 50, 35, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3064(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 21, 22, 7, 24, 15, 16, 17, 2, 3, 4, 3, 4, 5, 6, 23, 8, 63, 32, 33, 50, 51, 52, 51, 52, 53, 54, 39, 56, 47, 48, 49, 34, 35, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3065(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 21, 22, 7, 24, 31, 16, 17, 2, 3, 4, 3, 4, 5, 6, 23, 8, 47, 32, 33, 50, 51, 52, 51, 52, 53, 54, 39, 56, 63, 48, 49, 34, 35, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3066(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 21, 22, 7, 24, 15, 0, 17, 2, 3, 4, 3, 4, 5, 6, 23, 8, 63, 48, 33, 50, 51, 52, 51, 52, 53, 54, 39, 56, 47, 32, 49, 34, 35, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3067(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 21, 22, 7, 24, 31, 0, 17, 2, 3, 4, 3, 4, 5, 6, 23, 8, 47, 48, 33, 50, 51, 52, 51, 52, 53, 54, 39, 56, 63, 32, 49, 34, 35, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3068(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 21, 22, 7, 24, 15, 16, 1, 2, 3, 4, 3, 4, 5, 6, 23, 8, 63, 32, 49, 50, 51, 52, 51, 52, 53, 54, 39, 56, 47, 48, 33, 34, 35, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3069(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 21, 22, 7, 24, 31, 16, 1, 2, 3, 4, 3, 4, 5, 6, 23, 8, 47, 32, 49, 50, 51, 52, 51, 52, 53, 54, 39, 56, 63, 48, 33, 34, 35, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3070(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 21, 22, 7, 24, 15, 0, 1, 2, 3, 4, 3, 4, 5, 6, 23, 8, 63, 48, 49, 50, 51, 52, 51, 52, 53, 54, 39, 56, 47, 32, 33, 34, 35, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3071(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 21, 22, 7, 24, 31, 0, 1, 2, 3, 4, 3, 4, 5, 6, 23, 8, 47, 48, 49, 50, 51, 52, 51, 52, 53, 54, 39, 56, 63, 32, 33, 34, 35, 36, 35, 36, 37, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}


