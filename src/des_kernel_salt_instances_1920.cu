#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

#ifdef DESGPU_COMPILE_ALL_SALTS

void des_25_encrypt_salt1920(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 21, 22, 23, 8, 15, 16, 17, 18, 19, 20, 19, 4, 5, 6, 7, 24, 63, 32, 33, 34, 35, 36, 35, 52, 53, 54, 55, 40, 47, 48, 49, 50, 51, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1921(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 21, 22, 23, 8, 31, 16, 17, 18, 19, 20, 19, 4, 5, 6, 7, 24, 47, 32, 33, 34, 35, 36, 35, 52, 53, 54, 55, 40, 63, 48, 49, 50, 51, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1922(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 21, 22, 23, 8, 15, 0, 17, 18, 19, 20, 19, 4, 5, 6, 7, 24, 63, 48, 33, 34, 35, 36, 35, 52, 53, 54, 55, 40, 47, 32, 49, 50, 51, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1923(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 21, 22, 23, 8, 31, 0, 17, 18, 19, 20, 19, 4, 5, 6, 7, 24, 47, 48, 33, 34, 35, 36, 35, 52, 53, 54, 55, 40, 63, 32, 49, 50, 51, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1924(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 21, 22, 23, 8, 15, 16, 1, 18, 19, 20, 19, 4, 5, 6, 7, 24, 63, 32, 49, 34, 35, 36, 35, 52, 53, 54, 55, 40, 47, 48, 33, 50, 51, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1925(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 21, 22, 23, 8, 31, 16, 1, 18, 19, 20, 19, 4, 5, 6, 7, 24, 47, 32, 49, 34, 35, 36, 35, 52, 53, 54, 55, 40, 63, 48, 33, 50, 51, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1926(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 21, 22, 23, 8, 15, 0, 1, 18, 19, 20, 19, 4, 5, 6, 7, 24, 63, 48, 49, 34, 35, 36, 35, 52, 53, 54, 55, 40, 47, 32, 33, 50, 51, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1927(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 21, 22, 23, 8, 31, 0, 1, 18, 19, 20, 19, 4, 5, 6, 7, 24, 47, 48, 49, 34, 35, 36, 35, 52, 53, 54, 55, 40, 63, 32, 33, 50, 51, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1928(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 21, 22, 23, 8, 15, 16, 17, 2, 19, 20, 19, 4, 5, 6, 7, 24, 63, 32, 33, 50, 35, 36, 35, 52, 53, 54, 55, 40, 47, 48, 49, 34, 51, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1929(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 21, 22, 23, 8, 31, 16, 17, 2, 19, 20, 19, 4, 5, 6, 7, 24, 47, 32, 33, 50, 35, 36, 35, 52, 53, 54, 55, 40, 63, 48, 49, 34, 51, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1930(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 21, 22, 23, 8, 15, 0, 17, 2, 19, 20, 19, 4, 5, 6, 7, 24, 63, 48, 33, 50, 35, 36, 35, 52, 53, 54, 55, 40, 47, 32, 49, 34, 51, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1931(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 21, 22, 23, 8, 31, 0, 17, 2, 19, 20, 19, 4, 5, 6, 7, 24, 47, 48, 33, 50, 35, 36, 35, 52, 53, 54, 55, 40, 63, 32, 49, 34, 51, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1932(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 21, 22, 23, 8, 15, 16, 1, 2, 19, 20, 19, 4, 5, 6, 7, 24, 63, 32, 49, 50, 35, 36, 35, 52, 53, 54, 55, 40, 47, 48, 33, 34, 51, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1933(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 21, 22, 23, 8, 31, 16, 1, 2, 19, 20, 19, 4, 5, 6, 7, 24, 47, 32, 49, 50, 35, 36, 35, 52, 53, 54, 55, 40, 63, 48, 33, 34, 51, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1934(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 21, 22, 23, 8, 15, 0, 1, 2, 19, 20, 19, 4, 5, 6, 7, 24, 63, 48, 49, 50, 35, 36, 35, 52, 53, 54, 55, 40, 47, 32, 33, 34, 51, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1935(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 21, 22, 23, 8, 31, 0, 1, 2, 19, 20, 19, 4, 5, 6, 7, 24, 47, 48, 49, 50, 35, 36, 35, 52, 53, 54, 55, 40, 63, 32, 33, 34, 51, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1936(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 21, 22, 23, 8, 15, 16, 17, 18, 3, 20, 19, 4, 5, 6, 7, 24, 63, 32, 33, 34, 51, 36, 35, 52, 53, 54, 55, 40, 47, 48, 49, 50, 35, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1937(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 21, 22, 23, 8, 31, 16, 17, 18, 3, 20, 19, 4, 5, 6, 7, 24, 47, 32, 33, 34, 51, 36, 35, 52, 53, 54, 55, 40, 63, 48, 49, 50, 35, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1938(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 21, 22, 23, 8, 15, 0, 17, 18, 3, 20, 19, 4, 5, 6, 7, 24, 63, 48, 33, 34, 51, 36, 35, 52, 53, 54, 55, 40, 47, 32, 49, 50, 35, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1939(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 21, 22, 23, 8, 31, 0, 17, 18, 3, 20, 19, 4, 5, 6, 7, 24, 47, 48, 33, 34, 51, 36, 35, 52, 53, 54, 55, 40, 63, 32, 49, 50, 35, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1940(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 21, 22, 23, 8, 15, 16, 1, 18, 3, 20, 19, 4, 5, 6, 7, 24, 63, 32, 49, 34, 51, 36, 35, 52, 53, 54, 55, 40, 47, 48, 33, 50, 35, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1941(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 21, 22, 23, 8, 31, 16, 1, 18, 3, 20, 19, 4, 5, 6, 7, 24, 47, 32, 49, 34, 51, 36, 35, 52, 53, 54, 55, 40, 63, 48, 33, 50, 35, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1942(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 21, 22, 23, 8, 15, 0, 1, 18, 3, 20, 19, 4, 5, 6, 7, 24, 63, 48, 49, 34, 51, 36, 35, 52, 53, 54, 55, 40, 47, 32, 33, 50, 35, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1943(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 21, 22, 23, 8, 31, 0, 1, 18, 3, 20, 19, 4, 5, 6, 7, 24, 47, 48, 49, 34, 51, 36, 35, 52, 53, 54, 55, 40, 63, 32, 33, 50, 35, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1944(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 21, 22, 23, 8, 15, 16, 17, 2, 3, 20, 19, 4, 5, 6, 7, 24, 63, 32, 33, 50, 51, 36, 35, 52, 53, 54, 55, 40, 47, 48, 49, 34, 35, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1945(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 21, 22, 23, 8, 31, 16, 17, 2, 3, 20, 19, 4, 5, 6, 7, 24, 47, 32, 33, 50, 51, 36, 35, 52, 53, 54, 55, 40, 63, 48, 49, 34, 35, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1946(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 21, 22, 23, 8, 15, 0, 17, 2, 3, 20, 19, 4, 5, 6, 7, 24, 63, 48, 33, 50, 51, 36, 35, 52, 53, 54, 55, 40, 47, 32, 49, 34, 35, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1947(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 21, 22, 23, 8, 31, 0, 17, 2, 3, 20, 19, 4, 5, 6, 7, 24, 47, 48, 33, 50, 51, 36, 35, 52, 53, 54, 55, 40, 63, 32, 49, 34, 35, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1948(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 21, 22, 23, 8, 15, 16, 1, 2, 3, 20, 19, 4, 5, 6, 7, 24, 63, 32, 49, 50, 51, 36, 35, 52, 53, 54, 55, 40, 47, 48, 33, 34, 35, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1949(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 21, 22, 23, 8, 31, 16, 1, 2, 3, 20, 19, 4, 5, 6, 7, 24, 47, 32, 49, 50, 51, 36, 35, 52, 53, 54, 55, 40, 63, 48, 33, 34, 35, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1950(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 21, 22, 23, 8, 15, 0, 1, 2, 3, 20, 19, 4, 5, 6, 7, 24, 63, 48, 49, 50, 51, 36, 35, 52, 53, 54, 55, 40, 47, 32, 33, 34, 35, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1951(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 21, 22, 23, 8, 31, 0, 1, 2, 3, 20, 19, 4, 5, 6, 7, 24, 47, 48, 49, 50, 51, 36, 35, 52, 53, 54, 55, 40, 63, 32, 33, 34, 35, 52, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1952(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 21, 22, 23, 8, 15, 16, 17, 18, 19, 4, 19, 4, 5, 6, 7, 24, 63, 32, 33, 34, 35, 52, 35, 52, 53, 54, 55, 40, 47, 48, 49, 50, 51, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1953(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 21, 22, 23, 8, 31, 16, 17, 18, 19, 4, 19, 4, 5, 6, 7, 24, 47, 32, 33, 34, 35, 52, 35, 52, 53, 54, 55, 40, 63, 48, 49, 50, 51, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1954(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 21, 22, 23, 8, 15, 0, 17, 18, 19, 4, 19, 4, 5, 6, 7, 24, 63, 48, 33, 34, 35, 52, 35, 52, 53, 54, 55, 40, 47, 32, 49, 50, 51, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1955(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 21, 22, 23, 8, 31, 0, 17, 18, 19, 4, 19, 4, 5, 6, 7, 24, 47, 48, 33, 34, 35, 52, 35, 52, 53, 54, 55, 40, 63, 32, 49, 50, 51, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1956(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 21, 22, 23, 8, 15, 16, 1, 18, 19, 4, 19, 4, 5, 6, 7, 24, 63, 32, 49, 34, 35, 52, 35, 52, 53, 54, 55, 40, 47, 48, 33, 50, 51, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1957(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 21, 22, 23, 8, 31, 16, 1, 18, 19, 4, 19, 4, 5, 6, 7, 24, 47, 32, 49, 34, 35, 52, 35, 52, 53, 54, 55, 40, 63, 48, 33, 50, 51, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1958(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 21, 22, 23, 8, 15, 0, 1, 18, 19, 4, 19, 4, 5, 6, 7, 24, 63, 48, 49, 34, 35, 52, 35, 52, 53, 54, 55, 40, 47, 32, 33, 50, 51, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1959(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 21, 22, 23, 8, 31, 0, 1, 18, 19, 4, 19, 4, 5, 6, 7, 24, 47, 48, 49, 34, 35, 52, 35, 52, 53, 54, 55, 40, 63, 32, 33, 50, 51, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1960(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 21, 22, 23, 8, 15, 16, 17, 2, 19, 4, 19, 4, 5, 6, 7, 24, 63, 32, 33, 50, 35, 52, 35, 52, 53, 54, 55, 40, 47, 48, 49, 34, 51, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1961(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 21, 22, 23, 8, 31, 16, 17, 2, 19, 4, 19, 4, 5, 6, 7, 24, 47, 32, 33, 50, 35, 52, 35, 52, 53, 54, 55, 40, 63, 48, 49, 34, 51, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1962(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 21, 22, 23, 8, 15, 0, 17, 2, 19, 4, 19, 4, 5, 6, 7, 24, 63, 48, 33, 50, 35, 52, 35, 52, 53, 54, 55, 40, 47, 32, 49, 34, 51, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1963(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 21, 22, 23, 8, 31, 0, 17, 2, 19, 4, 19, 4, 5, 6, 7, 24, 47, 48, 33, 50, 35, 52, 35, 52, 53, 54, 55, 40, 63, 32, 49, 34, 51, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1964(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 21, 22, 23, 8, 15, 16, 1, 2, 19, 4, 19, 4, 5, 6, 7, 24, 63, 32, 49, 50, 35, 52, 35, 52, 53, 54, 55, 40, 47, 48, 33, 34, 51, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1965(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 21, 22, 23, 8, 31, 16, 1, 2, 19, 4, 19, 4, 5, 6, 7, 24, 47, 32, 49, 50, 35, 52, 35, 52, 53, 54, 55, 40, 63, 48, 33, 34, 51, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1966(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 21, 22, 23, 8, 15, 0, 1, 2, 19, 4, 19, 4, 5, 6, 7, 24, 63, 48, 49, 50, 35, 52, 35, 52, 53, 54, 55, 40, 47, 32, 33, 34, 51, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1967(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 21, 22, 23, 8, 31, 0, 1, 2, 19, 4, 19, 4, 5, 6, 7, 24, 47, 48, 49, 50, 35, 52, 35, 52, 53, 54, 55, 40, 63, 32, 33, 34, 51, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1968(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 21, 22, 23, 8, 15, 16, 17, 18, 3, 4, 19, 4, 5, 6, 7, 24, 63, 32, 33, 34, 51, 52, 35, 52, 53, 54, 55, 40, 47, 48, 49, 50, 35, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1969(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 21, 22, 23, 8, 31, 16, 17, 18, 3, 4, 19, 4, 5, 6, 7, 24, 47, 32, 33, 34, 51, 52, 35, 52, 53, 54, 55, 40, 63, 48, 49, 50, 35, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1970(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 21, 22, 23, 8, 15, 0, 17, 18, 3, 4, 19, 4, 5, 6, 7, 24, 63, 48, 33, 34, 51, 52, 35, 52, 53, 54, 55, 40, 47, 32, 49, 50, 35, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1971(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 21, 22, 23, 8, 31, 0, 17, 18, 3, 4, 19, 4, 5, 6, 7, 24, 47, 48, 33, 34, 51, 52, 35, 52, 53, 54, 55, 40, 63, 32, 49, 50, 35, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1972(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 21, 22, 23, 8, 15, 16, 1, 18, 3, 4, 19, 4, 5, 6, 7, 24, 63, 32, 49, 34, 51, 52, 35, 52, 53, 54, 55, 40, 47, 48, 33, 50, 35, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1973(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 21, 22, 23, 8, 31, 16, 1, 18, 3, 4, 19, 4, 5, 6, 7, 24, 47, 32, 49, 34, 51, 52, 35, 52, 53, 54, 55, 40, 63, 48, 33, 50, 35, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1974(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 21, 22, 23, 8, 15, 0, 1, 18, 3, 4, 19, 4, 5, 6, 7, 24, 63, 48, 49, 34, 51, 52, 35, 52, 53, 54, 55, 40, 47, 32, 33, 50, 35, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1975(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 21, 22, 23, 8, 31, 0, 1, 18, 3, 4, 19, 4, 5, 6, 7, 24, 47, 48, 49, 34, 51, 52, 35, 52, 53, 54, 55, 40, 63, 32, 33, 50, 35, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1976(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 21, 22, 23, 8, 15, 16, 17, 2, 3, 4, 19, 4, 5, 6, 7, 24, 63, 32, 33, 50, 51, 52, 35, 52, 53, 54, 55, 40, 47, 48, 49, 34, 35, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1977(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 21, 22, 23, 8, 31, 16, 17, 2, 3, 4, 19, 4, 5, 6, 7, 24, 47, 32, 33, 50, 51, 52, 35, 52, 53, 54, 55, 40, 63, 48, 49, 34, 35, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1978(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 21, 22, 23, 8, 15, 0, 17, 2, 3, 4, 19, 4, 5, 6, 7, 24, 63, 48, 33, 50, 51, 52, 35, 52, 53, 54, 55, 40, 47, 32, 49, 34, 35, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1979(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 21, 22, 23, 8, 31, 0, 17, 2, 3, 4, 19, 4, 5, 6, 7, 24, 47, 48, 33, 50, 51, 52, 35, 52, 53, 54, 55, 40, 63, 32, 49, 34, 35, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1980(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 21, 22, 23, 8, 15, 16, 1, 2, 3, 4, 19, 4, 5, 6, 7, 24, 63, 32, 49, 50, 51, 52, 35, 52, 53, 54, 55, 40, 47, 48, 33, 34, 35, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1981(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 21, 22, 23, 8, 31, 16, 1, 2, 3, 4, 19, 4, 5, 6, 7, 24, 47, 32, 49, 50, 51, 52, 35, 52, 53, 54, 55, 40, 63, 48, 33, 34, 35, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1982(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 21, 22, 23, 8, 15, 0, 1, 2, 3, 4, 19, 4, 5, 6, 7, 24, 63, 48, 49, 50, 51, 52, 35, 52, 53, 54, 55, 40, 47, 32, 33, 34, 35, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1983(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 21, 22, 23, 8, 31, 0, 1, 2, 3, 4, 19, 4, 5, 6, 7, 24, 47, 48, 49, 50, 51, 52, 35, 52, 53, 54, 55, 40, 63, 32, 33, 34, 35, 36, 51, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1984(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 21, 22, 23, 8, 15, 16, 17, 18, 19, 20, 3, 4, 5, 6, 7, 24, 63, 32, 33, 34, 35, 36, 51, 52, 53, 54, 55, 40, 47, 48, 49, 50, 51, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1985(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 21, 22, 23, 8, 31, 16, 17, 18, 19, 20, 3, 4, 5, 6, 7, 24, 47, 32, 33, 34, 35, 36, 51, 52, 53, 54, 55, 40, 63, 48, 49, 50, 51, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1986(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 21, 22, 23, 8, 15, 0, 17, 18, 19, 20, 3, 4, 5, 6, 7, 24, 63, 48, 33, 34, 35, 36, 51, 52, 53, 54, 55, 40, 47, 32, 49, 50, 51, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1987(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 21, 22, 23, 8, 31, 0, 17, 18, 19, 20, 3, 4, 5, 6, 7, 24, 47, 48, 33, 34, 35, 36, 51, 52, 53, 54, 55, 40, 63, 32, 49, 50, 51, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1988(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 21, 22, 23, 8, 15, 16, 1, 18, 19, 20, 3, 4, 5, 6, 7, 24, 63, 32, 49, 34, 35, 36, 51, 52, 53, 54, 55, 40, 47, 48, 33, 50, 51, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1989(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 21, 22, 23, 8, 31, 16, 1, 18, 19, 20, 3, 4, 5, 6, 7, 24, 47, 32, 49, 34, 35, 36, 51, 52, 53, 54, 55, 40, 63, 48, 33, 50, 51, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1990(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 21, 22, 23, 8, 15, 0, 1, 18, 19, 20, 3, 4, 5, 6, 7, 24, 63, 48, 49, 34, 35, 36, 51, 52, 53, 54, 55, 40, 47, 32, 33, 50, 51, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1991(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 21, 22, 23, 8, 31, 0, 1, 18, 19, 20, 3, 4, 5, 6, 7, 24, 47, 48, 49, 34, 35, 36, 51, 52, 53, 54, 55, 40, 63, 32, 33, 50, 51, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1992(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 21, 22, 23, 8, 15, 16, 17, 2, 19, 20, 3, 4, 5, 6, 7, 24, 63, 32, 33, 50, 35, 36, 51, 52, 53, 54, 55, 40, 47, 48, 49, 34, 51, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1993(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 21, 22, 23, 8, 31, 16, 17, 2, 19, 20, 3, 4, 5, 6, 7, 24, 47, 32, 33, 50, 35, 36, 51, 52, 53, 54, 55, 40, 63, 48, 49, 34, 51, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1994(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 21, 22, 23, 8, 15, 0, 17, 2, 19, 20, 3, 4, 5, 6, 7, 24, 63, 48, 33, 50, 35, 36, 51, 52, 53, 54, 55, 40, 47, 32, 49, 34, 51, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1995(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 21, 22, 23, 8, 31, 0, 17, 2, 19, 20, 3, 4, 5, 6, 7, 24, 47, 48, 33, 50, 35, 36, 51, 52, 53, 54, 55, 40, 63, 32, 49, 34, 51, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1996(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 21, 22, 23, 8, 15, 16, 1, 2, 19, 20, 3, 4, 5, 6, 7, 24, 63, 32, 49, 50, 35, 36, 51, 52, 53, 54, 55, 40, 47, 48, 33, 34, 51, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1997(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 21, 22, 23, 8, 31, 16, 1, 2, 19, 20, 3, 4, 5, 6, 7, 24, 47, 32, 49, 50, 35, 36, 51, 52, 53, 54, 55, 40, 63, 48, 33, 34, 51, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1998(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 21, 22, 23, 8, 15, 0, 1, 2, 19, 20, 3, 4, 5, 6, 7, 24, 63, 48, 49, 50, 35, 36, 51, 52, 53, 54, 55, 40, 47, 32, 33, 34, 51, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1999(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 21, 22, 23, 8, 31, 0, 1, 2, 19, 20, 3, 4, 5, 6, 7, 24, 47, 48, 49, 50, 35, 36, 51, 52, 53, 54, 55, 40, 63, 32, 33, 34, 51, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2000(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 21, 22, 23, 8, 15, 16, 17, 18, 3, 20, 3, 4, 5, 6, 7, 24, 63, 32, 33, 34, 51, 36, 51, 52, 53, 54, 55, 40, 47, 48, 49, 50, 35, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2001(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 21, 22, 23, 8, 31, 16, 17, 18, 3, 20, 3, 4, 5, 6, 7, 24, 47, 32, 33, 34, 51, 36, 51, 52, 53, 54, 55, 40, 63, 48, 49, 50, 35, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2002(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 21, 22, 23, 8, 15, 0, 17, 18, 3, 20, 3, 4, 5, 6, 7, 24, 63, 48, 33, 34, 51, 36, 51, 52, 53, 54, 55, 40, 47, 32, 49, 50, 35, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2003(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 21, 22, 23, 8, 31, 0, 17, 18, 3, 20, 3, 4, 5, 6, 7, 24, 47, 48, 33, 34, 51, 36, 51, 52, 53, 54, 55, 40, 63, 32, 49, 50, 35, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2004(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 21, 22, 23, 8, 15, 16, 1, 18, 3, 20, 3, 4, 5, 6, 7, 24, 63, 32, 49, 34, 51, 36, 51, 52, 53, 54, 55, 40, 47, 48, 33, 50, 35, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2005(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 21, 22, 23, 8, 31, 16, 1, 18, 3, 20, 3, 4, 5, 6, 7, 24, 47, 32, 49, 34, 51, 36, 51, 52, 53, 54, 55, 40, 63, 48, 33, 50, 35, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2006(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 21, 22, 23, 8, 15, 0, 1, 18, 3, 20, 3, 4, 5, 6, 7, 24, 63, 48, 49, 34, 51, 36, 51, 52, 53, 54, 55, 40, 47, 32, 33, 50, 35, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2007(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 21, 22, 23, 8, 31, 0, 1, 18, 3, 20, 3, 4, 5, 6, 7, 24, 47, 48, 49, 34, 51, 36, 51, 52, 53, 54, 55, 40, 63, 32, 33, 50, 35, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2008(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 21, 22, 23, 8, 15, 16, 17, 2, 3, 20, 3, 4, 5, 6, 7, 24, 63, 32, 33, 50, 51, 36, 51, 52, 53, 54, 55, 40, 47, 48, 49, 34, 35, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2009(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 21, 22, 23, 8, 31, 16, 17, 2, 3, 20, 3, 4, 5, 6, 7, 24, 47, 32, 33, 50, 51, 36, 51, 52, 53, 54, 55, 40, 63, 48, 49, 34, 35, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2010(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 21, 22, 23, 8, 15, 0, 17, 2, 3, 20, 3, 4, 5, 6, 7, 24, 63, 48, 33, 50, 51, 36, 51, 52, 53, 54, 55, 40, 47, 32, 49, 34, 35, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2011(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 21, 22, 23, 8, 31, 0, 17, 2, 3, 20, 3, 4, 5, 6, 7, 24, 47, 48, 33, 50, 51, 36, 51, 52, 53, 54, 55, 40, 63, 32, 49, 34, 35, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2012(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 21, 22, 23, 8, 15, 16, 1, 2, 3, 20, 3, 4, 5, 6, 7, 24, 63, 32, 49, 50, 51, 36, 51, 52, 53, 54, 55, 40, 47, 48, 33, 34, 35, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2013(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 21, 22, 23, 8, 31, 16, 1, 2, 3, 20, 3, 4, 5, 6, 7, 24, 47, 32, 49, 50, 51, 36, 51, 52, 53, 54, 55, 40, 63, 48, 33, 34, 35, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2014(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 21, 22, 23, 8, 15, 0, 1, 2, 3, 20, 3, 4, 5, 6, 7, 24, 63, 48, 49, 50, 51, 36, 51, 52, 53, 54, 55, 40, 47, 32, 33, 34, 35, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2015(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 21, 22, 23, 8, 31, 0, 1, 2, 3, 20, 3, 4, 5, 6, 7, 24, 47, 48, 49, 50, 51, 36, 51, 52, 53, 54, 55, 40, 63, 32, 33, 34, 35, 52, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2016(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 21, 22, 23, 8, 15, 16, 17, 18, 19, 4, 3, 4, 5, 6, 7, 24, 63, 32, 33, 34, 35, 52, 51, 52, 53, 54, 55, 40, 47, 48, 49, 50, 51, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2017(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 21, 22, 23, 8, 31, 16, 17, 18, 19, 4, 3, 4, 5, 6, 7, 24, 47, 32, 33, 34, 35, 52, 51, 52, 53, 54, 55, 40, 63, 48, 49, 50, 51, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2018(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 21, 22, 23, 8, 15, 0, 17, 18, 19, 4, 3, 4, 5, 6, 7, 24, 63, 48, 33, 34, 35, 52, 51, 52, 53, 54, 55, 40, 47, 32, 49, 50, 51, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2019(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 21, 22, 23, 8, 31, 0, 17, 18, 19, 4, 3, 4, 5, 6, 7, 24, 47, 48, 33, 34, 35, 52, 51, 52, 53, 54, 55, 40, 63, 32, 49, 50, 51, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2020(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 21, 22, 23, 8, 15, 16, 1, 18, 19, 4, 3, 4, 5, 6, 7, 24, 63, 32, 49, 34, 35, 52, 51, 52, 53, 54, 55, 40, 47, 48, 33, 50, 51, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2021(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 21, 22, 23, 8, 31, 16, 1, 18, 19, 4, 3, 4, 5, 6, 7, 24, 47, 32, 49, 34, 35, 52, 51, 52, 53, 54, 55, 40, 63, 48, 33, 50, 51, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2022(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 21, 22, 23, 8, 15, 0, 1, 18, 19, 4, 3, 4, 5, 6, 7, 24, 63, 48, 49, 34, 35, 52, 51, 52, 53, 54, 55, 40, 47, 32, 33, 50, 51, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2023(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 21, 22, 23, 8, 31, 0, 1, 18, 19, 4, 3, 4, 5, 6, 7, 24, 47, 48, 49, 34, 35, 52, 51, 52, 53, 54, 55, 40, 63, 32, 33, 50, 51, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2024(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 21, 22, 23, 8, 15, 16, 17, 2, 19, 4, 3, 4, 5, 6, 7, 24, 63, 32, 33, 50, 35, 52, 51, 52, 53, 54, 55, 40, 47, 48, 49, 34, 51, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2025(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 21, 22, 23, 8, 31, 16, 17, 2, 19, 4, 3, 4, 5, 6, 7, 24, 47, 32, 33, 50, 35, 52, 51, 52, 53, 54, 55, 40, 63, 48, 49, 34, 51, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2026(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 21, 22, 23, 8, 15, 0, 17, 2, 19, 4, 3, 4, 5, 6, 7, 24, 63, 48, 33, 50, 35, 52, 51, 52, 53, 54, 55, 40, 47, 32, 49, 34, 51, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2027(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 21, 22, 23, 8, 31, 0, 17, 2, 19, 4, 3, 4, 5, 6, 7, 24, 47, 48, 33, 50, 35, 52, 51, 52, 53, 54, 55, 40, 63, 32, 49, 34, 51, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2028(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 21, 22, 23, 8, 15, 16, 1, 2, 19, 4, 3, 4, 5, 6, 7, 24, 63, 32, 49, 50, 35, 52, 51, 52, 53, 54, 55, 40, 47, 48, 33, 34, 51, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2029(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 21, 22, 23, 8, 31, 16, 1, 2, 19, 4, 3, 4, 5, 6, 7, 24, 47, 32, 49, 50, 35, 52, 51, 52, 53, 54, 55, 40, 63, 48, 33, 34, 51, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2030(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 21, 22, 23, 8, 15, 0, 1, 2, 19, 4, 3, 4, 5, 6, 7, 24, 63, 48, 49, 50, 35, 52, 51, 52, 53, 54, 55, 40, 47, 32, 33, 34, 51, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2031(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 21, 22, 23, 8, 31, 0, 1, 2, 19, 4, 3, 4, 5, 6, 7, 24, 47, 48, 49, 50, 35, 52, 51, 52, 53, 54, 55, 40, 63, 32, 33, 34, 51, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2032(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 21, 22, 23, 8, 15, 16, 17, 18, 3, 4, 3, 4, 5, 6, 7, 24, 63, 32, 33, 34, 51, 52, 51, 52, 53, 54, 55, 40, 47, 48, 49, 50, 35, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2033(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 21, 22, 23, 8, 31, 16, 17, 18, 3, 4, 3, 4, 5, 6, 7, 24, 47, 32, 33, 34, 51, 52, 51, 52, 53, 54, 55, 40, 63, 48, 49, 50, 35, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2034(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 21, 22, 23, 8, 15, 0, 17, 18, 3, 4, 3, 4, 5, 6, 7, 24, 63, 48, 33, 34, 51, 52, 51, 52, 53, 54, 55, 40, 47, 32, 49, 50, 35, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2035(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 21, 22, 23, 8, 31, 0, 17, 18, 3, 4, 3, 4, 5, 6, 7, 24, 47, 48, 33, 34, 51, 52, 51, 52, 53, 54, 55, 40, 63, 32, 49, 50, 35, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2036(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 21, 22, 23, 8, 15, 16, 1, 18, 3, 4, 3, 4, 5, 6, 7, 24, 63, 32, 49, 34, 51, 52, 51, 52, 53, 54, 55, 40, 47, 48, 33, 50, 35, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2037(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 21, 22, 23, 8, 31, 16, 1, 18, 3, 4, 3, 4, 5, 6, 7, 24, 47, 32, 49, 34, 51, 52, 51, 52, 53, 54, 55, 40, 63, 48, 33, 50, 35, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2038(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 21, 22, 23, 8, 15, 0, 1, 18, 3, 4, 3, 4, 5, 6, 7, 24, 63, 48, 49, 34, 51, 52, 51, 52, 53, 54, 55, 40, 47, 32, 33, 50, 35, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2039(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 21, 22, 23, 8, 31, 0, 1, 18, 3, 4, 3, 4, 5, 6, 7, 24, 47, 48, 49, 34, 51, 52, 51, 52, 53, 54, 55, 40, 63, 32, 33, 50, 35, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2040(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 21, 22, 23, 8, 15, 16, 17, 2, 3, 4, 3, 4, 5, 6, 7, 24, 63, 32, 33, 50, 51, 52, 51, 52, 53, 54, 55, 40, 47, 48, 49, 34, 35, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2041(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 21, 22, 23, 8, 31, 16, 17, 2, 3, 4, 3, 4, 5, 6, 7, 24, 47, 32, 33, 50, 51, 52, 51, 52, 53, 54, 55, 40, 63, 48, 49, 34, 35, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2042(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 21, 22, 23, 8, 15, 0, 17, 2, 3, 4, 3, 4, 5, 6, 7, 24, 63, 48, 33, 50, 51, 52, 51, 52, 53, 54, 55, 40, 47, 32, 49, 34, 35, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2043(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 21, 22, 23, 8, 31, 0, 17, 2, 3, 4, 3, 4, 5, 6, 7, 24, 47, 48, 33, 50, 51, 52, 51, 52, 53, 54, 55, 40, 63, 32, 49, 34, 35, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2044(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 21, 22, 23, 8, 15, 16, 1, 2, 3, 4, 3, 4, 5, 6, 7, 24, 63, 32, 49, 50, 51, 52, 51, 52, 53, 54, 55, 40, 47, 48, 33, 34, 35, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2045(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 21, 22, 23, 8, 31, 16, 1, 2, 3, 4, 3, 4, 5, 6, 7, 24, 47, 32, 49, 50, 51, 52, 51, 52, 53, 54, 55, 40, 63, 48, 33, 34, 35, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2046(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 21, 22, 23, 8, 15, 0, 1, 2, 3, 4, 3, 4, 5, 6, 7, 24, 63, 48, 49, 50, 51, 52, 51, 52, 53, 54, 55, 40, 47, 32, 33, 34, 35, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2047(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 21, 22, 23, 8, 31, 0, 1, 2, 3, 4, 3, 4, 5, 6, 7, 24, 47, 48, 49, 50, 51, 52, 51, 52, 53, 54, 55, 40, 63, 32, 33, 34, 35, 36, 35, 36, 37, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

#endif // DESGPU_COMPILE_ALL_SALTS


