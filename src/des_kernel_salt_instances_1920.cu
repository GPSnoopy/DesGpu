#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt1920(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 21, 22, 23, 8, 15, 16, 17, 18, 19, 20, 19, 4, 5, 6, 7, 24, 63, 32, 33, 34, 35, 36, 35, 52, 53, 54, 55, 40, 47, 48, 49, 50, 51, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1921(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 21, 22, 23, 8, 31, 16, 17, 18, 19, 20, 19, 4, 5, 6, 7, 24, 47, 32, 33, 34, 35, 36, 35, 52, 53, 54, 55, 40, 63, 48, 49, 50, 51, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1922(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 21, 22, 23, 8, 15, 0, 17, 18, 19, 20, 19, 4, 5, 6, 7, 24, 63, 48, 33, 34, 35, 36, 35, 52, 53, 54, 55, 40, 47, 32, 49, 50, 51, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1923(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 21, 22, 23, 8, 31, 0, 17, 18, 19, 20, 19, 4, 5, 6, 7, 24, 47, 48, 33, 34, 35, 36, 35, 52, 53, 54, 55, 40, 63, 32, 49, 50, 51, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1924(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 21, 22, 23, 8, 15, 16, 1, 18, 19, 20, 19, 4, 5, 6, 7, 24, 63, 32, 49, 34, 35, 36, 35, 52, 53, 54, 55, 40, 47, 48, 33, 50, 51, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1925(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 21, 22, 23, 8, 31, 16, 1, 18, 19, 20, 19, 4, 5, 6, 7, 24, 47, 32, 49, 34, 35, 36, 35, 52, 53, 54, 55, 40, 63, 48, 33, 50, 51, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1926(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 21, 22, 23, 8, 15, 0, 1, 18, 19, 20, 19, 4, 5, 6, 7, 24, 63, 48, 49, 34, 35, 36, 35, 52, 53, 54, 55, 40, 47, 32, 33, 50, 51, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1927(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 21, 22, 23, 8, 31, 0, 1, 18, 19, 20, 19, 4, 5, 6, 7, 24, 47, 48, 49, 34, 35, 36, 35, 52, 53, 54, 55, 40, 63, 32, 33, 50, 51, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1928(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 21, 22, 23, 8, 15, 16, 17, 2, 19, 20, 19, 4, 5, 6, 7, 24, 63, 32, 33, 50, 35, 36, 35, 52, 53, 54, 55, 40, 47, 48, 49, 34, 51, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1929(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 21, 22, 23, 8, 31, 16, 17, 2, 19, 20, 19, 4, 5, 6, 7, 24, 47, 32, 33, 50, 35, 36, 35, 52, 53, 54, 55, 40, 63, 48, 49, 34, 51, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1930(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 21, 22, 23, 8, 15, 0, 17, 2, 19, 20, 19, 4, 5, 6, 7, 24, 63, 48, 33, 50, 35, 36, 35, 52, 53, 54, 55, 40, 47, 32, 49, 34, 51, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1931(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 21, 22, 23, 8, 31, 0, 17, 2, 19, 20, 19, 4, 5, 6, 7, 24, 47, 48, 33, 50, 35, 36, 35, 52, 53, 54, 55, 40, 63, 32, 49, 34, 51, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1932(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 21, 22, 23, 8, 15, 16, 1, 2, 19, 20, 19, 4, 5, 6, 7, 24, 63, 32, 49, 50, 35, 36, 35, 52, 53, 54, 55, 40, 47, 48, 33, 34, 51, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1933(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 21, 22, 23, 8, 31, 16, 1, 2, 19, 20, 19, 4, 5, 6, 7, 24, 47, 32, 49, 50, 35, 36, 35, 52, 53, 54, 55, 40, 63, 48, 33, 34, 51, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1934(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 21, 22, 23, 8, 15, 0, 1, 2, 19, 20, 19, 4, 5, 6, 7, 24, 63, 48, 49, 50, 35, 36, 35, 52, 53, 54, 55, 40, 47, 32, 33, 34, 51, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1935(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 21, 22, 23, 8, 31, 0, 1, 2, 19, 20, 19, 4, 5, 6, 7, 24, 47, 48, 49, 50, 35, 36, 35, 52, 53, 54, 55, 40, 63, 32, 33, 34, 51, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1936(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 21, 22, 23, 8, 15, 16, 17, 18, 3, 20, 19, 4, 5, 6, 7, 24, 63, 32, 33, 34, 51, 36, 35, 52, 53, 54, 55, 40, 47, 48, 49, 50, 35, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1937(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 21, 22, 23, 8, 31, 16, 17, 18, 3, 20, 19, 4, 5, 6, 7, 24, 47, 32, 33, 34, 51, 36, 35, 52, 53, 54, 55, 40, 63, 48, 49, 50, 35, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1938(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 21, 22, 23, 8, 15, 0, 17, 18, 3, 20, 19, 4, 5, 6, 7, 24, 63, 48, 33, 34, 51, 36, 35, 52, 53, 54, 55, 40, 47, 32, 49, 50, 35, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1939(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 21, 22, 23, 8, 31, 0, 17, 18, 3, 20, 19, 4, 5, 6, 7, 24, 47, 48, 33, 34, 51, 36, 35, 52, 53, 54, 55, 40, 63, 32, 49, 50, 35, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1940(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 21, 22, 23, 8, 15, 16, 1, 18, 3, 20, 19, 4, 5, 6, 7, 24, 63, 32, 49, 34, 51, 36, 35, 52, 53, 54, 55, 40, 47, 48, 33, 50, 35, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1941(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 21, 22, 23, 8, 31, 16, 1, 18, 3, 20, 19, 4, 5, 6, 7, 24, 47, 32, 49, 34, 51, 36, 35, 52, 53, 54, 55, 40, 63, 48, 33, 50, 35, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1942(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 21, 22, 23, 8, 15, 0, 1, 18, 3, 20, 19, 4, 5, 6, 7, 24, 63, 48, 49, 34, 51, 36, 35, 52, 53, 54, 55, 40, 47, 32, 33, 50, 35, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1943(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 21, 22, 23, 8, 31, 0, 1, 18, 3, 20, 19, 4, 5, 6, 7, 24, 47, 48, 49, 34, 51, 36, 35, 52, 53, 54, 55, 40, 63, 32, 33, 50, 35, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1944(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 21, 22, 23, 8, 15, 16, 17, 2, 3, 20, 19, 4, 5, 6, 7, 24, 63, 32, 33, 50, 51, 36, 35, 52, 53, 54, 55, 40, 47, 48, 49, 34, 35, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1945(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 21, 22, 23, 8, 31, 16, 17, 2, 3, 20, 19, 4, 5, 6, 7, 24, 47, 32, 33, 50, 51, 36, 35, 52, 53, 54, 55, 40, 63, 48, 49, 34, 35, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1946(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 21, 22, 23, 8, 15, 0, 17, 2, 3, 20, 19, 4, 5, 6, 7, 24, 63, 48, 33, 50, 51, 36, 35, 52, 53, 54, 55, 40, 47, 32, 49, 34, 35, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1947(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 21, 22, 23, 8, 31, 0, 17, 2, 3, 20, 19, 4, 5, 6, 7, 24, 47, 48, 33, 50, 51, 36, 35, 52, 53, 54, 55, 40, 63, 32, 49, 34, 35, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1948(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 21, 22, 23, 8, 15, 16, 1, 2, 3, 20, 19, 4, 5, 6, 7, 24, 63, 32, 49, 50, 51, 36, 35, 52, 53, 54, 55, 40, 47, 48, 33, 34, 35, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1949(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 21, 22, 23, 8, 31, 16, 1, 2, 3, 20, 19, 4, 5, 6, 7, 24, 47, 32, 49, 50, 51, 36, 35, 52, 53, 54, 55, 40, 63, 48, 33, 34, 35, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1950(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 21, 22, 23, 8, 15, 0, 1, 2, 3, 20, 19, 4, 5, 6, 7, 24, 63, 48, 49, 50, 51, 36, 35, 52, 53, 54, 55, 40, 47, 32, 33, 34, 35, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1951(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 21, 22, 23, 8, 31, 0, 1, 2, 3, 20, 19, 4, 5, 6, 7, 24, 47, 48, 49, 50, 51, 36, 35, 52, 53, 54, 55, 40, 63, 32, 33, 34, 35, 52, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1952(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 21, 22, 23, 8, 15, 16, 17, 18, 19, 4, 19, 4, 5, 6, 7, 24, 63, 32, 33, 34, 35, 52, 35, 52, 53, 54, 55, 40, 47, 48, 49, 50, 51, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1953(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 21, 22, 23, 8, 31, 16, 17, 18, 19, 4, 19, 4, 5, 6, 7, 24, 47, 32, 33, 34, 35, 52, 35, 52, 53, 54, 55, 40, 63, 48, 49, 50, 51, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1954(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 21, 22, 23, 8, 15, 0, 17, 18, 19, 4, 19, 4, 5, 6, 7, 24, 63, 48, 33, 34, 35, 52, 35, 52, 53, 54, 55, 40, 47, 32, 49, 50, 51, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1955(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 21, 22, 23, 8, 31, 0, 17, 18, 19, 4, 19, 4, 5, 6, 7, 24, 47, 48, 33, 34, 35, 52, 35, 52, 53, 54, 55, 40, 63, 32, 49, 50, 51, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1956(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 21, 22, 23, 8, 15, 16, 1, 18, 19, 4, 19, 4, 5, 6, 7, 24, 63, 32, 49, 34, 35, 52, 35, 52, 53, 54, 55, 40, 47, 48, 33, 50, 51, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1957(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 21, 22, 23, 8, 31, 16, 1, 18, 19, 4, 19, 4, 5, 6, 7, 24, 47, 32, 49, 34, 35, 52, 35, 52, 53, 54, 55, 40, 63, 48, 33, 50, 51, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1958(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 21, 22, 23, 8, 15, 0, 1, 18, 19, 4, 19, 4, 5, 6, 7, 24, 63, 48, 49, 34, 35, 52, 35, 52, 53, 54, 55, 40, 47, 32, 33, 50, 51, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1959(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 21, 22, 23, 8, 31, 0, 1, 18, 19, 4, 19, 4, 5, 6, 7, 24, 47, 48, 49, 34, 35, 52, 35, 52, 53, 54, 55, 40, 63, 32, 33, 50, 51, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1960(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 21, 22, 23, 8, 15, 16, 17, 2, 19, 4, 19, 4, 5, 6, 7, 24, 63, 32, 33, 50, 35, 52, 35, 52, 53, 54, 55, 40, 47, 48, 49, 34, 51, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1961(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 21, 22, 23, 8, 31, 16, 17, 2, 19, 4, 19, 4, 5, 6, 7, 24, 47, 32, 33, 50, 35, 52, 35, 52, 53, 54, 55, 40, 63, 48, 49, 34, 51, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1962(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 21, 22, 23, 8, 15, 0, 17, 2, 19, 4, 19, 4, 5, 6, 7, 24, 63, 48, 33, 50, 35, 52, 35, 52, 53, 54, 55, 40, 47, 32, 49, 34, 51, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1963(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 21, 22, 23, 8, 31, 0, 17, 2, 19, 4, 19, 4, 5, 6, 7, 24, 47, 48, 33, 50, 35, 52, 35, 52, 53, 54, 55, 40, 63, 32, 49, 34, 51, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1964(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 21, 22, 23, 8, 15, 16, 1, 2, 19, 4, 19, 4, 5, 6, 7, 24, 63, 32, 49, 50, 35, 52, 35, 52, 53, 54, 55, 40, 47, 48, 33, 34, 51, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1965(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 21, 22, 23, 8, 31, 16, 1, 2, 19, 4, 19, 4, 5, 6, 7, 24, 47, 32, 49, 50, 35, 52, 35, 52, 53, 54, 55, 40, 63, 48, 33, 34, 51, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1966(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 21, 22, 23, 8, 15, 0, 1, 2, 19, 4, 19, 4, 5, 6, 7, 24, 63, 48, 49, 50, 35, 52, 35, 52, 53, 54, 55, 40, 47, 32, 33, 34, 51, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1967(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 21, 22, 23, 8, 31, 0, 1, 2, 19, 4, 19, 4, 5, 6, 7, 24, 47, 48, 49, 50, 35, 52, 35, 52, 53, 54, 55, 40, 63, 32, 33, 34, 51, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1968(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 21, 22, 23, 8, 15, 16, 17, 18, 3, 4, 19, 4, 5, 6, 7, 24, 63, 32, 33, 34, 51, 52, 35, 52, 53, 54, 55, 40, 47, 48, 49, 50, 35, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1969(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 21, 22, 23, 8, 31, 16, 17, 18, 3, 4, 19, 4, 5, 6, 7, 24, 47, 32, 33, 34, 51, 52, 35, 52, 53, 54, 55, 40, 63, 48, 49, 50, 35, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1970(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 21, 22, 23, 8, 15, 0, 17, 18, 3, 4, 19, 4, 5, 6, 7, 24, 63, 48, 33, 34, 51, 52, 35, 52, 53, 54, 55, 40, 47, 32, 49, 50, 35, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1971(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 21, 22, 23, 8, 31, 0, 17, 18, 3, 4, 19, 4, 5, 6, 7, 24, 47, 48, 33, 34, 51, 52, 35, 52, 53, 54, 55, 40, 63, 32, 49, 50, 35, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1972(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 21, 22, 23, 8, 15, 16, 1, 18, 3, 4, 19, 4, 5, 6, 7, 24, 63, 32, 49, 34, 51, 52, 35, 52, 53, 54, 55, 40, 47, 48, 33, 50, 35, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1973(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 21, 22, 23, 8, 31, 16, 1, 18, 3, 4, 19, 4, 5, 6, 7, 24, 47, 32, 49, 34, 51, 52, 35, 52, 53, 54, 55, 40, 63, 48, 33, 50, 35, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1974(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 21, 22, 23, 8, 15, 0, 1, 18, 3, 4, 19, 4, 5, 6, 7, 24, 63, 48, 49, 34, 51, 52, 35, 52, 53, 54, 55, 40, 47, 32, 33, 50, 35, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1975(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 21, 22, 23, 8, 31, 0, 1, 18, 3, 4, 19, 4, 5, 6, 7, 24, 47, 48, 49, 34, 51, 52, 35, 52, 53, 54, 55, 40, 63, 32, 33, 50, 35, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1976(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 21, 22, 23, 8, 15, 16, 17, 2, 3, 4, 19, 4, 5, 6, 7, 24, 63, 32, 33, 50, 51, 52, 35, 52, 53, 54, 55, 40, 47, 48, 49, 34, 35, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1977(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 21, 22, 23, 8, 31, 16, 17, 2, 3, 4, 19, 4, 5, 6, 7, 24, 47, 32, 33, 50, 51, 52, 35, 52, 53, 54, 55, 40, 63, 48, 49, 34, 35, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1978(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 21, 22, 23, 8, 15, 0, 17, 2, 3, 4, 19, 4, 5, 6, 7, 24, 63, 48, 33, 50, 51, 52, 35, 52, 53, 54, 55, 40, 47, 32, 49, 34, 35, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1979(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 21, 22, 23, 8, 31, 0, 17, 2, 3, 4, 19, 4, 5, 6, 7, 24, 47, 48, 33, 50, 51, 52, 35, 52, 53, 54, 55, 40, 63, 32, 49, 34, 35, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1980(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 21, 22, 23, 8, 15, 16, 1, 2, 3, 4, 19, 4, 5, 6, 7, 24, 63, 32, 49, 50, 51, 52, 35, 52, 53, 54, 55, 40, 47, 48, 33, 34, 35, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1981(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 21, 22, 23, 8, 31, 16, 1, 2, 3, 4, 19, 4, 5, 6, 7, 24, 47, 32, 49, 50, 51, 52, 35, 52, 53, 54, 55, 40, 63, 48, 33, 34, 35, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1982(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 21, 22, 23, 8, 15, 0, 1, 2, 3, 4, 19, 4, 5, 6, 7, 24, 63, 48, 49, 50, 51, 52, 35, 52, 53, 54, 55, 40, 47, 32, 33, 34, 35, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1983(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 21, 22, 23, 8, 31, 0, 1, 2, 3, 4, 19, 4, 5, 6, 7, 24, 47, 48, 49, 50, 51, 52, 35, 52, 53, 54, 55, 40, 63, 32, 33, 34, 35, 36, 51, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1984(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 21, 22, 23, 8, 15, 16, 17, 18, 19, 20, 3, 4, 5, 6, 7, 24, 63, 32, 33, 34, 35, 36, 51, 52, 53, 54, 55, 40, 47, 48, 49, 50, 51, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1985(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 21, 22, 23, 8, 31, 16, 17, 18, 19, 20, 3, 4, 5, 6, 7, 24, 47, 32, 33, 34, 35, 36, 51, 52, 53, 54, 55, 40, 63, 48, 49, 50, 51, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1986(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 21, 22, 23, 8, 15, 0, 17, 18, 19, 20, 3, 4, 5, 6, 7, 24, 63, 48, 33, 34, 35, 36, 51, 52, 53, 54, 55, 40, 47, 32, 49, 50, 51, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1987(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 21, 22, 23, 8, 31, 0, 17, 18, 19, 20, 3, 4, 5, 6, 7, 24, 47, 48, 33, 34, 35, 36, 51, 52, 53, 54, 55, 40, 63, 32, 49, 50, 51, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1988(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 21, 22, 23, 8, 15, 16, 1, 18, 19, 20, 3, 4, 5, 6, 7, 24, 63, 32, 49, 34, 35, 36, 51, 52, 53, 54, 55, 40, 47, 48, 33, 50, 51, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1989(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 21, 22, 23, 8, 31, 16, 1, 18, 19, 20, 3, 4, 5, 6, 7, 24, 47, 32, 49, 34, 35, 36, 51, 52, 53, 54, 55, 40, 63, 48, 33, 50, 51, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1990(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 21, 22, 23, 8, 15, 0, 1, 18, 19, 20, 3, 4, 5, 6, 7, 24, 63, 48, 49, 34, 35, 36, 51, 52, 53, 54, 55, 40, 47, 32, 33, 50, 51, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1991(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 21, 22, 23, 8, 31, 0, 1, 18, 19, 20, 3, 4, 5, 6, 7, 24, 47, 48, 49, 34, 35, 36, 51, 52, 53, 54, 55, 40, 63, 32, 33, 50, 51, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1992(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 21, 22, 23, 8, 15, 16, 17, 2, 19, 20, 3, 4, 5, 6, 7, 24, 63, 32, 33, 50, 35, 36, 51, 52, 53, 54, 55, 40, 47, 48, 49, 34, 51, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1993(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 21, 22, 23, 8, 31, 16, 17, 2, 19, 20, 3, 4, 5, 6, 7, 24, 47, 32, 33, 50, 35, 36, 51, 52, 53, 54, 55, 40, 63, 48, 49, 34, 51, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1994(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 21, 22, 23, 8, 15, 0, 17, 2, 19, 20, 3, 4, 5, 6, 7, 24, 63, 48, 33, 50, 35, 36, 51, 52, 53, 54, 55, 40, 47, 32, 49, 34, 51, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1995(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 21, 22, 23, 8, 31, 0, 17, 2, 19, 20, 3, 4, 5, 6, 7, 24, 47, 48, 33, 50, 35, 36, 51, 52, 53, 54, 55, 40, 63, 32, 49, 34, 51, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1996(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 21, 22, 23, 8, 15, 16, 1, 2, 19, 20, 3, 4, 5, 6, 7, 24, 63, 32, 49, 50, 35, 36, 51, 52, 53, 54, 55, 40, 47, 48, 33, 34, 51, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1997(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 21, 22, 23, 8, 31, 16, 1, 2, 19, 20, 3, 4, 5, 6, 7, 24, 47, 32, 49, 50, 35, 36, 51, 52, 53, 54, 55, 40, 63, 48, 33, 34, 51, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1998(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 21, 22, 23, 8, 15, 0, 1, 2, 19, 20, 3, 4, 5, 6, 7, 24, 63, 48, 49, 50, 35, 36, 51, 52, 53, 54, 55, 40, 47, 32, 33, 34, 51, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1999(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 21, 22, 23, 8, 31, 0, 1, 2, 19, 20, 3, 4, 5, 6, 7, 24, 47, 48, 49, 50, 35, 36, 51, 52, 53, 54, 55, 40, 63, 32, 33, 34, 51, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2000(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 21, 22, 23, 8, 15, 16, 17, 18, 3, 20, 3, 4, 5, 6, 7, 24, 63, 32, 33, 34, 51, 36, 51, 52, 53, 54, 55, 40, 47, 48, 49, 50, 35, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2001(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 21, 22, 23, 8, 31, 16, 17, 18, 3, 20, 3, 4, 5, 6, 7, 24, 47, 32, 33, 34, 51, 36, 51, 52, 53, 54, 55, 40, 63, 48, 49, 50, 35, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2002(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 21, 22, 23, 8, 15, 0, 17, 18, 3, 20, 3, 4, 5, 6, 7, 24, 63, 48, 33, 34, 51, 36, 51, 52, 53, 54, 55, 40, 47, 32, 49, 50, 35, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2003(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 21, 22, 23, 8, 31, 0, 17, 18, 3, 20, 3, 4, 5, 6, 7, 24, 47, 48, 33, 34, 51, 36, 51, 52, 53, 54, 55, 40, 63, 32, 49, 50, 35, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2004(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 21, 22, 23, 8, 15, 16, 1, 18, 3, 20, 3, 4, 5, 6, 7, 24, 63, 32, 49, 34, 51, 36, 51, 52, 53, 54, 55, 40, 47, 48, 33, 50, 35, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2005(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 21, 22, 23, 8, 31, 16, 1, 18, 3, 20, 3, 4, 5, 6, 7, 24, 47, 32, 49, 34, 51, 36, 51, 52, 53, 54, 55, 40, 63, 48, 33, 50, 35, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2006(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 21, 22, 23, 8, 15, 0, 1, 18, 3, 20, 3, 4, 5, 6, 7, 24, 63, 48, 49, 34, 51, 36, 51, 52, 53, 54, 55, 40, 47, 32, 33, 50, 35, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2007(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 21, 22, 23, 8, 31, 0, 1, 18, 3, 20, 3, 4, 5, 6, 7, 24, 47, 48, 49, 34, 51, 36, 51, 52, 53, 54, 55, 40, 63, 32, 33, 50, 35, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2008(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 21, 22, 23, 8, 15, 16, 17, 2, 3, 20, 3, 4, 5, 6, 7, 24, 63, 32, 33, 50, 51, 36, 51, 52, 53, 54, 55, 40, 47, 48, 49, 34, 35, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2009(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 21, 22, 23, 8, 31, 16, 17, 2, 3, 20, 3, 4, 5, 6, 7, 24, 47, 32, 33, 50, 51, 36, 51, 52, 53, 54, 55, 40, 63, 48, 49, 34, 35, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2010(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 21, 22, 23, 8, 15, 0, 17, 2, 3, 20, 3, 4, 5, 6, 7, 24, 63, 48, 33, 50, 51, 36, 51, 52, 53, 54, 55, 40, 47, 32, 49, 34, 35, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2011(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 21, 22, 23, 8, 31, 0, 17, 2, 3, 20, 3, 4, 5, 6, 7, 24, 47, 48, 33, 50, 51, 36, 51, 52, 53, 54, 55, 40, 63, 32, 49, 34, 35, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2012(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 21, 22, 23, 8, 15, 16, 1, 2, 3, 20, 3, 4, 5, 6, 7, 24, 63, 32, 49, 50, 51, 36, 51, 52, 53, 54, 55, 40, 47, 48, 33, 34, 35, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2013(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 21, 22, 23, 8, 31, 16, 1, 2, 3, 20, 3, 4, 5, 6, 7, 24, 47, 32, 49, 50, 51, 36, 51, 52, 53, 54, 55, 40, 63, 48, 33, 34, 35, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2014(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 21, 22, 23, 8, 15, 0, 1, 2, 3, 20, 3, 4, 5, 6, 7, 24, 63, 48, 49, 50, 51, 36, 51, 52, 53, 54, 55, 40, 47, 32, 33, 34, 35, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2015(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 21, 22, 23, 8, 31, 0, 1, 2, 3, 20, 3, 4, 5, 6, 7, 24, 47, 48, 49, 50, 51, 36, 51, 52, 53, 54, 55, 40, 63, 32, 33, 34, 35, 52, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2016(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 21, 22, 23, 8, 15, 16, 17, 18, 19, 4, 3, 4, 5, 6, 7, 24, 63, 32, 33, 34, 35, 52, 51, 52, 53, 54, 55, 40, 47, 48, 49, 50, 51, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2017(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 21, 22, 23, 8, 31, 16, 17, 18, 19, 4, 3, 4, 5, 6, 7, 24, 47, 32, 33, 34, 35, 52, 51, 52, 53, 54, 55, 40, 63, 48, 49, 50, 51, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2018(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 21, 22, 23, 8, 15, 0, 17, 18, 19, 4, 3, 4, 5, 6, 7, 24, 63, 48, 33, 34, 35, 52, 51, 52, 53, 54, 55, 40, 47, 32, 49, 50, 51, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2019(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 21, 22, 23, 8, 31, 0, 17, 18, 19, 4, 3, 4, 5, 6, 7, 24, 47, 48, 33, 34, 35, 52, 51, 52, 53, 54, 55, 40, 63, 32, 49, 50, 51, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2020(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 21, 22, 23, 8, 15, 16, 1, 18, 19, 4, 3, 4, 5, 6, 7, 24, 63, 32, 49, 34, 35, 52, 51, 52, 53, 54, 55, 40, 47, 48, 33, 50, 51, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2021(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 21, 22, 23, 8, 31, 16, 1, 18, 19, 4, 3, 4, 5, 6, 7, 24, 47, 32, 49, 34, 35, 52, 51, 52, 53, 54, 55, 40, 63, 48, 33, 50, 51, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2022(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 21, 22, 23, 8, 15, 0, 1, 18, 19, 4, 3, 4, 5, 6, 7, 24, 63, 48, 49, 34, 35, 52, 51, 52, 53, 54, 55, 40, 47, 32, 33, 50, 51, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2023(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 21, 22, 23, 8, 31, 0, 1, 18, 19, 4, 3, 4, 5, 6, 7, 24, 47, 48, 49, 34, 35, 52, 51, 52, 53, 54, 55, 40, 63, 32, 33, 50, 51, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2024(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 21, 22, 23, 8, 15, 16, 17, 2, 19, 4, 3, 4, 5, 6, 7, 24, 63, 32, 33, 50, 35, 52, 51, 52, 53, 54, 55, 40, 47, 48, 49, 34, 51, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2025(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 21, 22, 23, 8, 31, 16, 17, 2, 19, 4, 3, 4, 5, 6, 7, 24, 47, 32, 33, 50, 35, 52, 51, 52, 53, 54, 55, 40, 63, 48, 49, 34, 51, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2026(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 21, 22, 23, 8, 15, 0, 17, 2, 19, 4, 3, 4, 5, 6, 7, 24, 63, 48, 33, 50, 35, 52, 51, 52, 53, 54, 55, 40, 47, 32, 49, 34, 51, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2027(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 21, 22, 23, 8, 31, 0, 17, 2, 19, 4, 3, 4, 5, 6, 7, 24, 47, 48, 33, 50, 35, 52, 51, 52, 53, 54, 55, 40, 63, 32, 49, 34, 51, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2028(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 21, 22, 23, 8, 15, 16, 1, 2, 19, 4, 3, 4, 5, 6, 7, 24, 63, 32, 49, 50, 35, 52, 51, 52, 53, 54, 55, 40, 47, 48, 33, 34, 51, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2029(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 21, 22, 23, 8, 31, 16, 1, 2, 19, 4, 3, 4, 5, 6, 7, 24, 47, 32, 49, 50, 35, 52, 51, 52, 53, 54, 55, 40, 63, 48, 33, 34, 51, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2030(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 21, 22, 23, 8, 15, 0, 1, 2, 19, 4, 3, 4, 5, 6, 7, 24, 63, 48, 49, 50, 35, 52, 51, 52, 53, 54, 55, 40, 47, 32, 33, 34, 51, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2031(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 21, 22, 23, 8, 31, 0, 1, 2, 19, 4, 3, 4, 5, 6, 7, 24, 47, 48, 49, 50, 35, 52, 51, 52, 53, 54, 55, 40, 63, 32, 33, 34, 51, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2032(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 21, 22, 23, 8, 15, 16, 17, 18, 3, 4, 3, 4, 5, 6, 7, 24, 63, 32, 33, 34, 51, 52, 51, 52, 53, 54, 55, 40, 47, 48, 49, 50, 35, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2033(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 21, 22, 23, 8, 31, 16, 17, 18, 3, 4, 3, 4, 5, 6, 7, 24, 47, 32, 33, 34, 51, 52, 51, 52, 53, 54, 55, 40, 63, 48, 49, 50, 35, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2034(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 21, 22, 23, 8, 15, 0, 17, 18, 3, 4, 3, 4, 5, 6, 7, 24, 63, 48, 33, 34, 51, 52, 51, 52, 53, 54, 55, 40, 47, 32, 49, 50, 35, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2035(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 21, 22, 23, 8, 31, 0, 17, 18, 3, 4, 3, 4, 5, 6, 7, 24, 47, 48, 33, 34, 51, 52, 51, 52, 53, 54, 55, 40, 63, 32, 49, 50, 35, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2036(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 21, 22, 23, 8, 15, 16, 1, 18, 3, 4, 3, 4, 5, 6, 7, 24, 63, 32, 49, 34, 51, 52, 51, 52, 53, 54, 55, 40, 47, 48, 33, 50, 35, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2037(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 21, 22, 23, 8, 31, 16, 1, 18, 3, 4, 3, 4, 5, 6, 7, 24, 47, 32, 49, 34, 51, 52, 51, 52, 53, 54, 55, 40, 63, 48, 33, 50, 35, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2038(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 21, 22, 23, 8, 15, 0, 1, 18, 3, 4, 3, 4, 5, 6, 7, 24, 63, 48, 49, 34, 51, 52, 51, 52, 53, 54, 55, 40, 47, 32, 33, 50, 35, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2039(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 21, 22, 23, 8, 31, 0, 1, 18, 3, 4, 3, 4, 5, 6, 7, 24, 47, 48, 49, 34, 51, 52, 51, 52, 53, 54, 55, 40, 63, 32, 33, 50, 35, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2040(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 21, 22, 23, 8, 15, 16, 17, 2, 3, 4, 3, 4, 5, 6, 7, 24, 63, 32, 33, 50, 51, 52, 51, 52, 53, 54, 55, 40, 47, 48, 49, 34, 35, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2041(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 21, 22, 23, 8, 31, 16, 17, 2, 3, 4, 3, 4, 5, 6, 7, 24, 47, 32, 33, 50, 51, 52, 51, 52, 53, 54, 55, 40, 63, 48, 49, 34, 35, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2042(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 21, 22, 23, 8, 15, 0, 17, 2, 3, 4, 3, 4, 5, 6, 7, 24, 63, 48, 33, 50, 51, 52, 51, 52, 53, 54, 55, 40, 47, 32, 49, 34, 35, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2043(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 21, 22, 23, 8, 31, 0, 17, 2, 3, 4, 3, 4, 5, 6, 7, 24, 47, 48, 33, 50, 51, 52, 51, 52, 53, 54, 55, 40, 63, 32, 49, 34, 35, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2044(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 21, 22, 23, 8, 15, 16, 1, 2, 3, 4, 3, 4, 5, 6, 7, 24, 63, 32, 49, 50, 51, 52, 51, 52, 53, 54, 55, 40, 47, 48, 33, 34, 35, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2045(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 21, 22, 23, 8, 31, 16, 1, 2, 3, 4, 3, 4, 5, 6, 7, 24, 47, 32, 49, 50, 51, 52, 51, 52, 53, 54, 55, 40, 63, 48, 33, 34, 35, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2046(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 21, 22, 23, 8, 15, 0, 1, 2, 3, 4, 3, 4, 5, 6, 7, 24, 63, 48, 49, 50, 51, 52, 51, 52, 53, 54, 55, 40, 47, 32, 33, 34, 35, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2047(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 21, 22, 23, 8, 31, 0, 1, 2, 3, 4, 3, 4, 5, 6, 7, 24, 47, 48, 49, 50, 51, 52, 51, 52, 53, 54, 55, 40, 63, 32, 33, 34, 35, 36, 35, 36, 37, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}


