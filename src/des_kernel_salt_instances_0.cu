#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt0(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 4, 5, 6, 7, 8, 15, 16, 17, 18, 19, 20, 19, 20, 21, 22, 23, 24, 63, 32, 33, 34, 35, 36, 35, 36, 37, 38, 39, 40, 47, 48, 49, 50, 51, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 4, 5, 6, 7, 8, 31, 16, 17, 18, 19, 20, 19, 20, 21, 22, 23, 24, 47, 32, 33, 34, 35, 36, 35, 36, 37, 38, 39, 40, 63, 48, 49, 50, 51, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 4, 5, 6, 7, 8, 15, 0, 17, 18, 19, 20, 19, 20, 21, 22, 23, 24, 63, 48, 33, 34, 35, 36, 35, 36, 37, 38, 39, 40, 47, 32, 49, 50, 51, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt3(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 4, 5, 6, 7, 8, 31, 0, 17, 18, 19, 20, 19, 20, 21, 22, 23, 24, 47, 48, 33, 34, 35, 36, 35, 36, 37, 38, 39, 40, 63, 32, 49, 50, 51, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt4(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 4, 5, 6, 7, 8, 15, 16, 1, 18, 19, 20, 19, 20, 21, 22, 23, 24, 63, 32, 49, 34, 35, 36, 35, 36, 37, 38, 39, 40, 47, 48, 33, 50, 51, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt5(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 4, 5, 6, 7, 8, 31, 16, 1, 18, 19, 20, 19, 20, 21, 22, 23, 24, 47, 32, 49, 34, 35, 36, 35, 36, 37, 38, 39, 40, 63, 48, 33, 50, 51, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt6(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 4, 5, 6, 7, 8, 15, 0, 1, 18, 19, 20, 19, 20, 21, 22, 23, 24, 63, 48, 49, 34, 35, 36, 35, 36, 37, 38, 39, 40, 47, 32, 33, 50, 51, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt7(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 4, 5, 6, 7, 8, 31, 0, 1, 18, 19, 20, 19, 20, 21, 22, 23, 24, 47, 48, 49, 34, 35, 36, 35, 36, 37, 38, 39, 40, 63, 32, 33, 50, 51, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt8(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 4, 5, 6, 7, 8, 15, 16, 17, 2, 19, 20, 19, 20, 21, 22, 23, 24, 63, 32, 33, 50, 35, 36, 35, 36, 37, 38, 39, 40, 47, 48, 49, 34, 51, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt9(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 4, 5, 6, 7, 8, 31, 16, 17, 2, 19, 20, 19, 20, 21, 22, 23, 24, 47, 32, 33, 50, 35, 36, 35, 36, 37, 38, 39, 40, 63, 48, 49, 34, 51, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt10(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 4, 5, 6, 7, 8, 15, 0, 17, 2, 19, 20, 19, 20, 21, 22, 23, 24, 63, 48, 33, 50, 35, 36, 35, 36, 37, 38, 39, 40, 47, 32, 49, 34, 51, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt11(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 4, 5, 6, 7, 8, 31, 0, 17, 2, 19, 20, 19, 20, 21, 22, 23, 24, 47, 48, 33, 50, 35, 36, 35, 36, 37, 38, 39, 40, 63, 32, 49, 34, 51, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt12(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 4, 5, 6, 7, 8, 15, 16, 1, 2, 19, 20, 19, 20, 21, 22, 23, 24, 63, 32, 49, 50, 35, 36, 35, 36, 37, 38, 39, 40, 47, 48, 33, 34, 51, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt13(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 4, 5, 6, 7, 8, 31, 16, 1, 2, 19, 20, 19, 20, 21, 22, 23, 24, 47, 32, 49, 50, 35, 36, 35, 36, 37, 38, 39, 40, 63, 48, 33, 34, 51, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt14(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 4, 5, 6, 7, 8, 15, 0, 1, 2, 19, 20, 19, 20, 21, 22, 23, 24, 63, 48, 49, 50, 35, 36, 35, 36, 37, 38, 39, 40, 47, 32, 33, 34, 51, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt15(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 4, 5, 6, 7, 8, 31, 0, 1, 2, 19, 20, 19, 20, 21, 22, 23, 24, 47, 48, 49, 50, 35, 36, 35, 36, 37, 38, 39, 40, 63, 32, 33, 34, 51, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt16(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 4, 5, 6, 7, 8, 15, 16, 17, 18, 3, 20, 19, 20, 21, 22, 23, 24, 63, 32, 33, 34, 51, 36, 35, 36, 37, 38, 39, 40, 47, 48, 49, 50, 35, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt17(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 4, 5, 6, 7, 8, 31, 16, 17, 18, 3, 20, 19, 20, 21, 22, 23, 24, 47, 32, 33, 34, 51, 36, 35, 36, 37, 38, 39, 40, 63, 48, 49, 50, 35, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt18(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 4, 5, 6, 7, 8, 15, 0, 17, 18, 3, 20, 19, 20, 21, 22, 23, 24, 63, 48, 33, 34, 51, 36, 35, 36, 37, 38, 39, 40, 47, 32, 49, 50, 35, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt19(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 4, 5, 6, 7, 8, 31, 0, 17, 18, 3, 20, 19, 20, 21, 22, 23, 24, 47, 48, 33, 34, 51, 36, 35, 36, 37, 38, 39, 40, 63, 32, 49, 50, 35, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt20(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 4, 5, 6, 7, 8, 15, 16, 1, 18, 3, 20, 19, 20, 21, 22, 23, 24, 63, 32, 49, 34, 51, 36, 35, 36, 37, 38, 39, 40, 47, 48, 33, 50, 35, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt21(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 4, 5, 6, 7, 8, 31, 16, 1, 18, 3, 20, 19, 20, 21, 22, 23, 24, 47, 32, 49, 34, 51, 36, 35, 36, 37, 38, 39, 40, 63, 48, 33, 50, 35, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt22(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 4, 5, 6, 7, 8, 15, 0, 1, 18, 3, 20, 19, 20, 21, 22, 23, 24, 63, 48, 49, 34, 51, 36, 35, 36, 37, 38, 39, 40, 47, 32, 33, 50, 35, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt23(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 4, 5, 6, 7, 8, 31, 0, 1, 18, 3, 20, 19, 20, 21, 22, 23, 24, 47, 48, 49, 34, 51, 36, 35, 36, 37, 38, 39, 40, 63, 32, 33, 50, 35, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt24(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 4, 5, 6, 7, 8, 15, 16, 17, 2, 3, 20, 19, 20, 21, 22, 23, 24, 63, 32, 33, 50, 51, 36, 35, 36, 37, 38, 39, 40, 47, 48, 49, 34, 35, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt25(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 4, 5, 6, 7, 8, 31, 16, 17, 2, 3, 20, 19, 20, 21, 22, 23, 24, 47, 32, 33, 50, 51, 36, 35, 36, 37, 38, 39, 40, 63, 48, 49, 34, 35, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt26(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 4, 5, 6, 7, 8, 15, 0, 17, 2, 3, 20, 19, 20, 21, 22, 23, 24, 63, 48, 33, 50, 51, 36, 35, 36, 37, 38, 39, 40, 47, 32, 49, 34, 35, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt27(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 4, 5, 6, 7, 8, 31, 0, 17, 2, 3, 20, 19, 20, 21, 22, 23, 24, 47, 48, 33, 50, 51, 36, 35, 36, 37, 38, 39, 40, 63, 32, 49, 34, 35, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt28(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 4, 5, 6, 7, 8, 15, 16, 1, 2, 3, 20, 19, 20, 21, 22, 23, 24, 63, 32, 49, 50, 51, 36, 35, 36, 37, 38, 39, 40, 47, 48, 33, 34, 35, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt29(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 4, 5, 6, 7, 8, 31, 16, 1, 2, 3, 20, 19, 20, 21, 22, 23, 24, 47, 32, 49, 50, 51, 36, 35, 36, 37, 38, 39, 40, 63, 48, 33, 34, 35, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt30(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 4, 5, 6, 7, 8, 15, 0, 1, 2, 3, 20, 19, 20, 21, 22, 23, 24, 63, 48, 49, 50, 51, 36, 35, 36, 37, 38, 39, 40, 47, 32, 33, 34, 35, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt31(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 4, 5, 6, 7, 8, 31, 0, 1, 2, 3, 20, 19, 20, 21, 22, 23, 24, 47, 48, 49, 50, 51, 36, 35, 36, 37, 38, 39, 40, 63, 32, 33, 34, 35, 52, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt32(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 4, 5, 6, 7, 8, 15, 16, 17, 18, 19, 4, 19, 20, 21, 22, 23, 24, 63, 32, 33, 34, 35, 52, 35, 36, 37, 38, 39, 40, 47, 48, 49, 50, 51, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt33(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 4, 5, 6, 7, 8, 31, 16, 17, 18, 19, 4, 19, 20, 21, 22, 23, 24, 47, 32, 33, 34, 35, 52, 35, 36, 37, 38, 39, 40, 63, 48, 49, 50, 51, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt34(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 4, 5, 6, 7, 8, 15, 0, 17, 18, 19, 4, 19, 20, 21, 22, 23, 24, 63, 48, 33, 34, 35, 52, 35, 36, 37, 38, 39, 40, 47, 32, 49, 50, 51, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt35(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 4, 5, 6, 7, 8, 31, 0, 17, 18, 19, 4, 19, 20, 21, 22, 23, 24, 47, 48, 33, 34, 35, 52, 35, 36, 37, 38, 39, 40, 63, 32, 49, 50, 51, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt36(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 4, 5, 6, 7, 8, 15, 16, 1, 18, 19, 4, 19, 20, 21, 22, 23, 24, 63, 32, 49, 34, 35, 52, 35, 36, 37, 38, 39, 40, 47, 48, 33, 50, 51, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt37(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 4, 5, 6, 7, 8, 31, 16, 1, 18, 19, 4, 19, 20, 21, 22, 23, 24, 47, 32, 49, 34, 35, 52, 35, 36, 37, 38, 39, 40, 63, 48, 33, 50, 51, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt38(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 4, 5, 6, 7, 8, 15, 0, 1, 18, 19, 4, 19, 20, 21, 22, 23, 24, 63, 48, 49, 34, 35, 52, 35, 36, 37, 38, 39, 40, 47, 32, 33, 50, 51, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt39(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 4, 5, 6, 7, 8, 31, 0, 1, 18, 19, 4, 19, 20, 21, 22, 23, 24, 47, 48, 49, 34, 35, 52, 35, 36, 37, 38, 39, 40, 63, 32, 33, 50, 51, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt40(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 4, 5, 6, 7, 8, 15, 16, 17, 2, 19, 4, 19, 20, 21, 22, 23, 24, 63, 32, 33, 50, 35, 52, 35, 36, 37, 38, 39, 40, 47, 48, 49, 34, 51, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt41(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 4, 5, 6, 7, 8, 31, 16, 17, 2, 19, 4, 19, 20, 21, 22, 23, 24, 47, 32, 33, 50, 35, 52, 35, 36, 37, 38, 39, 40, 63, 48, 49, 34, 51, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt42(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 4, 5, 6, 7, 8, 15, 0, 17, 2, 19, 4, 19, 20, 21, 22, 23, 24, 63, 48, 33, 50, 35, 52, 35, 36, 37, 38, 39, 40, 47, 32, 49, 34, 51, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt43(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 4, 5, 6, 7, 8, 31, 0, 17, 2, 19, 4, 19, 20, 21, 22, 23, 24, 47, 48, 33, 50, 35, 52, 35, 36, 37, 38, 39, 40, 63, 32, 49, 34, 51, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt44(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 4, 5, 6, 7, 8, 15, 16, 1, 2, 19, 4, 19, 20, 21, 22, 23, 24, 63, 32, 49, 50, 35, 52, 35, 36, 37, 38, 39, 40, 47, 48, 33, 34, 51, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt45(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 4, 5, 6, 7, 8, 31, 16, 1, 2, 19, 4, 19, 20, 21, 22, 23, 24, 47, 32, 49, 50, 35, 52, 35, 36, 37, 38, 39, 40, 63, 48, 33, 34, 51, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt46(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 4, 5, 6, 7, 8, 15, 0, 1, 2, 19, 4, 19, 20, 21, 22, 23, 24, 63, 48, 49, 50, 35, 52, 35, 36, 37, 38, 39, 40, 47, 32, 33, 34, 51, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt47(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 4, 5, 6, 7, 8, 31, 0, 1, 2, 19, 4, 19, 20, 21, 22, 23, 24, 47, 48, 49, 50, 35, 52, 35, 36, 37, 38, 39, 40, 63, 32, 33, 34, 51, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt48(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 4, 5, 6, 7, 8, 15, 16, 17, 18, 3, 4, 19, 20, 21, 22, 23, 24, 63, 32, 33, 34, 51, 52, 35, 36, 37, 38, 39, 40, 47, 48, 49, 50, 35, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt49(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 4, 5, 6, 7, 8, 31, 16, 17, 18, 3, 4, 19, 20, 21, 22, 23, 24, 47, 32, 33, 34, 51, 52, 35, 36, 37, 38, 39, 40, 63, 48, 49, 50, 35, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt50(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 4, 5, 6, 7, 8, 15, 0, 17, 18, 3, 4, 19, 20, 21, 22, 23, 24, 63, 48, 33, 34, 51, 52, 35, 36, 37, 38, 39, 40, 47, 32, 49, 50, 35, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt51(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 4, 5, 6, 7, 8, 31, 0, 17, 18, 3, 4, 19, 20, 21, 22, 23, 24, 47, 48, 33, 34, 51, 52, 35, 36, 37, 38, 39, 40, 63, 32, 49, 50, 35, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt52(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 4, 5, 6, 7, 8, 15, 16, 1, 18, 3, 4, 19, 20, 21, 22, 23, 24, 63, 32, 49, 34, 51, 52, 35, 36, 37, 38, 39, 40, 47, 48, 33, 50, 35, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt53(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 4, 5, 6, 7, 8, 31, 16, 1, 18, 3, 4, 19, 20, 21, 22, 23, 24, 47, 32, 49, 34, 51, 52, 35, 36, 37, 38, 39, 40, 63, 48, 33, 50, 35, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt54(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 4, 5, 6, 7, 8, 15, 0, 1, 18, 3, 4, 19, 20, 21, 22, 23, 24, 63, 48, 49, 34, 51, 52, 35, 36, 37, 38, 39, 40, 47, 32, 33, 50, 35, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt55(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 4, 5, 6, 7, 8, 31, 0, 1, 18, 3, 4, 19, 20, 21, 22, 23, 24, 47, 48, 49, 34, 51, 52, 35, 36, 37, 38, 39, 40, 63, 32, 33, 50, 35, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt56(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 4, 5, 6, 7, 8, 15, 16, 17, 2, 3, 4, 19, 20, 21, 22, 23, 24, 63, 32, 33, 50, 51, 52, 35, 36, 37, 38, 39, 40, 47, 48, 49, 34, 35, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt57(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 4, 5, 6, 7, 8, 31, 16, 17, 2, 3, 4, 19, 20, 21, 22, 23, 24, 47, 32, 33, 50, 51, 52, 35, 36, 37, 38, 39, 40, 63, 48, 49, 34, 35, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt58(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 4, 5, 6, 7, 8, 15, 0, 17, 2, 3, 4, 19, 20, 21, 22, 23, 24, 63, 48, 33, 50, 51, 52, 35, 36, 37, 38, 39, 40, 47, 32, 49, 34, 35, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt59(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 4, 5, 6, 7, 8, 31, 0, 17, 2, 3, 4, 19, 20, 21, 22, 23, 24, 47, 48, 33, 50, 51, 52, 35, 36, 37, 38, 39, 40, 63, 32, 49, 34, 35, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt60(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 4, 5, 6, 7, 8, 15, 16, 1, 2, 3, 4, 19, 20, 21, 22, 23, 24, 63, 32, 49, 50, 51, 52, 35, 36, 37, 38, 39, 40, 47, 48, 33, 34, 35, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt61(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 4, 5, 6, 7, 8, 31, 16, 1, 2, 3, 4, 19, 20, 21, 22, 23, 24, 47, 32, 49, 50, 51, 52, 35, 36, 37, 38, 39, 40, 63, 48, 33, 34, 35, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt62(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 4, 5, 6, 7, 8, 15, 0, 1, 2, 3, 4, 19, 20, 21, 22, 23, 24, 63, 48, 49, 50, 51, 52, 35, 36, 37, 38, 39, 40, 47, 32, 33, 34, 35, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt63(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 4, 5, 6, 7, 8, 31, 0, 1, 2, 3, 4, 19, 20, 21, 22, 23, 24, 47, 48, 49, 50, 51, 52, 35, 36, 37, 38, 39, 40, 63, 32, 33, 34, 35, 36, 51, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt64(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 4, 5, 6, 7, 8, 15, 16, 17, 18, 19, 20, 3, 20, 21, 22, 23, 24, 63, 32, 33, 34, 35, 36, 51, 36, 37, 38, 39, 40, 47, 48, 49, 50, 51, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt65(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 4, 5, 6, 7, 8, 31, 16, 17, 18, 19, 20, 3, 20, 21, 22, 23, 24, 47, 32, 33, 34, 35, 36, 51, 36, 37, 38, 39, 40, 63, 48, 49, 50, 51, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt66(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 4, 5, 6, 7, 8, 15, 0, 17, 18, 19, 20, 3, 20, 21, 22, 23, 24, 63, 48, 33, 34, 35, 36, 51, 36, 37, 38, 39, 40, 47, 32, 49, 50, 51, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt67(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 4, 5, 6, 7, 8, 31, 0, 17, 18, 19, 20, 3, 20, 21, 22, 23, 24, 47, 48, 33, 34, 35, 36, 51, 36, 37, 38, 39, 40, 63, 32, 49, 50, 51, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt68(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 4, 5, 6, 7, 8, 15, 16, 1, 18, 19, 20, 3, 20, 21, 22, 23, 24, 63, 32, 49, 34, 35, 36, 51, 36, 37, 38, 39, 40, 47, 48, 33, 50, 51, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt69(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 4, 5, 6, 7, 8, 31, 16, 1, 18, 19, 20, 3, 20, 21, 22, 23, 24, 47, 32, 49, 34, 35, 36, 51, 36, 37, 38, 39, 40, 63, 48, 33, 50, 51, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt70(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 4, 5, 6, 7, 8, 15, 0, 1, 18, 19, 20, 3, 20, 21, 22, 23, 24, 63, 48, 49, 34, 35, 36, 51, 36, 37, 38, 39, 40, 47, 32, 33, 50, 51, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt71(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 4, 5, 6, 7, 8, 31, 0, 1, 18, 19, 20, 3, 20, 21, 22, 23, 24, 47, 48, 49, 34, 35, 36, 51, 36, 37, 38, 39, 40, 63, 32, 33, 50, 51, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt72(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 4, 5, 6, 7, 8, 15, 16, 17, 2, 19, 20, 3, 20, 21, 22, 23, 24, 63, 32, 33, 50, 35, 36, 51, 36, 37, 38, 39, 40, 47, 48, 49, 34, 51, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt73(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 4, 5, 6, 7, 8, 31, 16, 17, 2, 19, 20, 3, 20, 21, 22, 23, 24, 47, 32, 33, 50, 35, 36, 51, 36, 37, 38, 39, 40, 63, 48, 49, 34, 51, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt74(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 4, 5, 6, 7, 8, 15, 0, 17, 2, 19, 20, 3, 20, 21, 22, 23, 24, 63, 48, 33, 50, 35, 36, 51, 36, 37, 38, 39, 40, 47, 32, 49, 34, 51, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt75(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 4, 5, 6, 7, 8, 31, 0, 17, 2, 19, 20, 3, 20, 21, 22, 23, 24, 47, 48, 33, 50, 35, 36, 51, 36, 37, 38, 39, 40, 63, 32, 49, 34, 51, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt76(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 4, 5, 6, 7, 8, 15, 16, 1, 2, 19, 20, 3, 20, 21, 22, 23, 24, 63, 32, 49, 50, 35, 36, 51, 36, 37, 38, 39, 40, 47, 48, 33, 34, 51, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt77(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 4, 5, 6, 7, 8, 31, 16, 1, 2, 19, 20, 3, 20, 21, 22, 23, 24, 47, 32, 49, 50, 35, 36, 51, 36, 37, 38, 39, 40, 63, 48, 33, 34, 51, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt78(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 4, 5, 6, 7, 8, 15, 0, 1, 2, 19, 20, 3, 20, 21, 22, 23, 24, 63, 48, 49, 50, 35, 36, 51, 36, 37, 38, 39, 40, 47, 32, 33, 34, 51, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt79(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 4, 5, 6, 7, 8, 31, 0, 1, 2, 19, 20, 3, 20, 21, 22, 23, 24, 47, 48, 49, 50, 35, 36, 51, 36, 37, 38, 39, 40, 63, 32, 33, 34, 51, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt80(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 4, 5, 6, 7, 8, 15, 16, 17, 18, 3, 20, 3, 20, 21, 22, 23, 24, 63, 32, 33, 34, 51, 36, 51, 36, 37, 38, 39, 40, 47, 48, 49, 50, 35, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt81(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 4, 5, 6, 7, 8, 31, 16, 17, 18, 3, 20, 3, 20, 21, 22, 23, 24, 47, 32, 33, 34, 51, 36, 51, 36, 37, 38, 39, 40, 63, 48, 49, 50, 35, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt82(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 4, 5, 6, 7, 8, 15, 0, 17, 18, 3, 20, 3, 20, 21, 22, 23, 24, 63, 48, 33, 34, 51, 36, 51, 36, 37, 38, 39, 40, 47, 32, 49, 50, 35, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt83(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 4, 5, 6, 7, 8, 31, 0, 17, 18, 3, 20, 3, 20, 21, 22, 23, 24, 47, 48, 33, 34, 51, 36, 51, 36, 37, 38, 39, 40, 63, 32, 49, 50, 35, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt84(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 4, 5, 6, 7, 8, 15, 16, 1, 18, 3, 20, 3, 20, 21, 22, 23, 24, 63, 32, 49, 34, 51, 36, 51, 36, 37, 38, 39, 40, 47, 48, 33, 50, 35, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt85(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 4, 5, 6, 7, 8, 31, 16, 1, 18, 3, 20, 3, 20, 21, 22, 23, 24, 47, 32, 49, 34, 51, 36, 51, 36, 37, 38, 39, 40, 63, 48, 33, 50, 35, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt86(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 4, 5, 6, 7, 8, 15, 0, 1, 18, 3, 20, 3, 20, 21, 22, 23, 24, 63, 48, 49, 34, 51, 36, 51, 36, 37, 38, 39, 40, 47, 32, 33, 50, 35, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt87(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 4, 5, 6, 7, 8, 31, 0, 1, 18, 3, 20, 3, 20, 21, 22, 23, 24, 47, 48, 49, 34, 51, 36, 51, 36, 37, 38, 39, 40, 63, 32, 33, 50, 35, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt88(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 4, 5, 6, 7, 8, 15, 16, 17, 2, 3, 20, 3, 20, 21, 22, 23, 24, 63, 32, 33, 50, 51, 36, 51, 36, 37, 38, 39, 40, 47, 48, 49, 34, 35, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt89(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 4, 5, 6, 7, 8, 31, 16, 17, 2, 3, 20, 3, 20, 21, 22, 23, 24, 47, 32, 33, 50, 51, 36, 51, 36, 37, 38, 39, 40, 63, 48, 49, 34, 35, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt90(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 4, 5, 6, 7, 8, 15, 0, 17, 2, 3, 20, 3, 20, 21, 22, 23, 24, 63, 48, 33, 50, 51, 36, 51, 36, 37, 38, 39, 40, 47, 32, 49, 34, 35, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt91(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 4, 5, 6, 7, 8, 31, 0, 17, 2, 3, 20, 3, 20, 21, 22, 23, 24, 47, 48, 33, 50, 51, 36, 51, 36, 37, 38, 39, 40, 63, 32, 49, 34, 35, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt92(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 4, 5, 6, 7, 8, 15, 16, 1, 2, 3, 20, 3, 20, 21, 22, 23, 24, 63, 32, 49, 50, 51, 36, 51, 36, 37, 38, 39, 40, 47, 48, 33, 34, 35, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt93(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 4, 5, 6, 7, 8, 31, 16, 1, 2, 3, 20, 3, 20, 21, 22, 23, 24, 47, 32, 49, 50, 51, 36, 51, 36, 37, 38, 39, 40, 63, 48, 33, 34, 35, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt94(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 4, 5, 6, 7, 8, 15, 0, 1, 2, 3, 20, 3, 20, 21, 22, 23, 24, 63, 48, 49, 50, 51, 36, 51, 36, 37, 38, 39, 40, 47, 32, 33, 34, 35, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt95(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 4, 5, 6, 7, 8, 31, 0, 1, 2, 3, 20, 3, 20, 21, 22, 23, 24, 47, 48, 49, 50, 51, 36, 51, 36, 37, 38, 39, 40, 63, 32, 33, 34, 35, 52, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt96(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 4, 5, 6, 7, 8, 15, 16, 17, 18, 19, 4, 3, 20, 21, 22, 23, 24, 63, 32, 33, 34, 35, 52, 51, 36, 37, 38, 39, 40, 47, 48, 49, 50, 51, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt97(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 4, 5, 6, 7, 8, 31, 16, 17, 18, 19, 4, 3, 20, 21, 22, 23, 24, 47, 32, 33, 34, 35, 52, 51, 36, 37, 38, 39, 40, 63, 48, 49, 50, 51, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt98(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 4, 5, 6, 7, 8, 15, 0, 17, 18, 19, 4, 3, 20, 21, 22, 23, 24, 63, 48, 33, 34, 35, 52, 51, 36, 37, 38, 39, 40, 47, 32, 49, 50, 51, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt99(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 4, 5, 6, 7, 8, 31, 0, 17, 18, 19, 4, 3, 20, 21, 22, 23, 24, 47, 48, 33, 34, 35, 52, 51, 36, 37, 38, 39, 40, 63, 32, 49, 50, 51, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt100(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 4, 5, 6, 7, 8, 15, 16, 1, 18, 19, 4, 3, 20, 21, 22, 23, 24, 63, 32, 49, 34, 35, 52, 51, 36, 37, 38, 39, 40, 47, 48, 33, 50, 51, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt101(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 4, 5, 6, 7, 8, 31, 16, 1, 18, 19, 4, 3, 20, 21, 22, 23, 24, 47, 32, 49, 34, 35, 52, 51, 36, 37, 38, 39, 40, 63, 48, 33, 50, 51, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt102(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 4, 5, 6, 7, 8, 15, 0, 1, 18, 19, 4, 3, 20, 21, 22, 23, 24, 63, 48, 49, 34, 35, 52, 51, 36, 37, 38, 39, 40, 47, 32, 33, 50, 51, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt103(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 4, 5, 6, 7, 8, 31, 0, 1, 18, 19, 4, 3, 20, 21, 22, 23, 24, 47, 48, 49, 34, 35, 52, 51, 36, 37, 38, 39, 40, 63, 32, 33, 50, 51, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt104(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 4, 5, 6, 7, 8, 15, 16, 17, 2, 19, 4, 3, 20, 21, 22, 23, 24, 63, 32, 33, 50, 35, 52, 51, 36, 37, 38, 39, 40, 47, 48, 49, 34, 51, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt105(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 4, 5, 6, 7, 8, 31, 16, 17, 2, 19, 4, 3, 20, 21, 22, 23, 24, 47, 32, 33, 50, 35, 52, 51, 36, 37, 38, 39, 40, 63, 48, 49, 34, 51, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt106(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 4, 5, 6, 7, 8, 15, 0, 17, 2, 19, 4, 3, 20, 21, 22, 23, 24, 63, 48, 33, 50, 35, 52, 51, 36, 37, 38, 39, 40, 47, 32, 49, 34, 51, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt107(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 4, 5, 6, 7, 8, 31, 0, 17, 2, 19, 4, 3, 20, 21, 22, 23, 24, 47, 48, 33, 50, 35, 52, 51, 36, 37, 38, 39, 40, 63, 32, 49, 34, 51, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt108(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 4, 5, 6, 7, 8, 15, 16, 1, 2, 19, 4, 3, 20, 21, 22, 23, 24, 63, 32, 49, 50, 35, 52, 51, 36, 37, 38, 39, 40, 47, 48, 33, 34, 51, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt109(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 4, 5, 6, 7, 8, 31, 16, 1, 2, 19, 4, 3, 20, 21, 22, 23, 24, 47, 32, 49, 50, 35, 52, 51, 36, 37, 38, 39, 40, 63, 48, 33, 34, 51, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt110(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 4, 5, 6, 7, 8, 15, 0, 1, 2, 19, 4, 3, 20, 21, 22, 23, 24, 63, 48, 49, 50, 35, 52, 51, 36, 37, 38, 39, 40, 47, 32, 33, 34, 51, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt111(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 4, 5, 6, 7, 8, 31, 0, 1, 2, 19, 4, 3, 20, 21, 22, 23, 24, 47, 48, 49, 50, 35, 52, 51, 36, 37, 38, 39, 40, 63, 32, 33, 34, 51, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt112(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 4, 5, 6, 7, 8, 15, 16, 17, 18, 3, 4, 3, 20, 21, 22, 23, 24, 63, 32, 33, 34, 51, 52, 51, 36, 37, 38, 39, 40, 47, 48, 49, 50, 35, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt113(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 4, 5, 6, 7, 8, 31, 16, 17, 18, 3, 4, 3, 20, 21, 22, 23, 24, 47, 32, 33, 34, 51, 52, 51, 36, 37, 38, 39, 40, 63, 48, 49, 50, 35, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt114(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 4, 5, 6, 7, 8, 15, 0, 17, 18, 3, 4, 3, 20, 21, 22, 23, 24, 63, 48, 33, 34, 51, 52, 51, 36, 37, 38, 39, 40, 47, 32, 49, 50, 35, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt115(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 4, 5, 6, 7, 8, 31, 0, 17, 18, 3, 4, 3, 20, 21, 22, 23, 24, 47, 48, 33, 34, 51, 52, 51, 36, 37, 38, 39, 40, 63, 32, 49, 50, 35, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt116(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 4, 5, 6, 7, 8, 15, 16, 1, 18, 3, 4, 3, 20, 21, 22, 23, 24, 63, 32, 49, 34, 51, 52, 51, 36, 37, 38, 39, 40, 47, 48, 33, 50, 35, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt117(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 4, 5, 6, 7, 8, 31, 16, 1, 18, 3, 4, 3, 20, 21, 22, 23, 24, 47, 32, 49, 34, 51, 52, 51, 36, 37, 38, 39, 40, 63, 48, 33, 50, 35, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt118(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 4, 5, 6, 7, 8, 15, 0, 1, 18, 3, 4, 3, 20, 21, 22, 23, 24, 63, 48, 49, 34, 51, 52, 51, 36, 37, 38, 39, 40, 47, 32, 33, 50, 35, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt119(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 4, 5, 6, 7, 8, 31, 0, 1, 18, 3, 4, 3, 20, 21, 22, 23, 24, 47, 48, 49, 34, 51, 52, 51, 36, 37, 38, 39, 40, 63, 32, 33, 50, 35, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt120(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 4, 5, 6, 7, 8, 15, 16, 17, 2, 3, 4, 3, 20, 21, 22, 23, 24, 63, 32, 33, 50, 51, 52, 51, 36, 37, 38, 39, 40, 47, 48, 49, 34, 35, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt121(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 4, 5, 6, 7, 8, 31, 16, 17, 2, 3, 4, 3, 20, 21, 22, 23, 24, 47, 32, 33, 50, 51, 52, 51, 36, 37, 38, 39, 40, 63, 48, 49, 34, 35, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt122(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 4, 5, 6, 7, 8, 15, 0, 17, 2, 3, 4, 3, 20, 21, 22, 23, 24, 63, 48, 33, 50, 51, 52, 51, 36, 37, 38, 39, 40, 47, 32, 49, 34, 35, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt123(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 4, 5, 6, 7, 8, 31, 0, 17, 2, 3, 4, 3, 20, 21, 22, 23, 24, 47, 48, 33, 50, 51, 52, 51, 36, 37, 38, 39, 40, 63, 32, 49, 34, 35, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt124(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 4, 5, 6, 7, 8, 15, 16, 1, 2, 3, 4, 3, 20, 21, 22, 23, 24, 63, 32, 49, 50, 51, 52, 51, 36, 37, 38, 39, 40, 47, 48, 33, 34, 35, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt125(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 4, 5, 6, 7, 8, 31, 16, 1, 2, 3, 4, 3, 20, 21, 22, 23, 24, 47, 32, 49, 50, 51, 52, 51, 36, 37, 38, 39, 40, 63, 48, 33, 34, 35, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt126(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 4, 5, 6, 7, 8, 15, 0, 1, 2, 3, 4, 3, 20, 21, 22, 23, 24, 63, 48, 49, 50, 51, 52, 51, 36, 37, 38, 39, 40, 47, 32, 33, 34, 35, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt127(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 4, 5, 6, 7, 8, 31, 0, 1, 2, 3, 4, 3, 20, 21, 22, 23, 24, 47, 48, 49, 50, 51, 52, 51, 36, 37, 38, 39, 40, 63, 32, 33, 34, 35, 36, 35, 52, 53, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}


