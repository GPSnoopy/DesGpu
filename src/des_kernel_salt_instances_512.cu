#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt512(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 4, 5, 22, 7, 8, 15, 16, 17, 18, 19, 20, 19, 20, 21, 6, 23, 24, 63, 32, 33, 34, 35, 36, 35, 36, 37, 54, 39, 40, 47, 48, 49, 50, 51, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt513(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 4, 5, 22, 7, 8, 31, 16, 17, 18, 19, 20, 19, 20, 21, 6, 23, 24, 47, 32, 33, 34, 35, 36, 35, 36, 37, 54, 39, 40, 63, 48, 49, 50, 51, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt514(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 4, 5, 22, 7, 8, 15, 0, 17, 18, 19, 20, 19, 20, 21, 6, 23, 24, 63, 48, 33, 34, 35, 36, 35, 36, 37, 54, 39, 40, 47, 32, 49, 50, 51, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt515(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 4, 5, 22, 7, 8, 31, 0, 17, 18, 19, 20, 19, 20, 21, 6, 23, 24, 47, 48, 33, 34, 35, 36, 35, 36, 37, 54, 39, 40, 63, 32, 49, 50, 51, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt516(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 4, 5, 22, 7, 8, 15, 16, 1, 18, 19, 20, 19, 20, 21, 6, 23, 24, 63, 32, 49, 34, 35, 36, 35, 36, 37, 54, 39, 40, 47, 48, 33, 50, 51, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt517(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 4, 5, 22, 7, 8, 31, 16, 1, 18, 19, 20, 19, 20, 21, 6, 23, 24, 47, 32, 49, 34, 35, 36, 35, 36, 37, 54, 39, 40, 63, 48, 33, 50, 51, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt518(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 4, 5, 22, 7, 8, 15, 0, 1, 18, 19, 20, 19, 20, 21, 6, 23, 24, 63, 48, 49, 34, 35, 36, 35, 36, 37, 54, 39, 40, 47, 32, 33, 50, 51, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt519(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 4, 5, 22, 7, 8, 31, 0, 1, 18, 19, 20, 19, 20, 21, 6, 23, 24, 47, 48, 49, 34, 35, 36, 35, 36, 37, 54, 39, 40, 63, 32, 33, 50, 51, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt520(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 4, 5, 22, 7, 8, 15, 16, 17, 2, 19, 20, 19, 20, 21, 6, 23, 24, 63, 32, 33, 50, 35, 36, 35, 36, 37, 54, 39, 40, 47, 48, 49, 34, 51, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt521(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 4, 5, 22, 7, 8, 31, 16, 17, 2, 19, 20, 19, 20, 21, 6, 23, 24, 47, 32, 33, 50, 35, 36, 35, 36, 37, 54, 39, 40, 63, 48, 49, 34, 51, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt522(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 4, 5, 22, 7, 8, 15, 0, 17, 2, 19, 20, 19, 20, 21, 6, 23, 24, 63, 48, 33, 50, 35, 36, 35, 36, 37, 54, 39, 40, 47, 32, 49, 34, 51, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt523(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 4, 5, 22, 7, 8, 31, 0, 17, 2, 19, 20, 19, 20, 21, 6, 23, 24, 47, 48, 33, 50, 35, 36, 35, 36, 37, 54, 39, 40, 63, 32, 49, 34, 51, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt524(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 4, 5, 22, 7, 8, 15, 16, 1, 2, 19, 20, 19, 20, 21, 6, 23, 24, 63, 32, 49, 50, 35, 36, 35, 36, 37, 54, 39, 40, 47, 48, 33, 34, 51, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt525(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 4, 5, 22, 7, 8, 31, 16, 1, 2, 19, 20, 19, 20, 21, 6, 23, 24, 47, 32, 49, 50, 35, 36, 35, 36, 37, 54, 39, 40, 63, 48, 33, 34, 51, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt526(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 4, 5, 22, 7, 8, 15, 0, 1, 2, 19, 20, 19, 20, 21, 6, 23, 24, 63, 48, 49, 50, 35, 36, 35, 36, 37, 54, 39, 40, 47, 32, 33, 34, 51, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt527(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 4, 5, 22, 7, 8, 31, 0, 1, 2, 19, 20, 19, 20, 21, 6, 23, 24, 47, 48, 49, 50, 35, 36, 35, 36, 37, 54, 39, 40, 63, 32, 33, 34, 51, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt528(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 4, 5, 22, 7, 8, 15, 16, 17, 18, 3, 20, 19, 20, 21, 6, 23, 24, 63, 32, 33, 34, 51, 36, 35, 36, 37, 54, 39, 40, 47, 48, 49, 50, 35, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt529(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 4, 5, 22, 7, 8, 31, 16, 17, 18, 3, 20, 19, 20, 21, 6, 23, 24, 47, 32, 33, 34, 51, 36, 35, 36, 37, 54, 39, 40, 63, 48, 49, 50, 35, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt530(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 4, 5, 22, 7, 8, 15, 0, 17, 18, 3, 20, 19, 20, 21, 6, 23, 24, 63, 48, 33, 34, 51, 36, 35, 36, 37, 54, 39, 40, 47, 32, 49, 50, 35, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt531(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 4, 5, 22, 7, 8, 31, 0, 17, 18, 3, 20, 19, 20, 21, 6, 23, 24, 47, 48, 33, 34, 51, 36, 35, 36, 37, 54, 39, 40, 63, 32, 49, 50, 35, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt532(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 4, 5, 22, 7, 8, 15, 16, 1, 18, 3, 20, 19, 20, 21, 6, 23, 24, 63, 32, 49, 34, 51, 36, 35, 36, 37, 54, 39, 40, 47, 48, 33, 50, 35, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt533(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 4, 5, 22, 7, 8, 31, 16, 1, 18, 3, 20, 19, 20, 21, 6, 23, 24, 47, 32, 49, 34, 51, 36, 35, 36, 37, 54, 39, 40, 63, 48, 33, 50, 35, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt534(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 4, 5, 22, 7, 8, 15, 0, 1, 18, 3, 20, 19, 20, 21, 6, 23, 24, 63, 48, 49, 34, 51, 36, 35, 36, 37, 54, 39, 40, 47, 32, 33, 50, 35, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt535(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 4, 5, 22, 7, 8, 31, 0, 1, 18, 3, 20, 19, 20, 21, 6, 23, 24, 47, 48, 49, 34, 51, 36, 35, 36, 37, 54, 39, 40, 63, 32, 33, 50, 35, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt536(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 4, 5, 22, 7, 8, 15, 16, 17, 2, 3, 20, 19, 20, 21, 6, 23, 24, 63, 32, 33, 50, 51, 36, 35, 36, 37, 54, 39, 40, 47, 48, 49, 34, 35, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt537(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 4, 5, 22, 7, 8, 31, 16, 17, 2, 3, 20, 19, 20, 21, 6, 23, 24, 47, 32, 33, 50, 51, 36, 35, 36, 37, 54, 39, 40, 63, 48, 49, 34, 35, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt538(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 4, 5, 22, 7, 8, 15, 0, 17, 2, 3, 20, 19, 20, 21, 6, 23, 24, 63, 48, 33, 50, 51, 36, 35, 36, 37, 54, 39, 40, 47, 32, 49, 34, 35, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt539(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 4, 5, 22, 7, 8, 31, 0, 17, 2, 3, 20, 19, 20, 21, 6, 23, 24, 47, 48, 33, 50, 51, 36, 35, 36, 37, 54, 39, 40, 63, 32, 49, 34, 35, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt540(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 4, 5, 22, 7, 8, 15, 16, 1, 2, 3, 20, 19, 20, 21, 6, 23, 24, 63, 32, 49, 50, 51, 36, 35, 36, 37, 54, 39, 40, 47, 48, 33, 34, 35, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt541(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 4, 5, 22, 7, 8, 31, 16, 1, 2, 3, 20, 19, 20, 21, 6, 23, 24, 47, 32, 49, 50, 51, 36, 35, 36, 37, 54, 39, 40, 63, 48, 33, 34, 35, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt542(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 4, 5, 22, 7, 8, 15, 0, 1, 2, 3, 20, 19, 20, 21, 6, 23, 24, 63, 48, 49, 50, 51, 36, 35, 36, 37, 54, 39, 40, 47, 32, 33, 34, 35, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt543(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 4, 5, 22, 7, 8, 31, 0, 1, 2, 3, 20, 19, 20, 21, 6, 23, 24, 47, 48, 49, 50, 51, 36, 35, 36, 37, 54, 39, 40, 63, 32, 33, 34, 35, 52, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt544(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 4, 5, 22, 7, 8, 15, 16, 17, 18, 19, 4, 19, 20, 21, 6, 23, 24, 63, 32, 33, 34, 35, 52, 35, 36, 37, 54, 39, 40, 47, 48, 49, 50, 51, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt545(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 4, 5, 22, 7, 8, 31, 16, 17, 18, 19, 4, 19, 20, 21, 6, 23, 24, 47, 32, 33, 34, 35, 52, 35, 36, 37, 54, 39, 40, 63, 48, 49, 50, 51, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt546(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 4, 5, 22, 7, 8, 15, 0, 17, 18, 19, 4, 19, 20, 21, 6, 23, 24, 63, 48, 33, 34, 35, 52, 35, 36, 37, 54, 39, 40, 47, 32, 49, 50, 51, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt547(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 4, 5, 22, 7, 8, 31, 0, 17, 18, 19, 4, 19, 20, 21, 6, 23, 24, 47, 48, 33, 34, 35, 52, 35, 36, 37, 54, 39, 40, 63, 32, 49, 50, 51, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt548(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 4, 5, 22, 7, 8, 15, 16, 1, 18, 19, 4, 19, 20, 21, 6, 23, 24, 63, 32, 49, 34, 35, 52, 35, 36, 37, 54, 39, 40, 47, 48, 33, 50, 51, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt549(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 4, 5, 22, 7, 8, 31, 16, 1, 18, 19, 4, 19, 20, 21, 6, 23, 24, 47, 32, 49, 34, 35, 52, 35, 36, 37, 54, 39, 40, 63, 48, 33, 50, 51, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt550(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 4, 5, 22, 7, 8, 15, 0, 1, 18, 19, 4, 19, 20, 21, 6, 23, 24, 63, 48, 49, 34, 35, 52, 35, 36, 37, 54, 39, 40, 47, 32, 33, 50, 51, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt551(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 4, 5, 22, 7, 8, 31, 0, 1, 18, 19, 4, 19, 20, 21, 6, 23, 24, 47, 48, 49, 34, 35, 52, 35, 36, 37, 54, 39, 40, 63, 32, 33, 50, 51, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt552(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 4, 5, 22, 7, 8, 15, 16, 17, 2, 19, 4, 19, 20, 21, 6, 23, 24, 63, 32, 33, 50, 35, 52, 35, 36, 37, 54, 39, 40, 47, 48, 49, 34, 51, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt553(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 4, 5, 22, 7, 8, 31, 16, 17, 2, 19, 4, 19, 20, 21, 6, 23, 24, 47, 32, 33, 50, 35, 52, 35, 36, 37, 54, 39, 40, 63, 48, 49, 34, 51, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt554(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 4, 5, 22, 7, 8, 15, 0, 17, 2, 19, 4, 19, 20, 21, 6, 23, 24, 63, 48, 33, 50, 35, 52, 35, 36, 37, 54, 39, 40, 47, 32, 49, 34, 51, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt555(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 4, 5, 22, 7, 8, 31, 0, 17, 2, 19, 4, 19, 20, 21, 6, 23, 24, 47, 48, 33, 50, 35, 52, 35, 36, 37, 54, 39, 40, 63, 32, 49, 34, 51, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt556(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 4, 5, 22, 7, 8, 15, 16, 1, 2, 19, 4, 19, 20, 21, 6, 23, 24, 63, 32, 49, 50, 35, 52, 35, 36, 37, 54, 39, 40, 47, 48, 33, 34, 51, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt557(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 4, 5, 22, 7, 8, 31, 16, 1, 2, 19, 4, 19, 20, 21, 6, 23, 24, 47, 32, 49, 50, 35, 52, 35, 36, 37, 54, 39, 40, 63, 48, 33, 34, 51, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt558(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 4, 5, 22, 7, 8, 15, 0, 1, 2, 19, 4, 19, 20, 21, 6, 23, 24, 63, 48, 49, 50, 35, 52, 35, 36, 37, 54, 39, 40, 47, 32, 33, 34, 51, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt559(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 4, 5, 22, 7, 8, 31, 0, 1, 2, 19, 4, 19, 20, 21, 6, 23, 24, 47, 48, 49, 50, 35, 52, 35, 36, 37, 54, 39, 40, 63, 32, 33, 34, 51, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt560(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 4, 5, 22, 7, 8, 15, 16, 17, 18, 3, 4, 19, 20, 21, 6, 23, 24, 63, 32, 33, 34, 51, 52, 35, 36, 37, 54, 39, 40, 47, 48, 49, 50, 35, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt561(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 4, 5, 22, 7, 8, 31, 16, 17, 18, 3, 4, 19, 20, 21, 6, 23, 24, 47, 32, 33, 34, 51, 52, 35, 36, 37, 54, 39, 40, 63, 48, 49, 50, 35, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt562(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 4, 5, 22, 7, 8, 15, 0, 17, 18, 3, 4, 19, 20, 21, 6, 23, 24, 63, 48, 33, 34, 51, 52, 35, 36, 37, 54, 39, 40, 47, 32, 49, 50, 35, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt563(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 4, 5, 22, 7, 8, 31, 0, 17, 18, 3, 4, 19, 20, 21, 6, 23, 24, 47, 48, 33, 34, 51, 52, 35, 36, 37, 54, 39, 40, 63, 32, 49, 50, 35, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt564(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 4, 5, 22, 7, 8, 15, 16, 1, 18, 3, 4, 19, 20, 21, 6, 23, 24, 63, 32, 49, 34, 51, 52, 35, 36, 37, 54, 39, 40, 47, 48, 33, 50, 35, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt565(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 4, 5, 22, 7, 8, 31, 16, 1, 18, 3, 4, 19, 20, 21, 6, 23, 24, 47, 32, 49, 34, 51, 52, 35, 36, 37, 54, 39, 40, 63, 48, 33, 50, 35, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt566(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 4, 5, 22, 7, 8, 15, 0, 1, 18, 3, 4, 19, 20, 21, 6, 23, 24, 63, 48, 49, 34, 51, 52, 35, 36, 37, 54, 39, 40, 47, 32, 33, 50, 35, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt567(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 4, 5, 22, 7, 8, 31, 0, 1, 18, 3, 4, 19, 20, 21, 6, 23, 24, 47, 48, 49, 34, 51, 52, 35, 36, 37, 54, 39, 40, 63, 32, 33, 50, 35, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt568(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 4, 5, 22, 7, 8, 15, 16, 17, 2, 3, 4, 19, 20, 21, 6, 23, 24, 63, 32, 33, 50, 51, 52, 35, 36, 37, 54, 39, 40, 47, 48, 49, 34, 35, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt569(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 4, 5, 22, 7, 8, 31, 16, 17, 2, 3, 4, 19, 20, 21, 6, 23, 24, 47, 32, 33, 50, 51, 52, 35, 36, 37, 54, 39, 40, 63, 48, 49, 34, 35, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt570(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 4, 5, 22, 7, 8, 15, 0, 17, 2, 3, 4, 19, 20, 21, 6, 23, 24, 63, 48, 33, 50, 51, 52, 35, 36, 37, 54, 39, 40, 47, 32, 49, 34, 35, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt571(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 4, 5, 22, 7, 8, 31, 0, 17, 2, 3, 4, 19, 20, 21, 6, 23, 24, 47, 48, 33, 50, 51, 52, 35, 36, 37, 54, 39, 40, 63, 32, 49, 34, 35, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt572(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 4, 5, 22, 7, 8, 15, 16, 1, 2, 3, 4, 19, 20, 21, 6, 23, 24, 63, 32, 49, 50, 51, 52, 35, 36, 37, 54, 39, 40, 47, 48, 33, 34, 35, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt573(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 4, 5, 22, 7, 8, 31, 16, 1, 2, 3, 4, 19, 20, 21, 6, 23, 24, 47, 32, 49, 50, 51, 52, 35, 36, 37, 54, 39, 40, 63, 48, 33, 34, 35, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt574(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 4, 5, 22, 7, 8, 15, 0, 1, 2, 3, 4, 19, 20, 21, 6, 23, 24, 63, 48, 49, 50, 51, 52, 35, 36, 37, 54, 39, 40, 47, 32, 33, 34, 35, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt575(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 4, 5, 22, 7, 8, 31, 0, 1, 2, 3, 4, 19, 20, 21, 6, 23, 24, 47, 48, 49, 50, 51, 52, 35, 36, 37, 54, 39, 40, 63, 32, 33, 34, 35, 36, 51, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt576(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 4, 5, 22, 7, 8, 15, 16, 17, 18, 19, 20, 3, 20, 21, 6, 23, 24, 63, 32, 33, 34, 35, 36, 51, 36, 37, 54, 39, 40, 47, 48, 49, 50, 51, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt577(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 4, 5, 22, 7, 8, 31, 16, 17, 18, 19, 20, 3, 20, 21, 6, 23, 24, 47, 32, 33, 34, 35, 36, 51, 36, 37, 54, 39, 40, 63, 48, 49, 50, 51, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt578(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 4, 5, 22, 7, 8, 15, 0, 17, 18, 19, 20, 3, 20, 21, 6, 23, 24, 63, 48, 33, 34, 35, 36, 51, 36, 37, 54, 39, 40, 47, 32, 49, 50, 51, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt579(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 4, 5, 22, 7, 8, 31, 0, 17, 18, 19, 20, 3, 20, 21, 6, 23, 24, 47, 48, 33, 34, 35, 36, 51, 36, 37, 54, 39, 40, 63, 32, 49, 50, 51, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt580(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 4, 5, 22, 7, 8, 15, 16, 1, 18, 19, 20, 3, 20, 21, 6, 23, 24, 63, 32, 49, 34, 35, 36, 51, 36, 37, 54, 39, 40, 47, 48, 33, 50, 51, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt581(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 4, 5, 22, 7, 8, 31, 16, 1, 18, 19, 20, 3, 20, 21, 6, 23, 24, 47, 32, 49, 34, 35, 36, 51, 36, 37, 54, 39, 40, 63, 48, 33, 50, 51, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt582(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 4, 5, 22, 7, 8, 15, 0, 1, 18, 19, 20, 3, 20, 21, 6, 23, 24, 63, 48, 49, 34, 35, 36, 51, 36, 37, 54, 39, 40, 47, 32, 33, 50, 51, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt583(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 4, 5, 22, 7, 8, 31, 0, 1, 18, 19, 20, 3, 20, 21, 6, 23, 24, 47, 48, 49, 34, 35, 36, 51, 36, 37, 54, 39, 40, 63, 32, 33, 50, 51, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt584(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 4, 5, 22, 7, 8, 15, 16, 17, 2, 19, 20, 3, 20, 21, 6, 23, 24, 63, 32, 33, 50, 35, 36, 51, 36, 37, 54, 39, 40, 47, 48, 49, 34, 51, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt585(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 4, 5, 22, 7, 8, 31, 16, 17, 2, 19, 20, 3, 20, 21, 6, 23, 24, 47, 32, 33, 50, 35, 36, 51, 36, 37, 54, 39, 40, 63, 48, 49, 34, 51, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt586(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 4, 5, 22, 7, 8, 15, 0, 17, 2, 19, 20, 3, 20, 21, 6, 23, 24, 63, 48, 33, 50, 35, 36, 51, 36, 37, 54, 39, 40, 47, 32, 49, 34, 51, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt587(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 4, 5, 22, 7, 8, 31, 0, 17, 2, 19, 20, 3, 20, 21, 6, 23, 24, 47, 48, 33, 50, 35, 36, 51, 36, 37, 54, 39, 40, 63, 32, 49, 34, 51, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt588(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 4, 5, 22, 7, 8, 15, 16, 1, 2, 19, 20, 3, 20, 21, 6, 23, 24, 63, 32, 49, 50, 35, 36, 51, 36, 37, 54, 39, 40, 47, 48, 33, 34, 51, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt589(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 4, 5, 22, 7, 8, 31, 16, 1, 2, 19, 20, 3, 20, 21, 6, 23, 24, 47, 32, 49, 50, 35, 36, 51, 36, 37, 54, 39, 40, 63, 48, 33, 34, 51, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt590(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 4, 5, 22, 7, 8, 15, 0, 1, 2, 19, 20, 3, 20, 21, 6, 23, 24, 63, 48, 49, 50, 35, 36, 51, 36, 37, 54, 39, 40, 47, 32, 33, 34, 51, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt591(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 4, 5, 22, 7, 8, 31, 0, 1, 2, 19, 20, 3, 20, 21, 6, 23, 24, 47, 48, 49, 50, 35, 36, 51, 36, 37, 54, 39, 40, 63, 32, 33, 34, 51, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt592(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 4, 5, 22, 7, 8, 15, 16, 17, 18, 3, 20, 3, 20, 21, 6, 23, 24, 63, 32, 33, 34, 51, 36, 51, 36, 37, 54, 39, 40, 47, 48, 49, 50, 35, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt593(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 4, 5, 22, 7, 8, 31, 16, 17, 18, 3, 20, 3, 20, 21, 6, 23, 24, 47, 32, 33, 34, 51, 36, 51, 36, 37, 54, 39, 40, 63, 48, 49, 50, 35, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt594(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 4, 5, 22, 7, 8, 15, 0, 17, 18, 3, 20, 3, 20, 21, 6, 23, 24, 63, 48, 33, 34, 51, 36, 51, 36, 37, 54, 39, 40, 47, 32, 49, 50, 35, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt595(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 4, 5, 22, 7, 8, 31, 0, 17, 18, 3, 20, 3, 20, 21, 6, 23, 24, 47, 48, 33, 34, 51, 36, 51, 36, 37, 54, 39, 40, 63, 32, 49, 50, 35, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt596(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 4, 5, 22, 7, 8, 15, 16, 1, 18, 3, 20, 3, 20, 21, 6, 23, 24, 63, 32, 49, 34, 51, 36, 51, 36, 37, 54, 39, 40, 47, 48, 33, 50, 35, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt597(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 4, 5, 22, 7, 8, 31, 16, 1, 18, 3, 20, 3, 20, 21, 6, 23, 24, 47, 32, 49, 34, 51, 36, 51, 36, 37, 54, 39, 40, 63, 48, 33, 50, 35, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt598(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 4, 5, 22, 7, 8, 15, 0, 1, 18, 3, 20, 3, 20, 21, 6, 23, 24, 63, 48, 49, 34, 51, 36, 51, 36, 37, 54, 39, 40, 47, 32, 33, 50, 35, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt599(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 4, 5, 22, 7, 8, 31, 0, 1, 18, 3, 20, 3, 20, 21, 6, 23, 24, 47, 48, 49, 34, 51, 36, 51, 36, 37, 54, 39, 40, 63, 32, 33, 50, 35, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt600(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 4, 5, 22, 7, 8, 15, 16, 17, 2, 3, 20, 3, 20, 21, 6, 23, 24, 63, 32, 33, 50, 51, 36, 51, 36, 37, 54, 39, 40, 47, 48, 49, 34, 35, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt601(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 4, 5, 22, 7, 8, 31, 16, 17, 2, 3, 20, 3, 20, 21, 6, 23, 24, 47, 32, 33, 50, 51, 36, 51, 36, 37, 54, 39, 40, 63, 48, 49, 34, 35, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt602(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 4, 5, 22, 7, 8, 15, 0, 17, 2, 3, 20, 3, 20, 21, 6, 23, 24, 63, 48, 33, 50, 51, 36, 51, 36, 37, 54, 39, 40, 47, 32, 49, 34, 35, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt603(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 4, 5, 22, 7, 8, 31, 0, 17, 2, 3, 20, 3, 20, 21, 6, 23, 24, 47, 48, 33, 50, 51, 36, 51, 36, 37, 54, 39, 40, 63, 32, 49, 34, 35, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt604(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 4, 5, 22, 7, 8, 15, 16, 1, 2, 3, 20, 3, 20, 21, 6, 23, 24, 63, 32, 49, 50, 51, 36, 51, 36, 37, 54, 39, 40, 47, 48, 33, 34, 35, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt605(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 4, 5, 22, 7, 8, 31, 16, 1, 2, 3, 20, 3, 20, 21, 6, 23, 24, 47, 32, 49, 50, 51, 36, 51, 36, 37, 54, 39, 40, 63, 48, 33, 34, 35, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt606(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 4, 5, 22, 7, 8, 15, 0, 1, 2, 3, 20, 3, 20, 21, 6, 23, 24, 63, 48, 49, 50, 51, 36, 51, 36, 37, 54, 39, 40, 47, 32, 33, 34, 35, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt607(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 4, 5, 22, 7, 8, 31, 0, 1, 2, 3, 20, 3, 20, 21, 6, 23, 24, 47, 48, 49, 50, 51, 36, 51, 36, 37, 54, 39, 40, 63, 32, 33, 34, 35, 52, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt608(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 4, 5, 22, 7, 8, 15, 16, 17, 18, 19, 4, 3, 20, 21, 6, 23, 24, 63, 32, 33, 34, 35, 52, 51, 36, 37, 54, 39, 40, 47, 48, 49, 50, 51, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt609(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 4, 5, 22, 7, 8, 31, 16, 17, 18, 19, 4, 3, 20, 21, 6, 23, 24, 47, 32, 33, 34, 35, 52, 51, 36, 37, 54, 39, 40, 63, 48, 49, 50, 51, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt610(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 4, 5, 22, 7, 8, 15, 0, 17, 18, 19, 4, 3, 20, 21, 6, 23, 24, 63, 48, 33, 34, 35, 52, 51, 36, 37, 54, 39, 40, 47, 32, 49, 50, 51, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt611(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 4, 5, 22, 7, 8, 31, 0, 17, 18, 19, 4, 3, 20, 21, 6, 23, 24, 47, 48, 33, 34, 35, 52, 51, 36, 37, 54, 39, 40, 63, 32, 49, 50, 51, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt612(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 4, 5, 22, 7, 8, 15, 16, 1, 18, 19, 4, 3, 20, 21, 6, 23, 24, 63, 32, 49, 34, 35, 52, 51, 36, 37, 54, 39, 40, 47, 48, 33, 50, 51, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt613(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 4, 5, 22, 7, 8, 31, 16, 1, 18, 19, 4, 3, 20, 21, 6, 23, 24, 47, 32, 49, 34, 35, 52, 51, 36, 37, 54, 39, 40, 63, 48, 33, 50, 51, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt614(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 4, 5, 22, 7, 8, 15, 0, 1, 18, 19, 4, 3, 20, 21, 6, 23, 24, 63, 48, 49, 34, 35, 52, 51, 36, 37, 54, 39, 40, 47, 32, 33, 50, 51, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt615(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 4, 5, 22, 7, 8, 31, 0, 1, 18, 19, 4, 3, 20, 21, 6, 23, 24, 47, 48, 49, 34, 35, 52, 51, 36, 37, 54, 39, 40, 63, 32, 33, 50, 51, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt616(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 4, 5, 22, 7, 8, 15, 16, 17, 2, 19, 4, 3, 20, 21, 6, 23, 24, 63, 32, 33, 50, 35, 52, 51, 36, 37, 54, 39, 40, 47, 48, 49, 34, 51, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt617(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 4, 5, 22, 7, 8, 31, 16, 17, 2, 19, 4, 3, 20, 21, 6, 23, 24, 47, 32, 33, 50, 35, 52, 51, 36, 37, 54, 39, 40, 63, 48, 49, 34, 51, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt618(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 4, 5, 22, 7, 8, 15, 0, 17, 2, 19, 4, 3, 20, 21, 6, 23, 24, 63, 48, 33, 50, 35, 52, 51, 36, 37, 54, 39, 40, 47, 32, 49, 34, 51, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt619(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 4, 5, 22, 7, 8, 31, 0, 17, 2, 19, 4, 3, 20, 21, 6, 23, 24, 47, 48, 33, 50, 35, 52, 51, 36, 37, 54, 39, 40, 63, 32, 49, 34, 51, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt620(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 4, 5, 22, 7, 8, 15, 16, 1, 2, 19, 4, 3, 20, 21, 6, 23, 24, 63, 32, 49, 50, 35, 52, 51, 36, 37, 54, 39, 40, 47, 48, 33, 34, 51, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt621(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 4, 5, 22, 7, 8, 31, 16, 1, 2, 19, 4, 3, 20, 21, 6, 23, 24, 47, 32, 49, 50, 35, 52, 51, 36, 37, 54, 39, 40, 63, 48, 33, 34, 51, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt622(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 4, 5, 22, 7, 8, 15, 0, 1, 2, 19, 4, 3, 20, 21, 6, 23, 24, 63, 48, 49, 50, 35, 52, 51, 36, 37, 54, 39, 40, 47, 32, 33, 34, 51, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt623(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 4, 5, 22, 7, 8, 31, 0, 1, 2, 19, 4, 3, 20, 21, 6, 23, 24, 47, 48, 49, 50, 35, 52, 51, 36, 37, 54, 39, 40, 63, 32, 33, 34, 51, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt624(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 4, 5, 22, 7, 8, 15, 16, 17, 18, 3, 4, 3, 20, 21, 6, 23, 24, 63, 32, 33, 34, 51, 52, 51, 36, 37, 54, 39, 40, 47, 48, 49, 50, 35, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt625(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 4, 5, 22, 7, 8, 31, 16, 17, 18, 3, 4, 3, 20, 21, 6, 23, 24, 47, 32, 33, 34, 51, 52, 51, 36, 37, 54, 39, 40, 63, 48, 49, 50, 35, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt626(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 4, 5, 22, 7, 8, 15, 0, 17, 18, 3, 4, 3, 20, 21, 6, 23, 24, 63, 48, 33, 34, 51, 52, 51, 36, 37, 54, 39, 40, 47, 32, 49, 50, 35, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt627(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 4, 5, 22, 7, 8, 31, 0, 17, 18, 3, 4, 3, 20, 21, 6, 23, 24, 47, 48, 33, 34, 51, 52, 51, 36, 37, 54, 39, 40, 63, 32, 49, 50, 35, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt628(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 4, 5, 22, 7, 8, 15, 16, 1, 18, 3, 4, 3, 20, 21, 6, 23, 24, 63, 32, 49, 34, 51, 52, 51, 36, 37, 54, 39, 40, 47, 48, 33, 50, 35, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt629(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 4, 5, 22, 7, 8, 31, 16, 1, 18, 3, 4, 3, 20, 21, 6, 23, 24, 47, 32, 49, 34, 51, 52, 51, 36, 37, 54, 39, 40, 63, 48, 33, 50, 35, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt630(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 4, 5, 22, 7, 8, 15, 0, 1, 18, 3, 4, 3, 20, 21, 6, 23, 24, 63, 48, 49, 34, 51, 52, 51, 36, 37, 54, 39, 40, 47, 32, 33, 50, 35, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt631(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 4, 5, 22, 7, 8, 31, 0, 1, 18, 3, 4, 3, 20, 21, 6, 23, 24, 47, 48, 49, 34, 51, 52, 51, 36, 37, 54, 39, 40, 63, 32, 33, 50, 35, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt632(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 4, 5, 22, 7, 8, 15, 16, 17, 2, 3, 4, 3, 20, 21, 6, 23, 24, 63, 32, 33, 50, 51, 52, 51, 36, 37, 54, 39, 40, 47, 48, 49, 34, 35, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt633(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 4, 5, 22, 7, 8, 31, 16, 17, 2, 3, 4, 3, 20, 21, 6, 23, 24, 47, 32, 33, 50, 51, 52, 51, 36, 37, 54, 39, 40, 63, 48, 49, 34, 35, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt634(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 4, 5, 22, 7, 8, 15, 0, 17, 2, 3, 4, 3, 20, 21, 6, 23, 24, 63, 48, 33, 50, 51, 52, 51, 36, 37, 54, 39, 40, 47, 32, 49, 34, 35, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt635(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 4, 5, 22, 7, 8, 31, 0, 17, 2, 3, 4, 3, 20, 21, 6, 23, 24, 47, 48, 33, 50, 51, 52, 51, 36, 37, 54, 39, 40, 63, 32, 49, 34, 35, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt636(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 4, 5, 22, 7, 8, 15, 16, 1, 2, 3, 4, 3, 20, 21, 6, 23, 24, 63, 32, 49, 50, 51, 52, 51, 36, 37, 54, 39, 40, 47, 48, 33, 34, 35, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt637(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 4, 5, 22, 7, 8, 31, 16, 1, 2, 3, 4, 3, 20, 21, 6, 23, 24, 47, 32, 49, 50, 51, 52, 51, 36, 37, 54, 39, 40, 63, 48, 33, 34, 35, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt638(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 4, 5, 22, 7, 8, 15, 0, 1, 2, 3, 4, 3, 20, 21, 6, 23, 24, 63, 48, 49, 50, 51, 52, 51, 36, 37, 54, 39, 40, 47, 32, 33, 34, 35, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt639(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 4, 5, 22, 7, 8, 31, 0, 1, 2, 3, 4, 3, 20, 21, 6, 23, 24, 47, 48, 49, 50, 51, 52, 51, 36, 37, 54, 39, 40, 63, 32, 33, 34, 35, 36, 35, 52, 53, 38, 55, 56>(unchecked_hashes, bitsplitted_keys);
}


