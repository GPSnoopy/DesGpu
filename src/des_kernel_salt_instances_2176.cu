#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt2176(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 5, 6, 7, 24, 15, 16, 17, 18, 19, 20, 19, 4, 21, 22, 23, 8, 63, 32, 33, 34, 35, 36, 35, 52, 37, 38, 39, 56, 47, 48, 49, 50, 51, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2177(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 5, 6, 7, 24, 31, 16, 17, 18, 19, 20, 19, 4, 21, 22, 23, 8, 47, 32, 33, 34, 35, 36, 35, 52, 37, 38, 39, 56, 63, 48, 49, 50, 51, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2178(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 5, 6, 7, 24, 15, 0, 17, 18, 19, 20, 19, 4, 21, 22, 23, 8, 63, 48, 33, 34, 35, 36, 35, 52, 37, 38, 39, 56, 47, 32, 49, 50, 51, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2179(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 5, 6, 7, 24, 31, 0, 17, 18, 19, 20, 19, 4, 21, 22, 23, 8, 47, 48, 33, 34, 35, 36, 35, 52, 37, 38, 39, 56, 63, 32, 49, 50, 51, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2180(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 5, 6, 7, 24, 15, 16, 1, 18, 19, 20, 19, 4, 21, 22, 23, 8, 63, 32, 49, 34, 35, 36, 35, 52, 37, 38, 39, 56, 47, 48, 33, 50, 51, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2181(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 5, 6, 7, 24, 31, 16, 1, 18, 19, 20, 19, 4, 21, 22, 23, 8, 47, 32, 49, 34, 35, 36, 35, 52, 37, 38, 39, 56, 63, 48, 33, 50, 51, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2182(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 5, 6, 7, 24, 15, 0, 1, 18, 19, 20, 19, 4, 21, 22, 23, 8, 63, 48, 49, 34, 35, 36, 35, 52, 37, 38, 39, 56, 47, 32, 33, 50, 51, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2183(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 5, 6, 7, 24, 31, 0, 1, 18, 19, 20, 19, 4, 21, 22, 23, 8, 47, 48, 49, 34, 35, 36, 35, 52, 37, 38, 39, 56, 63, 32, 33, 50, 51, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2184(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 5, 6, 7, 24, 15, 16, 17, 2, 19, 20, 19, 4, 21, 22, 23, 8, 63, 32, 33, 50, 35, 36, 35, 52, 37, 38, 39, 56, 47, 48, 49, 34, 51, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2185(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 5, 6, 7, 24, 31, 16, 17, 2, 19, 20, 19, 4, 21, 22, 23, 8, 47, 32, 33, 50, 35, 36, 35, 52, 37, 38, 39, 56, 63, 48, 49, 34, 51, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2186(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 5, 6, 7, 24, 15, 0, 17, 2, 19, 20, 19, 4, 21, 22, 23, 8, 63, 48, 33, 50, 35, 36, 35, 52, 37, 38, 39, 56, 47, 32, 49, 34, 51, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2187(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 5, 6, 7, 24, 31, 0, 17, 2, 19, 20, 19, 4, 21, 22, 23, 8, 47, 48, 33, 50, 35, 36, 35, 52, 37, 38, 39, 56, 63, 32, 49, 34, 51, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2188(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 5, 6, 7, 24, 15, 16, 1, 2, 19, 20, 19, 4, 21, 22, 23, 8, 63, 32, 49, 50, 35, 36, 35, 52, 37, 38, 39, 56, 47, 48, 33, 34, 51, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2189(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 5, 6, 7, 24, 31, 16, 1, 2, 19, 20, 19, 4, 21, 22, 23, 8, 47, 32, 49, 50, 35, 36, 35, 52, 37, 38, 39, 56, 63, 48, 33, 34, 51, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2190(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 5, 6, 7, 24, 15, 0, 1, 2, 19, 20, 19, 4, 21, 22, 23, 8, 63, 48, 49, 50, 35, 36, 35, 52, 37, 38, 39, 56, 47, 32, 33, 34, 51, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2191(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 5, 6, 7, 24, 31, 0, 1, 2, 19, 20, 19, 4, 21, 22, 23, 8, 47, 48, 49, 50, 35, 36, 35, 52, 37, 38, 39, 56, 63, 32, 33, 34, 51, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2192(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 5, 6, 7, 24, 15, 16, 17, 18, 3, 20, 19, 4, 21, 22, 23, 8, 63, 32, 33, 34, 51, 36, 35, 52, 37, 38, 39, 56, 47, 48, 49, 50, 35, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2193(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 5, 6, 7, 24, 31, 16, 17, 18, 3, 20, 19, 4, 21, 22, 23, 8, 47, 32, 33, 34, 51, 36, 35, 52, 37, 38, 39, 56, 63, 48, 49, 50, 35, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2194(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 5, 6, 7, 24, 15, 0, 17, 18, 3, 20, 19, 4, 21, 22, 23, 8, 63, 48, 33, 34, 51, 36, 35, 52, 37, 38, 39, 56, 47, 32, 49, 50, 35, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2195(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 5, 6, 7, 24, 31, 0, 17, 18, 3, 20, 19, 4, 21, 22, 23, 8, 47, 48, 33, 34, 51, 36, 35, 52, 37, 38, 39, 56, 63, 32, 49, 50, 35, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2196(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 5, 6, 7, 24, 15, 16, 1, 18, 3, 20, 19, 4, 21, 22, 23, 8, 63, 32, 49, 34, 51, 36, 35, 52, 37, 38, 39, 56, 47, 48, 33, 50, 35, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2197(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 5, 6, 7, 24, 31, 16, 1, 18, 3, 20, 19, 4, 21, 22, 23, 8, 47, 32, 49, 34, 51, 36, 35, 52, 37, 38, 39, 56, 63, 48, 33, 50, 35, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2198(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 5, 6, 7, 24, 15, 0, 1, 18, 3, 20, 19, 4, 21, 22, 23, 8, 63, 48, 49, 34, 51, 36, 35, 52, 37, 38, 39, 56, 47, 32, 33, 50, 35, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2199(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 5, 6, 7, 24, 31, 0, 1, 18, 3, 20, 19, 4, 21, 22, 23, 8, 47, 48, 49, 34, 51, 36, 35, 52, 37, 38, 39, 56, 63, 32, 33, 50, 35, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2200(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 5, 6, 7, 24, 15, 16, 17, 2, 3, 20, 19, 4, 21, 22, 23, 8, 63, 32, 33, 50, 51, 36, 35, 52, 37, 38, 39, 56, 47, 48, 49, 34, 35, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2201(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 5, 6, 7, 24, 31, 16, 17, 2, 3, 20, 19, 4, 21, 22, 23, 8, 47, 32, 33, 50, 51, 36, 35, 52, 37, 38, 39, 56, 63, 48, 49, 34, 35, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2202(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 5, 6, 7, 24, 15, 0, 17, 2, 3, 20, 19, 4, 21, 22, 23, 8, 63, 48, 33, 50, 51, 36, 35, 52, 37, 38, 39, 56, 47, 32, 49, 34, 35, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2203(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 5, 6, 7, 24, 31, 0, 17, 2, 3, 20, 19, 4, 21, 22, 23, 8, 47, 48, 33, 50, 51, 36, 35, 52, 37, 38, 39, 56, 63, 32, 49, 34, 35, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2204(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 5, 6, 7, 24, 15, 16, 1, 2, 3, 20, 19, 4, 21, 22, 23, 8, 63, 32, 49, 50, 51, 36, 35, 52, 37, 38, 39, 56, 47, 48, 33, 34, 35, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2205(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 5, 6, 7, 24, 31, 16, 1, 2, 3, 20, 19, 4, 21, 22, 23, 8, 47, 32, 49, 50, 51, 36, 35, 52, 37, 38, 39, 56, 63, 48, 33, 34, 35, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2206(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 5, 6, 7, 24, 15, 0, 1, 2, 3, 20, 19, 4, 21, 22, 23, 8, 63, 48, 49, 50, 51, 36, 35, 52, 37, 38, 39, 56, 47, 32, 33, 34, 35, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2207(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 5, 6, 7, 24, 31, 0, 1, 2, 3, 20, 19, 4, 21, 22, 23, 8, 47, 48, 49, 50, 51, 36, 35, 52, 37, 38, 39, 56, 63, 32, 33, 34, 35, 52, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2208(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 5, 6, 7, 24, 15, 16, 17, 18, 19, 4, 19, 4, 21, 22, 23, 8, 63, 32, 33, 34, 35, 52, 35, 52, 37, 38, 39, 56, 47, 48, 49, 50, 51, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2209(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 5, 6, 7, 24, 31, 16, 17, 18, 19, 4, 19, 4, 21, 22, 23, 8, 47, 32, 33, 34, 35, 52, 35, 52, 37, 38, 39, 56, 63, 48, 49, 50, 51, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2210(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 5, 6, 7, 24, 15, 0, 17, 18, 19, 4, 19, 4, 21, 22, 23, 8, 63, 48, 33, 34, 35, 52, 35, 52, 37, 38, 39, 56, 47, 32, 49, 50, 51, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2211(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 5, 6, 7, 24, 31, 0, 17, 18, 19, 4, 19, 4, 21, 22, 23, 8, 47, 48, 33, 34, 35, 52, 35, 52, 37, 38, 39, 56, 63, 32, 49, 50, 51, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2212(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 5, 6, 7, 24, 15, 16, 1, 18, 19, 4, 19, 4, 21, 22, 23, 8, 63, 32, 49, 34, 35, 52, 35, 52, 37, 38, 39, 56, 47, 48, 33, 50, 51, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2213(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 5, 6, 7, 24, 31, 16, 1, 18, 19, 4, 19, 4, 21, 22, 23, 8, 47, 32, 49, 34, 35, 52, 35, 52, 37, 38, 39, 56, 63, 48, 33, 50, 51, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2214(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 5, 6, 7, 24, 15, 0, 1, 18, 19, 4, 19, 4, 21, 22, 23, 8, 63, 48, 49, 34, 35, 52, 35, 52, 37, 38, 39, 56, 47, 32, 33, 50, 51, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2215(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 5, 6, 7, 24, 31, 0, 1, 18, 19, 4, 19, 4, 21, 22, 23, 8, 47, 48, 49, 34, 35, 52, 35, 52, 37, 38, 39, 56, 63, 32, 33, 50, 51, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2216(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 5, 6, 7, 24, 15, 16, 17, 2, 19, 4, 19, 4, 21, 22, 23, 8, 63, 32, 33, 50, 35, 52, 35, 52, 37, 38, 39, 56, 47, 48, 49, 34, 51, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2217(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 5, 6, 7, 24, 31, 16, 17, 2, 19, 4, 19, 4, 21, 22, 23, 8, 47, 32, 33, 50, 35, 52, 35, 52, 37, 38, 39, 56, 63, 48, 49, 34, 51, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2218(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 5, 6, 7, 24, 15, 0, 17, 2, 19, 4, 19, 4, 21, 22, 23, 8, 63, 48, 33, 50, 35, 52, 35, 52, 37, 38, 39, 56, 47, 32, 49, 34, 51, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2219(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 5, 6, 7, 24, 31, 0, 17, 2, 19, 4, 19, 4, 21, 22, 23, 8, 47, 48, 33, 50, 35, 52, 35, 52, 37, 38, 39, 56, 63, 32, 49, 34, 51, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2220(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 5, 6, 7, 24, 15, 16, 1, 2, 19, 4, 19, 4, 21, 22, 23, 8, 63, 32, 49, 50, 35, 52, 35, 52, 37, 38, 39, 56, 47, 48, 33, 34, 51, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2221(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 5, 6, 7, 24, 31, 16, 1, 2, 19, 4, 19, 4, 21, 22, 23, 8, 47, 32, 49, 50, 35, 52, 35, 52, 37, 38, 39, 56, 63, 48, 33, 34, 51, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2222(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 5, 6, 7, 24, 15, 0, 1, 2, 19, 4, 19, 4, 21, 22, 23, 8, 63, 48, 49, 50, 35, 52, 35, 52, 37, 38, 39, 56, 47, 32, 33, 34, 51, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2223(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 5, 6, 7, 24, 31, 0, 1, 2, 19, 4, 19, 4, 21, 22, 23, 8, 47, 48, 49, 50, 35, 52, 35, 52, 37, 38, 39, 56, 63, 32, 33, 34, 51, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2224(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 5, 6, 7, 24, 15, 16, 17, 18, 3, 4, 19, 4, 21, 22, 23, 8, 63, 32, 33, 34, 51, 52, 35, 52, 37, 38, 39, 56, 47, 48, 49, 50, 35, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2225(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 5, 6, 7, 24, 31, 16, 17, 18, 3, 4, 19, 4, 21, 22, 23, 8, 47, 32, 33, 34, 51, 52, 35, 52, 37, 38, 39, 56, 63, 48, 49, 50, 35, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2226(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 5, 6, 7, 24, 15, 0, 17, 18, 3, 4, 19, 4, 21, 22, 23, 8, 63, 48, 33, 34, 51, 52, 35, 52, 37, 38, 39, 56, 47, 32, 49, 50, 35, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2227(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 5, 6, 7, 24, 31, 0, 17, 18, 3, 4, 19, 4, 21, 22, 23, 8, 47, 48, 33, 34, 51, 52, 35, 52, 37, 38, 39, 56, 63, 32, 49, 50, 35, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2228(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 5, 6, 7, 24, 15, 16, 1, 18, 3, 4, 19, 4, 21, 22, 23, 8, 63, 32, 49, 34, 51, 52, 35, 52, 37, 38, 39, 56, 47, 48, 33, 50, 35, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2229(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 5, 6, 7, 24, 31, 16, 1, 18, 3, 4, 19, 4, 21, 22, 23, 8, 47, 32, 49, 34, 51, 52, 35, 52, 37, 38, 39, 56, 63, 48, 33, 50, 35, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2230(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 5, 6, 7, 24, 15, 0, 1, 18, 3, 4, 19, 4, 21, 22, 23, 8, 63, 48, 49, 34, 51, 52, 35, 52, 37, 38, 39, 56, 47, 32, 33, 50, 35, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2231(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 5, 6, 7, 24, 31, 0, 1, 18, 3, 4, 19, 4, 21, 22, 23, 8, 47, 48, 49, 34, 51, 52, 35, 52, 37, 38, 39, 56, 63, 32, 33, 50, 35, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2232(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 5, 6, 7, 24, 15, 16, 17, 2, 3, 4, 19, 4, 21, 22, 23, 8, 63, 32, 33, 50, 51, 52, 35, 52, 37, 38, 39, 56, 47, 48, 49, 34, 35, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2233(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 5, 6, 7, 24, 31, 16, 17, 2, 3, 4, 19, 4, 21, 22, 23, 8, 47, 32, 33, 50, 51, 52, 35, 52, 37, 38, 39, 56, 63, 48, 49, 34, 35, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2234(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 5, 6, 7, 24, 15, 0, 17, 2, 3, 4, 19, 4, 21, 22, 23, 8, 63, 48, 33, 50, 51, 52, 35, 52, 37, 38, 39, 56, 47, 32, 49, 34, 35, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2235(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 5, 6, 7, 24, 31, 0, 17, 2, 3, 4, 19, 4, 21, 22, 23, 8, 47, 48, 33, 50, 51, 52, 35, 52, 37, 38, 39, 56, 63, 32, 49, 34, 35, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2236(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 5, 6, 7, 24, 15, 16, 1, 2, 3, 4, 19, 4, 21, 22, 23, 8, 63, 32, 49, 50, 51, 52, 35, 52, 37, 38, 39, 56, 47, 48, 33, 34, 35, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2237(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 5, 6, 7, 24, 31, 16, 1, 2, 3, 4, 19, 4, 21, 22, 23, 8, 47, 32, 49, 50, 51, 52, 35, 52, 37, 38, 39, 56, 63, 48, 33, 34, 35, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2238(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 5, 6, 7, 24, 15, 0, 1, 2, 3, 4, 19, 4, 21, 22, 23, 8, 63, 48, 49, 50, 51, 52, 35, 52, 37, 38, 39, 56, 47, 32, 33, 34, 35, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2239(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 5, 6, 7, 24, 31, 0, 1, 2, 3, 4, 19, 4, 21, 22, 23, 8, 47, 48, 49, 50, 51, 52, 35, 52, 37, 38, 39, 56, 63, 32, 33, 34, 35, 36, 51, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2240(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 5, 6, 7, 24, 15, 16, 17, 18, 19, 20, 3, 4, 21, 22, 23, 8, 63, 32, 33, 34, 35, 36, 51, 52, 37, 38, 39, 56, 47, 48, 49, 50, 51, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2241(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 5, 6, 7, 24, 31, 16, 17, 18, 19, 20, 3, 4, 21, 22, 23, 8, 47, 32, 33, 34, 35, 36, 51, 52, 37, 38, 39, 56, 63, 48, 49, 50, 51, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2242(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 5, 6, 7, 24, 15, 0, 17, 18, 19, 20, 3, 4, 21, 22, 23, 8, 63, 48, 33, 34, 35, 36, 51, 52, 37, 38, 39, 56, 47, 32, 49, 50, 51, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2243(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 5, 6, 7, 24, 31, 0, 17, 18, 19, 20, 3, 4, 21, 22, 23, 8, 47, 48, 33, 34, 35, 36, 51, 52, 37, 38, 39, 56, 63, 32, 49, 50, 51, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2244(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 5, 6, 7, 24, 15, 16, 1, 18, 19, 20, 3, 4, 21, 22, 23, 8, 63, 32, 49, 34, 35, 36, 51, 52, 37, 38, 39, 56, 47, 48, 33, 50, 51, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2245(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 5, 6, 7, 24, 31, 16, 1, 18, 19, 20, 3, 4, 21, 22, 23, 8, 47, 32, 49, 34, 35, 36, 51, 52, 37, 38, 39, 56, 63, 48, 33, 50, 51, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2246(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 5, 6, 7, 24, 15, 0, 1, 18, 19, 20, 3, 4, 21, 22, 23, 8, 63, 48, 49, 34, 35, 36, 51, 52, 37, 38, 39, 56, 47, 32, 33, 50, 51, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2247(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 5, 6, 7, 24, 31, 0, 1, 18, 19, 20, 3, 4, 21, 22, 23, 8, 47, 48, 49, 34, 35, 36, 51, 52, 37, 38, 39, 56, 63, 32, 33, 50, 51, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2248(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 5, 6, 7, 24, 15, 16, 17, 2, 19, 20, 3, 4, 21, 22, 23, 8, 63, 32, 33, 50, 35, 36, 51, 52, 37, 38, 39, 56, 47, 48, 49, 34, 51, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2249(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 5, 6, 7, 24, 31, 16, 17, 2, 19, 20, 3, 4, 21, 22, 23, 8, 47, 32, 33, 50, 35, 36, 51, 52, 37, 38, 39, 56, 63, 48, 49, 34, 51, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2250(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 5, 6, 7, 24, 15, 0, 17, 2, 19, 20, 3, 4, 21, 22, 23, 8, 63, 48, 33, 50, 35, 36, 51, 52, 37, 38, 39, 56, 47, 32, 49, 34, 51, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2251(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 5, 6, 7, 24, 31, 0, 17, 2, 19, 20, 3, 4, 21, 22, 23, 8, 47, 48, 33, 50, 35, 36, 51, 52, 37, 38, 39, 56, 63, 32, 49, 34, 51, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2252(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 5, 6, 7, 24, 15, 16, 1, 2, 19, 20, 3, 4, 21, 22, 23, 8, 63, 32, 49, 50, 35, 36, 51, 52, 37, 38, 39, 56, 47, 48, 33, 34, 51, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2253(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 5, 6, 7, 24, 31, 16, 1, 2, 19, 20, 3, 4, 21, 22, 23, 8, 47, 32, 49, 50, 35, 36, 51, 52, 37, 38, 39, 56, 63, 48, 33, 34, 51, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2254(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 5, 6, 7, 24, 15, 0, 1, 2, 19, 20, 3, 4, 21, 22, 23, 8, 63, 48, 49, 50, 35, 36, 51, 52, 37, 38, 39, 56, 47, 32, 33, 34, 51, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2255(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 5, 6, 7, 24, 31, 0, 1, 2, 19, 20, 3, 4, 21, 22, 23, 8, 47, 48, 49, 50, 35, 36, 51, 52, 37, 38, 39, 56, 63, 32, 33, 34, 51, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2256(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 5, 6, 7, 24, 15, 16, 17, 18, 3, 20, 3, 4, 21, 22, 23, 8, 63, 32, 33, 34, 51, 36, 51, 52, 37, 38, 39, 56, 47, 48, 49, 50, 35, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2257(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 5, 6, 7, 24, 31, 16, 17, 18, 3, 20, 3, 4, 21, 22, 23, 8, 47, 32, 33, 34, 51, 36, 51, 52, 37, 38, 39, 56, 63, 48, 49, 50, 35, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2258(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 5, 6, 7, 24, 15, 0, 17, 18, 3, 20, 3, 4, 21, 22, 23, 8, 63, 48, 33, 34, 51, 36, 51, 52, 37, 38, 39, 56, 47, 32, 49, 50, 35, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2259(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 5, 6, 7, 24, 31, 0, 17, 18, 3, 20, 3, 4, 21, 22, 23, 8, 47, 48, 33, 34, 51, 36, 51, 52, 37, 38, 39, 56, 63, 32, 49, 50, 35, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2260(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 5, 6, 7, 24, 15, 16, 1, 18, 3, 20, 3, 4, 21, 22, 23, 8, 63, 32, 49, 34, 51, 36, 51, 52, 37, 38, 39, 56, 47, 48, 33, 50, 35, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2261(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 5, 6, 7, 24, 31, 16, 1, 18, 3, 20, 3, 4, 21, 22, 23, 8, 47, 32, 49, 34, 51, 36, 51, 52, 37, 38, 39, 56, 63, 48, 33, 50, 35, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2262(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 5, 6, 7, 24, 15, 0, 1, 18, 3, 20, 3, 4, 21, 22, 23, 8, 63, 48, 49, 34, 51, 36, 51, 52, 37, 38, 39, 56, 47, 32, 33, 50, 35, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2263(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 5, 6, 7, 24, 31, 0, 1, 18, 3, 20, 3, 4, 21, 22, 23, 8, 47, 48, 49, 34, 51, 36, 51, 52, 37, 38, 39, 56, 63, 32, 33, 50, 35, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2264(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 5, 6, 7, 24, 15, 16, 17, 2, 3, 20, 3, 4, 21, 22, 23, 8, 63, 32, 33, 50, 51, 36, 51, 52, 37, 38, 39, 56, 47, 48, 49, 34, 35, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2265(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 5, 6, 7, 24, 31, 16, 17, 2, 3, 20, 3, 4, 21, 22, 23, 8, 47, 32, 33, 50, 51, 36, 51, 52, 37, 38, 39, 56, 63, 48, 49, 34, 35, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2266(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 5, 6, 7, 24, 15, 0, 17, 2, 3, 20, 3, 4, 21, 22, 23, 8, 63, 48, 33, 50, 51, 36, 51, 52, 37, 38, 39, 56, 47, 32, 49, 34, 35, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2267(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 5, 6, 7, 24, 31, 0, 17, 2, 3, 20, 3, 4, 21, 22, 23, 8, 47, 48, 33, 50, 51, 36, 51, 52, 37, 38, 39, 56, 63, 32, 49, 34, 35, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2268(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 5, 6, 7, 24, 15, 16, 1, 2, 3, 20, 3, 4, 21, 22, 23, 8, 63, 32, 49, 50, 51, 36, 51, 52, 37, 38, 39, 56, 47, 48, 33, 34, 35, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2269(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 5, 6, 7, 24, 31, 16, 1, 2, 3, 20, 3, 4, 21, 22, 23, 8, 47, 32, 49, 50, 51, 36, 51, 52, 37, 38, 39, 56, 63, 48, 33, 34, 35, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2270(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 5, 6, 7, 24, 15, 0, 1, 2, 3, 20, 3, 4, 21, 22, 23, 8, 63, 48, 49, 50, 51, 36, 51, 52, 37, 38, 39, 56, 47, 32, 33, 34, 35, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2271(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 5, 6, 7, 24, 31, 0, 1, 2, 3, 20, 3, 4, 21, 22, 23, 8, 47, 48, 49, 50, 51, 36, 51, 52, 37, 38, 39, 56, 63, 32, 33, 34, 35, 52, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2272(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 5, 6, 7, 24, 15, 16, 17, 18, 19, 4, 3, 4, 21, 22, 23, 8, 63, 32, 33, 34, 35, 52, 51, 52, 37, 38, 39, 56, 47, 48, 49, 50, 51, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2273(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 5, 6, 7, 24, 31, 16, 17, 18, 19, 4, 3, 4, 21, 22, 23, 8, 47, 32, 33, 34, 35, 52, 51, 52, 37, 38, 39, 56, 63, 48, 49, 50, 51, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2274(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 5, 6, 7, 24, 15, 0, 17, 18, 19, 4, 3, 4, 21, 22, 23, 8, 63, 48, 33, 34, 35, 52, 51, 52, 37, 38, 39, 56, 47, 32, 49, 50, 51, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2275(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 5, 6, 7, 24, 31, 0, 17, 18, 19, 4, 3, 4, 21, 22, 23, 8, 47, 48, 33, 34, 35, 52, 51, 52, 37, 38, 39, 56, 63, 32, 49, 50, 51, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2276(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 5, 6, 7, 24, 15, 16, 1, 18, 19, 4, 3, 4, 21, 22, 23, 8, 63, 32, 49, 34, 35, 52, 51, 52, 37, 38, 39, 56, 47, 48, 33, 50, 51, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2277(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 5, 6, 7, 24, 31, 16, 1, 18, 19, 4, 3, 4, 21, 22, 23, 8, 47, 32, 49, 34, 35, 52, 51, 52, 37, 38, 39, 56, 63, 48, 33, 50, 51, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2278(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 5, 6, 7, 24, 15, 0, 1, 18, 19, 4, 3, 4, 21, 22, 23, 8, 63, 48, 49, 34, 35, 52, 51, 52, 37, 38, 39, 56, 47, 32, 33, 50, 51, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2279(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 5, 6, 7, 24, 31, 0, 1, 18, 19, 4, 3, 4, 21, 22, 23, 8, 47, 48, 49, 34, 35, 52, 51, 52, 37, 38, 39, 56, 63, 32, 33, 50, 51, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2280(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 5, 6, 7, 24, 15, 16, 17, 2, 19, 4, 3, 4, 21, 22, 23, 8, 63, 32, 33, 50, 35, 52, 51, 52, 37, 38, 39, 56, 47, 48, 49, 34, 51, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2281(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 5, 6, 7, 24, 31, 16, 17, 2, 19, 4, 3, 4, 21, 22, 23, 8, 47, 32, 33, 50, 35, 52, 51, 52, 37, 38, 39, 56, 63, 48, 49, 34, 51, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2282(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 5, 6, 7, 24, 15, 0, 17, 2, 19, 4, 3, 4, 21, 22, 23, 8, 63, 48, 33, 50, 35, 52, 51, 52, 37, 38, 39, 56, 47, 32, 49, 34, 51, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2283(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 5, 6, 7, 24, 31, 0, 17, 2, 19, 4, 3, 4, 21, 22, 23, 8, 47, 48, 33, 50, 35, 52, 51, 52, 37, 38, 39, 56, 63, 32, 49, 34, 51, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2284(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 5, 6, 7, 24, 15, 16, 1, 2, 19, 4, 3, 4, 21, 22, 23, 8, 63, 32, 49, 50, 35, 52, 51, 52, 37, 38, 39, 56, 47, 48, 33, 34, 51, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2285(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 5, 6, 7, 24, 31, 16, 1, 2, 19, 4, 3, 4, 21, 22, 23, 8, 47, 32, 49, 50, 35, 52, 51, 52, 37, 38, 39, 56, 63, 48, 33, 34, 51, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2286(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 5, 6, 7, 24, 15, 0, 1, 2, 19, 4, 3, 4, 21, 22, 23, 8, 63, 48, 49, 50, 35, 52, 51, 52, 37, 38, 39, 56, 47, 32, 33, 34, 51, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2287(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 5, 6, 7, 24, 31, 0, 1, 2, 19, 4, 3, 4, 21, 22, 23, 8, 47, 48, 49, 50, 35, 52, 51, 52, 37, 38, 39, 56, 63, 32, 33, 34, 51, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2288(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 5, 6, 7, 24, 15, 16, 17, 18, 3, 4, 3, 4, 21, 22, 23, 8, 63, 32, 33, 34, 51, 52, 51, 52, 37, 38, 39, 56, 47, 48, 49, 50, 35, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2289(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 5, 6, 7, 24, 31, 16, 17, 18, 3, 4, 3, 4, 21, 22, 23, 8, 47, 32, 33, 34, 51, 52, 51, 52, 37, 38, 39, 56, 63, 48, 49, 50, 35, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2290(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 5, 6, 7, 24, 15, 0, 17, 18, 3, 4, 3, 4, 21, 22, 23, 8, 63, 48, 33, 34, 51, 52, 51, 52, 37, 38, 39, 56, 47, 32, 49, 50, 35, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2291(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 5, 6, 7, 24, 31, 0, 17, 18, 3, 4, 3, 4, 21, 22, 23, 8, 47, 48, 33, 34, 51, 52, 51, 52, 37, 38, 39, 56, 63, 32, 49, 50, 35, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2292(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 5, 6, 7, 24, 15, 16, 1, 18, 3, 4, 3, 4, 21, 22, 23, 8, 63, 32, 49, 34, 51, 52, 51, 52, 37, 38, 39, 56, 47, 48, 33, 50, 35, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2293(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 5, 6, 7, 24, 31, 16, 1, 18, 3, 4, 3, 4, 21, 22, 23, 8, 47, 32, 49, 34, 51, 52, 51, 52, 37, 38, 39, 56, 63, 48, 33, 50, 35, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2294(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 5, 6, 7, 24, 15, 0, 1, 18, 3, 4, 3, 4, 21, 22, 23, 8, 63, 48, 49, 34, 51, 52, 51, 52, 37, 38, 39, 56, 47, 32, 33, 50, 35, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2295(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 5, 6, 7, 24, 31, 0, 1, 18, 3, 4, 3, 4, 21, 22, 23, 8, 47, 48, 49, 34, 51, 52, 51, 52, 37, 38, 39, 56, 63, 32, 33, 50, 35, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2296(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 5, 6, 7, 24, 15, 16, 17, 2, 3, 4, 3, 4, 21, 22, 23, 8, 63, 32, 33, 50, 51, 52, 51, 52, 37, 38, 39, 56, 47, 48, 49, 34, 35, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2297(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 5, 6, 7, 24, 31, 16, 17, 2, 3, 4, 3, 4, 21, 22, 23, 8, 47, 32, 33, 50, 51, 52, 51, 52, 37, 38, 39, 56, 63, 48, 49, 34, 35, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2298(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 5, 6, 7, 24, 15, 0, 17, 2, 3, 4, 3, 4, 21, 22, 23, 8, 63, 48, 33, 50, 51, 52, 51, 52, 37, 38, 39, 56, 47, 32, 49, 34, 35, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2299(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 5, 6, 7, 24, 31, 0, 17, 2, 3, 4, 3, 4, 21, 22, 23, 8, 47, 48, 33, 50, 51, 52, 51, 52, 37, 38, 39, 56, 63, 32, 49, 34, 35, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2300(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 5, 6, 7, 24, 15, 16, 1, 2, 3, 4, 3, 4, 21, 22, 23, 8, 63, 32, 49, 50, 51, 52, 51, 52, 37, 38, 39, 56, 47, 48, 33, 34, 35, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2301(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 5, 6, 7, 24, 31, 16, 1, 2, 3, 4, 3, 4, 21, 22, 23, 8, 47, 32, 49, 50, 51, 52, 51, 52, 37, 38, 39, 56, 63, 48, 33, 34, 35, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2302(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 5, 6, 7, 24, 15, 0, 1, 2, 3, 4, 3, 4, 21, 22, 23, 8, 63, 48, 49, 50, 51, 52, 51, 52, 37, 38, 39, 56, 47, 32, 33, 34, 35, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2303(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 5, 6, 7, 24, 31, 0, 1, 2, 3, 4, 3, 4, 21, 22, 23, 8, 47, 48, 49, 50, 51, 52, 51, 52, 37, 38, 39, 56, 63, 32, 33, 34, 35, 36, 35, 36, 53, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}


