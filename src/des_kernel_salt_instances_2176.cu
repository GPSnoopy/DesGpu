#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

#ifdef DESGPU_COMPILE_ALL_SALTS

void des_25_encrypt_salt2176(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 5, 6, 7, 24, 15, 16, 17, 18, 19, 20, 19, 4, 21, 22, 23, 8, 63, 32, 33, 34, 35, 36, 35, 52, 37, 38, 39, 56, 47, 48, 49, 50, 51, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2177(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 5, 6, 7, 24, 31, 16, 17, 18, 19, 20, 19, 4, 21, 22, 23, 8, 47, 32, 33, 34, 35, 36, 35, 52, 37, 38, 39, 56, 63, 48, 49, 50, 51, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2178(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 5, 6, 7, 24, 15, 0, 17, 18, 19, 20, 19, 4, 21, 22, 23, 8, 63, 48, 33, 34, 35, 36, 35, 52, 37, 38, 39, 56, 47, 32, 49, 50, 51, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2179(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 5, 6, 7, 24, 31, 0, 17, 18, 19, 20, 19, 4, 21, 22, 23, 8, 47, 48, 33, 34, 35, 36, 35, 52, 37, 38, 39, 56, 63, 32, 49, 50, 51, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2180(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 5, 6, 7, 24, 15, 16, 1, 18, 19, 20, 19, 4, 21, 22, 23, 8, 63, 32, 49, 34, 35, 36, 35, 52, 37, 38, 39, 56, 47, 48, 33, 50, 51, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2181(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 5, 6, 7, 24, 31, 16, 1, 18, 19, 20, 19, 4, 21, 22, 23, 8, 47, 32, 49, 34, 35, 36, 35, 52, 37, 38, 39, 56, 63, 48, 33, 50, 51, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2182(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 5, 6, 7, 24, 15, 0, 1, 18, 19, 20, 19, 4, 21, 22, 23, 8, 63, 48, 49, 34, 35, 36, 35, 52, 37, 38, 39, 56, 47, 32, 33, 50, 51, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2183(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 5, 6, 7, 24, 31, 0, 1, 18, 19, 20, 19, 4, 21, 22, 23, 8, 47, 48, 49, 34, 35, 36, 35, 52, 37, 38, 39, 56, 63, 32, 33, 50, 51, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2184(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 5, 6, 7, 24, 15, 16, 17, 2, 19, 20, 19, 4, 21, 22, 23, 8, 63, 32, 33, 50, 35, 36, 35, 52, 37, 38, 39, 56, 47, 48, 49, 34, 51, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2185(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 5, 6, 7, 24, 31, 16, 17, 2, 19, 20, 19, 4, 21, 22, 23, 8, 47, 32, 33, 50, 35, 36, 35, 52, 37, 38, 39, 56, 63, 48, 49, 34, 51, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2186(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 5, 6, 7, 24, 15, 0, 17, 2, 19, 20, 19, 4, 21, 22, 23, 8, 63, 48, 33, 50, 35, 36, 35, 52, 37, 38, 39, 56, 47, 32, 49, 34, 51, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2187(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 5, 6, 7, 24, 31, 0, 17, 2, 19, 20, 19, 4, 21, 22, 23, 8, 47, 48, 33, 50, 35, 36, 35, 52, 37, 38, 39, 56, 63, 32, 49, 34, 51, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2188(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 5, 6, 7, 24, 15, 16, 1, 2, 19, 20, 19, 4, 21, 22, 23, 8, 63, 32, 49, 50, 35, 36, 35, 52, 37, 38, 39, 56, 47, 48, 33, 34, 51, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2189(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 5, 6, 7, 24, 31, 16, 1, 2, 19, 20, 19, 4, 21, 22, 23, 8, 47, 32, 49, 50, 35, 36, 35, 52, 37, 38, 39, 56, 63, 48, 33, 34, 51, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2190(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 5, 6, 7, 24, 15, 0, 1, 2, 19, 20, 19, 4, 21, 22, 23, 8, 63, 48, 49, 50, 35, 36, 35, 52, 37, 38, 39, 56, 47, 32, 33, 34, 51, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2191(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 5, 6, 7, 24, 31, 0, 1, 2, 19, 20, 19, 4, 21, 22, 23, 8, 47, 48, 49, 50, 35, 36, 35, 52, 37, 38, 39, 56, 63, 32, 33, 34, 51, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2192(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 5, 6, 7, 24, 15, 16, 17, 18, 3, 20, 19, 4, 21, 22, 23, 8, 63, 32, 33, 34, 51, 36, 35, 52, 37, 38, 39, 56, 47, 48, 49, 50, 35, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2193(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 5, 6, 7, 24, 31, 16, 17, 18, 3, 20, 19, 4, 21, 22, 23, 8, 47, 32, 33, 34, 51, 36, 35, 52, 37, 38, 39, 56, 63, 48, 49, 50, 35, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2194(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 5, 6, 7, 24, 15, 0, 17, 18, 3, 20, 19, 4, 21, 22, 23, 8, 63, 48, 33, 34, 51, 36, 35, 52, 37, 38, 39, 56, 47, 32, 49, 50, 35, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2195(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 5, 6, 7, 24, 31, 0, 17, 18, 3, 20, 19, 4, 21, 22, 23, 8, 47, 48, 33, 34, 51, 36, 35, 52, 37, 38, 39, 56, 63, 32, 49, 50, 35, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2196(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 5, 6, 7, 24, 15, 16, 1, 18, 3, 20, 19, 4, 21, 22, 23, 8, 63, 32, 49, 34, 51, 36, 35, 52, 37, 38, 39, 56, 47, 48, 33, 50, 35, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2197(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 5, 6, 7, 24, 31, 16, 1, 18, 3, 20, 19, 4, 21, 22, 23, 8, 47, 32, 49, 34, 51, 36, 35, 52, 37, 38, 39, 56, 63, 48, 33, 50, 35, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2198(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 5, 6, 7, 24, 15, 0, 1, 18, 3, 20, 19, 4, 21, 22, 23, 8, 63, 48, 49, 34, 51, 36, 35, 52, 37, 38, 39, 56, 47, 32, 33, 50, 35, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2199(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 5, 6, 7, 24, 31, 0, 1, 18, 3, 20, 19, 4, 21, 22, 23, 8, 47, 48, 49, 34, 51, 36, 35, 52, 37, 38, 39, 56, 63, 32, 33, 50, 35, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2200(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 5, 6, 7, 24, 15, 16, 17, 2, 3, 20, 19, 4, 21, 22, 23, 8, 63, 32, 33, 50, 51, 36, 35, 52, 37, 38, 39, 56, 47, 48, 49, 34, 35, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2201(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 5, 6, 7, 24, 31, 16, 17, 2, 3, 20, 19, 4, 21, 22, 23, 8, 47, 32, 33, 50, 51, 36, 35, 52, 37, 38, 39, 56, 63, 48, 49, 34, 35, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2202(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 5, 6, 7, 24, 15, 0, 17, 2, 3, 20, 19, 4, 21, 22, 23, 8, 63, 48, 33, 50, 51, 36, 35, 52, 37, 38, 39, 56, 47, 32, 49, 34, 35, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2203(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 5, 6, 7, 24, 31, 0, 17, 2, 3, 20, 19, 4, 21, 22, 23, 8, 47, 48, 33, 50, 51, 36, 35, 52, 37, 38, 39, 56, 63, 32, 49, 34, 35, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2204(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 5, 6, 7, 24, 15, 16, 1, 2, 3, 20, 19, 4, 21, 22, 23, 8, 63, 32, 49, 50, 51, 36, 35, 52, 37, 38, 39, 56, 47, 48, 33, 34, 35, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2205(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 5, 6, 7, 24, 31, 16, 1, 2, 3, 20, 19, 4, 21, 22, 23, 8, 47, 32, 49, 50, 51, 36, 35, 52, 37, 38, 39, 56, 63, 48, 33, 34, 35, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2206(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 5, 6, 7, 24, 15, 0, 1, 2, 3, 20, 19, 4, 21, 22, 23, 8, 63, 48, 49, 50, 51, 36, 35, 52, 37, 38, 39, 56, 47, 32, 33, 34, 35, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2207(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 5, 6, 7, 24, 31, 0, 1, 2, 3, 20, 19, 4, 21, 22, 23, 8, 47, 48, 49, 50, 51, 36, 35, 52, 37, 38, 39, 56, 63, 32, 33, 34, 35, 52, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2208(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 5, 6, 7, 24, 15, 16, 17, 18, 19, 4, 19, 4, 21, 22, 23, 8, 63, 32, 33, 34, 35, 52, 35, 52, 37, 38, 39, 56, 47, 48, 49, 50, 51, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2209(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 5, 6, 7, 24, 31, 16, 17, 18, 19, 4, 19, 4, 21, 22, 23, 8, 47, 32, 33, 34, 35, 52, 35, 52, 37, 38, 39, 56, 63, 48, 49, 50, 51, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2210(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 5, 6, 7, 24, 15, 0, 17, 18, 19, 4, 19, 4, 21, 22, 23, 8, 63, 48, 33, 34, 35, 52, 35, 52, 37, 38, 39, 56, 47, 32, 49, 50, 51, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2211(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 5, 6, 7, 24, 31, 0, 17, 18, 19, 4, 19, 4, 21, 22, 23, 8, 47, 48, 33, 34, 35, 52, 35, 52, 37, 38, 39, 56, 63, 32, 49, 50, 51, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2212(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 5, 6, 7, 24, 15, 16, 1, 18, 19, 4, 19, 4, 21, 22, 23, 8, 63, 32, 49, 34, 35, 52, 35, 52, 37, 38, 39, 56, 47, 48, 33, 50, 51, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2213(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 5, 6, 7, 24, 31, 16, 1, 18, 19, 4, 19, 4, 21, 22, 23, 8, 47, 32, 49, 34, 35, 52, 35, 52, 37, 38, 39, 56, 63, 48, 33, 50, 51, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2214(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 5, 6, 7, 24, 15, 0, 1, 18, 19, 4, 19, 4, 21, 22, 23, 8, 63, 48, 49, 34, 35, 52, 35, 52, 37, 38, 39, 56, 47, 32, 33, 50, 51, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2215(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 5, 6, 7, 24, 31, 0, 1, 18, 19, 4, 19, 4, 21, 22, 23, 8, 47, 48, 49, 34, 35, 52, 35, 52, 37, 38, 39, 56, 63, 32, 33, 50, 51, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2216(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 5, 6, 7, 24, 15, 16, 17, 2, 19, 4, 19, 4, 21, 22, 23, 8, 63, 32, 33, 50, 35, 52, 35, 52, 37, 38, 39, 56, 47, 48, 49, 34, 51, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2217(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 5, 6, 7, 24, 31, 16, 17, 2, 19, 4, 19, 4, 21, 22, 23, 8, 47, 32, 33, 50, 35, 52, 35, 52, 37, 38, 39, 56, 63, 48, 49, 34, 51, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2218(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 5, 6, 7, 24, 15, 0, 17, 2, 19, 4, 19, 4, 21, 22, 23, 8, 63, 48, 33, 50, 35, 52, 35, 52, 37, 38, 39, 56, 47, 32, 49, 34, 51, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2219(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 5, 6, 7, 24, 31, 0, 17, 2, 19, 4, 19, 4, 21, 22, 23, 8, 47, 48, 33, 50, 35, 52, 35, 52, 37, 38, 39, 56, 63, 32, 49, 34, 51, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2220(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 5, 6, 7, 24, 15, 16, 1, 2, 19, 4, 19, 4, 21, 22, 23, 8, 63, 32, 49, 50, 35, 52, 35, 52, 37, 38, 39, 56, 47, 48, 33, 34, 51, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2221(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 5, 6, 7, 24, 31, 16, 1, 2, 19, 4, 19, 4, 21, 22, 23, 8, 47, 32, 49, 50, 35, 52, 35, 52, 37, 38, 39, 56, 63, 48, 33, 34, 51, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2222(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 5, 6, 7, 24, 15, 0, 1, 2, 19, 4, 19, 4, 21, 22, 23, 8, 63, 48, 49, 50, 35, 52, 35, 52, 37, 38, 39, 56, 47, 32, 33, 34, 51, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2223(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 5, 6, 7, 24, 31, 0, 1, 2, 19, 4, 19, 4, 21, 22, 23, 8, 47, 48, 49, 50, 35, 52, 35, 52, 37, 38, 39, 56, 63, 32, 33, 34, 51, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2224(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 5, 6, 7, 24, 15, 16, 17, 18, 3, 4, 19, 4, 21, 22, 23, 8, 63, 32, 33, 34, 51, 52, 35, 52, 37, 38, 39, 56, 47, 48, 49, 50, 35, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2225(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 5, 6, 7, 24, 31, 16, 17, 18, 3, 4, 19, 4, 21, 22, 23, 8, 47, 32, 33, 34, 51, 52, 35, 52, 37, 38, 39, 56, 63, 48, 49, 50, 35, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2226(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 5, 6, 7, 24, 15, 0, 17, 18, 3, 4, 19, 4, 21, 22, 23, 8, 63, 48, 33, 34, 51, 52, 35, 52, 37, 38, 39, 56, 47, 32, 49, 50, 35, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2227(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 5, 6, 7, 24, 31, 0, 17, 18, 3, 4, 19, 4, 21, 22, 23, 8, 47, 48, 33, 34, 51, 52, 35, 52, 37, 38, 39, 56, 63, 32, 49, 50, 35, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2228(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 5, 6, 7, 24, 15, 16, 1, 18, 3, 4, 19, 4, 21, 22, 23, 8, 63, 32, 49, 34, 51, 52, 35, 52, 37, 38, 39, 56, 47, 48, 33, 50, 35, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2229(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 5, 6, 7, 24, 31, 16, 1, 18, 3, 4, 19, 4, 21, 22, 23, 8, 47, 32, 49, 34, 51, 52, 35, 52, 37, 38, 39, 56, 63, 48, 33, 50, 35, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2230(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 5, 6, 7, 24, 15, 0, 1, 18, 3, 4, 19, 4, 21, 22, 23, 8, 63, 48, 49, 34, 51, 52, 35, 52, 37, 38, 39, 56, 47, 32, 33, 50, 35, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2231(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 5, 6, 7, 24, 31, 0, 1, 18, 3, 4, 19, 4, 21, 22, 23, 8, 47, 48, 49, 34, 51, 52, 35, 52, 37, 38, 39, 56, 63, 32, 33, 50, 35, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2232(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 5, 6, 7, 24, 15, 16, 17, 2, 3, 4, 19, 4, 21, 22, 23, 8, 63, 32, 33, 50, 51, 52, 35, 52, 37, 38, 39, 56, 47, 48, 49, 34, 35, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2233(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 5, 6, 7, 24, 31, 16, 17, 2, 3, 4, 19, 4, 21, 22, 23, 8, 47, 32, 33, 50, 51, 52, 35, 52, 37, 38, 39, 56, 63, 48, 49, 34, 35, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2234(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 5, 6, 7, 24, 15, 0, 17, 2, 3, 4, 19, 4, 21, 22, 23, 8, 63, 48, 33, 50, 51, 52, 35, 52, 37, 38, 39, 56, 47, 32, 49, 34, 35, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2235(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 5, 6, 7, 24, 31, 0, 17, 2, 3, 4, 19, 4, 21, 22, 23, 8, 47, 48, 33, 50, 51, 52, 35, 52, 37, 38, 39, 56, 63, 32, 49, 34, 35, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2236(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 5, 6, 7, 24, 15, 16, 1, 2, 3, 4, 19, 4, 21, 22, 23, 8, 63, 32, 49, 50, 51, 52, 35, 52, 37, 38, 39, 56, 47, 48, 33, 34, 35, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2237(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 5, 6, 7, 24, 31, 16, 1, 2, 3, 4, 19, 4, 21, 22, 23, 8, 47, 32, 49, 50, 51, 52, 35, 52, 37, 38, 39, 56, 63, 48, 33, 34, 35, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2238(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 5, 6, 7, 24, 15, 0, 1, 2, 3, 4, 19, 4, 21, 22, 23, 8, 63, 48, 49, 50, 51, 52, 35, 52, 37, 38, 39, 56, 47, 32, 33, 34, 35, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2239(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 5, 6, 7, 24, 31, 0, 1, 2, 3, 4, 19, 4, 21, 22, 23, 8, 47, 48, 49, 50, 51, 52, 35, 52, 37, 38, 39, 56, 63, 32, 33, 34, 35, 36, 51, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2240(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 5, 6, 7, 24, 15, 16, 17, 18, 19, 20, 3, 4, 21, 22, 23, 8, 63, 32, 33, 34, 35, 36, 51, 52, 37, 38, 39, 56, 47, 48, 49, 50, 51, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2241(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 5, 6, 7, 24, 31, 16, 17, 18, 19, 20, 3, 4, 21, 22, 23, 8, 47, 32, 33, 34, 35, 36, 51, 52, 37, 38, 39, 56, 63, 48, 49, 50, 51, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2242(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 5, 6, 7, 24, 15, 0, 17, 18, 19, 20, 3, 4, 21, 22, 23, 8, 63, 48, 33, 34, 35, 36, 51, 52, 37, 38, 39, 56, 47, 32, 49, 50, 51, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2243(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 5, 6, 7, 24, 31, 0, 17, 18, 19, 20, 3, 4, 21, 22, 23, 8, 47, 48, 33, 34, 35, 36, 51, 52, 37, 38, 39, 56, 63, 32, 49, 50, 51, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2244(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 5, 6, 7, 24, 15, 16, 1, 18, 19, 20, 3, 4, 21, 22, 23, 8, 63, 32, 49, 34, 35, 36, 51, 52, 37, 38, 39, 56, 47, 48, 33, 50, 51, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2245(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 5, 6, 7, 24, 31, 16, 1, 18, 19, 20, 3, 4, 21, 22, 23, 8, 47, 32, 49, 34, 35, 36, 51, 52, 37, 38, 39, 56, 63, 48, 33, 50, 51, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2246(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 5, 6, 7, 24, 15, 0, 1, 18, 19, 20, 3, 4, 21, 22, 23, 8, 63, 48, 49, 34, 35, 36, 51, 52, 37, 38, 39, 56, 47, 32, 33, 50, 51, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2247(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 5, 6, 7, 24, 31, 0, 1, 18, 19, 20, 3, 4, 21, 22, 23, 8, 47, 48, 49, 34, 35, 36, 51, 52, 37, 38, 39, 56, 63, 32, 33, 50, 51, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2248(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 5, 6, 7, 24, 15, 16, 17, 2, 19, 20, 3, 4, 21, 22, 23, 8, 63, 32, 33, 50, 35, 36, 51, 52, 37, 38, 39, 56, 47, 48, 49, 34, 51, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2249(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 5, 6, 7, 24, 31, 16, 17, 2, 19, 20, 3, 4, 21, 22, 23, 8, 47, 32, 33, 50, 35, 36, 51, 52, 37, 38, 39, 56, 63, 48, 49, 34, 51, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2250(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 5, 6, 7, 24, 15, 0, 17, 2, 19, 20, 3, 4, 21, 22, 23, 8, 63, 48, 33, 50, 35, 36, 51, 52, 37, 38, 39, 56, 47, 32, 49, 34, 51, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2251(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 5, 6, 7, 24, 31, 0, 17, 2, 19, 20, 3, 4, 21, 22, 23, 8, 47, 48, 33, 50, 35, 36, 51, 52, 37, 38, 39, 56, 63, 32, 49, 34, 51, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2252(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 5, 6, 7, 24, 15, 16, 1, 2, 19, 20, 3, 4, 21, 22, 23, 8, 63, 32, 49, 50, 35, 36, 51, 52, 37, 38, 39, 56, 47, 48, 33, 34, 51, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2253(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 5, 6, 7, 24, 31, 16, 1, 2, 19, 20, 3, 4, 21, 22, 23, 8, 47, 32, 49, 50, 35, 36, 51, 52, 37, 38, 39, 56, 63, 48, 33, 34, 51, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2254(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 5, 6, 7, 24, 15, 0, 1, 2, 19, 20, 3, 4, 21, 22, 23, 8, 63, 48, 49, 50, 35, 36, 51, 52, 37, 38, 39, 56, 47, 32, 33, 34, 51, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2255(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 5, 6, 7, 24, 31, 0, 1, 2, 19, 20, 3, 4, 21, 22, 23, 8, 47, 48, 49, 50, 35, 36, 51, 52, 37, 38, 39, 56, 63, 32, 33, 34, 51, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2256(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 5, 6, 7, 24, 15, 16, 17, 18, 3, 20, 3, 4, 21, 22, 23, 8, 63, 32, 33, 34, 51, 36, 51, 52, 37, 38, 39, 56, 47, 48, 49, 50, 35, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2257(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 5, 6, 7, 24, 31, 16, 17, 18, 3, 20, 3, 4, 21, 22, 23, 8, 47, 32, 33, 34, 51, 36, 51, 52, 37, 38, 39, 56, 63, 48, 49, 50, 35, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2258(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 5, 6, 7, 24, 15, 0, 17, 18, 3, 20, 3, 4, 21, 22, 23, 8, 63, 48, 33, 34, 51, 36, 51, 52, 37, 38, 39, 56, 47, 32, 49, 50, 35, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2259(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 5, 6, 7, 24, 31, 0, 17, 18, 3, 20, 3, 4, 21, 22, 23, 8, 47, 48, 33, 34, 51, 36, 51, 52, 37, 38, 39, 56, 63, 32, 49, 50, 35, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2260(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 5, 6, 7, 24, 15, 16, 1, 18, 3, 20, 3, 4, 21, 22, 23, 8, 63, 32, 49, 34, 51, 36, 51, 52, 37, 38, 39, 56, 47, 48, 33, 50, 35, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2261(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 5, 6, 7, 24, 31, 16, 1, 18, 3, 20, 3, 4, 21, 22, 23, 8, 47, 32, 49, 34, 51, 36, 51, 52, 37, 38, 39, 56, 63, 48, 33, 50, 35, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2262(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 5, 6, 7, 24, 15, 0, 1, 18, 3, 20, 3, 4, 21, 22, 23, 8, 63, 48, 49, 34, 51, 36, 51, 52, 37, 38, 39, 56, 47, 32, 33, 50, 35, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2263(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 5, 6, 7, 24, 31, 0, 1, 18, 3, 20, 3, 4, 21, 22, 23, 8, 47, 48, 49, 34, 51, 36, 51, 52, 37, 38, 39, 56, 63, 32, 33, 50, 35, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2264(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 5, 6, 7, 24, 15, 16, 17, 2, 3, 20, 3, 4, 21, 22, 23, 8, 63, 32, 33, 50, 51, 36, 51, 52, 37, 38, 39, 56, 47, 48, 49, 34, 35, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2265(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 5, 6, 7, 24, 31, 16, 17, 2, 3, 20, 3, 4, 21, 22, 23, 8, 47, 32, 33, 50, 51, 36, 51, 52, 37, 38, 39, 56, 63, 48, 49, 34, 35, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2266(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 5, 6, 7, 24, 15, 0, 17, 2, 3, 20, 3, 4, 21, 22, 23, 8, 63, 48, 33, 50, 51, 36, 51, 52, 37, 38, 39, 56, 47, 32, 49, 34, 35, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2267(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 5, 6, 7, 24, 31, 0, 17, 2, 3, 20, 3, 4, 21, 22, 23, 8, 47, 48, 33, 50, 51, 36, 51, 52, 37, 38, 39, 56, 63, 32, 49, 34, 35, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2268(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 5, 6, 7, 24, 15, 16, 1, 2, 3, 20, 3, 4, 21, 22, 23, 8, 63, 32, 49, 50, 51, 36, 51, 52, 37, 38, 39, 56, 47, 48, 33, 34, 35, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2269(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 5, 6, 7, 24, 31, 16, 1, 2, 3, 20, 3, 4, 21, 22, 23, 8, 47, 32, 49, 50, 51, 36, 51, 52, 37, 38, 39, 56, 63, 48, 33, 34, 35, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2270(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 5, 6, 7, 24, 15, 0, 1, 2, 3, 20, 3, 4, 21, 22, 23, 8, 63, 48, 49, 50, 51, 36, 51, 52, 37, 38, 39, 56, 47, 32, 33, 34, 35, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2271(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 5, 6, 7, 24, 31, 0, 1, 2, 3, 20, 3, 4, 21, 22, 23, 8, 47, 48, 49, 50, 51, 36, 51, 52, 37, 38, 39, 56, 63, 32, 33, 34, 35, 52, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2272(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 5, 6, 7, 24, 15, 16, 17, 18, 19, 4, 3, 4, 21, 22, 23, 8, 63, 32, 33, 34, 35, 52, 51, 52, 37, 38, 39, 56, 47, 48, 49, 50, 51, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2273(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 5, 6, 7, 24, 31, 16, 17, 18, 19, 4, 3, 4, 21, 22, 23, 8, 47, 32, 33, 34, 35, 52, 51, 52, 37, 38, 39, 56, 63, 48, 49, 50, 51, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2274(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 5, 6, 7, 24, 15, 0, 17, 18, 19, 4, 3, 4, 21, 22, 23, 8, 63, 48, 33, 34, 35, 52, 51, 52, 37, 38, 39, 56, 47, 32, 49, 50, 51, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2275(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 5, 6, 7, 24, 31, 0, 17, 18, 19, 4, 3, 4, 21, 22, 23, 8, 47, 48, 33, 34, 35, 52, 51, 52, 37, 38, 39, 56, 63, 32, 49, 50, 51, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2276(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 5, 6, 7, 24, 15, 16, 1, 18, 19, 4, 3, 4, 21, 22, 23, 8, 63, 32, 49, 34, 35, 52, 51, 52, 37, 38, 39, 56, 47, 48, 33, 50, 51, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2277(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 5, 6, 7, 24, 31, 16, 1, 18, 19, 4, 3, 4, 21, 22, 23, 8, 47, 32, 49, 34, 35, 52, 51, 52, 37, 38, 39, 56, 63, 48, 33, 50, 51, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2278(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 5, 6, 7, 24, 15, 0, 1, 18, 19, 4, 3, 4, 21, 22, 23, 8, 63, 48, 49, 34, 35, 52, 51, 52, 37, 38, 39, 56, 47, 32, 33, 50, 51, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2279(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 5, 6, 7, 24, 31, 0, 1, 18, 19, 4, 3, 4, 21, 22, 23, 8, 47, 48, 49, 34, 35, 52, 51, 52, 37, 38, 39, 56, 63, 32, 33, 50, 51, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2280(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 5, 6, 7, 24, 15, 16, 17, 2, 19, 4, 3, 4, 21, 22, 23, 8, 63, 32, 33, 50, 35, 52, 51, 52, 37, 38, 39, 56, 47, 48, 49, 34, 51, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2281(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 5, 6, 7, 24, 31, 16, 17, 2, 19, 4, 3, 4, 21, 22, 23, 8, 47, 32, 33, 50, 35, 52, 51, 52, 37, 38, 39, 56, 63, 48, 49, 34, 51, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2282(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 5, 6, 7, 24, 15, 0, 17, 2, 19, 4, 3, 4, 21, 22, 23, 8, 63, 48, 33, 50, 35, 52, 51, 52, 37, 38, 39, 56, 47, 32, 49, 34, 51, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2283(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 5, 6, 7, 24, 31, 0, 17, 2, 19, 4, 3, 4, 21, 22, 23, 8, 47, 48, 33, 50, 35, 52, 51, 52, 37, 38, 39, 56, 63, 32, 49, 34, 51, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2284(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 5, 6, 7, 24, 15, 16, 1, 2, 19, 4, 3, 4, 21, 22, 23, 8, 63, 32, 49, 50, 35, 52, 51, 52, 37, 38, 39, 56, 47, 48, 33, 34, 51, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2285(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 5, 6, 7, 24, 31, 16, 1, 2, 19, 4, 3, 4, 21, 22, 23, 8, 47, 32, 49, 50, 35, 52, 51, 52, 37, 38, 39, 56, 63, 48, 33, 34, 51, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2286(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 5, 6, 7, 24, 15, 0, 1, 2, 19, 4, 3, 4, 21, 22, 23, 8, 63, 48, 49, 50, 35, 52, 51, 52, 37, 38, 39, 56, 47, 32, 33, 34, 51, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2287(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 5, 6, 7, 24, 31, 0, 1, 2, 19, 4, 3, 4, 21, 22, 23, 8, 47, 48, 49, 50, 35, 52, 51, 52, 37, 38, 39, 56, 63, 32, 33, 34, 51, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2288(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 5, 6, 7, 24, 15, 16, 17, 18, 3, 4, 3, 4, 21, 22, 23, 8, 63, 32, 33, 34, 51, 52, 51, 52, 37, 38, 39, 56, 47, 48, 49, 50, 35, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2289(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 5, 6, 7, 24, 31, 16, 17, 18, 3, 4, 3, 4, 21, 22, 23, 8, 47, 32, 33, 34, 51, 52, 51, 52, 37, 38, 39, 56, 63, 48, 49, 50, 35, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2290(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 5, 6, 7, 24, 15, 0, 17, 18, 3, 4, 3, 4, 21, 22, 23, 8, 63, 48, 33, 34, 51, 52, 51, 52, 37, 38, 39, 56, 47, 32, 49, 50, 35, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2291(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 5, 6, 7, 24, 31, 0, 17, 18, 3, 4, 3, 4, 21, 22, 23, 8, 47, 48, 33, 34, 51, 52, 51, 52, 37, 38, 39, 56, 63, 32, 49, 50, 35, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2292(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 5, 6, 7, 24, 15, 16, 1, 18, 3, 4, 3, 4, 21, 22, 23, 8, 63, 32, 49, 34, 51, 52, 51, 52, 37, 38, 39, 56, 47, 48, 33, 50, 35, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2293(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 5, 6, 7, 24, 31, 16, 1, 18, 3, 4, 3, 4, 21, 22, 23, 8, 47, 32, 49, 34, 51, 52, 51, 52, 37, 38, 39, 56, 63, 48, 33, 50, 35, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2294(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 5, 6, 7, 24, 15, 0, 1, 18, 3, 4, 3, 4, 21, 22, 23, 8, 63, 48, 49, 34, 51, 52, 51, 52, 37, 38, 39, 56, 47, 32, 33, 50, 35, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2295(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 5, 6, 7, 24, 31, 0, 1, 18, 3, 4, 3, 4, 21, 22, 23, 8, 47, 48, 49, 34, 51, 52, 51, 52, 37, 38, 39, 56, 63, 32, 33, 50, 35, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2296(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 5, 6, 7, 24, 15, 16, 17, 2, 3, 4, 3, 4, 21, 22, 23, 8, 63, 32, 33, 50, 51, 52, 51, 52, 37, 38, 39, 56, 47, 48, 49, 34, 35, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2297(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 5, 6, 7, 24, 31, 16, 17, 2, 3, 4, 3, 4, 21, 22, 23, 8, 47, 32, 33, 50, 51, 52, 51, 52, 37, 38, 39, 56, 63, 48, 49, 34, 35, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2298(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 5, 6, 7, 24, 15, 0, 17, 2, 3, 4, 3, 4, 21, 22, 23, 8, 63, 48, 33, 50, 51, 52, 51, 52, 37, 38, 39, 56, 47, 32, 49, 34, 35, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2299(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 5, 6, 7, 24, 31, 0, 17, 2, 3, 4, 3, 4, 21, 22, 23, 8, 47, 48, 33, 50, 51, 52, 51, 52, 37, 38, 39, 56, 63, 32, 49, 34, 35, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2300(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 5, 6, 7, 24, 15, 16, 1, 2, 3, 4, 3, 4, 21, 22, 23, 8, 63, 32, 49, 50, 51, 52, 51, 52, 37, 38, 39, 56, 47, 48, 33, 34, 35, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2301(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 5, 6, 7, 24, 31, 16, 1, 2, 3, 4, 3, 4, 21, 22, 23, 8, 47, 32, 49, 50, 51, 52, 51, 52, 37, 38, 39, 56, 63, 48, 33, 34, 35, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2302(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 5, 6, 7, 24, 15, 0, 1, 2, 3, 4, 3, 4, 21, 22, 23, 8, 63, 48, 49, 50, 51, 52, 51, 52, 37, 38, 39, 56, 47, 32, 33, 34, 35, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt2303(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 5, 6, 7, 24, 31, 0, 1, 2, 3, 4, 3, 4, 21, 22, 23, 8, 47, 48, 49, 50, 51, 52, 51, 52, 37, 38, 39, 56, 63, 32, 33, 34, 35, 36, 35, 36, 53, 54, 55, 40><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

#endif // DESGPU_COMPILE_ALL_SALTS


