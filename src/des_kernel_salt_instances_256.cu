#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt256(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 4, 21, 6, 7, 8, 15, 16, 17, 18, 19, 20, 19, 20, 5, 22, 23, 24, 63, 32, 33, 34, 35, 36, 35, 36, 53, 38, 39, 40, 47, 48, 49, 50, 51, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt257(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 4, 21, 6, 7, 8, 31, 16, 17, 18, 19, 20, 19, 20, 5, 22, 23, 24, 47, 32, 33, 34, 35, 36, 35, 36, 53, 38, 39, 40, 63, 48, 49, 50, 51, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt258(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 4, 21, 6, 7, 8, 15, 0, 17, 18, 19, 20, 19, 20, 5, 22, 23, 24, 63, 48, 33, 34, 35, 36, 35, 36, 53, 38, 39, 40, 47, 32, 49, 50, 51, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt259(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 4, 21, 6, 7, 8, 31, 0, 17, 18, 19, 20, 19, 20, 5, 22, 23, 24, 47, 48, 33, 34, 35, 36, 35, 36, 53, 38, 39, 40, 63, 32, 49, 50, 51, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt260(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 4, 21, 6, 7, 8, 15, 16, 1, 18, 19, 20, 19, 20, 5, 22, 23, 24, 63, 32, 49, 34, 35, 36, 35, 36, 53, 38, 39, 40, 47, 48, 33, 50, 51, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt261(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 4, 21, 6, 7, 8, 31, 16, 1, 18, 19, 20, 19, 20, 5, 22, 23, 24, 47, 32, 49, 34, 35, 36, 35, 36, 53, 38, 39, 40, 63, 48, 33, 50, 51, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt262(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 4, 21, 6, 7, 8, 15, 0, 1, 18, 19, 20, 19, 20, 5, 22, 23, 24, 63, 48, 49, 34, 35, 36, 35, 36, 53, 38, 39, 40, 47, 32, 33, 50, 51, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt263(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 4, 21, 6, 7, 8, 31, 0, 1, 18, 19, 20, 19, 20, 5, 22, 23, 24, 47, 48, 49, 34, 35, 36, 35, 36, 53, 38, 39, 40, 63, 32, 33, 50, 51, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt264(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 4, 21, 6, 7, 8, 15, 16, 17, 2, 19, 20, 19, 20, 5, 22, 23, 24, 63, 32, 33, 50, 35, 36, 35, 36, 53, 38, 39, 40, 47, 48, 49, 34, 51, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt265(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 4, 21, 6, 7, 8, 31, 16, 17, 2, 19, 20, 19, 20, 5, 22, 23, 24, 47, 32, 33, 50, 35, 36, 35, 36, 53, 38, 39, 40, 63, 48, 49, 34, 51, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt266(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 4, 21, 6, 7, 8, 15, 0, 17, 2, 19, 20, 19, 20, 5, 22, 23, 24, 63, 48, 33, 50, 35, 36, 35, 36, 53, 38, 39, 40, 47, 32, 49, 34, 51, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt267(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 4, 21, 6, 7, 8, 31, 0, 17, 2, 19, 20, 19, 20, 5, 22, 23, 24, 47, 48, 33, 50, 35, 36, 35, 36, 53, 38, 39, 40, 63, 32, 49, 34, 51, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt268(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 4, 21, 6, 7, 8, 15, 16, 1, 2, 19, 20, 19, 20, 5, 22, 23, 24, 63, 32, 49, 50, 35, 36, 35, 36, 53, 38, 39, 40, 47, 48, 33, 34, 51, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt269(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 4, 21, 6, 7, 8, 31, 16, 1, 2, 19, 20, 19, 20, 5, 22, 23, 24, 47, 32, 49, 50, 35, 36, 35, 36, 53, 38, 39, 40, 63, 48, 33, 34, 51, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt270(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 4, 21, 6, 7, 8, 15, 0, 1, 2, 19, 20, 19, 20, 5, 22, 23, 24, 63, 48, 49, 50, 35, 36, 35, 36, 53, 38, 39, 40, 47, 32, 33, 34, 51, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt271(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 4, 21, 6, 7, 8, 31, 0, 1, 2, 19, 20, 19, 20, 5, 22, 23, 24, 47, 48, 49, 50, 35, 36, 35, 36, 53, 38, 39, 40, 63, 32, 33, 34, 51, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt272(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 4, 21, 6, 7, 8, 15, 16, 17, 18, 3, 20, 19, 20, 5, 22, 23, 24, 63, 32, 33, 34, 51, 36, 35, 36, 53, 38, 39, 40, 47, 48, 49, 50, 35, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt273(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 4, 21, 6, 7, 8, 31, 16, 17, 18, 3, 20, 19, 20, 5, 22, 23, 24, 47, 32, 33, 34, 51, 36, 35, 36, 53, 38, 39, 40, 63, 48, 49, 50, 35, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt274(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 4, 21, 6, 7, 8, 15, 0, 17, 18, 3, 20, 19, 20, 5, 22, 23, 24, 63, 48, 33, 34, 51, 36, 35, 36, 53, 38, 39, 40, 47, 32, 49, 50, 35, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt275(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 4, 21, 6, 7, 8, 31, 0, 17, 18, 3, 20, 19, 20, 5, 22, 23, 24, 47, 48, 33, 34, 51, 36, 35, 36, 53, 38, 39, 40, 63, 32, 49, 50, 35, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt276(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 4, 21, 6, 7, 8, 15, 16, 1, 18, 3, 20, 19, 20, 5, 22, 23, 24, 63, 32, 49, 34, 51, 36, 35, 36, 53, 38, 39, 40, 47, 48, 33, 50, 35, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt277(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 4, 21, 6, 7, 8, 31, 16, 1, 18, 3, 20, 19, 20, 5, 22, 23, 24, 47, 32, 49, 34, 51, 36, 35, 36, 53, 38, 39, 40, 63, 48, 33, 50, 35, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt278(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 4, 21, 6, 7, 8, 15, 0, 1, 18, 3, 20, 19, 20, 5, 22, 23, 24, 63, 48, 49, 34, 51, 36, 35, 36, 53, 38, 39, 40, 47, 32, 33, 50, 35, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt279(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 4, 21, 6, 7, 8, 31, 0, 1, 18, 3, 20, 19, 20, 5, 22, 23, 24, 47, 48, 49, 34, 51, 36, 35, 36, 53, 38, 39, 40, 63, 32, 33, 50, 35, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt280(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 4, 21, 6, 7, 8, 15, 16, 17, 2, 3, 20, 19, 20, 5, 22, 23, 24, 63, 32, 33, 50, 51, 36, 35, 36, 53, 38, 39, 40, 47, 48, 49, 34, 35, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt281(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 4, 21, 6, 7, 8, 31, 16, 17, 2, 3, 20, 19, 20, 5, 22, 23, 24, 47, 32, 33, 50, 51, 36, 35, 36, 53, 38, 39, 40, 63, 48, 49, 34, 35, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt282(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 4, 21, 6, 7, 8, 15, 0, 17, 2, 3, 20, 19, 20, 5, 22, 23, 24, 63, 48, 33, 50, 51, 36, 35, 36, 53, 38, 39, 40, 47, 32, 49, 34, 35, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt283(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 4, 21, 6, 7, 8, 31, 0, 17, 2, 3, 20, 19, 20, 5, 22, 23, 24, 47, 48, 33, 50, 51, 36, 35, 36, 53, 38, 39, 40, 63, 32, 49, 34, 35, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt284(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 4, 21, 6, 7, 8, 15, 16, 1, 2, 3, 20, 19, 20, 5, 22, 23, 24, 63, 32, 49, 50, 51, 36, 35, 36, 53, 38, 39, 40, 47, 48, 33, 34, 35, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt285(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 4, 21, 6, 7, 8, 31, 16, 1, 2, 3, 20, 19, 20, 5, 22, 23, 24, 47, 32, 49, 50, 51, 36, 35, 36, 53, 38, 39, 40, 63, 48, 33, 34, 35, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt286(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 4, 21, 6, 7, 8, 15, 0, 1, 2, 3, 20, 19, 20, 5, 22, 23, 24, 63, 48, 49, 50, 51, 36, 35, 36, 53, 38, 39, 40, 47, 32, 33, 34, 35, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt287(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 4, 21, 6, 7, 8, 31, 0, 1, 2, 3, 20, 19, 20, 5, 22, 23, 24, 47, 48, 49, 50, 51, 36, 35, 36, 53, 38, 39, 40, 63, 32, 33, 34, 35, 52, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt288(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 4, 21, 6, 7, 8, 15, 16, 17, 18, 19, 4, 19, 20, 5, 22, 23, 24, 63, 32, 33, 34, 35, 52, 35, 36, 53, 38, 39, 40, 47, 48, 49, 50, 51, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt289(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 4, 21, 6, 7, 8, 31, 16, 17, 18, 19, 4, 19, 20, 5, 22, 23, 24, 47, 32, 33, 34, 35, 52, 35, 36, 53, 38, 39, 40, 63, 48, 49, 50, 51, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt290(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 4, 21, 6, 7, 8, 15, 0, 17, 18, 19, 4, 19, 20, 5, 22, 23, 24, 63, 48, 33, 34, 35, 52, 35, 36, 53, 38, 39, 40, 47, 32, 49, 50, 51, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt291(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 4, 21, 6, 7, 8, 31, 0, 17, 18, 19, 4, 19, 20, 5, 22, 23, 24, 47, 48, 33, 34, 35, 52, 35, 36, 53, 38, 39, 40, 63, 32, 49, 50, 51, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt292(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 4, 21, 6, 7, 8, 15, 16, 1, 18, 19, 4, 19, 20, 5, 22, 23, 24, 63, 32, 49, 34, 35, 52, 35, 36, 53, 38, 39, 40, 47, 48, 33, 50, 51, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt293(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 4, 21, 6, 7, 8, 31, 16, 1, 18, 19, 4, 19, 20, 5, 22, 23, 24, 47, 32, 49, 34, 35, 52, 35, 36, 53, 38, 39, 40, 63, 48, 33, 50, 51, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt294(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 4, 21, 6, 7, 8, 15, 0, 1, 18, 19, 4, 19, 20, 5, 22, 23, 24, 63, 48, 49, 34, 35, 52, 35, 36, 53, 38, 39, 40, 47, 32, 33, 50, 51, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt295(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 4, 21, 6, 7, 8, 31, 0, 1, 18, 19, 4, 19, 20, 5, 22, 23, 24, 47, 48, 49, 34, 35, 52, 35, 36, 53, 38, 39, 40, 63, 32, 33, 50, 51, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt296(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 4, 21, 6, 7, 8, 15, 16, 17, 2, 19, 4, 19, 20, 5, 22, 23, 24, 63, 32, 33, 50, 35, 52, 35, 36, 53, 38, 39, 40, 47, 48, 49, 34, 51, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt297(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 4, 21, 6, 7, 8, 31, 16, 17, 2, 19, 4, 19, 20, 5, 22, 23, 24, 47, 32, 33, 50, 35, 52, 35, 36, 53, 38, 39, 40, 63, 48, 49, 34, 51, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt298(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 4, 21, 6, 7, 8, 15, 0, 17, 2, 19, 4, 19, 20, 5, 22, 23, 24, 63, 48, 33, 50, 35, 52, 35, 36, 53, 38, 39, 40, 47, 32, 49, 34, 51, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt299(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 4, 21, 6, 7, 8, 31, 0, 17, 2, 19, 4, 19, 20, 5, 22, 23, 24, 47, 48, 33, 50, 35, 52, 35, 36, 53, 38, 39, 40, 63, 32, 49, 34, 51, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt300(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 4, 21, 6, 7, 8, 15, 16, 1, 2, 19, 4, 19, 20, 5, 22, 23, 24, 63, 32, 49, 50, 35, 52, 35, 36, 53, 38, 39, 40, 47, 48, 33, 34, 51, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt301(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 4, 21, 6, 7, 8, 31, 16, 1, 2, 19, 4, 19, 20, 5, 22, 23, 24, 47, 32, 49, 50, 35, 52, 35, 36, 53, 38, 39, 40, 63, 48, 33, 34, 51, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt302(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 4, 21, 6, 7, 8, 15, 0, 1, 2, 19, 4, 19, 20, 5, 22, 23, 24, 63, 48, 49, 50, 35, 52, 35, 36, 53, 38, 39, 40, 47, 32, 33, 34, 51, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt303(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 4, 21, 6, 7, 8, 31, 0, 1, 2, 19, 4, 19, 20, 5, 22, 23, 24, 47, 48, 49, 50, 35, 52, 35, 36, 53, 38, 39, 40, 63, 32, 33, 34, 51, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt304(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 4, 21, 6, 7, 8, 15, 16, 17, 18, 3, 4, 19, 20, 5, 22, 23, 24, 63, 32, 33, 34, 51, 52, 35, 36, 53, 38, 39, 40, 47, 48, 49, 50, 35, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt305(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 4, 21, 6, 7, 8, 31, 16, 17, 18, 3, 4, 19, 20, 5, 22, 23, 24, 47, 32, 33, 34, 51, 52, 35, 36, 53, 38, 39, 40, 63, 48, 49, 50, 35, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt306(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 4, 21, 6, 7, 8, 15, 0, 17, 18, 3, 4, 19, 20, 5, 22, 23, 24, 63, 48, 33, 34, 51, 52, 35, 36, 53, 38, 39, 40, 47, 32, 49, 50, 35, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt307(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 4, 21, 6, 7, 8, 31, 0, 17, 18, 3, 4, 19, 20, 5, 22, 23, 24, 47, 48, 33, 34, 51, 52, 35, 36, 53, 38, 39, 40, 63, 32, 49, 50, 35, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt308(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 4, 21, 6, 7, 8, 15, 16, 1, 18, 3, 4, 19, 20, 5, 22, 23, 24, 63, 32, 49, 34, 51, 52, 35, 36, 53, 38, 39, 40, 47, 48, 33, 50, 35, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt309(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 4, 21, 6, 7, 8, 31, 16, 1, 18, 3, 4, 19, 20, 5, 22, 23, 24, 47, 32, 49, 34, 51, 52, 35, 36, 53, 38, 39, 40, 63, 48, 33, 50, 35, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt310(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 4, 21, 6, 7, 8, 15, 0, 1, 18, 3, 4, 19, 20, 5, 22, 23, 24, 63, 48, 49, 34, 51, 52, 35, 36, 53, 38, 39, 40, 47, 32, 33, 50, 35, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt311(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 4, 21, 6, 7, 8, 31, 0, 1, 18, 3, 4, 19, 20, 5, 22, 23, 24, 47, 48, 49, 34, 51, 52, 35, 36, 53, 38, 39, 40, 63, 32, 33, 50, 35, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt312(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 4, 21, 6, 7, 8, 15, 16, 17, 2, 3, 4, 19, 20, 5, 22, 23, 24, 63, 32, 33, 50, 51, 52, 35, 36, 53, 38, 39, 40, 47, 48, 49, 34, 35, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt313(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 4, 21, 6, 7, 8, 31, 16, 17, 2, 3, 4, 19, 20, 5, 22, 23, 24, 47, 32, 33, 50, 51, 52, 35, 36, 53, 38, 39, 40, 63, 48, 49, 34, 35, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt314(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 4, 21, 6, 7, 8, 15, 0, 17, 2, 3, 4, 19, 20, 5, 22, 23, 24, 63, 48, 33, 50, 51, 52, 35, 36, 53, 38, 39, 40, 47, 32, 49, 34, 35, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt315(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 4, 21, 6, 7, 8, 31, 0, 17, 2, 3, 4, 19, 20, 5, 22, 23, 24, 47, 48, 33, 50, 51, 52, 35, 36, 53, 38, 39, 40, 63, 32, 49, 34, 35, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt316(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 4, 21, 6, 7, 8, 15, 16, 1, 2, 3, 4, 19, 20, 5, 22, 23, 24, 63, 32, 49, 50, 51, 52, 35, 36, 53, 38, 39, 40, 47, 48, 33, 34, 35, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt317(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 4, 21, 6, 7, 8, 31, 16, 1, 2, 3, 4, 19, 20, 5, 22, 23, 24, 47, 32, 49, 50, 51, 52, 35, 36, 53, 38, 39, 40, 63, 48, 33, 34, 35, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt318(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 4, 21, 6, 7, 8, 15, 0, 1, 2, 3, 4, 19, 20, 5, 22, 23, 24, 63, 48, 49, 50, 51, 52, 35, 36, 53, 38, 39, 40, 47, 32, 33, 34, 35, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt319(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 4, 21, 6, 7, 8, 31, 0, 1, 2, 3, 4, 19, 20, 5, 22, 23, 24, 47, 48, 49, 50, 51, 52, 35, 36, 53, 38, 39, 40, 63, 32, 33, 34, 35, 36, 51, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt320(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 4, 21, 6, 7, 8, 15, 16, 17, 18, 19, 20, 3, 20, 5, 22, 23, 24, 63, 32, 33, 34, 35, 36, 51, 36, 53, 38, 39, 40, 47, 48, 49, 50, 51, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt321(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 4, 21, 6, 7, 8, 31, 16, 17, 18, 19, 20, 3, 20, 5, 22, 23, 24, 47, 32, 33, 34, 35, 36, 51, 36, 53, 38, 39, 40, 63, 48, 49, 50, 51, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt322(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 4, 21, 6, 7, 8, 15, 0, 17, 18, 19, 20, 3, 20, 5, 22, 23, 24, 63, 48, 33, 34, 35, 36, 51, 36, 53, 38, 39, 40, 47, 32, 49, 50, 51, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt323(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 4, 21, 6, 7, 8, 31, 0, 17, 18, 19, 20, 3, 20, 5, 22, 23, 24, 47, 48, 33, 34, 35, 36, 51, 36, 53, 38, 39, 40, 63, 32, 49, 50, 51, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt324(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 4, 21, 6, 7, 8, 15, 16, 1, 18, 19, 20, 3, 20, 5, 22, 23, 24, 63, 32, 49, 34, 35, 36, 51, 36, 53, 38, 39, 40, 47, 48, 33, 50, 51, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt325(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 4, 21, 6, 7, 8, 31, 16, 1, 18, 19, 20, 3, 20, 5, 22, 23, 24, 47, 32, 49, 34, 35, 36, 51, 36, 53, 38, 39, 40, 63, 48, 33, 50, 51, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt326(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 4, 21, 6, 7, 8, 15, 0, 1, 18, 19, 20, 3, 20, 5, 22, 23, 24, 63, 48, 49, 34, 35, 36, 51, 36, 53, 38, 39, 40, 47, 32, 33, 50, 51, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt327(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 4, 21, 6, 7, 8, 31, 0, 1, 18, 19, 20, 3, 20, 5, 22, 23, 24, 47, 48, 49, 34, 35, 36, 51, 36, 53, 38, 39, 40, 63, 32, 33, 50, 51, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt328(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 4, 21, 6, 7, 8, 15, 16, 17, 2, 19, 20, 3, 20, 5, 22, 23, 24, 63, 32, 33, 50, 35, 36, 51, 36, 53, 38, 39, 40, 47, 48, 49, 34, 51, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt329(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 4, 21, 6, 7, 8, 31, 16, 17, 2, 19, 20, 3, 20, 5, 22, 23, 24, 47, 32, 33, 50, 35, 36, 51, 36, 53, 38, 39, 40, 63, 48, 49, 34, 51, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt330(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 4, 21, 6, 7, 8, 15, 0, 17, 2, 19, 20, 3, 20, 5, 22, 23, 24, 63, 48, 33, 50, 35, 36, 51, 36, 53, 38, 39, 40, 47, 32, 49, 34, 51, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt331(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 4, 21, 6, 7, 8, 31, 0, 17, 2, 19, 20, 3, 20, 5, 22, 23, 24, 47, 48, 33, 50, 35, 36, 51, 36, 53, 38, 39, 40, 63, 32, 49, 34, 51, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt332(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 4, 21, 6, 7, 8, 15, 16, 1, 2, 19, 20, 3, 20, 5, 22, 23, 24, 63, 32, 49, 50, 35, 36, 51, 36, 53, 38, 39, 40, 47, 48, 33, 34, 51, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt333(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 4, 21, 6, 7, 8, 31, 16, 1, 2, 19, 20, 3, 20, 5, 22, 23, 24, 47, 32, 49, 50, 35, 36, 51, 36, 53, 38, 39, 40, 63, 48, 33, 34, 51, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt334(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 4, 21, 6, 7, 8, 15, 0, 1, 2, 19, 20, 3, 20, 5, 22, 23, 24, 63, 48, 49, 50, 35, 36, 51, 36, 53, 38, 39, 40, 47, 32, 33, 34, 51, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt335(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 4, 21, 6, 7, 8, 31, 0, 1, 2, 19, 20, 3, 20, 5, 22, 23, 24, 47, 48, 49, 50, 35, 36, 51, 36, 53, 38, 39, 40, 63, 32, 33, 34, 51, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt336(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 4, 21, 6, 7, 8, 15, 16, 17, 18, 3, 20, 3, 20, 5, 22, 23, 24, 63, 32, 33, 34, 51, 36, 51, 36, 53, 38, 39, 40, 47, 48, 49, 50, 35, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt337(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 4, 21, 6, 7, 8, 31, 16, 17, 18, 3, 20, 3, 20, 5, 22, 23, 24, 47, 32, 33, 34, 51, 36, 51, 36, 53, 38, 39, 40, 63, 48, 49, 50, 35, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt338(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 4, 21, 6, 7, 8, 15, 0, 17, 18, 3, 20, 3, 20, 5, 22, 23, 24, 63, 48, 33, 34, 51, 36, 51, 36, 53, 38, 39, 40, 47, 32, 49, 50, 35, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt339(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 4, 21, 6, 7, 8, 31, 0, 17, 18, 3, 20, 3, 20, 5, 22, 23, 24, 47, 48, 33, 34, 51, 36, 51, 36, 53, 38, 39, 40, 63, 32, 49, 50, 35, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt340(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 4, 21, 6, 7, 8, 15, 16, 1, 18, 3, 20, 3, 20, 5, 22, 23, 24, 63, 32, 49, 34, 51, 36, 51, 36, 53, 38, 39, 40, 47, 48, 33, 50, 35, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt341(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 4, 21, 6, 7, 8, 31, 16, 1, 18, 3, 20, 3, 20, 5, 22, 23, 24, 47, 32, 49, 34, 51, 36, 51, 36, 53, 38, 39, 40, 63, 48, 33, 50, 35, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt342(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 4, 21, 6, 7, 8, 15, 0, 1, 18, 3, 20, 3, 20, 5, 22, 23, 24, 63, 48, 49, 34, 51, 36, 51, 36, 53, 38, 39, 40, 47, 32, 33, 50, 35, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt343(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 4, 21, 6, 7, 8, 31, 0, 1, 18, 3, 20, 3, 20, 5, 22, 23, 24, 47, 48, 49, 34, 51, 36, 51, 36, 53, 38, 39, 40, 63, 32, 33, 50, 35, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt344(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 4, 21, 6, 7, 8, 15, 16, 17, 2, 3, 20, 3, 20, 5, 22, 23, 24, 63, 32, 33, 50, 51, 36, 51, 36, 53, 38, 39, 40, 47, 48, 49, 34, 35, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt345(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 4, 21, 6, 7, 8, 31, 16, 17, 2, 3, 20, 3, 20, 5, 22, 23, 24, 47, 32, 33, 50, 51, 36, 51, 36, 53, 38, 39, 40, 63, 48, 49, 34, 35, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt346(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 4, 21, 6, 7, 8, 15, 0, 17, 2, 3, 20, 3, 20, 5, 22, 23, 24, 63, 48, 33, 50, 51, 36, 51, 36, 53, 38, 39, 40, 47, 32, 49, 34, 35, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt347(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 4, 21, 6, 7, 8, 31, 0, 17, 2, 3, 20, 3, 20, 5, 22, 23, 24, 47, 48, 33, 50, 51, 36, 51, 36, 53, 38, 39, 40, 63, 32, 49, 34, 35, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt348(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 4, 21, 6, 7, 8, 15, 16, 1, 2, 3, 20, 3, 20, 5, 22, 23, 24, 63, 32, 49, 50, 51, 36, 51, 36, 53, 38, 39, 40, 47, 48, 33, 34, 35, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt349(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 4, 21, 6, 7, 8, 31, 16, 1, 2, 3, 20, 3, 20, 5, 22, 23, 24, 47, 32, 49, 50, 51, 36, 51, 36, 53, 38, 39, 40, 63, 48, 33, 34, 35, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt350(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 4, 21, 6, 7, 8, 15, 0, 1, 2, 3, 20, 3, 20, 5, 22, 23, 24, 63, 48, 49, 50, 51, 36, 51, 36, 53, 38, 39, 40, 47, 32, 33, 34, 35, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt351(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 4, 21, 6, 7, 8, 31, 0, 1, 2, 3, 20, 3, 20, 5, 22, 23, 24, 47, 48, 49, 50, 51, 36, 51, 36, 53, 38, 39, 40, 63, 32, 33, 34, 35, 52, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt352(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 4, 21, 6, 7, 8, 15, 16, 17, 18, 19, 4, 3, 20, 5, 22, 23, 24, 63, 32, 33, 34, 35, 52, 51, 36, 53, 38, 39, 40, 47, 48, 49, 50, 51, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt353(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 4, 21, 6, 7, 8, 31, 16, 17, 18, 19, 4, 3, 20, 5, 22, 23, 24, 47, 32, 33, 34, 35, 52, 51, 36, 53, 38, 39, 40, 63, 48, 49, 50, 51, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt354(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 4, 21, 6, 7, 8, 15, 0, 17, 18, 19, 4, 3, 20, 5, 22, 23, 24, 63, 48, 33, 34, 35, 52, 51, 36, 53, 38, 39, 40, 47, 32, 49, 50, 51, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt355(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 4, 21, 6, 7, 8, 31, 0, 17, 18, 19, 4, 3, 20, 5, 22, 23, 24, 47, 48, 33, 34, 35, 52, 51, 36, 53, 38, 39, 40, 63, 32, 49, 50, 51, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt356(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 4, 21, 6, 7, 8, 15, 16, 1, 18, 19, 4, 3, 20, 5, 22, 23, 24, 63, 32, 49, 34, 35, 52, 51, 36, 53, 38, 39, 40, 47, 48, 33, 50, 51, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt357(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 4, 21, 6, 7, 8, 31, 16, 1, 18, 19, 4, 3, 20, 5, 22, 23, 24, 47, 32, 49, 34, 35, 52, 51, 36, 53, 38, 39, 40, 63, 48, 33, 50, 51, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt358(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 4, 21, 6, 7, 8, 15, 0, 1, 18, 19, 4, 3, 20, 5, 22, 23, 24, 63, 48, 49, 34, 35, 52, 51, 36, 53, 38, 39, 40, 47, 32, 33, 50, 51, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt359(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 4, 21, 6, 7, 8, 31, 0, 1, 18, 19, 4, 3, 20, 5, 22, 23, 24, 47, 48, 49, 34, 35, 52, 51, 36, 53, 38, 39, 40, 63, 32, 33, 50, 51, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt360(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 4, 21, 6, 7, 8, 15, 16, 17, 2, 19, 4, 3, 20, 5, 22, 23, 24, 63, 32, 33, 50, 35, 52, 51, 36, 53, 38, 39, 40, 47, 48, 49, 34, 51, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt361(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 4, 21, 6, 7, 8, 31, 16, 17, 2, 19, 4, 3, 20, 5, 22, 23, 24, 47, 32, 33, 50, 35, 52, 51, 36, 53, 38, 39, 40, 63, 48, 49, 34, 51, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt362(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 4, 21, 6, 7, 8, 15, 0, 17, 2, 19, 4, 3, 20, 5, 22, 23, 24, 63, 48, 33, 50, 35, 52, 51, 36, 53, 38, 39, 40, 47, 32, 49, 34, 51, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt363(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 4, 21, 6, 7, 8, 31, 0, 17, 2, 19, 4, 3, 20, 5, 22, 23, 24, 47, 48, 33, 50, 35, 52, 51, 36, 53, 38, 39, 40, 63, 32, 49, 34, 51, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt364(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 4, 21, 6, 7, 8, 15, 16, 1, 2, 19, 4, 3, 20, 5, 22, 23, 24, 63, 32, 49, 50, 35, 52, 51, 36, 53, 38, 39, 40, 47, 48, 33, 34, 51, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt365(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 4, 21, 6, 7, 8, 31, 16, 1, 2, 19, 4, 3, 20, 5, 22, 23, 24, 47, 32, 49, 50, 35, 52, 51, 36, 53, 38, 39, 40, 63, 48, 33, 34, 51, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt366(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 4, 21, 6, 7, 8, 15, 0, 1, 2, 19, 4, 3, 20, 5, 22, 23, 24, 63, 48, 49, 50, 35, 52, 51, 36, 53, 38, 39, 40, 47, 32, 33, 34, 51, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt367(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 4, 21, 6, 7, 8, 31, 0, 1, 2, 19, 4, 3, 20, 5, 22, 23, 24, 47, 48, 49, 50, 35, 52, 51, 36, 53, 38, 39, 40, 63, 32, 33, 34, 51, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt368(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 4, 21, 6, 7, 8, 15, 16, 17, 18, 3, 4, 3, 20, 5, 22, 23, 24, 63, 32, 33, 34, 51, 52, 51, 36, 53, 38, 39, 40, 47, 48, 49, 50, 35, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt369(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 4, 21, 6, 7, 8, 31, 16, 17, 18, 3, 4, 3, 20, 5, 22, 23, 24, 47, 32, 33, 34, 51, 52, 51, 36, 53, 38, 39, 40, 63, 48, 49, 50, 35, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt370(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 4, 21, 6, 7, 8, 15, 0, 17, 18, 3, 4, 3, 20, 5, 22, 23, 24, 63, 48, 33, 34, 51, 52, 51, 36, 53, 38, 39, 40, 47, 32, 49, 50, 35, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt371(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 4, 21, 6, 7, 8, 31, 0, 17, 18, 3, 4, 3, 20, 5, 22, 23, 24, 47, 48, 33, 34, 51, 52, 51, 36, 53, 38, 39, 40, 63, 32, 49, 50, 35, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt372(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 4, 21, 6, 7, 8, 15, 16, 1, 18, 3, 4, 3, 20, 5, 22, 23, 24, 63, 32, 49, 34, 51, 52, 51, 36, 53, 38, 39, 40, 47, 48, 33, 50, 35, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt373(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 4, 21, 6, 7, 8, 31, 16, 1, 18, 3, 4, 3, 20, 5, 22, 23, 24, 47, 32, 49, 34, 51, 52, 51, 36, 53, 38, 39, 40, 63, 48, 33, 50, 35, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt374(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 4, 21, 6, 7, 8, 15, 0, 1, 18, 3, 4, 3, 20, 5, 22, 23, 24, 63, 48, 49, 34, 51, 52, 51, 36, 53, 38, 39, 40, 47, 32, 33, 50, 35, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt375(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 4, 21, 6, 7, 8, 31, 0, 1, 18, 3, 4, 3, 20, 5, 22, 23, 24, 47, 48, 49, 34, 51, 52, 51, 36, 53, 38, 39, 40, 63, 32, 33, 50, 35, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt376(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 4, 21, 6, 7, 8, 15, 16, 17, 2, 3, 4, 3, 20, 5, 22, 23, 24, 63, 32, 33, 50, 51, 52, 51, 36, 53, 38, 39, 40, 47, 48, 49, 34, 35, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt377(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 4, 21, 6, 7, 8, 31, 16, 17, 2, 3, 4, 3, 20, 5, 22, 23, 24, 47, 32, 33, 50, 51, 52, 51, 36, 53, 38, 39, 40, 63, 48, 49, 34, 35, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt378(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 4, 21, 6, 7, 8, 15, 0, 17, 2, 3, 4, 3, 20, 5, 22, 23, 24, 63, 48, 33, 50, 51, 52, 51, 36, 53, 38, 39, 40, 47, 32, 49, 34, 35, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt379(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 4, 21, 6, 7, 8, 31, 0, 17, 2, 3, 4, 3, 20, 5, 22, 23, 24, 47, 48, 33, 50, 51, 52, 51, 36, 53, 38, 39, 40, 63, 32, 49, 34, 35, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt380(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 4, 21, 6, 7, 8, 15, 16, 1, 2, 3, 4, 3, 20, 5, 22, 23, 24, 63, 32, 49, 50, 51, 52, 51, 36, 53, 38, 39, 40, 47, 48, 33, 34, 35, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt381(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 4, 21, 6, 7, 8, 31, 16, 1, 2, 3, 4, 3, 20, 5, 22, 23, 24, 47, 32, 49, 50, 51, 52, 51, 36, 53, 38, 39, 40, 63, 48, 33, 34, 35, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt382(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 4, 21, 6, 7, 8, 15, 0, 1, 2, 3, 4, 3, 20, 5, 22, 23, 24, 63, 48, 49, 50, 51, 52, 51, 36, 53, 38, 39, 40, 47, 32, 33, 34, 35, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt383(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 4, 21, 6, 7, 8, 31, 0, 1, 2, 3, 4, 3, 20, 5, 22, 23, 24, 47, 48, 49, 50, 51, 52, 51, 36, 53, 38, 39, 40, 63, 32, 33, 34, 35, 36, 35, 52, 37, 54, 55, 56>(unchecked_hashes, bitsplitted_keys);
}


