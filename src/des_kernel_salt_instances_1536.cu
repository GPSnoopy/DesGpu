#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt1536(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 4, 5, 22, 23, 8, 15, 16, 17, 18, 19, 20, 19, 20, 21, 6, 7, 24, 63, 32, 33, 34, 35, 36, 35, 36, 37, 54, 55, 40, 47, 48, 49, 50, 51, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1537(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 4, 5, 22, 23, 8, 31, 16, 17, 18, 19, 20, 19, 20, 21, 6, 7, 24, 47, 32, 33, 34, 35, 36, 35, 36, 37, 54, 55, 40, 63, 48, 49, 50, 51, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1538(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 4, 5, 22, 23, 8, 15, 0, 17, 18, 19, 20, 19, 20, 21, 6, 7, 24, 63, 48, 33, 34, 35, 36, 35, 36, 37, 54, 55, 40, 47, 32, 49, 50, 51, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1539(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 4, 5, 22, 23, 8, 31, 0, 17, 18, 19, 20, 19, 20, 21, 6, 7, 24, 47, 48, 33, 34, 35, 36, 35, 36, 37, 54, 55, 40, 63, 32, 49, 50, 51, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1540(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 4, 5, 22, 23, 8, 15, 16, 1, 18, 19, 20, 19, 20, 21, 6, 7, 24, 63, 32, 49, 34, 35, 36, 35, 36, 37, 54, 55, 40, 47, 48, 33, 50, 51, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1541(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 4, 5, 22, 23, 8, 31, 16, 1, 18, 19, 20, 19, 20, 21, 6, 7, 24, 47, 32, 49, 34, 35, 36, 35, 36, 37, 54, 55, 40, 63, 48, 33, 50, 51, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1542(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 4, 5, 22, 23, 8, 15, 0, 1, 18, 19, 20, 19, 20, 21, 6, 7, 24, 63, 48, 49, 34, 35, 36, 35, 36, 37, 54, 55, 40, 47, 32, 33, 50, 51, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1543(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 4, 5, 22, 23, 8, 31, 0, 1, 18, 19, 20, 19, 20, 21, 6, 7, 24, 47, 48, 49, 34, 35, 36, 35, 36, 37, 54, 55, 40, 63, 32, 33, 50, 51, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1544(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 4, 5, 22, 23, 8, 15, 16, 17, 2, 19, 20, 19, 20, 21, 6, 7, 24, 63, 32, 33, 50, 35, 36, 35, 36, 37, 54, 55, 40, 47, 48, 49, 34, 51, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1545(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 4, 5, 22, 23, 8, 31, 16, 17, 2, 19, 20, 19, 20, 21, 6, 7, 24, 47, 32, 33, 50, 35, 36, 35, 36, 37, 54, 55, 40, 63, 48, 49, 34, 51, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1546(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 4, 5, 22, 23, 8, 15, 0, 17, 2, 19, 20, 19, 20, 21, 6, 7, 24, 63, 48, 33, 50, 35, 36, 35, 36, 37, 54, 55, 40, 47, 32, 49, 34, 51, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1547(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 4, 5, 22, 23, 8, 31, 0, 17, 2, 19, 20, 19, 20, 21, 6, 7, 24, 47, 48, 33, 50, 35, 36, 35, 36, 37, 54, 55, 40, 63, 32, 49, 34, 51, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1548(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 4, 5, 22, 23, 8, 15, 16, 1, 2, 19, 20, 19, 20, 21, 6, 7, 24, 63, 32, 49, 50, 35, 36, 35, 36, 37, 54, 55, 40, 47, 48, 33, 34, 51, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1549(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 4, 5, 22, 23, 8, 31, 16, 1, 2, 19, 20, 19, 20, 21, 6, 7, 24, 47, 32, 49, 50, 35, 36, 35, 36, 37, 54, 55, 40, 63, 48, 33, 34, 51, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1550(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 4, 5, 22, 23, 8, 15, 0, 1, 2, 19, 20, 19, 20, 21, 6, 7, 24, 63, 48, 49, 50, 35, 36, 35, 36, 37, 54, 55, 40, 47, 32, 33, 34, 51, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1551(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 4, 5, 22, 23, 8, 31, 0, 1, 2, 19, 20, 19, 20, 21, 6, 7, 24, 47, 48, 49, 50, 35, 36, 35, 36, 37, 54, 55, 40, 63, 32, 33, 34, 51, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1552(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 4, 5, 22, 23, 8, 15, 16, 17, 18, 3, 20, 19, 20, 21, 6, 7, 24, 63, 32, 33, 34, 51, 36, 35, 36, 37, 54, 55, 40, 47, 48, 49, 50, 35, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1553(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 4, 5, 22, 23, 8, 31, 16, 17, 18, 3, 20, 19, 20, 21, 6, 7, 24, 47, 32, 33, 34, 51, 36, 35, 36, 37, 54, 55, 40, 63, 48, 49, 50, 35, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1554(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 4, 5, 22, 23, 8, 15, 0, 17, 18, 3, 20, 19, 20, 21, 6, 7, 24, 63, 48, 33, 34, 51, 36, 35, 36, 37, 54, 55, 40, 47, 32, 49, 50, 35, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1555(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 4, 5, 22, 23, 8, 31, 0, 17, 18, 3, 20, 19, 20, 21, 6, 7, 24, 47, 48, 33, 34, 51, 36, 35, 36, 37, 54, 55, 40, 63, 32, 49, 50, 35, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1556(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 4, 5, 22, 23, 8, 15, 16, 1, 18, 3, 20, 19, 20, 21, 6, 7, 24, 63, 32, 49, 34, 51, 36, 35, 36, 37, 54, 55, 40, 47, 48, 33, 50, 35, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1557(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 4, 5, 22, 23, 8, 31, 16, 1, 18, 3, 20, 19, 20, 21, 6, 7, 24, 47, 32, 49, 34, 51, 36, 35, 36, 37, 54, 55, 40, 63, 48, 33, 50, 35, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1558(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 4, 5, 22, 23, 8, 15, 0, 1, 18, 3, 20, 19, 20, 21, 6, 7, 24, 63, 48, 49, 34, 51, 36, 35, 36, 37, 54, 55, 40, 47, 32, 33, 50, 35, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1559(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 4, 5, 22, 23, 8, 31, 0, 1, 18, 3, 20, 19, 20, 21, 6, 7, 24, 47, 48, 49, 34, 51, 36, 35, 36, 37, 54, 55, 40, 63, 32, 33, 50, 35, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1560(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 4, 5, 22, 23, 8, 15, 16, 17, 2, 3, 20, 19, 20, 21, 6, 7, 24, 63, 32, 33, 50, 51, 36, 35, 36, 37, 54, 55, 40, 47, 48, 49, 34, 35, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1561(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 4, 5, 22, 23, 8, 31, 16, 17, 2, 3, 20, 19, 20, 21, 6, 7, 24, 47, 32, 33, 50, 51, 36, 35, 36, 37, 54, 55, 40, 63, 48, 49, 34, 35, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1562(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 4, 5, 22, 23, 8, 15, 0, 17, 2, 3, 20, 19, 20, 21, 6, 7, 24, 63, 48, 33, 50, 51, 36, 35, 36, 37, 54, 55, 40, 47, 32, 49, 34, 35, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1563(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 4, 5, 22, 23, 8, 31, 0, 17, 2, 3, 20, 19, 20, 21, 6, 7, 24, 47, 48, 33, 50, 51, 36, 35, 36, 37, 54, 55, 40, 63, 32, 49, 34, 35, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1564(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 4, 5, 22, 23, 8, 15, 16, 1, 2, 3, 20, 19, 20, 21, 6, 7, 24, 63, 32, 49, 50, 51, 36, 35, 36, 37, 54, 55, 40, 47, 48, 33, 34, 35, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1565(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 4, 5, 22, 23, 8, 31, 16, 1, 2, 3, 20, 19, 20, 21, 6, 7, 24, 47, 32, 49, 50, 51, 36, 35, 36, 37, 54, 55, 40, 63, 48, 33, 34, 35, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1566(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 4, 5, 22, 23, 8, 15, 0, 1, 2, 3, 20, 19, 20, 21, 6, 7, 24, 63, 48, 49, 50, 51, 36, 35, 36, 37, 54, 55, 40, 47, 32, 33, 34, 35, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1567(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 4, 5, 22, 23, 8, 31, 0, 1, 2, 3, 20, 19, 20, 21, 6, 7, 24, 47, 48, 49, 50, 51, 36, 35, 36, 37, 54, 55, 40, 63, 32, 33, 34, 35, 52, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1568(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 4, 5, 22, 23, 8, 15, 16, 17, 18, 19, 4, 19, 20, 21, 6, 7, 24, 63, 32, 33, 34, 35, 52, 35, 36, 37, 54, 55, 40, 47, 48, 49, 50, 51, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1569(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 4, 5, 22, 23, 8, 31, 16, 17, 18, 19, 4, 19, 20, 21, 6, 7, 24, 47, 32, 33, 34, 35, 52, 35, 36, 37, 54, 55, 40, 63, 48, 49, 50, 51, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1570(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 4, 5, 22, 23, 8, 15, 0, 17, 18, 19, 4, 19, 20, 21, 6, 7, 24, 63, 48, 33, 34, 35, 52, 35, 36, 37, 54, 55, 40, 47, 32, 49, 50, 51, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1571(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 4, 5, 22, 23, 8, 31, 0, 17, 18, 19, 4, 19, 20, 21, 6, 7, 24, 47, 48, 33, 34, 35, 52, 35, 36, 37, 54, 55, 40, 63, 32, 49, 50, 51, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1572(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 4, 5, 22, 23, 8, 15, 16, 1, 18, 19, 4, 19, 20, 21, 6, 7, 24, 63, 32, 49, 34, 35, 52, 35, 36, 37, 54, 55, 40, 47, 48, 33, 50, 51, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1573(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 4, 5, 22, 23, 8, 31, 16, 1, 18, 19, 4, 19, 20, 21, 6, 7, 24, 47, 32, 49, 34, 35, 52, 35, 36, 37, 54, 55, 40, 63, 48, 33, 50, 51, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1574(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 4, 5, 22, 23, 8, 15, 0, 1, 18, 19, 4, 19, 20, 21, 6, 7, 24, 63, 48, 49, 34, 35, 52, 35, 36, 37, 54, 55, 40, 47, 32, 33, 50, 51, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1575(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 4, 5, 22, 23, 8, 31, 0, 1, 18, 19, 4, 19, 20, 21, 6, 7, 24, 47, 48, 49, 34, 35, 52, 35, 36, 37, 54, 55, 40, 63, 32, 33, 50, 51, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1576(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 4, 5, 22, 23, 8, 15, 16, 17, 2, 19, 4, 19, 20, 21, 6, 7, 24, 63, 32, 33, 50, 35, 52, 35, 36, 37, 54, 55, 40, 47, 48, 49, 34, 51, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1577(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 4, 5, 22, 23, 8, 31, 16, 17, 2, 19, 4, 19, 20, 21, 6, 7, 24, 47, 32, 33, 50, 35, 52, 35, 36, 37, 54, 55, 40, 63, 48, 49, 34, 51, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1578(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 4, 5, 22, 23, 8, 15, 0, 17, 2, 19, 4, 19, 20, 21, 6, 7, 24, 63, 48, 33, 50, 35, 52, 35, 36, 37, 54, 55, 40, 47, 32, 49, 34, 51, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1579(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 4, 5, 22, 23, 8, 31, 0, 17, 2, 19, 4, 19, 20, 21, 6, 7, 24, 47, 48, 33, 50, 35, 52, 35, 36, 37, 54, 55, 40, 63, 32, 49, 34, 51, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1580(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 4, 5, 22, 23, 8, 15, 16, 1, 2, 19, 4, 19, 20, 21, 6, 7, 24, 63, 32, 49, 50, 35, 52, 35, 36, 37, 54, 55, 40, 47, 48, 33, 34, 51, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1581(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 4, 5, 22, 23, 8, 31, 16, 1, 2, 19, 4, 19, 20, 21, 6, 7, 24, 47, 32, 49, 50, 35, 52, 35, 36, 37, 54, 55, 40, 63, 48, 33, 34, 51, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1582(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 4, 5, 22, 23, 8, 15, 0, 1, 2, 19, 4, 19, 20, 21, 6, 7, 24, 63, 48, 49, 50, 35, 52, 35, 36, 37, 54, 55, 40, 47, 32, 33, 34, 51, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1583(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 4, 5, 22, 23, 8, 31, 0, 1, 2, 19, 4, 19, 20, 21, 6, 7, 24, 47, 48, 49, 50, 35, 52, 35, 36, 37, 54, 55, 40, 63, 32, 33, 34, 51, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1584(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 4, 5, 22, 23, 8, 15, 16, 17, 18, 3, 4, 19, 20, 21, 6, 7, 24, 63, 32, 33, 34, 51, 52, 35, 36, 37, 54, 55, 40, 47, 48, 49, 50, 35, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1585(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 4, 5, 22, 23, 8, 31, 16, 17, 18, 3, 4, 19, 20, 21, 6, 7, 24, 47, 32, 33, 34, 51, 52, 35, 36, 37, 54, 55, 40, 63, 48, 49, 50, 35, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1586(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 4, 5, 22, 23, 8, 15, 0, 17, 18, 3, 4, 19, 20, 21, 6, 7, 24, 63, 48, 33, 34, 51, 52, 35, 36, 37, 54, 55, 40, 47, 32, 49, 50, 35, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1587(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 4, 5, 22, 23, 8, 31, 0, 17, 18, 3, 4, 19, 20, 21, 6, 7, 24, 47, 48, 33, 34, 51, 52, 35, 36, 37, 54, 55, 40, 63, 32, 49, 50, 35, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1588(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 4, 5, 22, 23, 8, 15, 16, 1, 18, 3, 4, 19, 20, 21, 6, 7, 24, 63, 32, 49, 34, 51, 52, 35, 36, 37, 54, 55, 40, 47, 48, 33, 50, 35, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1589(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 4, 5, 22, 23, 8, 31, 16, 1, 18, 3, 4, 19, 20, 21, 6, 7, 24, 47, 32, 49, 34, 51, 52, 35, 36, 37, 54, 55, 40, 63, 48, 33, 50, 35, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1590(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 4, 5, 22, 23, 8, 15, 0, 1, 18, 3, 4, 19, 20, 21, 6, 7, 24, 63, 48, 49, 34, 51, 52, 35, 36, 37, 54, 55, 40, 47, 32, 33, 50, 35, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1591(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 4, 5, 22, 23, 8, 31, 0, 1, 18, 3, 4, 19, 20, 21, 6, 7, 24, 47, 48, 49, 34, 51, 52, 35, 36, 37, 54, 55, 40, 63, 32, 33, 50, 35, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1592(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 4, 5, 22, 23, 8, 15, 16, 17, 2, 3, 4, 19, 20, 21, 6, 7, 24, 63, 32, 33, 50, 51, 52, 35, 36, 37, 54, 55, 40, 47, 48, 49, 34, 35, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1593(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 4, 5, 22, 23, 8, 31, 16, 17, 2, 3, 4, 19, 20, 21, 6, 7, 24, 47, 32, 33, 50, 51, 52, 35, 36, 37, 54, 55, 40, 63, 48, 49, 34, 35, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1594(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 4, 5, 22, 23, 8, 15, 0, 17, 2, 3, 4, 19, 20, 21, 6, 7, 24, 63, 48, 33, 50, 51, 52, 35, 36, 37, 54, 55, 40, 47, 32, 49, 34, 35, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1595(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 4, 5, 22, 23, 8, 31, 0, 17, 2, 3, 4, 19, 20, 21, 6, 7, 24, 47, 48, 33, 50, 51, 52, 35, 36, 37, 54, 55, 40, 63, 32, 49, 34, 35, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1596(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 4, 5, 22, 23, 8, 15, 16, 1, 2, 3, 4, 19, 20, 21, 6, 7, 24, 63, 32, 49, 50, 51, 52, 35, 36, 37, 54, 55, 40, 47, 48, 33, 34, 35, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1597(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 4, 5, 22, 23, 8, 31, 16, 1, 2, 3, 4, 19, 20, 21, 6, 7, 24, 47, 32, 49, 50, 51, 52, 35, 36, 37, 54, 55, 40, 63, 48, 33, 34, 35, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1598(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 4, 5, 22, 23, 8, 15, 0, 1, 2, 3, 4, 19, 20, 21, 6, 7, 24, 63, 48, 49, 50, 51, 52, 35, 36, 37, 54, 55, 40, 47, 32, 33, 34, 35, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1599(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 4, 5, 22, 23, 8, 31, 0, 1, 2, 3, 4, 19, 20, 21, 6, 7, 24, 47, 48, 49, 50, 51, 52, 35, 36, 37, 54, 55, 40, 63, 32, 33, 34, 35, 36, 51, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1600(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 4, 5, 22, 23, 8, 15, 16, 17, 18, 19, 20, 3, 20, 21, 6, 7, 24, 63, 32, 33, 34, 35, 36, 51, 36, 37, 54, 55, 40, 47, 48, 49, 50, 51, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1601(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 4, 5, 22, 23, 8, 31, 16, 17, 18, 19, 20, 3, 20, 21, 6, 7, 24, 47, 32, 33, 34, 35, 36, 51, 36, 37, 54, 55, 40, 63, 48, 49, 50, 51, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1602(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 4, 5, 22, 23, 8, 15, 0, 17, 18, 19, 20, 3, 20, 21, 6, 7, 24, 63, 48, 33, 34, 35, 36, 51, 36, 37, 54, 55, 40, 47, 32, 49, 50, 51, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1603(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 4, 5, 22, 23, 8, 31, 0, 17, 18, 19, 20, 3, 20, 21, 6, 7, 24, 47, 48, 33, 34, 35, 36, 51, 36, 37, 54, 55, 40, 63, 32, 49, 50, 51, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1604(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 4, 5, 22, 23, 8, 15, 16, 1, 18, 19, 20, 3, 20, 21, 6, 7, 24, 63, 32, 49, 34, 35, 36, 51, 36, 37, 54, 55, 40, 47, 48, 33, 50, 51, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1605(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 4, 5, 22, 23, 8, 31, 16, 1, 18, 19, 20, 3, 20, 21, 6, 7, 24, 47, 32, 49, 34, 35, 36, 51, 36, 37, 54, 55, 40, 63, 48, 33, 50, 51, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1606(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 4, 5, 22, 23, 8, 15, 0, 1, 18, 19, 20, 3, 20, 21, 6, 7, 24, 63, 48, 49, 34, 35, 36, 51, 36, 37, 54, 55, 40, 47, 32, 33, 50, 51, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1607(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 4, 5, 22, 23, 8, 31, 0, 1, 18, 19, 20, 3, 20, 21, 6, 7, 24, 47, 48, 49, 34, 35, 36, 51, 36, 37, 54, 55, 40, 63, 32, 33, 50, 51, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1608(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 4, 5, 22, 23, 8, 15, 16, 17, 2, 19, 20, 3, 20, 21, 6, 7, 24, 63, 32, 33, 50, 35, 36, 51, 36, 37, 54, 55, 40, 47, 48, 49, 34, 51, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1609(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 4, 5, 22, 23, 8, 31, 16, 17, 2, 19, 20, 3, 20, 21, 6, 7, 24, 47, 32, 33, 50, 35, 36, 51, 36, 37, 54, 55, 40, 63, 48, 49, 34, 51, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1610(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 4, 5, 22, 23, 8, 15, 0, 17, 2, 19, 20, 3, 20, 21, 6, 7, 24, 63, 48, 33, 50, 35, 36, 51, 36, 37, 54, 55, 40, 47, 32, 49, 34, 51, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1611(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 4, 5, 22, 23, 8, 31, 0, 17, 2, 19, 20, 3, 20, 21, 6, 7, 24, 47, 48, 33, 50, 35, 36, 51, 36, 37, 54, 55, 40, 63, 32, 49, 34, 51, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1612(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 4, 5, 22, 23, 8, 15, 16, 1, 2, 19, 20, 3, 20, 21, 6, 7, 24, 63, 32, 49, 50, 35, 36, 51, 36, 37, 54, 55, 40, 47, 48, 33, 34, 51, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1613(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 4, 5, 22, 23, 8, 31, 16, 1, 2, 19, 20, 3, 20, 21, 6, 7, 24, 47, 32, 49, 50, 35, 36, 51, 36, 37, 54, 55, 40, 63, 48, 33, 34, 51, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1614(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 4, 5, 22, 23, 8, 15, 0, 1, 2, 19, 20, 3, 20, 21, 6, 7, 24, 63, 48, 49, 50, 35, 36, 51, 36, 37, 54, 55, 40, 47, 32, 33, 34, 51, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1615(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 4, 5, 22, 23, 8, 31, 0, 1, 2, 19, 20, 3, 20, 21, 6, 7, 24, 47, 48, 49, 50, 35, 36, 51, 36, 37, 54, 55, 40, 63, 32, 33, 34, 51, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1616(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 4, 5, 22, 23, 8, 15, 16, 17, 18, 3, 20, 3, 20, 21, 6, 7, 24, 63, 32, 33, 34, 51, 36, 51, 36, 37, 54, 55, 40, 47, 48, 49, 50, 35, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1617(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 4, 5, 22, 23, 8, 31, 16, 17, 18, 3, 20, 3, 20, 21, 6, 7, 24, 47, 32, 33, 34, 51, 36, 51, 36, 37, 54, 55, 40, 63, 48, 49, 50, 35, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1618(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 4, 5, 22, 23, 8, 15, 0, 17, 18, 3, 20, 3, 20, 21, 6, 7, 24, 63, 48, 33, 34, 51, 36, 51, 36, 37, 54, 55, 40, 47, 32, 49, 50, 35, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1619(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 4, 5, 22, 23, 8, 31, 0, 17, 18, 3, 20, 3, 20, 21, 6, 7, 24, 47, 48, 33, 34, 51, 36, 51, 36, 37, 54, 55, 40, 63, 32, 49, 50, 35, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1620(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 4, 5, 22, 23, 8, 15, 16, 1, 18, 3, 20, 3, 20, 21, 6, 7, 24, 63, 32, 49, 34, 51, 36, 51, 36, 37, 54, 55, 40, 47, 48, 33, 50, 35, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1621(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 4, 5, 22, 23, 8, 31, 16, 1, 18, 3, 20, 3, 20, 21, 6, 7, 24, 47, 32, 49, 34, 51, 36, 51, 36, 37, 54, 55, 40, 63, 48, 33, 50, 35, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1622(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 4, 5, 22, 23, 8, 15, 0, 1, 18, 3, 20, 3, 20, 21, 6, 7, 24, 63, 48, 49, 34, 51, 36, 51, 36, 37, 54, 55, 40, 47, 32, 33, 50, 35, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1623(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 4, 5, 22, 23, 8, 31, 0, 1, 18, 3, 20, 3, 20, 21, 6, 7, 24, 47, 48, 49, 34, 51, 36, 51, 36, 37, 54, 55, 40, 63, 32, 33, 50, 35, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1624(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 4, 5, 22, 23, 8, 15, 16, 17, 2, 3, 20, 3, 20, 21, 6, 7, 24, 63, 32, 33, 50, 51, 36, 51, 36, 37, 54, 55, 40, 47, 48, 49, 34, 35, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1625(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 4, 5, 22, 23, 8, 31, 16, 17, 2, 3, 20, 3, 20, 21, 6, 7, 24, 47, 32, 33, 50, 51, 36, 51, 36, 37, 54, 55, 40, 63, 48, 49, 34, 35, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1626(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 4, 5, 22, 23, 8, 15, 0, 17, 2, 3, 20, 3, 20, 21, 6, 7, 24, 63, 48, 33, 50, 51, 36, 51, 36, 37, 54, 55, 40, 47, 32, 49, 34, 35, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1627(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 4, 5, 22, 23, 8, 31, 0, 17, 2, 3, 20, 3, 20, 21, 6, 7, 24, 47, 48, 33, 50, 51, 36, 51, 36, 37, 54, 55, 40, 63, 32, 49, 34, 35, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1628(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 4, 5, 22, 23, 8, 15, 16, 1, 2, 3, 20, 3, 20, 21, 6, 7, 24, 63, 32, 49, 50, 51, 36, 51, 36, 37, 54, 55, 40, 47, 48, 33, 34, 35, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1629(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 4, 5, 22, 23, 8, 31, 16, 1, 2, 3, 20, 3, 20, 21, 6, 7, 24, 47, 32, 49, 50, 51, 36, 51, 36, 37, 54, 55, 40, 63, 48, 33, 34, 35, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1630(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 4, 5, 22, 23, 8, 15, 0, 1, 2, 3, 20, 3, 20, 21, 6, 7, 24, 63, 48, 49, 50, 51, 36, 51, 36, 37, 54, 55, 40, 47, 32, 33, 34, 35, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1631(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 4, 5, 22, 23, 8, 31, 0, 1, 2, 3, 20, 3, 20, 21, 6, 7, 24, 47, 48, 49, 50, 51, 36, 51, 36, 37, 54, 55, 40, 63, 32, 33, 34, 35, 52, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1632(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 4, 5, 22, 23, 8, 15, 16, 17, 18, 19, 4, 3, 20, 21, 6, 7, 24, 63, 32, 33, 34, 35, 52, 51, 36, 37, 54, 55, 40, 47, 48, 49, 50, 51, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1633(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 4, 5, 22, 23, 8, 31, 16, 17, 18, 19, 4, 3, 20, 21, 6, 7, 24, 47, 32, 33, 34, 35, 52, 51, 36, 37, 54, 55, 40, 63, 48, 49, 50, 51, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1634(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 4, 5, 22, 23, 8, 15, 0, 17, 18, 19, 4, 3, 20, 21, 6, 7, 24, 63, 48, 33, 34, 35, 52, 51, 36, 37, 54, 55, 40, 47, 32, 49, 50, 51, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1635(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 4, 5, 22, 23, 8, 31, 0, 17, 18, 19, 4, 3, 20, 21, 6, 7, 24, 47, 48, 33, 34, 35, 52, 51, 36, 37, 54, 55, 40, 63, 32, 49, 50, 51, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1636(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 4, 5, 22, 23, 8, 15, 16, 1, 18, 19, 4, 3, 20, 21, 6, 7, 24, 63, 32, 49, 34, 35, 52, 51, 36, 37, 54, 55, 40, 47, 48, 33, 50, 51, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1637(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 4, 5, 22, 23, 8, 31, 16, 1, 18, 19, 4, 3, 20, 21, 6, 7, 24, 47, 32, 49, 34, 35, 52, 51, 36, 37, 54, 55, 40, 63, 48, 33, 50, 51, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1638(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 4, 5, 22, 23, 8, 15, 0, 1, 18, 19, 4, 3, 20, 21, 6, 7, 24, 63, 48, 49, 34, 35, 52, 51, 36, 37, 54, 55, 40, 47, 32, 33, 50, 51, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1639(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 4, 5, 22, 23, 8, 31, 0, 1, 18, 19, 4, 3, 20, 21, 6, 7, 24, 47, 48, 49, 34, 35, 52, 51, 36, 37, 54, 55, 40, 63, 32, 33, 50, 51, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1640(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 4, 5, 22, 23, 8, 15, 16, 17, 2, 19, 4, 3, 20, 21, 6, 7, 24, 63, 32, 33, 50, 35, 52, 51, 36, 37, 54, 55, 40, 47, 48, 49, 34, 51, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1641(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 4, 5, 22, 23, 8, 31, 16, 17, 2, 19, 4, 3, 20, 21, 6, 7, 24, 47, 32, 33, 50, 35, 52, 51, 36, 37, 54, 55, 40, 63, 48, 49, 34, 51, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1642(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 4, 5, 22, 23, 8, 15, 0, 17, 2, 19, 4, 3, 20, 21, 6, 7, 24, 63, 48, 33, 50, 35, 52, 51, 36, 37, 54, 55, 40, 47, 32, 49, 34, 51, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1643(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 4, 5, 22, 23, 8, 31, 0, 17, 2, 19, 4, 3, 20, 21, 6, 7, 24, 47, 48, 33, 50, 35, 52, 51, 36, 37, 54, 55, 40, 63, 32, 49, 34, 51, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1644(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 4, 5, 22, 23, 8, 15, 16, 1, 2, 19, 4, 3, 20, 21, 6, 7, 24, 63, 32, 49, 50, 35, 52, 51, 36, 37, 54, 55, 40, 47, 48, 33, 34, 51, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1645(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 4, 5, 22, 23, 8, 31, 16, 1, 2, 19, 4, 3, 20, 21, 6, 7, 24, 47, 32, 49, 50, 35, 52, 51, 36, 37, 54, 55, 40, 63, 48, 33, 34, 51, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1646(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 4, 5, 22, 23, 8, 15, 0, 1, 2, 19, 4, 3, 20, 21, 6, 7, 24, 63, 48, 49, 50, 35, 52, 51, 36, 37, 54, 55, 40, 47, 32, 33, 34, 51, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1647(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 4, 5, 22, 23, 8, 31, 0, 1, 2, 19, 4, 3, 20, 21, 6, 7, 24, 47, 48, 49, 50, 35, 52, 51, 36, 37, 54, 55, 40, 63, 32, 33, 34, 51, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1648(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 4, 5, 22, 23, 8, 15, 16, 17, 18, 3, 4, 3, 20, 21, 6, 7, 24, 63, 32, 33, 34, 51, 52, 51, 36, 37, 54, 55, 40, 47, 48, 49, 50, 35, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1649(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 4, 5, 22, 23, 8, 31, 16, 17, 18, 3, 4, 3, 20, 21, 6, 7, 24, 47, 32, 33, 34, 51, 52, 51, 36, 37, 54, 55, 40, 63, 48, 49, 50, 35, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1650(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 4, 5, 22, 23, 8, 15, 0, 17, 18, 3, 4, 3, 20, 21, 6, 7, 24, 63, 48, 33, 34, 51, 52, 51, 36, 37, 54, 55, 40, 47, 32, 49, 50, 35, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1651(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 4, 5, 22, 23, 8, 31, 0, 17, 18, 3, 4, 3, 20, 21, 6, 7, 24, 47, 48, 33, 34, 51, 52, 51, 36, 37, 54, 55, 40, 63, 32, 49, 50, 35, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1652(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 4, 5, 22, 23, 8, 15, 16, 1, 18, 3, 4, 3, 20, 21, 6, 7, 24, 63, 32, 49, 34, 51, 52, 51, 36, 37, 54, 55, 40, 47, 48, 33, 50, 35, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1653(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 4, 5, 22, 23, 8, 31, 16, 1, 18, 3, 4, 3, 20, 21, 6, 7, 24, 47, 32, 49, 34, 51, 52, 51, 36, 37, 54, 55, 40, 63, 48, 33, 50, 35, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1654(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 4, 5, 22, 23, 8, 15, 0, 1, 18, 3, 4, 3, 20, 21, 6, 7, 24, 63, 48, 49, 34, 51, 52, 51, 36, 37, 54, 55, 40, 47, 32, 33, 50, 35, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1655(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 4, 5, 22, 23, 8, 31, 0, 1, 18, 3, 4, 3, 20, 21, 6, 7, 24, 47, 48, 49, 34, 51, 52, 51, 36, 37, 54, 55, 40, 63, 32, 33, 50, 35, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1656(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 4, 5, 22, 23, 8, 15, 16, 17, 2, 3, 4, 3, 20, 21, 6, 7, 24, 63, 32, 33, 50, 51, 52, 51, 36, 37, 54, 55, 40, 47, 48, 49, 34, 35, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1657(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 4, 5, 22, 23, 8, 31, 16, 17, 2, 3, 4, 3, 20, 21, 6, 7, 24, 47, 32, 33, 50, 51, 52, 51, 36, 37, 54, 55, 40, 63, 48, 49, 34, 35, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1658(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 4, 5, 22, 23, 8, 15, 0, 17, 2, 3, 4, 3, 20, 21, 6, 7, 24, 63, 48, 33, 50, 51, 52, 51, 36, 37, 54, 55, 40, 47, 32, 49, 34, 35, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1659(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 4, 5, 22, 23, 8, 31, 0, 17, 2, 3, 4, 3, 20, 21, 6, 7, 24, 47, 48, 33, 50, 51, 52, 51, 36, 37, 54, 55, 40, 63, 32, 49, 34, 35, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1660(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 4, 5, 22, 23, 8, 15, 16, 1, 2, 3, 4, 3, 20, 21, 6, 7, 24, 63, 32, 49, 50, 51, 52, 51, 36, 37, 54, 55, 40, 47, 48, 33, 34, 35, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1661(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 4, 5, 22, 23, 8, 31, 16, 1, 2, 3, 4, 3, 20, 21, 6, 7, 24, 47, 32, 49, 50, 51, 52, 51, 36, 37, 54, 55, 40, 63, 48, 33, 34, 35, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1662(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 4, 5, 22, 23, 8, 15, 0, 1, 2, 3, 4, 3, 20, 21, 6, 7, 24, 63, 48, 49, 50, 51, 52, 51, 36, 37, 54, 55, 40, 47, 32, 33, 34, 35, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1663(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 4, 5, 22, 23, 8, 31, 0, 1, 2, 3, 4, 3, 20, 21, 6, 7, 24, 47, 48, 49, 50, 51, 52, 51, 36, 37, 54, 55, 40, 63, 32, 33, 34, 35, 36, 35, 52, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}


