#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt2432(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 21, 6, 7, 24, 15, 16, 17, 18, 19, 20, 19, 4, 5, 22, 23, 8, 63, 32, 33, 34, 35, 36, 35, 52, 53, 38, 39, 56, 47, 48, 49, 50, 51, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2433(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 21, 6, 7, 24, 31, 16, 17, 18, 19, 20, 19, 4, 5, 22, 23, 8, 47, 32, 33, 34, 35, 36, 35, 52, 53, 38, 39, 56, 63, 48, 49, 50, 51, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2434(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 21, 6, 7, 24, 15, 0, 17, 18, 19, 20, 19, 4, 5, 22, 23, 8, 63, 48, 33, 34, 35, 36, 35, 52, 53, 38, 39, 56, 47, 32, 49, 50, 51, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2435(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 21, 6, 7, 24, 31, 0, 17, 18, 19, 20, 19, 4, 5, 22, 23, 8, 47, 48, 33, 34, 35, 36, 35, 52, 53, 38, 39, 56, 63, 32, 49, 50, 51, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2436(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 21, 6, 7, 24, 15, 16, 1, 18, 19, 20, 19, 4, 5, 22, 23, 8, 63, 32, 49, 34, 35, 36, 35, 52, 53, 38, 39, 56, 47, 48, 33, 50, 51, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2437(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 21, 6, 7, 24, 31, 16, 1, 18, 19, 20, 19, 4, 5, 22, 23, 8, 47, 32, 49, 34, 35, 36, 35, 52, 53, 38, 39, 56, 63, 48, 33, 50, 51, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2438(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 21, 6, 7, 24, 15, 0, 1, 18, 19, 20, 19, 4, 5, 22, 23, 8, 63, 48, 49, 34, 35, 36, 35, 52, 53, 38, 39, 56, 47, 32, 33, 50, 51, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2439(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 21, 6, 7, 24, 31, 0, 1, 18, 19, 20, 19, 4, 5, 22, 23, 8, 47, 48, 49, 34, 35, 36, 35, 52, 53, 38, 39, 56, 63, 32, 33, 50, 51, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2440(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 21, 6, 7, 24, 15, 16, 17, 2, 19, 20, 19, 4, 5, 22, 23, 8, 63, 32, 33, 50, 35, 36, 35, 52, 53, 38, 39, 56, 47, 48, 49, 34, 51, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2441(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 21, 6, 7, 24, 31, 16, 17, 2, 19, 20, 19, 4, 5, 22, 23, 8, 47, 32, 33, 50, 35, 36, 35, 52, 53, 38, 39, 56, 63, 48, 49, 34, 51, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2442(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 21, 6, 7, 24, 15, 0, 17, 2, 19, 20, 19, 4, 5, 22, 23, 8, 63, 48, 33, 50, 35, 36, 35, 52, 53, 38, 39, 56, 47, 32, 49, 34, 51, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2443(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 21, 6, 7, 24, 31, 0, 17, 2, 19, 20, 19, 4, 5, 22, 23, 8, 47, 48, 33, 50, 35, 36, 35, 52, 53, 38, 39, 56, 63, 32, 49, 34, 51, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2444(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 21, 6, 7, 24, 15, 16, 1, 2, 19, 20, 19, 4, 5, 22, 23, 8, 63, 32, 49, 50, 35, 36, 35, 52, 53, 38, 39, 56, 47, 48, 33, 34, 51, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2445(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 21, 6, 7, 24, 31, 16, 1, 2, 19, 20, 19, 4, 5, 22, 23, 8, 47, 32, 49, 50, 35, 36, 35, 52, 53, 38, 39, 56, 63, 48, 33, 34, 51, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2446(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 21, 6, 7, 24, 15, 0, 1, 2, 19, 20, 19, 4, 5, 22, 23, 8, 63, 48, 49, 50, 35, 36, 35, 52, 53, 38, 39, 56, 47, 32, 33, 34, 51, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2447(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 21, 6, 7, 24, 31, 0, 1, 2, 19, 20, 19, 4, 5, 22, 23, 8, 47, 48, 49, 50, 35, 36, 35, 52, 53, 38, 39, 56, 63, 32, 33, 34, 51, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2448(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 21, 6, 7, 24, 15, 16, 17, 18, 3, 20, 19, 4, 5, 22, 23, 8, 63, 32, 33, 34, 51, 36, 35, 52, 53, 38, 39, 56, 47, 48, 49, 50, 35, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2449(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 21, 6, 7, 24, 31, 16, 17, 18, 3, 20, 19, 4, 5, 22, 23, 8, 47, 32, 33, 34, 51, 36, 35, 52, 53, 38, 39, 56, 63, 48, 49, 50, 35, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2450(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 21, 6, 7, 24, 15, 0, 17, 18, 3, 20, 19, 4, 5, 22, 23, 8, 63, 48, 33, 34, 51, 36, 35, 52, 53, 38, 39, 56, 47, 32, 49, 50, 35, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2451(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 21, 6, 7, 24, 31, 0, 17, 18, 3, 20, 19, 4, 5, 22, 23, 8, 47, 48, 33, 34, 51, 36, 35, 52, 53, 38, 39, 56, 63, 32, 49, 50, 35, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2452(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 21, 6, 7, 24, 15, 16, 1, 18, 3, 20, 19, 4, 5, 22, 23, 8, 63, 32, 49, 34, 51, 36, 35, 52, 53, 38, 39, 56, 47, 48, 33, 50, 35, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2453(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 21, 6, 7, 24, 31, 16, 1, 18, 3, 20, 19, 4, 5, 22, 23, 8, 47, 32, 49, 34, 51, 36, 35, 52, 53, 38, 39, 56, 63, 48, 33, 50, 35, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2454(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 21, 6, 7, 24, 15, 0, 1, 18, 3, 20, 19, 4, 5, 22, 23, 8, 63, 48, 49, 34, 51, 36, 35, 52, 53, 38, 39, 56, 47, 32, 33, 50, 35, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2455(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 21, 6, 7, 24, 31, 0, 1, 18, 3, 20, 19, 4, 5, 22, 23, 8, 47, 48, 49, 34, 51, 36, 35, 52, 53, 38, 39, 56, 63, 32, 33, 50, 35, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2456(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 21, 6, 7, 24, 15, 16, 17, 2, 3, 20, 19, 4, 5, 22, 23, 8, 63, 32, 33, 50, 51, 36, 35, 52, 53, 38, 39, 56, 47, 48, 49, 34, 35, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2457(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 21, 6, 7, 24, 31, 16, 17, 2, 3, 20, 19, 4, 5, 22, 23, 8, 47, 32, 33, 50, 51, 36, 35, 52, 53, 38, 39, 56, 63, 48, 49, 34, 35, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2458(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 21, 6, 7, 24, 15, 0, 17, 2, 3, 20, 19, 4, 5, 22, 23, 8, 63, 48, 33, 50, 51, 36, 35, 52, 53, 38, 39, 56, 47, 32, 49, 34, 35, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2459(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 21, 6, 7, 24, 31, 0, 17, 2, 3, 20, 19, 4, 5, 22, 23, 8, 47, 48, 33, 50, 51, 36, 35, 52, 53, 38, 39, 56, 63, 32, 49, 34, 35, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2460(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 21, 6, 7, 24, 15, 16, 1, 2, 3, 20, 19, 4, 5, 22, 23, 8, 63, 32, 49, 50, 51, 36, 35, 52, 53, 38, 39, 56, 47, 48, 33, 34, 35, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2461(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 21, 6, 7, 24, 31, 16, 1, 2, 3, 20, 19, 4, 5, 22, 23, 8, 47, 32, 49, 50, 51, 36, 35, 52, 53, 38, 39, 56, 63, 48, 33, 34, 35, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2462(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 21, 6, 7, 24, 15, 0, 1, 2, 3, 20, 19, 4, 5, 22, 23, 8, 63, 48, 49, 50, 51, 36, 35, 52, 53, 38, 39, 56, 47, 32, 33, 34, 35, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2463(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 21, 6, 7, 24, 31, 0, 1, 2, 3, 20, 19, 4, 5, 22, 23, 8, 47, 48, 49, 50, 51, 36, 35, 52, 53, 38, 39, 56, 63, 32, 33, 34, 35, 52, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2464(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 21, 6, 7, 24, 15, 16, 17, 18, 19, 4, 19, 4, 5, 22, 23, 8, 63, 32, 33, 34, 35, 52, 35, 52, 53, 38, 39, 56, 47, 48, 49, 50, 51, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2465(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 21, 6, 7, 24, 31, 16, 17, 18, 19, 4, 19, 4, 5, 22, 23, 8, 47, 32, 33, 34, 35, 52, 35, 52, 53, 38, 39, 56, 63, 48, 49, 50, 51, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2466(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 21, 6, 7, 24, 15, 0, 17, 18, 19, 4, 19, 4, 5, 22, 23, 8, 63, 48, 33, 34, 35, 52, 35, 52, 53, 38, 39, 56, 47, 32, 49, 50, 51, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2467(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 21, 6, 7, 24, 31, 0, 17, 18, 19, 4, 19, 4, 5, 22, 23, 8, 47, 48, 33, 34, 35, 52, 35, 52, 53, 38, 39, 56, 63, 32, 49, 50, 51, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2468(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 21, 6, 7, 24, 15, 16, 1, 18, 19, 4, 19, 4, 5, 22, 23, 8, 63, 32, 49, 34, 35, 52, 35, 52, 53, 38, 39, 56, 47, 48, 33, 50, 51, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2469(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 21, 6, 7, 24, 31, 16, 1, 18, 19, 4, 19, 4, 5, 22, 23, 8, 47, 32, 49, 34, 35, 52, 35, 52, 53, 38, 39, 56, 63, 48, 33, 50, 51, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2470(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 21, 6, 7, 24, 15, 0, 1, 18, 19, 4, 19, 4, 5, 22, 23, 8, 63, 48, 49, 34, 35, 52, 35, 52, 53, 38, 39, 56, 47, 32, 33, 50, 51, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2471(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 21, 6, 7, 24, 31, 0, 1, 18, 19, 4, 19, 4, 5, 22, 23, 8, 47, 48, 49, 34, 35, 52, 35, 52, 53, 38, 39, 56, 63, 32, 33, 50, 51, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2472(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 21, 6, 7, 24, 15, 16, 17, 2, 19, 4, 19, 4, 5, 22, 23, 8, 63, 32, 33, 50, 35, 52, 35, 52, 53, 38, 39, 56, 47, 48, 49, 34, 51, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2473(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 21, 6, 7, 24, 31, 16, 17, 2, 19, 4, 19, 4, 5, 22, 23, 8, 47, 32, 33, 50, 35, 52, 35, 52, 53, 38, 39, 56, 63, 48, 49, 34, 51, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2474(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 21, 6, 7, 24, 15, 0, 17, 2, 19, 4, 19, 4, 5, 22, 23, 8, 63, 48, 33, 50, 35, 52, 35, 52, 53, 38, 39, 56, 47, 32, 49, 34, 51, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2475(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 21, 6, 7, 24, 31, 0, 17, 2, 19, 4, 19, 4, 5, 22, 23, 8, 47, 48, 33, 50, 35, 52, 35, 52, 53, 38, 39, 56, 63, 32, 49, 34, 51, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2476(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 21, 6, 7, 24, 15, 16, 1, 2, 19, 4, 19, 4, 5, 22, 23, 8, 63, 32, 49, 50, 35, 52, 35, 52, 53, 38, 39, 56, 47, 48, 33, 34, 51, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2477(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 21, 6, 7, 24, 31, 16, 1, 2, 19, 4, 19, 4, 5, 22, 23, 8, 47, 32, 49, 50, 35, 52, 35, 52, 53, 38, 39, 56, 63, 48, 33, 34, 51, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2478(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 21, 6, 7, 24, 15, 0, 1, 2, 19, 4, 19, 4, 5, 22, 23, 8, 63, 48, 49, 50, 35, 52, 35, 52, 53, 38, 39, 56, 47, 32, 33, 34, 51, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2479(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 21, 6, 7, 24, 31, 0, 1, 2, 19, 4, 19, 4, 5, 22, 23, 8, 47, 48, 49, 50, 35, 52, 35, 52, 53, 38, 39, 56, 63, 32, 33, 34, 51, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2480(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 21, 6, 7, 24, 15, 16, 17, 18, 3, 4, 19, 4, 5, 22, 23, 8, 63, 32, 33, 34, 51, 52, 35, 52, 53, 38, 39, 56, 47, 48, 49, 50, 35, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2481(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 21, 6, 7, 24, 31, 16, 17, 18, 3, 4, 19, 4, 5, 22, 23, 8, 47, 32, 33, 34, 51, 52, 35, 52, 53, 38, 39, 56, 63, 48, 49, 50, 35, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2482(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 21, 6, 7, 24, 15, 0, 17, 18, 3, 4, 19, 4, 5, 22, 23, 8, 63, 48, 33, 34, 51, 52, 35, 52, 53, 38, 39, 56, 47, 32, 49, 50, 35, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2483(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 21, 6, 7, 24, 31, 0, 17, 18, 3, 4, 19, 4, 5, 22, 23, 8, 47, 48, 33, 34, 51, 52, 35, 52, 53, 38, 39, 56, 63, 32, 49, 50, 35, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2484(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 21, 6, 7, 24, 15, 16, 1, 18, 3, 4, 19, 4, 5, 22, 23, 8, 63, 32, 49, 34, 51, 52, 35, 52, 53, 38, 39, 56, 47, 48, 33, 50, 35, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2485(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 21, 6, 7, 24, 31, 16, 1, 18, 3, 4, 19, 4, 5, 22, 23, 8, 47, 32, 49, 34, 51, 52, 35, 52, 53, 38, 39, 56, 63, 48, 33, 50, 35, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2486(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 21, 6, 7, 24, 15, 0, 1, 18, 3, 4, 19, 4, 5, 22, 23, 8, 63, 48, 49, 34, 51, 52, 35, 52, 53, 38, 39, 56, 47, 32, 33, 50, 35, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2487(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 21, 6, 7, 24, 31, 0, 1, 18, 3, 4, 19, 4, 5, 22, 23, 8, 47, 48, 49, 34, 51, 52, 35, 52, 53, 38, 39, 56, 63, 32, 33, 50, 35, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2488(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 21, 6, 7, 24, 15, 16, 17, 2, 3, 4, 19, 4, 5, 22, 23, 8, 63, 32, 33, 50, 51, 52, 35, 52, 53, 38, 39, 56, 47, 48, 49, 34, 35, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2489(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 21, 6, 7, 24, 31, 16, 17, 2, 3, 4, 19, 4, 5, 22, 23, 8, 47, 32, 33, 50, 51, 52, 35, 52, 53, 38, 39, 56, 63, 48, 49, 34, 35, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2490(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 21, 6, 7, 24, 15, 0, 17, 2, 3, 4, 19, 4, 5, 22, 23, 8, 63, 48, 33, 50, 51, 52, 35, 52, 53, 38, 39, 56, 47, 32, 49, 34, 35, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2491(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 21, 6, 7, 24, 31, 0, 17, 2, 3, 4, 19, 4, 5, 22, 23, 8, 47, 48, 33, 50, 51, 52, 35, 52, 53, 38, 39, 56, 63, 32, 49, 34, 35, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2492(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 21, 6, 7, 24, 15, 16, 1, 2, 3, 4, 19, 4, 5, 22, 23, 8, 63, 32, 49, 50, 51, 52, 35, 52, 53, 38, 39, 56, 47, 48, 33, 34, 35, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2493(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 21, 6, 7, 24, 31, 16, 1, 2, 3, 4, 19, 4, 5, 22, 23, 8, 47, 32, 49, 50, 51, 52, 35, 52, 53, 38, 39, 56, 63, 48, 33, 34, 35, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2494(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 21, 6, 7, 24, 15, 0, 1, 2, 3, 4, 19, 4, 5, 22, 23, 8, 63, 48, 49, 50, 51, 52, 35, 52, 53, 38, 39, 56, 47, 32, 33, 34, 35, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2495(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 21, 6, 7, 24, 31, 0, 1, 2, 3, 4, 19, 4, 5, 22, 23, 8, 47, 48, 49, 50, 51, 52, 35, 52, 53, 38, 39, 56, 63, 32, 33, 34, 35, 36, 51, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2496(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 21, 6, 7, 24, 15, 16, 17, 18, 19, 20, 3, 4, 5, 22, 23, 8, 63, 32, 33, 34, 35, 36, 51, 52, 53, 38, 39, 56, 47, 48, 49, 50, 51, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2497(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 21, 6, 7, 24, 31, 16, 17, 18, 19, 20, 3, 4, 5, 22, 23, 8, 47, 32, 33, 34, 35, 36, 51, 52, 53, 38, 39, 56, 63, 48, 49, 50, 51, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2498(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 21, 6, 7, 24, 15, 0, 17, 18, 19, 20, 3, 4, 5, 22, 23, 8, 63, 48, 33, 34, 35, 36, 51, 52, 53, 38, 39, 56, 47, 32, 49, 50, 51, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2499(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 21, 6, 7, 24, 31, 0, 17, 18, 19, 20, 3, 4, 5, 22, 23, 8, 47, 48, 33, 34, 35, 36, 51, 52, 53, 38, 39, 56, 63, 32, 49, 50, 51, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2500(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 21, 6, 7, 24, 15, 16, 1, 18, 19, 20, 3, 4, 5, 22, 23, 8, 63, 32, 49, 34, 35, 36, 51, 52, 53, 38, 39, 56, 47, 48, 33, 50, 51, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2501(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 21, 6, 7, 24, 31, 16, 1, 18, 19, 20, 3, 4, 5, 22, 23, 8, 47, 32, 49, 34, 35, 36, 51, 52, 53, 38, 39, 56, 63, 48, 33, 50, 51, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2502(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 21, 6, 7, 24, 15, 0, 1, 18, 19, 20, 3, 4, 5, 22, 23, 8, 63, 48, 49, 34, 35, 36, 51, 52, 53, 38, 39, 56, 47, 32, 33, 50, 51, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2503(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 21, 6, 7, 24, 31, 0, 1, 18, 19, 20, 3, 4, 5, 22, 23, 8, 47, 48, 49, 34, 35, 36, 51, 52, 53, 38, 39, 56, 63, 32, 33, 50, 51, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2504(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 21, 6, 7, 24, 15, 16, 17, 2, 19, 20, 3, 4, 5, 22, 23, 8, 63, 32, 33, 50, 35, 36, 51, 52, 53, 38, 39, 56, 47, 48, 49, 34, 51, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2505(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 21, 6, 7, 24, 31, 16, 17, 2, 19, 20, 3, 4, 5, 22, 23, 8, 47, 32, 33, 50, 35, 36, 51, 52, 53, 38, 39, 56, 63, 48, 49, 34, 51, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2506(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 21, 6, 7, 24, 15, 0, 17, 2, 19, 20, 3, 4, 5, 22, 23, 8, 63, 48, 33, 50, 35, 36, 51, 52, 53, 38, 39, 56, 47, 32, 49, 34, 51, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2507(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 21, 6, 7, 24, 31, 0, 17, 2, 19, 20, 3, 4, 5, 22, 23, 8, 47, 48, 33, 50, 35, 36, 51, 52, 53, 38, 39, 56, 63, 32, 49, 34, 51, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2508(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 21, 6, 7, 24, 15, 16, 1, 2, 19, 20, 3, 4, 5, 22, 23, 8, 63, 32, 49, 50, 35, 36, 51, 52, 53, 38, 39, 56, 47, 48, 33, 34, 51, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2509(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 21, 6, 7, 24, 31, 16, 1, 2, 19, 20, 3, 4, 5, 22, 23, 8, 47, 32, 49, 50, 35, 36, 51, 52, 53, 38, 39, 56, 63, 48, 33, 34, 51, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2510(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 21, 6, 7, 24, 15, 0, 1, 2, 19, 20, 3, 4, 5, 22, 23, 8, 63, 48, 49, 50, 35, 36, 51, 52, 53, 38, 39, 56, 47, 32, 33, 34, 51, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2511(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 21, 6, 7, 24, 31, 0, 1, 2, 19, 20, 3, 4, 5, 22, 23, 8, 47, 48, 49, 50, 35, 36, 51, 52, 53, 38, 39, 56, 63, 32, 33, 34, 51, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2512(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 21, 6, 7, 24, 15, 16, 17, 18, 3, 20, 3, 4, 5, 22, 23, 8, 63, 32, 33, 34, 51, 36, 51, 52, 53, 38, 39, 56, 47, 48, 49, 50, 35, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2513(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 21, 6, 7, 24, 31, 16, 17, 18, 3, 20, 3, 4, 5, 22, 23, 8, 47, 32, 33, 34, 51, 36, 51, 52, 53, 38, 39, 56, 63, 48, 49, 50, 35, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2514(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 21, 6, 7, 24, 15, 0, 17, 18, 3, 20, 3, 4, 5, 22, 23, 8, 63, 48, 33, 34, 51, 36, 51, 52, 53, 38, 39, 56, 47, 32, 49, 50, 35, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2515(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 21, 6, 7, 24, 31, 0, 17, 18, 3, 20, 3, 4, 5, 22, 23, 8, 47, 48, 33, 34, 51, 36, 51, 52, 53, 38, 39, 56, 63, 32, 49, 50, 35, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2516(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 21, 6, 7, 24, 15, 16, 1, 18, 3, 20, 3, 4, 5, 22, 23, 8, 63, 32, 49, 34, 51, 36, 51, 52, 53, 38, 39, 56, 47, 48, 33, 50, 35, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2517(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 21, 6, 7, 24, 31, 16, 1, 18, 3, 20, 3, 4, 5, 22, 23, 8, 47, 32, 49, 34, 51, 36, 51, 52, 53, 38, 39, 56, 63, 48, 33, 50, 35, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2518(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 21, 6, 7, 24, 15, 0, 1, 18, 3, 20, 3, 4, 5, 22, 23, 8, 63, 48, 49, 34, 51, 36, 51, 52, 53, 38, 39, 56, 47, 32, 33, 50, 35, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2519(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 21, 6, 7, 24, 31, 0, 1, 18, 3, 20, 3, 4, 5, 22, 23, 8, 47, 48, 49, 34, 51, 36, 51, 52, 53, 38, 39, 56, 63, 32, 33, 50, 35, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2520(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 21, 6, 7, 24, 15, 16, 17, 2, 3, 20, 3, 4, 5, 22, 23, 8, 63, 32, 33, 50, 51, 36, 51, 52, 53, 38, 39, 56, 47, 48, 49, 34, 35, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2521(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 21, 6, 7, 24, 31, 16, 17, 2, 3, 20, 3, 4, 5, 22, 23, 8, 47, 32, 33, 50, 51, 36, 51, 52, 53, 38, 39, 56, 63, 48, 49, 34, 35, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2522(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 21, 6, 7, 24, 15, 0, 17, 2, 3, 20, 3, 4, 5, 22, 23, 8, 63, 48, 33, 50, 51, 36, 51, 52, 53, 38, 39, 56, 47, 32, 49, 34, 35, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2523(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 21, 6, 7, 24, 31, 0, 17, 2, 3, 20, 3, 4, 5, 22, 23, 8, 47, 48, 33, 50, 51, 36, 51, 52, 53, 38, 39, 56, 63, 32, 49, 34, 35, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2524(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 21, 6, 7, 24, 15, 16, 1, 2, 3, 20, 3, 4, 5, 22, 23, 8, 63, 32, 49, 50, 51, 36, 51, 52, 53, 38, 39, 56, 47, 48, 33, 34, 35, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2525(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 21, 6, 7, 24, 31, 16, 1, 2, 3, 20, 3, 4, 5, 22, 23, 8, 47, 32, 49, 50, 51, 36, 51, 52, 53, 38, 39, 56, 63, 48, 33, 34, 35, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2526(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 21, 6, 7, 24, 15, 0, 1, 2, 3, 20, 3, 4, 5, 22, 23, 8, 63, 48, 49, 50, 51, 36, 51, 52, 53, 38, 39, 56, 47, 32, 33, 34, 35, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2527(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 21, 6, 7, 24, 31, 0, 1, 2, 3, 20, 3, 4, 5, 22, 23, 8, 47, 48, 49, 50, 51, 36, 51, 52, 53, 38, 39, 56, 63, 32, 33, 34, 35, 52, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2528(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 21, 6, 7, 24, 15, 16, 17, 18, 19, 4, 3, 4, 5, 22, 23, 8, 63, 32, 33, 34, 35, 52, 51, 52, 53, 38, 39, 56, 47, 48, 49, 50, 51, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2529(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 21, 6, 7, 24, 31, 16, 17, 18, 19, 4, 3, 4, 5, 22, 23, 8, 47, 32, 33, 34, 35, 52, 51, 52, 53, 38, 39, 56, 63, 48, 49, 50, 51, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2530(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 21, 6, 7, 24, 15, 0, 17, 18, 19, 4, 3, 4, 5, 22, 23, 8, 63, 48, 33, 34, 35, 52, 51, 52, 53, 38, 39, 56, 47, 32, 49, 50, 51, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2531(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 21, 6, 7, 24, 31, 0, 17, 18, 19, 4, 3, 4, 5, 22, 23, 8, 47, 48, 33, 34, 35, 52, 51, 52, 53, 38, 39, 56, 63, 32, 49, 50, 51, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2532(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 21, 6, 7, 24, 15, 16, 1, 18, 19, 4, 3, 4, 5, 22, 23, 8, 63, 32, 49, 34, 35, 52, 51, 52, 53, 38, 39, 56, 47, 48, 33, 50, 51, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2533(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 21, 6, 7, 24, 31, 16, 1, 18, 19, 4, 3, 4, 5, 22, 23, 8, 47, 32, 49, 34, 35, 52, 51, 52, 53, 38, 39, 56, 63, 48, 33, 50, 51, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2534(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 21, 6, 7, 24, 15, 0, 1, 18, 19, 4, 3, 4, 5, 22, 23, 8, 63, 48, 49, 34, 35, 52, 51, 52, 53, 38, 39, 56, 47, 32, 33, 50, 51, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2535(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 21, 6, 7, 24, 31, 0, 1, 18, 19, 4, 3, 4, 5, 22, 23, 8, 47, 48, 49, 34, 35, 52, 51, 52, 53, 38, 39, 56, 63, 32, 33, 50, 51, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2536(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 21, 6, 7, 24, 15, 16, 17, 2, 19, 4, 3, 4, 5, 22, 23, 8, 63, 32, 33, 50, 35, 52, 51, 52, 53, 38, 39, 56, 47, 48, 49, 34, 51, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2537(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 21, 6, 7, 24, 31, 16, 17, 2, 19, 4, 3, 4, 5, 22, 23, 8, 47, 32, 33, 50, 35, 52, 51, 52, 53, 38, 39, 56, 63, 48, 49, 34, 51, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2538(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 21, 6, 7, 24, 15, 0, 17, 2, 19, 4, 3, 4, 5, 22, 23, 8, 63, 48, 33, 50, 35, 52, 51, 52, 53, 38, 39, 56, 47, 32, 49, 34, 51, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2539(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 21, 6, 7, 24, 31, 0, 17, 2, 19, 4, 3, 4, 5, 22, 23, 8, 47, 48, 33, 50, 35, 52, 51, 52, 53, 38, 39, 56, 63, 32, 49, 34, 51, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2540(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 21, 6, 7, 24, 15, 16, 1, 2, 19, 4, 3, 4, 5, 22, 23, 8, 63, 32, 49, 50, 35, 52, 51, 52, 53, 38, 39, 56, 47, 48, 33, 34, 51, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2541(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 21, 6, 7, 24, 31, 16, 1, 2, 19, 4, 3, 4, 5, 22, 23, 8, 47, 32, 49, 50, 35, 52, 51, 52, 53, 38, 39, 56, 63, 48, 33, 34, 51, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2542(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 21, 6, 7, 24, 15, 0, 1, 2, 19, 4, 3, 4, 5, 22, 23, 8, 63, 48, 49, 50, 35, 52, 51, 52, 53, 38, 39, 56, 47, 32, 33, 34, 51, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2543(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 21, 6, 7, 24, 31, 0, 1, 2, 19, 4, 3, 4, 5, 22, 23, 8, 47, 48, 49, 50, 35, 52, 51, 52, 53, 38, 39, 56, 63, 32, 33, 34, 51, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2544(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 21, 6, 7, 24, 15, 16, 17, 18, 3, 4, 3, 4, 5, 22, 23, 8, 63, 32, 33, 34, 51, 52, 51, 52, 53, 38, 39, 56, 47, 48, 49, 50, 35, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2545(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 21, 6, 7, 24, 31, 16, 17, 18, 3, 4, 3, 4, 5, 22, 23, 8, 47, 32, 33, 34, 51, 52, 51, 52, 53, 38, 39, 56, 63, 48, 49, 50, 35, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2546(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 21, 6, 7, 24, 15, 0, 17, 18, 3, 4, 3, 4, 5, 22, 23, 8, 63, 48, 33, 34, 51, 52, 51, 52, 53, 38, 39, 56, 47, 32, 49, 50, 35, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2547(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 21, 6, 7, 24, 31, 0, 17, 18, 3, 4, 3, 4, 5, 22, 23, 8, 47, 48, 33, 34, 51, 52, 51, 52, 53, 38, 39, 56, 63, 32, 49, 50, 35, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2548(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 21, 6, 7, 24, 15, 16, 1, 18, 3, 4, 3, 4, 5, 22, 23, 8, 63, 32, 49, 34, 51, 52, 51, 52, 53, 38, 39, 56, 47, 48, 33, 50, 35, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2549(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 21, 6, 7, 24, 31, 16, 1, 18, 3, 4, 3, 4, 5, 22, 23, 8, 47, 32, 49, 34, 51, 52, 51, 52, 53, 38, 39, 56, 63, 48, 33, 50, 35, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2550(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 21, 6, 7, 24, 15, 0, 1, 18, 3, 4, 3, 4, 5, 22, 23, 8, 63, 48, 49, 34, 51, 52, 51, 52, 53, 38, 39, 56, 47, 32, 33, 50, 35, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2551(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 21, 6, 7, 24, 31, 0, 1, 18, 3, 4, 3, 4, 5, 22, 23, 8, 47, 48, 49, 34, 51, 52, 51, 52, 53, 38, 39, 56, 63, 32, 33, 50, 35, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2552(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 21, 6, 7, 24, 15, 16, 17, 2, 3, 4, 3, 4, 5, 22, 23, 8, 63, 32, 33, 50, 51, 52, 51, 52, 53, 38, 39, 56, 47, 48, 49, 34, 35, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2553(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 21, 6, 7, 24, 31, 16, 17, 2, 3, 4, 3, 4, 5, 22, 23, 8, 47, 32, 33, 50, 51, 52, 51, 52, 53, 38, 39, 56, 63, 48, 49, 34, 35, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2554(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 21, 6, 7, 24, 15, 0, 17, 2, 3, 4, 3, 4, 5, 22, 23, 8, 63, 48, 33, 50, 51, 52, 51, 52, 53, 38, 39, 56, 47, 32, 49, 34, 35, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2555(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 21, 6, 7, 24, 31, 0, 17, 2, 3, 4, 3, 4, 5, 22, 23, 8, 47, 48, 33, 50, 51, 52, 51, 52, 53, 38, 39, 56, 63, 32, 49, 34, 35, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2556(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 21, 6, 7, 24, 15, 16, 1, 2, 3, 4, 3, 4, 5, 22, 23, 8, 63, 32, 49, 50, 51, 52, 51, 52, 53, 38, 39, 56, 47, 48, 33, 34, 35, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2557(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 21, 6, 7, 24, 31, 16, 1, 2, 3, 4, 3, 4, 5, 22, 23, 8, 47, 32, 49, 50, 51, 52, 51, 52, 53, 38, 39, 56, 63, 48, 33, 34, 35, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2558(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 21, 6, 7, 24, 15, 0, 1, 2, 3, 4, 3, 4, 5, 22, 23, 8, 63, 48, 49, 50, 51, 52, 51, 52, 53, 38, 39, 56, 47, 32, 33, 34, 35, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2559(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 21, 6, 7, 24, 31, 0, 1, 2, 3, 4, 3, 4, 5, 22, 23, 8, 47, 48, 49, 50, 51, 52, 51, 52, 53, 38, 39, 56, 63, 32, 33, 34, 35, 36, 35, 36, 37, 54, 55, 40>(unchecked_hashes, bitsplitted_keys);
}


