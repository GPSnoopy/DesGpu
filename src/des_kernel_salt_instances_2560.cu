#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt2560(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 4, 5, 22, 7, 24, 15, 16, 17, 18, 19, 20, 19, 20, 21, 6, 23, 8, 63, 32, 33, 34, 35, 36, 35, 36, 37, 54, 39, 56, 47, 48, 49, 50, 51, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2561(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 4, 5, 22, 7, 24, 31, 16, 17, 18, 19, 20, 19, 20, 21, 6, 23, 8, 47, 32, 33, 34, 35, 36, 35, 36, 37, 54, 39, 56, 63, 48, 49, 50, 51, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2562(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 4, 5, 22, 7, 24, 15, 0, 17, 18, 19, 20, 19, 20, 21, 6, 23, 8, 63, 48, 33, 34, 35, 36, 35, 36, 37, 54, 39, 56, 47, 32, 49, 50, 51, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2563(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 4, 5, 22, 7, 24, 31, 0, 17, 18, 19, 20, 19, 20, 21, 6, 23, 8, 47, 48, 33, 34, 35, 36, 35, 36, 37, 54, 39, 56, 63, 32, 49, 50, 51, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2564(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 4, 5, 22, 7, 24, 15, 16, 1, 18, 19, 20, 19, 20, 21, 6, 23, 8, 63, 32, 49, 34, 35, 36, 35, 36, 37, 54, 39, 56, 47, 48, 33, 50, 51, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2565(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 4, 5, 22, 7, 24, 31, 16, 1, 18, 19, 20, 19, 20, 21, 6, 23, 8, 47, 32, 49, 34, 35, 36, 35, 36, 37, 54, 39, 56, 63, 48, 33, 50, 51, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2566(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 4, 5, 22, 7, 24, 15, 0, 1, 18, 19, 20, 19, 20, 21, 6, 23, 8, 63, 48, 49, 34, 35, 36, 35, 36, 37, 54, 39, 56, 47, 32, 33, 50, 51, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2567(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 4, 5, 22, 7, 24, 31, 0, 1, 18, 19, 20, 19, 20, 21, 6, 23, 8, 47, 48, 49, 34, 35, 36, 35, 36, 37, 54, 39, 56, 63, 32, 33, 50, 51, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2568(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 4, 5, 22, 7, 24, 15, 16, 17, 2, 19, 20, 19, 20, 21, 6, 23, 8, 63, 32, 33, 50, 35, 36, 35, 36, 37, 54, 39, 56, 47, 48, 49, 34, 51, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2569(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 4, 5, 22, 7, 24, 31, 16, 17, 2, 19, 20, 19, 20, 21, 6, 23, 8, 47, 32, 33, 50, 35, 36, 35, 36, 37, 54, 39, 56, 63, 48, 49, 34, 51, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2570(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 4, 5, 22, 7, 24, 15, 0, 17, 2, 19, 20, 19, 20, 21, 6, 23, 8, 63, 48, 33, 50, 35, 36, 35, 36, 37, 54, 39, 56, 47, 32, 49, 34, 51, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2571(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 4, 5, 22, 7, 24, 31, 0, 17, 2, 19, 20, 19, 20, 21, 6, 23, 8, 47, 48, 33, 50, 35, 36, 35, 36, 37, 54, 39, 56, 63, 32, 49, 34, 51, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2572(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 4, 5, 22, 7, 24, 15, 16, 1, 2, 19, 20, 19, 20, 21, 6, 23, 8, 63, 32, 49, 50, 35, 36, 35, 36, 37, 54, 39, 56, 47, 48, 33, 34, 51, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2573(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 4, 5, 22, 7, 24, 31, 16, 1, 2, 19, 20, 19, 20, 21, 6, 23, 8, 47, 32, 49, 50, 35, 36, 35, 36, 37, 54, 39, 56, 63, 48, 33, 34, 51, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2574(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 4, 5, 22, 7, 24, 15, 0, 1, 2, 19, 20, 19, 20, 21, 6, 23, 8, 63, 48, 49, 50, 35, 36, 35, 36, 37, 54, 39, 56, 47, 32, 33, 34, 51, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2575(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 4, 5, 22, 7, 24, 31, 0, 1, 2, 19, 20, 19, 20, 21, 6, 23, 8, 47, 48, 49, 50, 35, 36, 35, 36, 37, 54, 39, 56, 63, 32, 33, 34, 51, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2576(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 4, 5, 22, 7, 24, 15, 16, 17, 18, 3, 20, 19, 20, 21, 6, 23, 8, 63, 32, 33, 34, 51, 36, 35, 36, 37, 54, 39, 56, 47, 48, 49, 50, 35, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2577(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 4, 5, 22, 7, 24, 31, 16, 17, 18, 3, 20, 19, 20, 21, 6, 23, 8, 47, 32, 33, 34, 51, 36, 35, 36, 37, 54, 39, 56, 63, 48, 49, 50, 35, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2578(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 4, 5, 22, 7, 24, 15, 0, 17, 18, 3, 20, 19, 20, 21, 6, 23, 8, 63, 48, 33, 34, 51, 36, 35, 36, 37, 54, 39, 56, 47, 32, 49, 50, 35, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2579(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 4, 5, 22, 7, 24, 31, 0, 17, 18, 3, 20, 19, 20, 21, 6, 23, 8, 47, 48, 33, 34, 51, 36, 35, 36, 37, 54, 39, 56, 63, 32, 49, 50, 35, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2580(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 4, 5, 22, 7, 24, 15, 16, 1, 18, 3, 20, 19, 20, 21, 6, 23, 8, 63, 32, 49, 34, 51, 36, 35, 36, 37, 54, 39, 56, 47, 48, 33, 50, 35, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2581(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 4, 5, 22, 7, 24, 31, 16, 1, 18, 3, 20, 19, 20, 21, 6, 23, 8, 47, 32, 49, 34, 51, 36, 35, 36, 37, 54, 39, 56, 63, 48, 33, 50, 35, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2582(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 4, 5, 22, 7, 24, 15, 0, 1, 18, 3, 20, 19, 20, 21, 6, 23, 8, 63, 48, 49, 34, 51, 36, 35, 36, 37, 54, 39, 56, 47, 32, 33, 50, 35, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2583(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 4, 5, 22, 7, 24, 31, 0, 1, 18, 3, 20, 19, 20, 21, 6, 23, 8, 47, 48, 49, 34, 51, 36, 35, 36, 37, 54, 39, 56, 63, 32, 33, 50, 35, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2584(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 4, 5, 22, 7, 24, 15, 16, 17, 2, 3, 20, 19, 20, 21, 6, 23, 8, 63, 32, 33, 50, 51, 36, 35, 36, 37, 54, 39, 56, 47, 48, 49, 34, 35, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2585(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 4, 5, 22, 7, 24, 31, 16, 17, 2, 3, 20, 19, 20, 21, 6, 23, 8, 47, 32, 33, 50, 51, 36, 35, 36, 37, 54, 39, 56, 63, 48, 49, 34, 35, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2586(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 4, 5, 22, 7, 24, 15, 0, 17, 2, 3, 20, 19, 20, 21, 6, 23, 8, 63, 48, 33, 50, 51, 36, 35, 36, 37, 54, 39, 56, 47, 32, 49, 34, 35, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2587(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 4, 5, 22, 7, 24, 31, 0, 17, 2, 3, 20, 19, 20, 21, 6, 23, 8, 47, 48, 33, 50, 51, 36, 35, 36, 37, 54, 39, 56, 63, 32, 49, 34, 35, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2588(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 4, 5, 22, 7, 24, 15, 16, 1, 2, 3, 20, 19, 20, 21, 6, 23, 8, 63, 32, 49, 50, 51, 36, 35, 36, 37, 54, 39, 56, 47, 48, 33, 34, 35, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2589(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 4, 5, 22, 7, 24, 31, 16, 1, 2, 3, 20, 19, 20, 21, 6, 23, 8, 47, 32, 49, 50, 51, 36, 35, 36, 37, 54, 39, 56, 63, 48, 33, 34, 35, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2590(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 4, 5, 22, 7, 24, 15, 0, 1, 2, 3, 20, 19, 20, 21, 6, 23, 8, 63, 48, 49, 50, 51, 36, 35, 36, 37, 54, 39, 56, 47, 32, 33, 34, 35, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2591(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 4, 5, 22, 7, 24, 31, 0, 1, 2, 3, 20, 19, 20, 21, 6, 23, 8, 47, 48, 49, 50, 51, 36, 35, 36, 37, 54, 39, 56, 63, 32, 33, 34, 35, 52, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2592(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 4, 5, 22, 7, 24, 15, 16, 17, 18, 19, 4, 19, 20, 21, 6, 23, 8, 63, 32, 33, 34, 35, 52, 35, 36, 37, 54, 39, 56, 47, 48, 49, 50, 51, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2593(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 4, 5, 22, 7, 24, 31, 16, 17, 18, 19, 4, 19, 20, 21, 6, 23, 8, 47, 32, 33, 34, 35, 52, 35, 36, 37, 54, 39, 56, 63, 48, 49, 50, 51, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2594(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 4, 5, 22, 7, 24, 15, 0, 17, 18, 19, 4, 19, 20, 21, 6, 23, 8, 63, 48, 33, 34, 35, 52, 35, 36, 37, 54, 39, 56, 47, 32, 49, 50, 51, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2595(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 4, 5, 22, 7, 24, 31, 0, 17, 18, 19, 4, 19, 20, 21, 6, 23, 8, 47, 48, 33, 34, 35, 52, 35, 36, 37, 54, 39, 56, 63, 32, 49, 50, 51, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2596(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 4, 5, 22, 7, 24, 15, 16, 1, 18, 19, 4, 19, 20, 21, 6, 23, 8, 63, 32, 49, 34, 35, 52, 35, 36, 37, 54, 39, 56, 47, 48, 33, 50, 51, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2597(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 4, 5, 22, 7, 24, 31, 16, 1, 18, 19, 4, 19, 20, 21, 6, 23, 8, 47, 32, 49, 34, 35, 52, 35, 36, 37, 54, 39, 56, 63, 48, 33, 50, 51, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2598(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 4, 5, 22, 7, 24, 15, 0, 1, 18, 19, 4, 19, 20, 21, 6, 23, 8, 63, 48, 49, 34, 35, 52, 35, 36, 37, 54, 39, 56, 47, 32, 33, 50, 51, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2599(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 4, 5, 22, 7, 24, 31, 0, 1, 18, 19, 4, 19, 20, 21, 6, 23, 8, 47, 48, 49, 34, 35, 52, 35, 36, 37, 54, 39, 56, 63, 32, 33, 50, 51, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2600(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 4, 5, 22, 7, 24, 15, 16, 17, 2, 19, 4, 19, 20, 21, 6, 23, 8, 63, 32, 33, 50, 35, 52, 35, 36, 37, 54, 39, 56, 47, 48, 49, 34, 51, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2601(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 4, 5, 22, 7, 24, 31, 16, 17, 2, 19, 4, 19, 20, 21, 6, 23, 8, 47, 32, 33, 50, 35, 52, 35, 36, 37, 54, 39, 56, 63, 48, 49, 34, 51, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2602(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 4, 5, 22, 7, 24, 15, 0, 17, 2, 19, 4, 19, 20, 21, 6, 23, 8, 63, 48, 33, 50, 35, 52, 35, 36, 37, 54, 39, 56, 47, 32, 49, 34, 51, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2603(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 4, 5, 22, 7, 24, 31, 0, 17, 2, 19, 4, 19, 20, 21, 6, 23, 8, 47, 48, 33, 50, 35, 52, 35, 36, 37, 54, 39, 56, 63, 32, 49, 34, 51, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2604(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 4, 5, 22, 7, 24, 15, 16, 1, 2, 19, 4, 19, 20, 21, 6, 23, 8, 63, 32, 49, 50, 35, 52, 35, 36, 37, 54, 39, 56, 47, 48, 33, 34, 51, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2605(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 4, 5, 22, 7, 24, 31, 16, 1, 2, 19, 4, 19, 20, 21, 6, 23, 8, 47, 32, 49, 50, 35, 52, 35, 36, 37, 54, 39, 56, 63, 48, 33, 34, 51, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2606(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 4, 5, 22, 7, 24, 15, 0, 1, 2, 19, 4, 19, 20, 21, 6, 23, 8, 63, 48, 49, 50, 35, 52, 35, 36, 37, 54, 39, 56, 47, 32, 33, 34, 51, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2607(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 4, 5, 22, 7, 24, 31, 0, 1, 2, 19, 4, 19, 20, 21, 6, 23, 8, 47, 48, 49, 50, 35, 52, 35, 36, 37, 54, 39, 56, 63, 32, 33, 34, 51, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2608(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 4, 5, 22, 7, 24, 15, 16, 17, 18, 3, 4, 19, 20, 21, 6, 23, 8, 63, 32, 33, 34, 51, 52, 35, 36, 37, 54, 39, 56, 47, 48, 49, 50, 35, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2609(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 4, 5, 22, 7, 24, 31, 16, 17, 18, 3, 4, 19, 20, 21, 6, 23, 8, 47, 32, 33, 34, 51, 52, 35, 36, 37, 54, 39, 56, 63, 48, 49, 50, 35, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2610(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 4, 5, 22, 7, 24, 15, 0, 17, 18, 3, 4, 19, 20, 21, 6, 23, 8, 63, 48, 33, 34, 51, 52, 35, 36, 37, 54, 39, 56, 47, 32, 49, 50, 35, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2611(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 4, 5, 22, 7, 24, 31, 0, 17, 18, 3, 4, 19, 20, 21, 6, 23, 8, 47, 48, 33, 34, 51, 52, 35, 36, 37, 54, 39, 56, 63, 32, 49, 50, 35, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2612(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 4, 5, 22, 7, 24, 15, 16, 1, 18, 3, 4, 19, 20, 21, 6, 23, 8, 63, 32, 49, 34, 51, 52, 35, 36, 37, 54, 39, 56, 47, 48, 33, 50, 35, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2613(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 4, 5, 22, 7, 24, 31, 16, 1, 18, 3, 4, 19, 20, 21, 6, 23, 8, 47, 32, 49, 34, 51, 52, 35, 36, 37, 54, 39, 56, 63, 48, 33, 50, 35, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2614(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 4, 5, 22, 7, 24, 15, 0, 1, 18, 3, 4, 19, 20, 21, 6, 23, 8, 63, 48, 49, 34, 51, 52, 35, 36, 37, 54, 39, 56, 47, 32, 33, 50, 35, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2615(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 4, 5, 22, 7, 24, 31, 0, 1, 18, 3, 4, 19, 20, 21, 6, 23, 8, 47, 48, 49, 34, 51, 52, 35, 36, 37, 54, 39, 56, 63, 32, 33, 50, 35, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2616(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 4, 5, 22, 7, 24, 15, 16, 17, 2, 3, 4, 19, 20, 21, 6, 23, 8, 63, 32, 33, 50, 51, 52, 35, 36, 37, 54, 39, 56, 47, 48, 49, 34, 35, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2617(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 4, 5, 22, 7, 24, 31, 16, 17, 2, 3, 4, 19, 20, 21, 6, 23, 8, 47, 32, 33, 50, 51, 52, 35, 36, 37, 54, 39, 56, 63, 48, 49, 34, 35, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2618(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 4, 5, 22, 7, 24, 15, 0, 17, 2, 3, 4, 19, 20, 21, 6, 23, 8, 63, 48, 33, 50, 51, 52, 35, 36, 37, 54, 39, 56, 47, 32, 49, 34, 35, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2619(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 4, 5, 22, 7, 24, 31, 0, 17, 2, 3, 4, 19, 20, 21, 6, 23, 8, 47, 48, 33, 50, 51, 52, 35, 36, 37, 54, 39, 56, 63, 32, 49, 34, 35, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2620(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 4, 5, 22, 7, 24, 15, 16, 1, 2, 3, 4, 19, 20, 21, 6, 23, 8, 63, 32, 49, 50, 51, 52, 35, 36, 37, 54, 39, 56, 47, 48, 33, 34, 35, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2621(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 4, 5, 22, 7, 24, 31, 16, 1, 2, 3, 4, 19, 20, 21, 6, 23, 8, 47, 32, 49, 50, 51, 52, 35, 36, 37, 54, 39, 56, 63, 48, 33, 34, 35, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2622(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 4, 5, 22, 7, 24, 15, 0, 1, 2, 3, 4, 19, 20, 21, 6, 23, 8, 63, 48, 49, 50, 51, 52, 35, 36, 37, 54, 39, 56, 47, 32, 33, 34, 35, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2623(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 4, 5, 22, 7, 24, 31, 0, 1, 2, 3, 4, 19, 20, 21, 6, 23, 8, 47, 48, 49, 50, 51, 52, 35, 36, 37, 54, 39, 56, 63, 32, 33, 34, 35, 36, 51, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2624(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 4, 5, 22, 7, 24, 15, 16, 17, 18, 19, 20, 3, 20, 21, 6, 23, 8, 63, 32, 33, 34, 35, 36, 51, 36, 37, 54, 39, 56, 47, 48, 49, 50, 51, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2625(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 4, 5, 22, 7, 24, 31, 16, 17, 18, 19, 20, 3, 20, 21, 6, 23, 8, 47, 32, 33, 34, 35, 36, 51, 36, 37, 54, 39, 56, 63, 48, 49, 50, 51, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2626(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 4, 5, 22, 7, 24, 15, 0, 17, 18, 19, 20, 3, 20, 21, 6, 23, 8, 63, 48, 33, 34, 35, 36, 51, 36, 37, 54, 39, 56, 47, 32, 49, 50, 51, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2627(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 4, 5, 22, 7, 24, 31, 0, 17, 18, 19, 20, 3, 20, 21, 6, 23, 8, 47, 48, 33, 34, 35, 36, 51, 36, 37, 54, 39, 56, 63, 32, 49, 50, 51, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2628(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 4, 5, 22, 7, 24, 15, 16, 1, 18, 19, 20, 3, 20, 21, 6, 23, 8, 63, 32, 49, 34, 35, 36, 51, 36, 37, 54, 39, 56, 47, 48, 33, 50, 51, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2629(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 4, 5, 22, 7, 24, 31, 16, 1, 18, 19, 20, 3, 20, 21, 6, 23, 8, 47, 32, 49, 34, 35, 36, 51, 36, 37, 54, 39, 56, 63, 48, 33, 50, 51, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2630(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 4, 5, 22, 7, 24, 15, 0, 1, 18, 19, 20, 3, 20, 21, 6, 23, 8, 63, 48, 49, 34, 35, 36, 51, 36, 37, 54, 39, 56, 47, 32, 33, 50, 51, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2631(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 4, 5, 22, 7, 24, 31, 0, 1, 18, 19, 20, 3, 20, 21, 6, 23, 8, 47, 48, 49, 34, 35, 36, 51, 36, 37, 54, 39, 56, 63, 32, 33, 50, 51, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2632(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 4, 5, 22, 7, 24, 15, 16, 17, 2, 19, 20, 3, 20, 21, 6, 23, 8, 63, 32, 33, 50, 35, 36, 51, 36, 37, 54, 39, 56, 47, 48, 49, 34, 51, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2633(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 4, 5, 22, 7, 24, 31, 16, 17, 2, 19, 20, 3, 20, 21, 6, 23, 8, 47, 32, 33, 50, 35, 36, 51, 36, 37, 54, 39, 56, 63, 48, 49, 34, 51, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2634(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 4, 5, 22, 7, 24, 15, 0, 17, 2, 19, 20, 3, 20, 21, 6, 23, 8, 63, 48, 33, 50, 35, 36, 51, 36, 37, 54, 39, 56, 47, 32, 49, 34, 51, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2635(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 4, 5, 22, 7, 24, 31, 0, 17, 2, 19, 20, 3, 20, 21, 6, 23, 8, 47, 48, 33, 50, 35, 36, 51, 36, 37, 54, 39, 56, 63, 32, 49, 34, 51, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2636(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 4, 5, 22, 7, 24, 15, 16, 1, 2, 19, 20, 3, 20, 21, 6, 23, 8, 63, 32, 49, 50, 35, 36, 51, 36, 37, 54, 39, 56, 47, 48, 33, 34, 51, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2637(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 4, 5, 22, 7, 24, 31, 16, 1, 2, 19, 20, 3, 20, 21, 6, 23, 8, 47, 32, 49, 50, 35, 36, 51, 36, 37, 54, 39, 56, 63, 48, 33, 34, 51, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2638(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 4, 5, 22, 7, 24, 15, 0, 1, 2, 19, 20, 3, 20, 21, 6, 23, 8, 63, 48, 49, 50, 35, 36, 51, 36, 37, 54, 39, 56, 47, 32, 33, 34, 51, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2639(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 4, 5, 22, 7, 24, 31, 0, 1, 2, 19, 20, 3, 20, 21, 6, 23, 8, 47, 48, 49, 50, 35, 36, 51, 36, 37, 54, 39, 56, 63, 32, 33, 34, 51, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2640(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 4, 5, 22, 7, 24, 15, 16, 17, 18, 3, 20, 3, 20, 21, 6, 23, 8, 63, 32, 33, 34, 51, 36, 51, 36, 37, 54, 39, 56, 47, 48, 49, 50, 35, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2641(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 4, 5, 22, 7, 24, 31, 16, 17, 18, 3, 20, 3, 20, 21, 6, 23, 8, 47, 32, 33, 34, 51, 36, 51, 36, 37, 54, 39, 56, 63, 48, 49, 50, 35, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2642(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 4, 5, 22, 7, 24, 15, 0, 17, 18, 3, 20, 3, 20, 21, 6, 23, 8, 63, 48, 33, 34, 51, 36, 51, 36, 37, 54, 39, 56, 47, 32, 49, 50, 35, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2643(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 4, 5, 22, 7, 24, 31, 0, 17, 18, 3, 20, 3, 20, 21, 6, 23, 8, 47, 48, 33, 34, 51, 36, 51, 36, 37, 54, 39, 56, 63, 32, 49, 50, 35, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2644(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 4, 5, 22, 7, 24, 15, 16, 1, 18, 3, 20, 3, 20, 21, 6, 23, 8, 63, 32, 49, 34, 51, 36, 51, 36, 37, 54, 39, 56, 47, 48, 33, 50, 35, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2645(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 4, 5, 22, 7, 24, 31, 16, 1, 18, 3, 20, 3, 20, 21, 6, 23, 8, 47, 32, 49, 34, 51, 36, 51, 36, 37, 54, 39, 56, 63, 48, 33, 50, 35, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2646(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 4, 5, 22, 7, 24, 15, 0, 1, 18, 3, 20, 3, 20, 21, 6, 23, 8, 63, 48, 49, 34, 51, 36, 51, 36, 37, 54, 39, 56, 47, 32, 33, 50, 35, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2647(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 4, 5, 22, 7, 24, 31, 0, 1, 18, 3, 20, 3, 20, 21, 6, 23, 8, 47, 48, 49, 34, 51, 36, 51, 36, 37, 54, 39, 56, 63, 32, 33, 50, 35, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2648(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 4, 5, 22, 7, 24, 15, 16, 17, 2, 3, 20, 3, 20, 21, 6, 23, 8, 63, 32, 33, 50, 51, 36, 51, 36, 37, 54, 39, 56, 47, 48, 49, 34, 35, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2649(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 4, 5, 22, 7, 24, 31, 16, 17, 2, 3, 20, 3, 20, 21, 6, 23, 8, 47, 32, 33, 50, 51, 36, 51, 36, 37, 54, 39, 56, 63, 48, 49, 34, 35, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2650(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 4, 5, 22, 7, 24, 15, 0, 17, 2, 3, 20, 3, 20, 21, 6, 23, 8, 63, 48, 33, 50, 51, 36, 51, 36, 37, 54, 39, 56, 47, 32, 49, 34, 35, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2651(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 4, 5, 22, 7, 24, 31, 0, 17, 2, 3, 20, 3, 20, 21, 6, 23, 8, 47, 48, 33, 50, 51, 36, 51, 36, 37, 54, 39, 56, 63, 32, 49, 34, 35, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2652(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 4, 5, 22, 7, 24, 15, 16, 1, 2, 3, 20, 3, 20, 21, 6, 23, 8, 63, 32, 49, 50, 51, 36, 51, 36, 37, 54, 39, 56, 47, 48, 33, 34, 35, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2653(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 4, 5, 22, 7, 24, 31, 16, 1, 2, 3, 20, 3, 20, 21, 6, 23, 8, 47, 32, 49, 50, 51, 36, 51, 36, 37, 54, 39, 56, 63, 48, 33, 34, 35, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2654(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 4, 5, 22, 7, 24, 15, 0, 1, 2, 3, 20, 3, 20, 21, 6, 23, 8, 63, 48, 49, 50, 51, 36, 51, 36, 37, 54, 39, 56, 47, 32, 33, 34, 35, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2655(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 4, 5, 22, 7, 24, 31, 0, 1, 2, 3, 20, 3, 20, 21, 6, 23, 8, 47, 48, 49, 50, 51, 36, 51, 36, 37, 54, 39, 56, 63, 32, 33, 34, 35, 52, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2656(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 4, 5, 22, 7, 24, 15, 16, 17, 18, 19, 4, 3, 20, 21, 6, 23, 8, 63, 32, 33, 34, 35, 52, 51, 36, 37, 54, 39, 56, 47, 48, 49, 50, 51, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2657(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 4, 5, 22, 7, 24, 31, 16, 17, 18, 19, 4, 3, 20, 21, 6, 23, 8, 47, 32, 33, 34, 35, 52, 51, 36, 37, 54, 39, 56, 63, 48, 49, 50, 51, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2658(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 4, 5, 22, 7, 24, 15, 0, 17, 18, 19, 4, 3, 20, 21, 6, 23, 8, 63, 48, 33, 34, 35, 52, 51, 36, 37, 54, 39, 56, 47, 32, 49, 50, 51, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2659(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 4, 5, 22, 7, 24, 31, 0, 17, 18, 19, 4, 3, 20, 21, 6, 23, 8, 47, 48, 33, 34, 35, 52, 51, 36, 37, 54, 39, 56, 63, 32, 49, 50, 51, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2660(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 4, 5, 22, 7, 24, 15, 16, 1, 18, 19, 4, 3, 20, 21, 6, 23, 8, 63, 32, 49, 34, 35, 52, 51, 36, 37, 54, 39, 56, 47, 48, 33, 50, 51, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2661(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 4, 5, 22, 7, 24, 31, 16, 1, 18, 19, 4, 3, 20, 21, 6, 23, 8, 47, 32, 49, 34, 35, 52, 51, 36, 37, 54, 39, 56, 63, 48, 33, 50, 51, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2662(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 4, 5, 22, 7, 24, 15, 0, 1, 18, 19, 4, 3, 20, 21, 6, 23, 8, 63, 48, 49, 34, 35, 52, 51, 36, 37, 54, 39, 56, 47, 32, 33, 50, 51, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2663(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 4, 5, 22, 7, 24, 31, 0, 1, 18, 19, 4, 3, 20, 21, 6, 23, 8, 47, 48, 49, 34, 35, 52, 51, 36, 37, 54, 39, 56, 63, 32, 33, 50, 51, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2664(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 4, 5, 22, 7, 24, 15, 16, 17, 2, 19, 4, 3, 20, 21, 6, 23, 8, 63, 32, 33, 50, 35, 52, 51, 36, 37, 54, 39, 56, 47, 48, 49, 34, 51, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2665(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 4, 5, 22, 7, 24, 31, 16, 17, 2, 19, 4, 3, 20, 21, 6, 23, 8, 47, 32, 33, 50, 35, 52, 51, 36, 37, 54, 39, 56, 63, 48, 49, 34, 51, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2666(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 4, 5, 22, 7, 24, 15, 0, 17, 2, 19, 4, 3, 20, 21, 6, 23, 8, 63, 48, 33, 50, 35, 52, 51, 36, 37, 54, 39, 56, 47, 32, 49, 34, 51, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2667(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 4, 5, 22, 7, 24, 31, 0, 17, 2, 19, 4, 3, 20, 21, 6, 23, 8, 47, 48, 33, 50, 35, 52, 51, 36, 37, 54, 39, 56, 63, 32, 49, 34, 51, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2668(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 4, 5, 22, 7, 24, 15, 16, 1, 2, 19, 4, 3, 20, 21, 6, 23, 8, 63, 32, 49, 50, 35, 52, 51, 36, 37, 54, 39, 56, 47, 48, 33, 34, 51, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2669(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 4, 5, 22, 7, 24, 31, 16, 1, 2, 19, 4, 3, 20, 21, 6, 23, 8, 47, 32, 49, 50, 35, 52, 51, 36, 37, 54, 39, 56, 63, 48, 33, 34, 51, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2670(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 4, 5, 22, 7, 24, 15, 0, 1, 2, 19, 4, 3, 20, 21, 6, 23, 8, 63, 48, 49, 50, 35, 52, 51, 36, 37, 54, 39, 56, 47, 32, 33, 34, 51, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2671(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 4, 5, 22, 7, 24, 31, 0, 1, 2, 19, 4, 3, 20, 21, 6, 23, 8, 47, 48, 49, 50, 35, 52, 51, 36, 37, 54, 39, 56, 63, 32, 33, 34, 51, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2672(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 4, 5, 22, 7, 24, 15, 16, 17, 18, 3, 4, 3, 20, 21, 6, 23, 8, 63, 32, 33, 34, 51, 52, 51, 36, 37, 54, 39, 56, 47, 48, 49, 50, 35, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2673(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 4, 5, 22, 7, 24, 31, 16, 17, 18, 3, 4, 3, 20, 21, 6, 23, 8, 47, 32, 33, 34, 51, 52, 51, 36, 37, 54, 39, 56, 63, 48, 49, 50, 35, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2674(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 4, 5, 22, 7, 24, 15, 0, 17, 18, 3, 4, 3, 20, 21, 6, 23, 8, 63, 48, 33, 34, 51, 52, 51, 36, 37, 54, 39, 56, 47, 32, 49, 50, 35, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2675(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 4, 5, 22, 7, 24, 31, 0, 17, 18, 3, 4, 3, 20, 21, 6, 23, 8, 47, 48, 33, 34, 51, 52, 51, 36, 37, 54, 39, 56, 63, 32, 49, 50, 35, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2676(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 4, 5, 22, 7, 24, 15, 16, 1, 18, 3, 4, 3, 20, 21, 6, 23, 8, 63, 32, 49, 34, 51, 52, 51, 36, 37, 54, 39, 56, 47, 48, 33, 50, 35, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2677(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 4, 5, 22, 7, 24, 31, 16, 1, 18, 3, 4, 3, 20, 21, 6, 23, 8, 47, 32, 49, 34, 51, 52, 51, 36, 37, 54, 39, 56, 63, 48, 33, 50, 35, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2678(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 4, 5, 22, 7, 24, 15, 0, 1, 18, 3, 4, 3, 20, 21, 6, 23, 8, 63, 48, 49, 34, 51, 52, 51, 36, 37, 54, 39, 56, 47, 32, 33, 50, 35, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2679(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 4, 5, 22, 7, 24, 31, 0, 1, 18, 3, 4, 3, 20, 21, 6, 23, 8, 47, 48, 49, 34, 51, 52, 51, 36, 37, 54, 39, 56, 63, 32, 33, 50, 35, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2680(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 4, 5, 22, 7, 24, 15, 16, 17, 2, 3, 4, 3, 20, 21, 6, 23, 8, 63, 32, 33, 50, 51, 52, 51, 36, 37, 54, 39, 56, 47, 48, 49, 34, 35, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2681(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 4, 5, 22, 7, 24, 31, 16, 17, 2, 3, 4, 3, 20, 21, 6, 23, 8, 47, 32, 33, 50, 51, 52, 51, 36, 37, 54, 39, 56, 63, 48, 49, 34, 35, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2682(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 4, 5, 22, 7, 24, 15, 0, 17, 2, 3, 4, 3, 20, 21, 6, 23, 8, 63, 48, 33, 50, 51, 52, 51, 36, 37, 54, 39, 56, 47, 32, 49, 34, 35, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2683(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 4, 5, 22, 7, 24, 31, 0, 17, 2, 3, 4, 3, 20, 21, 6, 23, 8, 47, 48, 33, 50, 51, 52, 51, 36, 37, 54, 39, 56, 63, 32, 49, 34, 35, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2684(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 4, 5, 22, 7, 24, 15, 16, 1, 2, 3, 4, 3, 20, 21, 6, 23, 8, 63, 32, 49, 50, 51, 52, 51, 36, 37, 54, 39, 56, 47, 48, 33, 34, 35, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2685(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 4, 5, 22, 7, 24, 31, 16, 1, 2, 3, 4, 3, 20, 21, 6, 23, 8, 47, 32, 49, 50, 51, 52, 51, 36, 37, 54, 39, 56, 63, 48, 33, 34, 35, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2686(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 4, 5, 22, 7, 24, 15, 0, 1, 2, 3, 4, 3, 20, 21, 6, 23, 8, 63, 48, 49, 50, 51, 52, 51, 36, 37, 54, 39, 56, 47, 32, 33, 34, 35, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt2687(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 4, 5, 22, 7, 24, 31, 0, 1, 2, 3, 4, 3, 20, 21, 6, 23, 8, 47, 48, 49, 50, 51, 52, 51, 36, 37, 54, 39, 56, 63, 32, 33, 34, 35, 36, 35, 52, 53, 38, 55, 40>(unchecked_hashes, bitsplitted_keys);
}


