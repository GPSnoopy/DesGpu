#include "hip/hip_runtime.h"
#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

__global__ void des_25_encrypt_salt1664(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 5, 22, 23, 8, 15, 16, 17, 18, 19, 20, 19, 4, 21, 6, 7, 24, 63, 32, 33, 34, 35, 36, 35, 52, 37, 54, 55, 40, 47, 48, 49, 50, 51, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1665(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 5, 22, 23, 8, 31, 16, 17, 18, 19, 20, 19, 4, 21, 6, 7, 24, 47, 32, 33, 34, 35, 36, 35, 52, 37, 54, 55, 40, 63, 48, 49, 50, 51, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1666(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 5, 22, 23, 8, 15, 0, 17, 18, 19, 20, 19, 4, 21, 6, 7, 24, 63, 48, 33, 34, 35, 36, 35, 52, 37, 54, 55, 40, 47, 32, 49, 50, 51, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1667(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 5, 22, 23, 8, 31, 0, 17, 18, 19, 20, 19, 4, 21, 6, 7, 24, 47, 48, 33, 34, 35, 36, 35, 52, 37, 54, 55, 40, 63, 32, 49, 50, 51, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1668(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 5, 22, 23, 8, 15, 16, 1, 18, 19, 20, 19, 4, 21, 6, 7, 24, 63, 32, 49, 34, 35, 36, 35, 52, 37, 54, 55, 40, 47, 48, 33, 50, 51, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1669(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 5, 22, 23, 8, 31, 16, 1, 18, 19, 20, 19, 4, 21, 6, 7, 24, 47, 32, 49, 34, 35, 36, 35, 52, 37, 54, 55, 40, 63, 48, 33, 50, 51, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1670(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 5, 22, 23, 8, 15, 0, 1, 18, 19, 20, 19, 4, 21, 6, 7, 24, 63, 48, 49, 34, 35, 36, 35, 52, 37, 54, 55, 40, 47, 32, 33, 50, 51, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1671(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 5, 22, 23, 8, 31, 0, 1, 18, 19, 20, 19, 4, 21, 6, 7, 24, 47, 48, 49, 34, 35, 36, 35, 52, 37, 54, 55, 40, 63, 32, 33, 50, 51, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1672(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 5, 22, 23, 8, 15, 16, 17, 2, 19, 20, 19, 4, 21, 6, 7, 24, 63, 32, 33, 50, 35, 36, 35, 52, 37, 54, 55, 40, 47, 48, 49, 34, 51, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1673(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 5, 22, 23, 8, 31, 16, 17, 2, 19, 20, 19, 4, 21, 6, 7, 24, 47, 32, 33, 50, 35, 36, 35, 52, 37, 54, 55, 40, 63, 48, 49, 34, 51, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1674(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 5, 22, 23, 8, 15, 0, 17, 2, 19, 20, 19, 4, 21, 6, 7, 24, 63, 48, 33, 50, 35, 36, 35, 52, 37, 54, 55, 40, 47, 32, 49, 34, 51, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1675(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 5, 22, 23, 8, 31, 0, 17, 2, 19, 20, 19, 4, 21, 6, 7, 24, 47, 48, 33, 50, 35, 36, 35, 52, 37, 54, 55, 40, 63, 32, 49, 34, 51, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1676(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 5, 22, 23, 8, 15, 16, 1, 2, 19, 20, 19, 4, 21, 6, 7, 24, 63, 32, 49, 50, 35, 36, 35, 52, 37, 54, 55, 40, 47, 48, 33, 34, 51, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1677(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 5, 22, 23, 8, 31, 16, 1, 2, 19, 20, 19, 4, 21, 6, 7, 24, 47, 32, 49, 50, 35, 36, 35, 52, 37, 54, 55, 40, 63, 48, 33, 34, 51, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1678(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 5, 22, 23, 8, 15, 0, 1, 2, 19, 20, 19, 4, 21, 6, 7, 24, 63, 48, 49, 50, 35, 36, 35, 52, 37, 54, 55, 40, 47, 32, 33, 34, 51, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1679(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 5, 22, 23, 8, 31, 0, 1, 2, 19, 20, 19, 4, 21, 6, 7, 24, 47, 48, 49, 50, 35, 36, 35, 52, 37, 54, 55, 40, 63, 32, 33, 34, 51, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1680(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 5, 22, 23, 8, 15, 16, 17, 18, 3, 20, 19, 4, 21, 6, 7, 24, 63, 32, 33, 34, 51, 36, 35, 52, 37, 54, 55, 40, 47, 48, 49, 50, 35, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1681(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 5, 22, 23, 8, 31, 16, 17, 18, 3, 20, 19, 4, 21, 6, 7, 24, 47, 32, 33, 34, 51, 36, 35, 52, 37, 54, 55, 40, 63, 48, 49, 50, 35, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1682(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 5, 22, 23, 8, 15, 0, 17, 18, 3, 20, 19, 4, 21, 6, 7, 24, 63, 48, 33, 34, 51, 36, 35, 52, 37, 54, 55, 40, 47, 32, 49, 50, 35, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1683(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 5, 22, 23, 8, 31, 0, 17, 18, 3, 20, 19, 4, 21, 6, 7, 24, 47, 48, 33, 34, 51, 36, 35, 52, 37, 54, 55, 40, 63, 32, 49, 50, 35, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1684(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 5, 22, 23, 8, 15, 16, 1, 18, 3, 20, 19, 4, 21, 6, 7, 24, 63, 32, 49, 34, 51, 36, 35, 52, 37, 54, 55, 40, 47, 48, 33, 50, 35, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1685(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 5, 22, 23, 8, 31, 16, 1, 18, 3, 20, 19, 4, 21, 6, 7, 24, 47, 32, 49, 34, 51, 36, 35, 52, 37, 54, 55, 40, 63, 48, 33, 50, 35, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1686(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 5, 22, 23, 8, 15, 0, 1, 18, 3, 20, 19, 4, 21, 6, 7, 24, 63, 48, 49, 34, 51, 36, 35, 52, 37, 54, 55, 40, 47, 32, 33, 50, 35, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1687(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 5, 22, 23, 8, 31, 0, 1, 18, 3, 20, 19, 4, 21, 6, 7, 24, 47, 48, 49, 34, 51, 36, 35, 52, 37, 54, 55, 40, 63, 32, 33, 50, 35, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1688(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 5, 22, 23, 8, 15, 16, 17, 2, 3, 20, 19, 4, 21, 6, 7, 24, 63, 32, 33, 50, 51, 36, 35, 52, 37, 54, 55, 40, 47, 48, 49, 34, 35, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1689(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 5, 22, 23, 8, 31, 16, 17, 2, 3, 20, 19, 4, 21, 6, 7, 24, 47, 32, 33, 50, 51, 36, 35, 52, 37, 54, 55, 40, 63, 48, 49, 34, 35, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1690(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 5, 22, 23, 8, 15, 0, 17, 2, 3, 20, 19, 4, 21, 6, 7, 24, 63, 48, 33, 50, 51, 36, 35, 52, 37, 54, 55, 40, 47, 32, 49, 34, 35, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1691(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 5, 22, 23, 8, 31, 0, 17, 2, 3, 20, 19, 4, 21, 6, 7, 24, 47, 48, 33, 50, 51, 36, 35, 52, 37, 54, 55, 40, 63, 32, 49, 34, 35, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1692(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 5, 22, 23, 8, 15, 16, 1, 2, 3, 20, 19, 4, 21, 6, 7, 24, 63, 32, 49, 50, 51, 36, 35, 52, 37, 54, 55, 40, 47, 48, 33, 34, 35, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1693(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 5, 22, 23, 8, 31, 16, 1, 2, 3, 20, 19, 4, 21, 6, 7, 24, 47, 32, 49, 50, 51, 36, 35, 52, 37, 54, 55, 40, 63, 48, 33, 34, 35, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1694(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 5, 22, 23, 8, 15, 0, 1, 2, 3, 20, 19, 4, 21, 6, 7, 24, 63, 48, 49, 50, 51, 36, 35, 52, 37, 54, 55, 40, 47, 32, 33, 34, 35, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1695(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 5, 22, 23, 8, 31, 0, 1, 2, 3, 20, 19, 4, 21, 6, 7, 24, 47, 48, 49, 50, 51, 36, 35, 52, 37, 54, 55, 40, 63, 32, 33, 34, 35, 52, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1696(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 5, 22, 23, 8, 15, 16, 17, 18, 19, 4, 19, 4, 21, 6, 7, 24, 63, 32, 33, 34, 35, 52, 35, 52, 37, 54, 55, 40, 47, 48, 49, 50, 51, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1697(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 5, 22, 23, 8, 31, 16, 17, 18, 19, 4, 19, 4, 21, 6, 7, 24, 47, 32, 33, 34, 35, 52, 35, 52, 37, 54, 55, 40, 63, 48, 49, 50, 51, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1698(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 5, 22, 23, 8, 15, 0, 17, 18, 19, 4, 19, 4, 21, 6, 7, 24, 63, 48, 33, 34, 35, 52, 35, 52, 37, 54, 55, 40, 47, 32, 49, 50, 51, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1699(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 5, 22, 23, 8, 31, 0, 17, 18, 19, 4, 19, 4, 21, 6, 7, 24, 47, 48, 33, 34, 35, 52, 35, 52, 37, 54, 55, 40, 63, 32, 49, 50, 51, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1700(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 5, 22, 23, 8, 15, 16, 1, 18, 19, 4, 19, 4, 21, 6, 7, 24, 63, 32, 49, 34, 35, 52, 35, 52, 37, 54, 55, 40, 47, 48, 33, 50, 51, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1701(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 5, 22, 23, 8, 31, 16, 1, 18, 19, 4, 19, 4, 21, 6, 7, 24, 47, 32, 49, 34, 35, 52, 35, 52, 37, 54, 55, 40, 63, 48, 33, 50, 51, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1702(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 5, 22, 23, 8, 15, 0, 1, 18, 19, 4, 19, 4, 21, 6, 7, 24, 63, 48, 49, 34, 35, 52, 35, 52, 37, 54, 55, 40, 47, 32, 33, 50, 51, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1703(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 5, 22, 23, 8, 31, 0, 1, 18, 19, 4, 19, 4, 21, 6, 7, 24, 47, 48, 49, 34, 35, 52, 35, 52, 37, 54, 55, 40, 63, 32, 33, 50, 51, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1704(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 5, 22, 23, 8, 15, 16, 17, 2, 19, 4, 19, 4, 21, 6, 7, 24, 63, 32, 33, 50, 35, 52, 35, 52, 37, 54, 55, 40, 47, 48, 49, 34, 51, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1705(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 5, 22, 23, 8, 31, 16, 17, 2, 19, 4, 19, 4, 21, 6, 7, 24, 47, 32, 33, 50, 35, 52, 35, 52, 37, 54, 55, 40, 63, 48, 49, 34, 51, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1706(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 5, 22, 23, 8, 15, 0, 17, 2, 19, 4, 19, 4, 21, 6, 7, 24, 63, 48, 33, 50, 35, 52, 35, 52, 37, 54, 55, 40, 47, 32, 49, 34, 51, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1707(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 5, 22, 23, 8, 31, 0, 17, 2, 19, 4, 19, 4, 21, 6, 7, 24, 47, 48, 33, 50, 35, 52, 35, 52, 37, 54, 55, 40, 63, 32, 49, 34, 51, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1708(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 5, 22, 23, 8, 15, 16, 1, 2, 19, 4, 19, 4, 21, 6, 7, 24, 63, 32, 49, 50, 35, 52, 35, 52, 37, 54, 55, 40, 47, 48, 33, 34, 51, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1709(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 5, 22, 23, 8, 31, 16, 1, 2, 19, 4, 19, 4, 21, 6, 7, 24, 47, 32, 49, 50, 35, 52, 35, 52, 37, 54, 55, 40, 63, 48, 33, 34, 51, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1710(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 5, 22, 23, 8, 15, 0, 1, 2, 19, 4, 19, 4, 21, 6, 7, 24, 63, 48, 49, 50, 35, 52, 35, 52, 37, 54, 55, 40, 47, 32, 33, 34, 51, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1711(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 5, 22, 23, 8, 31, 0, 1, 2, 19, 4, 19, 4, 21, 6, 7, 24, 47, 48, 49, 50, 35, 52, 35, 52, 37, 54, 55, 40, 63, 32, 33, 34, 51, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1712(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 5, 22, 23, 8, 15, 16, 17, 18, 3, 4, 19, 4, 21, 6, 7, 24, 63, 32, 33, 34, 51, 52, 35, 52, 37, 54, 55, 40, 47, 48, 49, 50, 35, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1713(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 5, 22, 23, 8, 31, 16, 17, 18, 3, 4, 19, 4, 21, 6, 7, 24, 47, 32, 33, 34, 51, 52, 35, 52, 37, 54, 55, 40, 63, 48, 49, 50, 35, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1714(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 5, 22, 23, 8, 15, 0, 17, 18, 3, 4, 19, 4, 21, 6, 7, 24, 63, 48, 33, 34, 51, 52, 35, 52, 37, 54, 55, 40, 47, 32, 49, 50, 35, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1715(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 5, 22, 23, 8, 31, 0, 17, 18, 3, 4, 19, 4, 21, 6, 7, 24, 47, 48, 33, 34, 51, 52, 35, 52, 37, 54, 55, 40, 63, 32, 49, 50, 35, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1716(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 5, 22, 23, 8, 15, 16, 1, 18, 3, 4, 19, 4, 21, 6, 7, 24, 63, 32, 49, 34, 51, 52, 35, 52, 37, 54, 55, 40, 47, 48, 33, 50, 35, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1717(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 5, 22, 23, 8, 31, 16, 1, 18, 3, 4, 19, 4, 21, 6, 7, 24, 47, 32, 49, 34, 51, 52, 35, 52, 37, 54, 55, 40, 63, 48, 33, 50, 35, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1718(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 5, 22, 23, 8, 15, 0, 1, 18, 3, 4, 19, 4, 21, 6, 7, 24, 63, 48, 49, 34, 51, 52, 35, 52, 37, 54, 55, 40, 47, 32, 33, 50, 35, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1719(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 5, 22, 23, 8, 31, 0, 1, 18, 3, 4, 19, 4, 21, 6, 7, 24, 47, 48, 49, 34, 51, 52, 35, 52, 37, 54, 55, 40, 63, 32, 33, 50, 35, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1720(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 5, 22, 23, 8, 15, 16, 17, 2, 3, 4, 19, 4, 21, 6, 7, 24, 63, 32, 33, 50, 51, 52, 35, 52, 37, 54, 55, 40, 47, 48, 49, 34, 35, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1721(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 5, 22, 23, 8, 31, 16, 17, 2, 3, 4, 19, 4, 21, 6, 7, 24, 47, 32, 33, 50, 51, 52, 35, 52, 37, 54, 55, 40, 63, 48, 49, 34, 35, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1722(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 5, 22, 23, 8, 15, 0, 17, 2, 3, 4, 19, 4, 21, 6, 7, 24, 63, 48, 33, 50, 51, 52, 35, 52, 37, 54, 55, 40, 47, 32, 49, 34, 35, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1723(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 5, 22, 23, 8, 31, 0, 17, 2, 3, 4, 19, 4, 21, 6, 7, 24, 47, 48, 33, 50, 51, 52, 35, 52, 37, 54, 55, 40, 63, 32, 49, 34, 35, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1724(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 5, 22, 23, 8, 15, 16, 1, 2, 3, 4, 19, 4, 21, 6, 7, 24, 63, 32, 49, 50, 51, 52, 35, 52, 37, 54, 55, 40, 47, 48, 33, 34, 35, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1725(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 5, 22, 23, 8, 31, 16, 1, 2, 3, 4, 19, 4, 21, 6, 7, 24, 47, 32, 49, 50, 51, 52, 35, 52, 37, 54, 55, 40, 63, 48, 33, 34, 35, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1726(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 5, 22, 23, 8, 15, 0, 1, 2, 3, 4, 19, 4, 21, 6, 7, 24, 63, 48, 49, 50, 51, 52, 35, 52, 37, 54, 55, 40, 47, 32, 33, 34, 35, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1727(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 5, 22, 23, 8, 31, 0, 1, 2, 3, 4, 19, 4, 21, 6, 7, 24, 47, 48, 49, 50, 51, 52, 35, 52, 37, 54, 55, 40, 63, 32, 33, 34, 35, 36, 51, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1728(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 5, 22, 23, 8, 15, 16, 17, 18, 19, 20, 3, 4, 21, 6, 7, 24, 63, 32, 33, 34, 35, 36, 51, 52, 37, 54, 55, 40, 47, 48, 49, 50, 51, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1729(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 5, 22, 23, 8, 31, 16, 17, 18, 19, 20, 3, 4, 21, 6, 7, 24, 47, 32, 33, 34, 35, 36, 51, 52, 37, 54, 55, 40, 63, 48, 49, 50, 51, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1730(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 5, 22, 23, 8, 15, 0, 17, 18, 19, 20, 3, 4, 21, 6, 7, 24, 63, 48, 33, 34, 35, 36, 51, 52, 37, 54, 55, 40, 47, 32, 49, 50, 51, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1731(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 5, 22, 23, 8, 31, 0, 17, 18, 19, 20, 3, 4, 21, 6, 7, 24, 47, 48, 33, 34, 35, 36, 51, 52, 37, 54, 55, 40, 63, 32, 49, 50, 51, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1732(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 5, 22, 23, 8, 15, 16, 1, 18, 19, 20, 3, 4, 21, 6, 7, 24, 63, 32, 49, 34, 35, 36, 51, 52, 37, 54, 55, 40, 47, 48, 33, 50, 51, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1733(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 5, 22, 23, 8, 31, 16, 1, 18, 19, 20, 3, 4, 21, 6, 7, 24, 47, 32, 49, 34, 35, 36, 51, 52, 37, 54, 55, 40, 63, 48, 33, 50, 51, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1734(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 5, 22, 23, 8, 15, 0, 1, 18, 19, 20, 3, 4, 21, 6, 7, 24, 63, 48, 49, 34, 35, 36, 51, 52, 37, 54, 55, 40, 47, 32, 33, 50, 51, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1735(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 5, 22, 23, 8, 31, 0, 1, 18, 19, 20, 3, 4, 21, 6, 7, 24, 47, 48, 49, 34, 35, 36, 51, 52, 37, 54, 55, 40, 63, 32, 33, 50, 51, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1736(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 5, 22, 23, 8, 15, 16, 17, 2, 19, 20, 3, 4, 21, 6, 7, 24, 63, 32, 33, 50, 35, 36, 51, 52, 37, 54, 55, 40, 47, 48, 49, 34, 51, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1737(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 5, 22, 23, 8, 31, 16, 17, 2, 19, 20, 3, 4, 21, 6, 7, 24, 47, 32, 33, 50, 35, 36, 51, 52, 37, 54, 55, 40, 63, 48, 49, 34, 51, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1738(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 5, 22, 23, 8, 15, 0, 17, 2, 19, 20, 3, 4, 21, 6, 7, 24, 63, 48, 33, 50, 35, 36, 51, 52, 37, 54, 55, 40, 47, 32, 49, 34, 51, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1739(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 5, 22, 23, 8, 31, 0, 17, 2, 19, 20, 3, 4, 21, 6, 7, 24, 47, 48, 33, 50, 35, 36, 51, 52, 37, 54, 55, 40, 63, 32, 49, 34, 51, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1740(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 5, 22, 23, 8, 15, 16, 1, 2, 19, 20, 3, 4, 21, 6, 7, 24, 63, 32, 49, 50, 35, 36, 51, 52, 37, 54, 55, 40, 47, 48, 33, 34, 51, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1741(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 5, 22, 23, 8, 31, 16, 1, 2, 19, 20, 3, 4, 21, 6, 7, 24, 47, 32, 49, 50, 35, 36, 51, 52, 37, 54, 55, 40, 63, 48, 33, 34, 51, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1742(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 5, 22, 23, 8, 15, 0, 1, 2, 19, 20, 3, 4, 21, 6, 7, 24, 63, 48, 49, 50, 35, 36, 51, 52, 37, 54, 55, 40, 47, 32, 33, 34, 51, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1743(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 5, 22, 23, 8, 31, 0, 1, 2, 19, 20, 3, 4, 21, 6, 7, 24, 47, 48, 49, 50, 35, 36, 51, 52, 37, 54, 55, 40, 63, 32, 33, 34, 51, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1744(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 5, 22, 23, 8, 15, 16, 17, 18, 3, 20, 3, 4, 21, 6, 7, 24, 63, 32, 33, 34, 51, 36, 51, 52, 37, 54, 55, 40, 47, 48, 49, 50, 35, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1745(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 5, 22, 23, 8, 31, 16, 17, 18, 3, 20, 3, 4, 21, 6, 7, 24, 47, 32, 33, 34, 51, 36, 51, 52, 37, 54, 55, 40, 63, 48, 49, 50, 35, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1746(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 5, 22, 23, 8, 15, 0, 17, 18, 3, 20, 3, 4, 21, 6, 7, 24, 63, 48, 33, 34, 51, 36, 51, 52, 37, 54, 55, 40, 47, 32, 49, 50, 35, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1747(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 5, 22, 23, 8, 31, 0, 17, 18, 3, 20, 3, 4, 21, 6, 7, 24, 47, 48, 33, 34, 51, 36, 51, 52, 37, 54, 55, 40, 63, 32, 49, 50, 35, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1748(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 5, 22, 23, 8, 15, 16, 1, 18, 3, 20, 3, 4, 21, 6, 7, 24, 63, 32, 49, 34, 51, 36, 51, 52, 37, 54, 55, 40, 47, 48, 33, 50, 35, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1749(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 5, 22, 23, 8, 31, 16, 1, 18, 3, 20, 3, 4, 21, 6, 7, 24, 47, 32, 49, 34, 51, 36, 51, 52, 37, 54, 55, 40, 63, 48, 33, 50, 35, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1750(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 5, 22, 23, 8, 15, 0, 1, 18, 3, 20, 3, 4, 21, 6, 7, 24, 63, 48, 49, 34, 51, 36, 51, 52, 37, 54, 55, 40, 47, 32, 33, 50, 35, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1751(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 5, 22, 23, 8, 31, 0, 1, 18, 3, 20, 3, 4, 21, 6, 7, 24, 47, 48, 49, 34, 51, 36, 51, 52, 37, 54, 55, 40, 63, 32, 33, 50, 35, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1752(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 5, 22, 23, 8, 15, 16, 17, 2, 3, 20, 3, 4, 21, 6, 7, 24, 63, 32, 33, 50, 51, 36, 51, 52, 37, 54, 55, 40, 47, 48, 49, 34, 35, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1753(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 5, 22, 23, 8, 31, 16, 17, 2, 3, 20, 3, 4, 21, 6, 7, 24, 47, 32, 33, 50, 51, 36, 51, 52, 37, 54, 55, 40, 63, 48, 49, 34, 35, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1754(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 5, 22, 23, 8, 15, 0, 17, 2, 3, 20, 3, 4, 21, 6, 7, 24, 63, 48, 33, 50, 51, 36, 51, 52, 37, 54, 55, 40, 47, 32, 49, 34, 35, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1755(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 5, 22, 23, 8, 31, 0, 17, 2, 3, 20, 3, 4, 21, 6, 7, 24, 47, 48, 33, 50, 51, 36, 51, 52, 37, 54, 55, 40, 63, 32, 49, 34, 35, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1756(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 5, 22, 23, 8, 15, 16, 1, 2, 3, 20, 3, 4, 21, 6, 7, 24, 63, 32, 49, 50, 51, 36, 51, 52, 37, 54, 55, 40, 47, 48, 33, 34, 35, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1757(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 5, 22, 23, 8, 31, 16, 1, 2, 3, 20, 3, 4, 21, 6, 7, 24, 47, 32, 49, 50, 51, 36, 51, 52, 37, 54, 55, 40, 63, 48, 33, 34, 35, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1758(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 5, 22, 23, 8, 15, 0, 1, 2, 3, 20, 3, 4, 21, 6, 7, 24, 63, 48, 49, 50, 51, 36, 51, 52, 37, 54, 55, 40, 47, 32, 33, 34, 35, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1759(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 5, 22, 23, 8, 31, 0, 1, 2, 3, 20, 3, 4, 21, 6, 7, 24, 47, 48, 49, 50, 51, 36, 51, 52, 37, 54, 55, 40, 63, 32, 33, 34, 35, 52, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1760(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 5, 22, 23, 8, 15, 16, 17, 18, 19, 4, 3, 4, 21, 6, 7, 24, 63, 32, 33, 34, 35, 52, 51, 52, 37, 54, 55, 40, 47, 48, 49, 50, 51, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1761(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 5, 22, 23, 8, 31, 16, 17, 18, 19, 4, 3, 4, 21, 6, 7, 24, 47, 32, 33, 34, 35, 52, 51, 52, 37, 54, 55, 40, 63, 48, 49, 50, 51, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1762(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 5, 22, 23, 8, 15, 0, 17, 18, 19, 4, 3, 4, 21, 6, 7, 24, 63, 48, 33, 34, 35, 52, 51, 52, 37, 54, 55, 40, 47, 32, 49, 50, 51, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1763(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 5, 22, 23, 8, 31, 0, 17, 18, 19, 4, 3, 4, 21, 6, 7, 24, 47, 48, 33, 34, 35, 52, 51, 52, 37, 54, 55, 40, 63, 32, 49, 50, 51, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1764(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 5, 22, 23, 8, 15, 16, 1, 18, 19, 4, 3, 4, 21, 6, 7, 24, 63, 32, 49, 34, 35, 52, 51, 52, 37, 54, 55, 40, 47, 48, 33, 50, 51, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1765(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 5, 22, 23, 8, 31, 16, 1, 18, 19, 4, 3, 4, 21, 6, 7, 24, 47, 32, 49, 34, 35, 52, 51, 52, 37, 54, 55, 40, 63, 48, 33, 50, 51, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1766(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 5, 22, 23, 8, 15, 0, 1, 18, 19, 4, 3, 4, 21, 6, 7, 24, 63, 48, 49, 34, 35, 52, 51, 52, 37, 54, 55, 40, 47, 32, 33, 50, 51, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1767(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 5, 22, 23, 8, 31, 0, 1, 18, 19, 4, 3, 4, 21, 6, 7, 24, 47, 48, 49, 34, 35, 52, 51, 52, 37, 54, 55, 40, 63, 32, 33, 50, 51, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1768(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 5, 22, 23, 8, 15, 16, 17, 2, 19, 4, 3, 4, 21, 6, 7, 24, 63, 32, 33, 50, 35, 52, 51, 52, 37, 54, 55, 40, 47, 48, 49, 34, 51, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1769(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 5, 22, 23, 8, 31, 16, 17, 2, 19, 4, 3, 4, 21, 6, 7, 24, 47, 32, 33, 50, 35, 52, 51, 52, 37, 54, 55, 40, 63, 48, 49, 34, 51, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1770(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 5, 22, 23, 8, 15, 0, 17, 2, 19, 4, 3, 4, 21, 6, 7, 24, 63, 48, 33, 50, 35, 52, 51, 52, 37, 54, 55, 40, 47, 32, 49, 34, 51, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1771(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 5, 22, 23, 8, 31, 0, 17, 2, 19, 4, 3, 4, 21, 6, 7, 24, 47, 48, 33, 50, 35, 52, 51, 52, 37, 54, 55, 40, 63, 32, 49, 34, 51, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1772(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 5, 22, 23, 8, 15, 16, 1, 2, 19, 4, 3, 4, 21, 6, 7, 24, 63, 32, 49, 50, 35, 52, 51, 52, 37, 54, 55, 40, 47, 48, 33, 34, 51, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1773(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 5, 22, 23, 8, 31, 16, 1, 2, 19, 4, 3, 4, 21, 6, 7, 24, 47, 32, 49, 50, 35, 52, 51, 52, 37, 54, 55, 40, 63, 48, 33, 34, 51, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1774(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 5, 22, 23, 8, 15, 0, 1, 2, 19, 4, 3, 4, 21, 6, 7, 24, 63, 48, 49, 50, 35, 52, 51, 52, 37, 54, 55, 40, 47, 32, 33, 34, 51, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1775(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 5, 22, 23, 8, 31, 0, 1, 2, 19, 4, 3, 4, 21, 6, 7, 24, 47, 48, 49, 50, 35, 52, 51, 52, 37, 54, 55, 40, 63, 32, 33, 34, 51, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1776(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 5, 22, 23, 8, 15, 16, 17, 18, 3, 4, 3, 4, 21, 6, 7, 24, 63, 32, 33, 34, 51, 52, 51, 52, 37, 54, 55, 40, 47, 48, 49, 50, 35, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1777(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 5, 22, 23, 8, 31, 16, 17, 18, 3, 4, 3, 4, 21, 6, 7, 24, 47, 32, 33, 34, 51, 52, 51, 52, 37, 54, 55, 40, 63, 48, 49, 50, 35, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1778(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 5, 22, 23, 8, 15, 0, 17, 18, 3, 4, 3, 4, 21, 6, 7, 24, 63, 48, 33, 34, 51, 52, 51, 52, 37, 54, 55, 40, 47, 32, 49, 50, 35, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1779(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 5, 22, 23, 8, 31, 0, 17, 18, 3, 4, 3, 4, 21, 6, 7, 24, 47, 48, 33, 34, 51, 52, 51, 52, 37, 54, 55, 40, 63, 32, 49, 50, 35, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1780(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 5, 22, 23, 8, 15, 16, 1, 18, 3, 4, 3, 4, 21, 6, 7, 24, 63, 32, 49, 34, 51, 52, 51, 52, 37, 54, 55, 40, 47, 48, 33, 50, 35, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1781(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 5, 22, 23, 8, 31, 16, 1, 18, 3, 4, 3, 4, 21, 6, 7, 24, 47, 32, 49, 34, 51, 52, 51, 52, 37, 54, 55, 40, 63, 48, 33, 50, 35, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1782(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 5, 22, 23, 8, 15, 0, 1, 18, 3, 4, 3, 4, 21, 6, 7, 24, 63, 48, 49, 34, 51, 52, 51, 52, 37, 54, 55, 40, 47, 32, 33, 50, 35, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1783(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 5, 22, 23, 8, 31, 0, 1, 18, 3, 4, 3, 4, 21, 6, 7, 24, 47, 48, 49, 34, 51, 52, 51, 52, 37, 54, 55, 40, 63, 32, 33, 50, 35, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1784(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 5, 22, 23, 8, 15, 16, 17, 2, 3, 4, 3, 4, 21, 6, 7, 24, 63, 32, 33, 50, 51, 52, 51, 52, 37, 54, 55, 40, 47, 48, 49, 34, 35, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1785(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 5, 22, 23, 8, 31, 16, 17, 2, 3, 4, 3, 4, 21, 6, 7, 24, 47, 32, 33, 50, 51, 52, 51, 52, 37, 54, 55, 40, 63, 48, 49, 34, 35, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1786(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 5, 22, 23, 8, 15, 0, 17, 2, 3, 4, 3, 4, 21, 6, 7, 24, 63, 48, 33, 50, 51, 52, 51, 52, 37, 54, 55, 40, 47, 32, 49, 34, 35, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1787(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 5, 22, 23, 8, 31, 0, 17, 2, 3, 4, 3, 4, 21, 6, 7, 24, 47, 48, 33, 50, 51, 52, 51, 52, 37, 54, 55, 40, 63, 32, 49, 34, 35, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1788(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 5, 22, 23, 8, 15, 16, 1, 2, 3, 4, 3, 4, 21, 6, 7, 24, 63, 32, 49, 50, 51, 52, 51, 52, 37, 54, 55, 40, 47, 48, 33, 34, 35, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1789(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 5, 22, 23, 8, 31, 16, 1, 2, 3, 4, 3, 4, 21, 6, 7, 24, 47, 32, 49, 50, 51, 52, 51, 52, 37, 54, 55, 40, 63, 48, 33, 34, 35, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1790(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 5, 22, 23, 8, 15, 0, 1, 2, 3, 4, 3, 4, 21, 6, 7, 24, 63, 48, 49, 50, 51, 52, 51, 52, 37, 54, 55, 40, 47, 32, 33, 34, 35, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}

__global__ void des_25_encrypt_salt1791(vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 5, 22, 23, 8, 31, 0, 1, 2, 3, 4, 3, 4, 21, 6, 7, 24, 47, 48, 49, 50, 51, 52, 51, 52, 37, 54, 55, 40, 63, 32, 33, 34, 35, 36, 35, 36, 53, 38, 39, 56>(unchecked_hashes, bitsplitted_keys);
}


