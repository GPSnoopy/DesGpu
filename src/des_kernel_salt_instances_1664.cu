#include "des_kernel_encrypt.h"
#include "des_kernel_salt_instances.h"

#ifdef DESGPU_COMPILE_ALL_SALTS

void des_25_encrypt_salt1664(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 3, 20, 5, 22, 23, 8, 15, 16, 17, 18, 19, 20, 19, 4, 21, 6, 7, 24, 63, 32, 33, 34, 35, 36, 35, 52, 37, 54, 55, 40, 47, 48, 49, 50, 51, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1665(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 3, 20, 5, 22, 23, 8, 31, 16, 17, 18, 19, 20, 19, 4, 21, 6, 7, 24, 47, 32, 33, 34, 35, 36, 35, 52, 37, 54, 55, 40, 63, 48, 49, 50, 51, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1666(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 3, 20, 5, 22, 23, 8, 15, 0, 17, 18, 19, 20, 19, 4, 21, 6, 7, 24, 63, 48, 33, 34, 35, 36, 35, 52, 37, 54, 55, 40, 47, 32, 49, 50, 51, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1667(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 3, 20, 5, 22, 23, 8, 31, 0, 17, 18, 19, 20, 19, 4, 21, 6, 7, 24, 47, 48, 33, 34, 35, 36, 35, 52, 37, 54, 55, 40, 63, 32, 49, 50, 51, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1668(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 3, 20, 5, 22, 23, 8, 15, 16, 1, 18, 19, 20, 19, 4, 21, 6, 7, 24, 63, 32, 49, 34, 35, 36, 35, 52, 37, 54, 55, 40, 47, 48, 33, 50, 51, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1669(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 3, 20, 5, 22, 23, 8, 31, 16, 1, 18, 19, 20, 19, 4, 21, 6, 7, 24, 47, 32, 49, 34, 35, 36, 35, 52, 37, 54, 55, 40, 63, 48, 33, 50, 51, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1670(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 3, 20, 5, 22, 23, 8, 15, 0, 1, 18, 19, 20, 19, 4, 21, 6, 7, 24, 63, 48, 49, 34, 35, 36, 35, 52, 37, 54, 55, 40, 47, 32, 33, 50, 51, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1671(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 3, 20, 5, 22, 23, 8, 31, 0, 1, 18, 19, 20, 19, 4, 21, 6, 7, 24, 47, 48, 49, 34, 35, 36, 35, 52, 37, 54, 55, 40, 63, 32, 33, 50, 51, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1672(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 3, 20, 5, 22, 23, 8, 15, 16, 17, 2, 19, 20, 19, 4, 21, 6, 7, 24, 63, 32, 33, 50, 35, 36, 35, 52, 37, 54, 55, 40, 47, 48, 49, 34, 51, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1673(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 3, 20, 5, 22, 23, 8, 31, 16, 17, 2, 19, 20, 19, 4, 21, 6, 7, 24, 47, 32, 33, 50, 35, 36, 35, 52, 37, 54, 55, 40, 63, 48, 49, 34, 51, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1674(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 3, 20, 5, 22, 23, 8, 15, 0, 17, 2, 19, 20, 19, 4, 21, 6, 7, 24, 63, 48, 33, 50, 35, 36, 35, 52, 37, 54, 55, 40, 47, 32, 49, 34, 51, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1675(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 3, 20, 5, 22, 23, 8, 31, 0, 17, 2, 19, 20, 19, 4, 21, 6, 7, 24, 47, 48, 33, 50, 35, 36, 35, 52, 37, 54, 55, 40, 63, 32, 49, 34, 51, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1676(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 3, 20, 5, 22, 23, 8, 15, 16, 1, 2, 19, 20, 19, 4, 21, 6, 7, 24, 63, 32, 49, 50, 35, 36, 35, 52, 37, 54, 55, 40, 47, 48, 33, 34, 51, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1677(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 3, 20, 5, 22, 23, 8, 31, 16, 1, 2, 19, 20, 19, 4, 21, 6, 7, 24, 47, 32, 49, 50, 35, 36, 35, 52, 37, 54, 55, 40, 63, 48, 33, 34, 51, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1678(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 3, 20, 5, 22, 23, 8, 15, 0, 1, 2, 19, 20, 19, 4, 21, 6, 7, 24, 63, 48, 49, 50, 35, 36, 35, 52, 37, 54, 55, 40, 47, 32, 33, 34, 51, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1679(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 3, 20, 5, 22, 23, 8, 31, 0, 1, 2, 19, 20, 19, 4, 21, 6, 7, 24, 47, 48, 49, 50, 35, 36, 35, 52, 37, 54, 55, 40, 63, 32, 33, 34, 51, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1680(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 3, 20, 5, 22, 23, 8, 15, 16, 17, 18, 3, 20, 19, 4, 21, 6, 7, 24, 63, 32, 33, 34, 51, 36, 35, 52, 37, 54, 55, 40, 47, 48, 49, 50, 35, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1681(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 3, 20, 5, 22, 23, 8, 31, 16, 17, 18, 3, 20, 19, 4, 21, 6, 7, 24, 47, 32, 33, 34, 51, 36, 35, 52, 37, 54, 55, 40, 63, 48, 49, 50, 35, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1682(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 3, 20, 5, 22, 23, 8, 15, 0, 17, 18, 3, 20, 19, 4, 21, 6, 7, 24, 63, 48, 33, 34, 51, 36, 35, 52, 37, 54, 55, 40, 47, 32, 49, 50, 35, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1683(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 3, 20, 5, 22, 23, 8, 31, 0, 17, 18, 3, 20, 19, 4, 21, 6, 7, 24, 47, 48, 33, 34, 51, 36, 35, 52, 37, 54, 55, 40, 63, 32, 49, 50, 35, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1684(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 3, 20, 5, 22, 23, 8, 15, 16, 1, 18, 3, 20, 19, 4, 21, 6, 7, 24, 63, 32, 49, 34, 51, 36, 35, 52, 37, 54, 55, 40, 47, 48, 33, 50, 35, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1685(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 3, 20, 5, 22, 23, 8, 31, 16, 1, 18, 3, 20, 19, 4, 21, 6, 7, 24, 47, 32, 49, 34, 51, 36, 35, 52, 37, 54, 55, 40, 63, 48, 33, 50, 35, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1686(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 3, 20, 5, 22, 23, 8, 15, 0, 1, 18, 3, 20, 19, 4, 21, 6, 7, 24, 63, 48, 49, 34, 51, 36, 35, 52, 37, 54, 55, 40, 47, 32, 33, 50, 35, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1687(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 3, 20, 5, 22, 23, 8, 31, 0, 1, 18, 3, 20, 19, 4, 21, 6, 7, 24, 47, 48, 49, 34, 51, 36, 35, 52, 37, 54, 55, 40, 63, 32, 33, 50, 35, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1688(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 3, 20, 5, 22, 23, 8, 15, 16, 17, 2, 3, 20, 19, 4, 21, 6, 7, 24, 63, 32, 33, 50, 51, 36, 35, 52, 37, 54, 55, 40, 47, 48, 49, 34, 35, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1689(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 3, 20, 5, 22, 23, 8, 31, 16, 17, 2, 3, 20, 19, 4, 21, 6, 7, 24, 47, 32, 33, 50, 51, 36, 35, 52, 37, 54, 55, 40, 63, 48, 49, 34, 35, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1690(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 3, 20, 5, 22, 23, 8, 15, 0, 17, 2, 3, 20, 19, 4, 21, 6, 7, 24, 63, 48, 33, 50, 51, 36, 35, 52, 37, 54, 55, 40, 47, 32, 49, 34, 35, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1691(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 3, 20, 5, 22, 23, 8, 31, 0, 17, 2, 3, 20, 19, 4, 21, 6, 7, 24, 47, 48, 33, 50, 51, 36, 35, 52, 37, 54, 55, 40, 63, 32, 49, 34, 35, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1692(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 3, 20, 5, 22, 23, 8, 15, 16, 1, 2, 3, 20, 19, 4, 21, 6, 7, 24, 63, 32, 49, 50, 51, 36, 35, 52, 37, 54, 55, 40, 47, 48, 33, 34, 35, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1693(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 3, 20, 5, 22, 23, 8, 31, 16, 1, 2, 3, 20, 19, 4, 21, 6, 7, 24, 47, 32, 49, 50, 51, 36, 35, 52, 37, 54, 55, 40, 63, 48, 33, 34, 35, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1694(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 3, 20, 5, 22, 23, 8, 15, 0, 1, 2, 3, 20, 19, 4, 21, 6, 7, 24, 63, 48, 49, 50, 51, 36, 35, 52, 37, 54, 55, 40, 47, 32, 33, 34, 35, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1695(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 3, 20, 5, 22, 23, 8, 31, 0, 1, 2, 3, 20, 19, 4, 21, 6, 7, 24, 47, 48, 49, 50, 51, 36, 35, 52, 37, 54, 55, 40, 63, 32, 33, 34, 35, 52, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1696(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 3, 20, 5, 22, 23, 8, 15, 16, 17, 18, 19, 4, 19, 4, 21, 6, 7, 24, 63, 32, 33, 34, 35, 52, 35, 52, 37, 54, 55, 40, 47, 48, 49, 50, 51, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1697(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 3, 20, 5, 22, 23, 8, 31, 16, 17, 18, 19, 4, 19, 4, 21, 6, 7, 24, 47, 32, 33, 34, 35, 52, 35, 52, 37, 54, 55, 40, 63, 48, 49, 50, 51, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1698(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 3, 20, 5, 22, 23, 8, 15, 0, 17, 18, 19, 4, 19, 4, 21, 6, 7, 24, 63, 48, 33, 34, 35, 52, 35, 52, 37, 54, 55, 40, 47, 32, 49, 50, 51, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1699(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 3, 20, 5, 22, 23, 8, 31, 0, 17, 18, 19, 4, 19, 4, 21, 6, 7, 24, 47, 48, 33, 34, 35, 52, 35, 52, 37, 54, 55, 40, 63, 32, 49, 50, 51, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1700(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 3, 20, 5, 22, 23, 8, 15, 16, 1, 18, 19, 4, 19, 4, 21, 6, 7, 24, 63, 32, 49, 34, 35, 52, 35, 52, 37, 54, 55, 40, 47, 48, 33, 50, 51, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1701(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 3, 20, 5, 22, 23, 8, 31, 16, 1, 18, 19, 4, 19, 4, 21, 6, 7, 24, 47, 32, 49, 34, 35, 52, 35, 52, 37, 54, 55, 40, 63, 48, 33, 50, 51, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1702(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 3, 20, 5, 22, 23, 8, 15, 0, 1, 18, 19, 4, 19, 4, 21, 6, 7, 24, 63, 48, 49, 34, 35, 52, 35, 52, 37, 54, 55, 40, 47, 32, 33, 50, 51, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1703(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 3, 20, 5, 22, 23, 8, 31, 0, 1, 18, 19, 4, 19, 4, 21, 6, 7, 24, 47, 48, 49, 34, 35, 52, 35, 52, 37, 54, 55, 40, 63, 32, 33, 50, 51, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1704(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 3, 20, 5, 22, 23, 8, 15, 16, 17, 2, 19, 4, 19, 4, 21, 6, 7, 24, 63, 32, 33, 50, 35, 52, 35, 52, 37, 54, 55, 40, 47, 48, 49, 34, 51, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1705(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 3, 20, 5, 22, 23, 8, 31, 16, 17, 2, 19, 4, 19, 4, 21, 6, 7, 24, 47, 32, 33, 50, 35, 52, 35, 52, 37, 54, 55, 40, 63, 48, 49, 34, 51, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1706(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 3, 20, 5, 22, 23, 8, 15, 0, 17, 2, 19, 4, 19, 4, 21, 6, 7, 24, 63, 48, 33, 50, 35, 52, 35, 52, 37, 54, 55, 40, 47, 32, 49, 34, 51, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1707(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 3, 20, 5, 22, 23, 8, 31, 0, 17, 2, 19, 4, 19, 4, 21, 6, 7, 24, 47, 48, 33, 50, 35, 52, 35, 52, 37, 54, 55, 40, 63, 32, 49, 34, 51, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1708(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 3, 20, 5, 22, 23, 8, 15, 16, 1, 2, 19, 4, 19, 4, 21, 6, 7, 24, 63, 32, 49, 50, 35, 52, 35, 52, 37, 54, 55, 40, 47, 48, 33, 34, 51, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1709(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 3, 20, 5, 22, 23, 8, 31, 16, 1, 2, 19, 4, 19, 4, 21, 6, 7, 24, 47, 32, 49, 50, 35, 52, 35, 52, 37, 54, 55, 40, 63, 48, 33, 34, 51, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1710(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 3, 20, 5, 22, 23, 8, 15, 0, 1, 2, 19, 4, 19, 4, 21, 6, 7, 24, 63, 48, 49, 50, 35, 52, 35, 52, 37, 54, 55, 40, 47, 32, 33, 34, 51, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1711(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 3, 20, 5, 22, 23, 8, 31, 0, 1, 2, 19, 4, 19, 4, 21, 6, 7, 24, 47, 48, 49, 50, 35, 52, 35, 52, 37, 54, 55, 40, 63, 32, 33, 34, 51, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1712(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 3, 20, 5, 22, 23, 8, 15, 16, 17, 18, 3, 4, 19, 4, 21, 6, 7, 24, 63, 32, 33, 34, 51, 52, 35, 52, 37, 54, 55, 40, 47, 48, 49, 50, 35, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1713(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 3, 20, 5, 22, 23, 8, 31, 16, 17, 18, 3, 4, 19, 4, 21, 6, 7, 24, 47, 32, 33, 34, 51, 52, 35, 52, 37, 54, 55, 40, 63, 48, 49, 50, 35, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1714(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 3, 20, 5, 22, 23, 8, 15, 0, 17, 18, 3, 4, 19, 4, 21, 6, 7, 24, 63, 48, 33, 34, 51, 52, 35, 52, 37, 54, 55, 40, 47, 32, 49, 50, 35, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1715(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 3, 20, 5, 22, 23, 8, 31, 0, 17, 18, 3, 4, 19, 4, 21, 6, 7, 24, 47, 48, 33, 34, 51, 52, 35, 52, 37, 54, 55, 40, 63, 32, 49, 50, 35, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1716(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 3, 20, 5, 22, 23, 8, 15, 16, 1, 18, 3, 4, 19, 4, 21, 6, 7, 24, 63, 32, 49, 34, 51, 52, 35, 52, 37, 54, 55, 40, 47, 48, 33, 50, 35, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1717(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 3, 20, 5, 22, 23, 8, 31, 16, 1, 18, 3, 4, 19, 4, 21, 6, 7, 24, 47, 32, 49, 34, 51, 52, 35, 52, 37, 54, 55, 40, 63, 48, 33, 50, 35, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1718(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 3, 20, 5, 22, 23, 8, 15, 0, 1, 18, 3, 4, 19, 4, 21, 6, 7, 24, 63, 48, 49, 34, 51, 52, 35, 52, 37, 54, 55, 40, 47, 32, 33, 50, 35, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1719(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 3, 20, 5, 22, 23, 8, 31, 0, 1, 18, 3, 4, 19, 4, 21, 6, 7, 24, 47, 48, 49, 34, 51, 52, 35, 52, 37, 54, 55, 40, 63, 32, 33, 50, 35, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1720(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 3, 20, 5, 22, 23, 8, 15, 16, 17, 2, 3, 4, 19, 4, 21, 6, 7, 24, 63, 32, 33, 50, 51, 52, 35, 52, 37, 54, 55, 40, 47, 48, 49, 34, 35, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1721(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 3, 20, 5, 22, 23, 8, 31, 16, 17, 2, 3, 4, 19, 4, 21, 6, 7, 24, 47, 32, 33, 50, 51, 52, 35, 52, 37, 54, 55, 40, 63, 48, 49, 34, 35, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1722(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 3, 20, 5, 22, 23, 8, 15, 0, 17, 2, 3, 4, 19, 4, 21, 6, 7, 24, 63, 48, 33, 50, 51, 52, 35, 52, 37, 54, 55, 40, 47, 32, 49, 34, 35, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1723(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 3, 20, 5, 22, 23, 8, 31, 0, 17, 2, 3, 4, 19, 4, 21, 6, 7, 24, 47, 48, 33, 50, 51, 52, 35, 52, 37, 54, 55, 40, 63, 32, 49, 34, 35, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1724(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 3, 20, 5, 22, 23, 8, 15, 16, 1, 2, 3, 4, 19, 4, 21, 6, 7, 24, 63, 32, 49, 50, 51, 52, 35, 52, 37, 54, 55, 40, 47, 48, 33, 34, 35, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1725(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 3, 20, 5, 22, 23, 8, 31, 16, 1, 2, 3, 4, 19, 4, 21, 6, 7, 24, 47, 32, 49, 50, 51, 52, 35, 52, 37, 54, 55, 40, 63, 48, 33, 34, 35, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1726(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 3, 20, 5, 22, 23, 8, 15, 0, 1, 2, 3, 4, 19, 4, 21, 6, 7, 24, 63, 48, 49, 50, 51, 52, 35, 52, 37, 54, 55, 40, 47, 32, 33, 34, 35, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1727(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 3, 20, 5, 22, 23, 8, 31, 0, 1, 2, 3, 4, 19, 4, 21, 6, 7, 24, 47, 48, 49, 50, 51, 52, 35, 52, 37, 54, 55, 40, 63, 32, 33, 34, 35, 36, 51, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1728(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 4, 19, 20, 5, 22, 23, 8, 15, 16, 17, 18, 19, 20, 3, 4, 21, 6, 7, 24, 63, 32, 33, 34, 35, 36, 51, 52, 37, 54, 55, 40, 47, 48, 49, 50, 51, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1729(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 4, 19, 20, 5, 22, 23, 8, 31, 16, 17, 18, 19, 20, 3, 4, 21, 6, 7, 24, 47, 32, 33, 34, 35, 36, 51, 52, 37, 54, 55, 40, 63, 48, 49, 50, 51, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1730(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 4, 19, 20, 5, 22, 23, 8, 15, 0, 17, 18, 19, 20, 3, 4, 21, 6, 7, 24, 63, 48, 33, 34, 35, 36, 51, 52, 37, 54, 55, 40, 47, 32, 49, 50, 51, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1731(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 4, 19, 20, 5, 22, 23, 8, 31, 0, 17, 18, 19, 20, 3, 4, 21, 6, 7, 24, 47, 48, 33, 34, 35, 36, 51, 52, 37, 54, 55, 40, 63, 32, 49, 50, 51, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1732(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 4, 19, 20, 5, 22, 23, 8, 15, 16, 1, 18, 19, 20, 3, 4, 21, 6, 7, 24, 63, 32, 49, 34, 35, 36, 51, 52, 37, 54, 55, 40, 47, 48, 33, 50, 51, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1733(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 4, 19, 20, 5, 22, 23, 8, 31, 16, 1, 18, 19, 20, 3, 4, 21, 6, 7, 24, 47, 32, 49, 34, 35, 36, 51, 52, 37, 54, 55, 40, 63, 48, 33, 50, 51, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1734(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 4, 19, 20, 5, 22, 23, 8, 15, 0, 1, 18, 19, 20, 3, 4, 21, 6, 7, 24, 63, 48, 49, 34, 35, 36, 51, 52, 37, 54, 55, 40, 47, 32, 33, 50, 51, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1735(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 4, 19, 20, 5, 22, 23, 8, 31, 0, 1, 18, 19, 20, 3, 4, 21, 6, 7, 24, 47, 48, 49, 34, 35, 36, 51, 52, 37, 54, 55, 40, 63, 32, 33, 50, 51, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1736(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 4, 19, 20, 5, 22, 23, 8, 15, 16, 17, 2, 19, 20, 3, 4, 21, 6, 7, 24, 63, 32, 33, 50, 35, 36, 51, 52, 37, 54, 55, 40, 47, 48, 49, 34, 51, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1737(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 4, 19, 20, 5, 22, 23, 8, 31, 16, 17, 2, 19, 20, 3, 4, 21, 6, 7, 24, 47, 32, 33, 50, 35, 36, 51, 52, 37, 54, 55, 40, 63, 48, 49, 34, 51, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1738(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 4, 19, 20, 5, 22, 23, 8, 15, 0, 17, 2, 19, 20, 3, 4, 21, 6, 7, 24, 63, 48, 33, 50, 35, 36, 51, 52, 37, 54, 55, 40, 47, 32, 49, 34, 51, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1739(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 4, 19, 20, 5, 22, 23, 8, 31, 0, 17, 2, 19, 20, 3, 4, 21, 6, 7, 24, 47, 48, 33, 50, 35, 36, 51, 52, 37, 54, 55, 40, 63, 32, 49, 34, 51, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1740(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 4, 19, 20, 5, 22, 23, 8, 15, 16, 1, 2, 19, 20, 3, 4, 21, 6, 7, 24, 63, 32, 49, 50, 35, 36, 51, 52, 37, 54, 55, 40, 47, 48, 33, 34, 51, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1741(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 4, 19, 20, 5, 22, 23, 8, 31, 16, 1, 2, 19, 20, 3, 4, 21, 6, 7, 24, 47, 32, 49, 50, 35, 36, 51, 52, 37, 54, 55, 40, 63, 48, 33, 34, 51, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1742(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 4, 19, 20, 5, 22, 23, 8, 15, 0, 1, 2, 19, 20, 3, 4, 21, 6, 7, 24, 63, 48, 49, 50, 35, 36, 51, 52, 37, 54, 55, 40, 47, 32, 33, 34, 51, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1743(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 4, 19, 20, 5, 22, 23, 8, 31, 0, 1, 2, 19, 20, 3, 4, 21, 6, 7, 24, 47, 48, 49, 50, 35, 36, 51, 52, 37, 54, 55, 40, 63, 32, 33, 34, 51, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1744(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 4, 19, 20, 5, 22, 23, 8, 15, 16, 17, 18, 3, 20, 3, 4, 21, 6, 7, 24, 63, 32, 33, 34, 51, 36, 51, 52, 37, 54, 55, 40, 47, 48, 49, 50, 35, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1745(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 4, 19, 20, 5, 22, 23, 8, 31, 16, 17, 18, 3, 20, 3, 4, 21, 6, 7, 24, 47, 32, 33, 34, 51, 36, 51, 52, 37, 54, 55, 40, 63, 48, 49, 50, 35, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1746(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 4, 19, 20, 5, 22, 23, 8, 15, 0, 17, 18, 3, 20, 3, 4, 21, 6, 7, 24, 63, 48, 33, 34, 51, 36, 51, 52, 37, 54, 55, 40, 47, 32, 49, 50, 35, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1747(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 4, 19, 20, 5, 22, 23, 8, 31, 0, 17, 18, 3, 20, 3, 4, 21, 6, 7, 24, 47, 48, 33, 34, 51, 36, 51, 52, 37, 54, 55, 40, 63, 32, 49, 50, 35, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1748(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 4, 19, 20, 5, 22, 23, 8, 15, 16, 1, 18, 3, 20, 3, 4, 21, 6, 7, 24, 63, 32, 49, 34, 51, 36, 51, 52, 37, 54, 55, 40, 47, 48, 33, 50, 35, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1749(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 4, 19, 20, 5, 22, 23, 8, 31, 16, 1, 18, 3, 20, 3, 4, 21, 6, 7, 24, 47, 32, 49, 34, 51, 36, 51, 52, 37, 54, 55, 40, 63, 48, 33, 50, 35, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1750(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 4, 19, 20, 5, 22, 23, 8, 15, 0, 1, 18, 3, 20, 3, 4, 21, 6, 7, 24, 63, 48, 49, 34, 51, 36, 51, 52, 37, 54, 55, 40, 47, 32, 33, 50, 35, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1751(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 4, 19, 20, 5, 22, 23, 8, 31, 0, 1, 18, 3, 20, 3, 4, 21, 6, 7, 24, 47, 48, 49, 34, 51, 36, 51, 52, 37, 54, 55, 40, 63, 32, 33, 50, 35, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1752(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 4, 19, 20, 5, 22, 23, 8, 15, 16, 17, 2, 3, 20, 3, 4, 21, 6, 7, 24, 63, 32, 33, 50, 51, 36, 51, 52, 37, 54, 55, 40, 47, 48, 49, 34, 35, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1753(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 4, 19, 20, 5, 22, 23, 8, 31, 16, 17, 2, 3, 20, 3, 4, 21, 6, 7, 24, 47, 32, 33, 50, 51, 36, 51, 52, 37, 54, 55, 40, 63, 48, 49, 34, 35, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1754(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 4, 19, 20, 5, 22, 23, 8, 15, 0, 17, 2, 3, 20, 3, 4, 21, 6, 7, 24, 63, 48, 33, 50, 51, 36, 51, 52, 37, 54, 55, 40, 47, 32, 49, 34, 35, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1755(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 4, 19, 20, 5, 22, 23, 8, 31, 0, 17, 2, 3, 20, 3, 4, 21, 6, 7, 24, 47, 48, 33, 50, 51, 36, 51, 52, 37, 54, 55, 40, 63, 32, 49, 34, 35, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1756(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 4, 19, 20, 5, 22, 23, 8, 15, 16, 1, 2, 3, 20, 3, 4, 21, 6, 7, 24, 63, 32, 49, 50, 51, 36, 51, 52, 37, 54, 55, 40, 47, 48, 33, 34, 35, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1757(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 4, 19, 20, 5, 22, 23, 8, 31, 16, 1, 2, 3, 20, 3, 4, 21, 6, 7, 24, 47, 32, 49, 50, 51, 36, 51, 52, 37, 54, 55, 40, 63, 48, 33, 34, 35, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1758(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 4, 19, 20, 5, 22, 23, 8, 15, 0, 1, 2, 3, 20, 3, 4, 21, 6, 7, 24, 63, 48, 49, 50, 51, 36, 51, 52, 37, 54, 55, 40, 47, 32, 33, 34, 35, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1759(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 4, 19, 20, 5, 22, 23, 8, 31, 0, 1, 2, 3, 20, 3, 4, 21, 6, 7, 24, 47, 48, 49, 50, 51, 36, 51, 52, 37, 54, 55, 40, 63, 32, 33, 34, 35, 52, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1760(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 3, 20, 19, 20, 5, 22, 23, 8, 15, 16, 17, 18, 19, 4, 3, 4, 21, 6, 7, 24, 63, 32, 33, 34, 35, 52, 51, 52, 37, 54, 55, 40, 47, 48, 49, 50, 51, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1761(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 3, 20, 19, 20, 5, 22, 23, 8, 31, 16, 17, 18, 19, 4, 3, 4, 21, 6, 7, 24, 47, 32, 33, 34, 35, 52, 51, 52, 37, 54, 55, 40, 63, 48, 49, 50, 51, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1762(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 3, 20, 19, 20, 5, 22, 23, 8, 15, 0, 17, 18, 19, 4, 3, 4, 21, 6, 7, 24, 63, 48, 33, 34, 35, 52, 51, 52, 37, 54, 55, 40, 47, 32, 49, 50, 51, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1763(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 3, 20, 19, 20, 5, 22, 23, 8, 31, 0, 17, 18, 19, 4, 3, 4, 21, 6, 7, 24, 47, 48, 33, 34, 35, 52, 51, 52, 37, 54, 55, 40, 63, 32, 49, 50, 51, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1764(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 3, 20, 19, 20, 5, 22, 23, 8, 15, 16, 1, 18, 19, 4, 3, 4, 21, 6, 7, 24, 63, 32, 49, 34, 35, 52, 51, 52, 37, 54, 55, 40, 47, 48, 33, 50, 51, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1765(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 3, 20, 19, 20, 5, 22, 23, 8, 31, 16, 1, 18, 19, 4, 3, 4, 21, 6, 7, 24, 47, 32, 49, 34, 35, 52, 51, 52, 37, 54, 55, 40, 63, 48, 33, 50, 51, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1766(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 3, 20, 19, 20, 5, 22, 23, 8, 15, 0, 1, 18, 19, 4, 3, 4, 21, 6, 7, 24, 63, 48, 49, 34, 35, 52, 51, 52, 37, 54, 55, 40, 47, 32, 33, 50, 51, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1767(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 3, 20, 19, 20, 5, 22, 23, 8, 31, 0, 1, 18, 19, 4, 3, 4, 21, 6, 7, 24, 47, 48, 49, 34, 35, 52, 51, 52, 37, 54, 55, 40, 63, 32, 33, 50, 51, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1768(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 3, 20, 19, 20, 5, 22, 23, 8, 15, 16, 17, 2, 19, 4, 3, 4, 21, 6, 7, 24, 63, 32, 33, 50, 35, 52, 51, 52, 37, 54, 55, 40, 47, 48, 49, 34, 51, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1769(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 3, 20, 19, 20, 5, 22, 23, 8, 31, 16, 17, 2, 19, 4, 3, 4, 21, 6, 7, 24, 47, 32, 33, 50, 35, 52, 51, 52, 37, 54, 55, 40, 63, 48, 49, 34, 51, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1770(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 3, 20, 19, 20, 5, 22, 23, 8, 15, 0, 17, 2, 19, 4, 3, 4, 21, 6, 7, 24, 63, 48, 33, 50, 35, 52, 51, 52, 37, 54, 55, 40, 47, 32, 49, 34, 51, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1771(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 3, 20, 19, 20, 5, 22, 23, 8, 31, 0, 17, 2, 19, 4, 3, 4, 21, 6, 7, 24, 47, 48, 33, 50, 35, 52, 51, 52, 37, 54, 55, 40, 63, 32, 49, 34, 51, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1772(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 3, 20, 19, 20, 5, 22, 23, 8, 15, 16, 1, 2, 19, 4, 3, 4, 21, 6, 7, 24, 63, 32, 49, 50, 35, 52, 51, 52, 37, 54, 55, 40, 47, 48, 33, 34, 51, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1773(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 3, 20, 19, 20, 5, 22, 23, 8, 31, 16, 1, 2, 19, 4, 3, 4, 21, 6, 7, 24, 47, 32, 49, 50, 35, 52, 51, 52, 37, 54, 55, 40, 63, 48, 33, 34, 51, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1774(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 3, 20, 19, 20, 5, 22, 23, 8, 15, 0, 1, 2, 19, 4, 3, 4, 21, 6, 7, 24, 63, 48, 49, 50, 35, 52, 51, 52, 37, 54, 55, 40, 47, 32, 33, 34, 51, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1775(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 3, 20, 19, 20, 5, 22, 23, 8, 31, 0, 1, 2, 19, 4, 3, 4, 21, 6, 7, 24, 47, 48, 49, 50, 35, 52, 51, 52, 37, 54, 55, 40, 63, 32, 33, 34, 51, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1776(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 2, 19, 20, 19, 20, 5, 22, 23, 8, 15, 16, 17, 18, 3, 4, 3, 4, 21, 6, 7, 24, 63, 32, 33, 34, 51, 52, 51, 52, 37, 54, 55, 40, 47, 48, 49, 50, 35, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1777(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 2, 19, 20, 19, 20, 5, 22, 23, 8, 31, 16, 17, 18, 3, 4, 3, 4, 21, 6, 7, 24, 47, 32, 33, 34, 51, 52, 51, 52, 37, 54, 55, 40, 63, 48, 49, 50, 35, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1778(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 2, 19, 20, 19, 20, 5, 22, 23, 8, 15, 0, 17, 18, 3, 4, 3, 4, 21, 6, 7, 24, 63, 48, 33, 34, 51, 52, 51, 52, 37, 54, 55, 40, 47, 32, 49, 50, 35, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1779(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 2, 19, 20, 19, 20, 5, 22, 23, 8, 31, 0, 17, 18, 3, 4, 3, 4, 21, 6, 7, 24, 47, 48, 33, 34, 51, 52, 51, 52, 37, 54, 55, 40, 63, 32, 49, 50, 35, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1780(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 2, 19, 20, 19, 20, 5, 22, 23, 8, 15, 16, 1, 18, 3, 4, 3, 4, 21, 6, 7, 24, 63, 32, 49, 34, 51, 52, 51, 52, 37, 54, 55, 40, 47, 48, 33, 50, 35, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1781(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 2, 19, 20, 19, 20, 5, 22, 23, 8, 31, 16, 1, 18, 3, 4, 3, 4, 21, 6, 7, 24, 47, 32, 49, 34, 51, 52, 51, 52, 37, 54, 55, 40, 63, 48, 33, 50, 35, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1782(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 2, 19, 20, 19, 20, 5, 22, 23, 8, 15, 0, 1, 18, 3, 4, 3, 4, 21, 6, 7, 24, 63, 48, 49, 34, 51, 52, 51, 52, 37, 54, 55, 40, 47, 32, 33, 50, 35, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1783(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 2, 19, 20, 19, 20, 5, 22, 23, 8, 31, 0, 1, 18, 3, 4, 3, 4, 21, 6, 7, 24, 47, 48, 49, 34, 51, 52, 51, 52, 37, 54, 55, 40, 63, 32, 33, 50, 35, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1784(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 1, 18, 19, 20, 19, 20, 5, 22, 23, 8, 15, 16, 17, 2, 3, 4, 3, 4, 21, 6, 7, 24, 63, 32, 33, 50, 51, 52, 51, 52, 37, 54, 55, 40, 47, 48, 49, 34, 35, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1785(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 1, 18, 19, 20, 19, 20, 5, 22, 23, 8, 31, 16, 17, 2, 3, 4, 3, 4, 21, 6, 7, 24, 47, 32, 33, 50, 51, 52, 51, 52, 37, 54, 55, 40, 63, 48, 49, 34, 35, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1786(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 1, 18, 19, 20, 19, 20, 5, 22, 23, 8, 15, 0, 17, 2, 3, 4, 3, 4, 21, 6, 7, 24, 63, 48, 33, 50, 51, 52, 51, 52, 37, 54, 55, 40, 47, 32, 49, 34, 35, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1787(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 1, 18, 19, 20, 19, 20, 5, 22, 23, 8, 31, 0, 17, 2, 3, 4, 3, 4, 21, 6, 7, 24, 47, 48, 33, 50, 51, 52, 51, 52, 37, 54, 55, 40, 63, 32, 49, 34, 35, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1788(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 0, 17, 18, 19, 20, 19, 20, 5, 22, 23, 8, 15, 16, 1, 2, 3, 4, 3, 4, 21, 6, 7, 24, 63, 32, 49, 50, 51, 52, 51, 52, 37, 54, 55, 40, 47, 48, 33, 34, 35, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1789(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 0, 17, 18, 19, 20, 19, 20, 5, 22, 23, 8, 31, 16, 1, 2, 3, 4, 3, 4, 21, 6, 7, 24, 47, 32, 49, 50, 51, 52, 51, 52, 37, 54, 55, 40, 63, 48, 33, 34, 35, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1790(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<31, 16, 17, 18, 19, 20, 19, 20, 5, 22, 23, 8, 15, 0, 1, 2, 3, 4, 3, 4, 21, 6, 7, 24, 63, 48, 49, 50, 51, 52, 51, 52, 37, 54, 55, 40, 47, 32, 33, 34, 35, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

void des_25_encrypt_salt1791(const size_t num_blocks, const size_t threads_per_block, vtype* const unchecked_hashes, const bs_vector* const bitsplitted_keys)
{
	des_25_encrypt<15, 16, 17, 18, 19, 20, 19, 20, 5, 22, 23, 8, 31, 0, 1, 2, 3, 4, 3, 4, 21, 6, 7, 24, 47, 48, 49, 50, 51, 52, 51, 52, 37, 54, 55, 40, 63, 32, 33, 34, 35, 36, 35, 36, 53, 38, 39, 56><<<num_blocks, threads_per_block>>>(unchecked_hashes, bitsplitted_keys);
}

#endif // DESGPU_COMPILE_ALL_SALTS


